/*
  Copyright (C) 2010,2011,2012,2013 The ESPResSo project
  
  This file is part of ESPResSo.
  
  ESPResSo is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.
  
  ESPResSo is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  
  You should have received a copy of the GNU General Public License
  along with this program.  If not, see <http://www.gnu.org/licenses/>. 
*/


// CUDA code is always interpreted as C++, so we need the extern C interface
extern "C" {

#include "utils.h"
#include "cuda_init.h"

}

#ifdef CUDA

#include <hip/hip_runtime.h>

/** \name minimally required compute capability. */
/*@{*/
static const int computeCapabilityMinMajor = 1;
static const int computeCapabilityMinMinor = 1;
/*@}*/

const char *cuda_error;

/// get the number of CUDA devices.
int cuda_get_n_gpus()
{
  int deviceCount;
  hipError_t error = hipGetDeviceCount(&deviceCount);
  if (error != hipSuccess) {
    cuda_error = hipGetErrorString(error);
    return -1;
  }
  return deviceCount;
}

int cuda_check_gpu(int dev)
{
  hipDeviceProp_t deviceProp;
  hipError_t error = hipGetDeviceProperties(&deviceProp, dev);
  if (error != hipSuccess) {
    cuda_error = hipGetErrorString(error);
    return ES_ERROR;
  }
  if (deviceProp.major < computeCapabilityMinMajor ||
      (deviceProp.major == computeCapabilityMinMajor &&
       deviceProp.minor < computeCapabilityMinMinor)) {
    cuda_error = "compute capability insufficient";
    return ES_ERROR;
  }
  return ES_OK;
}

void cuda_get_gpu_name(int dev, char name[64])
{
  hipDeviceProp_t deviceProp;
  hipError_t error = hipGetDeviceProperties(&deviceProp, dev);
  if (error != hipSuccess) {
    cuda_error = hipGetErrorString(error);
    strcpy(name, "no GPU");
    return;
  }
  strncpy(name, deviceProp.name, 63);
  name[63] = 0;
}

int cuda_set_device(int dev)
{
  hipError_t error = hipSetDevice(dev);
  if (error != hipSuccess) {
    cuda_error = hipGetErrorString(error);
    return ES_ERROR;
  }
  else
    return ES_OK;
}

int cuda_get_device()
{
  int dev;
  hipError_t error = hipGetDevice(&dev);
  if (error != hipSuccess) {
    cuda_error = hipGetErrorString(error);
    return -1;
  }
  else
    return dev;
}

#endif /* defined(CUDA) */
