#include "hip/hip_runtime.h"
//======================================================================================================================
//
//  This file is part of waLBerla. waLBerla is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  waLBerla is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
//  for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with waLBerla (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \\file Dynamic_UBB_double_precisionCUDA.cpp
//! \\author pystencils
//======================================================================================================================

// kernel generated with pystencils v1.3.3, lbmpy v1.3.3, lbmpy_walberla/pystencils_walberla from waLBerla commit 04f4adbdfc0af983e2d9b72e244d775f37d77034

#include "Dynamic_UBB_double_precisionCUDA.h"
#include "core/DataTypes.h"
#include "core/Macros.h"
#include "gpu/ErrorChecking.h"

#define FUNC_PREFIX __global__

using namespace std;

namespace walberla {
namespace lbm {

#ifdef __GNUC__
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wstrict-aliasing"
#pragma GCC diagnostic ignored "-Wunused-variable"
#pragma GCC diagnostic ignored "-Wconversion"
#endif

#ifdef __HIPCC__

#ifdef __NVCC_DIAG_PRAGMA_SUPPORT__
#pragma nv_diag_suppress 177
#else

#endif
#endif
// NOLINTBEGIN(readability-non-const-parameter*)
namespace internal_dynamic_ubb_double_precisioncuda_boundary_Dynamic_UBB_double_precisionCUDA {
static FUNC_PREFIX __launch_bounds__(256) void dynamic_ubb_double_precisioncuda_boundary_Dynamic_UBB_double_precisionCUDA(uint8_t *RESTRICT const _data_indexVector, double *RESTRICT _data_pdfs, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3, int32_t indexVectorSize) {

  const int32_t f_in_inv_dir_idx[] = {0, 2, 1, 4, 3, 6, 5, 10, 9, 8, 7, 16, 15, 18, 17, 12, 11, 14, 13};
  const int32_t f_in_inv_offsets_x[] = {0, 0, 0, -1, 1, 0, 0, -1, 1, -1, 1, 0, 0, -1, 1, 0, 0, -1, 1};
  const int32_t f_in_inv_offsets_y[] = {0, 1, -1, 0, 0, 0, 0, 1, 1, -1, -1, 1, -1, 0, 0, 1, -1, 0, 0};
  const int32_t f_in_inv_offsets_z[] = {0, 0, 0, 0, 0, 1, -1, 0, 0, 0, 0, 1, 1, 1, 1, -1, -1, -1, -1};

  const double weights[] = {0.33333333333333333, 0.055555555555555556, 0.055555555555555556, 0.055555555555555556, 0.055555555555555556, 0.055555555555555556, 0.055555555555555556, 0.027777777777777778, 0.027777777777777778, 0.027777777777777778, 0.027777777777777778, 0.027777777777777778, 0.027777777777777778, 0.027777777777777778, 0.027777777777777778, 0.027777777777777778, 0.027777777777777778, 0.027777777777777778, 0.027777777777777778};

  const int32_t neighbour_offset_x[] = {0, 0, 0, -1, 1, 0, 0, -1, 1, -1, 1, 0, 0, -1, 1, 0, 0, -1, 1};
  const int32_t neighbour_offset_y[] = {0, 1, -1, 0, 0, 0, 0, 1, 1, -1, -1, 1, -1, 0, 0, 1, -1, 0, 0};
  const int32_t neighbour_offset_z[] = {0, 0, 0, 0, 0, 1, -1, 0, 0, 0, 0, 1, 1, 1, 1, -1, -1, -1, -1};

  if (blockDim.x * blockIdx.x + threadIdx.x < indexVectorSize) {
    uint8_t *RESTRICT _data_indexVector_10 = _data_indexVector;
    const int32_t x = *((int32_t *)(&_data_indexVector_10[40 * blockDim.x * blockIdx.x + 40 * threadIdx.x]));
    uint8_t *RESTRICT _data_indexVector_14 = _data_indexVector + 4;
    const int32_t y = *((int32_t *)(&_data_indexVector_14[40 * blockDim.x * blockIdx.x + 40 * threadIdx.x]));
    uint8_t *RESTRICT _data_indexVector_18 = _data_indexVector + 8;
    const int32_t z = *((int32_t *)(&_data_indexVector_18[40 * blockDim.x * blockIdx.x + 40 * threadIdx.x]));
    uint8_t *RESTRICT _data_indexVector_112 = _data_indexVector + 12;
    const int32_t dir = *((int32_t *)(&_data_indexVector_112[40 * blockDim.x * blockIdx.x + 40 * threadIdx.x]));
    double *RESTRICT _data_pdfs_10_20_310 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + 10 * _stride_pdfs_3;
    double *RESTRICT _data_pdfs_10_20_314 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + 14 * _stride_pdfs_3;
    double *RESTRICT _data_pdfs_10_20_318 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + 18 * _stride_pdfs_3;
    double *RESTRICT _data_pdfs_10_20_34 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + 4 * _stride_pdfs_3;
    double *RESTRICT _data_pdfs_10_20_38 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + 8 * _stride_pdfs_3;
    const double vel0Term = _data_pdfs_10_20_310[_stride_pdfs_0 * x] + _data_pdfs_10_20_314[_stride_pdfs_0 * x] + _data_pdfs_10_20_318[_stride_pdfs_0 * x] + _data_pdfs_10_20_34[_stride_pdfs_0 * x] + _data_pdfs_10_20_38[_stride_pdfs_0 * x];
    double *RESTRICT _data_pdfs_10_20_31 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + _stride_pdfs_3;
    double *RESTRICT _data_pdfs_10_20_311 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + 11 * _stride_pdfs_3;
    double *RESTRICT _data_pdfs_10_20_315 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + 15 * _stride_pdfs_3;
    double *RESTRICT _data_pdfs_10_20_37 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + 7 * _stride_pdfs_3;
    const double vel1Term = _data_pdfs_10_20_311[_stride_pdfs_0 * x] + _data_pdfs_10_20_315[_stride_pdfs_0 * x] + _data_pdfs_10_20_31[_stride_pdfs_0 * x] + _data_pdfs_10_20_37[_stride_pdfs_0 * x];
    double *RESTRICT _data_pdfs_10_20_312 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + 12 * _stride_pdfs_3;
    double *RESTRICT _data_pdfs_10_20_313 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + 13 * _stride_pdfs_3;
    double *RESTRICT _data_pdfs_10_20_35 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + 5 * _stride_pdfs_3;
    const double vel2Term = _data_pdfs_10_20_312[_stride_pdfs_0 * x] + _data_pdfs_10_20_313[_stride_pdfs_0 * x] + _data_pdfs_10_20_35[_stride_pdfs_0 * x];
    double *RESTRICT _data_pdfs_10_20_30 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z;
    double *RESTRICT _data_pdfs_10_20_316 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + 16 * _stride_pdfs_3;
    double *RESTRICT _data_pdfs_10_20_317 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + 17 * _stride_pdfs_3;
    double *RESTRICT _data_pdfs_10_20_32 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + 2 * _stride_pdfs_3;
    double *RESTRICT _data_pdfs_10_20_33 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + 3 * _stride_pdfs_3;
    double *RESTRICT _data_pdfs_10_20_36 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + 6 * _stride_pdfs_3;
    double *RESTRICT _data_pdfs_10_20_39 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + 9 * _stride_pdfs_3;
    const double rho = vel0Term + vel1Term + vel2Term + _data_pdfs_10_20_30[_stride_pdfs_0 * x] + _data_pdfs_10_20_316[_stride_pdfs_0 * x] + _data_pdfs_10_20_317[_stride_pdfs_0 * x] + _data_pdfs_10_20_32[_stride_pdfs_0 * x] + _data_pdfs_10_20_33[_stride_pdfs_0 * x] + _data_pdfs_10_20_36[_stride_pdfs_0 * x] + _data_pdfs_10_20_39[_stride_pdfs_0 * x];
    double *RESTRICT _data_pdfs00178f3386915a72 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_1 * f_in_inv_offsets_y[dir] + _stride_pdfs_2 * z + _stride_pdfs_2 * f_in_inv_offsets_z[dir] + _stride_pdfs_3 * f_in_inv_dir_idx[dir];
    uint8_t *RESTRICT _data_indexVector_116 = _data_indexVector + 16;
    uint8_t *RESTRICT _data_indexVector_124 = _data_indexVector + 24;
    uint8_t *RESTRICT _data_indexVector_132 = _data_indexVector + 32;
    double *RESTRICT _data_pdfs_10_20c107984fcf09b503 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + _stride_pdfs_3 * dir;
    _data_pdfs00178f3386915a72[_stride_pdfs_0 * x + _stride_pdfs_0 * f_in_inv_offsets_x[dir]] = -rho * (6.0 * ((double)(neighbour_offset_x[dir])) * *((double *)(&_data_indexVector_116[40 * blockDim.x * blockIdx.x + 40 * threadIdx.x])) + 6.0 * ((double)(neighbour_offset_y[dir])) * *((double *)(&_data_indexVector_124[40 * blockDim.x * blockIdx.x + 40 * threadIdx.x])) + 6.0 * ((double)(neighbour_offset_z[dir])) * *((double *)(&_data_indexVector_132[40 * blockDim.x * blockIdx.x + 40 * threadIdx.x]))) * weights[dir] + _data_pdfs_10_20c107984fcf09b503[_stride_pdfs_0 * x];
  }
}
} // namespace internal_dynamic_ubb_double_precisioncuda_boundary_Dynamic_UBB_double_precisionCUDA

// NOLINTEND(readability-non-const-parameter*)
#ifdef __GNUC__
#pragma GCC diagnostic pop
#endif

#ifdef __HIPCC__

#endif

void Dynamic_UBB_double_precisionCUDA::run_impl(IBlock *block, IndexVectors::Type type, gpuStream_t stream) {
  auto *indexVectors = block->getData<IndexVectors>(indexVectorID);
  int32_t indexVectorSize = int32_c(indexVectors->indexVector(type).size());
  if (indexVectorSize == 0)
    return;

  auto pointer = indexVectors->pointerGpu(type);

  uint8_t *_data_indexVector = reinterpret_cast<uint8_t *>(pointer);

  auto pdfs = block->getData<gpu::GPUField<double>>(pdfsID);

  WALBERLA_ASSERT_GREATER_EQUAL(0, -int_c(pdfs->nrOfGhostLayers()))
  double *RESTRICT _data_pdfs = pdfs->dataAt(0, 0, 0, 0);
  const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
  const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
  const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
  const int64_t _stride_pdfs_3 = int64_t(1 * int64_t(pdfs->fStride()));
  dim3 _block(uint32_c(((256 < indexVectorSize) ? 256 : indexVectorSize)), uint32_c(1), uint32_c(1));
  dim3 _grid(uint32_c(((indexVectorSize) % (((256 < indexVectorSize) ? 256 : indexVectorSize)) == 0 ? (int64_t)(indexVectorSize) / (int64_t)(((256 < indexVectorSize) ? 256 : indexVectorSize)) : ((int64_t)(indexVectorSize) / (int64_t)(((256 < indexVectorSize) ? 256 : indexVectorSize))) + 1)), uint32_c(1), uint32_c(1));
  internal_dynamic_ubb_double_precisioncuda_boundary_Dynamic_UBB_double_precisionCUDA::dynamic_ubb_double_precisioncuda_boundary_Dynamic_UBB_double_precisionCUDA<<<_grid, _block, 0, stream>>>(_data_indexVector, _data_pdfs, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3, indexVectorSize);
}

void Dynamic_UBB_double_precisionCUDA::run(IBlock *block, gpuStream_t stream) {
  run_impl(block, IndexVectors::ALL, stream);
}

void Dynamic_UBB_double_precisionCUDA::inner(IBlock *block, gpuStream_t stream) {
  run_impl(block, IndexVectors::INNER, stream);
}

void Dynamic_UBB_double_precisionCUDA::outer(IBlock *block, gpuStream_t stream) {
  run_impl(block, IndexVectors::OUTER, stream);
}

} // namespace lbm
} // namespace walberla
