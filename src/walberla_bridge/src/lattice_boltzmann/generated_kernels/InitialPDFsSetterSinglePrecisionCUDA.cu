#include "hip/hip_runtime.h"
//======================================================================================================================
//
//  This file is part of waLBerla. waLBerla is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  waLBerla is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
//  for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with waLBerla (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \\file InitialPDFsSetterSinglePrecisionCUDA.cpp
//! \\author pystencils
//======================================================================================================================

// kernel generated with pystencils v1.3.3, lbmpy v1.3.3, lbmpy_walberla/pystencils_walberla from waLBerla commit 04f4adbdfc0af983e2d9b72e244d775f37d77034

#include <cmath>

#include "InitialPDFsSetterSinglePrecisionCUDA.h"
#include "core/DataTypes.h"
#include "core/Macros.h"

#define FUNC_PREFIX __global__

#if (defined WALBERLA_CXX_COMPILER_IS_GNU) || (defined WALBERLA_CXX_COMPILER_IS_CLANG)
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wfloat-equal"
#pragma GCC diagnostic ignored "-Wshadow"
#pragma GCC diagnostic ignored "-Wconversion"
#pragma GCC diagnostic ignored "-Wunused-variable"
#endif

#if (defined WALBERLA_CXX_COMPILER_IS_INTEL)
#pragma warning push
#pragma warning(disable : 1599)
#endif

using namespace std;

namespace walberla {
namespace pystencils {

namespace internal_initialpdfssettersingleprecisioncuda_initialpdfssettersingleprecisioncuda {
static FUNC_PREFIX __launch_bounds__(256) void initialpdfssettersingleprecisioncuda_initialpdfssettersingleprecisioncuda(float *RESTRICT const _data_force, float *RESTRICT _data_pdfs, float *RESTRICT const _data_velocity, int64_t const _size_force_0, int64_t const _size_force_1, int64_t const _size_force_2, int64_t const _stride_force_0, int64_t const _stride_force_1, int64_t const _stride_force_2, int64_t const _stride_force_3, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3, int64_t const _stride_velocity_0, int64_t const _stride_velocity_1, int64_t const _stride_velocity_2, int64_t const _stride_velocity_3, float rho_0) {
  if (blockDim.x * blockIdx.x + threadIdx.x < _size_force_0 && blockDim.y * blockIdx.y + threadIdx.y < _size_force_1 && blockDim.z * blockIdx.z + threadIdx.z < _size_force_2) {
    const int64_t ctr_0 = blockDim.x * blockIdx.x + threadIdx.x;
    const int64_t ctr_1 = blockDim.y * blockIdx.y + threadIdx.y;
    const int64_t ctr_2 = blockDim.z * blockIdx.z + threadIdx.z;
    const float rho = rho_0;
    const float u_0 = -0.5f * ((1.0f) / (rho)) * _data_force[_stride_force_0 * ctr_0 + _stride_force_1 * ctr_1 + _stride_force_2 * ctr_2] + _data_velocity[_stride_velocity_0 * ctr_0 + _stride_velocity_1 * ctr_1 + _stride_velocity_2 * ctr_2];
    const float u_1 = -0.5f * ((1.0f) / (rho)) * _data_force[_stride_force_0 * ctr_0 + _stride_force_1 * ctr_1 + _stride_force_2 * ctr_2 + _stride_force_3] + _data_velocity[_stride_velocity_0 * ctr_0 + _stride_velocity_1 * ctr_1 + _stride_velocity_2 * ctr_2 + _stride_velocity_3];
    const float u_2 = -0.5f * ((1.0f) / (rho)) * _data_force[_stride_force_0 * ctr_0 + _stride_force_1 * ctr_1 + _stride_force_2 * ctr_2 + 2 * _stride_force_3] + _data_velocity[_stride_velocity_0 * ctr_0 + _stride_velocity_1 * ctr_1 + _stride_velocity_2 * ctr_2 + 2 * _stride_velocity_3];
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2] = rho * -0.33333333333333331f * (u_0 * u_0) + rho * -0.33333333333333331f * (u_1 * u_1) + rho * -0.33333333333333331f * (u_2 * u_2) + rho * 0.33333333333333331f;
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + _stride_pdfs_3] = rho * u_1 * 0.16666666666666666f + rho * -0.16666666666666666f * (u_0 * u_0) + rho * -0.16666666666666666f * (u_2 * u_2) + rho * 0.055555555555555552f + rho * 0.16666666666666666f * (u_1 * u_1);
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 2 * _stride_pdfs_3] = rho * u_1 * -0.16666666666666666f + rho * -0.16666666666666666f * (u_0 * u_0) + rho * -0.16666666666666666f * (u_2 * u_2) + rho * 0.055555555555555552f + rho * 0.16666666666666666f * (u_1 * u_1);
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 3 * _stride_pdfs_3] = rho * u_0 * -0.16666666666666666f + rho * -0.16666666666666666f * (u_1 * u_1) + rho * -0.16666666666666666f * (u_2 * u_2) + rho * 0.055555555555555552f + rho * 0.16666666666666666f * (u_0 * u_0);
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 4 * _stride_pdfs_3] = rho * u_0 * 0.16666666666666666f + rho * -0.16666666666666666f * (u_1 * u_1) + rho * -0.16666666666666666f * (u_2 * u_2) + rho * 0.055555555555555552f + rho * 0.16666666666666666f * (u_0 * u_0);
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 5 * _stride_pdfs_3] = rho * u_2 * 0.16666666666666666f + rho * -0.16666666666666666f * (u_0 * u_0) + rho * -0.16666666666666666f * (u_1 * u_1) + rho * 0.055555555555555552f + rho * 0.16666666666666666f * (u_2 * u_2);
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 6 * _stride_pdfs_3] = rho * u_2 * -0.16666666666666666f + rho * -0.16666666666666666f * (u_0 * u_0) + rho * -0.16666666666666666f * (u_1 * u_1) + rho * 0.055555555555555552f + rho * 0.16666666666666666f * (u_2 * u_2);
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 7 * _stride_pdfs_3] = rho * u_0 * u_1 * -0.25f + rho * u_0 * -0.083333333333333329f + rho * u_1 * 0.083333333333333329f + rho * 0.027777777777777776f + rho * 0.083333333333333329f * (u_0 * u_0) + rho * 0.083333333333333329f * (u_1 * u_1);
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 8 * _stride_pdfs_3] = rho * u_0 * u_1 * 0.25f + rho * u_0 * 0.083333333333333329f + rho * u_1 * 0.083333333333333329f + rho * 0.027777777777777776f + rho * 0.083333333333333329f * (u_0 * u_0) + rho * 0.083333333333333329f * (u_1 * u_1);
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 9 * _stride_pdfs_3] = rho * u_0 * u_1 * 0.25f + rho * u_0 * -0.083333333333333329f + rho * u_1 * -0.083333333333333329f + rho * 0.027777777777777776f + rho * 0.083333333333333329f * (u_0 * u_0) + rho * 0.083333333333333329f * (u_1 * u_1);
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 10 * _stride_pdfs_3] = rho * u_0 * u_1 * -0.25f + rho * u_0 * 0.083333333333333329f + rho * u_1 * -0.083333333333333329f + rho * 0.027777777777777776f + rho * 0.083333333333333329f * (u_0 * u_0) + rho * 0.083333333333333329f * (u_1 * u_1);
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 11 * _stride_pdfs_3] = rho * u_1 * u_2 * 0.25f + rho * u_1 * 0.083333333333333329f + rho * u_2 * 0.083333333333333329f + rho * 0.027777777777777776f + rho * 0.083333333333333329f * (u_1 * u_1) + rho * 0.083333333333333329f * (u_2 * u_2);
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 12 * _stride_pdfs_3] = rho * u_1 * u_2 * -0.25f + rho * u_1 * -0.083333333333333329f + rho * u_2 * 0.083333333333333329f + rho * 0.027777777777777776f + rho * 0.083333333333333329f * (u_1 * u_1) + rho * 0.083333333333333329f * (u_2 * u_2);
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 13 * _stride_pdfs_3] = rho * u_0 * u_2 * -0.25f + rho * u_0 * -0.083333333333333329f + rho * u_2 * 0.083333333333333329f + rho * 0.027777777777777776f + rho * 0.083333333333333329f * (u_0 * u_0) + rho * 0.083333333333333329f * (u_2 * u_2);
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 14 * _stride_pdfs_3] = rho * u_0 * u_2 * 0.25f + rho * u_0 * 0.083333333333333329f + rho * u_2 * 0.083333333333333329f + rho * 0.027777777777777776f + rho * 0.083333333333333329f * (u_0 * u_0) + rho * 0.083333333333333329f * (u_2 * u_2);
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 15 * _stride_pdfs_3] = rho * u_1 * u_2 * -0.25f + rho * u_1 * 0.083333333333333329f + rho * u_2 * -0.083333333333333329f + rho * 0.027777777777777776f + rho * 0.083333333333333329f * (u_1 * u_1) + rho * 0.083333333333333329f * (u_2 * u_2);
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 16 * _stride_pdfs_3] = rho * u_1 * u_2 * 0.25f + rho * u_1 * -0.083333333333333329f + rho * u_2 * -0.083333333333333329f + rho * 0.027777777777777776f + rho * 0.083333333333333329f * (u_1 * u_1) + rho * 0.083333333333333329f * (u_2 * u_2);
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 17 * _stride_pdfs_3] = rho * u_0 * u_2 * 0.25f + rho * u_0 * -0.083333333333333329f + rho * u_2 * -0.083333333333333329f + rho * 0.027777777777777776f + rho * 0.083333333333333329f * (u_0 * u_0) + rho * 0.083333333333333329f * (u_2 * u_2);
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 18 * _stride_pdfs_3] = rho * u_0 * u_2 * -0.25f + rho * u_0 * 0.083333333333333329f + rho * u_2 * -0.083333333333333329f + rho * 0.027777777777777776f + rho * 0.083333333333333329f * (u_0 * u_0) + rho * 0.083333333333333329f * (u_2 * u_2);
  }
}
} // namespace internal_initialpdfssettersingleprecisioncuda_initialpdfssettersingleprecisioncuda

void InitialPDFsSetterSinglePrecisionCUDA::run(IBlock *block, gpuStream_t stream) {

  auto force = block->getData<gpu::GPUField<float>>(forceID);
  auto velocity = block->getData<gpu::GPUField<float>>(velocityID);
  auto pdfs = block->getData<gpu::GPUField<float>>(pdfsID);

  auto &rho_0 = this->rho_0_;
  WALBERLA_ASSERT_GREATER_EQUAL(0, -int_c(force->nrOfGhostLayers()))
  float *RESTRICT const _data_force = force->dataAt(0, 0, 0, 0);
  WALBERLA_ASSERT_EQUAL(force->layout(), field::fzyx)
  WALBERLA_ASSERT_GREATER_EQUAL(0, -int_c(pdfs->nrOfGhostLayers()))
  float *RESTRICT _data_pdfs = pdfs->dataAt(0, 0, 0, 0);
  WALBERLA_ASSERT_EQUAL(pdfs->layout(), field::fzyx)
  WALBERLA_ASSERT_GREATER_EQUAL(0, -int_c(velocity->nrOfGhostLayers()))
  float *RESTRICT const _data_velocity = velocity->dataAt(0, 0, 0, 0);
  WALBERLA_ASSERT_EQUAL(velocity->layout(), field::fzyx)
  WALBERLA_ASSERT_GREATER_EQUAL(force->xSizeWithGhostLayer(), int64_t(int64_c(force->xSize()) + 0))
  const int64_t _size_force_0 = int64_t(int64_c(force->xSize()) + 0);
  WALBERLA_ASSERT_EQUAL(force->layout(), field::fzyx)
  WALBERLA_ASSERT_GREATER_EQUAL(force->ySizeWithGhostLayer(), int64_t(int64_c(force->ySize()) + 0))
  const int64_t _size_force_1 = int64_t(int64_c(force->ySize()) + 0);
  WALBERLA_ASSERT_EQUAL(force->layout(), field::fzyx)
  WALBERLA_ASSERT_GREATER_EQUAL(force->zSizeWithGhostLayer(), int64_t(int64_c(force->zSize()) + 0))
  const int64_t _size_force_2 = int64_t(int64_c(force->zSize()) + 0);
  WALBERLA_ASSERT_EQUAL(force->layout(), field::fzyx)
  const int64_t _stride_force_0 = int64_t(force->xStride());
  const int64_t _stride_force_1 = int64_t(force->yStride());
  const int64_t _stride_force_2 = int64_t(force->zStride());
  const int64_t _stride_force_3 = int64_t(1 * int64_t(force->fStride()));
  const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
  const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
  const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
  const int64_t _stride_pdfs_3 = int64_t(1 * int64_t(pdfs->fStride()));
  const int64_t _stride_velocity_0 = int64_t(velocity->xStride());
  const int64_t _stride_velocity_1 = int64_t(velocity->yStride());
  const int64_t _stride_velocity_2 = int64_t(velocity->zStride());
  const int64_t _stride_velocity_3 = int64_t(1 * int64_t(velocity->fStride()));
  dim3 _block(uint32_c(((128 < _size_force_0) ? 128 : _size_force_0)), uint32_c(((1024 < ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))) ? 1024 : ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))), uint32_c(((64 < ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))))) ? 64 : ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))))));
  dim3 _grid(uint32_c(((_size_force_0) % (((128 < _size_force_0) ? 128 : _size_force_0)) == 0 ? (int64_t)(_size_force_0) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)) : ((int64_t)(_size_force_0) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))) + 1)), uint32_c(((_size_force_1) % (((1024 < ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))) ? 1024 : ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))) == 0 ? (int64_t)(_size_force_1) / (int64_t)(((1024 < ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))) ? 1024 : ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))) : ((int64_t)(_size_force_1) / (int64_t)(((1024 < ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))) ? 1024 : ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) + 1)), uint32_c(((_size_force_2) % (((64 < ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))))) ? 64 : ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))))) == 0 ? (int64_t)(_size_force_2) / (int64_t)(((64 < ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))))) ? 64 : ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))))) : ((int64_t)(_size_force_2) / (int64_t)(((64 < ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))))) ? 64 : ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))))))) + 1)));
  internal_initialpdfssettersingleprecisioncuda_initialpdfssettersingleprecisioncuda::initialpdfssettersingleprecisioncuda_initialpdfssettersingleprecisioncuda<<<_grid, _block, 0, stream>>>(_data_force, _data_pdfs, _data_velocity, _size_force_0, _size_force_1, _size_force_2, _stride_force_0, _stride_force_1, _stride_force_2, _stride_force_3, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3, _stride_velocity_0, _stride_velocity_1, _stride_velocity_2, _stride_velocity_3, rho_0);
}

void InitialPDFsSetterSinglePrecisionCUDA::runOnCellInterval(const shared_ptr<StructuredBlockStorage> &blocks, const CellInterval &globalCellInterval, cell_idx_t ghostLayers, IBlock *block, gpuStream_t stream) {

  CellInterval ci = globalCellInterval;
  CellInterval blockBB = blocks->getBlockCellBB(*block);
  blockBB.expand(ghostLayers);
  ci.intersect(blockBB);
  blocks->transformGlobalToBlockLocalCellInterval(ci, *block);
  if (ci.empty())
    return;

  auto force = block->getData<gpu::GPUField<float>>(forceID);
  auto velocity = block->getData<gpu::GPUField<float>>(velocityID);
  auto pdfs = block->getData<gpu::GPUField<float>>(pdfsID);

  auto &rho_0 = this->rho_0_;
  WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(force->nrOfGhostLayers()))
  WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(force->nrOfGhostLayers()))
  WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(force->nrOfGhostLayers()))
  float *RESTRICT const _data_force = force->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
  WALBERLA_ASSERT_EQUAL(force->layout(), field::fzyx)
  WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()))
  WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()))
  WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()))
  float *RESTRICT _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
  WALBERLA_ASSERT_EQUAL(pdfs->layout(), field::fzyx)
  WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(velocity->nrOfGhostLayers()))
  WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(velocity->nrOfGhostLayers()))
  WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(velocity->nrOfGhostLayers()))
  float *RESTRICT const _data_velocity = velocity->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
  WALBERLA_ASSERT_EQUAL(velocity->layout(), field::fzyx)
  WALBERLA_ASSERT_GREATER_EQUAL(force->xSizeWithGhostLayer(), int64_t(int64_c(ci.xSize()) + 0))
  const int64_t _size_force_0 = int64_t(int64_c(ci.xSize()) + 0);
  WALBERLA_ASSERT_EQUAL(force->layout(), field::fzyx)
  WALBERLA_ASSERT_GREATER_EQUAL(force->ySizeWithGhostLayer(), int64_t(int64_c(ci.ySize()) + 0))
  const int64_t _size_force_1 = int64_t(int64_c(ci.ySize()) + 0);
  WALBERLA_ASSERT_EQUAL(force->layout(), field::fzyx)
  WALBERLA_ASSERT_GREATER_EQUAL(force->zSizeWithGhostLayer(), int64_t(int64_c(ci.zSize()) + 0))
  const int64_t _size_force_2 = int64_t(int64_c(ci.zSize()) + 0);
  WALBERLA_ASSERT_EQUAL(force->layout(), field::fzyx)
  const int64_t _stride_force_0 = int64_t(force->xStride());
  const int64_t _stride_force_1 = int64_t(force->yStride());
  const int64_t _stride_force_2 = int64_t(force->zStride());
  const int64_t _stride_force_3 = int64_t(1 * int64_t(force->fStride()));
  const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
  const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
  const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
  const int64_t _stride_pdfs_3 = int64_t(1 * int64_t(pdfs->fStride()));
  const int64_t _stride_velocity_0 = int64_t(velocity->xStride());
  const int64_t _stride_velocity_1 = int64_t(velocity->yStride());
  const int64_t _stride_velocity_2 = int64_t(velocity->zStride());
  const int64_t _stride_velocity_3 = int64_t(1 * int64_t(velocity->fStride()));
  dim3 _block(uint32_c(((128 < _size_force_0) ? 128 : _size_force_0)), uint32_c(((1024 < ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))) ? 1024 : ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))), uint32_c(((64 < ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))))) ? 64 : ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))))));
  dim3 _grid(uint32_c(((_size_force_0) % (((128 < _size_force_0) ? 128 : _size_force_0)) == 0 ? (int64_t)(_size_force_0) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)) : ((int64_t)(_size_force_0) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))) + 1)), uint32_c(((_size_force_1) % (((1024 < ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))) ? 1024 : ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))) == 0 ? (int64_t)(_size_force_1) / (int64_t)(((1024 < ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))) ? 1024 : ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))) : ((int64_t)(_size_force_1) / (int64_t)(((1024 < ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))) ? 1024 : ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) + 1)), uint32_c(((_size_force_2) % (((64 < ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))))) ? 64 : ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))))) == 0 ? (int64_t)(_size_force_2) / (int64_t)(((64 < ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))))) ? 64 : ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))))) : ((int64_t)(_size_force_2) / (int64_t)(((64 < ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))))) ? 64 : ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))))))) + 1)));
  internal_initialpdfssettersingleprecisioncuda_initialpdfssettersingleprecisioncuda::initialpdfssettersingleprecisioncuda_initialpdfssettersingleprecisioncuda<<<_grid, _block, 0, stream>>>(_data_force, _data_pdfs, _data_velocity, _size_force_0, _size_force_1, _size_force_2, _stride_force_0, _stride_force_1, _stride_force_2, _stride_force_3, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3, _stride_velocity_0, _stride_velocity_1, _stride_velocity_2, _stride_velocity_3, rho_0);
}

} // namespace pystencils
} // namespace walberla

#if (defined WALBERLA_CXX_COMPILER_IS_GNU) || (defined WALBERLA_CXX_COMPILER_IS_CLANG)
#pragma GCC diagnostic pop
#endif

#if (defined WALBERLA_CXX_COMPILER_IS_INTEL)
#pragma warning pop
#endif
