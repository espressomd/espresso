#include "hip/hip_runtime.h"
//======================================================================================================================
//
//  This file is part of waLBerla. waLBerla is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  waLBerla is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
//  for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with waLBerla (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \\file CollideSweepDoublePrecisionThermalizedCUDA.cpp
//! \\author pystencils
//======================================================================================================================

// kernel generated with pystencils v1.3.3, lbmpy v1.3.3, lbmpy_walberla/pystencils_walberla from waLBerla commit 04f4adbdfc0af983e2d9b72e244d775f37d77034

#include <cmath>

#include "CollideSweepDoublePrecisionThermalizedCUDA.h"
#include "core/DataTypes.h"
#include "core/Macros.h"

#include "philox_rand.h"

#define FUNC_PREFIX __global__

#if (defined WALBERLA_CXX_COMPILER_IS_GNU) || (defined WALBERLA_CXX_COMPILER_IS_CLANG)
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wfloat-equal"
#pragma GCC diagnostic ignored "-Wshadow"
#pragma GCC diagnostic ignored "-Wconversion"
#pragma GCC diagnostic ignored "-Wunused-variable"
#endif

#if (defined WALBERLA_CXX_COMPILER_IS_INTEL)
#pragma warning push
#pragma warning(disable : 1599)
#endif

using namespace std;

namespace walberla {
namespace pystencils {

namespace internal_collidesweepdoubleprecisionthermalizedcuda_collidesweepdoubleprecisionthermalizedcuda {
static FUNC_PREFIX __launch_bounds__(256) void collidesweepdoubleprecisionthermalizedcuda_collidesweepdoubleprecisionthermalizedcuda(double *RESTRICT const _data_force, double *RESTRICT _data_pdfs, int64_t const _size_force_0, int64_t const _size_force_1, int64_t const _size_force_2, int64_t const _stride_force_0, int64_t const _stride_force_1, int64_t const _stride_force_2, int64_t const _stride_force_3, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3, uint32_t block_offset_0, uint32_t block_offset_1, uint32_t block_offset_2, double kT, double omega_bulk, double omega_even, double omega_odd, double omega_shear, uint32_t seed, uint32_t time_step) {
  if (blockDim.x * blockIdx.x + threadIdx.x < _size_force_0 && blockDim.y * blockIdx.y + threadIdx.y < _size_force_1 && blockDim.z * blockIdx.z + threadIdx.z < _size_force_2) {
    const int64_t ctr_0 = blockDim.x * blockIdx.x + threadIdx.x;
    const int64_t ctr_1 = blockDim.y * blockIdx.y + threadIdx.y;
    const int64_t ctr_2 = blockDim.z * blockIdx.z + threadIdx.z;
    const double xi_244 = _data_force[_stride_force_0 * ctr_0 + _stride_force_1 * ctr_1 + _stride_force_2 * ctr_2 + 2 * _stride_force_3];
    const double xi_245 = _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 13 * _stride_pdfs_3];
    const double xi_246 = _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 10 * _stride_pdfs_3];
    const double xi_247 = _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 3 * _stride_pdfs_3];
    const double xi_248 = _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 17 * _stride_pdfs_3];
    const double xi_249 = _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 11 * _stride_pdfs_3];
    const double xi_250 = _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 6 * _stride_pdfs_3];
    const double xi_251 = _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 8 * _stride_pdfs_3];
    const double xi_252 = _data_force[_stride_force_0 * ctr_0 + _stride_force_1 * ctr_1 + _stride_force_2 * ctr_2 + _stride_force_3];
    const double xi_253 = _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 5 * _stride_pdfs_3];
    const double xi_254 = _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 14 * _stride_pdfs_3];
    const double xi_255 = _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 15 * _stride_pdfs_3];
    const double xi_256 = _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 16 * _stride_pdfs_3];
    const double xi_257 = _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 4 * _stride_pdfs_3];
    const double xi_258 = _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 2 * _stride_pdfs_3];
    const double xi_259 = _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 9 * _stride_pdfs_3];
    const double xi_260 = _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 18 * _stride_pdfs_3];
    const double xi_261 = _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 12 * _stride_pdfs_3];
    const double xi_262 = _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2];
    const double xi_263 = _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + _stride_pdfs_3];
    const double xi_264 = _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 7 * _stride_pdfs_3];
    const double xi_265 = _data_force[_stride_force_0 * ctr_0 + _stride_force_1 * ctr_1 + _stride_force_2 * ctr_2];

    double random_7_0{};
    double random_7_1{};
    if (kT > 0.) {
      philox_double2(time_step, block_offset_0 + ctr_0, block_offset_1 + ctr_1, block_offset_2 + ctr_2, 7, seed, random_7_0, random_7_1);
    }

    double random_6_0{};
    double random_6_1{};
    if (kT > 0.) {
      philox_double2(time_step, block_offset_0 + ctr_0, block_offset_1 + ctr_1, block_offset_2 + ctr_2, 6, seed, random_6_0, random_6_1);
    }

    double random_5_0{};
    double random_5_1{};
    if (kT > 0.) {
      philox_double2(time_step, block_offset_0 + ctr_0, block_offset_1 + ctr_1, block_offset_2 + ctr_2, 5, seed, random_5_0, random_5_1);
    }

    double random_4_0{};
    double random_4_1{};
    if (kT > 0.) {
      philox_double2(time_step, block_offset_0 + ctr_0, block_offset_1 + ctr_1, block_offset_2 + ctr_2, 4, seed, random_4_0, random_4_1);
    }

    double random_3_0{};
    double random_3_1{};
    if (kT > 0.) {
      philox_double2(time_step, block_offset_0 + ctr_0, block_offset_1 + ctr_1, block_offset_2 + ctr_2, 3, seed, random_3_0, random_3_1);
    }

    double random_2_0{};
    double random_2_1{};
    if (kT > 0.) {
      philox_double2(time_step, block_offset_0 + ctr_0, block_offset_1 + ctr_1, block_offset_2 + ctr_2, 2, seed, random_2_0, random_2_1);
    }

    double random_1_0{};
    double random_1_1{};
    if (kT > 0.) {
      philox_double2(time_step, block_offset_0 + ctr_0, block_offset_1 + ctr_1, block_offset_2 + ctr_2, 1, seed, random_1_0, random_1_1);
    }

    double random_0_0{};
    double random_0_1{};
    if (kT > 0.) {
      philox_double2(time_step, block_offset_0 + ctr_0, block_offset_1 + ctr_1, block_offset_2 + ctr_2, 0, seed, random_0_0, random_0_1);
    }
    const double xi_2 = xi_254 + xi_260;
    const double xi_3 = xi_2 + xi_257;
    const double xi_4 = xi_249 + xi_255 + xi_263;
    const double xi_5 = xi_253 + xi_261;
    const double xi_6 = xi_248 + xi_250;
    const double xi_8 = -xi_259;
    const double xi_9 = -xi_264;
    const double xi_10 = -xi_248;
    const double xi_11 = -xi_245;
    const double xi_12 = -xi_247;
    const double xi_13 = xi_10 + xi_11 + xi_12;
    const double xi_14 = -xi_258;
    const double xi_15 = -xi_246;
    const double xi_16 = xi_14 + xi_15;
    const double xi_17 = -xi_256;
    const double xi_18 = -xi_261;
    const double xi_19 = xi_17 + xi_18;
    const double xi_20 = -xi_260;
    const double xi_21 = xi_10 + xi_20;
    const double xi_22 = -xi_255;
    const double xi_23 = -xi_250;
    const double xi_24 = xi_17 + xi_22 + xi_23 + xi_249;
    const double xi_28 = omega_bulk * 0.5;
    const double xi_29 = xi_252 * 0.16666666666666666;
    const double xi_30 = xi_252 * 0.083333333333333329;
    const double xi_42 = xi_265 * 0.16666666666666666;
    const double xi_43 = xi_265 * 0.083333333333333329;
    const double xi_49 = xi_244 * 0.16666666666666666;
    const double xi_50 = xi_244 * 0.083333333333333329;
    const double xi_55 = omega_shear * 0.041666666666666664;
    const double xi_60 = omega_bulk * 0.041666666666666664;
    const double xi_67 = xi_252 * 0.25;
    const double xi_71 = omega_shear * 0.125;
    const double xi_72 = xi_252 * xi_71;
    const double xi_109 = 2.4494897427831779;
    const double xi_114 = -xi_262;
    const double xi_118 = -xi_249;
    const double xi_119 = xi_118 + xi_18;
    const double xi_120 = -xi_251 + xi_8;
    const double xi_122 = -xi_254;
    const double xi_123 = xi_11 + xi_122 + xi_15 + xi_21;
    const double xi_125 = xi_249 * 2.0 + xi_255 * 2.0 + xi_256 * 2.0 + xi_261 * 2.0;
    const double xi_126 = xi_125 + xi_247 * 5.0 + xi_257 * 5.0;
    const double xi_128 = xi_245 * 2.0;
    const double xi_129 = xi_254 * 2.0;
    const double xi_130 = xi_248 * 2.0 + xi_260 * 2.0;
    const double xi_132 = xi_118 + xi_261;
    const double xi_133 = xi_132 + xi_14 + xi_22 + xi_256 + xi_263;
    const double xi_134 = omega_odd * 0.25;
    const double xi_135 = xi_133 * xi_134;
    const double xi_136 = random_5_1 - 0.5;
    const double xi_141 = xi_264 * 2.0;
    const double xi_142 = xi_246 * 2.0;
    const double xi_143 = xi_251 * -2.0 + xi_259 * 2.0;
    const double xi_144 = xi_14 - xi_141 + xi_142 + xi_143 + xi_19 + xi_4;
    const double xi_145 = omega_odd * 0.083333333333333329;
    const double xi_146 = xi_144 * xi_145;
    const double xi_147 = random_3_0 - 0.5;
    const double xi_152 = random_0_1 - 0.5;
    const double xi_166 = xi_122 + xi_245;
    const double xi_167 = xi_12 + xi_166 + xi_20 + xi_248 + xi_257;
    const double xi_168 = xi_134 * xi_167;
    const double xi_169 = random_4_1 - 0.5;
    const double xi_171 = xi_13 + xi_141 - xi_142 + xi_143 + xi_3;
    const double xi_172 = xi_145 * xi_171;
    const double xi_173 = random_4_0 - 0.5;
    const double xi_178 = xi_119 + xi_23 + xi_253 + xi_255 + xi_256;
    const double xi_179 = xi_134 * xi_178;
    const double xi_180 = random_5_0 - 0.5;
    const double xi_182 = -xi_128 - xi_129 + xi_130 + xi_24 + xi_5;
    const double xi_183 = xi_145 * xi_182;
    const double xi_184 = random_3_1 - 0.5;
    const double xi_198 = omega_shear * 0.25;
    const double xi_211 = omega_odd * 0.041666666666666664;
    const double xi_212 = xi_182 * xi_211;
    const double xi_213 = omega_odd * 0.125;
    const double xi_214 = xi_178 * xi_213;
    const double xi_220 = xi_144 * xi_211;
    const double xi_221 = xi_133 * xi_213;
    const double xi_235 = xi_167 * xi_213;
    const double xi_236 = xi_171 * xi_211;
    const double rr_0 = 0.0;
    const double xi_31 = rr_0 * xi_30;
    const double xi_44 = rr_0 * xi_43;
    const double xi_51 = rr_0 * xi_50;
    const double xi_53 = rr_0 * 0.041666666666666664;
    const double xi_54 = xi_265 * xi_53;
    const double xi_59 = xi_252 * xi_53;
    const double xi_81 = xi_244 * xi_53;
    const double vel0Term = xi_246 + xi_251 + xi_3;
    const double vel1Term = xi_264 + xi_4;
    const double vel2Term = xi_245 + xi_5;
    const double rho = vel0Term + vel1Term + vel2Term + xi_247 + xi_256 + xi_258 + xi_259 + xi_262 + xi_6;
    const double xi_105 = kT * rho;
    const double xi_106 = pow(xi_105 * (1.0 - (-omega_even + 1.0) * (-omega_even + 1.0)), 0.5);
    const double xi_107 = xi_106 * (random_6_0 - 0.5) * 3.7416573867739413;
    const double xi_108 = xi_106 * (random_7_0 - 0.5) * 5.4772255750516612;
    const double xi_110 = xi_109 * (random_2_1 - 0.5) * pow(xi_105 * (1.0 - (-omega_bulk + 1.0) * (-omega_bulk + 1.0)), 0.5);
    const double xi_111 = xi_106 * (random_6_1 - 0.5) * 8.3666002653407556;
    const double xi_137 = pow(xi_105 * (1.0 - (-omega_odd + 1.0) * (-omega_odd + 1.0)), 0.5);
    const double xi_138 = xi_137 * 1.4142135623730951;
    const double xi_139 = xi_138 * 0.5;
    const double xi_140 = xi_136 * xi_139;
    const double xi_148 = xi_109 * xi_137;
    const double xi_149 = xi_148 * 0.16666666666666666;
    const double xi_150 = xi_147 * xi_149;
    const double xi_151 = -xi_146 - xi_150;
    const double xi_153 = pow(xi_105 * (1.0 - (-omega_shear + 1.0) * (-omega_shear + 1.0)), 0.5);
    const double xi_154 = xi_153 * 0.5;
    const double xi_155 = xi_152 * xi_154;
    const double xi_161 = xi_153 * (random_0_0 - 0.5) * 1.7320508075688772;
    const double xi_165 = xi_146 + xi_150;
    const double xi_170 = xi_139 * xi_169;
    const double xi_174 = xi_149 * xi_173;
    const double xi_175 = xi_172 + xi_174;
    const double xi_177 = -xi_172 - xi_174;
    const double xi_181 = xi_139 * xi_180;
    const double xi_185 = xi_149 * xi_184;
    const double xi_186 = -xi_183 - xi_185;
    const double xi_188 = xi_183 + xi_185;
    const double xi_189 = xi_152 * xi_153 * 0.25;
    const double xi_192 = xi_107 * 0.083333333333333329;
    const double xi_196 = xi_154 * (random_1_0 - 0.5);
    const double xi_203 = xi_154 * (random_2_0 - 0.5);
    const double xi_207 = xi_111 * -0.014285714285714285;
    const double xi_208 = xi_108 * 0.050000000000000003;
    const double xi_215 = xi_148 * 0.083333333333333329;
    const double xi_216 = xi_184 * xi_215;
    const double xi_217 = xi_138 * 0.25;
    const double xi_218 = xi_180 * xi_217;
    const double xi_219 = -xi_212 + xi_214 - xi_216 + xi_218;
    const double xi_222 = xi_147 * xi_215;
    const double xi_223 = xi_136 * xi_217;
    const double xi_224 = -xi_220 + xi_221 - xi_222 + xi_223;
    const double xi_225 = xi_220 - xi_221 + xi_222 - xi_223;
    const double xi_227 = -xi_189;
    const double xi_230 = xi_111 * 0.035714285714285712;
    const double xi_232 = xi_154 * (random_1_1 - 0.5);
    const double xi_237 = xi_169 * xi_217;
    const double xi_238 = xi_173 * xi_215;
    const double xi_239 = -xi_235 + xi_236 - xi_237 + xi_238;
    const double xi_241 = xi_235 - xi_236 + xi_237 - xi_238;
    const double xi_242 = xi_212 - xi_214 + xi_216 - xi_218;
    const double xi_0 = ((1.0) / (rho));
    const double xi_7 = xi_0 * 0.5;
    const double u_0 = xi_0 * (vel0Term + xi_13 + xi_8 + xi_9) + xi_265 * xi_7;
    const double xi_25 = u_0 * xi_265;
    const double xi_37 = xi_25 * 0.16666666666666666;
    const double xi_38 = xi_25 * 0.083333333333333329;
    const double xi_39 = omega_shear * xi_38;
    const double xi_40 = -xi_37 + xi_39;
    const double xi_56 = -xi_25 * xi_55 + xi_37;
    const double xi_57 = -xi_43 + xi_54 + xi_56;
    const double xi_61 = -xi_25 * xi_60;
    const double xi_68 = u_0 * xi_67;
    const double xi_73 = u_0 * xi_72;
    const double xi_77 = xi_43 - xi_54 + xi_56;
    const double xi_84 = -xi_38;
    const double xi_95 = u_0 * xi_244;
    const double xi_96 = xi_95 * 0.25;
    const double xi_99 = xi_71 * xi_95;
    const double xi_113 = rho * (u_0 * u_0);
    const double u_1 = xi_0 * (vel1Term + xi_16 + xi_19 + xi_251 + xi_8) + xi_252 * xi_7;
    const double xi_26 = u_1 * xi_252;
    const double xi_32 = xi_26 * 0.16666666666666666;
    const double xi_45 = xi_26 * 0.083333333333333329;
    const double xi_46 = omega_shear * xi_45;
    const double xi_47 = -xi_32 + xi_46;
    const double xi_62 = -xi_26 * xi_60;
    const double xi_69 = u_1 * 0.25;
    const double xi_70 = xi_265 * xi_69;
    const double xi_74 = u_1 * xi_71;
    const double xi_75 = xi_265 * xi_74;
    const double xi_76 = -xi_68 - xi_70 + xi_73 + xi_75;
    const double xi_78 = xi_68 + xi_70 - xi_73 - xi_75;
    const double xi_86 = xi_244 * xi_69;
    const double xi_88 = xi_244 * xi_74;
    const double xi_93 = -xi_45;
    const double xi_112 = rho * (u_1 * u_1);
    const double xi_121 = xi_112 + xi_120 + xi_9;
    const double xi_197 = rho * u_1;
    const double xi_199 = xi_198 * (u_0 * xi_197 + xi_120 + xi_246 + xi_264);
    const double xi_200 = -xi_196 - xi_199;
    const double xi_201 = xi_196 + xi_199;
    const double u_2 = xi_0 * (vel2Term + xi_21 + xi_24 + xi_254) + xi_244 * xi_7;
    const double xi_27 = u_2 * xi_244;
    const double xi_33 = xi_27 * 0.16666666666666666;
    const double xi_34 = xi_27 * 0.083333333333333329;
    const double xi_35 = omega_shear * xi_34;
    const double xi_36 = -xi_33 + xi_35;
    const double xi_41 = -omega_shear * xi_32 + xi_26 * 0.33333333333333331 + xi_36 + xi_40;
    const double xi_48 = -omega_shear * xi_37 + xi_25 * 0.33333333333333331 + xi_36 + xi_47;
    const double xi_52 = -omega_shear * xi_33 + xi_27 * 0.33333333333333331 + xi_40 + xi_47;
    const double xi_58 = -xi_34;
    const double xi_63 = -xi_27 * xi_60;
    const double xi_64 = -xi_26 * xi_55 + xi_32 + xi_61 + xi_62 + xi_63;
    const double xi_65 = xi_30 - xi_59 + xi_64;
    const double xi_66 = xi_35 + xi_58 + xi_65;
    const double xi_79 = -xi_30 + xi_59 + xi_64;
    const double xi_80 = xi_35 + xi_58 + xi_79;
    const double xi_82 = -xi_27 * xi_55 + xi_33;
    const double xi_83 = xi_50 - xi_81 + xi_82;
    const double xi_85 = xi_39 + xi_65 + xi_84;
    const double xi_87 = u_2 * xi_67;
    const double xi_89 = u_2 * xi_72;
    const double xi_90 = xi_86 + xi_87 - xi_88 - xi_89;
    const double xi_91 = xi_39 + xi_79 + xi_84;
    const double xi_92 = -xi_86 - xi_87 + xi_88 + xi_89;
    const double xi_94 = xi_46 + xi_61 + xi_62 + xi_63 + xi_83 + xi_93;
    const double xi_97 = u_2 * xi_265;
    const double xi_98 = xi_97 * 0.25;
    const double xi_100 = xi_71 * xi_97;
    const double xi_101 = xi_100 - xi_96 - xi_98 + xi_99;
    const double xi_102 = -xi_100 + xi_96 + xi_98 - xi_99;
    const double xi_103 = -xi_50 + xi_81 + xi_82;
    const double xi_104 = xi_103 + xi_46 + xi_61 + xi_62 + xi_63 + xi_93;
    const double xi_115 = rho * (u_2 * u_2);
    const double xi_116 = xi_114 + xi_115 * 0.66666666666666663 + xi_250 * 3.0 + xi_253 * 3.0;
    const double xi_117 = omega_even * (xi_112 * 0.66666666666666663 + xi_113 * 1.6666666666666667 + xi_116 + xi_249 * -3.0 + xi_255 * -3.0 + xi_256 * -3.0 + xi_258 * 3.0 + xi_261 * -3.0 + xi_263 * 3.0);
    const double xi_124 = omega_bulk * (xi_113 + xi_115 + xi_119 + xi_121 + xi_123 + xi_17 + xi_22 + xi_262);
    const double xi_127 = omega_even * (xi_112 * 2.3333333333333335 + xi_116 + xi_126 + xi_245 * -5.0 + xi_248 * -5.0 + xi_254 * -5.0 + xi_258 * -2.0 + xi_260 * -5.0 + xi_263 * -2.0);
    const double xi_131 = omega_even * (xi_114 + xi_115 * 3.0 + xi_126 + xi_128 + xi_129 + xi_130 + xi_246 * -7.0 + xi_250 * -4.0 + xi_251 * -7.0 + xi_253 * -4.0 + xi_258 * 5.0 + xi_259 * -7.0 + xi_263 * 5.0 + xi_264 * -7.0);
    const double xi_156 = -xi_115 + xi_253;
    const double xi_157 = omega_shear * (xi_121 + xi_156 + xi_16 + xi_2 + xi_245 - xi_263 + xi_6);
    const double xi_158 = xi_157 * 0.125;
    const double xi_159 = xi_107 * -0.11904761904761904 + xi_131 * -0.01984126984126984;
    const double xi_160 = omega_shear * (-xi_112 + xi_113 * 2.0 + xi_120 + xi_123 + xi_125 + xi_156 + xi_247 * -2.0 + xi_250 + xi_257 * -2.0 + xi_258 + xi_263 + xi_9);
    const double xi_162 = xi_160 * -0.041666666666666664 + xi_161 * -0.16666666666666666;
    const double xi_163 = xi_108 * -0.10000000000000001 + xi_117 * -0.050000000000000003 + xi_162;
    const double xi_164 = xi_111 * 0.028571428571428571 + xi_127 * 0.014285714285714285 + xi_155 + xi_158 + xi_159 + xi_163;
    const double xi_176 = xi_111 * -0.071428571428571425 + xi_127 * -0.035714285714285712 + xi_159 + xi_160 * 0.083333333333333329 + xi_161 * 0.33333333333333331;
    const double xi_187 = xi_107 * 0.095238095238095233 + xi_111 * -0.042857142857142858 + xi_127 * -0.021428571428571429 + xi_131 * 0.015873015873015872 - xi_155 - xi_158 + xi_163;
    const double xi_190 = xi_157 * 0.0625;
    const double xi_191 = xi_131 * 0.013888888888888888;
    const double xi_193 = xi_110 * 0.083333333333333329 + xi_124 * 0.041666666666666664;
    const double xi_194 = xi_160 * 0.020833333333333332 + xi_161 * 0.083333333333333329 + xi_193;
    const double xi_195 = xi_165 + xi_189 + xi_190 + xi_191 + xi_192 + xi_194;
    const double xi_202 = xi_151 + xi_189 + xi_190 + xi_191 + xi_192 + xi_194;
    const double xi_204 = xi_127 * -0.0071428571428571426;
    const double xi_205 = xi_198 * (u_2 * xi_197 + xi_132 + xi_17 + xi_255);
    const double xi_206 = xi_117 * 0.025000000000000001;
    const double xi_209 = xi_107 * -0.023809523809523808 + xi_131 * -0.003968253968253968;
    const double xi_210 = xi_162 + xi_193 + xi_203 + xi_204 + xi_205 + xi_206 + xi_207 + xi_208 + xi_209;
    const double xi_226 = xi_162 + xi_193 - xi_203 + xi_204 - xi_205 + xi_206 + xi_207 + xi_208 + xi_209;
    const double xi_228 = -xi_190;
    const double xi_229 = xi_127 * 0.017857142857142856;
    const double xi_231 = xi_188 + xi_194 + xi_209 + xi_227 + xi_228 + xi_229 + xi_230;
    const double xi_233 = xi_198 * (rho * u_0 * u_2 + xi_10 + xi_166 + xi_260);
    const double xi_234 = -xi_232 - xi_233;
    const double xi_240 = xi_232 + xi_233;
    const double xi_243 = xi_186 + xi_194 + xi_209 + xi_227 + xi_228 + xi_229 + xi_230;
    const double forceTerm_0 = xi_25 * xi_28 - xi_25 + xi_26 * xi_28 - xi_26 + xi_27 * xi_28 - xi_27;
    const double forceTerm_1 = xi_29 - xi_31 + xi_41;
    const double forceTerm_2 = -xi_29 + xi_31 + xi_41;
    const double forceTerm_3 = -xi_42 + xi_44 + xi_48;
    const double forceTerm_4 = xi_42 - xi_44 + xi_48;
    const double forceTerm_5 = xi_49 - xi_51 + xi_52;
    const double forceTerm_6 = -xi_49 + xi_51 + xi_52;
    const double forceTerm_7 = xi_57 + xi_66 + xi_76;
    const double forceTerm_8 = xi_66 + xi_77 + xi_78;
    const double forceTerm_9 = xi_57 + xi_78 + xi_80;
    const double forceTerm_10 = xi_76 + xi_77 + xi_80;
    const double forceTerm_11 = xi_83 + xi_85 + xi_90;
    const double forceTerm_12 = xi_83 + xi_91 + xi_92;
    const double forceTerm_13 = xi_101 + xi_57 + xi_94;
    const double forceTerm_14 = xi_102 + xi_77 + xi_94;
    const double forceTerm_15 = xi_103 + xi_85 + xi_92;
    const double forceTerm_16 = xi_103 + xi_90 + xi_91;
    const double forceTerm_17 = xi_102 + xi_104 + xi_57;
    const double forceTerm_18 = xi_101 + xi_104 + xi_77;
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2] = forceTerm_0 + xi_107 * 0.14285714285714285 + xi_108 * 0.20000000000000001 - xi_110 + xi_111 * 0.085714285714285715 + xi_117 * 0.10000000000000001 + xi_124 * -0.5 + xi_127 * 0.042857142857142858 + xi_131 * 0.023809523809523808 + xi_262;
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + _stride_pdfs_3] = forceTerm_1 - xi_135 - xi_140 + xi_151 + xi_164 + xi_263;
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 2 * _stride_pdfs_3] = forceTerm_2 + xi_135 + xi_140 + xi_164 + xi_165 + xi_258;
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 3 * _stride_pdfs_3] = forceTerm_3 + xi_168 + xi_170 + xi_175 + xi_176 + xi_247;
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 4 * _stride_pdfs_3] = forceTerm_4 - xi_168 - xi_170 + xi_176 + xi_177 + xi_257;
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 5 * _stride_pdfs_3] = forceTerm_5 - xi_179 - xi_181 + xi_186 + xi_187 + xi_253;
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 6 * _stride_pdfs_3] = forceTerm_6 + xi_179 + xi_181 + xi_187 + xi_188 + xi_250;
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 7 * _stride_pdfs_3] = forceTerm_7 + xi_177 + xi_195 + xi_200 + xi_264;
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 8 * _stride_pdfs_3] = forceTerm_8 + xi_175 + xi_195 + xi_201 + xi_251;
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 9 * _stride_pdfs_3] = forceTerm_9 + xi_177 + xi_201 + xi_202 + xi_259;
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 10 * _stride_pdfs_3] = forceTerm_10 + xi_175 + xi_200 + xi_202 + xi_246;
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 11 * _stride_pdfs_3] = forceTerm_11 + xi_210 + xi_219 + xi_224 + xi_249;
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 12 * _stride_pdfs_3] = forceTerm_12 + xi_219 + xi_225 + xi_226 + xi_261;
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 13 * _stride_pdfs_3] = forceTerm_13 + xi_231 + xi_234 + xi_239 + xi_245;
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 14 * _stride_pdfs_3] = forceTerm_14 + xi_231 + xi_240 + xi_241 + xi_254;
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 15 * _stride_pdfs_3] = forceTerm_15 + xi_224 + xi_226 + xi_242 + xi_255;
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 16 * _stride_pdfs_3] = forceTerm_16 + xi_210 + xi_225 + xi_242 + xi_256;
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 17 * _stride_pdfs_3] = forceTerm_17 + xi_239 + xi_240 + xi_243 + xi_248;
    _data_pdfs[_stride_pdfs_0 * ctr_0 + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 18 * _stride_pdfs_3] = forceTerm_18 + xi_234 + xi_241 + xi_243 + xi_260;
  }
}
} // namespace internal_collidesweepdoubleprecisionthermalizedcuda_collidesweepdoubleprecisionthermalizedcuda

void CollideSweepDoublePrecisionThermalizedCUDA::run(IBlock *block, gpuStream_t stream) {
  if (!this->configured_)
    WALBERLA_ABORT("This Sweep contains a configure function that needs to be called manually")

  auto force = block->getData<gpu::GPUField<double>>(forceID);
  auto pdfs = block->getData<gpu::GPUField<double>>(pdfsID);

  auto &omega_bulk = this->omega_bulk_;
  auto &kT = this->kT_;
  auto &block_offset_0 = this->block_offset_0_;
  auto &block_offset_2 = this->block_offset_2_;
  auto &block_offset_1 = this->block_offset_1_;
  auto &omega_even = this->omega_even_;
  auto &seed = this->seed_;
  auto &omega_odd = this->omega_odd_;
  auto &time_step = this->time_step_;
  auto &omega_shear = this->omega_shear_;
  WALBERLA_ASSERT_GREATER_EQUAL(0, -int_c(force->nrOfGhostLayers()))
  double *RESTRICT const _data_force = force->dataAt(0, 0, 0, 0);
  WALBERLA_ASSERT_EQUAL(force->layout(), field::fzyx)
  WALBERLA_ASSERT_GREATER_EQUAL(0, -int_c(pdfs->nrOfGhostLayers()))
  double *RESTRICT _data_pdfs = pdfs->dataAt(0, 0, 0, 0);
  WALBERLA_ASSERT_EQUAL(pdfs->layout(), field::fzyx)
  WALBERLA_ASSERT_GREATER_EQUAL(force->xSizeWithGhostLayer(), int64_t(int64_c(force->xSize()) + 0))
  const int64_t _size_force_0 = int64_t(int64_c(force->xSize()) + 0);
  WALBERLA_ASSERT_EQUAL(force->layout(), field::fzyx)
  WALBERLA_ASSERT_GREATER_EQUAL(force->ySizeWithGhostLayer(), int64_t(int64_c(force->ySize()) + 0))
  const int64_t _size_force_1 = int64_t(int64_c(force->ySize()) + 0);
  WALBERLA_ASSERT_EQUAL(force->layout(), field::fzyx)
  WALBERLA_ASSERT_GREATER_EQUAL(force->zSizeWithGhostLayer(), int64_t(int64_c(force->zSize()) + 0))
  const int64_t _size_force_2 = int64_t(int64_c(force->zSize()) + 0);
  WALBERLA_ASSERT_EQUAL(force->layout(), field::fzyx)
  const int64_t _stride_force_0 = int64_t(force->xStride());
  const int64_t _stride_force_1 = int64_t(force->yStride());
  const int64_t _stride_force_2 = int64_t(force->zStride());
  const int64_t _stride_force_3 = int64_t(1 * int64_t(force->fStride()));
  const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
  const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
  const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
  const int64_t _stride_pdfs_3 = int64_t(1 * int64_t(pdfs->fStride()));
  dim3 _block(uint32_c(((128 < _size_force_0) ? 128 : _size_force_0)), uint32_c(((1024 < ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))) ? 1024 : ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))), uint32_c(((64 < ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))))) ? 64 : ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))))));
  dim3 _grid(uint32_c(((_size_force_0) % (((128 < _size_force_0) ? 128 : _size_force_0)) == 0 ? (int64_t)(_size_force_0) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)) : ((int64_t)(_size_force_0) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))) + 1)), uint32_c(((_size_force_1) % (((1024 < ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))) ? 1024 : ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))) == 0 ? (int64_t)(_size_force_1) / (int64_t)(((1024 < ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))) ? 1024 : ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))) : ((int64_t)(_size_force_1) / (int64_t)(((1024 < ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))) ? 1024 : ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) + 1)), uint32_c(((_size_force_2) % (((64 < ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))))) ? 64 : ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))))) == 0 ? (int64_t)(_size_force_2) / (int64_t)(((64 < ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))))) ? 64 : ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))))) : ((int64_t)(_size_force_2) / (int64_t)(((64 < ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))))) ? 64 : ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))))))) + 1)));
  internal_collidesweepdoubleprecisionthermalizedcuda_collidesweepdoubleprecisionthermalizedcuda::collidesweepdoubleprecisionthermalizedcuda_collidesweepdoubleprecisionthermalizedcuda<<<_grid, _block, 0, stream>>>(_data_force, _data_pdfs, _size_force_0, _size_force_1, _size_force_2, _stride_force_0, _stride_force_1, _stride_force_2, _stride_force_3, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3, block_offset_0, block_offset_1, block_offset_2, kT, omega_bulk, omega_even, omega_odd, omega_shear, seed, time_step);
}

void CollideSweepDoublePrecisionThermalizedCUDA::runOnCellInterval(const shared_ptr<StructuredBlockStorage> &blocks, const CellInterval &globalCellInterval, cell_idx_t ghostLayers, IBlock *block, gpuStream_t stream) {
  if (!this->configured_)
    WALBERLA_ABORT("This Sweep contains a configure function that needs to be called manually")

  CellInterval ci = globalCellInterval;
  CellInterval blockBB = blocks->getBlockCellBB(*block);
  blockBB.expand(ghostLayers);
  ci.intersect(blockBB);
  blocks->transformGlobalToBlockLocalCellInterval(ci, *block);
  if (ci.empty())
    return;

  auto force = block->getData<gpu::GPUField<double>>(forceID);
  auto pdfs = block->getData<gpu::GPUField<double>>(pdfsID);

  auto &omega_bulk = this->omega_bulk_;
  auto &kT = this->kT_;
  auto &block_offset_0 = this->block_offset_0_;
  auto &block_offset_2 = this->block_offset_2_;
  auto &block_offset_1 = this->block_offset_1_;
  auto &omega_even = this->omega_even_;
  auto &seed = this->seed_;
  auto &omega_odd = this->omega_odd_;
  auto &time_step = this->time_step_;
  auto &omega_shear = this->omega_shear_;
  WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(force->nrOfGhostLayers()))
  WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(force->nrOfGhostLayers()))
  WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(force->nrOfGhostLayers()))
  double *RESTRICT const _data_force = force->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
  WALBERLA_ASSERT_EQUAL(force->layout(), field::fzyx)
  WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()))
  WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()))
  WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()))
  double *RESTRICT _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
  WALBERLA_ASSERT_EQUAL(pdfs->layout(), field::fzyx)
  WALBERLA_ASSERT_GREATER_EQUAL(force->xSizeWithGhostLayer(), int64_t(int64_c(ci.xSize()) + 0))
  const int64_t _size_force_0 = int64_t(int64_c(ci.xSize()) + 0);
  WALBERLA_ASSERT_EQUAL(force->layout(), field::fzyx)
  WALBERLA_ASSERT_GREATER_EQUAL(force->ySizeWithGhostLayer(), int64_t(int64_c(ci.ySize()) + 0))
  const int64_t _size_force_1 = int64_t(int64_c(ci.ySize()) + 0);
  WALBERLA_ASSERT_EQUAL(force->layout(), field::fzyx)
  WALBERLA_ASSERT_GREATER_EQUAL(force->zSizeWithGhostLayer(), int64_t(int64_c(ci.zSize()) + 0))
  const int64_t _size_force_2 = int64_t(int64_c(ci.zSize()) + 0);
  WALBERLA_ASSERT_EQUAL(force->layout(), field::fzyx)
  const int64_t _stride_force_0 = int64_t(force->xStride());
  const int64_t _stride_force_1 = int64_t(force->yStride());
  const int64_t _stride_force_2 = int64_t(force->zStride());
  const int64_t _stride_force_3 = int64_t(1 * int64_t(force->fStride()));
  const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
  const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
  const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
  const int64_t _stride_pdfs_3 = int64_t(1 * int64_t(pdfs->fStride()));
  dim3 _block(uint32_c(((128 < _size_force_0) ? 128 : _size_force_0)), uint32_c(((1024 < ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))) ? 1024 : ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))), uint32_c(((64 < ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))))) ? 64 : ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))))));
  dim3 _grid(uint32_c(((_size_force_0) % (((128 < _size_force_0) ? 128 : _size_force_0)) == 0 ? (int64_t)(_size_force_0) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)) : ((int64_t)(_size_force_0) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))) + 1)), uint32_c(((_size_force_1) % (((1024 < ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))) ? 1024 : ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))) == 0 ? (int64_t)(_size_force_1) / (int64_t)(((1024 < ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))) ? 1024 : ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))) : ((int64_t)(_size_force_1) / (int64_t)(((1024 < ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))) ? 1024 : ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) + 1)), uint32_c(((_size_force_2) % (((64 < ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))))) ? 64 : ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))))) == 0 ? (int64_t)(_size_force_2) / (int64_t)(((64 < ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))))) ? 64 : ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))))) : ((int64_t)(_size_force_2) / (int64_t)(((64 < ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))))) ? 64 : ((_size_force_2 < ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0) * ((_size_force_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))) ? _size_force_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_force_0) ? 128 : _size_force_0)))))))))) + 1)));
  internal_collidesweepdoubleprecisionthermalizedcuda_collidesweepdoubleprecisionthermalizedcuda::collidesweepdoubleprecisionthermalizedcuda_collidesweepdoubleprecisionthermalizedcuda<<<_grid, _block, 0, stream>>>(_data_force, _data_pdfs, _size_force_0, _size_force_1, _size_force_2, _stride_force_0, _stride_force_1, _stride_force_2, _stride_force_3, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3, block_offset_0, block_offset_1, block_offset_2, kT, omega_bulk, omega_even, omega_odd, omega_shear, seed, time_step);
}

} // namespace pystencils
} // namespace walberla

#if (defined WALBERLA_CXX_COMPILER_IS_GNU) || (defined WALBERLA_CXX_COMPILER_IS_CLANG)
#pragma GCC diagnostic pop
#endif

#if (defined WALBERLA_CXX_COMPILER_IS_INTEL)
#pragma warning pop
#endif
