#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023-2024 The ESPResSo project
 * Copyright (C) 2020 The waLBerla project
 *
 * This file is part of ESPResSo.
 *
 * ESPResSo is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * ESPResSo is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

// kernel generated with pystencils v1.3.3, lbmpy v1.3.3, lbmpy_walberla/pystencils_walberla from waLBerla commit 04f4adbdfc0af983e2d9b72e244d775f37d77034

/**
 * @file
 * Lattice field accessors.
 * Adapted from the waLBerla source file
 * https://i10git.cs.fau.de/walberla/walberla/-/blob/a16141524c58ab88386e2a0f8fdd7c63c5edd704/python/lbmpy_walberla/templates/LatticeModel.tmpl.h
 */

#include <core/DataTypes.h>
#include <core/cell/Cell.h>
#include <core/cell/CellInterval.h>
#include <core/math/Matrix3.h>
#include <core/math/Vector3.h>

#include <field/iterators/IteratorMacros.h>

#include <gpu/FieldAccessor.h>
#include <gpu/FieldIndexing.h>
#include <gpu/GPUField.h>
#include <gpu/Kernel.h>

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>

#include <array>
#include <vector>

#if defined(__NVCC__)
#define RESTRICT __restrict__
#pragma nv_diagnostic push
#pragma nv_diag_suppress 177 // unused variable
#elif defined(__clang__)
#if defined(__CUDA__)
#if defined(__CUDA_ARCH__)
// clang compiling CUDA code in device mode
#define RESTRICT __restrict__
#pragma clang diagnostic push
#pragma clang diagnostic ignored "-Wunused-variable"
#else
// clang compiling CUDA code in host mode
#define RESTRICT __restrict__
#pragma clang diagnostic push
#pragma clang diagnostic ignored "-Wunused-variable"
#endif
#endif
#elif defined(__GNUC__) or defined(__GNUG__)
#define RESTRICT __restrict__
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wunused-variable"
#elif defined(_MSC_VER)
#define RESTRICT __restrict
#else
#define RESTRICT
#endif

/** @brief Get linear index of flattened data with original layout @c fzyx. */
static __forceinline__ __device__ uint getLinearIndex(uint3 blockIdx, uint3 threadIdx, uint3 gridDim, uint3 blockDim, uint fOffset) {
  auto const x = threadIdx.x;
  auto const y = blockIdx.x;
  auto const z = blockIdx.y;
  auto const f = blockIdx.z;
  auto const ySize = gridDim.x;
  auto const zSize = gridDim.y;
  auto const fSize = fOffset;
  return f +
         z * fSize +
         y * fSize * zSize +
         x * fSize * zSize * ySize;
}

namespace walberla {
namespace lbm {
namespace accessor {

namespace Population {
// LCOV_EXCL_START
__global__ void kernel_get(
    gpu::FieldAccessor<double> pdf,
    double *RESTRICT pop) {
  auto const offset = getLinearIndex(blockIdx, threadIdx, gridDim, blockDim, 19u);
  pdf.set(blockIdx, threadIdx);
  pop += offset;
  if (pdf.isValidPosition()) {
    pop[0u] = pdf.get(0u);
    pop[1u] = pdf.get(1u);
    pop[2u] = pdf.get(2u);
    pop[3u] = pdf.get(3u);
    pop[4u] = pdf.get(4u);
    pop[5u] = pdf.get(5u);
    pop[6u] = pdf.get(6u);
    pop[7u] = pdf.get(7u);
    pop[8u] = pdf.get(8u);
    pop[9u] = pdf.get(9u);
    pop[10u] = pdf.get(10u);
    pop[11u] = pdf.get(11u);
    pop[12u] = pdf.get(12u);
    pop[13u] = pdf.get(13u);
    pop[14u] = pdf.get(14u);
    pop[15u] = pdf.get(15u);
    pop[16u] = pdf.get(16u);
    pop[17u] = pdf.get(17u);
    pop[18u] = pdf.get(18u);
  }
}

__global__ void kernel_set(
    gpu::FieldAccessor<double> pdf,
    double const *RESTRICT pop) {
  auto const offset = getLinearIndex(blockIdx, threadIdx, gridDim, blockDim, 19u);
  pdf.set(blockIdx, threadIdx);
  pop += offset;
  if (pdf.isValidPosition()) {
    pdf.get(0u) = pop[0u];
    pdf.get(1u) = pop[1u];
    pdf.get(2u) = pop[2u];
    pdf.get(3u) = pop[3u];
    pdf.get(4u) = pop[4u];
    pdf.get(5u) = pop[5u];
    pdf.get(6u) = pop[6u];
    pdf.get(7u) = pop[7u];
    pdf.get(8u) = pop[8u];
    pdf.get(9u) = pop[9u];
    pdf.get(10u) = pop[10u];
    pdf.get(11u) = pop[11u];
    pdf.get(12u) = pop[12u];
    pdf.get(13u) = pop[13u];
    pdf.get(14u) = pop[14u];
    pdf.get(15u) = pop[15u];
    pdf.get(16u) = pop[16u];
    pdf.get(17u) = pop[17u];
    pdf.get(18u) = pop[18u];
  }
}

__global__ void kernel_broadcast(
    gpu::FieldAccessor<double> pdf,
    double const *RESTRICT pop) {
  pdf.set(blockIdx, threadIdx);
  if (pdf.isValidPosition()) {
    pdf.get(0u) = pop[0u];
    pdf.get(1u) = pop[1u];
    pdf.get(2u) = pop[2u];
    pdf.get(3u) = pop[3u];
    pdf.get(4u) = pop[4u];
    pdf.get(5u) = pop[5u];
    pdf.get(6u) = pop[6u];
    pdf.get(7u) = pop[7u];
    pdf.get(8u) = pop[8u];
    pdf.get(9u) = pop[9u];
    pdf.get(10u) = pop[10u];
    pdf.get(11u) = pop[11u];
    pdf.get(12u) = pop[12u];
    pdf.get(13u) = pop[13u];
    pdf.get(14u) = pop[14u];
    pdf.get(15u) = pop[15u];
    pdf.get(16u) = pop[16u];
    pdf.get(17u) = pop[17u];
    pdf.get(18u) = pop[18u];
  }
}

__global__ void kernel_set_vel(
    gpu::FieldAccessor<double> pdf,
    gpu::FieldAccessor<double> velocity,
    gpu::FieldAccessor<double> force,
    double const *RESTRICT pop) {
  auto const offset = getLinearIndex(blockIdx, threadIdx, gridDim, blockDim, 19u);
  pdf.set(blockIdx, threadIdx);
  velocity.set(blockIdx, threadIdx);
  force.set(blockIdx, threadIdx);
  pop += offset;
  if (pdf.isValidPosition()) {
    const double f_0 = pdf.get(0u) = pop[0u];
    const double f_1 = pdf.get(1u) = pop[1u];
    const double f_2 = pdf.get(2u) = pop[2u];
    const double f_3 = pdf.get(3u) = pop[3u];
    const double f_4 = pdf.get(4u) = pop[4u];
    const double f_5 = pdf.get(5u) = pop[5u];
    const double f_6 = pdf.get(6u) = pop[6u];
    const double f_7 = pdf.get(7u) = pop[7u];
    const double f_8 = pdf.get(8u) = pop[8u];
    const double f_9 = pdf.get(9u) = pop[9u];
    const double f_10 = pdf.get(10u) = pop[10u];
    const double f_11 = pdf.get(11u) = pop[11u];
    const double f_12 = pdf.get(12u) = pop[12u];
    const double f_13 = pdf.get(13u) = pop[13u];
    const double f_14 = pdf.get(14u) = pop[14u];
    const double f_15 = pdf.get(15u) = pop[15u];
    const double f_16 = pdf.get(16u) = pop[16u];
    const double f_17 = pdf.get(17u) = pop[17u];
    const double f_18 = pdf.get(18u) = pop[18u];
    const double vel0Term = f_10 + f_14 + f_18 + f_4 + f_8;
    const double momdensity_0 = -f_13 - f_17 - f_3 - f_7 - f_9 + vel0Term;
    const double vel1Term = f_1 + f_11 + f_15 + f_7;
    const double momdensity_1 = -f_10 - f_12 - f_16 - f_2 + f_8 - f_9 + vel1Term;
    const double vel2Term = f_12 + f_13 + f_5;
    const double momdensity_2 = f_11 + f_14 - f_15 - f_16 - f_17 - f_18 - f_6 + vel2Term;
    const double rho = f_0 + f_16 + f_17 + f_2 + f_3 + f_6 + f_9 + vel0Term + vel1Term + vel2Term;
    const double md_0 = force.get(0) * 0.50000000000000000 + momdensity_0;
    const double md_1 = force.get(1) * 0.50000000000000000 + momdensity_1;
    const double md_2 = force.get(2) * 0.50000000000000000 + momdensity_2;
    const double rho_inv = double{1} / rho;
    velocity.get(0u) = md_0 * rho_inv;
    velocity.get(1u) = md_1 * rho_inv;
    velocity.get(2u) = md_2 * rho_inv;
  }
}
// LCOV_EXCL_STOP

std::array<double, 19u> get(
    gpu::GPUField<double> const *pdf_field,
    Cell const &cell) {
  CellInterval ci(cell, cell);
  thrust::device_vector<double> dev_data(19u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_get);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addParam(dev_data_ptr);
  kernel();
  std::array<double, 19u> pop;
  thrust::copy(dev_data.begin(), dev_data.end(), pop.data());
  return pop;
}

void set(
    gpu::GPUField<double> *pdf_field,
    std::array<double, 19u> const &pop,
    Cell const &cell) {
  thrust::device_vector<double> dev_data(pop.begin(), pop.end());
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  CellInterval ci(cell, cell);
  auto kernel = gpu::make_kernel(kernel_set);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addParam(const_cast<const double *>(dev_data_ptr));
  kernel();
}

void set(
    gpu::GPUField<double> *pdf_field,
    gpu::GPUField<double> *velocity_field,
    gpu::GPUField<double> const *force_field,
    std::array<double, 19u> const &pop,
    Cell const &cell) {
  thrust::device_vector<double> dev_data(pop.begin(), pop.end());
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  CellInterval ci(cell, cell);
  auto kernel = gpu::make_kernel(kernel_set_vel);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*velocity_field, ci));
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*force_field, ci));
  kernel.addParam(const_cast<const double *>(dev_data_ptr));
  kernel();
}

void initialize(
    gpu::GPUField<double> *pdf_field,
    std::array<double, 19u> const &pop) {
  CellInterval ci = pdf_field->xyzSizeWithGhostLayer();
  thrust::device_vector<double> dev_data(pop.begin(), pop.end());
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_broadcast);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addParam(const_cast<const double *>(dev_data_ptr));
  kernel();
}

std::vector<double> get(
    gpu::GPUField<double> const *pdf_field,
    CellInterval const &ci) {
  thrust::device_vector<double> dev_data(ci.numCells() * 19u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_get);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addParam(dev_data_ptr);
  kernel();
  std::vector<double> out(ci.numCells() * 19u);
  thrust::copy(dev_data.begin(), dev_data.end(), out.data());
  return out;
}

void set(
    gpu::GPUField<double> *pdf_field,
    std::vector<double> const &values,
    CellInterval const &ci) {
  thrust::device_vector<double> dev_data(values.begin(), values.end());
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_set);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addParam(const_cast<const double *>(dev_data_ptr));
  kernel();
}

void set(
    gpu::GPUField<double> *pdf_field,
    gpu::GPUField<double> *velocity_field,
    gpu::GPUField<double> const *force_field,
    std::vector<double> const &values,
    CellInterval const &ci) {
  thrust::device_vector<double> dev_data(values.begin(), values.end());
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_set_vel);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*velocity_field, ci));
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*force_field, ci));
  kernel.addParam(const_cast<const double *>(dev_data_ptr));
  kernel();
}
} // namespace Population

namespace Vector {
// LCOV_EXCL_START
__global__ void kernel_get(
    gpu::FieldAccessor<double> vec,
    double *u_out) {
  auto const offset = getLinearIndex(blockIdx, threadIdx, gridDim, blockDim, 3u);
  vec.set(blockIdx, threadIdx);
  u_out += offset;
  if (vec.isValidPosition()) {
    u_out[0u] = vec.get(0u);
    u_out[1u] = vec.get(1u);
    u_out[2u] = vec.get(2u);
  }
}

__global__ void kernel_set(
    gpu::FieldAccessor<double> vec,
    double const *RESTRICT u_in) {
  auto const offset = getLinearIndex(blockIdx, threadIdx, gridDim, blockDim, 3u);
  vec.set(blockIdx, threadIdx);
  u_in += offset;
  if (vec.isValidPosition()) {
    vec.get(0u) = u_in[0u];
    vec.get(1u) = u_in[1u];
    vec.get(2u) = u_in[2u];
  }
}

__global__ void kernel_broadcast(
    gpu::FieldAccessor<double> vec,
    double const *RESTRICT u_in) {
  vec.set(blockIdx, threadIdx);
  if (vec.isValidPosition()) {
    vec.get(0u) = u_in[0u];
    vec.get(1u) = u_in[1u];
    vec.get(2u) = u_in[2u];
  }
}

__global__ void kernel_add(
    gpu::FieldAccessor<double> vec,
    double const *RESTRICT u_in) {
  auto const offset = getLinearIndex(blockIdx, threadIdx, gridDim, blockDim, 3u);
  vec.set(blockIdx, threadIdx);
  u_in += offset;
  if (vec.isValidPosition()) {
    vec.get(0u) += u_in[0u];
    vec.get(1u) += u_in[1u];
    vec.get(2u) += u_in[2u];
  }
}

__global__ void kernel_broadcast_add(
    gpu::FieldAccessor<double> vec,
    double const *RESTRICT u_in) {
  vec.set(blockIdx, threadIdx);
  if (vec.isValidPosition()) {
    vec.get(0u) += u_in[0u];
    vec.get(1u) += u_in[1u];
    vec.get(2u) += u_in[2u];
  }
}
// LCOV_EXCL_STOP

Vector3<double> get(
    gpu::GPUField<double> const *vec_field,
    Cell const &cell) {
  CellInterval ci(cell, cell);
  thrust::device_vector<double> dev_data(3u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_get);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*vec_field, ci));
  kernel.addParam(dev_data_ptr);
  kernel();
  Vector3<double> vec;
  thrust::copy(dev_data.begin(), dev_data.end(), vec.data());
  return vec;
}

void set(
    gpu::GPUField<double> *vec_field,
    Vector3<double> const &vec,
    Cell const &cell) {
  CellInterval ci(cell, cell);
  thrust::device_vector<double> dev_data(vec.data(), vec.data() + 3u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_set);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*vec_field, ci));
  kernel.addParam(const_cast<const double *>(dev_data_ptr));
  kernel();
}

void add(
    gpu::GPUField<double> *vec_field,
    Vector3<double> const &vec,
    Cell const &cell) {
  CellInterval ci(cell, cell);
  thrust::device_vector<double> dev_data(vec.data(), vec.data() + 3u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_add);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*vec_field, ci));
  kernel.addParam(const_cast<const double *>(dev_data_ptr));
  kernel();
}

void initialize(
    gpu::GPUField<double> *vec_field,
    Vector3<double> const &vec) {
  CellInterval ci = vec_field->xyzSizeWithGhostLayer();
  thrust::device_vector<double> dev_data(vec.data(), vec.data() + 3u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_broadcast);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*vec_field, ci));
  kernel.addParam(const_cast<const double *>(dev_data_ptr));
  kernel();
}

void add_to_all(
    gpu::GPUField<double> *vec_field,
    Vector3<double> const &vec) {
  CellInterval ci = vec_field->xyzSizeWithGhostLayer();
  thrust::device_vector<double> dev_data(vec.data(), vec.data() + 3u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_broadcast_add);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*vec_field, ci));
  kernel.addParam(const_cast<const double *>(dev_data_ptr));
  kernel();
}

std::vector<double> get(
    gpu::GPUField<double> const *vec_field,
    CellInterval const &ci) {
  thrust::device_vector<double> dev_data(ci.numCells() * 3u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_get);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*vec_field, ci));
  kernel.addParam(dev_data_ptr);
  kernel();
  std::vector<double> out(ci.numCells() * 3u);
  thrust::copy(dev_data.begin(), dev_data.end(), out.data());
  return out;
}

void set(
    gpu::GPUField<double> *vec_field,
    std::vector<double> const &values,
    CellInterval const &ci) {
  thrust::device_vector<double> dev_data(values.begin(), values.end());
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_set);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*vec_field, ci));
  kernel.addParam(const_cast<const double *>(dev_data_ptr));
  kernel();
}
} // namespace Vector

namespace Interpolation {
// LCOV_EXCL_START
/** @brief Calculate interpolation weights. */
static __forceinline__ __device__ void calculate_weights(
    double const *RESTRICT const pos,
    int *RESTRICT const corner,
    double *RESTRICT const weights,
    uint gl) {
#pragma unroll
  for (int dim = 0; dim < 3; ++dim) {
    auto const fractional_index = pos[dim] - double{0.5};
    auto const nmp = floorf(fractional_index);
    auto const distance = fractional_index - nmp - double{0.5};
    corner[dim] = __double2int_rn(nmp) + static_cast<int>(gl);
    weights[dim * 2 + 0] = double{0.5} - distance;
    weights[dim * 2 + 1] = double{0.5} + distance;
  }
}

__global__ void kernel_get(
    gpu::FieldAccessor<double> vec,
    double const *RESTRICT const pos,
    double *RESTRICT const vel,
    uint n_pos,
    uint gl) {

  uint pos_index = blockIdx.y * gridDim.x * blockDim.x +
                   blockDim.x * blockIdx.x + threadIdx.x;

  vec.set({0u, 0u, 0u}, {0u, 0u, 0u});
  if (vec.isValidPosition() and pos_index < n_pos) {
    auto const array_offset = pos_index * uint(3u);
    int corner[3];
    double weights[3][2];
    calculate_weights(pos + array_offset, corner, &weights[0][0], gl);
#pragma unroll
    for (int i = 0; i < 2; i++) {
      auto const cx = corner[0] + i;
      auto const wx = weights[0][i];
#pragma unroll
      for (int j = 0; j < 2; j++) {
        auto const cy = corner[1] + j;
        auto const wxy = wx * weights[1][j];
#pragma unroll
        for (int k = 0; k < 2; k++) {
          auto const cz = corner[2] + k;
          auto const weight = wxy * weights[2][k];
          vel[array_offset + 0u] += weight * vec.getNeighbor(cx, cy, cz, 0u);
          vel[array_offset + 1u] += weight * vec.getNeighbor(cx, cy, cz, 1u);
          vel[array_offset + 2u] += weight * vec.getNeighbor(cx, cy, cz, 2u);
        }
      }
    }
  }
}

__global__ void kernel_set(
    gpu::FieldAccessor<double> vec,
    double const *RESTRICT const pos,
    double const *RESTRICT const forces,
    uint n_pos,
    uint gl) {

  uint pos_index = blockIdx.y * gridDim.x * blockDim.x +
                   blockDim.x * blockIdx.x + threadIdx.x;

  vec.set({0u, 0u, 0u}, {0u, 0u, 0u});
  if (vec.isValidPosition() and pos_index < n_pos) {
    auto const array_offset = pos_index * uint(3u);
    int corner[3];
    double weights[3][2];
    calculate_weights(pos + array_offset, corner, &weights[0][0], gl);
#pragma unroll
    for (int i = 0; i < 2; i++) {
      auto const cx = corner[0] + i;
      auto const wx = weights[0][i];
#pragma unroll
      for (int j = 0; j < 2; j++) {
        auto const cy = corner[1] + j;
        auto const wxy = wx * weights[1][j];
#pragma unroll
        for (int k = 0; k < 2; k++) {
          auto const cz = corner[2] + k;
          auto const weight = wxy * weights[2][k];
          atomicAdd(&vec.getNeighbor(cx, cy, cz, 0u),
                    weight * forces[array_offset + 0u]);
          atomicAdd(&vec.getNeighbor(cx, cy, cz, 1u),
                    weight * forces[array_offset + 1u]);
          atomicAdd(&vec.getNeighbor(cx, cy, cz, 2u),
                    weight * forces[array_offset + 2u]);
        }
      }
    }
  }
}
// LCOV_EXCL_STOP

static dim3 calculate_dim_grid(uint const threads_x,
                               uint const blocks_per_grid_y,
                               uint const threads_per_block) {
  assert(threads_x >= 1u);
  assert(blocks_per_grid_y >= 1u);
  assert(threads_per_block >= 1u);
  auto const threads_y = threads_per_block * blocks_per_grid_y;
  auto const blocks_per_grid_x = (threads_x + threads_y - 1) / threads_y;
  return make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);
}

std::vector<double>
get(
    gpu::GPUField<double> const *vec_field,
    std::vector<double> const &pos,
    uint gl) {
  thrust::device_vector<double> dev_pos(pos.begin(), pos.end());
  thrust::device_vector<double> dev_vel(pos.size());
  auto const dev_pos_ptr = thrust::raw_pointer_cast(dev_pos.data());
  auto const dev_vel_ptr = thrust::raw_pointer_cast(dev_vel.data());

  auto const threads_per_block = uint(64u);
  auto const n_pos = static_cast<uint>(pos.size() / 3ul);
  auto const dim_grid = calculate_dim_grid(n_pos, 4u, threads_per_block);
  kernel_get<<<dim_grid, threads_per_block, 0u, nullptr>>>(
      gpu::FieldIndexing<double>::withGhostLayerXYZ(*vec_field, gl).gpuAccess(),
      dev_pos_ptr, dev_vel_ptr, n_pos, gl);

  std::vector<double> out(pos.size());
  thrust::copy(dev_vel.begin(), dev_vel.end(), out.data());
  return out;
}

void set(
    gpu::GPUField<double> const *vec_field,
    std::vector<double> const &pos,
    std::vector<double> const &forces,
    uint gl) {
  thrust::device_vector<double> dev_pos(pos.begin(), pos.end());
  thrust::device_vector<double> dev_for(forces.begin(), forces.end());
  auto const dev_pos_ptr = thrust::raw_pointer_cast(dev_pos.data());
  auto const dev_for_ptr = thrust::raw_pointer_cast(dev_for.data());

  auto const threads_per_block = uint(64u);
  auto const n_pos = static_cast<uint>(pos.size() / 3ul);
  auto const dim_grid = calculate_dim_grid(n_pos, 4u, threads_per_block);
  kernel_set<<<dim_grid, threads_per_block, 0u, nullptr>>>(
      gpu::FieldIndexing<double>::withGhostLayerXYZ(*vec_field, gl).gpuAccess(),
      dev_pos_ptr, dev_for_ptr, n_pos, gl);
}
} // namespace Interpolation

namespace Equilibrium {
// LCOV_EXCL_START
__device__ void kernel_set_device(
    gpu::FieldAccessor<double> pdf,
    double const *RESTRICT const u,
    double rho) {

  pdf.get(0u) = rho * -0.33333333333333331 * (u[0] * u[0]) + rho * -0.33333333333333331 * (u[1] * u[1]) + rho * -0.33333333333333331 * (u[2] * u[2]) + rho * 0.33333333333333331;
  pdf.get(1u) = rho * -0.16666666666666666 * (u[0] * u[0]) + rho * -0.16666666666666666 * (u[2] * u[2]) + rho * 0.055555555555555552 + rho * 0.16666666666666666 * u[1] + rho * 0.16666666666666666 * (u[1] * u[1]);
  pdf.get(2u) = rho * -0.16666666666666666 * u[1] + rho * -0.16666666666666666 * (u[0] * u[0]) + rho * -0.16666666666666666 * (u[2] * u[2]) + rho * 0.055555555555555552 + rho * 0.16666666666666666 * (u[1] * u[1]);
  pdf.get(3u) = rho * -0.16666666666666666 * u[0] + rho * -0.16666666666666666 * (u[1] * u[1]) + rho * -0.16666666666666666 * (u[2] * u[2]) + rho * 0.055555555555555552 + rho * 0.16666666666666666 * (u[0] * u[0]);
  pdf.get(4u) = rho * -0.16666666666666666 * (u[1] * u[1]) + rho * -0.16666666666666666 * (u[2] * u[2]) + rho * 0.055555555555555552 + rho * 0.16666666666666666 * u[0] + rho * 0.16666666666666666 * (u[0] * u[0]);
  pdf.get(5u) = rho * -0.16666666666666666 * (u[0] * u[0]) + rho * -0.16666666666666666 * (u[1] * u[1]) + rho * 0.055555555555555552 + rho * 0.16666666666666666 * u[2] + rho * 0.16666666666666666 * (u[2] * u[2]);
  pdf.get(6u) = rho * -0.16666666666666666 * u[2] + rho * -0.16666666666666666 * (u[0] * u[0]) + rho * -0.16666666666666666 * (u[1] * u[1]) + rho * 0.055555555555555552 + rho * 0.16666666666666666 * (u[2] * u[2]);
  pdf.get(7u) = rho * -0.083333333333333329 * u[0] + rho * -0.25 * u[0] * u[1] + rho * 0.027777777777777776 + rho * 0.083333333333333329 * u[1] + rho * 0.083333333333333329 * (u[0] * u[0]) + rho * 0.083333333333333329 * (u[1] * u[1]);
  pdf.get(8u) = rho * 0.027777777777777776 + rho * 0.083333333333333329 * u[0] + rho * 0.083333333333333329 * u[1] + rho * 0.083333333333333329 * (u[0] * u[0]) + rho * 0.083333333333333329 * (u[1] * u[1]) + rho * 0.25 * u[0] * u[1];
  pdf.get(9u) = rho * -0.083333333333333329 * u[0] + rho * -0.083333333333333329 * u[1] + rho * 0.027777777777777776 + rho * 0.083333333333333329 * (u[0] * u[0]) + rho * 0.083333333333333329 * (u[1] * u[1]) + rho * 0.25 * u[0] * u[1];
  pdf.get(10u) = rho * -0.083333333333333329 * u[1] + rho * -0.25 * u[0] * u[1] + rho * 0.027777777777777776 + rho * 0.083333333333333329 * u[0] + rho * 0.083333333333333329 * (u[0] * u[0]) + rho * 0.083333333333333329 * (u[1] * u[1]);
  pdf.get(11u) = rho * 0.027777777777777776 + rho * 0.083333333333333329 * u[1] + rho * 0.083333333333333329 * u[2] + rho * 0.083333333333333329 * (u[1] * u[1]) + rho * 0.083333333333333329 * (u[2] * u[2]) + rho * 0.25 * u[1] * u[2];
  pdf.get(12u) = rho * -0.083333333333333329 * u[1] + rho * -0.25 * u[1] * u[2] + rho * 0.027777777777777776 + rho * 0.083333333333333329 * u[2] + rho * 0.083333333333333329 * (u[1] * u[1]) + rho * 0.083333333333333329 * (u[2] * u[2]);
  pdf.get(13u) = rho * -0.083333333333333329 * u[0] + rho * -0.25 * u[0] * u[2] + rho * 0.027777777777777776 + rho * 0.083333333333333329 * u[2] + rho * 0.083333333333333329 * (u[0] * u[0]) + rho * 0.083333333333333329 * (u[2] * u[2]);
  pdf.get(14u) = rho * 0.027777777777777776 + rho * 0.083333333333333329 * u[0] + rho * 0.083333333333333329 * u[2] + rho * 0.083333333333333329 * (u[0] * u[0]) + rho * 0.083333333333333329 * (u[2] * u[2]) + rho * 0.25 * u[0] * u[2];
  pdf.get(15u) = rho * -0.083333333333333329 * u[2] + rho * -0.25 * u[1] * u[2] + rho * 0.027777777777777776 + rho * 0.083333333333333329 * u[1] + rho * 0.083333333333333329 * (u[1] * u[1]) + rho * 0.083333333333333329 * (u[2] * u[2]);
  pdf.get(16u) = rho * -0.083333333333333329 * u[1] + rho * -0.083333333333333329 * u[2] + rho * 0.027777777777777776 + rho * 0.083333333333333329 * (u[1] * u[1]) + rho * 0.083333333333333329 * (u[2] * u[2]) + rho * 0.25 * u[1] * u[2];
  pdf.get(17u) = rho * -0.083333333333333329 * u[0] + rho * -0.083333333333333329 * u[2] + rho * 0.027777777777777776 + rho * 0.083333333333333329 * (u[0] * u[0]) + rho * 0.083333333333333329 * (u[2] * u[2]) + rho * 0.25 * u[0] * u[2];
  pdf.get(18u) = rho * -0.083333333333333329 * u[2] + rho * -0.25 * u[0] * u[2] + rho * 0.027777777777777776 + rho * 0.083333333333333329 * u[0] + rho * 0.083333333333333329 * (u[0] * u[0]) + rho * 0.083333333333333329 * (u[2] * u[2]);
}
// LCOV_EXCL_STOP
} // namespace Equilibrium

namespace Density {
// LCOV_EXCL_START
__global__ void kernel_get(
    gpu::FieldAccessor<double> pdf,
    double *RESTRICT rho_out) {
  auto const offset = getLinearIndex(blockIdx, threadIdx, gridDim, blockDim, 1u);
  pdf.set(blockIdx, threadIdx);
  rho_out += offset;
  if (pdf.isValidPosition()) {
    double const f_0 = pdf.get(0u);
    double const f_1 = pdf.get(1u);
    double const f_2 = pdf.get(2u);
    double const f_3 = pdf.get(3u);
    double const f_4 = pdf.get(4u);
    double const f_5 = pdf.get(5u);
    double const f_6 = pdf.get(6u);
    double const f_7 = pdf.get(7u);
    double const f_8 = pdf.get(8u);
    double const f_9 = pdf.get(9u);
    double const f_10 = pdf.get(10u);
    double const f_11 = pdf.get(11u);
    double const f_12 = pdf.get(12u);
    double const f_13 = pdf.get(13u);
    double const f_14 = pdf.get(14u);
    double const f_15 = pdf.get(15u);
    double const f_16 = pdf.get(16u);
    double const f_17 = pdf.get(17u);
    double const f_18 = pdf.get(18u);
    const double vel0Term = f_10 + f_14 + f_18 + f_4 + f_8;
    const double vel1Term = f_1 + f_11 + f_15 + f_7;
    const double vel2Term = f_12 + f_13 + f_5;
    const double rho = f_0 + f_16 + f_17 + f_2 + f_3 + f_6 + f_9 + vel0Term + vel1Term + vel2Term;
    rho_out[0u] = rho;
  }
}

__global__ void kernel_set(
    gpu::FieldAccessor<double> pdf,
    double const *RESTRICT rho_in) {
  auto const offset = getLinearIndex(blockIdx, threadIdx, gridDim, blockDim, 1u);
  pdf.set(blockIdx, threadIdx);
  rho_in += offset;
  if (pdf.isValidPosition()) {
    double const f_0 = pdf.get(0u);
    double const f_1 = pdf.get(1u);
    double const f_2 = pdf.get(2u);
    double const f_3 = pdf.get(3u);
    double const f_4 = pdf.get(4u);
    double const f_5 = pdf.get(5u);
    double const f_6 = pdf.get(6u);
    double const f_7 = pdf.get(7u);
    double const f_8 = pdf.get(8u);
    double const f_9 = pdf.get(9u);
    double const f_10 = pdf.get(10u);
    double const f_11 = pdf.get(11u);
    double const f_12 = pdf.get(12u);
    double const f_13 = pdf.get(13u);
    double const f_14 = pdf.get(14u);
    double const f_15 = pdf.get(15u);
    double const f_16 = pdf.get(16u);
    double const f_17 = pdf.get(17u);
    double const f_18 = pdf.get(18u);
    const double vel0Term = f_10 + f_14 + f_18 + f_4 + f_8;
    const double momdensity_0 = -f_13 - f_17 - f_3 - f_7 - f_9 + vel0Term;
    const double vel1Term = f_1 + f_11 + f_15 + f_7;
    const double momdensity_1 = -f_10 - f_12 - f_16 - f_2 + f_8 - f_9 + vel1Term;
    const double vel2Term = f_12 + f_13 + f_5;
    const double momdensity_2 = f_11 + f_14 - f_15 - f_16 - f_17 - f_18 - f_6 + vel2Term;
    const double rho = f_0 + f_16 + f_17 + f_2 + f_3 + f_6 + f_9 + vel0Term + vel1Term + vel2Term;

    // calculate current velocity (before density change)
    double const rho_inv = double{1} / rho;
    double const u_old[3] = {momdensity_0 * rho_inv, momdensity_1 * rho_inv, momdensity_2 * rho_inv};

    Equilibrium::kernel_set_device(pdf, u_old, rho_in[0u]);
  }
}
// LCOV_EXCL_STOP

double get(
    gpu::GPUField<double> const *pdf_field,
    Cell const &cell) {
  CellInterval ci(cell, cell);
  thrust::device_vector<double> dev_data(1u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_get);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addParam(dev_data_ptr);
  kernel();
  double rho = dev_data[0u];
  return rho;
}

std::vector<double> get(
    gpu::GPUField<double> const *pdf_field,
    CellInterval const &ci) {
  thrust::device_vector<double> dev_data(ci.numCells());
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_get);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addParam(dev_data_ptr);
  kernel();
  std::vector<double> out(dev_data.size());
  thrust::copy(dev_data.begin(), dev_data.end(), out.begin());
  return out;
}

void set(
    gpu::GPUField<double> *pdf_field,
    const double rho,
    Cell const &cell) {
  CellInterval ci(cell, cell);
  thrust::device_vector<double> dev_data(1u, rho);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_set);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addParam(const_cast<const double *>(dev_data_ptr));
  kernel();
}

void set(
    gpu::GPUField<double> *pdf_field,
    std::vector<double> const &values,
    CellInterval const &ci) {
  thrust::device_vector<double> dev_data(values.begin(), values.end());
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_set);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addParam(const_cast<const double *>(dev_data_ptr));
  kernel();
}
} // namespace Density

namespace Velocity {
// LCOV_EXCL_START
__global__ void kernel_get(
    gpu::FieldAccessor<double> pdf,
    gpu::FieldAccessor<double> force,
    double *RESTRICT u_out) {
  auto const offset = getLinearIndex(blockIdx, threadIdx, gridDim, blockDim, 3u);
  pdf.set(blockIdx, threadIdx);
  force.set(blockIdx, threadIdx);
  u_out += offset;
  if (pdf.isValidPosition()) {
    double const f_0 = pdf.get(0u);
    double const f_1 = pdf.get(1u);
    double const f_2 = pdf.get(2u);
    double const f_3 = pdf.get(3u);
    double const f_4 = pdf.get(4u);
    double const f_5 = pdf.get(5u);
    double const f_6 = pdf.get(6u);
    double const f_7 = pdf.get(7u);
    double const f_8 = pdf.get(8u);
    double const f_9 = pdf.get(9u);
    double const f_10 = pdf.get(10u);
    double const f_11 = pdf.get(11u);
    double const f_12 = pdf.get(12u);
    double const f_13 = pdf.get(13u);
    double const f_14 = pdf.get(14u);
    double const f_15 = pdf.get(15u);
    double const f_16 = pdf.get(16u);
    double const f_17 = pdf.get(17u);
    double const f_18 = pdf.get(18u);
    const double vel0Term = f_10 + f_14 + f_18 + f_4 + f_8;
    const double momdensity_0 = -f_13 - f_17 - f_3 - f_7 - f_9 + vel0Term;
    const double vel1Term = f_1 + f_11 + f_15 + f_7;
    const double momdensity_1 = -f_10 - f_12 - f_16 - f_2 + f_8 - f_9 + vel1Term;
    const double vel2Term = f_12 + f_13 + f_5;
    const double momdensity_2 = f_11 + f_14 - f_15 - f_16 - f_17 - f_18 - f_6 + vel2Term;
    const double rho = f_0 + f_16 + f_17 + f_2 + f_3 + f_6 + f_9 + vel0Term + vel1Term + vel2Term;
    const double md_0 = force.get(0) * 0.50000000000000000 + momdensity_0;
    const double md_1 = force.get(1) * 0.50000000000000000 + momdensity_1;
    const double md_2 = force.get(2) * 0.50000000000000000 + momdensity_2;
    auto const rho_inv = double{1} / rho;
    u_out[0u] = md_0 * rho_inv;
    u_out[1u] = md_1 * rho_inv;
    u_out[2u] = md_2 * rho_inv;
  }
}

__global__ void kernel_set(
    gpu::FieldAccessor<double> pdf,
    gpu::FieldAccessor<double> velocity,
    gpu::FieldAccessor<double> force,
    double const *RESTRICT u_in) {
  auto const offset = getLinearIndex(blockIdx, threadIdx, gridDim, blockDim, 3u);
  pdf.set(blockIdx, threadIdx);
  velocity.set(blockIdx, threadIdx);
  force.set(blockIdx, threadIdx);
  u_in += offset;
  if (pdf.isValidPosition()) {
    double const f_0 = pdf.get(0u);
    double const f_1 = pdf.get(1u);
    double const f_2 = pdf.get(2u);
    double const f_3 = pdf.get(3u);
    double const f_4 = pdf.get(4u);
    double const f_5 = pdf.get(5u);
    double const f_6 = pdf.get(6u);
    double const f_7 = pdf.get(7u);
    double const f_8 = pdf.get(8u);
    double const f_9 = pdf.get(9u);
    double const f_10 = pdf.get(10u);
    double const f_11 = pdf.get(11u);
    double const f_12 = pdf.get(12u);
    double const f_13 = pdf.get(13u);
    double const f_14 = pdf.get(14u);
    double const f_15 = pdf.get(15u);
    double const f_16 = pdf.get(16u);
    double const f_17 = pdf.get(17u);
    double const f_18 = pdf.get(18u);
    double const *RESTRICT const u = u_in;
    const double vel0Term = f_10 + f_14 + f_18 + f_4 + f_8;
    const double vel1Term = f_1 + f_11 + f_15 + f_7;
    const double vel2Term = f_12 + f_13 + f_5;
    const double rho = f_0 + f_16 + f_17 + f_2 + f_3 + f_6 + f_9 + vel0Term + vel1Term + vel2Term;
    const double u_0 = -force.get(0) * 0.50000000000000000 / rho + u[0];
    const double u_1 = -force.get(1) * 0.50000000000000000 / rho + u[1];
    const double u_2 = -force.get(2) * 0.50000000000000000 / rho + u[2];
    velocity.get(0u) = u_in[0u];
    velocity.get(1u) = u_in[1u];
    velocity.get(2u) = u_in[2u];

    double u_new[3] = {u_0, u_1, u_2};

    Equilibrium::kernel_set_device(pdf, u_new, rho);
  }
}
// LCOV_EXCL_STOP

Vector3<double> get(
    gpu::GPUField<double> const *pdf_field,
    gpu::GPUField<double> const *force_field,
    Cell const &cell) {
  CellInterval ci(cell, cell);
  thrust::device_vector<double> dev_data(3u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_get);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*force_field, ci));
  kernel.addParam(dev_data_ptr);
  kernel();
  Vector3<double> vec;
  thrust::copy(dev_data.begin(), dev_data.end(), vec.data());
  return vec;
}

std::vector<double> get(
    gpu::GPUField<double> const *pdf_field,
    gpu::GPUField<double> const *force_field,
    CellInterval const &ci) {
  thrust::device_vector<double> dev_data(3u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_get);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*force_field, ci));
  kernel.addParam(dev_data_ptr);
  kernel();
  std::vector<double> out(dev_data.size());
  thrust::copy(dev_data.begin(), dev_data.end(), out.data());
  return out;
}

void set(
    gpu::GPUField<double> *pdf_field,
    gpu::GPUField<double> *velocity_field,
    gpu::GPUField<double> const *force_field,
    Vector3<double> const &u,
    Cell const &cell) {
  CellInterval ci(cell, cell);
  thrust::device_vector<double> dev_data(u.data(), u.data() + 3u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_set);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*velocity_field, ci));
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*force_field, ci));
  kernel.addParam(const_cast<const double *>(dev_data_ptr));
  kernel();
}

void set(
    gpu::GPUField<double> *pdf_field,
    gpu::GPUField<double> *velocity_field,
    gpu::GPUField<double> const *force_field,
    std::vector<double> const &values,
    CellInterval const &ci) {
  thrust::device_vector<double> dev_data(values.begin(), values.end());
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_set);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*velocity_field, ci));
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*force_field, ci));
  kernel.addParam(const_cast<const double *>(dev_data_ptr));
  kernel();
}
} // namespace Velocity

namespace Force {
// LCOV_EXCL_START
__global__ void kernel_set(
    gpu::FieldAccessor<double> pdf,
    gpu::FieldAccessor<double> velocity,
    gpu::FieldAccessor<double> force,
    double const *RESTRICT f_in) {
  auto const offset = getLinearIndex(blockIdx, threadIdx, gridDim, blockDim, 3u);
  pdf.set(blockIdx, threadIdx);
  velocity.set(blockIdx, threadIdx);
  force.set(blockIdx, threadIdx);
  f_in += offset;
  if (pdf.isValidPosition()) {
    double const f_0 = pdf.get(0u);
    double const f_1 = pdf.get(1u);
    double const f_2 = pdf.get(2u);
    double const f_3 = pdf.get(3u);
    double const f_4 = pdf.get(4u);
    double const f_5 = pdf.get(5u);
    double const f_6 = pdf.get(6u);
    double const f_7 = pdf.get(7u);
    double const f_8 = pdf.get(8u);
    double const f_9 = pdf.get(9u);
    double const f_10 = pdf.get(10u);
    double const f_11 = pdf.get(11u);
    double const f_12 = pdf.get(12u);
    double const f_13 = pdf.get(13u);
    double const f_14 = pdf.get(14u);
    double const f_15 = pdf.get(15u);
    double const f_16 = pdf.get(16u);
    double const f_17 = pdf.get(17u);
    double const f_18 = pdf.get(18u);
    const double vel0Term = f_10 + f_14 + f_18 + f_4 + f_8;
    const double momdensity_0 = -f_13 - f_17 - f_3 - f_7 - f_9 + vel0Term;
    const double vel1Term = f_1 + f_11 + f_15 + f_7;
    const double momdensity_1 = -f_10 - f_12 - f_16 - f_2 + f_8 - f_9 + vel1Term;
    const double vel2Term = f_12 + f_13 + f_5;
    const double momdensity_2 = f_11 + f_14 - f_15 - f_16 - f_17 - f_18 - f_6 + vel2Term;
    const double rho = f_0 + f_16 + f_17 + f_2 + f_3 + f_6 + f_9 + vel0Term + vel1Term + vel2Term;
    const double md_0 = f_in[0u] * 0.50000000000000000 + momdensity_0;
    const double md_1 = f_in[1u] * 0.50000000000000000 + momdensity_1;
    const double md_2 = f_in[2u] * 0.50000000000000000 + momdensity_2;
    auto const rho_inv = double{1} / rho;

    force.get(0u) = f_in[0u];
    force.get(1u) = f_in[1u];
    force.get(2u) = f_in[2u];

    velocity.get(0u) = md_0 * rho_inv;
    velocity.get(1u) = md_1 * rho_inv;
    velocity.get(2u) = md_2 * rho_inv;
  }
}
// LCOV_EXCL_STOP

void set(gpu::GPUField<double> const *pdf_field,
         gpu::GPUField<double> *velocity_field,
         gpu::GPUField<double> *force_field,
         Vector3<double> const &u,
         Cell const &cell) {
  CellInterval ci(cell, cell);
  thrust::device_vector<double> dev_data(u.data(), u.data() + 3u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_set);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*velocity_field, ci));
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*force_field, ci));
  kernel.addParam(const_cast<const double *>(dev_data_ptr));
  kernel();
}

void set(gpu::GPUField<double> const *pdf_field,
         gpu::GPUField<double> *velocity_field,
         gpu::GPUField<double> *force_field,
         std::vector<double> const &values,
         CellInterval const &ci) {
  thrust::device_vector<double> dev_data(values.begin(), values.end());
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_set);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*velocity_field, ci));
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*force_field, ci));
  kernel.addParam(const_cast<const double *>(dev_data_ptr));
  kernel();
}
} // namespace Force

namespace MomentumDensity {
// LCOV_EXCL_START
__global__ void kernel_sum(
    gpu::FieldAccessor<double> pdf,
    gpu::FieldAccessor<double> force,
    double *RESTRICT out) {
  auto const offset = getLinearIndex(blockIdx, threadIdx, gridDim, blockDim, 3u);
  pdf.set(blockIdx, threadIdx);
  force.set(blockIdx, threadIdx);
  out += offset;
  if (pdf.isValidPosition()) {
    double const f_0 = pdf.get(0u);
    double const f_1 = pdf.get(1u);
    double const f_2 = pdf.get(2u);
    double const f_3 = pdf.get(3u);
    double const f_4 = pdf.get(4u);
    double const f_5 = pdf.get(5u);
    double const f_6 = pdf.get(6u);
    double const f_7 = pdf.get(7u);
    double const f_8 = pdf.get(8u);
    double const f_9 = pdf.get(9u);
    double const f_10 = pdf.get(10u);
    double const f_11 = pdf.get(11u);
    double const f_12 = pdf.get(12u);
    double const f_13 = pdf.get(13u);
    double const f_14 = pdf.get(14u);
    double const f_15 = pdf.get(15u);
    double const f_16 = pdf.get(16u);
    double const f_17 = pdf.get(17u);
    double const f_18 = pdf.get(18u);
    const double vel0Term = f_10 + f_14 + f_18 + f_4 + f_8;
    const double momdensity_0 = -f_13 - f_17 - f_3 - f_7 - f_9 + vel0Term;
    const double vel1Term = f_1 + f_11 + f_15 + f_7;
    const double momdensity_1 = -f_10 - f_12 - f_16 - f_2 + f_8 - f_9 + vel1Term;
    const double vel2Term = f_12 + f_13 + f_5;
    const double momdensity_2 = f_11 + f_14 - f_15 - f_16 - f_17 - f_18 - f_6 + vel2Term;
    const double rho = f_0 + f_16 + f_17 + f_2 + f_3 + f_6 + f_9 + vel0Term + vel1Term + vel2Term;
    const double md_0 = force.get(0) * 0.50000000000000000 + momdensity_0;
    const double md_1 = force.get(1) * 0.50000000000000000 + momdensity_1;
    const double md_2 = force.get(2) * 0.50000000000000000 + momdensity_2;
    out[0u] += md_0;
    out[1u] += md_1;
    out[2u] += md_2;
  }
}
// LCOV_EXCL_STOP

Vector3<double> reduce(
    gpu::GPUField<double> const *pdf_field,
    gpu::GPUField<double> const *force_field) {
  thrust::device_vector<double> dev_data(3u, double{0});
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  WALBERLA_FOR_ALL_CELLS_XYZ(pdf_field, {
    Cell cell(x, y, z);
    CellInterval ci(cell, cell);
    auto kernel = gpu::make_kernel(kernel_sum);
    kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
    kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*force_field, ci));
    kernel.addParam(dev_data_ptr);
    kernel();
  });
  Vector3<double> mom(double{0});
  thrust::copy(dev_data.begin(), dev_data.begin() + 3u, mom.data());
  return mom;
}
} // namespace MomentumDensity

namespace PressureTensor {
// LCOV_EXCL_START
__global__ void kernel_get(
    gpu::FieldAccessor<double> pdf,
    double *RESTRICT p_out) {
  auto const offset = getLinearIndex(blockIdx, threadIdx, gridDim, blockDim, 9u);
  pdf.set(blockIdx, threadIdx);
  p_out += offset;
  if (pdf.isValidPosition()) {
    double const f_0 = pdf.get(0u);
    double const f_1 = pdf.get(1u);
    double const f_2 = pdf.get(2u);
    double const f_3 = pdf.get(3u);
    double const f_4 = pdf.get(4u);
    double const f_5 = pdf.get(5u);
    double const f_6 = pdf.get(6u);
    double const f_7 = pdf.get(7u);
    double const f_8 = pdf.get(8u);
    double const f_9 = pdf.get(9u);
    double const f_10 = pdf.get(10u);
    double const f_11 = pdf.get(11u);
    double const f_12 = pdf.get(12u);
    double const f_13 = pdf.get(13u);
    double const f_14 = pdf.get(14u);
    double const f_15 = pdf.get(15u);
    double const f_16 = pdf.get(16u);
    double const f_17 = pdf.get(17u);
    double const f_18 = pdf.get(18u);
    const double p_0 = f_10 + f_13 + f_14 + f_17 + f_18 + f_3 + f_4 + f_7 + f_8 + f_9;
    const double p_1 = -f_10 - f_7 + f_8 + f_9;
    const double p_2 = -f_13 + f_14 + f_17 - f_18;
    const double p_3 = -f_10 - f_7 + f_8 + f_9;
    const double p_4 = f_1 + f_10 + f_11 + f_12 + f_15 + f_16 + f_2 + f_7 + f_8 + f_9;
    const double p_5 = f_11 - f_12 - f_15 + f_16;
    const double p_6 = -f_13 + f_14 + f_17 - f_18;
    const double p_7 = f_11 - f_12 - f_15 + f_16;
    const double p_8 = f_11 + f_12 + f_13 + f_14 + f_15 + f_16 + f_17 + f_18 + f_5 + f_6;
    p_out[0u] = p_0;
    p_out[1u] = p_1;
    p_out[2u] = p_2;
    p_out[3u] = p_3;
    p_out[4u] = p_4;
    p_out[5u] = p_5;
    p_out[6u] = p_6;
    p_out[7u] = p_7;
    p_out[8u] = p_8;
  }
}
// LCOV_EXCL_STOP

Matrix3<double> get(
    gpu::GPUField<double> const *pdf_field,
    Cell const &cell) {
  CellInterval ci(cell, cell);
  thrust::device_vector<double> dev_data(9u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_get);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addParam(dev_data_ptr);
  kernel();
  Matrix3<double> out;
  thrust::copy(dev_data.begin(), dev_data.end(), out.data());
  return out;
}

std::vector<double> get(
    gpu::GPUField<double> const *pdf_field,
    CellInterval const &ci) {
  thrust::device_vector<double> dev_data(9u * ci.numCells());
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_get);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addParam(dev_data_ptr);
  kernel();
  std::vector<double> out(dev_data.size());
  thrust::copy(dev_data.begin(), dev_data.end(), out.data());
  return out;
}
} // namespace PressureTensor

} // namespace accessor
} // namespace lbm
} // namespace walberla
