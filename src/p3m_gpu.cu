#include "hip/hip_runtime.h"
/* 
   Copyright (C) 2010,2011,2012,2013 The ESPResSo project

   This file is part of ESPResSo.
  
   ESPResSo is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.
   
   ESPResSo is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.
   
   You should have received a copy of the GNU General Public License
   along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

/** \file lbgpu.cu
 *
 * Cuda (.cu) file for the Lattice Boltzmann implementation on GPUs.
 * Header file for \ref lbgpu.h.
 */ 

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <hipfft/hipfft.h>
#include "cuda_common.h"
#include "config.h"

#ifdef __cplusplus
extern "C" {
#endif
#include "p3m_gpu.h"
#include "utils.h"
#ifdef __cplusplus
}
#endif

#ifdef ELECTROSTATICS

struct dummytypename {
  CUFFT_TYPE_COMPLEX *charge_mesh;
  CUFFT_TYPE_COMPLEX *force_mesh;
  REAL_TYPE *G_hat, *G_hat_host;
  hipfftHandle fft_plan;
  int cao, mesh;
  REAL_TYPE alpha;
  int npart;
  REAL_TYPE box;
} p3m_gpu_data;


#define SQR(A) ((A)*(A))

void static Aliasing_sums_ik ( int cao, REAL_TYPE box, REAL_TYPE alpha, int mesh, int NX, int NY, int NZ,
                        REAL_TYPE *Zaehler, REAL_TYPE *Nenner ) {
    REAL_TYPE S1,S2,S3;
    REAL_TYPE fak1,fak2,zwi;
    int    MX,MY,MZ;
    REAL_TYPE NMX,NMY,NMZ;
    REAL_TYPE NM2;
    REAL_TYPE expo, TE;
    REAL_TYPE Leni = 1.0/box;

    fak1 = 1.0/ ( REAL_TYPE ) mesh;
    fak2 = SQR ( PI/ ( alpha ) );

    Zaehler[0] = Zaehler[1] = Zaehler[2] = *Nenner = 0.0;

    for ( MX = -P3M_BRILLOUIN; MX <= P3M_BRILLOUIN; MX++ ) {
      NMX = ( ( NX > mesh/2 ) ? NX - mesh : NX ) + mesh*MX;
      S1 = pow ( sinc(fak1*NMX ), 2*cao );
      for ( MY = -P3M_BRILLOUIN; MY <= P3M_BRILLOUIN; MY++ ) {
	NMY = ( ( NY > mesh/2 ) ? NY - mesh : NY ) + mesh*MY;
	S2   = S1*pow ( sinc (fak1*NMY ), 2*cao );
	for ( MZ = -P3M_BRILLOUIN; MZ <= P3M_BRILLOUIN; MZ++ ) {
	  NMZ = ( ( NZ > mesh/2 ) ? NZ - mesh : NZ ) + mesh*MZ;
	  S3   = S2*pow ( sinc( fak1*NMZ ), 2*cao );

	  NM2 = SQR ( NMX*Leni ) + SQR ( NMY*Leni ) + SQR ( NMZ*Leni );
	  *Nenner += S3;

	  expo = fak2*NM2;
	  TE = exp ( -expo );
	  zwi  = S3 * TE/NM2;
	  Zaehler[0] += NMX*zwi*Leni;
	  Zaehler[1] += NMY*zwi*Leni;
	  Zaehler[2] += NMZ*zwi*Leni;
	}
      }
    }
}

/* Calculate influence function */
void static calculate_influence_function ( int cao, int mesh, REAL_TYPE box, REAL_TYPE alpha, REAL_TYPE *G_hat ) {

  int    NX,NY,NZ;
  REAL_TYPE Dnx,Dny,Dnz;
  REAL_TYPE Zaehler[3]={0.0,0.0,0.0},Nenner=0.0;
  REAL_TYPE zwi;
  int ind = 0;
  REAL_TYPE Leni = 1.0/box;

  for ( NX=0; NX<mesh; NX++ ) {
    for ( NY=0; NY<mesh; NY++ ) {
      for ( NZ=0; NZ<mesh; NZ++ ) {
	ind = NX*mesh*mesh + NY * mesh + NZ;
	  
	if ( ( NX==0 ) && ( NY==0 ) && ( NZ==0 ) )
	  G_hat[ind]=0.0;
	else if ( ( NX% ( mesh/2 ) == 0 ) && ( NY% ( mesh/2 ) == 0 ) && ( NZ% ( mesh/2 ) == 0 ) )
	  G_hat[ind]=0.0;
	else {
	  Aliasing_sums_ik ( cao, box, alpha, mesh, NX, NY, NZ, Zaehler, &Nenner );
		  
	  Dnx = ( NX > mesh/2 ) ? NX - mesh : NX;
	  Dny = ( NY > mesh/2 ) ? NY - mesh : NY;
	  Dnz = ( NZ > mesh/2 ) ? NZ - mesh : NZ;
	    
	  zwi  = Dnx*Zaehler[0]*Leni + Dny*Zaehler[1]*Leni + Dnz*Zaehler[2]*Leni;
	  zwi /= ( ( SQR ( Dnx*Leni ) + SQR ( Dny*Leni ) + SQR ( Dnz*Leni ) ) * SQR ( Nenner ) );
	  G_hat[ind] = 2.0 * zwi / PI;
	}
      }
    }
  }
}

//NOTE :if one wants to use the function below it requires cuda compute capability 1.3
#ifdef _P3M_GPU_REAL_DOUBLE
__device__ double atomicAdd (double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}
#endif

/** atomic add function for several cuda architectures 
*/

#if !defined __CUDA_ARCH__ || __CUDA_ARCH__ >= 200 // for Fermi, atomicAdd supports floats
//atomicAdd supports floats already, do nothing
#elif __CUDA_ARCH__ >= 110
#warning Using slower atomicAdd emulation
__device__ inline void atomicAdd(float* address, float value){
  // float-atomic-add from 
// [url="http://forums.nvidia.com/index.php?showtopic=158039&view=findpost&p=991561"]
  float old = value;
  while ((old = atomicExch(address, atomicExch(address, 0.0f)+old))!=0.0f);
}
#else
#error I need at least compute capability 1.1
#endif



__device__ unsigned int getThreadIndexP3M() { //rename is dumb but can't import same fnc from cuda_common

  return blockIdx.y * gridDim.x * blockDim.x +
         blockDim.x * blockIdx.x +
         threadIdx.x;
}


// __global__ void add_p3m_farfield_force_gpu( LB_parameters_gpu* lb_parameters_gpu,
//                                             CUDA_particle_data* lb_particle_gpu,
//                                             CUDA_particle_force* lb_particle_force_gpu
//                                           ) {

//   unsigned int index = getThreadIndex();

//   if( index < lb_parameters_gpu->number_of_particles ) {
    
//     lb_particle_force_gpu[ index ].f[0] = 1.0f;
//     lb_particle_force_gpu[ index ].f[1] = 2.0f;
//     lb_particle_force_gpu[ index ].f[2] = 3.0f;
//   }
// }


template<int dim>
__global__ void apply_diff_op( CUFFT_TYPE_COMPLEX *mesh, const int mesh_size, CUFFT_TYPE_COMPLEX *force_mesh,  const REAL_TYPE box ) {
  int linear_index = mesh_size*mesh_size*blockIdx.x + mesh_size * blockIdx.y + threadIdx.x;
  int n;

  switch( dim ) {
  case 0:
    n = blockIdx.x;
    break;
  case 1:
    n = blockIdx.y;
    break;
  case 2:
    n = threadIdx.x;
    break;
  }

  n = ( n == mesh_size/2 ) ? 0.0 : n;
  n = ( n > mesh_size/2) ? n - mesh_size : n;
 
  force_mesh[linear_index].x =  -2.0 * PI * n * mesh[linear_index].y / box;
  force_mesh[linear_index].y =   2.0 * PI * n * mesh[linear_index].x / box;
}


__device__ inline int wrap_index(const int ind, const int mesh) {
  if(ind < 0)
    return ind + mesh;
  else if(ind >= mesh)
    return ind - mesh;
  else 
    return ind;	   
}

__device__ REAL_TYPE caf(int i, REAL_TYPE x, int cao_value) {
  switch (cao_value) {
  case 1 : return 1.0;
  case 2 : {
    switch (i) {
    case 0: return 0.5-x;
    case 1: return 0.5+x;
    default:
      return 0.0;
    }
  } 
  case 3 : { 
    switch (i) {
    case 0: return 0.5*SQR(0.5 - x);
    case 1: return 0.75 - SQR(x);
    case 2: return 0.5*SQR(0.5 + x);
    default:
      return 0.0;
    }
  case 4 : { 
    switch (i) {
    case 0: return ( 1.0+x*( -6.0+x*( 12.0-x* 8.0)))/48.0;
    case 1: return (23.0+x*(-30.0+x*(-12.0+x*24.0)))/48.0;
    case 2: return (23.0+x*( 30.0+x*(-12.0-x*24.0)))/48.0;
    case 3: return ( 1.0+x*(  6.0+x*( 12.0+x* 8.0)))/48.0;
    default:
      return 0.0;
    }
  }
  case 5 : {
    switch (i) {
    case 0: return (  1.0+x*( -8.0+x*(  24.0+x*(-32.0+x*16.0))))/384.0;
    case 1: return ( 19.0+x*(-44.0+x*(  24.0+x*( 16.0-x*16.0))))/ 96.0;
    case 2: return (115.0+x*       x*(-120.0+x*       x*48.0))  /192.0;
    case 3: return ( 19.0+x*( 44.0+x*(  24.0+x*(-16.0-x*16.0))))/ 96.0;
    case 4: return (  1.0+x*(  8.0+x*(  24.0+x*( 32.0+x*16.0))))/384.0;
    default:
      return 0.0;
    }
  }
  case 6 : {
    switch (i) {
    case 0: return (  1.0+x*( -10.0+x*(  40.0+x*( -80.0+x*(  80.0-x* 32.0)))))/3840.0;
    case 1: return (237.0+x*(-750.0+x*( 840.0+x*(-240.0+x*(-240.0+x*160.0)))))/3840.0;
    case 2: return (841.0+x*(-770.0+x*(-440.0+x*( 560.0+x*(  80.0-x*160.0)))))/1920.0;
    case 3: return (841.0+x*(+770.0+x*(-440.0+x*(-560.0+x*(  80.0+x*160.0)))))/1920.0;
    case 4: return (237.0+x*( 750.0+x*( 840.0+x*( 240.0+x*(-240.0-x*160.0)))))/3840.0;
    case 5: return (  1.0+x*(  10.0+x*(  40.0+x*(  80.0+x*(  80.0+x* 32.0)))))/3840.0;
    default:
      return 0.0;
    }
  }
  case 7 : {
    switch (i) {
    case 0: return (    1.0+x*(   -12.0+x*(   60.0+x*( -160.0+x*(  240.0+x*(-192.0+x* 64.0))))))/46080.0;
    case 1: return (  361.0+x*( -1416.0+x*( 2220.0+x*(-1600.0+x*(  240.0+x*( 384.0-x*192.0))))))/23040.0;
    case 2: return (10543.0+x*(-17340.0+x*( 4740.0+x*( 6880.0+x*(-4080.0+x*(-960.0+x*960.0))))))/46080.0;
    case 3: return ( 5887.0+x*          x*(-4620.0+x*         x*( 1680.0-x*        x*320.0)))   /11520.0;
    case 4: return (10543.0+x*( 17340.0+x*( 4740.0+x*(-6880.0+x*(-4080.0+x*( 960.0+x*960.0))))))/46080.0;
    case 5: return (  361.0+x*(  1416.0+x*( 2220.0+x*( 1600.0+x*(  240.0+x*(-384.0-x*192.0))))))/23040.0;
    case 6: return (    1.0+x*(    12.0+x*(   60.0+x*(  160.0+x*(  240.0+x*( 192.0+x* 64.0))))))/46080.0;
    default:
      return 0.0;
    }
  }
  }}
  return 0.0;
}

__global__ void apply_influence_function( CUFFT_TYPE_COMPLEX *mesh, int mesh_size, REAL_TYPE *G_hat ) {
  int linear_index = mesh_size*mesh_size*blockIdx.x + mesh_size * blockIdx.y + threadIdx.x;
  mesh[linear_index].x *= G_hat[linear_index];
  mesh[linear_index].y *= G_hat[linear_index];
}

__global__ void assign_charges(const CUDA_particle_data * const pdata,
CUFFT_TYPE_COMPLEX *mesh, const int m_size, const int cao, const REAL_TYPE pos_shift, const
REAL_TYPE hi) {
      /** id of the particle **/
      int id = blockIdx.x;
      /** position relative to the closest gird point **/
      REAL_TYPE m_pos[3];
      /** index of the nearest mesh point **/
      int nmp_x, nmp_y, nmp_z;      
      
      CUDA_particle_data p = pdata[id];

      m_pos[0] = p.p[0] * hi - pos_shift;
      m_pos[1] = p.p[1] * hi - pos_shift;
      m_pos[2] = p.p[2] * hi - pos_shift;

      nmp_x = (int) floor(m_pos[0] + 0.5);
      nmp_y = (int) floor(m_pos[1] + 0.5);
      nmp_z = (int) floor(m_pos[2] + 0.5);

      m_pos[0] -= nmp_x;
      m_pos[1] -= nmp_y;
      m_pos[2] -= nmp_z;

      nmp_x = wrap_index(nmp_x + threadIdx.x, m_size);
      nmp_y = wrap_index(nmp_y + threadIdx.y, m_size);
      nmp_z = wrap_index(nmp_z + threadIdx.z, m_size);

      atomicAdd( &(mesh[m_size*m_size*nmp_x +  m_size*nmp_y + nmp_z].x), caf(threadIdx.x, m_pos[0], cao)*caf(threadIdx.y, m_pos[1], cao)*caf(threadIdx.z, m_pos[2], cao)*p.q);
}

__global__ void assign_forces(const CUDA_particle_data * const pdata, CUFFT_TYPE_COMPLEX *mesh, const int m_size, const int cao, const REAL_TYPE pos_shift, const
			      REAL_TYPE hi, CUDA_particle_force * lb_particle_force_gpu, REAL_TYPE prefactor, int dim) {
      /** id of the particle **/
      int id = blockIdx.x;
      /** position relative to the closest gird point **/
      REAL_TYPE m_pos[3];
      /** index of the nearest mesh point **/
      int nmp_x, nmp_y, nmp_z;      

      CUDA_particle_data p = pdata[id];

      m_pos[0] = p.p[0] * hi - pos_shift;
      m_pos[1] = p.p[1] * hi - pos_shift;
      m_pos[2] = p.p[2] * hi - pos_shift;

      nmp_x = (int) floor(m_pos[0] + 0.5);
      nmp_y = (int) floor(m_pos[1] + 0.5);
      nmp_z = (int) floor(m_pos[2] + 0.5);

      m_pos[0] -= nmp_x;
      m_pos[1] -= nmp_y;
      m_pos[2] -= nmp_z;

      nmp_x = wrap_index(nmp_x + threadIdx.x, m_size);
      nmp_y = wrap_index(nmp_y + threadIdx.y, m_size);
      nmp_z = wrap_index(nmp_z + threadIdx.z, m_size);

      atomicAdd( &(lb_particle_force_gpu[id].f[dim]), (float)(-prefactor*mesh[m_size*m_size*nmp_x +  m_size*nmp_y + nmp_z].x*caf(threadIdx.x, m_pos[0], cao)*caf(threadIdx.y, m_pos[1], cao)*caf(threadIdx.z, m_pos[2], cao)*p.q));
      
}

extern "C" {

  /* Init the internal datastructures of the P3M GPU.
   * Mainly allocation on the device and influence function calculation.
   * Be advised: this needs mesh^3*5*sizeof(REAL_TYPE) of device memory. 
   */

  void p3m_gpu_init(int cao, int mesh, REAL_TYPE alpha, REAL_TYPE box) {
    gpu_init_particle_comm();
    
 
    if ( this_node == 0 ) {
      p3m_gpu_data.npart = gpu_get_global_particle_vars_pointer_host()->number_of_particles;
      p3m_gpu_data.alpha = alpha;
      p3m_gpu_data.cao = cao;
      p3m_gpu_data.mesh = mesh;
      p3m_gpu_data.box = box;
      int mesh3 = mesh*mesh*mesh;

      hipMalloc((void **)&(p3m_gpu_data.charge_mesh), mesh3*sizeof(CUFFT_TYPE_COMPLEX));
      hipMalloc((void **)&(p3m_gpu_data.force_mesh), mesh3*sizeof(CUFFT_TYPE_COMPLEX));
      hipMalloc((void **)&(p3m_gpu_data.G_hat), mesh3*sizeof(REAL_TYPE));

      p3m_gpu_data.G_hat_host = (REAL_TYPE *)malloc(mesh3*sizeof(REAL_TYPE));

      // Calculate influence function of host.
      calculate_influence_function( cao, mesh, box, alpha, p3m_gpu_data.G_hat_host);

      // Copy influence function to device.
      hipMemcpy( p3m_gpu_data.G_hat, p3m_gpu_data.G_hat_host, mesh3*sizeof(REAL_TYPE), hipMemcpyHostToDevice);

      hipfftPlan3d(&(p3m_gpu_data.fft_plan), mesh, mesh, mesh, CUFFT_PLAN_FLAG);
    }
  }

void p3m_gpu_add_farfield_force() {

  CUDA_particle_data* lb_particle_gpu;
  CUDA_particle_force* lb_particle_force_gpu;
  
  int mesh = p3m_gpu_data.mesh;
  int mesh3 = mesh*mesh*mesh;
  int cao = p3m_gpu_data.cao;
  REAL_TYPE box = p3m_gpu_data.box;

  lb_particle_gpu = gpu_get_particle_pointer();
  lb_particle_force_gpu = gpu_get_particle_force_pointer();

  p3m_gpu_data.npart = gpu_get_global_particle_vars_pointer_host()->number_of_particles;

  if(p3m_gpu_data.npart == 0)
    return;

  //printf("p3m params: mesh %d npart %d cao %d\n", mesh, p3m_gpu_data.npart, cao); //TODO delete

  dim3 gridAssignment(p3m_gpu_data.npart,1,1);
  dim3 threadsAssignment(cao,cao,cao);
  
  dim3 gridConv(mesh,mesh,1);
  dim3 threadsConv(mesh,1,1);

  REAL_TYPE pos_shift = (REAL_TYPE)((cao-1)/2);
  REAL_TYPE hi = mesh/box;
  REAL_TYPE prefactor = 1.0/(box*box*box*2.0);

  hipMemset( p3m_gpu_data.charge_mesh, 0, mesh3*sizeof(CUFFT_TYPE_COMPLEX));

  KERNELCALL(assign_charges, gridAssignment, threadsAssignment, (lb_particle_gpu,p3m_gpu_data.charge_mesh,mesh,cao,pos_shift,hi));

  hipDeviceSynchronize();

  if (CUFFT_FFT(p3m_gpu_data.fft_plan, p3m_gpu_data.charge_mesh, p3m_gpu_data.charge_mesh, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: ExecZ2Z Forward failed\n");
    return;
  }

  KERNELCALL( apply_influence_function, gridConv, threadsConv, (p3m_gpu_data.charge_mesh, mesh, p3m_gpu_data.G_hat));

  KERNELCALL(apply_diff_op<0>, gridConv, threadsConv, (p3m_gpu_data.charge_mesh, mesh, p3m_gpu_data.force_mesh, box));
  
  CUFFT_FFT(p3m_gpu_data.fft_plan, p3m_gpu_data.force_mesh, p3m_gpu_data.force_mesh, HIPFFT_BACKWARD);

  KERNELCALL(assign_forces, gridAssignment, threadsAssignment, (lb_particle_gpu, p3m_gpu_data.force_mesh, mesh, cao, pos_shift, hi, lb_particle_force_gpu, prefactor, 0));

  KERNELCALL(apply_diff_op<1>, gridConv, threadsConv, (p3m_gpu_data.charge_mesh, mesh, p3m_gpu_data.force_mesh, box));

  CUFFT_FFT(p3m_gpu_data.fft_plan, p3m_gpu_data.force_mesh, p3m_gpu_data.force_mesh, HIPFFT_BACKWARD);
  
  KERNELCALL(assign_forces, gridAssignment, threadsAssignment, (lb_particle_gpu, p3m_gpu_data.force_mesh, mesh, cao, pos_shift, hi, lb_particle_force_gpu, prefactor, 1));

  KERNELCALL(apply_diff_op<2>, gridConv, threadsConv, (p3m_gpu_data.charge_mesh, mesh, p3m_gpu_data.force_mesh, box));

  CUFFT_FFT(p3m_gpu_data.fft_plan, p3m_gpu_data.force_mesh, p3m_gpu_data.force_mesh, HIPFFT_BACKWARD);
  
  KERNELCALL(assign_forces, gridAssignment, threadsAssignment, (lb_particle_gpu, p3m_gpu_data.force_mesh, mesh, cao, pos_shift, hi, lb_particle_force_gpu, prefactor, 2));


  // KERNELCALL( add_p3m_farfield_force_gpu, dim_grid, threads_per_block, ( lb_parameters_gpu, lb_particle_gpu, lb_particle_force_gpu ) );
}

}

#endif /* ELECTROSTATICS */
