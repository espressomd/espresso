#include "hip/hip_runtime.h"
/* 
   Copyright (C) 2010,2011,2012,2013 The ESPResSo project

   This file is part of ESPResSo.
  
   ESPResSo is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.
   
   ESPResSo is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.
   
   You should have received a copy of the GNU General Public License
   along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/


#include "cuda_common.h" //I can't go in extern C


#include "grid.h"
extern "C" {

#include "config.h"
#include "random.h"
#include "particle_data.h"
#include "interaction_data.h"


  
  static int max_ran = 1000000;
  static CUDA_global_part_vars global_part_vars_host = {0,0,0};
  static __device__ __constant__ CUDA_global_part_vars global_part_vars_device;
  
  /** struct for particle force */
  static CUDA_particle_force *particle_forces_device = NULL;
  /** struct for particle position and veloctiy */
  static CUDA_particle_data *particle_data_device = NULL;
  /** struct for storing particle rn seed */
  static CUDA_particle_seed *particle_seeds_device = NULL;

  CUDA_particle_data *particle_data_host = NULL;
  CUDA_particle_force *particle_forces_host = NULL;

  /**cuda streams for parallel computing on cpu and gpu */
  hipStream_t stream[1];

  hipError_t err;
  hipError_t _err;
  
}

void _cuda_safe_mem(hipError_t err, char *file, unsigned int line){
  if( hipSuccess != err) {                                             
    fprintf(stderr, "Cuda Memory error at %s:%u.\n", file, line);
    printf("CUDA error: %s\n", hipGetErrorString(err));
    if ( err == hipErrorInvalidValue )
      fprintf(stderr, "You may have tried to allocate zero memory at %s:%u.\n", file, line);
    exit(EXIT_FAILURE);
  } else {
    _err=hipGetLastError();
    if (_err != hipSuccess) {
      fprintf(stderr, "Error found during memory operation. Possibly however from an failed operation before. %s:%u.\n", file, line);
      printf("CUDA error: %s\n", hipGetErrorString(err));
      if ( _err == hipErrorInvalidValue )
	fprintf(stderr, "You may have tried to allocate zero memory before %s:%u.\n", file, line);
      exit(EXIT_FAILURE);
    }
  }
}


__device__ unsigned int getThreadIndex() {

  return blockIdx.y * gridDim.x * blockDim.x +
         blockDim.x * blockIdx.x +
         threadIdx.x;
}



/** kernel for the initalisation of the particle force array
 * @param *particle_forces_device	    Pointer to local particle force (Output)
 * @param *particle_seeds_device			Pointer to the particle rn seed storearray (Output)
*/
__global__ void init_particle_force(CUDA_particle_force *particle_forces_device, CUDA_particle_seed *particle_seeds_device){

  unsigned int part_index = getThreadIndex();

  if(part_index<global_part_vars_device.number_of_particles){
    particle_forces_device[part_index].f[0] = 0.0f;
    particle_forces_device[part_index].f[1] = 0.0f;
    particle_forces_device[part_index].f[2] = 0.0f;

    particle_seeds_device[part_index].seed = global_part_vars_device.seed + part_index;
  }

}


/** kernel for the initalisation of the partikel force array
 * @param *particle_forces_device	pointer to local particle force (Input)
*/
__global__ void reset_particle_force(CUDA_particle_force *particle_forces_device){
	
  unsigned int part_index = getThreadIndex();
	
  if(part_index<global_part_vars_device.number_of_particles){
    particle_forces_device[part_index].f[0] = 0.0f;
    particle_forces_device[part_index].f[1] = 0.0f;
    particle_forces_device[part_index].f[2] = 0.0f;
  }			
}


extern "C" {


  /** change number of particles to be communicated to the GPU
   *  Note that in addition to calling this function the parameters must be broadcast with either:
   * 1) cuda_bcast_global_part_params(); (when just being executed on the master node) or
   * 2) MPI_Bcast(gpu_get_global_particle_vars_pointer_host(), sizeof(CUDA_global_part_vars), MPI_BYTE, 0, comm_cart); (when executed on all nodes)
   */
  void gpu_change_number_of_part_to_comm() {
    //we only run the function if there are new particles which have been created since the last call of this function

    if ( global_part_vars_host.number_of_particles != n_total_particles && global_part_vars_host.communication_enabled == 1 && this_node == 0) {
      
      global_part_vars_host.seed = (unsigned int)i_random(max_ran);
      global_part_vars_host.number_of_particles = n_total_particles;

      cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(global_part_vars_device), &global_part_vars_host, sizeof(CUDA_global_part_vars)));

      if ( particle_forces_host )    hipHostFree(particle_forces_host); //if the arrays exists free them to prevent memory leaks
      if ( particle_data_host )      hipHostFree(particle_data_host);
      if ( particle_forces_device )  hipFree(particle_forces_device);
      if ( particle_data_device )    hipFree(particle_data_device);
      if ( particle_seeds_device )   hipFree(particle_seeds_device);


      if ( global_part_vars_host.number_of_particles ) {

    #if !defined __CUDA_ARCH__ || __CUDA_ARCH__ >= 200
        /**pinned memory mode - use special function to get OS-pinned memory*/
        hipHostAlloc((void**)&particle_data_host, global_part_vars_host.number_of_particles * sizeof(CUDA_particle_data), hipHostMallocWriteCombined);
        hipHostAlloc((void**)&particle_forces_host, global_part_vars_host.number_of_particles * sizeof(CUDA_particle_force), hipHostMallocWriteCombined);
    #else
        hipHostMalloc((void**)&particle_data_host, global_part_vars_host.number_of_particles * sizeof(CUDA_particle_data));
        hipHostMalloc((void**)&particle_forces_host, global_part_vars_host.number_of_particles * sizeof(CUDA_particle_force));
    #endif

        cuda_safe_mem(hipMalloc((void**)&particle_forces_device, global_part_vars_host.number_of_particles * sizeof(CUDA_particle_force)));
        cuda_safe_mem(hipMalloc((void**)&particle_data_device, global_part_vars_host.number_of_particles * sizeof(CUDA_particle_data)));
        cuda_safe_mem(hipMalloc((void**)&particle_seeds_device, global_part_vars_host.number_of_particles * sizeof(CUDA_particle_seed)));
        
        /** values for the particle kernel */
        int threads_per_block_particles = 64;
        int blocks_per_grid_particles_y = 4;
        int blocks_per_grid_particles_x = (global_part_vars_host.number_of_particles + threads_per_block_particles * blocks_per_grid_particles_y - 1)/(threads_per_block_particles * blocks_per_grid_particles_y);
        dim3 dim_grid_particles = make_uint3(blocks_per_grid_particles_x, blocks_per_grid_particles_y, 1);

        KERNELCALL(init_particle_force, dim_grid_particles, threads_per_block_particles, (particle_forces_device, particle_seeds_device));
      }

    }

  }

  /** setup and call particle reallocation from the host
   *  Note that in addition to calling this function the parameters must be broadcast with either:
   * 1) cuda_bcast_global_part_params(); (when just being executed on the master node) or
   * 2) MPI_Bcast(gpu_get_global_particle_vars_pointer_host(), sizeof(CUDA_global_part_vars), MPI_BYTE, 0, comm_cart); (when executed on all nodes)
   */
  void gpu_init_particle_comm() {
    if ( this_node == 0  && global_part_vars_host.communication_enabled == 0 ) {
      if( cuda_get_n_gpus() == -1 ) {
        fprintf(stderr, "Unable to initialize CUDA as no sufficient GPU is available.\n");
        exit(0);
      }
      if (cuda_get_n_gpus()>1) {
        fprintf (stderr, "More than one GPU detected, please note Espresso uses device 0 by default regardless of usage or capability\n");
        fprintf (stderr, "Note that the GPU to be used can be modified using cuda setdevice <int>\n");
        if (cuda_check_gpu(0)!=ES_OK) {
          fprintf (stderr, "WARNING!  CUDA device 0 is not capable of running Espresso but is used by default.  Espresso has detected a CUDA capable card but it is not the one used by Espresso by default\n");
          fprintf (stderr, "Please set the GPU to use with the cuda setdevice <int> command.\n");
          fprintf (stderr, "A list of available GPUs can be accessed using cuda list.\n");
        }
      }
    }
    global_part_vars_host.communication_enabled = 1;

    gpu_change_number_of_part_to_comm();

  }

  CUDA_particle_data* gpu_get_particle_pointer() {
    return particle_data_device;
  }
  CUDA_global_part_vars* gpu_get_global_particle_vars_pointer_host() {
    return &global_part_vars_host;
  }  
  CUDA_global_part_vars* gpu_get_global_particle_vars_pointer() {
    return &global_part_vars_device;
  }
  CUDA_particle_force* gpu_get_particle_force_pointer() {
    return particle_forces_device;
  }

  CUDA_particle_seed* gpu_get_particle_seed_pointer() {
    return particle_seeds_device;
  }

  void copy_part_data_to_gpu() {

    if ( global_part_vars_host.communication_enabled == 1 && global_part_vars_host.number_of_particles ) {
     
      cuda_mpi_get_particles(particle_data_host);

      /** get espresso md particle values*/
      if ( this_node == 0 ) hipMemcpyAsync(particle_data_device, particle_data_host, global_part_vars_host.number_of_particles * sizeof(CUDA_particle_data), hipMemcpyHostToDevice, stream[0]);

    }
  }



  /** setup and call kernel to copy particle forces to host
  */
  void copy_forces_from_GPU() {

    if ( global_part_vars_host.communication_enabled == 1 && global_part_vars_host.number_of_particles ) {

      /** Copy result from device memory to host memory*/
      if ( this_node == 0 ) {
        cuda_safe_mem (hipMemcpy(particle_forces_host, particle_forces_device, global_part_vars_host.number_of_particles * sizeof(CUDA_particle_force), hipMemcpyDeviceToHost));


        /** values for the particle kernel */
        int threads_per_block_particles = 64;
        int blocks_per_grid_particles_y = 4;
        int blocks_per_grid_particles_x = (global_part_vars_host.number_of_particles + threads_per_block_particles * blocks_per_grid_particles_y - 1)/(threads_per_block_particles * blocks_per_grid_particles_y);
        dim3 dim_grid_particles = make_uint3(blocks_per_grid_particles_x, blocks_per_grid_particles_y, 1);

        /** reset part forces with zero*/

        KERNELCALL(reset_particle_force, dim_grid_particles, threads_per_block_particles, (particle_forces_device));
        hipDeviceSynchronize();
      }
      cuda_mpi_send_forces(particle_forces_host);
    }
  }

  
}
