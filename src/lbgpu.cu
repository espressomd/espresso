#include "hip/hip_runtime.h"
/* 
   Copyright (C) 2010,2011,2012 The ESPResSo project

   This file is part of ESPResSo.
  
   ESPResSo is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.
   
   ESPResSo is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.
   
   You should have received a copy of the GNU General Public License
   along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

/** \file lbgpu.cu
 *
 * Cuda (.cu) file for the Lattice Boltzmann implementation on GPUs.
 * Header file for \ref lbgpu.h.
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

extern "C" {
#include "lbgpu.h"
}

#ifdef LB_GPU
#ifndef GAUSSRANDOM
#define GAUSSRANDOM
#endif

/**defining structures residing in global memory */
/** struct for phys. values */
static LB_values_gpu *device_values = NULL;
/** structs for velocity densities */
static LB_nodes_gpu nodes_a;
static LB_nodes_gpu nodes_b;
/** struct for particle force */
static LB_particle_force_gpu *particle_force = NULL;
/** struct for particle position and veloctiy */
static LB_particle_gpu *particle_data = NULL;
/** struct for node force */
static LB_node_force_gpu node_f;
/** struct for storing particle rn seed */
static LB_particle_seed_gpu *part = NULL;

static LB_extern_nodeforce_gpu *extern_nodeforces = NULL;
#ifdef LB_BOUNDARIES_GPU
/** pointer for bound index array*/
static int *boundindex;
static size_t size_of_boundindex;
#endif
/** pointers for additional cuda check flag*/
static int *gpu_check = NULL;
static int *h_gpu_check = NULL;

static unsigned int intflag = 1;
static LB_nodes_gpu *current_nodes = NULL;
/**defining size values for allocating global memory */
static size_t size_of_values;
static size_t size_of_forces;
static size_t size_of_positions;
static size_t size_of_seed;
static size_t size_of_extern_nodeforces;

/**parameters residing in constant memory */
static __device__ __constant__ LB_parameters_gpu para;
static const float c_sound_sq = 1.f/3.f;
/**cudasteams for parallel computing on cpu and gpu */
hipStream_t stream[1];

hipError_t err;
hipError_t _err;
/*-------------------------------------------------------*/
/*********************************************************/
/** \name device funktions called by kernel funktions */
/*********************************************************/
/*-------------------------------------------------------*/

/*-------------------------------------------------------*/

/** atomic add function for sveral cuda architectures 
*/
__device__ inline void atomicadd(float* address, float value){
#if !defined __CUDA_ARCH__ || __CUDA_ARCH__ >= 200 // for Fermi, atomicAdd supports floats
  atomicAdd(address, value);
#elif __CUDA_ARCH__ >= 110
#warning Using slower atomicAdd emulation
// float-atomic-add from 
// [url="http://forums.nvidia.com/index.php?showtopic=158039&view=findpost&p=991561"]
  float old = value;
  while ((old = atomicExch(address, atomicExch(address, 0.0f)+old))!=0.0f);
#else
#error I need at least compute capability 1.1
#endif
}

/**randomgenerator which generates numbers [0,1]
 * @param *rn	Pointer to randomnumber array of the local node or particle 
*/
__device__ void random_01(LB_randomnr_gpu *rn){

  const float mxi = 1.f/(float)(1ul<<31);
  unsigned int curr = rn->seed;

  curr = 1103515245 * curr + 12345;
  rn->randomnr[0] = (float)(curr & ((1ul<<31)-1))*mxi;
  curr = 1103515245 * curr + 12345;
  rn->randomnr[1] = (float)(curr & ((1ul<<31)-1))*mxi;
  rn->seed = curr;

}

/** gaussian random nummber generator for thermalisation
 * @param *rn	Pointer to randomnumber array of the local node node or particle 
*/
__device__ void gaussian_random(LB_randomnr_gpu *rn){

  float x1, x2;
  float r2, fac;
  /** On every second call two gaussian random numbers are calculated
   via the Box-Muller transformation.*/
  /** draw two uniform random numbers in the unit circle */
  do {
    random_01(rn);
    x1 = 2.f*rn->randomnr[0]-1.f;
    x2 = 2.f*rn->randomnr[1]-1.f;
    r2 = x1*x1 + x2*x2;
  } while (r2 >= 1.f || r2 == 0.f);

  /** perform Box-Muller transformation */
  fac = sqrtf(-2.f*__logf(r2)/r2);
  rn->randomnr[0] = x2*fac;
  rn->randomnr[1] = x1*fac;
  
}

/**tranformation from 1d array-index to xyz
 * @param index		node index / thread index (Input)
 * @param xyz		Pointer to calculated xyz array (Output)
 */
__device__ void index_to_xyz(unsigned int index, unsigned int *xyz){

  xyz[0] = index%para.dim_x;
  index /= para.dim_x;
  xyz[1] = index%para.dim_y;
  index /= para.dim_y;
  xyz[2] = index;
}

/**calculation of the modes from the velocitydensities (space-transform.)
 * @param n_a		Pointer to local node residing in array a (Input)
 * @param index		node index / thread index (Input)
 * @param mode		Pointer to the local register values mode (Output)
*/
__device__ void calc_m_from_n(LB_nodes_gpu n_a, unsigned int index, float *mode){

  /* mass mode */
  mode[0] = n_a.vd[0*para.number_of_nodes + index] + n_a.vd[1*para.number_of_nodes + index] + n_a.vd[2*para.number_of_nodes + index]
          + n_a.vd[3*para.number_of_nodes + index] + n_a.vd[4*para.number_of_nodes + index] + n_a.vd[5*para.number_of_nodes + index]
          + n_a.vd[6*para.number_of_nodes + index] + n_a.vd[7*para.number_of_nodes + index] + n_a.vd[8*para.number_of_nodes + index]
          + n_a.vd[9*para.number_of_nodes + index] + n_a.vd[10*para.number_of_nodes + index] + n_a.vd[11*para.number_of_nodes + index] + n_a.vd[12*para.number_of_nodes + index]
          + n_a.vd[13*para.number_of_nodes + index] + n_a.vd[14*para.number_of_nodes + index] + n_a.vd[15*para.number_of_nodes + index] + n_a.vd[16*para.number_of_nodes + index]
          + n_a.vd[17*para.number_of_nodes + index] + n_a.vd[18*para.number_of_nodes + index];

  /* momentum modes */
  mode[1] = (n_a.vd[1*para.number_of_nodes + index] - n_a.vd[2*para.number_of_nodes + index]) + (n_a.vd[7*para.number_of_nodes + index] - n_a.vd[8*para.number_of_nodes + index])
          + (n_a.vd[9*para.number_of_nodes + index] - n_a.vd[10*para.number_of_nodes + index]) + (n_a.vd[11*para.number_of_nodes + index] - n_a.vd[12*para.number_of_nodes + index])
          + (n_a.vd[13*para.number_of_nodes + index] - n_a.vd[14*para.number_of_nodes + index]);
  mode[2] = (n_a.vd[3*para.number_of_nodes + index] - n_a.vd[4*para.number_of_nodes + index]) + (n_a.vd[7*para.number_of_nodes + index] - n_a.vd[8*para.number_of_nodes + index])
          - (n_a.vd[9*para.number_of_nodes + index] - n_a.vd[10*para.number_of_nodes + index]) + (n_a.vd[15*para.number_of_nodes + index] - n_a.vd[16*para.number_of_nodes + index])
          + (n_a.vd[17*para.number_of_nodes + index] - n_a.vd[18*para.number_of_nodes + index]);
  mode[3] = (n_a.vd[5*para.number_of_nodes + index] - n_a.vd[6*para.number_of_nodes + index]) + (n_a.vd[11*para.number_of_nodes + index] - n_a.vd[12*para.number_of_nodes + index])
          - (n_a.vd[13*para.number_of_nodes + index] - n_a.vd[14*para.number_of_nodes + index]) + (n_a.vd[15*para.number_of_nodes + index] - n_a.vd[16*para.number_of_nodes + index])
          - (n_a.vd[17*para.number_of_nodes + index] - n_a.vd[18*para.number_of_nodes + index]);

  /* stress modes */
  mode[4] = -(n_a.vd[0*para.number_of_nodes + index]) + n_a.vd[7*para.number_of_nodes + index] + n_a.vd[8*para.number_of_nodes + index] + n_a.vd[9*para.number_of_nodes + index] + n_a.vd[10*para.number_of_nodes + index]
          + n_a.vd[11*para.number_of_nodes + index] + n_a.vd[12*para.number_of_nodes + index] + n_a.vd[13*para.number_of_nodes + index] + n_a.vd[14*para.number_of_nodes + index]
          + n_a.vd[15*para.number_of_nodes + index] + n_a.vd[16*para.number_of_nodes + index] + n_a.vd[17*para.number_of_nodes + index] + n_a.vd[18*para.number_of_nodes + index];
  mode[5] = n_a.vd[1*para.number_of_nodes + index] + n_a.vd[2*para.number_of_nodes + index] - (n_a.vd[3*para.number_of_nodes + index] + n_a.vd[4*para.number_of_nodes + index])
          + (n_a.vd[11*para.number_of_nodes + index] + n_a.vd[12*para.number_of_nodes + index]) + (n_a.vd[13*para.number_of_nodes + index] + n_a.vd[14*para.number_of_nodes + index])
          - (n_a.vd[15*para.number_of_nodes + index] + n_a.vd[16*para.number_of_nodes + index]) - (n_a.vd[17*para.number_of_nodes + index] + n_a.vd[18*para.number_of_nodes + index]);
  mode[6] = (n_a.vd[1*para.number_of_nodes + index] + n_a.vd[2*para.number_of_nodes + index]) + (n_a.vd[3*para.number_of_nodes + index] + n_a.vd[4*para.number_of_nodes + index])
          - (n_a.vd[11*para.number_of_nodes + index] + n_a.vd[12*para.number_of_nodes + index]) - (n_a.vd[13*para.number_of_nodes + index] + n_a.vd[14*para.number_of_nodes + index])
          - (n_a.vd[15*para.number_of_nodes + index] + n_a.vd[16*para.number_of_nodes + index]) - (n_a.vd[17*para.number_of_nodes + index] + n_a.vd[18*para.number_of_nodes + index])
          - 2.f*(n_a.vd[5*para.number_of_nodes + index] + n_a.vd[6*para.number_of_nodes + index] - (n_a.vd[7*para.number_of_nodes + index] + n_a.vd[8*para.number_of_nodes + index])
          - (n_a.vd[9*para.number_of_nodes + index] +n_a.vd[10*para.number_of_nodes + index]));
  mode[7] = n_a.vd[7*para.number_of_nodes + index] + n_a.vd[8*para.number_of_nodes + index] - (n_a.vd[9*para.number_of_nodes + index] + n_a.vd[10*para.number_of_nodes + index]);
  mode[8] = n_a.vd[11*para.number_of_nodes + index] + n_a.vd[12*para.number_of_nodes + index] - (n_a.vd[13*para.number_of_nodes + index] + n_a.vd[14*para.number_of_nodes + index]);
  mode[9] = n_a.vd[15*para.number_of_nodes + index] + n_a.vd[16*para.number_of_nodes + index] - (n_a.vd[17*para.number_of_nodes + index] + n_a.vd[18*para.number_of_nodes + index]);

  /* kinetic modes */
  mode[10] = -2.f*(n_a.vd[1*para.number_of_nodes + index] - n_a.vd[2*para.number_of_nodes + index]) + (n_a.vd[7*para.number_of_nodes + index] - n_a.vd[8*para.number_of_nodes + index])
           + (n_a.vd[9*para.number_of_nodes + index] - n_a.vd[10*para.number_of_nodes + index]) + (n_a.vd[11*para.number_of_nodes + index] - n_a.vd[12*para.number_of_nodes + index])
           + (n_a.vd[13*para.number_of_nodes + index] - n_a.vd[14*para.number_of_nodes + index]);
  mode[11] = -2.f*(n_a.vd[3*para.number_of_nodes + index] - n_a.vd[4*para.number_of_nodes + index]) + (n_a.vd[7*para.number_of_nodes + index] - n_a.vd[8*para.number_of_nodes + index])
           - (n_a.vd[9*para.number_of_nodes + index] - n_a.vd[10*para.number_of_nodes + index]) + (n_a.vd[15*para.number_of_nodes + index] - n_a.vd[16*para.number_of_nodes + index])
           + (n_a.vd[17*para.number_of_nodes + index] - n_a.vd[18*para.number_of_nodes + index]);
  mode[12] = -2.f*(n_a.vd[5*para.number_of_nodes + index] - n_a.vd[6*para.number_of_nodes + index]) + (n_a.vd[11*para.number_of_nodes + index] - n_a.vd[12*para.number_of_nodes + index])
           - (n_a.vd[13*para.number_of_nodes + index] - n_a.vd[14*para.number_of_nodes + index]) + (n_a.vd[15*para.number_of_nodes + index] - n_a.vd[16*para.number_of_nodes + index])
           - (n_a.vd[17*para.number_of_nodes + index] - n_a.vd[18*para.number_of_nodes + index]);
  mode[13] = (n_a.vd[7*para.number_of_nodes + index] - n_a.vd[8*para.number_of_nodes + index]) + (n_a.vd[9*para.number_of_nodes + index] - n_a.vd[10*para.number_of_nodes + index])
           - (n_a.vd[11*para.number_of_nodes + index] - n_a.vd[12*para.number_of_nodes + index]) - (n_a.vd[13*para.number_of_nodes + index] - n_a.vd[14*para.number_of_nodes + index]);
  mode[14] = (n_a.vd[7*para.number_of_nodes + index] - n_a.vd[8*para.number_of_nodes + index]) - (n_a.vd[9*para.number_of_nodes + index] - n_a.vd[10*para.number_of_nodes + index])
           - (n_a.vd[15*para.number_of_nodes + index] - n_a.vd[16*para.number_of_nodes + index]) - (n_a.vd[17*para.number_of_nodes + index] - n_a.vd[18*para.number_of_nodes + index]);
  mode[15] = (n_a.vd[11*para.number_of_nodes + index] - n_a.vd[12*para.number_of_nodes + index]) - (n_a.vd[13*para.number_of_nodes + index] - n_a.vd[14*para.number_of_nodes + index])
           - (n_a.vd[15*para.number_of_nodes + index] - n_a.vd[16*para.number_of_nodes + index]) + (n_a.vd[17*para.number_of_nodes + index] - n_a.vd[18*para.number_of_nodes + index]);
  mode[16] = n_a.vd[0*para.number_of_nodes + index] + n_a.vd[7*para.number_of_nodes + index] + n_a.vd[8*para.number_of_nodes + index] + n_a.vd[9*para.number_of_nodes + index] + n_a.vd[10*para.number_of_nodes + index]
           + n_a.vd[11*para.number_of_nodes + index] + n_a.vd[12*para.number_of_nodes + index] + n_a.vd[13*para.number_of_nodes + index] + n_a.vd[14*para.number_of_nodes + index]
           + n_a.vd[15*para.number_of_nodes + index] + n_a.vd[16*para.number_of_nodes + index] + n_a.vd[17*para.number_of_nodes + index] + n_a.vd[18*para.number_of_nodes + index]
           - 2.f*((n_a.vd[1*para.number_of_nodes + index] + n_a.vd[2*para.number_of_nodes + index]) + (n_a.vd[3*para.number_of_nodes + index] + n_a.vd[4*para.number_of_nodes + index])
           + (n_a.vd[5*para.number_of_nodes + index] + n_a.vd[6*para.number_of_nodes + index]));
  mode[17] = -(n_a.vd[1*para.number_of_nodes + index] + n_a.vd[2*para.number_of_nodes + index]) + (n_a.vd[3*para.number_of_nodes + index] + n_a.vd[4*para.number_of_nodes + index])
           + (n_a.vd[11*para.number_of_nodes + index] + n_a.vd[12*para.number_of_nodes + index]) + (n_a.vd[13*para.number_of_nodes + index] + n_a.vd[14*para.number_of_nodes + index])
           - (n_a.vd[15*para.number_of_nodes + index] + n_a.vd[16*para.number_of_nodes + index]) - (n_a.vd[17*para.number_of_nodes + index] + n_a.vd[18*para.number_of_nodes + index]);
  mode[18] = -(n_a.vd[1*para.number_of_nodes + index] + n_a.vd[2*para.number_of_nodes + index]) - (n_a.vd[3*para.number_of_nodes + index] + n_a.vd[4*para.number_of_nodes + index])
           - (n_a.vd[11*para.number_of_nodes + index] + n_a.vd[12*para.number_of_nodes + index]) - (n_a.vd[13*para.number_of_nodes + index] + n_a.vd[14*para.number_of_nodes + index])
           - (n_a.vd[15*para.number_of_nodes + index] + n_a.vd[16*para.number_of_nodes + index]) - (n_a.vd[17*para.number_of_nodes + index] + n_a.vd[18*para.number_of_nodes + index])
           + 2.f*((n_a.vd[5*para.number_of_nodes + index] + n_a.vd[6*para.number_of_nodes + index]) + (n_a.vd[7*para.number_of_nodes + index] + n_a.vd[8*para.number_of_nodes + index])
           + (n_a.vd[9*para.number_of_nodes + index] + n_a.vd[10*para.number_of_nodes + index]));

}

/**lb_relax_modes, means collision update of the modes
 * @param index		node index / thread index (Input)
 * @param mode		Pointer to the local register values mode (Input/Output)
 * @param node_f	Pointer to local node force (Input)
*/
__device__ void relax_modes(float *mode, unsigned int index, LB_node_force_gpu node_f){

  float Rho = mode[0] + para.rho*para.agrid*para.agrid*para.agrid;
  float j[3], pi_eq[6];

  /** re-construct the real density
  * remember that the populations are stored as differences to their
  * equilibrium value */

  j[0] = mode[1];
  j[1] = mode[2];
  j[2] = mode[3];

  /** if forces are present, the momentum density is redefined to
  * inlcude one half-step of the force action.  See the
  * Chapman-Enskog expansion in [Ladd & Verberg]. */

  j[0] += 0.5f*node_f.force[0*para.number_of_nodes + index];
  j[1] += 0.5f*node_f.force[1*para.number_of_nodes + index];
  j[2] += 0.5f*node_f.force[2*para.number_of_nodes + index];

  /** equilibrium part of the stress modes (eq13 schiller)*/
  pi_eq[0] = ((j[0]*j[0])+(j[1]*j[1])+(j[2]*j[2]))/Rho;
  pi_eq[1] = ((j[0]*j[0])-(j[1]*j[1]))/Rho;
  pi_eq[2] = (((j[0]*j[0])+(j[1]*j[1])+(j[2]*j[2])) - 3.0f*(j[2]*j[2]))/Rho;
  pi_eq[3] = j[0]*j[1]/Rho;
  pi_eq[4] = j[0]*j[2]/Rho;
  pi_eq[5] = j[1]*j[2]/Rho;

  /** relax the stress modes (eq14 schiller)*/
  mode[4] = pi_eq[0] + para.gamma_bulk*(mode[4] - pi_eq[0]);
  mode[5] = pi_eq[1] + para.gamma_shear*(mode[5] - pi_eq[1]);
  mode[6] = pi_eq[2] + para.gamma_shear*(mode[6] - pi_eq[2]);
  mode[7] = pi_eq[3] + para.gamma_shear*(mode[7] - pi_eq[3]);
  mode[8] = pi_eq[4] + para.gamma_shear*(mode[8] - pi_eq[4]);
  mode[9] = pi_eq[5] + para.gamma_shear*(mode[9] - pi_eq[5]);

  /** relax the ghost modes (project them out) */
  /** ghost modes have no equilibrium part due to orthogonality */
  mode[10] = para.gamma_odd*mode[10];
  mode[11] = para.gamma_odd*mode[11];
  mode[12] = para.gamma_odd*mode[12];
  mode[13] = para.gamma_odd*mode[13];
  mode[14] = para.gamma_odd*mode[14];
  mode[15] = para.gamma_odd*mode[15];
  mode[16] = para.gamma_even*mode[16];
  mode[17] = para.gamma_even*mode[17];
  mode[18] = para.gamma_even*mode[18];

}

/**thermalization of the modes with gaussian random numbers
 * @param index		node index / thread index (Input)
 * @param mode		Pointer to the local register values mode (Input/Output)
 * @param *rn		Pointer to randomnumber array of the local node
*/
__device__ void thermalize_modes(float *mode, unsigned int index, LB_randomnr_gpu *rn){

  float Rho = mode[0] + para.rho*para.agrid*para.agrid*para.agrid;

#ifdef GAUSSRANDOM
  /** stress modes */
  gaussian_random(rn);
  mode[4] += sqrt(Rho*(para.mu*(2.f/3.f)*(1.f-(para.gamma_bulk*para.gamma_bulk)))) * rn->randomnr[1];
  mode[5] += sqrt(Rho*(para.mu*(4.f/9.f)*(1.f-(para.gamma_shear*para.gamma_shear)))) * rn->randomnr[0];

  gaussian_random(rn);
  mode[6] += sqrt(Rho*(para.mu*(4.f/3.f)*(1.f-(para.gamma_shear*para.gamma_shear)))) * rn->randomnr[1];
  mode[7] += sqrt(Rho*(para.mu*(1.f/9.f)*(1.f-(para.gamma_shear*para.gamma_shear)))) * rn->randomnr[0];

  gaussian_random(rn);
  mode[8] += sqrt(Rho*(para.mu*(1.f/9.f)*(1.f-(para.gamma_shear*para.gamma_shear)))) * rn->randomnr[1];
  mode[9] += sqrt(Rho*(para.mu*(1.f/9.f)*(1.f-(para.gamma_shear*para.gamma_shear)))) * rn->randomnr[0];
 
  /** ghost modes */
  gaussian_random(rn);
  mode[10] += sqrt(Rho*(para.mu*(2.f/3.f))) * rn->randomnr[1];
  mode[11] += sqrt(Rho*(para.mu*(2.f/3.f))) * rn->randomnr[0];

  gaussian_random(rn);
  mode[12] += sqrt(Rho*(para.mu*(2.f/3.f))) * rn->randomnr[1];
  mode[13] += sqrt(Rho*(para.mu*(2.f/9.f))) * rn->randomnr[0];

  gaussian_random(rn);
  mode[14] += sqrt(Rho*(para.mu*(2.f/9.f))) * rn->randomnr[1];
  mode[15] += sqrt(Rho*(para.mu*(2.f/9.f))) * rn->randomnr[0];

  gaussian_random(rn);
  mode[16] += sqrt(Rho*(para.mu*(2.f))) * rn->randomnr[1];
  mode[17] += sqrt(Rho*(para.mu*(4.f/9.f))) * rn->randomnr[0];

  gaussian_random(rn);
  mode[18] += sqrt(Rho*(para.mu*(4.f/3.f))) * rn->randomnr[1];
#else
  /** stress modes */
  random_01(rn);
  mode[4] += sqrt(12.f*Rho*para.mu*(2.f/3.f)*(1.f-(para.gamma_bulk*para.gamma_bulk))) * (rn->randomnr[1]-0.5f);
  mode[5] += sqrt(12.f*Rho*para.mu*(4.f/9.f)*(1.f-(para.gamma_shear*para.gamma_shear))) * (rn->randomnr[0]-0.5f);

  random_01(rn);
  mode[6] += sqrt(12.f*Rho*para.mu*(4.f/3.f)*(1.f-(para.gamma_shear*para.gamma_shear))) * (rn->randomnr[1]-0.5f);
  mode[7] += sqrt(12.f*Rho*para.mu*(1.f/9.f)*(1.f-(para.gamma_shear*para.gamma_shear))) * (rn->randomnr[0]-0.5f);

  random_01(rn);
  mode[8] += sqrt(12.f*para.mu*(1.f/9.f)*(1.f-(para.gamma_shear*para.gamma_shear))) * (rn->randomnr[1]-0.5f);
  mode[9] += sqrt(12.f*para.mu*(1.f/9.f)*(1.f-(para.gamma_shear*para.gamma_shear))) * (rn->randomnr[0]-0.5f);
 
  /** ghost modes */
  random_01(rn);
  mode[10] += sqrt(12.f*Rho*para.mu*(2.f/3.f)) * (rn->randomnr[1]-0.5f);
  mode[11] += sqrt(12.f*Rho*para.mu*(2.f/3.f)) * (rn->randomnr[0]-0.5f);

  random_01(rn);
  mode[12] += sqrt(12.f*Rho*para.mu*(2.f/3.f)) * (rn->randomnr[1]-0.5f);
  mode[13] += sqrt(12.f*Rho*para.mu*(2.f/9.f)) * (rn->randomnr[0]-0.5f);

  random_01(rn);
  mode[14] += sqrt(12.f*Rho*para.mu*(2.f/9.f)) * (rn->randomnr[1]-0.5f);
  mode[15] += sqrt(12.f*Rho*para.mu*(2.f/9.f)) * (rn->randomnr[0]-0.5f);

  random_01(rn);
  mode[16] += sqrt(12.f*Rho*para.mu*(2.f)) * (rn->randomnr[1]-0.5f);
  mode[17] += sqrt(12.f*Rho*para.mu*(4.f/9.f)) * (rn->randomnr[0]-0.5f);

  random_01(rn);
  mode[18] += sqrt(12.f*Rho*para.mu*(4.f/3.f)) * (rn->randomnr[1]-0.5f);
#endif
}
/*-------------------------------------------------------*/
/**normalization of the modes need befor backtransformation into velocity space
 * @param mode		Pointer to the local register values mode (Input/Output)
*/
__device__ void normalize_modes(float* mode){

  /** normalization factors enter in the back transformation */
  mode[0] *= 1.f;
  mode[1] *= 3.f;
  mode[2] *= 3.f;
  mode[3] *= 3.f;
  mode[4] *= 3.f/2.f;
  mode[5] *= 9.f/4.f;
  mode[6] *= 3.f/4.f;
  mode[7] *= 9.f;
  mode[8] *= 9.f;
  mode[9] *= 9.f;
  mode[10] *= 3.f/2.f;
  mode[11] *= 3.f/2.f;
  mode[12] *= 3.f/2.f;
  mode[13] *= 9.f/2.f;
  mode[14] *= 9.f/2.f;
  mode[15] *= 9.f/2.f;
  mode[16] *= 1.f/2.f;
  mode[17] *= 9.f/4.f;
  mode[18] *= 3.f/4.f;

}
/*-------------------------------------------------------*/
/**backtransformation from modespace to desityspace and streaming with the push method using pbc
 * @param index		node index / thread index (Input)
 * @param mode		Pointer to the local register values mode (Input)
 * @param *n_b		Pointer to local node residing in array b (Output)
*/
__device__ void calc_n_from_modes_push(LB_nodes_gpu n_b, float *mode, unsigned int index){

  unsigned int xyz[3];
  index_to_xyz(index, xyz);
  unsigned int x = xyz[0];
  unsigned int y = xyz[1];
  unsigned int z = xyz[2];

  n_b.vd[0*para.number_of_nodes + x + para.dim_x*y + para.dim_x*para.dim_y*z] = 1.f/3.f * (mode[0] - mode[4] + mode[16]);
  n_b.vd[1*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z] = 1.f/18.f * (mode[0] + mode[1] + mode[5] + mode[6] - mode[17] - mode[18] - 2.f*(mode[10] + mode[16]));
  n_b.vd[2*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z] = 1.f/18.f * (mode[0] - mode[1] + mode[5] + mode[6] - mode[17] - mode[18] + 2.f*(mode[10] - mode[16]));
  n_b.vd[3*para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = 1.f/18.f * (mode[0] + mode[2] - mode[5] + mode[6] + mode[17] - mode[18] - 2.f*(mode[11] + mode[16]));
  n_b.vd[4*para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = 1.f/18.f * (mode[0] - mode[2] - mode[5] + mode[6] + mode[17] - mode[18] + 2.f*(mode[11] - mode[16]));
  n_b.vd[5*para.number_of_nodes + x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = 1.f/18.f * (mode[0] + mode[3] - 2.f*(mode[6] + mode[12] + mode[16] - mode[18]));
  n_b.vd[6*para.number_of_nodes + x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = 1.f/18.f * (mode[0] - mode[3] - 2.f*(mode[6] - mode[12] + mode[16] - mode[18]));
  n_b.vd[7*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = 1.f/36.f * (mode[0] + mode[1] + mode[2] + mode[4] + 2.f*mode[6] + mode[7] + mode[10] + mode[11] + mode[13] + mode[14] + mode[16] + 2.f*mode[18]);
  n_b.vd[8*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = 1.f/36.f * (mode[0] - mode[1] - mode[2] + mode[4] + 2.f*mode[6] + mode[7] - mode[10] - mode[11] - mode[13] - mode[14] + mode[16] + 2.f*mode[18]);
  n_b.vd[9*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = 1.f/36.f * (mode[0] + mode[1] - mode[2] + mode[4] + 2.f*mode[6] - mode[7] + mode[10] - mode[11] + mode[13] - mode[14] + mode[16] + 2.f*mode[18]);
  n_b.vd[10*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = 1.f/36.f * (mode[0] - mode[1] + mode[2] + mode[4] + 2.f*mode[6] - mode[7] - mode[10] + mode[11] - mode[13] + mode[14] + mode[16] + 2.f*mode[18]);
  n_b.vd[11*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = 1.f/36.f * (mode[0] + mode[1] + mode[3] + mode[4] + mode[5] - mode[6] + mode[8] + mode[10] + mode[12] - mode[13] + mode[15] + mode[16] + mode[17] - mode[18]);
  n_b.vd[12*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = 1.f/36.f * (mode[0] - mode[1] - mode[3] + mode[4] + mode[5] - mode[6] + mode[8] - mode[10] - mode[12] + mode[13] - mode[15] + mode[16] + mode[17] - mode[18]);
  n_b.vd[13*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = 1.f/36.f * (mode[0] + mode[1] - mode[3] + mode[4] + mode[5] - mode[6] - mode[8] + mode[10] - mode[12] - mode[13] - mode[15] + mode[16] + mode[17] - mode[18]);
  n_b.vd[14*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = 1.f/36.f * (mode[0] - mode[1] + mode[3] + mode[4] + mode[5] - mode[6] - mode[8] - mode[10] + mode[12] + mode[13] + mode[15] + mode[16] + mode[17] - mode[18]);
  n_b.vd[15*para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = 1.f/36.f * (mode[0] + mode[2] + mode[3] + mode[4] - mode[5] - mode[6] + mode[9] + mode[11] + mode[12] - mode[14] - mode[15] + mode[16] - mode[17] - mode[18]);
  n_b.vd[16*para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = 1.f/36.f * (mode[0] - mode[2] - mode[3] + mode[4] - mode[5] - mode[6] + mode[9] - mode[11] - mode[12] + mode[14] + mode[15] + mode[16] - mode[17] - mode[18]);
  n_b.vd[17*para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = 1.f/36.f * (mode[0] + mode[2] - mode[3] + mode[4] - mode[5] - mode[6] - mode[9] + mode[11] - mode[12] - mode[14] + mode[15] + mode[16] - mode[17] - mode[18]);
  n_b.vd[18*para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = 1.f/36.f * (mode[0] - mode[2] + mode[3] + mode[4] - mode[5] - mode[6] - mode[9] - mode[11] + mode[12] + mode[14] - mode[15] + mode[16] - mode[17] - mode[18]);

}

/** Bounce back boundary conditions.
 * The populations that have propagated into a boundary node
 * are bounced back to the node they came from. This results
 * in no slip boundary conditions.
 *
 * [cf. Ladd and Verberg, J. Stat. Phys. 104(5/6):1191-1251, 2001]
 * @param index			node index / thread index (Input)
 * @param n_b			Pointer to local node residing in array b (Input)
 * @param n_a			Pointer to local node residing in array a (Output) (temp stored in buffer a)
*/
__device__ void bounce_back_read(LB_nodes_gpu n_b, LB_nodes_gpu n_a, unsigned int index){
    
  unsigned int xyz[3];

  if(n_b.boundary[index] == 1){
    index_to_xyz(index, xyz);
    unsigned int x = xyz[0];
    unsigned int y = xyz[1];
    unsigned int z = xyz[2];

    /** store vd temporary in second lattice to avoid race conditions */
    n_a.vd[1*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z] = n_b.vd[2*para.number_of_nodes + index];
    n_a.vd[2*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z] = n_b.vd[1*para.number_of_nodes + index];
    n_a.vd[3*para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_b.vd[4*para.number_of_nodes + index];
    n_a.vd[4*para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_b.vd[3*para.number_of_nodes + index];
    n_a.vd[5*para.number_of_nodes + x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_b.vd[6*para.number_of_nodes + index];
    n_a.vd[6*para.number_of_nodes + x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_b.vd[5*para.number_of_nodes + index];
    n_a.vd[7*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_b.vd[8*para.number_of_nodes + index];
    n_a.vd[8*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_b.vd[7*para.number_of_nodes + index];
    n_a.vd[9*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_b.vd[10*para.number_of_nodes + index];
    n_a.vd[10*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_b.vd[9*para.number_of_nodes + index];
    n_a.vd[11*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_b.vd[12*para.number_of_nodes + index];
    n_a.vd[12*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_b.vd[11*para.number_of_nodes + index]; 
    n_a.vd[13*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_b.vd[14*para.number_of_nodes + index]; 
    n_a.vd[14*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_b.vd[13*para.number_of_nodes + index]; 
    n_a.vd[15*para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_b.vd[16*para.number_of_nodes + index];
    n_a.vd[16*para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_b.vd[15*para.number_of_nodes + index];
    n_a.vd[17*para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_b.vd[18*para.number_of_nodes + index]; 
    n_a.vd[18*para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_b.vd[17*para.number_of_nodes + index];
  }
}
/**bounce back read kernel needed to avoid raceconditions
 * @param index			node index / thread index (Input)
 * @param n_b			Pointer to local node residing in array b (Input)
 * @param n_a			Pointer to local node residing in array a (Output) (temp stored in buffer a)
*/
__device__ void bounce_back_write(LB_nodes_gpu n_b, LB_nodes_gpu n_a, unsigned int index){

  unsigned int xyz[3];

  if(n_b.boundary[index] == 1){
    index_to_xyz(index, xyz);
    unsigned int x = xyz[0];
    unsigned int y = xyz[1];
    unsigned int z = xyz[2];

    /** stream vd from boundary node back to origin node */
    n_b.vd[1*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z] = n_a.vd[1*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z];
    n_b.vd[2*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z] = n_a.vd[2*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z];
    n_b.vd[3*para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_a.vd[3*para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z];
    n_b.vd[4*para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_a.vd[4*para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z];
    n_b.vd[5*para.number_of_nodes + x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_a.vd[5*para.number_of_nodes + x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)];
    n_b.vd[6*para.number_of_nodes + x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_a.vd[6*para.number_of_nodes + x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)];
    n_b.vd[7*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_a.vd[7*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z];
    n_b.vd[8*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_a.vd[8*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z];
    n_b.vd[9*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_a.vd[9*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z];
    n_b.vd[10*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_a.vd[10*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z];
    n_b.vd[11*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_a.vd[11*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)];
    n_b.vd[12*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_a.vd[12*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)];
    n_b.vd[13*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_a.vd[13*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)];
    n_b.vd[14*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_a.vd[14*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)];
    n_b.vd[15*para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_a.vd[15*para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)];
    n_b.vd[16*para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_a.vd[16*para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)];
    n_b.vd[17*para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_a.vd[17*para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)];
    n_b.vd[18*para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_a.vd[18*para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)];
  }
}
/** add of (external) forces within the modespace, needed for particle-interaction
 * @param index		node index / thread index (Input)
 * @param mode		Pointer to the local register values mode (Input/Output)
 * @param node_f	Pointer to local node force (Input)
*/
__device__ void apply_forces(unsigned int index, float *mode, LB_node_force_gpu node_f) {

  float Rho, u[3], C[6];
  Rho = mode[0] + para.rho*para.agrid*para.agrid*para.agrid;

  /** hydrodynamic momentum density is redefined when forces present */
  u[0] = (mode[1] + 0.5f*node_f.force[0*para.number_of_nodes + index])/Rho;
  u[1] = (mode[2] + 0.5f*node_f.force[1*para.number_of_nodes + index])/Rho;
  u[2] = (mode[3] + 0.5f*node_f.force[2*para.number_of_nodes + index])/Rho;

  C[0] = (1.f + para.gamma_bulk)*u[0]*node_f.force[0*para.number_of_nodes + index] + 1.f/3.f*(para.gamma_bulk-para.gamma_shear)*(u[0]*node_f.force[0*para.number_of_nodes + index] + u[1]*node_f.force[1*para.number_of_nodes + index] + u[2]*node_f.force[2*para.number_of_nodes + index]);
  C[2] = (1.f + para.gamma_bulk)*u[1]*node_f.force[1*para.number_of_nodes + index] + 1.f/3.f*(para.gamma_bulk-para.gamma_shear)*(u[0]*node_f.force[0*para.number_of_nodes + index] + u[1]*node_f.force[1*para.number_of_nodes + index] + u[2]*node_f.force[2*para.number_of_nodes + index]);
  C[5] = (1.f + para.gamma_bulk)*u[2]*node_f.force[2*para.number_of_nodes + index] + 1.f/3.f*(para.gamma_bulk-para.gamma_shear)*(u[0]*node_f.force[0*para.number_of_nodes + index] + u[1]*node_f.force[1*para.number_of_nodes + index] + u[2]*node_f.force[2*para.number_of_nodes + index]);
  C[1] = 1.f/2.f*(1.f+para.gamma_shear)*(u[0]*node_f.force[1*para.number_of_nodes + index]+u[1]*node_f.force[0*para.number_of_nodes + index]);
  C[3] = 1.f/2.f*(1.f+para.gamma_shear)*(u[0]*node_f.force[2*para.number_of_nodes + index]+u[2]*node_f.force[0*para.number_of_nodes + index]);
  C[4] = 1.f/2.f*(1.f+para.gamma_shear)*(u[1]*node_f.force[2*para.number_of_nodes + index]+u[2]*node_f.force[1*para.number_of_nodes + index]);

  /** update momentum modes */
  mode[1] += node_f.force[0*para.number_of_nodes + index];
  mode[2] += node_f.force[1*para.number_of_nodes + index];
  mode[3] += node_f.force[2*para.number_of_nodes + index];
  	
  /** update stress modes */
  mode[4] += C[0] + C[2] + C[5];
  mode[5] += C[0] - C[2];
  mode[6] += C[0] + C[2] - 2.f*C[5];
  mode[7] += C[1];
  mode[8] += C[3];
  mode[9] += C[4];

#ifdef EXTERNAL_FORCES
  if(para.external_force){
    node_f.force[0*para.number_of_nodes + index] = para.ext_force[0]*powf(para.agrid,4)*para.tau*para.tau;
    node_f.force[1*para.number_of_nodes + index] = para.ext_force[1]*powf(para.agrid,4)*para.tau*para.tau;
    node_f.force[2*para.number_of_nodes + index] = para.ext_force[2]*powf(para.agrid,4)*para.tau*para.tau;
  }
  else{
  node_f.force[0*para.number_of_nodes + index] = 0.f;
  node_f.force[1*para.number_of_nodes + index] = 0.f;
  node_f.force[2*para.number_of_nodes + index] = 0.f;
  }
#else
  /** reset force */
  node_f.force[0*para.number_of_nodes + index] = 0.f;
  node_f.force[1*para.number_of_nodes + index] = 0.f;
  node_f.force[2*para.number_of_nodes + index] = 0.f;
#endif
}

/**function used to calc physical values of every node
 * @param index		node index / thread index (Input)
 * @param mode		Pointer to the local register values mode (Input)
 * @param n_a		Pointer to local node residing in array a for boundary flag(Input)
 * @param *d_v		Pointer to local device values (Input/Output)
 * @param singlenode	Flag, if there is only one node
*/
__device__ void calc_values(LB_nodes_gpu n_a, float *mode, LB_values_gpu *d_v, unsigned int index, unsigned int singlenode){

  float Rho = mode[0] + para.rho*para.agrid*para.agrid*para.agrid;
	
  /**implemented due to the problem of division via zero*/
  if(n_a.boundary[index] == 1){
    Rho = 1.0f;
    mode[1] = 0.f;
    mode[2] = 0.f;
    mode[3] = 0.f;
  }

  if(singlenode == 1){
    d_v[0].rho = Rho;
    d_v[0].v[0] = mode[1]/Rho/para.agrid/para.tau;
    d_v[0].v[1] = mode[2]/Rho/para.agrid/para.tau;
    d_v[0].v[2] = mode[3]/Rho/para.agrid/para.tau;
  }
  else{
    d_v[index].rho = Rho;
    d_v[index].v[0] = mode[1]/Rho/para.agrid/para.tau;
    d_v[index].v[1] = mode[2]/Rho/para.agrid/para.tau;
    d_v[index].v[2] = mode[3]/Rho/para.agrid/para.tau;
  }
#if 0
  if(singlenode == 1){
    /** equilibrium part of the stress modes */
    /**to print out the stress tensor entries, ensure that in lbgpu.h struct the values are available*/
    d_v[0].pi[0] = ((mode[1]*mode[1]) + (mode[2]*mode[2]) + (mode[3]*mode[3]))/para.rho;
    d_v[0].pi[1] = ((mode[1]*mode[1]) - (mode[2]*mode[2]))/para.rho;
    d_v[0].pi[2] = ((mode[1]*mode[1]) + (mode[2]*mode[2])  + (mode[3]*mode[3])) - 3.0f*(mode[3]*mode[3]))/para.rho;
    d_v[0].pi[3] = mode[1]*mode[2]/para.rho;
    d_v[0].pi[4] = mode[1]*mode[3]/para.rho;
    d_v[0].pi[5] = mode[2]*mode[3]/para.rho;
  else{
    d_v[index].pi[0] = ((mode[1]*mode[1]) + (mode[2]*mode[2]) + (mode[3]*mode[3]))/para.rho;
    d_v[index].pi[1] = ((mode[1]*mode[1]) - (mode[2]*mode[2]))/para.rho;
    d_v[index].pi[2] = ((mode[1]*mode[1]) + (mode[2]*mode[2])  + (mode[3]*mode[3])) - 3.0f*(mode[3]*mode[3]))/para.rho;
    d_v[index].pi[3] = mode[1]*mode[2]/para.rho;
    d_v[index].pi[4] = mode[1]*mode[3]/para.rho;
    d_v[index].pi[5] = mode[2]*mode[3]/para.rho;
  }
#endif
}
/** 
 * @param node_index	node index around (8) particle (Input)
 * @param *mode			Pointer to the local register values mode (Output)
 * @param n_a			Pointer to local node residing in array a(Input)
*/
__device__ void calc_mode(float *mode, LB_nodes_gpu n_a, unsigned int node_index){
	
  /** mass mode */
  mode[0] = n_a.vd[0*para.number_of_nodes + node_index] + n_a.vd[1*para.number_of_nodes + node_index] + n_a.vd[2*para.number_of_nodes + node_index]
          + n_a.vd[3*para.number_of_nodes + node_index] + n_a.vd[4*para.number_of_nodes + node_index] + n_a.vd[5*para.number_of_nodes + node_index]
          + n_a.vd[6*para.number_of_nodes + node_index] + n_a.vd[7*para.number_of_nodes + node_index] + n_a.vd[8*para.number_of_nodes + node_index]
          + n_a.vd[9*para.number_of_nodes + node_index] + n_a.vd[10*para.number_of_nodes + node_index] + n_a.vd[11*para.number_of_nodes + node_index] + n_a.vd[12*para.number_of_nodes + node_index]
          + n_a.vd[13*para.number_of_nodes + node_index] + n_a.vd[14*para.number_of_nodes + node_index] + n_a.vd[15*para.number_of_nodes + node_index] + n_a.vd[16*para.number_of_nodes + node_index]
          + n_a.vd[17*para.number_of_nodes + node_index] + n_a.vd[18*para.number_of_nodes + node_index];

  /** momentum modes */
  mode[1] = (n_a.vd[1*para.number_of_nodes + node_index] - n_a.vd[2*para.number_of_nodes + node_index]) + (n_a.vd[7*para.number_of_nodes + node_index] - n_a.vd[8*para.number_of_nodes + node_index])
          + (n_a.vd[9*para.number_of_nodes + node_index] - n_a.vd[10*para.number_of_nodes + node_index]) + (n_a.vd[11*para.number_of_nodes + node_index] - n_a.vd[12*para.number_of_nodes + node_index])
          + (n_a.vd[13*para.number_of_nodes + node_index] - n_a.vd[14*para.number_of_nodes + node_index]);
  mode[2] = (n_a.vd[3*para.number_of_nodes + node_index] - n_a.vd[4*para.number_of_nodes + node_index]) + (n_a.vd[7*para.number_of_nodes + node_index] - n_a.vd[8*para.number_of_nodes + node_index])
          - (n_a.vd[9*para.number_of_nodes + node_index] - n_a.vd[10*para.number_of_nodes + node_index]) + (n_a.vd[15*para.number_of_nodes + node_index] - n_a.vd[16*para.number_of_nodes + node_index])
          + (n_a.vd[17*para.number_of_nodes + node_index] - n_a.vd[18*para.number_of_nodes + node_index]);
  mode[3] = (n_a.vd[5*para.number_of_nodes + node_index] - n_a.vd[6*para.number_of_nodes + node_index]) + (n_a.vd[11*para.number_of_nodes + node_index] - n_a.vd[12*para.number_of_nodes + node_index])
          - (n_a.vd[13*para.number_of_nodes + node_index] - n_a.vd[14*para.number_of_nodes + node_index]) + (n_a.vd[15*para.number_of_nodes + node_index] - n_a.vd[16*para.number_of_nodes + node_index])
          - (n_a.vd[17*para.number_of_nodes + node_index] - n_a.vd[18*para.number_of_nodes + node_index]);
}
/*********************************************************/
/** \name Coupling part */
/*********************************************************/
/**(Eq. (12) Ahlrichs and Duenweg, JCP 111(17):8225 (1999))
 * @param n_a			Pointer to local node residing in array a (Input)
 * @param *delta		Pointer for the weighting of particle position (Output)
 * @param *delta_j		Pointer for the weighting of particle momentum (Output)
 * @param *particle_data	Pointer to the particle position and velocity (Input)
 * @param *particle_force	Pointer to the particle force (Input)
 * @param part_index		particle id / thread id (Input)
 * @param *rn_part		Pointer to randomnumber array of the particle
 * @param node_index		node index around (8) particle (Output)
*/
__device__ void calc_viscous_force(LB_nodes_gpu n_a, float *delta, LB_particle_gpu *particle_data, LB_particle_force_gpu *particle_force, unsigned int part_index, LB_randomnr_gpu *rn_part, float *delta_j, unsigned int *node_index){
	
  float mode[4];
  int my_left[3];
  float interpolated_u1, interpolated_u2, interpolated_u3;
  float Rho;
  interpolated_u1 = interpolated_u2 = interpolated_u3 = 0.f;

  float temp_delta[6];
  float temp_delta_half[6];

  /** see ahlrichs + duennweg page 8227 equ (10) and (11) */
  #pragma unroll
  for(int i=0; i<3; ++i){
    float scaledpos = particle_data[part_index].p[i]/para.agrid;
    my_left[i] = (int)(floorf(scaledpos - 0.5f));
    //printf("scaledpos %f \t myleft: %u \n", scaledpos, my_left[i]);
    temp_delta[3+i] = scaledpos - my_left[i];
    temp_delta[i] = 1.f - temp_delta[3+i];
    /**further value used for interpolation of fluid velocity at part pos near boundaries */
    temp_delta_half[3+i] = (scaledpos - my_left[i])*2.f;
    temp_delta_half[i] = 2.f - temp_delta_half[3+i];
  }

  delta[0] = temp_delta[0] * temp_delta[1] * temp_delta[2];
  delta[1] = temp_delta[3] * temp_delta[1] * temp_delta[2];
  delta[2] = temp_delta[0] * temp_delta[4] * temp_delta[2];
  delta[3] = temp_delta[3] * temp_delta[4] * temp_delta[2];
  delta[4] = temp_delta[0] * temp_delta[1] * temp_delta[5];
  delta[5] = temp_delta[3] * temp_delta[1] * temp_delta[5];
  delta[6] = temp_delta[0] * temp_delta[4] * temp_delta[5];
  delta[7] = temp_delta[3] * temp_delta[4] * temp_delta[5];

  int x = my_left[0];
  int y = my_left[1];
  int z = my_left[2];

  node_index[0] = x%para.dim_x     + para.dim_x*(y%para.dim_y)     + para.dim_x*para.dim_y*(z%para.dim_z);
  node_index[1] = (x+1)%para.dim_x + para.dim_x*(y%para.dim_y)     + para.dim_x*para.dim_y*(z%para.dim_z);
  node_index[2] = x%para.dim_x     + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*(z%para.dim_z);
  node_index[3] = (x+1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*(z%para.dim_z);
  node_index[4] = x%para.dim_x     + para.dim_x*(y%para.dim_y)     + para.dim_x*para.dim_y*((z+1)%para.dim_z);
  node_index[5] = (x+1)%para.dim_x + para.dim_x*(y%para.dim_y)     + para.dim_x*para.dim_y*((z+1)%para.dim_z);
  node_index[6] = x%para.dim_x     + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z);
  node_index[7] = (x+1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z);
  #pragma unroll
  for(int i=0; i<8; ++i){
    calc_mode(mode, n_a, node_index[i]);
    Rho = mode[0] + para.rho*para.agrid*para.agrid*para.agrid;	
    interpolated_u1 += delta[i]*mode[1]/(Rho);
    interpolated_u2 += delta[i]*mode[2]/(Rho);
    interpolated_u3 += delta[i]*mode[3]/(Rho);
  }


  /** calculate viscous force
   * take care to rescale velocities with time_step and transform to MD units
   * (Eq. (9) Ahlrichs and Duenweg, JCP 111(17):8225 (1999)) */
#ifdef LB_ELECTROHYDRODYNAMICS
  particle_force[part_index].f[0] = - para.friction * (particle_data[part_index].v[0]/para.time_step - interpolated_u1*para.agrid/para.tau - particle_data[part_index].mu_E[0]);
  particle_force[part_index].f[1] = - para.friction * (particle_data[part_index].v[1]/para.time_step - interpolated_u2*para.agrid/para.tau - particle_data[part_index].mu_E[1]);
  particle_force[part_index].f[2] = - para.friction * (particle_data[part_index].v[2]/para.time_step - interpolated_u3*para.agrid/para.tau - particle_data[part_index].mu_E[2]);
#else
  particle_force[part_index].f[0] = - para.friction * (particle_data[part_index].v[0]/para.time_step - interpolated_u1*para.agrid/para.tau);
  particle_force[part_index].f[1] = - para.friction * (particle_data[part_index].v[1]/para.time_step - interpolated_u2*para.agrid/para.tau);
  particle_force[part_index].f[2] = - para.friction * (particle_data[part_index].v[2]/para.time_step - interpolated_u3*para.agrid/para.tau);
#endif
  /** add stochastik force of zero mean (Ahlrichs, Duennweg equ. 15)*/
#ifdef GAUSSRANDOM
  gaussian_random(rn_part);
  particle_force[part_index].f[0] += para.lb_coupl_pref2*rn_part->randomnr[0];
  particle_force[part_index].f[1] += para.lb_coupl_pref2*rn_part->randomnr[1];
  gaussian_random(rn_part);
  particle_force[part_index].f[2] += para.lb_coupl_pref2*rn_part->randomnr[0];
#else
  random_01(rn_part);
  particle_force[part_index].f[0] += para.lb_coupl_pref*(rn_part->randomnr[0]-0.5f);
  particle_force[part_index].f[1] += para.lb_coupl_pref*(rn_part->randomnr[1]-0.5f);
  random_01(rn_part);
  particle_force[part_index].f[2] += para.lb_coupl_pref*(rn_part->randomnr[0]-0.5f);
#endif	  
  /** delta_j for transform momentum transfer to lattice units which is done in calc_node_force
  (Eq. (12) Ahlrichs and Duenweg, JCP 111(17):8225 (1999)) */
  delta_j[0] = - particle_force[part_index].f[0]*para.time_step*para.tau/para.agrid;
  delta_j[1] = - particle_force[part_index].f[1]*para.time_step*para.tau/para.agrid;
  delta_j[2] = - particle_force[part_index].f[2]*para.time_step*para.tau/para.agrid;  	
															  																	  
}

/**calcutlation of the node force caused by the particles, with atomicadd due to avoiding race conditions 
	(Eq. (14) Ahlrichs and Duenweg, JCP 111(17):8225 (1999))
 * @param *delta		Pointer for the weighting of particle position (Input)
 * @param *delta_j		Pointer for the weighting of particle momentum (Input)
 * @param node_index		node index around (8) particle (Input)
 * @param node_f    		Pointer to the node force (Output).
*/
__device__ void calc_node_force(float *delta, float *delta_j, unsigned int *node_index, LB_node_force_gpu node_f){

#if 1
  atomicadd(&(node_f.force[0*para.number_of_nodes + node_index[0]]), (delta[0]*delta_j[0]));
  atomicadd(&(node_f.force[1*para.number_of_nodes + node_index[0]]), (delta[0]*delta_j[1]));
  atomicadd(&(node_f.force[2*para.number_of_nodes + node_index[0]]), (delta[0]*delta_j[2]));

  atomicadd(&(node_f.force[0*para.number_of_nodes + node_index[1]]), (delta[1]*delta_j[0]));
  atomicadd(&(node_f.force[1*para.number_of_nodes + node_index[1]]), (delta[1]*delta_j[1]));
  atomicadd(&(node_f.force[2*para.number_of_nodes + node_index[1]]), (delta[1]*delta_j[2]));

  atomicadd(&(node_f.force[0*para.number_of_nodes + node_index[2]]), (delta[2]*delta_j[0]));
  atomicadd(&(node_f.force[1*para.number_of_nodes + node_index[2]]), (delta[2]*delta_j[1]));
  atomicadd(&(node_f.force[2*para.number_of_nodes + node_index[2]]), (delta[2]*delta_j[2]));

  atomicadd(&(node_f.force[0*para.number_of_nodes + node_index[3]]), (delta[3]*delta_j[0]));
  atomicadd(&(node_f.force[1*para.number_of_nodes + node_index[3]]), (delta[3]*delta_j[1]));
  atomicadd(&(node_f.force[2*para.number_of_nodes + node_index[3]]), (delta[3]*delta_j[2]));

  atomicadd(&(node_f.force[0*para.number_of_nodes + node_index[4]]), (delta[4]*delta_j[0]));
  atomicadd(&(node_f.force[1*para.number_of_nodes + node_index[4]]), (delta[4]*delta_j[1]));
  atomicadd(&(node_f.force[2*para.number_of_nodes + node_index[4]]), (delta[4]*delta_j[2]));

  atomicadd(&(node_f.force[0*para.number_of_nodes + node_index[5]]), (delta[5]*delta_j[0]));
  atomicadd(&(node_f.force[1*para.number_of_nodes + node_index[5]]), (delta[5]*delta_j[1]));
  atomicadd(&(node_f.force[2*para.number_of_nodes + node_index[5]]), (delta[5]*delta_j[2]));

  atomicadd(&(node_f.force[0*para.number_of_nodes + node_index[6]]), (delta[6]*delta_j[0]));
  atomicadd(&(node_f.force[1*para.number_of_nodes + node_index[6]]), (delta[6]*delta_j[1]));
  atomicadd(&(node_f.force[2*para.number_of_nodes + node_index[6]]), (delta[6]*delta_j[2]));

  atomicadd(&(node_f.force[0*para.number_of_nodes + node_index[7]]), (delta[7]*delta_j[0]));
  atomicadd(&(node_f.force[1*para.number_of_nodes + node_index[7]]), (delta[7]*delta_j[1]));
  atomicadd(&(node_f.force[2*para.number_of_nodes + node_index[7]]), (delta[7]*delta_j[2]));
#endif
}
/*********************************************************/
/** \name System setup and Kernel funktions */
/*********************************************************/
/**kernel to calculate local populations from hydrodynamic fields given by the tcl values.
 * The mapping is given in terms of the equilibrium distribution.
 *
 * Eq. (2.15) Ladd, J. Fluid Mech. 271, 295-309 (1994)
 * Eq. (4) in Berk Usta, Ladd and Butler, JCP 122, 094902 (2005)
 *
 * @param n_a		 Pointer to the lattice site (Input).
 * @param *gpu_check additional check if gpu kernel are executed(Input).
*/
__global__ void calc_n_equilibrium(LB_nodes_gpu n_a, int *gpu_check) {

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){

    /** default values for fields in lattice units */
    gpu_check[0] = 1;

    float Rho = para.rho*para.agrid*para.agrid*para.agrid;
    float v[3] = { 0.0f, 0.0f, 0.0f };
    float pi[6] = { Rho*c_sound_sq, 0.0f, Rho*c_sound_sq, 0.0f, 0.0f, Rho*c_sound_sq };

    float rhoc_sq = Rho*c_sound_sq;
    float avg_rho = para.rho*para.agrid*para.agrid*para.agrid;
    float local_rho, local_j[3], *local_pi, trace;

    local_rho  = Rho;

    local_j[0] = Rho * v[0];
    local_j[1] = Rho * v[1];
    local_j[2] = Rho * v[2];

    local_pi = pi;

    /** reduce the pressure tensor to the part needed here */
    local_pi[0] -= rhoc_sq;
    local_pi[2] -= rhoc_sq;
    local_pi[5] -= rhoc_sq;

    trace = local_pi[0] + local_pi[2] + local_pi[5];

    float rho_times_coeff;
    float tmp1,tmp2;

    /** update the q=0 sublattice */
    n_a.vd[0*para.number_of_nodes + index] = 1.f/3.f * (local_rho-avg_rho) - 1.f/2.f*trace;

    /** update the q=1 sublattice */
    rho_times_coeff = 1.f/18.f * (local_rho-avg_rho);

    n_a.vd[1*para.number_of_nodes + index] = rho_times_coeff + 1.f/6.f*local_j[0] + 1.f/4.f*local_pi[0] - 1.f/12.f*trace;
    n_a.vd[2*para.number_of_nodes + index] = rho_times_coeff - 1.f/6.f*local_j[0] + 1.f/4.f*local_pi[0] - 1.f/12.f*trace;
    n_a.vd[3*para.number_of_nodes + index] = rho_times_coeff + 1.f/6.f*local_j[1] + 1.f/4.f*local_pi[2] - 1.f/12.f*trace;
    n_a.vd[4*para.number_of_nodes + index] = rho_times_coeff - 1.f/6.f*local_j[1] + 1.f/4.f*local_pi[2] - 1.f/12.f*trace;
    n_a.vd[5*para.number_of_nodes + index] = rho_times_coeff + 1.f/6.f*local_j[2] + 1.f/4.f*local_pi[5] - 1.f/12.f*trace;
    n_a.vd[6*para.number_of_nodes + index] = rho_times_coeff - 1.f/6.f*local_j[2] + 1.f/4.f*local_pi[5] - 1.f/12.f*trace;

    /** update the q=2 sublattice */
    rho_times_coeff = 1.f/36.f * (local_rho-avg_rho);

    tmp1 = local_pi[0] + local_pi[2];
    tmp2 = 2.0f*local_pi[1];
    n_a.vd[7*para.number_of_nodes + index]  = rho_times_coeff + 1.f/12.f*(local_j[0]+local_j[1]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[8*para.number_of_nodes + index]  = rho_times_coeff - 1.f/12.f*(local_j[0]+local_j[1]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[9*para.number_of_nodes + index]  = rho_times_coeff + 1.f/12.f*(local_j[0]-local_j[1]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;
    n_a.vd[10*para.number_of_nodes + index] = rho_times_coeff - 1.f/12.f*(local_j[0]-local_j[1]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;

    tmp1 = local_pi[0] + local_pi[5];
    tmp2 = 2.0f*local_pi[3];

    n_a.vd[11*para.number_of_nodes + index] = rho_times_coeff + 1.f/12.f*(local_j[0]+local_j[2]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[12*para.number_of_nodes + index] = rho_times_coeff - 1.f/12.f*(local_j[0]+local_j[2]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[13*para.number_of_nodes + index] = rho_times_coeff + 1.f/12.f*(local_j[0]-local_j[2]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;
    n_a.vd[14*para.number_of_nodes + index] = rho_times_coeff - 1.f/12.f*(local_j[0]-local_j[2]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;

    tmp1 = local_pi[2] + local_pi[5];
    tmp2 = 2.0f*local_pi[4];

    n_a.vd[15*para.number_of_nodes + index] = rho_times_coeff + 1.f/12.f*(local_j[1]+local_j[2]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[16*para.number_of_nodes + index] = rho_times_coeff - 1.f/12.f*(local_j[1]+local_j[2]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[17*para.number_of_nodes + index] = rho_times_coeff + 1.f/12.f*(local_j[1]-local_j[2]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;
    n_a.vd[18*para.number_of_nodes + index] = rho_times_coeff - 1.f/12.f*(local_j[1]-local_j[2]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;

    /**set different seed for randomgen on every node */
    n_a.seed[index] = para.your_seed + index;
  }
}
/** kernel to calculate local populations from hydrodynamic fields
 * from given flow field velocities.  The mapping is given in terms of
 * the equilibrium distribution.
 *
 * Eq. (2.15) Ladd, J. Fluid Mech. 271, 295-309 (1994)
 * Eq. (4) in Berk Usta, Ladd and Butler, JCP 122, 094902 (2005)
 *
 * @param n_a		   the current nodes array (double buffering!)
 * @param single_nodeindex the node to set the velocity for
 * @param velocity         the velocity to set
 */
__global__ void set_u_equilibrium(LB_nodes_gpu n_a, int single_nodeindex,float *velocity) {

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index == 0){

    /** default values for fields in lattice units */
    float mode[19];
    calc_mode(mode, n_a, single_nodeindex);
    float Rho = mode[0] + para.rho*para.agrid*para.agrid*para.agrid;

    float v[3];
    v[0] = velocity[0];
    v[1] = velocity[1];
    v[2] = velocity[2];

    float pi[6] = { Rho*c_sound_sq, 0.0f, Rho*c_sound_sq, 0.0f, 0.0f, Rho*c_sound_sq };

    float rhoc_sq = Rho*c_sound_sq;
    float avg_rho = para.rho*para.agrid*para.agrid*para.agrid;
    float local_rho, local_j[3], *local_pi, trace;

    local_rho  = Rho;

    local_j[0] = Rho * v[0];
    local_j[1] = Rho * v[1];
    local_j[2] = Rho * v[2];

    local_pi = pi;

    /** reduce the pressure tensor to the part needed here */
    local_pi[0] -= rhoc_sq;
    local_pi[2] -= rhoc_sq;
    local_pi[5] -= rhoc_sq;

    trace = local_pi[0] + local_pi[2] + local_pi[5];

    float rho_times_coeff;
    float tmp1,tmp2;

    /** update the q=0 sublattice */
    n_a.vd[0*para.number_of_nodes + index] = 1.f/3.f * (local_rho-avg_rho) - 1.f/2.f*trace;

    /** update the q=1 sublattice */
    rho_times_coeff = 1.f/18.f * (local_rho-avg_rho);

    n_a.vd[1*para.number_of_nodes + single_nodeindex] = rho_times_coeff + 1.f/6.f*local_j[0] + 1.f/4.f*local_pi[0] - 1.f/12.f*trace;
    n_a.vd[2*para.number_of_nodes + single_nodeindex] = rho_times_coeff - 1.f/6.f*local_j[0] + 1.f/4.f*local_pi[0] - 1.f/12.f*trace;
    n_a.vd[3*para.number_of_nodes + single_nodeindex] = rho_times_coeff + 1.f/6.f*local_j[1] + 1.f/4.f*local_pi[2] - 1.f/12.f*trace;
    n_a.vd[4*para.number_of_nodes + single_nodeindex] = rho_times_coeff - 1.f/6.f*local_j[1] + 1.f/4.f*local_pi[2] - 1.f/12.f*trace;
    n_a.vd[5*para.number_of_nodes + single_nodeindex] = rho_times_coeff + 1.f/6.f*local_j[2] + 1.f/4.f*local_pi[5] - 1.f/12.f*trace;
    n_a.vd[6*para.number_of_nodes + single_nodeindex] = rho_times_coeff - 1.f/6.f*local_j[2] + 1.f/4.f*local_pi[5] - 1.f/12.f*trace;

    /** update the q=2 sublattice */
    rho_times_coeff = 1.f/36.f * (local_rho-avg_rho);

    tmp1 = local_pi[0] + local_pi[2];
    tmp2 = 2.0f*local_pi[1];
    n_a.vd[7*para.number_of_nodes + single_nodeindex]  = rho_times_coeff + 1.f/12.f*(local_j[0]+local_j[1]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[8*para.number_of_nodes + single_nodeindex]  = rho_times_coeff - 1.f/12.f*(local_j[0]+local_j[1]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[9*para.number_of_nodes + single_nodeindex]  = rho_times_coeff + 1.f/12.f*(local_j[0]-local_j[1]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;
    n_a.vd[10*para.number_of_nodes + single_nodeindex] = rho_times_coeff - 1.f/12.f*(local_j[0]-local_j[1]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;

    tmp1 = local_pi[0] + local_pi[5];
    tmp2 = 2.0f*local_pi[3];

    n_a.vd[11*para.number_of_nodes + single_nodeindex] = rho_times_coeff + 1.f/12.f*(local_j[0]+local_j[2]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[12*para.number_of_nodes + single_nodeindex] = rho_times_coeff - 1.f/12.f*(local_j[0]+local_j[2]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[13*para.number_of_nodes + single_nodeindex] = rho_times_coeff + 1.f/12.f*(local_j[0]-local_j[2]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;
    n_a.vd[14*para.number_of_nodes + single_nodeindex] = rho_times_coeff - 1.f/12.f*(local_j[0]-local_j[2]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;

    tmp1 = local_pi[2] + local_pi[5];
    tmp2 = 2.0f*local_pi[4];

    n_a.vd[15*para.number_of_nodes + single_nodeindex] = rho_times_coeff + 1.f/12.f*(local_j[1]+local_j[2]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[16*para.number_of_nodes + single_nodeindex] = rho_times_coeff - 1.f/12.f*(local_j[1]+local_j[2]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[17*para.number_of_nodes + single_nodeindex] = rho_times_coeff + 1.f/12.f*(local_j[1]-local_j[2]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;
    n_a.vd[18*para.number_of_nodes + single_nodeindex] = rho_times_coeff - 1.f/12.f*(local_j[1]-local_j[2]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;

  }
}
/** kernel for the initalisation of the particle force array
 * @param *particle_force	Pointer to local particle force (Output)
 * @param *part			Pointer to the particle rn seed storearray (Output)
*/
__global__ void init_particle_force(LB_particle_force_gpu *particle_force, LB_particle_seed_gpu *part){
	
  unsigned int part_index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
	
  if(part_index<para.number_of_particles){
    particle_force[part_index].f[0] = 0.0f;
    particle_force[part_index].f[1] = 0.0f;
    particle_force[part_index].f[2] = 0.0f;
	
    part[part_index].seed = para.your_seed + part_index;
  }
			
}

/** kernel for the initalisation of the partikel force array
 * @param *particle_force	pointer to local particle force (Input)
*/
__global__ void reset_particle_force(LB_particle_force_gpu *particle_force){
	
  unsigned int part_index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
	
  if(part_index<para.number_of_particles){
    particle_force[part_index].f[0] = 0.0f;
    particle_force[part_index].f[1] = 0.0f;
    particle_force[part_index].f[2] = 0.0f;
  }			
}

/** (re-)initialization of the node force / set up of external force in lb units
 * @param node_f		Pointer to local node force (Input)
*/
__global__ void reinit_node_force(LB_node_force_gpu node_f){

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
#ifdef EXTERNAL_FORCE
    if(para.external_force){
      node_f.force[0*para.number_of_nodes + index] = para.ext_force[0]*powf(para.agrid,4)*para.tau*para.tau;
      node_f.force[1*para.number_of_nodes + index] = para.ext_force[1]*powf(para.agrid,4)*para.tau*para.tau;
      node_f.force[2*para.number_of_nodes + index] = para.ext_force[2]*powf(para.agrid,4)*para.tau*para.tau;
    }
    else{
      node_f.force[0*para.number_of_nodes + index] = 0.0f;
      node_f.force[1*para.number_of_nodes + index] = 0.0f;
      node_f.force[2*para.number_of_nodes + index] = 0.0f;
    }
#else
    node_f.force[0*para.number_of_nodes + index] = 0.0f;
    node_f.force[1*para.number_of_nodes + index] = 0.0f;
    node_f.force[2*para.number_of_nodes + index] = 0.0f;
#endif
  }
}

/**set the boundary flag for all boundary nodes
 * @param *boundindex	     	Pointer to the 1d index of the boundnode (Input)
 * @param number_of_boundnodes	The number of boundary nodes
 * @param n_a			Pointer to local node residing in array a (Input)
 * @param n_b			Pointer to local node residing in array b (Input)
*/
__global__ void init_boundaries(int *boundindex, int number_of_boundnodes, LB_nodes_gpu n_a, LB_nodes_gpu n_b){

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<number_of_boundnodes){
    n_a.boundary[boundindex[index]] = n_b.boundary[boundindex[index]] = 1;
  }	
}

/**reset the boundary flag of every node
 * @param n_a		Pointer to local node residing in array a (Input)
 * @param n_b		Pointer to local node residing in array b (Input)	
*/
__global__ void reset_boundaries(LB_nodes_gpu n_a, LB_nodes_gpu n_b){

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
    n_a.boundary[index] = n_b.boundary[index] = 0;
  }
}

/** integrationstep of the lb-fluid-solver
 * @param n_a		Pointer to local node residing in array a (Input)
 * @param n_b		Pointer to local node residing in array b (Input)
 * @param *d_v		Pointer to local device values (Input)
 * @param node_f	Pointer to local node force (Input)
*/
__global__ void integrate(LB_nodes_gpu n_a, LB_nodes_gpu n_b, LB_values_gpu *d_v, LB_node_force_gpu node_f){
    
  /**every node is connected to a thread via the index*/
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
  /**the 19 moments (modes) are only temporary register values */
  float mode[19];
  LB_randomnr_gpu rng;

  if(index<para.number_of_nodes){
    /** storing the seed into a register value*/
    rng.seed = n_a.seed[index];
    /**calc_m_from_n*/
    calc_m_from_n(n_a, index, mode);
    /**lb_relax_modes*/
    relax_modes(mode, index, node_f);
    /**lb_thermalize_modes */
    if (para.fluct) thermalize_modes(mode, index, &rng);
#ifdef EXTERNAL_FORCES
    /**if external force is used apply node force */
    apply_forces(index, mode, node_f);
#else
    /**if partcles are used apply node forces*/
    if (para.number_of_particles) apply_forces(index, mode, node_f); 
#endif
    /**lb_calc_n_from_modes_push*/
    normalize_modes(mode);
    /**calc of velocity densities and streaming with pbc*/
    calc_n_from_modes_push(n_b, mode, index);
    /** rewriting the seed back to the global memory*/
    n_b.seed[index] = rng.seed;
  }  
}

/** part interaction kernel
 * @param n_a				Pointer to local node residing in array a (Input)
 * @param *particle_data		Pointer to the particle position and velocity (Input)
 * @param *particle_force		Pointer to the particle force (Input)
 * @param *part				Pointer to the rn array of the particles (Input)
 * @param node_f			Pointer to local node force (Input)
*/
__global__ void calc_fluid_particle_ia(LB_nodes_gpu n_a, LB_particle_gpu *particle_data, LB_particle_force_gpu *particle_force, LB_node_force_gpu node_f, LB_particle_seed_gpu *part){
	
  unsigned int part_index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int node_index[8];
  float delta[8];
  float delta_j[3];
  LB_randomnr_gpu rng_part;
	
  if(part_index<para.number_of_particles){

    rng_part.seed = part[part_index].seed;
    /**calc of the force which act on the particle */
    calc_viscous_force(n_a, delta, particle_data, particle_force, part_index, &rng_part, delta_j, node_index);
    /**calc of the force which acts back to the fluid node */
    calc_node_force(delta, delta_j, node_index, node_f);
    part[part_index].seed = rng_part.seed;		
  }
}

/**Bounce back boundary read kernel
 * @param n_a					Pointer to local node residing in array a (Input)
 * @param n_b					Pointer to local node residing in array b (Input)
*/
__global__ void bb_read(LB_nodes_gpu n_a, LB_nodes_gpu n_b){

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
    bounce_back_read(n_b, n_a, index);
  }
}

/**Bounce back boundary write kernel
 * @param n_a					Pointer to local node residing in array a (Input)
 * @param n_b					Pointer to local node residing in array b (Input)
*/
__global__ void bb_write(LB_nodes_gpu n_a, LB_nodes_gpu n_b){

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
    bounce_back_write(n_b, n_a, index);
  }
}

/** get physical values of the nodes (density, velocity, ...)
 * @param n_a		Pointer to local node residing in array a (Input)
 * @param *d_v		Pointer to local device values (Input)
*/
__global__ void values(LB_nodes_gpu n_a, LB_values_gpu *d_v){

  float mode[19];
  unsigned int singlenode = 0;
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
    calc_mode(mode, n_a, index);
    calc_values(n_a, mode, d_v, index, singlenode);
  }
}

/** get boundary flags
 *  @param n_a	              Pointer to local node residing in array a (Input)
 *  @param device_bound_array Pointer to local device values (Input)
 */
__global__ void lb_get_boundaries(LB_nodes_gpu n_a, unsigned int *device_bound_array){

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
   device_bound_array[index] = n_a.boundary[index];
  }
}

/**set extern force on single nodes kernel
 * @param n_extern_nodeforces		number of nodes (Input)
 * @param *extern_nodeforces		Pointer to extern node force array (Input)
 * @param node_f			node force struct (Output)
*/
__global__ void init_extern_nodeforces(int n_extern_nodeforces, LB_extern_nodeforce_gpu *extern_nodeforces, LB_node_force_gpu node_f){

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<n_extern_nodeforces){
    node_f.force[0*para.number_of_nodes + extern_nodeforces[index].index] = extern_nodeforces[index].force[0]*powf(para.agrid,4)*para.tau*para.tau;
    node_f.force[1*para.number_of_nodes + extern_nodeforces[index].index] = extern_nodeforces[index].force[1]*powf(para.agrid,4)*para.tau*para.tau;
    node_f.force[2*para.number_of_nodes + extern_nodeforces[index].index] = extern_nodeforces[index].force[2]*powf(para.agrid,4)*para.tau*para.tau;
  }
}

/**print single node values kernel
 * @param single_nodeindex		index of the node (Input)
 * @param *d_p_v			Pointer to result storage array (Input)
 * @param n_a				Pointer to local node residing in array a (Input)
*/
__global__ void lb_print_node(int single_nodeindex, LB_values_gpu *d_p_v, LB_nodes_gpu n_a){
	
  float mode[19];
  unsigned int singlenode = 1;
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index == 0){
    calc_mode(mode, n_a, single_nodeindex);
    calc_values(n_a, mode, d_p_v, single_nodeindex, singlenode);
  }	
}
/**calculate mass of the hole fluid kernel
 * @param *sum				Pointer to result storage value (Output)
 * @param n_a				Pointer to local node residing in array a (Input)
*/
__global__ void calc_mass(LB_nodes_gpu n_a, float *sum) {
  float mode[1];

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
    calc_mode(mode, n_a, index);
    float Rho = mode[0] + para.rho*para.agrid*para.agrid*para.agrid;
    //if(n_a.boundary[index]){
      //mode[0] = 0.f;
    //}
    atomicadd(&(sum[0]), Rho);
  }
}
/**calculate momentum of the hole fluid kernel
 * @param node_f			node force struct (Input)
 * @param *sum				Pointer to result storage value (Output)
 * @param n_a				Pointer to local node residing in array a (Input)
*/
__global__ void momentum(LB_nodes_gpu n_a, float *sum, LB_node_force_gpu node_f) {
  float mode[4];

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
    calc_mode(mode, n_a, index);
    if(n_a.boundary[index]){
      mode[1] = mode[2] = mode[3] = 0.f;
    }
    atomicadd(&(sum[0]), mode[1]+node_f.force[0*para.number_of_nodes + index]);
    atomicadd(&(sum[1]), mode[2]+node_f.force[1*para.number_of_nodes + index]);
    atomicadd(&(sum[2]), mode[3]+node_f.force[2*para.number_of_nodes + index]);
  }
}

/**calculate temperature of the fluid kernel
 * @param *cpu_jsquared			Pointer to result storage value (Output)
 * @param n_a				Pointer to local node residing in array a (Input)
*/
__global__ void temperature(LB_nodes_gpu n_a, float *cpu_jsquared) {
  float mode[4];
  float jsquared = 0.f;
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
    calc_mode(mode, n_a, index);
    if(n_a.boundary[index]){
      jsquared = 0.f;
    }
    else{
      jsquared = mode[1]*mode[1]+mode[2]*mode[2]+mode[3]*mode[3];
    }
    atomicadd(cpu_jsquared, jsquared);
  }
}
/**print single node boundary flag
 * @param single_nodeindex		index of the node (Input)
 * @param *device_flag			Pointer to result storage array (Input)
 * @param n_a				Pointer to local node residing in array a (Input)
*/
__global__ void lb_get_boundary_flag(int single_nodeindex, unsigned int *device_flag, LB_nodes_gpu n_a){
	
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index == 0){
    device_flag[0] = n_a.boundary[single_nodeindex];
  }	
}
/**erroroutput for memory allocation and memory copy 
 * @param err cuda error code
 * @param *file .cu file were the error took place
 * @param line line of the file were the error took place
*/
void _cuda_safe_mem(hipError_t err, char *file, unsigned int line){
    if( hipSuccess != err) {                                             
      fprintf(stderr, "Could not allocate gpu memory at %s:%u.\n", file, line);
      printf("CUDA error: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
}
#define cuda_safe_mem(a) _cuda_safe_mem((a), __FILE__, __LINE__)
#define KERNELCALL(_f, _a, _b, _params) \
_f<<<_a, _b, 0, stream[0]>>>_params; \
_err=hipGetLastError(); \
if (_err!=hipSuccess){ \
  printf("CUDA error: %s\n", hipGetErrorString(_err)); \
  fprintf(stderr, "error calling %s with #thpb %d in %s:%u\n", #_f, _b, __FILE__, __LINE__); \
  exit(EXIT_FAILURE); \
}
/*********************************************************/
/** \name Host functions to setup and call kernels */
/*********************************************************/
/**********************************************************************/
/* Host funktions to setup and call kernels*/
/**********************************************************************/

/**initialization for the lb gpu fluid called from host
 * @param *lbpar_gpu	Pointer to parameters to setup the lb field
*/
void lb_init_GPU(LB_parameters_gpu *lbpar_gpu){

  /** Allocate structs in device memory*/
  size_of_values = lbpar_gpu->number_of_nodes * sizeof(LB_values_gpu);
  size_of_forces = lbpar_gpu->number_of_particles * sizeof(LB_particle_force_gpu);
  size_of_positions = lbpar_gpu->number_of_particles * sizeof(LB_particle_gpu);
  size_of_seed = lbpar_gpu->number_of_particles * sizeof(LB_particle_seed_gpu);

  cuda_safe_mem(hipMalloc((void**)&device_values, size_of_values));


  cuda_safe_mem(hipMalloc((void**)&nodes_a.vd, lbpar_gpu->number_of_nodes * 19 * sizeof(float)));
  cuda_safe_mem(hipMalloc((void**)&nodes_b.vd, lbpar_gpu->number_of_nodes * 19 * sizeof(float)));                                           

  cuda_safe_mem(hipMalloc((void**)&nodes_a.seed, lbpar_gpu->number_of_nodes * sizeof(unsigned int)));
  cuda_safe_mem(hipMalloc((void**)&nodes_a.boundary, lbpar_gpu->number_of_nodes * sizeof(unsigned int)));
  cuda_safe_mem(hipMalloc((void**)&nodes_b.seed, lbpar_gpu->number_of_nodes * sizeof(unsigned int)));
  cuda_safe_mem(hipMalloc((void**)&nodes_b.boundary, lbpar_gpu->number_of_nodes * sizeof(unsigned int)));

  cuda_safe_mem(hipMalloc((void**)&node_f.force, lbpar_gpu->number_of_nodes * 3 * sizeof(float)));
//maybe coalesced alloc  
  cuda_safe_mem(hipMalloc((void**)&particle_force, size_of_forces));
  cuda_safe_mem(hipMalloc((void**)&particle_data, size_of_positions));
	
  cuda_safe_mem(hipMalloc((void**)&part, size_of_seed));
	
  /**write parameters in const memory*/
  cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu, sizeof(LB_parameters_gpu)));
  /**check flag if lb gpu init works*/
  cuda_safe_mem(hipMalloc((void**)&gpu_check, sizeof(int)));
  h_gpu_check = (int*)malloc(sizeof(int));

  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu->number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  hipStreamCreate(&stream[0]);
  /** values for the particle kernel */
  int threads_per_block_particles = 64;
  int blocks_per_grid_particles_y = 4;
  int blocks_per_grid_particles_x = (lbpar_gpu->number_of_particles + threads_per_block_particles * blocks_per_grid_particles_y - 1)/(threads_per_block_particles * blocks_per_grid_particles_y);
  dim3 dim_grid_particles = make_uint3(blocks_per_grid_particles_x, blocks_per_grid_particles_y, 1);

  KERNELCALL(reset_boundaries, dim_grid, threads_per_block, (nodes_a, nodes_b));

  /** calc of veloctiydensities from given parameters and initialize the Node_Force array with zero */
  KERNELCALL(calc_n_equilibrium, dim_grid, threads_per_block, (nodes_a, gpu_check));	
  /** init part forces with zero*/
  if(lbpar_gpu->number_of_particles) KERNELCALL(init_particle_force, dim_grid_particles, threads_per_block_particles, (particle_force, part));
  KERNELCALL(reinit_node_force, dim_grid, threads_per_block, (node_f));

  intflag = 1;
  current_nodes = &nodes_a;
  h_gpu_check[0] = 0;
  cuda_safe_mem(hipMemcpy(h_gpu_check, gpu_check, sizeof(int), hipMemcpyDeviceToHost));
//fprintf(stderr, "initialization of lb gpu code %i\n", lbpar_gpu->number_of_nodes);
  hipDeviceSynchronize();
  if(!h_gpu_check[0]){
    fprintf(stderr, "initialization of lb gpu code failed! \n");
    errexit();	
  }	
}
/** reinitialization for the lb gpu fluid called from host
 * @param *lbpar_gpu	Pointer to parameters to setup the lb field
*/
void lb_reinit_GPU(LB_parameters_gpu *lbpar_gpu){

  /**write parameters in const memory*/
  cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu, sizeof(LB_parameters_gpu)));
  
  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu->number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  /** calc of veloctiydensities from given parameters and initialize the Node_Force array with zero */
  KERNELCALL(calc_n_equilibrium, dim_grid, threads_per_block, (nodes_a, gpu_check));
}

/**setup and call particle reallocation from the host
 * @param *lbpar_gpu	Pointer to parameters to setup the lb field
 * @param **host_data	Pointer to host information data
*/
void lb_realloc_particle_GPU(LB_parameters_gpu *lbpar_gpu, LB_particle_gpu **host_data){

  /** Allocate struct for particle positions */
  size_of_forces = lbpar_gpu->number_of_particles * sizeof(LB_particle_force_gpu);
  size_of_positions = lbpar_gpu->number_of_particles * sizeof(LB_particle_gpu);
  size_of_seed = lbpar_gpu->number_of_particles * sizeof(LB_particle_seed_gpu);

  hipHostFree(*host_data);

#if !defined __CUDA_ARCH__ || __CUDA_ARCH__ >= 200
  /**pinned memory mode - use special function to get OS-pinned memory*/
  hipHostAlloc((void**)host_data, size_of_positions, hipHostMallocWriteCombined);
#else
  hipHostMalloc((void**)host_data, size_of_positions);
#endif

  hipFree(particle_force);
  hipFree(particle_data);
  hipFree(part);

  cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu, sizeof(LB_parameters_gpu)));
 
  cuda_safe_mem(hipMalloc((void**)&particle_force, size_of_forces));
  cuda_safe_mem(hipMalloc((void**)&particle_data, size_of_positions));
  cuda_safe_mem(hipMalloc((void**)&part, size_of_seed));

  /** values for the particle kernel */
  int threads_per_block_particles = 64;
  int blocks_per_grid_particles_y = 4;
  int blocks_per_grid_particles_x = (lbpar_gpu->number_of_particles + threads_per_block_particles * blocks_per_grid_particles_y - 1)/(threads_per_block_particles * blocks_per_grid_particles_y);
  dim3 dim_grid_particles = make_uint3(blocks_per_grid_particles_x, blocks_per_grid_particles_y, 1);

  if(lbpar_gpu->number_of_particles) KERNELCALL(init_particle_force, dim_grid_particles, threads_per_block_particles, (particle_force, part));	
}
#ifdef LB_BOUNDARIES_GPU
/**setup and call boundaries from the host
 * @param *host_boundindex		Pointer to the host bound index
 * @param number_of_boundnodes	number of boundnodes
*/
void lb_init_boundaries_GPU(int number_of_boundnodes, int *host_boundindex){

  size_of_boundindex = number_of_boundnodes*sizeof(int);
  cuda_safe_mem(hipMalloc((void**)&boundindex, size_of_boundindex));
  hipMemcpy(boundindex, host_boundindex, size_of_boundindex, hipMemcpyHostToDevice);
  
  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(reset_boundaries, dim_grid, threads_per_block, (nodes_a, nodes_b));

  if (n_lb_boundaries == 0) {
    hipDeviceSynchronize();
    return;
  }
  if(number_of_boundnodes == 0){
    fprintf(stderr, "WARNING: boundary cmd executed but no boundary node found!\n");
  }
  else{
    int threads_per_block_bound = 64;
    int blocks_per_grid_bound_y = 4;
    int blocks_per_grid_bound_x = (number_of_boundnodes + threads_per_block_bound * blocks_per_grid_bound_y - 1) /(threads_per_block_bound * blocks_per_grid_bound_y);
    dim3 dim_grid_bound = make_uint3(blocks_per_grid_bound_x, blocks_per_grid_bound_y, 1);

    KERNELCALL(init_boundaries, dim_grid_bound, threads_per_block_bound, (boundindex, number_of_boundnodes, nodes_a, nodes_b));
  }

  hipDeviceSynchronize();
}
#endif
/**setup and call extern single node force initialization from the host
 * @param *lbpar_gpu				Pointer to host parameter struct
*/
void lb_reinit_extern_nodeforce_GPU(LB_parameters_gpu *lbpar_gpu){

  cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu, sizeof(LB_parameters_gpu))); 

  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu->number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(reinit_node_force, dim_grid, threads_per_block, (node_f));

}
/**setup and call extern single node force initialization from the host
 * @param n_extern_nodeforces			number of nodes on which the external force has to be applied
 * @param *host_extern_nodeforces		Pointer to the host extern node forces
 * @param *lbpar_gpu				Pointer to host parameter struct
*/
void lb_init_extern_nodeforces_GPU(int n_extern_nodeforces, LB_extern_nodeforce_gpu *host_extern_nodeforces, LB_parameters_gpu *lbpar_gpu){

  size_of_extern_nodeforces = n_extern_nodeforces*sizeof(LB_extern_nodeforce_gpu);
  cuda_safe_mem(hipMalloc((void**)&extern_nodeforces, size_of_extern_nodeforces));
  hipMemcpy(extern_nodeforces, host_extern_nodeforces, size_of_extern_nodeforces, hipMemcpyHostToDevice);

  if(para.external_force == 0)cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu, sizeof(LB_parameters_gpu))); 

  int threads_per_block_exf = 64;
  int blocks_per_grid_exf_y = 4;
  int blocks_per_grid_exf_x = (n_extern_nodeforces + threads_per_block_exf * blocks_per_grid_exf_y - 1) /(threads_per_block_exf * blocks_per_grid_exf_y);
  dim3 dim_grid_exf = make_uint3(blocks_per_grid_exf_x, blocks_per_grid_exf_y, 1);
	
  KERNELCALL(init_extern_nodeforces, dim_grid_exf, threads_per_block_exf, (n_extern_nodeforces, extern_nodeforces, node_f));
  hipFree(extern_nodeforces);
}

/**setup and call particle kernel from the host
 * @param **host_data		Pointer to the host particle positions and velocities
*/
void lb_particle_GPU(LB_particle_gpu *host_data){
  
  /** get espresso md particle values*/
  hipMemcpyAsync(particle_data, host_data, size_of_positions, hipMemcpyHostToDevice, stream[0]);
  /** call of the particle kernel */
  /** values for the particle kernel */
  int threads_per_block_particles = 64;
  int blocks_per_grid_particles_y = 4;
  int blocks_per_grid_particles_x = (lbpar_gpu.number_of_particles + threads_per_block_particles * blocks_per_grid_particles_y - 1)/(threads_per_block_particles * blocks_per_grid_particles_y);
  dim3 dim_grid_particles = make_uint3(blocks_per_grid_particles_x, blocks_per_grid_particles_y, 1);

  KERNELCALL(calc_fluid_particle_ia, dim_grid_particles, threads_per_block_particles, (*current_nodes, particle_data, particle_force, node_f, part));
}
/** setup and call kernel to copy particle forces to host
 * @param *host_forces contains the particle force computed on the GPU
*/
void lb_copy_forces_GPU(LB_particle_force_gpu *host_forces){

  /** Copy result from device memory to host memory*/
  hipMemcpy(host_forces, particle_force, size_of_forces, hipMemcpyDeviceToHost);

    /** values for the particle kernel */
  int threads_per_block_particles = 64;
  int blocks_per_grid_particles_y = 4;
  int blocks_per_grid_particles_x = (lbpar_gpu.number_of_particles + threads_per_block_particles * blocks_per_grid_particles_y - 1)/(threads_per_block_particles * blocks_per_grid_particles_y);
  dim3 dim_grid_particles = make_uint3(blocks_per_grid_particles_x, blocks_per_grid_particles_y, 1);

  /** reset part forces with zero*/
  KERNELCALL(reset_particle_force, dim_grid_particles, threads_per_block_particles, (particle_force));
	
  hipDeviceSynchronize();
}

/** setup and call kernel for getting macroscopic fluid values of all nodes
 * @param *host_values struct to save the gpu values
*/
void lb_get_values_GPU(LB_values_gpu *host_values){

  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(values, dim_grid, threads_per_block, (*current_nodes, device_values));
  hipMemcpy(host_values, device_values, size_of_values, hipMemcpyDeviceToHost);

}

/** get all the boundary flags for all nodes
 *  @param host_bound_array here go the values of the boundary flag
 */
void lb_get_boundary_flags_GPU(unsigned int* host_bound_array){
   
  unsigned int* device_bound_array;
  cuda_safe_mem(hipMalloc((void**)&device_bound_array, lbpar_gpu.number_of_nodes*sizeof(unsigned int)));	
  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(lb_get_boundaries, dim_grid, threads_per_block, (*current_nodes, device_bound_array));

  hipMemcpy(host_bound_array, device_bound_array, lbpar_gpu.number_of_nodes*sizeof(unsigned int), hipMemcpyDeviceToHost);

  hipFree(device_bound_array);

}

/** setup and call kernel for getting macroscopic fluid values of a single node*/
void lb_print_node_GPU(int single_nodeindex, LB_values_gpu *host_print_values){ 
      
  LB_values_gpu *device_print_values;
  cuda_safe_mem(hipMalloc((void**)&device_print_values, sizeof(LB_values_gpu)));	
  int threads_per_block_print = 1;
  int blocks_per_grid_print_y = 1;
  int blocks_per_grid_print_x = 1;
  dim3 dim_grid_print = make_uint3(blocks_per_grid_print_x, blocks_per_grid_print_y, 1);

  KERNELCALL(lb_print_node, dim_grid_print, threads_per_block_print, (single_nodeindex, device_print_values, *current_nodes));

  hipMemcpy(host_print_values, device_print_values, sizeof(LB_values_gpu), hipMemcpyDeviceToHost);
  hipFree(device_print_values);

}
/** setup and call kernel to calculate the total momentum of the hole fluid
 * @param *mass value of the mass calcutated on the GPU
*/
void lb_calc_fluid_mass_GPU(double* mass){

  float* tot_mass;
  float cpu_mass =  0.f ;
  cuda_safe_mem(hipMalloc((void**)&tot_mass, sizeof(float)));
  hipMemcpy(tot_mass, &cpu_mass, sizeof(float), hipMemcpyHostToDevice);

  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(calc_mass, dim_grid, threads_per_block,(*current_nodes, tot_mass));

  hipMemcpy(&cpu_mass, tot_mass, sizeof(float), hipMemcpyDeviceToHost);
  
  hipFree(tot_mass);
  mass[0] = (double)(cpu_mass);
}

/** setup and call kernel to calculate the total momentum of the hole fluid
 *  @param host_mom value of the momentum calcutated on the GPU
 */
void lb_calc_fluid_momentum_GPU(double* host_mom){

  float* tot_momentum;
  float host_momentum[3] = { 0.f, 0.f, 0.f};
  cuda_safe_mem(hipMalloc((void**)&tot_momentum, 3*sizeof(float)));
  hipMemcpy(tot_momentum, host_momentum, 3*sizeof(float), hipMemcpyHostToDevice);

  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(momentum, dim_grid, threads_per_block,(*current_nodes, tot_momentum, node_f));
  
  hipMemcpy(host_momentum, tot_momentum, 3*sizeof(float), hipMemcpyDeviceToHost);
  
  hipFree(tot_momentum);
  host_mom[0] = (double)(host_momentum[0]* lbpar_gpu.agrid/lbpar_gpu.tau);
  host_mom[1] = (double)(host_momentum[1]* lbpar_gpu.agrid/lbpar_gpu.tau);
  host_mom[2] = (double)(host_momentum[2]* lbpar_gpu.agrid/lbpar_gpu.tau);
}
/** setup and call kernel to calculate the temperature of the hole fluid
 *  @param host_temp value of the temperatur calcutated on the GPU
*/
void lb_calc_fluid_temperature_GPU(double* host_temp){
  float host_jsquared = 0.f;
  float* device_jsquared;
  cuda_safe_mem(hipMalloc((void**)&device_jsquared, sizeof(float)));
  hipMemcpy(device_jsquared, &host_jsquared, sizeof(float), hipMemcpyHostToDevice);

  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(temperature, dim_grid, threads_per_block,(*current_nodes, device_jsquared));

  hipMemcpy(&host_jsquared, device_jsquared, sizeof(float), hipMemcpyDeviceToHost);

  host_temp[0] = (double)(host_jsquared*1./(3.f*lbpar_gpu.rho*lbpar_gpu.dim_x*lbpar_gpu.dim_y*lbpar_gpu.dim_z*lbpar_gpu.tau*lbpar_gpu.tau*lbpar_gpu.agrid));
}

/** setup and call kernel to get the boundary flag of a single node
 *  @param single_nodeindex number of the node to get the flag for
 *  @param host_flag her goes the value of the boundary flag
 */
void lb_get_boundary_flag_GPU(int single_nodeindex, unsigned int* host_flag){
   
  unsigned int* device_flag;
  cuda_safe_mem(hipMalloc((void**)&device_flag, sizeof(unsigned int)));	
  int threads_per_block_flag = 1;
  int blocks_per_grid_flag_y = 1;
  int blocks_per_grid_flag_x = 1;
  dim3 dim_grid_flag = make_uint3(blocks_per_grid_flag_x, blocks_per_grid_flag_y, 1);

  KERNELCALL(lb_get_boundary_flag, dim_grid_flag, threads_per_block_flag, (single_nodeindex, device_flag, *current_nodes));

  hipMemcpy(host_flag, device_flag, sizeof(unsigned int), hipMemcpyDeviceToHost);

  hipFree(device_flag);

}
/** set the net velocity at a single node
 *  @param single_nodeindex the node to set the velocity for 
 *  @param host_velocity the velocity to set
 */
void lb_set_node_velocity_GPU(int single_nodeindex, float* host_velocity){
   
  float* device_velocity;
  cuda_safe_mem(hipMalloc((void**)&device_velocity, 3*sizeof(float)));	
  hipMemcpy(device_velocity, host_velocity, 3*sizeof(float), hipMemcpyHostToDevice);
  int threads_per_block_flag = 1;
  int blocks_per_grid_flag_y = 1;
  int blocks_per_grid_flag_x = 1;
  dim3 dim_grid_flag = make_uint3(blocks_per_grid_flag_x, blocks_per_grid_flag_y, 1);

  KERNELCALL(set_u_equilibrium, dim_grid_flag, threads_per_block_flag, (*current_nodes, single_nodeindex, device_velocity));

  hipFree(device_velocity);

}
/** reinit of params 
 * @param *lbpar_gpu struct containing the paramters of the fluid
*/
void reinit_parameters_GPU(LB_parameters_gpu *lbpar_gpu){

  /**write parameters in const memory*/
  cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu, sizeof(LB_parameters_gpu)));
}
/**integration kernel for the lb gpu fluid update called from host */
void lb_integrate_GPU(){

  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  /**call of fluid step*/
  if (intflag == 1){
    KERNELCALL(integrate, dim_grid, threads_per_block, (nodes_a, nodes_b, device_values, node_f));
    current_nodes = &nodes_b;
#ifdef LB_BOUNDARIES_GPU		
    if (n_lb_boundaries > 0) KERNELCALL(bb_read, dim_grid, threads_per_block, (nodes_a, nodes_b));
			
    if (n_lb_boundaries > 0) KERNELCALL(bb_write, dim_grid, threads_per_block, (nodes_a, nodes_b));
#endif
    intflag = 0;
  }
  else{
    KERNELCALL(integrate, dim_grid, threads_per_block, (nodes_b, nodes_a, device_values, node_f));
    current_nodes = &nodes_a;
#ifdef LB_BOUNDARIES_GPU		
    if (n_lb_boundaries > 0) KERNELCALL(bb_read, dim_grid, threads_per_block, (nodes_b, nodes_a));
			
    if (n_lb_boundaries > 0) KERNELCALL(bb_write, dim_grid, threads_per_block, (nodes_b, nodes_a));
#endif
    intflag = 1;
  }             
}

/** free gpu memory kernel called from the host (not used anymore) */
void lb_free_GPU(){
  // Free device memory
  hipFree(device_values);
  hipFree(&para);
  hipFree(&nodes_a);
  hipFree(&nodes_b);
  hipFree(particle_force);
  hipFree(particle_data);
  hipFree(&node_f);
  hipFree(part);
  hipStreamDestroy(stream[0]);
}
#endif /* LB_GPU */
