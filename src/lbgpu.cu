#include "hip/hip_runtime.h"
/* 
   Copyright (C) 2010,2011,2012,2013 The ESPResSo project

   This file is part of ESPResSo.
  
   ESPResSo is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.
   
   ESPResSo is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.
   
   You should have received a copy of the GNU General Public License
   along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

/** \file lbgpu.cu
 *
 * Cuda (.cu) file for the Lattice Boltzmann implementation on GPUs.
 * Header file for \ref lbgpu.h.
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

extern "C" {
#include "lbgpu.h"
#include "config.h"
//#include "cuda_common.h"
}

#ifdef LB_GPU
#ifndef GAUSSRANDOM
#define GAUSSRANDOM
#endif

int extended_values_flag=0; /* TODO: this has to be set to one by
                               appropriate functions if there is 
                               the need to compute pi at every 
                               step (e.g. moving boundaries)*/

/**defining structures residing in global memory */

/** device_rho_v: struct for hydrodynamic fields: this is for internal use 
    (i.e. stores values in LB units) and should not used for 
    printing values  */
static LB_rho_v_gpu *device_rho_v= NULL;

/** device_rho_v_pi: extended struct for hydrodynamic fields: this is the interface
    to tcl, and stores values in MD units. It should not used
    as an input for any LB calculations. TODO: This structure is not yet 
    used, and it is here to allow access to the stress tensor at any
    timestep, e.g. for future implementations of moving boundary codes */
static LB_rho_v_gpu *device_rho_v_pi= NULL;

/** print_rho_v_pi: struct for hydrodynamic fields: this is the interface
    to tcl, and stores values in MD units. It should not used
    as an input for any LB calculations. TODO: in the future,
    one might want to have several structures for printing 
    separately rho, v, pi without having to compute/store 
    the complete set. */
static LB_rho_v_pi_gpu *print_rho_v_pi= NULL;

/** structs for velocity densities */
static LB_nodes_gpu nodes_a = {.vd=NULL,.seed=NULL,.boundary=NULL};
static LB_nodes_gpu nodes_b = {.vd=NULL,.seed=NULL,.boundary=NULL};;
/** struct for node force */
static LB_node_force_gpu node_f = {.force=NULL} ;

static LB_extern_nodeforce_gpu *extern_nodeforces = NULL;

#ifdef LB_BOUNDARIES_GPU
static float* LB_boundary_force = NULL;
static float* LB_boundary_velocity = NULL;
/** pointer for bound index array*/
static int *boundary_node_list;
static int *boundary_index_list;
static __device__ __constant__ int n_lb_boundaries_gpu = 0;
static size_t size_of_boundindex;
#endif
/** pointers for additional cuda check flag*/
static int *gpu_check = NULL;
static int *h_gpu_check = NULL;

static unsigned int intflag = 1;
static LB_nodes_gpu *current_nodes = NULL;
/**defining size values for allocating global memory */
static size_t size_of_rho_v;
static size_t size_of_rho_v_pi;
static size_t size_of_extern_nodeforces;

/**parameters residing in constant memory */
static __device__ __constant__ LB_parameters_gpu para;
static const float c_sound_sq = 1.f/3.f;

/**cuda streams for parallel computing on cpu and gpu */
//extern hipStream_t stream[1];

//extern hipError_t err;
//extern hipError_t _err;

/*-------------------------------------------------------*/
/*********************************************************/
/** \name device functions called by kernel functions */
/*********************************************************/
/*-------------------------------------------------------*/

/*-------------------------------------------------------*/

/** atomic add function for sveral cuda architectures 
*/
__device__ inline void atomicadd(float* address, float value){
#if !defined __CUDA_ARCH__ || __CUDA_ARCH__ >= 200 // for Fermi, atomicAdd supports floats
  atomicAdd(address, value);
#elif __CUDA_ARCH__ >= 110
#warning Using slower atomicAdd emulation
// float-atomic-add from 
// [url="http://forums.nvidia.com/index.php?showtopic=158039&view=findpost&p=991561"]
  float old = value;
  while ((old = atomicExch(address, atomicExch(address, 0.0f)+old))!=0.0f);
#else
#error I need at least compute capability 1.1
#endif
}

/**randomgenerator which generates numbers [0,1]
 * @param *rn	Pointer to randomnumber array of the local node or particle 
*/
__device__ void random_01(LB_randomnr_gpu *rn){

  const float mxi = 1.f/(float)(1ul<<31);
  unsigned int curr = rn->seed;

  curr = 1103515245 * curr + 12345;
  rn->randomnr[0] = (float)(curr & ((1ul<<31)-1))*mxi;
  curr = 1103515245 * curr + 12345;
  rn->randomnr[1] = (float)(curr & ((1ul<<31)-1))*mxi;
  rn->seed = curr;

}

/** gaussian random nummber generator for thermalisation
 * @param *rn	Pointer to randomnumber array of the local node node or particle 
*/
__device__ void gaussian_random(LB_randomnr_gpu *rn){

  float x1, x2;
  float r2, fac;
  /** On every second call two gaussian random numbers are calculated
   via the Box-Muller transformation.*/
  /** draw two uniform random numbers in the unit circle */
  do {
    random_01(rn);
    x1 = 2.f*rn->randomnr[0]-1.f;
    x2 = 2.f*rn->randomnr[1]-1.f;
    r2 = x1*x1 + x2*x2;
  } while (r2 >= 1.f || r2 == 0.f);

  /** perform Box-Muller transformation */
  fac = sqrtf(-2.f*__logf(r2)/r2);
  rn->randomnr[0] = x2*fac;
  rn->randomnr[1] = x1*fac;
  
}
/* wrapper */
__device__ void random_wrapper(LB_randomnr_gpu *rn) { 

#ifdef GAUSSRANDOM
	gaussian_random(rn);	
#else 
#define sqrt12i 0.288675134594813f
        random_01(rn);
        rn->randomnr[0]-=0.5f;
        rn->randomnr[0]*=sqrt12i;
        rn->randomnr[1]-=0.5f;
        rn->randomnr[1]*=sqrt12i;
#endif   
}


/**tranformation from 1d array-index to xyz
 * @param index		node index / thread index (Input)
 * @param xyz		Pointer to calculated xyz array (Output)
 */
__device__ void index_to_xyz(unsigned int index, unsigned int *xyz){

  xyz[0] = index%para.dim_x;
  index /= para.dim_x;
  xyz[1] = index%para.dim_y;
  index /= para.dim_y;
  xyz[2] = index;
}

/**calculation of the modes from the velocitydensities (space-transform.)
 * @param n_a		Pointer to local node residing in array a (Input)
 * @param index		node index / thread index (Input)
 * @param mode		Pointer to the local register values mode (Output)
*/
__device__ void calc_m_from_n(LB_nodes_gpu n_a, unsigned int index, float *mode){
  #pragma unroll
  for(int ii=0;ii<LB_COMPONENTS;++ii) { 
  /* mass mode */
  mode[0 + ii * LBQ] = n_a.vd[(0 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(1 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(2 + ii*LBQ ) * para.number_of_nodes + index]
          + n_a.vd[(3 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(4 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(5 + ii*LBQ ) * para.number_of_nodes + index]
          + n_a.vd[(6 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(7 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(8 + ii*LBQ ) * para.number_of_nodes + index]
          + n_a.vd[(9 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index]
          + n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index]
          + n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index];

  /* momentum modes */
  mode[1 + ii * LBQ] = (n_a.vd[(1 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(2 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(7 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(8 + ii*LBQ ) * para.number_of_nodes + index])
          + (n_a.vd[(9 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index])
          + (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index]);
  mode[2 + ii * LBQ] = (n_a.vd[(3 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(4 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(7 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(8 + ii*LBQ ) * para.number_of_nodes + index])
          - (n_a.vd[(9 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index])
          + (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]);
  mode[3 + ii * LBQ] = (n_a.vd[(5 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(6 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index])
          - (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index])
          - (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]);

  /* stress modes */
  mode[4 + ii * LBQ] = -(n_a.vd[(0 + ii*LBQ ) * para.number_of_nodes + index]) + n_a.vd[(7 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(8 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(9 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index]
          + n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index]
          + n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index];
  mode[5 + ii * LBQ] = n_a.vd[(1 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(2 + ii*LBQ ) * para.number_of_nodes + index] - (n_a.vd[(3 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(4 + ii*LBQ ) * para.number_of_nodes + index])
          + (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index])
          - (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index]) - (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]);
  mode[6 + ii * LBQ] = (n_a.vd[(1 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(2 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(3 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(4 + ii*LBQ ) * para.number_of_nodes + index])
          - (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index]) - (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index])
          - (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index]) - (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index])
          - 2.f*(n_a.vd[(5 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(6 + ii*LBQ ) * para.number_of_nodes + index] - (n_a.vd[(7 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(8 + ii*LBQ ) * para.number_of_nodes + index])
          - (n_a.vd[(9 + ii*LBQ ) * para.number_of_nodes + index] +n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index]));
  mode[7 + ii * LBQ] = n_a.vd[(7 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(8 + ii*LBQ ) * para.number_of_nodes + index] - (n_a.vd[(9 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index]);
  mode[8 + ii * LBQ] = n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index] - (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index]);
  mode[9 + ii * LBQ] = n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index] - (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]);

  /* kinetic modes */
  mode[10 + ii * LBQ] = -2.f*(n_a.vd[(1 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(2 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(7 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(8 + ii*LBQ ) * para.number_of_nodes + index])
           + (n_a.vd[(9 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index])
           + (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index]);
  mode[11 + ii * LBQ] = -2.f*(n_a.vd[(3 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(4 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(7 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(8 + ii*LBQ ) * para.number_of_nodes + index])
           - (n_a.vd[(9 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index])
           + (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]);
  mode[12 + ii * LBQ] = -2.f*(n_a.vd[(5 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(6 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index])
           - (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index])
           - (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]);
  mode[13 + ii * LBQ] = (n_a.vd[(7 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(8 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(9 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index])
           - (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index]) - (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index]);
  mode[14 + ii * LBQ] = (n_a.vd[(7 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(8 + ii*LBQ ) * para.number_of_nodes + index]) - (n_a.vd[(9 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index])
           - (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index]) - (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]);
  mode[15 + ii * LBQ] = (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index]) - (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index])
           - (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]);
  mode[16 + ii * LBQ] = n_a.vd[(0 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(7 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(8 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(9 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index]
           + n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index]
           + n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]
           - 2.f*((n_a.vd[(1 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(2 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(3 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(4 + ii*LBQ ) * para.number_of_nodes + index])
           + (n_a.vd[(5 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(6 + ii*LBQ ) * para.number_of_nodes + index]));
  mode[17 + ii * LBQ] = -(n_a.vd[(1 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(2 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(3 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(4 + ii*LBQ ) * para.number_of_nodes + index])
           + (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index])
           - (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index]) - (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]);
  mode[18 + ii * LBQ] = -(n_a.vd[(1 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(2 + ii*LBQ ) * para.number_of_nodes + index]) - (n_a.vd[(3 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(4 + ii*LBQ ) * para.number_of_nodes + index])
           - (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index]) - (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index])
           - (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index]) - (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index])
           + 2.f*((n_a.vd[(5 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(6 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(7 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(8 + ii*LBQ ) * para.number_of_nodes + index])
           + (n_a.vd[(9 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index]));

 }
}

__device__ void update_rho_v(float *mode, unsigned int index, LB_node_force_gpu node_f, LB_rho_v_gpu *d_v){

  float Rho_tot=0.f;
  float u_tot[3]={0.f,0.f,0.f};
  
  #pragma unroll
  for(int ii=0;ii<LB_COMPONENTS;++ii) { 
      /** re-construct the real density
      * remember that the populations are stored as differences to their
      * equilibrium value */
      d_v[index].rho[ii]= mode[0 + ii * LBQ]+ para.rho[ii]*para.agrid*para.agrid*para.agrid;
      Rho_tot  += mode[0 + ii * LBQ]+ para.rho[ii]*para.agrid*para.agrid*para.agrid;
      u_tot[0] += mode[1 + ii * LBQ];
      u_tot[1] += mode[2 + ii * LBQ];
      u_tot[2] += mode[3 + ii * LBQ];

      /** if forces are present, the momentum density is redefined to
      * inlcude one half-step of the force action.  See the
      * Chapman-Enskog expansion in [Ladd & Verberg]. */
      u_tot[0] += 0.5f*node_f.force[(0+ii*3)*para.number_of_nodes + index];
      u_tot[1] += 0.5f*node_f.force[(1+ii*3)*para.number_of_nodes + index];
      u_tot[2] += 0.5f*node_f.force[(2+ii*3)*para.number_of_nodes + index];
  }
  u_tot[0]/=Rho_tot;
  u_tot[1]/=Rho_tot;
  u_tot[2]/=Rho_tot;

  d_v[index].v[0]=u_tot[0]; 
  d_v[index].v[1]=u_tot[1]; 
  d_v[index].v[2]=u_tot[2]; 
}

/**lb_relax_modes, means collision update of the modes
 * @param index		node index / thread index (Input)
 * @param mode		Pointer to the local register values mode (Input/Output)
 * @param node_f	Pointer to local node force (Input)
*/
__device__ void relax_modes(float *mode, unsigned int index, LB_node_force_gpu node_f, LB_rho_v_gpu *d_v){
  float u_tot[3]={0.f,0.f,0.f};

  update_rho_v(mode, index, node_f, d_v);
  u_tot[0]=d_v[index].v[0];  
  u_tot[1]=d_v[index].v[1];  
  u_tot[2]=d_v[index].v[2];  
 
  #pragma unroll
  for(int ii=0;ii<LB_COMPONENTS;++ii) { 
      float Rho; float j[3]; float pi_eq[6];

      Rho = mode[0 + ii * LBQ] + para.rho[ii]*para.agrid*para.agrid*para.agrid ;
      j[0] = Rho * u_tot[0];
      j[1] = Rho * u_tot[1];
      j[2] = Rho * u_tot[2];
      /** equilibrium part of the stress modes (eq13 schiller)*/

      pi_eq[0] = ((j[0]*j[0])+(j[1]*j[1])+(j[2]*j[2]))/Rho;
      pi_eq[1] = ((j[0]*j[0])-(j[1]*j[1]))/Rho;
      pi_eq[2] = (((j[0]*j[0])+(j[1]*j[1])+(j[2]*j[2])) - 3.0f*(j[2]*j[2]))/Rho;
      pi_eq[3] = j[0]*j[1]/Rho;
      pi_eq[4] = j[0]*j[2]/Rho;
      pi_eq[5] = j[1]*j[2]/Rho;
 
      /** in Shan-Chen we have to relax the momentum modes as well using the mobility, but
          the total momentum is conserved */  
#ifdef SHANCHEN
      mode[1 + ii * LBQ] = j[0] + para.gamma_mobility[0]*(mode[1 + ii * LBQ] - j[0]);
      mode[2 + ii * LBQ] = j[1] + para.gamma_mobility[0]*(mode[2 + ii * LBQ] - j[1]);
      mode[3 + ii * LBQ] = j[2] + para.gamma_mobility[0]*(mode[3 + ii * LBQ] - j[2]);
#endif
 
      /** relax the stress modes (eq14 schiller)*/
      mode[4 + ii * LBQ] = pi_eq[0] + para.gamma_bulk[ii]*(mode[4 + ii * LBQ] - pi_eq[0]);
      mode[5 + ii * LBQ] = pi_eq[1] + para.gamma_shear[ii]*(mode[5 + ii * LBQ] - pi_eq[1]);
      mode[6 + ii * LBQ] = pi_eq[2] + para.gamma_shear[ii]*(mode[6 + ii * LBQ] - pi_eq[2]);
      mode[7 + ii * LBQ] = pi_eq[3] + para.gamma_shear[ii]*(mode[7 + ii * LBQ] - pi_eq[3]);
      mode[8 + ii * LBQ] = pi_eq[4] + para.gamma_shear[ii]*(mode[8 + ii * LBQ] - pi_eq[4]);
      mode[9 + ii * LBQ] = pi_eq[5] + para.gamma_shear[ii]*(mode[9 + ii * LBQ] - pi_eq[5]);
    
      /** relax the ghost modes (project them out) */
      /** ghost modes have no equilibrium part due to orthogonality */
      mode[10 + ii * LBQ] = para.gamma_odd[ii]*mode[10 + ii * LBQ];
      mode[11 + ii * LBQ] = para.gamma_odd[ii]*mode[11 + ii * LBQ];
      mode[12 + ii * LBQ] = para.gamma_odd[ii]*mode[12 + ii * LBQ];
      mode[13 + ii * LBQ] = para.gamma_odd[ii]*mode[13 + ii * LBQ];
      mode[14 + ii * LBQ] = para.gamma_odd[ii]*mode[14 + ii * LBQ];
      mode[15 + ii * LBQ] = para.gamma_odd[ii]*mode[15 + ii * LBQ];
      mode[16 + ii * LBQ] = para.gamma_even[ii]*mode[16 + ii * LBQ];
      mode[17 + ii * LBQ] = para.gamma_even[ii]*mode[17 + ii * LBQ];
      mode[18 + ii * LBQ] = para.gamma_even[ii]*mode[18 + ii * LBQ];
 }
}


/**thermalization of the modes with gaussian random numbers
 * @param index		node index / thread index (Input)
 * @param mode		Pointer to the local register values mode (Input/Output)
 * @param *rn		Pointer to randomnumber array of the local node
*/
__device__ void thermalize_modes(float *mode, unsigned int index, LB_randomnr_gpu *rn){
  float Rho;
#ifdef SHANCHEN
  random_wrapper(rn);
  for(int ii=0;ii<LB_COMPONENTS;++ii) { 
      mode[1 + ii * LBQ] += sqrt((para.mu[ii]*(2.f/3.f)*(1.f-(para.gamma_mobility[0]*para.gamma_mobility[0])))) * (2*ii-1) * rn->randomnr[0];
      mode[2 + ii * LBQ] += sqrt((para.mu[ii]*(2.f/3.f)*(1.f-(para.gamma_mobility[0]*para.gamma_mobility[0])))) * (2*ii-1) * rn->randomnr[1];
  }
  random_wrapper(rn);
  for(int ii=0;ii<LB_COMPONENTS;++ii)  
      mode[3 + ii * LBQ] += sqrt((para.mu[ii]*(2.f/3.f)*(1.f-(para.gamma_mobility[0]*para.gamma_mobility[0])))) * (2*ii-1) * rn->randomnr[0];
#endif
  
  
  for(int ii=0;ii<LB_COMPONENTS;++ii) {  
      
      Rho = mode[0 + ii * LBQ] + para.rho[ii]*para.agrid*para.agrid*para.agrid;
      /** momentum modes */
      random_wrapper(rn);
      /** stress modes */
      mode[4 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(2.f/3.f)*(1.f-(para.gamma_bulk[ii]*para.gamma_bulk[ii])))) * rn->randomnr[0];
      mode[5 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(4.f/9.f)*(1.f-(para.gamma_shear[ii]*para.gamma_shear[ii])))) * rn->randomnr[1];
      random_wrapper(rn);
      mode[6 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(4.f/3.f)*(1.f-(para.gamma_shear[ii]*para.gamma_shear[ii])))) * rn->randomnr[0];
      mode[7 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(1.f/9.f)*(1.f-(para.gamma_shear[ii]*para.gamma_shear[ii])))) * rn->randomnr[1];
      random_wrapper(rn);
      mode[8 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(1.f/9.f)*(1.f-(para.gamma_shear[ii]*para.gamma_shear[ii])))) * rn->randomnr[0];
      mode[9 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(1.f/9.f)*(1.f-(para.gamma_shear[ii]*para.gamma_shear[ii])))) * rn->randomnr[1];
      /** ghost modes */
      random_wrapper(rn);
      mode[10 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(2.f/3.f))) * rn->randomnr[0];
      mode[11 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(2.f/3.f))) * rn->randomnr[1];
      random_wrapper(rn);
      mode[12 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(2.f/3.f))) * rn->randomnr[0];
      mode[13 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(2.f/9.f))) * rn->randomnr[1];
      random_wrapper(rn);
      mode[14 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(2.f/9.f))) * rn->randomnr[0];
      mode[15 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(2.f/9.f))) * rn->randomnr[1];
      random_wrapper(rn);
      mode[16 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(2.f)))     * rn->randomnr[0];
      mode[17 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(4.f/9.f))) * rn->randomnr[1];
      random_wrapper(rn);
      mode[18 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(4.f/3.f))) * rn->randomnr[0];
   }
}


/*-------------------------------------------------------*/
/**normalization of the modes need befor backtransformation into velocity space
 * @param mode		Pointer to the local register values mode (Input/Output)
*/
__device__ void normalize_modes(float* mode){
  #pragma unroll
  for(int ii=0;ii<LB_COMPONENTS;++ii) { 

      /** normalization factors enter in the back transformation */
      mode[0 + ii * LBQ] *= 1.f;
      mode[1 + ii * LBQ] *= 3.f;
      mode[2 + ii * LBQ] *= 3.f;
      mode[3 + ii * LBQ] *= 3.f;
      mode[4 + ii * LBQ] *= 3.f/2.f;
      mode[5 + ii * LBQ] *= 9.f/4.f;
      mode[6 + ii * LBQ] *= 3.f/4.f;
      mode[7 + ii * LBQ] *= 9.f;
      mode[8 + ii * LBQ] *= 9.f;
      mode[9 + ii * LBQ] *= 9.f;
      mode[10 + ii * LBQ] *= 3.f/2.f;
      mode[11 + ii * LBQ] *= 3.f/2.f;
      mode[12 + ii * LBQ] *= 3.f/2.f;
      mode[13 + ii * LBQ] *= 9.f/2.f;
      mode[14 + ii * LBQ] *= 9.f/2.f;
      mode[15 + ii * LBQ] *= 9.f/2.f;
      mode[16 + ii * LBQ] *= 1.f/2.f;
      mode[17 + ii * LBQ] *= 9.f/4.f;
      mode[18 + ii * LBQ] *= 3.f/4.f;
  }
}



/*-------------------------------------------------------*/
/**backtransformation from modespace to desityspace and streaming with the push method using pbc
 * @param index		node index / thread index (Input)
 * @param mode		Pointer to the local register values mode (Input)
 * @param *n_b		Pointer to local node residing in array b (Output)
*/
__device__ void calc_n_from_modes_push(LB_nodes_gpu n_b, float *mode, unsigned int index){

  unsigned int xyz[3];
  index_to_xyz(index, xyz);
  unsigned int x = xyz[0];
  unsigned int y = xyz[1];
  unsigned int z = xyz[2];

  #pragma unroll
  for(int ii=0;ii<LB_COMPONENTS;++ii) { 
  n_b.vd[(0 + ii*LBQ ) * para.number_of_nodes + x + para.dim_x*y + para.dim_x*para.dim_y*z] = 1.f/3.f * (mode[0 + ii * LBQ] - mode[4 + ii * LBQ] + mode[16 + ii * LBQ]);
  n_b.vd[(1 + ii*LBQ ) * para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z] = 1.f/18.f * (mode[0 + ii * LBQ] + mode[1 + ii * LBQ] + mode[5 + ii * LBQ] + mode[6 + ii * LBQ] - mode[17 + ii * LBQ] - mode[18 + ii * LBQ] - 2.f*(mode[10 + ii * LBQ] + mode[16 + ii * LBQ]));
  n_b.vd[(2 + ii*LBQ ) * para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z] = 1.f/18.f * (mode[0 + ii * LBQ] - mode[1 + ii * LBQ] + mode[5 + ii * LBQ] + mode[6 + ii * LBQ] - mode[17 + ii * LBQ] - mode[18 + ii * LBQ] + 2.f*(mode[10 + ii * LBQ] - mode[16 + ii * LBQ]));
  n_b.vd[(3 + ii*LBQ ) * para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = 1.f/18.f * (mode[0 + ii * LBQ] + mode[2 + ii * LBQ] - mode[5 + ii * LBQ] + mode[6 + ii * LBQ] + mode[17 + ii * LBQ] - mode[18 + ii * LBQ] - 2.f*(mode[11 + ii * LBQ] + mode[16 + ii * LBQ]));
  n_b.vd[(4 + ii*LBQ ) * para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = 1.f/18.f * (mode[0 + ii * LBQ] - mode[2 + ii * LBQ] - mode[5 + ii * LBQ] + mode[6 + ii * LBQ] + mode[17 + ii * LBQ] - mode[18 + ii * LBQ] + 2.f*(mode[11 + ii * LBQ] - mode[16 + ii * LBQ]));
  n_b.vd[(5 + ii*LBQ ) * para.number_of_nodes + x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = 1.f/18.f * (mode[0 + ii * LBQ] + mode[3 + ii * LBQ] - 2.f*(mode[6 + ii * LBQ] + mode[12 + ii * LBQ] + mode[16 + ii * LBQ] - mode[18 + ii * LBQ]));
  n_b.vd[(6 + ii*LBQ ) * para.number_of_nodes + x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = 1.f/18.f * (mode[0 + ii * LBQ] - mode[3 + ii * LBQ] - 2.f*(mode[6 + ii * LBQ] - mode[12 + ii * LBQ] + mode[16 + ii * LBQ] - mode[18 + ii * LBQ]));
  n_b.vd[(7 + ii*LBQ ) * para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = 1.f/36.f * (mode[0 + ii * LBQ] + mode[1 + ii * LBQ] + mode[2 + ii * LBQ] + mode[4 + ii * LBQ] + 2.f*mode[6 + ii * LBQ] + mode[7 + ii * LBQ] + mode[10 + ii * LBQ] + mode[11 + ii * LBQ] + mode[13 + ii * LBQ] + mode[14 + ii * LBQ] + mode[16 + ii * LBQ] + 2.f*mode[18 + ii * LBQ]);
  n_b.vd[(8 + ii*LBQ ) * para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = 1.f/36.f * (mode[0 + ii * LBQ] - mode[1 + ii * LBQ] - mode[2 + ii * LBQ] + mode[4 + ii * LBQ] + 2.f*mode[6 + ii * LBQ] + mode[7 + ii * LBQ] - mode[10 + ii * LBQ] - mode[11 + ii * LBQ] - mode[13 + ii * LBQ] - mode[14 + ii * LBQ] + mode[16 + ii * LBQ] + 2.f*mode[18 + ii * LBQ]);
  n_b.vd[(9 + ii*LBQ ) * para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = 1.f/36.f * (mode[0 + ii * LBQ] + mode[1 + ii * LBQ] - mode[2 + ii * LBQ] + mode[4 + ii * LBQ] + 2.f*mode[6 + ii * LBQ] - mode[7 + ii * LBQ] + mode[10 + ii * LBQ] - mode[11 + ii * LBQ] + mode[13 + ii * LBQ] - mode[14 + ii * LBQ] + mode[16 + ii * LBQ] + 2.f*mode[18 + ii * LBQ]);
  n_b.vd[(10 + ii*LBQ ) * para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = 1.f/36.f * (mode[0 + ii * LBQ] - mode[1 + ii * LBQ] + mode[2 + ii * LBQ] + mode[4 + ii * LBQ] + 2.f*mode[6 + ii * LBQ] - mode[7 + ii * LBQ] - mode[10 + ii * LBQ] + mode[11 + ii * LBQ] - mode[13 + ii * LBQ] + mode[14 + ii * LBQ] + mode[16 + ii * LBQ] + 2.f*mode[18 + ii * LBQ]);
  n_b.vd[(11 + ii*LBQ ) * para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = 1.f/36.f * (mode[0 + ii * LBQ] + mode[1 + ii * LBQ] + mode[3 + ii * LBQ] + mode[4 + ii * LBQ] + mode[5 + ii * LBQ] - mode[6 + ii * LBQ] + mode[8 + ii * LBQ] + mode[10 + ii * LBQ] + mode[12 + ii * LBQ] - mode[13 + ii * LBQ] + mode[15 + ii * LBQ] + mode[16 + ii * LBQ] + mode[17 + ii * LBQ] - mode[18 + ii * LBQ]);
  n_b.vd[(12 + ii*LBQ ) * para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = 1.f/36.f * (mode[0 + ii * LBQ] - mode[1 + ii * LBQ] - mode[3 + ii * LBQ] + mode[4 + ii * LBQ] + mode[5 + ii * LBQ] - mode[6 + ii * LBQ] + mode[8 + ii * LBQ] - mode[10 + ii * LBQ] - mode[12 + ii * LBQ] + mode[13 + ii * LBQ] - mode[15 + ii * LBQ] + mode[16 + ii * LBQ] + mode[17 + ii * LBQ] - mode[18 + ii * LBQ]);
  n_b.vd[(13 + ii*LBQ ) * para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = 1.f/36.f * (mode[0 + ii * LBQ] + mode[1 + ii * LBQ] - mode[3 + ii * LBQ] + mode[4 + ii * LBQ] + mode[5 + ii * LBQ] - mode[6 + ii * LBQ] - mode[8 + ii * LBQ] + mode[10 + ii * LBQ] - mode[12 + ii * LBQ] - mode[13 + ii * LBQ] - mode[15 + ii * LBQ] + mode[16 + ii * LBQ] + mode[17 + ii * LBQ] - mode[18 + ii * LBQ]);
  n_b.vd[(14 + ii*LBQ ) * para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = 1.f/36.f * (mode[0 + ii * LBQ] - mode[1 + ii * LBQ] + mode[3 + ii * LBQ] + mode[4 + ii * LBQ] + mode[5 + ii * LBQ] - mode[6 + ii * LBQ] - mode[8 + ii * LBQ] - mode[10 + ii * LBQ] + mode[12 + ii * LBQ] + mode[13 + ii * LBQ] + mode[15 + ii * LBQ] + mode[16 + ii * LBQ] + mode[17 + ii * LBQ] - mode[18 + ii * LBQ]);
  n_b.vd[(15 + ii*LBQ ) * para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = 1.f/36.f * (mode[0 + ii * LBQ] + mode[2 + ii * LBQ] + mode[3 + ii * LBQ] + mode[4 + ii * LBQ] - mode[5 + ii * LBQ] - mode[6 + ii * LBQ] + mode[9 + ii * LBQ] + mode[11 + ii * LBQ] + mode[12 + ii * LBQ] - mode[14 + ii * LBQ] - mode[15 + ii * LBQ] + mode[16 + ii * LBQ] - mode[17 + ii * LBQ] - mode[18 + ii * LBQ]);
  n_b.vd[(16 + ii*LBQ ) * para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = 1.f/36.f * (mode[0 + ii * LBQ] - mode[2 + ii * LBQ] - mode[3 + ii * LBQ] + mode[4 + ii * LBQ] - mode[5 + ii * LBQ] - mode[6 + ii * LBQ] + mode[9 + ii * LBQ] - mode[11 + ii * LBQ] - mode[12 + ii * LBQ] + mode[14 + ii * LBQ] + mode[15 + ii * LBQ] + mode[16 + ii * LBQ] - mode[17 + ii * LBQ] - mode[18 + ii * LBQ]);
  n_b.vd[(17 + ii*LBQ ) * para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = 1.f/36.f * (mode[0 + ii * LBQ] + mode[2 + ii * LBQ] - mode[3 + ii * LBQ] + mode[4 + ii * LBQ] - mode[5 + ii * LBQ] - mode[6 + ii * LBQ] - mode[9 + ii * LBQ] + mode[11 + ii * LBQ] - mode[12 + ii * LBQ] - mode[14 + ii * LBQ] + mode[15 + ii * LBQ] + mode[16 + ii * LBQ] - mode[17 + ii * LBQ] - mode[18 + ii * LBQ]);
  n_b.vd[(18 + ii*LBQ ) * para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = 1.f/36.f * (mode[0 + ii * LBQ] - mode[2 + ii * LBQ] + mode[3 + ii * LBQ] + mode[4 + ii * LBQ] - mode[5 + ii * LBQ] - mode[6 + ii * LBQ] - mode[9 + ii * LBQ] - mode[11 + ii * LBQ] + mode[12 + ii * LBQ] + mode[14 + ii * LBQ] - mode[15 + ii * LBQ] + mode[16 + ii * LBQ] - mode[17 + ii * LBQ] - mode[18 + ii * LBQ]);

}
}


#ifndef SHANCHEN

/** Bounce back boundary conditions.
 * The populations that have propagated into a boundary node
 * are bounced back to the node they came from. This results
 * in no slip boundary conditions.
 *
 * [cf. Ladd and Verberg, J. Stat. Phys. 104(5/6):1191-1251, 2001]
 * @param index			node index / thread index (Input)
 * @param n_b			Pointer to local node residing in array b (Input)
 * @param n_a			Pointer to local node residing in array a (Output) (temp stored in buffer a)
 * @param LB_boundary_velocity 			The constant velocity at the boundary, set by the user (Input)
 * @param LB_boundary_force 			The force on the boundary nodes (Output)
*/
__device__ void bounce_back_read(LB_nodes_gpu n_b, LB_nodes_gpu n_a, unsigned int index, \
    float* LB_boundary_velocity, float* LB_boundary_force){
    
  unsigned int xyz[3];
  int c[3];
  float v[3];
  float shift, weight, pop_to_bounce_back;
  float boundary_force[3] = {0,0,0};
  size_t to_index, to_index_x, to_index_y, to_index_z;
  int population, inverse;
  int boundary_index;


  boundary_index=n_b.boundary[index];
  if(boundary_index != 0){
    
    v[0]=LB_boundary_velocity[3*(boundary_index-1)+0];
    v[1]=LB_boundary_velocity[3*(boundary_index-1)+1];
    v[2]=LB_boundary_velocity[3*(boundary_index-1)+2];

    index_to_xyz(index, xyz);

    unsigned int x = xyz[0];
    unsigned int y = xyz[1];
    unsigned int z = xyz[2];

/* CPU analog of shift:
   lbpar.agrid*lbpar.agrid*lbpar.agrid*lbpar.rho*2*lbmodel.c[i][l]*lb_boundaries[lbfields[k].boundary-1].velocity[l] */
  
    /** store vd temporary in second lattice to avoid race conditions */
   // TODO: fix the multicomponent version (rho...)
#define BOUNCEBACK  \
  shift = para.agrid*para.agrid*para.agrid*para.agrid*para.rho[0]*2.*3.*weight*para.tau*(v[0]*c[0] + v[1]*c[1] + v[2]*c[2]); \
  pop_to_bounce_back = n_b.vd[population*para.number_of_nodes + index ]; \
  to_index_x = (x+c[0]+para.dim_x)%para.dim_x; \
  to_index_y = (y+c[1]+para.dim_y)%para.dim_y; \
  to_index_z = (z+c[2]+para.dim_z)%para.dim_z; \
  to_index = to_index_x + para.dim_x*to_index_y + para.dim_x*para.dim_y*to_index_z; \
  if (n_b.boundary[to_index] == 0) \
  { \
    boundary_force[0] += (2*pop_to_bounce_back+shift)*c[0]/para.tau/para.tau/para.agrid; \
    boundary_force[1] += (2*pop_to_bounce_back+shift)*c[1]/para.tau/para.tau/para.agrid; \
    boundary_force[2] += (2*pop_to_bounce_back+shift)*c[2]/para.tau/para.tau/para.agrid; \
    n_b.vd[inverse*para.number_of_nodes + to_index ] = pop_to_bounce_back + shift; \
  }

// ***** SHOULDN'T THERE BE AN ELSE STATMENT IN "BOUNCEBACK"?
// ***** THERE IS AN ODD FACTOR OF 2 THAT YOU INCUR IN THE FORCES FOR THE "lb_stokes_sphere_gpu.tcl" TEST CASE

    // the resting population does nothing.
    c[0]=1;c[1]=0;c[2]=0; weight=1./18.; population=2; inverse=1; 
    BOUNCEBACK
    
    c[0]=-1;c[1]=0;c[2]=0; weight=1./18.; population=1; inverse=2; 
    BOUNCEBACK
    
    c[0]=0;c[1]=1;c[2]=0;  weight=1./18.; population=4; inverse=3; 
    BOUNCEBACK

    c[0]=0;c[1]=-1;c[2]=0; weight=1./18.; population=3; inverse=4; 
    BOUNCEBACK
    
    c[0]=0;c[1]=0;c[2]=1; weight=1./18.; population=6; inverse=5; 
    BOUNCEBACK

    c[0]=0;c[1]=0;c[2]=-1; weight=1./18.; population=5; inverse=6; 
    BOUNCEBACK 
    
    c[0]=1;c[1]=1;c[2]=0; weight=1./36.; population=8; inverse=7; 
    BOUNCEBACK
    
    c[0]=-1;c[1]=-1;c[2]=0; weight=1./36.; population=7; inverse=8; 
    BOUNCEBACK
    
    c[0]=1;c[1]=-1;c[2]=0; weight=1./36.; population=10; inverse=9; 
    BOUNCEBACK

    c[0]=-1;c[1]=+1;c[2]=0; weight=1./36.; population=9; inverse=10; 
    BOUNCEBACK
    
    c[0]=1;c[1]=0;c[2]=1; weight=1./36.; population=12; inverse=11; 
    BOUNCEBACK
    
    c[0]=-1;c[1]=0;c[2]=-1; weight=1./36.; population=11; inverse=12; 
    BOUNCEBACK

    c[0]=1;c[1]=0;c[2]=-1; weight=1./36.; population=14; inverse=13; 
    BOUNCEBACK
    
    c[0]=-1;c[1]=0;c[2]=1; weight=1./36.; population=13; inverse=14; 
    BOUNCEBACK

    c[0]=0;c[1]=1;c[2]=1; weight=1./36.; population=16; inverse=15; 
    BOUNCEBACK
    
    c[0]=0;c[1]=-1;c[2]=-1; weight=1./36.; population=15; inverse=16; 
    BOUNCEBACK
    
    c[0]=0;c[1]=1;c[2]=-1; weight=1./36.; population=18; inverse=17; 
    BOUNCEBACK
    
    c[0]=0;c[1]=-1;c[2]=1; weight=1./36.; population=17; inverse=18; 
    BOUNCEBACK  
    
    atomicadd(&LB_boundary_force[3*(n_b.boundary[index]-1)+0], boundary_force[0]);
    atomicadd(&LB_boundary_force[3*(n_b.boundary[index]-1)+1], boundary_force[1]);
    atomicadd(&LB_boundary_force[3*(n_b.boundary[index]-1)+2], boundary_force[2]);
  }
}


#else  // SHANCHEN

// To be implemented


#endif // SHANCHEN

#ifndef SHANCHEN

/**bounce back read kernel needed to avoid raceconditions
 * @param index			node index / thread index (Input)
 * @param n_b			Pointer to local node residing in array b (Input)
 * @param n_a			Pointer to local node residing in array a (Output) (temp stored in buffer a)
*/
__device__ void bounce_back_write(LB_nodes_gpu n_b, LB_nodes_gpu n_a, unsigned int index){

  unsigned int xyz[3];

  if(n_b.boundary[index] != 0){
    index_to_xyz(index, xyz);
    unsigned int x = xyz[0];
    unsigned int y = xyz[1];
    unsigned int z = xyz[2];

    /** stream vd from boundary node back to origin node */
    n_b.vd[1*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z] = n_a.vd[1*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z];
    n_b.vd[2*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z] = n_a.vd[2*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z];
    n_b.vd[3*para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_a.vd[3*para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z];
    n_b.vd[4*para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_a.vd[4*para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z];
    n_b.vd[5*para.number_of_nodes + x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_a.vd[5*para.number_of_nodes + x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)];
    n_b.vd[6*para.number_of_nodes + x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_a.vd[6*para.number_of_nodes + x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)];
    n_b.vd[7*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_a.vd[7*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z];
    n_b.vd[8*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_a.vd[8*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z];
    n_b.vd[9*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_a.vd[9*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z];
    n_b.vd[10*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_a.vd[10*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z];
    n_b.vd[11*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_a.vd[11*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)];
    n_b.vd[12*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_a.vd[12*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)];
    n_b.vd[13*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_a.vd[13*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)];
    n_b.vd[14*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_a.vd[14*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)];
    n_b.vd[15*para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_a.vd[15*para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)];
    n_b.vd[16*para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_a.vd[16*para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)];
    n_b.vd[17*para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_a.vd[17*para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)];
    n_b.vd[18*para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_a.vd[18*para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)];
  }
}

#else // SHANCHEN

// to be implemented

#endif // SHANCHEN


/** add of (external) forces within the modespace, needed for particle-interaction
 * @param index		node index / thread index (Input)
 * @param mode		Pointer to the local register values mode (Input/Output)
 * @param node_f	Pointer to local node force (Input)
*/
__device__ void apply_forces(unsigned int index, float *mode, LB_node_force_gpu node_f, LB_rho_v_gpu *d_v) {
  
  float u[3]={0.f,0.f,0.f}, C[6]={0.f,0.f,0.f,0.f,0.f,0.f};
  float force_factor=powf(para.agrid,4)*para.tau*para.tau;
  /* Note: the values d_v were calculated in relax_modes() */

  u[0]=d_v[index].v[0]; 
  u[1]=d_v[index].v[1]; 
  u[2]=d_v[index].v[2]; 


  #pragma unroll
  for(int ii=0;ii<LB_COMPONENTS;++ii) {  
       C[0] += (1.f + para.gamma_bulk[ii])*u[0]*node_f.force[(0 + ii*3 ) * para.number_of_nodes + index] + 
                1.f/3.f*(para.gamma_bulk[ii]-para.gamma_shear[ii])*(u[0]*node_f.force[(0 + ii*3 ) * para.number_of_nodes + index] + 
                        u[1]*node_f.force[(1 + ii*3 ) * para.number_of_nodes + index] + 
                        u[2]*node_f.force[(2 + ii*3 ) * para.number_of_nodes + index]);
       C[2] += (1.f + para.gamma_bulk[ii])*u[1]*node_f.force[(1 + ii*3 ) * para.number_of_nodes + index] + 
                1.f/3.f*(para.gamma_bulk[ii]-para.gamma_shear[ii])*(u[0]*node_f.force[(0 + ii*3 ) * para.number_of_nodes + index] + 
                        u[1]*node_f.force[(1 + ii*3 ) * para.number_of_nodes + index] + 
                        u[2]*node_f.force[(2 + ii*3 ) * para.number_of_nodes + index]);
       C[5] += (1.f + para.gamma_bulk[ii])*u[2]*node_f.force[(2 + ii*3 ) * para.number_of_nodes + index] + 
                1.f/3.f*(para.gamma_bulk[ii]-para.gamma_shear[ii])*(u[0]*node_f.force[(0 + ii*3 ) * para.number_of_nodes + index] + 
                        u[1]*node_f.force[(1 + ii*3 ) * para.number_of_nodes + index] + 
                        u[2]*node_f.force[(2 + ii*3 ) * para.number_of_nodes + index]);
       C[1] += 1.f/2.f*(1.f+para.gamma_shear[ii])*(u[0]*node_f.force[(1 + ii*3 ) * para.number_of_nodes + index]+
                        u[1]*node_f.force[(0 + ii*3 ) * para.number_of_nodes + index]);
       C[3] += 1.f/2.f*(1.f+para.gamma_shear[ii])*(u[0]*node_f.force[(2 + ii*3 ) * para.number_of_nodes + index]+
                        u[2]*node_f.force[(0 + ii*3 ) * para.number_of_nodes + index]);
       C[4] += 1.f/2.f*(1.f+para.gamma_shear[ii])*(u[1]*node_f.force[(2 + ii*3 ) * para.number_of_nodes + index]+
                        u[2]*node_f.force[(1 + ii*3 ) * para.number_of_nodes + index]);
  }

  #pragma unroll
  for(int ii=0;ii<LB_COMPONENTS;++ii) {  
      /** update momentum modes */
#ifdef SHANCHEN
      float mobility_factor=1.f/2.f*(1.f+para.gamma_mobility[0]);
#else
      float mobility_factor=1.f;
#endif 
 /** update momentum modes */
      mode[1 + ii * LBQ] += mobility_factor * node_f.force[(0 + ii*3 ) * para.number_of_nodes + index];
      mode[2 + ii * LBQ] += mobility_factor * node_f.force[(1 + ii*3 ) * para.number_of_nodes + index];
      mode[3 + ii * LBQ] += mobility_factor * node_f.force[(2 + ii*3 ) * para.number_of_nodes + index];
      	
      /** update stress modes */
      mode[4 + ii * LBQ] += C[0] + C[2] + C[5];
      mode[5 + ii * LBQ] += C[0] - C[2];
      mode[6 + ii * LBQ] += C[0] + C[2] - 2.f*C[5];
      mode[7 + ii * LBQ] += C[1];
      mode[8 + ii * LBQ] += C[3];
      mode[9 + ii * LBQ] += C[4];
    
#ifdef EXTERNAL_FORCES
      if(para.external_force){
        node_f.force[(0 + ii*3 ) * para.number_of_nodes + index] = para.ext_force[0]*force_factor;
        node_f.force[(1 + ii*3 ) * para.number_of_nodes + index] = para.ext_force[1]*force_factor;
        node_f.force[(2 + ii*3 ) * para.number_of_nodes + index] = para.ext_force[2]*force_factor;
      }
      else{
      node_f.force[(0 + ii*3 ) * para.number_of_nodes + index] = 0.f;
      node_f.force[(1 + ii*3 ) * para.number_of_nodes + index] = 0.f;
      node_f.force[(2 + ii*3 ) * para.number_of_nodes + index] = 0.f;
      }
#else
      /** reset force */
      node_f.force[(0 + ii*3 ) * para.number_of_nodes + index] = 0.f;
      node_f.force[(1 + ii*3 ) * para.number_of_nodes + index] = 0.f;
      node_f.force[(2 + ii*3 ) * para.number_of_nodes + index] = 0.f;
#endif
  }
}

/**function used to calculate hydrodynamic fields in MD units.
 * @param n_a		Pointer to local node residing in array a for boundary flag(Input)
 * @param mode		Pointer to the local register values mode (Input)
 * @param d_p_v         Pointer to local print values (Output)
 * @param d_v           Pointer to local device values (Input)
 * @param index		node index / thread index (Input)
*/
__device__ void calc_values_in_MD_units(LB_nodes_gpu n_a, float *mode,  LB_rho_v_pi_gpu *d_p_v, LB_rho_v_gpu * d_v, unsigned int index, unsigned int print_index) {
  
  float j[3]; 
  float pi_eq[6] ; 
  float pi[6]={0.f,0.f,0.f,0.f,0.f,0.f};
  float rho_tot=0.f;

  if(n_a.boundary[index] == 0) {

    for(int ii= 0; ii < LB_COMPONENTS; ii++) {
   	  rho_tot += d_v[index].rho[ii];
      d_p_v[print_index].rho[ii] = d_v[index].rho[ii] / para.agrid / para.agrid / para.agrid;
    }
      
    d_p_v[print_index].v[0] = d_v[index].v[0] / para.tau / para.agrid;
    d_p_v[print_index].v[1] = d_v[index].v[1] / para.tau / para.agrid;
    d_p_v[print_index].v[2] = d_v[index].v[2] / para.tau / para.agrid;

    /* stress calculation */ 
    for(int ii = 0; ii < LB_COMPONENTS; ii++) {
      float Rho = d_v[index].rho[ii];
      
      /* note that d_v[index].v[] already includes the 1/2 f term, accounting for the pre- and post-collisional average */
      j[0] = Rho * d_v[index].v[0];
      j[1] = Rho * d_v[index].v[1];
      j[2] = Rho * d_v[index].v[2];
      
      /* equilibrium part of the stress modes */
      pi_eq[0] = ( j[0]*j[0] + j[1]*j[1] + j[2]*j[2] ) / Rho;
      pi_eq[1] = ( j[0]*j[0] - j[1]*j[1] )/ Rho;
      pi_eq[2] = ( j[0]*j[0] + j[1]*j[1] + j[2]*j[2] - 3.0*j[2]*j[2] ) / Rho;
      pi_eq[3] = j[0]*j[1] / Rho;
      pi_eq[4] = j[0]*j[2] / Rho;
      pi_eq[5] = j[1]*j[2] / Rho;
     
      /* Now we must predict the outcome of the next collision */
      /* We immediately average pre- and post-collision.  */
      /* TODO: need a reference for this.   */
      mode[4 + ii * LBQ ] = pi_eq[0] + (0.5 + 0.5*para.gamma_bulk[ii] ) * (mode[4 + ii * LBQ] - pi_eq[0]);
      mode[5 + ii * LBQ ] = pi_eq[1] + (0.5 + 0.5*para.gamma_shear[ii]) * (mode[5 + ii * LBQ] - pi_eq[1]);
      mode[6 + ii * LBQ ] = pi_eq[2] + (0.5 + 0.5*para.gamma_shear[ii]) * (mode[6 + ii * LBQ] - pi_eq[2]);
      mode[7 + ii * LBQ ] = pi_eq[3] + (0.5 + 0.5*para.gamma_shear[ii]) * (mode[7 + ii * LBQ] - pi_eq[3]);
      mode[8 + ii * LBQ ] = pi_eq[4] + (0.5 + 0.5*para.gamma_shear[ii]) * (mode[8 + ii * LBQ] - pi_eq[4]);
      mode[9 + ii * LBQ ] = pi_eq[5] + (0.5 + 0.5*para.gamma_shear[ii]) * (mode[9 + ii * LBQ] - pi_eq[5]);
     
      /* Now we have to transform to the "usual" stress tensor components */
      /* We use eq. 116ff in Duenweg Ladd for that. */
      pi[0] += ( mode[0 + ii * LBQ] + mode[4 + ii * LBQ] + mode[5 + ii * LBQ] ) / 3.0;
      pi[2] += ( 2*mode[0 + ii * LBQ] + 2*mode[4 + ii * LBQ] - mode[5 + ii * LBQ] + 3*mode[6 + ii * LBQ] ) / 6.;
      pi[5] += ( 2*mode[0 + ii * LBQ] + 2*mode[4 + ii * LBQ] - mode[5 + ii * LBQ] + 3*mode[6 + ii * LBQ ]) / 6.;
      pi[1] += mode[7 + ii * LBQ];
      pi[3] += mode[8 + ii * LBQ];
      pi[4] += mode[9 + ii * LBQ];
    }
     
    for(int i = 0; i < 6; i++) {
      d_p_v[print_index].pi[i] = pi[i]  /para.tau / para.tau / para.agrid / para.agrid / para.agrid;
    }
  }
  else {
    for(int ii = 0; ii < LB_COMPONENTS; ii++)
	    d_p_v[print_index].rho[ii] = 0.0f;
     
    for(int i = 0; i < 3; i++)
     	d_p_v[print_index].v[i] = 0.0f;
     	
    for(int i = 0; i < 6; i++)
     	d_p_v[print_index].pi[i] = 0.0f;
  }
}

/**function used to calc physical values of every node
 * @param n_a		Pointer to local node residing in array a for boundary flag(Input)
 * @param mode		Pointer to the local register values mode (Input)
 * @param d_v		Pointer to local device values (Input/Output)
 * @param index		node index / thread index (Input)
*/

/* FIXME this function is basically un-used, think about removing/replacing it */
__device__ void calc_values(LB_nodes_gpu n_a, float *mode, LB_rho_v_gpu *d_v, LB_node_force_gpu node_f, unsigned int index){ 

  float Rho_tot=0.f;
  float u_tot[3]={0.f,0.f,0.f};

  if(n_a.boundary[index] != 1){
      #pragma unroll
      for(int ii=0;ii<LB_COMPONENTS;++ii) { 
          /** re-construct the real density
          * remember that the populations are stored as differences to their
          * equilibrium value */
          d_v[index].rho[ii]= mode[0 + ii * 4]+ para.rho[ii]*para.agrid*para.agrid*para.agrid;
          Rho_tot  += mode[0 + ii * 4]+ para.rho[ii]*para.agrid*para.agrid*para.agrid;
          u_tot[0] += mode[1 + ii * 4];
          u_tot[1] += mode[2 + ii * 4];
          u_tot[2] += mode[3 + ii * 4];
    
          /** if forces are present, the momentum density is redefined to
          * inlcude one half-step of the force action.  See the
          * Chapman-Enskog expansion in [Ladd & Verberg]. */
    
          u_tot[0] += 0.5f*node_f.force[(0+ii*3)*para.number_of_nodes + index];
          u_tot[1] += 0.5f*node_f.force[(1+ii*3)*para.number_of_nodes + index];
          u_tot[2] += 0.5f*node_f.force[(2+ii*3)*para.number_of_nodes + index];
      }
      u_tot[0]/=Rho_tot;
      u_tot[1]/=Rho_tot;
      u_tot[2]/=Rho_tot;
    
      d_v[index].v[0]=u_tot[0]; 
      d_v[index].v[1]=u_tot[1]; 
      d_v[index].v[2]=u_tot[2]; 
  } else { 
    #pragma unroll
    for(int ii=0;ii<LB_COMPONENTS;++ii) { 
       d_v[index].rho[ii]   = 1.;
    }
    d_v[index].v[0] = 0.;
    d_v[index].v[1] = 0.; 
    d_v[index].v[2] = 0.; 
  }   
}


/** 
 * @param node_index	node index around (8) particle (Input)
 * @param *mode			Pointer to the local register values mode (Output)
 * @param n_a			Pointer to local node residing in array a(Input)
*/
__device__ void calc_mode(float *mode, LB_nodes_gpu n_a, unsigned int node_index, int component_index){
	
  /** mass mode */
  mode[0] = n_a.vd[(0 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(1 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(2 + component_index*LBQ ) * para.number_of_nodes + node_index] 
          + n_a.vd[(3 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(4 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(5 + component_index*LBQ ) * para.number_of_nodes + node_index]
          + n_a.vd[(6 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(7 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(8 + component_index*LBQ ) * para.number_of_nodes + node_index]
          + n_a.vd[(9 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(10 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(11 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(12 + component_index*LBQ ) * para.number_of_nodes + node_index]
          + n_a.vd[(13 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(14 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(15 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(16 + component_index*LBQ ) * para.number_of_nodes + node_index]
          + n_a.vd[(17 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(18 + component_index*LBQ ) * para.number_of_nodes + node_index];

  /** momentum modes */
  mode[1] = (n_a.vd[(1 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(2 + component_index*LBQ ) * para.number_of_nodes + node_index]) + (n_a.vd[(7 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(8 + component_index*LBQ ) * para.number_of_nodes + node_index])
          + (n_a.vd[(9 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(10 + component_index*LBQ ) * para.number_of_nodes + node_index]) + (n_a.vd[(11 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(12 + component_index*LBQ ) * para.number_of_nodes + node_index])
          + (n_a.vd[(13 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(14 + component_index*LBQ ) * para.number_of_nodes + node_index]);
  mode[2] = (n_a.vd[(3 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(4 + component_index*LBQ ) * para.number_of_nodes + node_index]) + (n_a.vd[(7 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(8 + component_index*LBQ ) * para.number_of_nodes + node_index])
          - (n_a.vd[(9 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(10 + component_index*LBQ ) * para.number_of_nodes + node_index]) + (n_a.vd[(15 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(16 + component_index*LBQ ) * para.number_of_nodes + node_index])
          + (n_a.vd[(17 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(18 + component_index*LBQ ) * para.number_of_nodes + node_index]);
  mode[3] = (n_a.vd[(5 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(6 + component_index*LBQ ) * para.number_of_nodes + node_index]) + (n_a.vd[(11 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(12 + component_index*LBQ ) * para.number_of_nodes + node_index])
          - (n_a.vd[(13 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(14 + component_index*LBQ ) * para.number_of_nodes + node_index]) + (n_a.vd[(15 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(16 + component_index*LBQ ) * para.number_of_nodes + node_index])
          - (n_a.vd[(17 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(18 + component_index*LBQ ) * para.number_of_nodes + node_index]);
}


/**calculate temperature of the fluid kernel
 * @param *cpu_jsquared			Pointer to result storage value (Output)
 * @param n_a				Pointer to local node residing in array a (Input)
*/
__global__ void temperature(LB_nodes_gpu n_a, float *cpu_jsquared) {
  float mode[4];
  float jsquared = 0.f;
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
    if(!n_a.boundary[index]){
     for(int ii=0;ii<LB_COMPONENTS;++ii) {  
         calc_mode(mode, n_a, index,ii);
         jsquared = mode[1]*mode[1]+mode[2]*mode[2]+mode[3]*mode[3];
         atomicadd(cpu_jsquared, jsquared);
     }
   }
 }
}


/*********************************************************/
/** \name Coupling part */
/*********************************************************/
/**(Eq. (12) Ahlrichs and Duenweg, JCP 111(17):8225 (1999))
 * @param n_a			Pointer to local node residing in array a (Input)
 * @param *delta		Pointer for the weighting of particle position (Output)
 * @param *delta_j		Pointer for the weighting of particle momentum (Output)
 * @param *particle_data	Pointer to the particle position and velocity (Input)
 * @param *particle_force	Pointer to the particle force (Input)
 * @param part_index		particle id / thread id (Input)
 * @param *rn_part		Pointer to randomnumber array of the particle
 * @param node_index		node index around (8) particle (Output)
*/
__device__ void calc_viscous_force(LB_nodes_gpu n_a, float *delta, float * partgrad1, float * partgrad2, float * partgrad3, CUDA_particle_data *particle_data, CUDA_particle_force *particle_force, unsigned int part_index, LB_randomnr_gpu *rn_part, float *delta_j, unsigned int *node_index, LB_rho_v_gpu *d_v){
	
 int my_left[3];
 float interpolated_u1, interpolated_u2, interpolated_u3;
 float interpolated_rho[LB_COMPONENTS];
 float temp_delta[6];
 float temp_delta_half[6];
 float viscforce[3*LB_COMPONENTS];
 float scforce[3*LB_COMPONENTS];
 float mode[19*LB_COMPONENTS];
#ifdef SHANCHEN
 float gradrho1, gradrho2, gradrho3;
 float Rho;
#endif 

 #pragma unroll
 for(int ii=0; ii<LB_COMPONENTS; ++ii){ 
   #pragma unroll
   for(int jj=0; jj<3; ++jj){ 
    scforce[jj+ii*3]  =0.f;
    viscforce[jj+ii*3]=0.f;
    delta_j[jj+ii*3]  =0.f;
   }
   #pragma unroll
   for(int jj=0; jj<8; ++jj){ 
    partgrad1[jj+ii*8]=0.f;
    partgrad2[jj+ii*8]=0.f;
    partgrad3[jj+ii*8]=0.f;
   }
 }
 /** see ahlrichs + duenweg page 8227 equ (10) and (11) */
 #pragma unroll
 for(int i=0; i<3; ++i){
   float scaledpos = particle_data[part_index].p[i]/para.agrid - 0.5f;
   my_left[i] = (int)(floorf(scaledpos));
   //printf("scaledpos %f \t myleft: %d \n", scaledpos, my_left[i]);
   temp_delta[3+i] = scaledpos - my_left[i];
   temp_delta[i] = 1.f - temp_delta[3+i];
   /**further value used for interpolation of fluid velocity at part pos near boundaries */
   temp_delta_half[3+i] = (scaledpos - my_left[i])*2.f;
   temp_delta_half[i] = 2.f - temp_delta_half[3+i];
 }

 delta[0] = temp_delta[0] * temp_delta[1] * temp_delta[2];
 delta[1] = temp_delta[3] * temp_delta[1] * temp_delta[2];
 delta[2] = temp_delta[0] * temp_delta[4] * temp_delta[2];
 delta[3] = temp_delta[3] * temp_delta[4] * temp_delta[2];
 delta[4] = temp_delta[0] * temp_delta[1] * temp_delta[5];
 delta[5] = temp_delta[3] * temp_delta[1] * temp_delta[5];
 delta[6] = temp_delta[0] * temp_delta[4] * temp_delta[5];
 delta[7] = temp_delta[3] * temp_delta[4] * temp_delta[5];

 // modulo for negative numbers is strange at best, shift to make sure we are positive
 int x = my_left[0] + para.dim_x;
 int y = my_left[1] + para.dim_y;
 int z = my_left[2] + para.dim_z;

 node_index[0] = x%para.dim_x     + para.dim_x*(y%para.dim_y)     + para.dim_x*para.dim_y*(z%para.dim_z);
 node_index[1] = (x+1)%para.dim_x + para.dim_x*(y%para.dim_y)     + para.dim_x*para.dim_y*(z%para.dim_z);
 node_index[2] = x%para.dim_x     + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*(z%para.dim_z);
 node_index[3] = (x+1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*(z%para.dim_z);
 node_index[4] = x%para.dim_x     + para.dim_x*(y%para.dim_y)     + para.dim_x*para.dim_y*((z+1)%para.dim_z);
 node_index[5] = (x+1)%para.dim_x + para.dim_x*(y%para.dim_y)     + para.dim_x*para.dim_y*((z+1)%para.dim_z);
 node_index[6] = x%para.dim_x     + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z);
 node_index[7] = (x+1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z);

 particle_force[part_index].f[0] = 0.f;
 particle_force[part_index].f[1] = 0.f;
 particle_force[part_index].f[2] = 0.f;

 interpolated_u1 = interpolated_u2 = interpolated_u3 = 0.f;
 #pragma unroll
 for(int i=0; i<8; ++i){
    float totmass=0.f;
    calc_m_from_n(n_a,node_index[i],mode);
    #pragma unroll
    for(int ii=0;ii<LB_COMPONENTS;ii++){
	totmass+=mode[0]+para.rho[ii]*para.agrid*para.agrid*para.agrid;
    } 
#ifndef SHANCHEN
    interpolated_u1 += (mode[1]/totmass)*delta[i];
    interpolated_u2 += (mode[2]/totmass)*delta[i];
    interpolated_u3 += (mode[3]/totmass)*delta[i];
#else //SHANCHEN
    interpolated_u1 += d_v[node_index[i]].v[0]/8.;  
    interpolated_u2 += d_v[node_index[i]].v[1]/8.;
    interpolated_u3 += d_v[node_index[i]].v[2]/8.;
#endif
 }

#ifdef SHANCHEN
 #pragma unroll
 for(int ii=0; ii<LB_COMPONENTS; ++ii){ 
  float solvation2 = particle_data[part_index].solvation[2*ii + 1];
   
  interpolated_rho[ii]  = 0.f;
  gradrho1 = gradrho2 = gradrho3 = 0.f;
  
 // TODO: should one introduce a density-dependent friction ?
  calc_mode(mode, n_a, node_index[0],ii);
  Rho = mode[0] + para.rho[ii]*para.agrid*para.agrid*para.agrid;
  interpolated_rho[ii] += delta[0] * Rho; 
  partgrad1[ii*8 + 0] += Rho * solvation2;
  partgrad2[ii*8 + 0] += Rho * solvation2;
  partgrad3[ii*8 + 0] += Rho * solvation2;
  gradrho1 -=(delta[0] + delta[1]) * Rho; 
  gradrho2 -=(delta[0] + delta[2]) * Rho; 
  gradrho3 -=(delta[0] + delta[4]) * Rho; 

  calc_mode(mode, n_a, node_index[1],ii); 
  Rho = mode[0] +  para.rho[ii]*para.agrid*para.agrid*para.agrid; 
  interpolated_rho[ii] += delta[1] * Rho; 
  partgrad1[ii*8 + 1] -= Rho * solvation2;
  partgrad2[ii*8 + 1] += Rho * solvation2;
  partgrad3[ii*8 + 1] += Rho * solvation2;
  gradrho1 +=(delta[1] + delta[0]) * Rho; 
  gradrho2 -=(delta[1] + delta[3]) * Rho; 
  gradrho3 -=(delta[1] + delta[5]) * Rho; 
  
  calc_mode(mode, n_a, node_index[2],ii);
  Rho = mode[0] + para.rho[ii]*para.agrid*para.agrid*para.agrid;	
  interpolated_rho[ii] += delta[2] * Rho; 
  partgrad1[ii*8 + 2] += Rho * solvation2;
  partgrad2[ii*8 + 2] -= Rho * solvation2;
  partgrad3[ii*8 + 2] += Rho * solvation2;
  gradrho1 -=(delta[2] + delta[3]) * Rho; 
  gradrho2 +=(delta[2] + delta[0]) * Rho; 
  gradrho3 -=(delta[2] + delta[6]) * Rho; 

  calc_mode(mode, n_a, node_index[3],ii);
  Rho = mode[0] + para.rho[ii]*para.agrid*para.agrid*para.agrid;	
  interpolated_rho[ii] += delta[3] * Rho; 
  partgrad1[ii*8 + 3] -= Rho * solvation2;
  partgrad2[ii*8 + 3] -= Rho * solvation2;
  partgrad3[ii*8 + 3] += Rho * solvation2;
  gradrho1 +=(delta[3] + delta[2]) * Rho; 
  gradrho2 +=(delta[3] + delta[1]) * Rho; 
  gradrho3 -=(delta[3] + delta[7]) * Rho; 

  calc_mode(mode, n_a, node_index[4],ii);
  Rho = mode[0] + para.rho[ii]*para.agrid*para.agrid*para.agrid;	
  interpolated_rho[ii] += delta[4] * Rho; 
  partgrad1[ii*8 + 4] += Rho * solvation2;
  partgrad2[ii*8 + 4] += Rho * solvation2;
  partgrad3[ii*8 + 4] -= Rho * solvation2;
  gradrho1 -=(delta[4] + delta[5]) * Rho; 
  gradrho2 -=(delta[4] + delta[6]) * Rho; 
  gradrho3 +=(delta[4] + delta[0]) * Rho; 

  calc_mode(mode, n_a, node_index[5],ii);
  Rho = mode[0] + para.rho[ii]*para.agrid*para.agrid*para.agrid;	
  interpolated_rho[ii] += delta[5] * Rho; 
  partgrad1[ii*8 + 5] -= Rho * solvation2;
  partgrad2[ii*8 + 5] += Rho * solvation2;
  partgrad3[ii*8 + 5] -= Rho * solvation2;
  gradrho1 +=(delta[5] + delta[4]) * Rho; 
  gradrho2 -=(delta[5] + delta[7]) * Rho; 
  gradrho3 +=(delta[5] + delta[1]) * Rho; 

  calc_mode(mode, n_a, node_index[6],ii);
  Rho = mode[0] + para.rho[ii]*para.agrid*para.agrid*para.agrid;	
  interpolated_rho[ii] += delta[6] * Rho; 
  partgrad1[ii*8 + 6] += Rho * solvation2;
  partgrad2[ii*8 + 6] -= Rho * solvation2;
  partgrad3[ii*8 + 6] -= Rho * solvation2;
  gradrho1 -=(delta[6] + delta[7]) * Rho; 
  gradrho2 +=(delta[6] + delta[4]) * Rho; 
  gradrho3 +=(delta[6] + delta[2]) * Rho; 

  calc_mode(mode, n_a, node_index[7],ii);
  Rho = mode[0] + para.rho[ii]*para.agrid*para.agrid*para.agrid;	
  interpolated_rho[ii] += delta[7] * Rho; 
  partgrad1[ii*8 + 7] -= Rho * solvation2;
  partgrad2[ii*8 + 7] -= Rho * solvation2;
  partgrad3[ii*8 + 7] -= Rho * solvation2;
  gradrho1 +=(delta[7] + delta[6]) * Rho; 
  gradrho2 +=(delta[7] + delta[5]) * Rho; 
  gradrho3 +=(delta[7] + delta[3]) * Rho; 

  /* normalize the gradient to md units TODO: is that correct?*/
  gradrho1 *= para.agrid; 
  gradrho2 *= para.agrid; 
  gradrho3 *= para.agrid; 

  scforce[0+ii*3] += particle_data[part_index].solvation[2*ii] * gradrho1 ; 
  scforce[1+ii*3] += particle_data[part_index].solvation[2*ii] * gradrho2 ;
  scforce[2+ii*3] += particle_data[part_index].solvation[2*ii] * gradrho3 ;
  /* scforce is used also later...*/
  particle_force[part_index].f[0] += scforce[0+ii*3];
  particle_force[part_index].f[1] += scforce[1+ii*3];
  particle_force[part_index].f[2] += scforce[2+ii*3];
 }

#else // SHANCHEN is not defined
 /* for LB we do not reweight the friction force */
 for(int ii=0; ii<LB_COMPONENTS; ++ii){ 
	interpolated_rho[ii]=1.0;
 }

#endif // SHANCHEN

  /** calculate viscous force
   * take care to rescale velocities with time_step and transform to MD units
   * (Eq. (9) Ahlrichs and Duenweg, JCP 111(17):8225 (1999)) */
 float rhotot=0;

 #pragma unroll
 for(int ii=0; ii<LB_COMPONENTS; ++ii){ 
	rhotot+=interpolated_rho[ii];
 }


 /* Viscous force */

 for(int ii=0; ii<LB_COMPONENTS; ++ii){ 
  viscforce[0+ii*3] -= interpolated_rho[ii]*para.friction[ii]*(particle_data[part_index].v[0]/para.time_step - interpolated_u1*para.agrid/para.tau)/rhotot;
  viscforce[1+ii*3] -= interpolated_rho[ii]*para.friction[ii]*(particle_data[part_index].v[1]/para.time_step - interpolated_u2*para.agrid/para.tau)/rhotot;
  viscforce[2+ii*3] -= interpolated_rho[ii]*para.friction[ii]*(particle_data[part_index].v[2]/para.time_step - interpolated_u3*para.agrid/para.tau)/rhotot;

#ifdef LB_ELECTROHYDRODYNAMICS
  viscforce[0+ii*3] += interpolated_rho[ii]*para.friction[ii] * particle_data[part_index].mu_E[0]/rhotot;
  viscforce[1+ii*3] += interpolated_rho[ii]*para.friction[ii] * particle_data[part_index].mu_E[1]/rhotot;
  viscforce[2+ii*3] += interpolated_rho[ii]*para.friction[ii] * particle_data[part_index].mu_E[2]/rhotot;
#endif

  /** add stochastic force of zero mean (Ahlrichs, Duenweg equ. 15)*/
#ifdef GAUSSRANDOM
  gaussian_random(rn_part);
  viscforce[0+ii*3] += para.lb_coupl_pref2[ii]*rn_part->randomnr[0];
  viscforce[1+ii*3] += para.lb_coupl_pref2[ii]*rn_part->randomnr[1];
  gaussian_random(rn_part);
  viscforce[2+ii*3] += para.lb_coupl_pref2[ii]*rn_part->randomnr[0];
#else
  random_01(rn_part);
  viscforce[0+ii*3] += para.lb_coupl_pref[ii]*(rn_part->randomnr[0]-0.5f);
  viscforce[1+ii*3] += para.lb_coupl_pref[ii]*(rn_part->randomnr[1]-0.5f);
  random_01(rn_part);
  viscforce[2+ii*3] += para.lb_coupl_pref[ii]*(rn_part->randomnr[0]-0.5f);
#endif	  
  /** delta_j for transform momentum transfer to lattice units which is done in calc_node_force
  (Eq. (12) Ahlrichs and Duenweg, JCP 111(17):8225 (1999)) */

  particle_force[part_index].f[0] += viscforce[0+ii*3];
  particle_force[part_index].f[1] += viscforce[1+ii*3];
  particle_force[part_index].f[2] += viscforce[2+ii*3];
  /* the average force from the particle to surrounding nodes is transmitted back to preserve momentum */
  for(int node=0 ; node < 8 ; node++ ) { 
     particle_force[part_index].f[0] -= partgrad1[node+ii*8]/8.;
     particle_force[part_index].f[1] -= partgrad2[node+ii*8]/8.;
     particle_force[part_index].f[2] -= partgrad3[node+ii*8]/8.;
  }
  /* note that scforce is zero if SHANCHEN is not #defined */
  delta_j[0+3*ii] -= (scforce[0+ii*3]+viscforce[0+ii*3])*para.time_step*para.tau/para.agrid;
  delta_j[1+3*ii] -= (scforce[1+ii*3]+viscforce[1+ii*3])*para.time_step*para.tau/para.agrid;
  delta_j[2+3*ii] -= (scforce[2+ii*3]+viscforce[2+ii*3])*para.time_step*para.tau/para.agrid;  	
 }
}

/**calcutlation of the node force caused by the particles, with atomicadd due to avoiding race conditions 
	(Eq. (14) Ahlrichs and Duenweg, JCP 111(17):8225 (1999))
 * @param *delta		Pointer for the weighting of particle position (Input)
 * @param *delta_j		Pointer for the weighting of particle momentum (Input)
 * @param node_index		node index around (8) particle (Input)
 * @param node_f    		Pointer to the node force (Output).
*/
__device__ void calc_node_force(float *delta, float *delta_j, float * partgrad1, float * partgrad2, float * partgrad3,  unsigned int *node_index, LB_node_force_gpu node_f){
/* TODO: should the drag depend on the density?? */
/* NOTE: partgrad is not zero only if SHANCHEN is defined. It is initialized in calc_node_force. Alternatively one could 
         specialize this function to the single component LB */ 
 for(int ii=0; ii < LB_COMPONENTS; ++ii) { 
  atomicadd(&(node_f.force[(0+ii*3)*para.number_of_nodes + node_index[0]]), (delta[0]*delta_j[0+ii*3] + partgrad1[ii*8+0]));
  atomicadd(&(node_f.force[(1+ii*3)*para.number_of_nodes + node_index[0]]), (delta[0]*delta_j[1+ii*3] + partgrad2[ii*8+0]));
  atomicadd(&(node_f.force[(2+ii*3)*para.number_of_nodes + node_index[0]]), (delta[0]*delta_j[2+ii*3] + partgrad3[ii*8+0]));
                                                                                                    
  atomicadd(&(node_f.force[(0+ii*3)*para.number_of_nodes + node_index[1]]), (delta[1]*delta_j[0+ii*3] + partgrad1[ii*8+1]));
  atomicadd(&(node_f.force[(1+ii*3)*para.number_of_nodes + node_index[1]]), (delta[1]*delta_j[1+ii*3] + partgrad2[ii*8+1]));
  atomicadd(&(node_f.force[(2+ii*3)*para.number_of_nodes + node_index[1]]), (delta[1]*delta_j[2+ii*3] + partgrad3[ii*8+1]));
                                                                                                    
  atomicadd(&(node_f.force[(0+ii*3)*para.number_of_nodes + node_index[2]]), (delta[2]*delta_j[0+ii*3] + partgrad1[ii*8+2]));
  atomicadd(&(node_f.force[(1+ii*3)*para.number_of_nodes + node_index[2]]), (delta[2]*delta_j[1+ii*3] + partgrad2[ii*8+2]));
  atomicadd(&(node_f.force[(2+ii*3)*para.number_of_nodes + node_index[2]]), (delta[2]*delta_j[2+ii*3] + partgrad3[ii*8+2]));
                                                                                                    
  atomicadd(&(node_f.force[(0+ii*3)*para.number_of_nodes + node_index[3]]), (delta[3]*delta_j[0+ii*3] + partgrad1[ii*8+3]));
  atomicadd(&(node_f.force[(1+ii*3)*para.number_of_nodes + node_index[3]]), (delta[3]*delta_j[1+ii*3] + partgrad2[ii*8+3]));
  atomicadd(&(node_f.force[(2+ii*3)*para.number_of_nodes + node_index[3]]), (delta[3]*delta_j[2+ii*3] + partgrad3[ii*8+3]));
                                                                                                    
  atomicadd(&(node_f.force[(0+ii*3)*para.number_of_nodes + node_index[4]]), (delta[4]*delta_j[0+ii*3] + partgrad1[ii*8+4]));
  atomicadd(&(node_f.force[(1+ii*3)*para.number_of_nodes + node_index[4]]), (delta[4]*delta_j[1+ii*3] + partgrad2[ii*8+4]));
  atomicadd(&(node_f.force[(2+ii*3)*para.number_of_nodes + node_index[4]]), (delta[4]*delta_j[2+ii*3] + partgrad3[ii*8+4]));
                                                                                                    
  atomicadd(&(node_f.force[(0+ii*3)*para.number_of_nodes + node_index[5]]), (delta[5]*delta_j[0+ii*3] + partgrad1[ii*8+5]));
  atomicadd(&(node_f.force[(1+ii*3)*para.number_of_nodes + node_index[5]]), (delta[5]*delta_j[1+ii*3] + partgrad2[ii*8+5]));
  atomicadd(&(node_f.force[(2+ii*3)*para.number_of_nodes + node_index[5]]), (delta[5]*delta_j[2+ii*3] + partgrad3[ii*8+5]));
                                                                                                    
  atomicadd(&(node_f.force[(0+ii*3)*para.number_of_nodes + node_index[6]]), (delta[6]*delta_j[0+ii*3] + partgrad1[ii*8+6]));
  atomicadd(&(node_f.force[(1+ii*3)*para.number_of_nodes + node_index[6]]), (delta[6]*delta_j[1+ii*3] + partgrad2[ii*8+6]));
  atomicadd(&(node_f.force[(2+ii*3)*para.number_of_nodes + node_index[6]]), (delta[6]*delta_j[2+ii*3] + partgrad3[ii*8+6]));
                                                                                                    
  atomicadd(&(node_f.force[(0+ii*3)*para.number_of_nodes + node_index[7]]), (delta[7]*delta_j[0+ii*3] + partgrad1[ii*8+7]));
  atomicadd(&(node_f.force[(1+ii*3)*para.number_of_nodes + node_index[7]]), (delta[7]*delta_j[1+ii*3] + partgrad2[ii*8+7]));
  atomicadd(&(node_f.force[(2+ii*3)*para.number_of_nodes + node_index[7]]), (delta[7]*delta_j[2+ii*3] + partgrad3[ii*8+7]));
 }
}


/*********************************************************/
/** \name System setup and Kernel functions */
/*********************************************************/

/**kernel to calculate local populations from hydrodynamic fields given by the tcl values.
 * The mapping is given in terms of the equilibrium distribution.
 *
 * Eq. (2.15) Ladd, J. Fluid Mech. 271, 295-309 (1994)
 * Eq. (4) in Berk Usta, Ladd and Butler, JCP 122, 094902 (2005)
 *
 * @param n_a		 Pointer to the lattice site (Input).
 * @param *gpu_check additional check if gpu kernel are executed(Input).
*/
__global__ void calc_n_equilibrium(LB_nodes_gpu n_a, LB_rho_v_gpu *d_v, LB_node_force_gpu node_f, int *gpu_check) {
   /* TODO: this can handle only a uniform density, somehting similar, but local, 
            has to be called every time the fields are set by the user ! */ 
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
  if(index<para.number_of_nodes){
       float mode[19*LB_COMPONENTS];
       #pragma unroll
       for(int ii=0;ii<LB_COMPONENTS;++ii) { 
     
         /** default values for fields in lattice units */
         gpu_check[0] = 1;
     
         float Rho = para.rho[ii]*para.agrid*para.agrid*para.agrid;
         float v[3] = { 0.0f, 0.0f, 0.0f };
         float pi[6] = { Rho*c_sound_sq, 0.0f, Rho*c_sound_sq, 0.0f, 0.0f, Rho*c_sound_sq };
     
         float rhoc_sq = Rho*c_sound_sq;
         float avg_rho = para.rho[ii]*para.agrid*para.agrid*para.agrid;
         float local_rho, local_j[3], *local_pi, trace;
     
         local_rho  = Rho;
     
         local_j[0] = Rho * v[0];
         local_j[1] = Rho * v[1];
         local_j[2] = Rho * v[2];
     
         local_pi = pi;
     
         /** reduce the pressure tensor to the part needed here. NOTE: this not true anymore for SHANCHEN if the densities are not uniform. FIXME*/
         local_pi[0] -= rhoc_sq;
         local_pi[2] -= rhoc_sq;
         local_pi[5] -= rhoc_sq;
     
         trace = local_pi[0] + local_pi[2] + local_pi[5];
     
         float rho_times_coeff;
         float tmp1,tmp2;
     
         /** update the q=0 sublattice */
         n_a.vd[(0 + ii*LBQ ) * para.number_of_nodes + index] = 1.f/3.f * (local_rho-avg_rho) - 1.f/2.f*trace;
     
         /** update the q=1 sublattice */
         rho_times_coeff = 1.f/18.f * (local_rho-avg_rho);
     
         n_a.vd[(1 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff + 1.f/6.f*local_j[0] + 1.f/4.f*local_pi[0] - 1.f/12.f*trace;
         n_a.vd[(2 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff - 1.f/6.f*local_j[0] + 1.f/4.f*local_pi[0] - 1.f/12.f*trace;
         n_a.vd[(3 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff + 1.f/6.f*local_j[1] + 1.f/4.f*local_pi[2] - 1.f/12.f*trace;
         n_a.vd[(4 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff - 1.f/6.f*local_j[1] + 1.f/4.f*local_pi[2] - 1.f/12.f*trace;
         n_a.vd[(5 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff + 1.f/6.f*local_j[2] + 1.f/4.f*local_pi[5] - 1.f/12.f*trace;
         n_a.vd[(6 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff - 1.f/6.f*local_j[2] + 1.f/4.f*local_pi[5] - 1.f/12.f*trace;
     
         /** update the q=2 sublattice */
         rho_times_coeff = 1.f/36.f * (local_rho-avg_rho);
     
         tmp1 = local_pi[0] + local_pi[2];
         tmp2 = 2.0f*local_pi[1];
         n_a.vd[(7 + ii*LBQ ) * para.number_of_nodes + index]  = rho_times_coeff + 1.f/12.f*(local_j[0]+local_j[1]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
         n_a.vd[(8 + ii*LBQ ) * para.number_of_nodes + index]  = rho_times_coeff - 1.f/12.f*(local_j[0]+local_j[1]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
         n_a.vd[(9 + ii*LBQ ) * para.number_of_nodes + index]  = rho_times_coeff + 1.f/12.f*(local_j[0]-local_j[1]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;
         n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff - 1.f/12.f*(local_j[0]-local_j[1]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;
     
         tmp1 = local_pi[0] + local_pi[5];
         tmp2 = 2.0f*local_pi[3];
     
         n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff + 1.f/12.f*(local_j[0]+local_j[2]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
         n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff - 1.f/12.f*(local_j[0]+local_j[2]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
         n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff + 1.f/12.f*(local_j[0]-local_j[2]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;
         n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff - 1.f/12.f*(local_j[0]-local_j[2]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;
     
         tmp1 = local_pi[2] + local_pi[5];
         tmp2 = 2.0f*local_pi[4];
     
         n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff + 1.f/12.f*(local_j[1]+local_j[2]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
         n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff - 1.f/12.f*(local_j[1]+local_j[2]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
         n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff + 1.f/12.f*(local_j[1]-local_j[2]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;
         n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff - 1.f/12.f*(local_j[1]-local_j[2]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;
     
         /**set different seed for randomgen on every node */
         n_a.seed[index] = para.your_seed + index;
       }
       calc_m_from_n(n_a,index,mode);
       update_rho_v(mode,index,node_f,d_v);
  }
}

/** kernel to calculate local populations from hydrodynamic fields
 * from given flow field velocities.  The mapping is given in terms of
 * the equilibrium distribution.
 *
 * Eq. (2.15) Ladd, J. Fluid Mech. 271, 295-309 (1994)
 * Eq. (4) in Berk Usta, Ladd and Butler, JCP 122, 094902 (2005)
 *
 * @param n_a		   the current nodes array (double buffering!)
 * @param single_nodeindex the node to set the velocity for
 * @param velocity         the velocity to set
 */
__global__ void set_u_equilibrium(LB_nodes_gpu n_a, int single_nodeindex,float *velocity) {

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index == 0){
  float v[3];
  float mode[4*LB_COMPONENTS];
  float rhoc_sq,avg_rho;
  float local_rho, local_j[3], *local_pi, trace;
  v[0] = velocity[0];
  v[1] = velocity[1];
  v[2] = velocity[2];
  #pragma unroll
  for(int ii=0;ii<LB_COMPONENTS;++ii) { 

    /** default values for fields in lattice units */
    calc_mode(&mode[4*ii], n_a, single_nodeindex,ii);
    float Rho = mode[0*4*ii] + para.rho[ii]*para.agrid*para.agrid*para.agrid; 

    float pi[6] = { Rho*c_sound_sq, 0.0f, Rho*c_sound_sq, 0.0f, 0.0f, Rho*c_sound_sq };

    rhoc_sq = Rho*c_sound_sq;
    avg_rho = para.rho[ii]*para.agrid*para.agrid*para.agrid;

    local_rho  = Rho;

    local_j[0] = Rho * v[0];
    local_j[1] = Rho * v[1];
    local_j[2] = Rho * v[2];


    local_pi = pi;

    /** reduce the pressure tensor to the part needed here. NOTE: this not true anymore for SHANCHEN if the densities are not uniform. FIXME*/
    /* there is much duplicated code from calc_n_equilibrium(). FIXME */
    local_pi[0] -= rhoc_sq; 
    local_pi[2] -= rhoc_sq;
    local_pi[5] -= rhoc_sq;

    trace = local_pi[0] + local_pi[2] + local_pi[5];

    float rho_times_coeff;
    float tmp1,tmp2;

    /** update the q=0 sublattice */
    n_a.vd[(0 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/3.f * (local_rho-avg_rho) - 1.f/2.f*trace;

    /** update the q=1 sublattice */
    rho_times_coeff = 1.f/18.f * (local_rho-avg_rho);

    n_a.vd[(1 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff + 1.f/6.f*local_j[0] + 1.f/4.f*local_pi[0] - 1.f/12.f*trace;
    n_a.vd[(2 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff - 1.f/6.f*local_j[0] + 1.f/4.f*local_pi[0] - 1.f/12.f*trace;
    n_a.vd[(3 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff + 1.f/6.f*local_j[1] + 1.f/4.f*local_pi[2] - 1.f/12.f*trace;
    n_a.vd[(4 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff - 1.f/6.f*local_j[1] + 1.f/4.f*local_pi[2] - 1.f/12.f*trace;
    n_a.vd[(5 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff + 1.f/6.f*local_j[2] + 1.f/4.f*local_pi[5] - 1.f/12.f*trace;
    n_a.vd[(6 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff - 1.f/6.f*local_j[2] + 1.f/4.f*local_pi[5] - 1.f/12.f*trace;

    /** update the q=2 sublattice */
    rho_times_coeff = 1.f/36.f * (local_rho-avg_rho);

    tmp1 = local_pi[0] + local_pi[2];
    tmp2 = 2.0f*local_pi[1];
    n_a.vd[(7 + ii*LBQ ) * para.number_of_nodes + single_nodeindex]  = rho_times_coeff + 1.f/12.f*(local_j[0]+local_j[1]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[(8 + ii*LBQ ) * para.number_of_nodes + single_nodeindex]  = rho_times_coeff - 1.f/12.f*(local_j[0]+local_j[1]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[(9 + ii*LBQ ) * para.number_of_nodes + single_nodeindex]  = rho_times_coeff + 1.f/12.f*(local_j[0]-local_j[1]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;
    n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff - 1.f/12.f*(local_j[0]-local_j[1]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;

    tmp1 = local_pi[0] + local_pi[5];
    tmp2 = 2.0f*local_pi[3];

    n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff + 1.f/12.f*(local_j[0]+local_j[2]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff - 1.f/12.f*(local_j[0]+local_j[2]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff + 1.f/12.f*(local_j[0]-local_j[2]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;
    n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff - 1.f/12.f*(local_j[0]-local_j[2]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;

    tmp1 = local_pi[2] + local_pi[5];
    tmp2 = 2.0f*local_pi[4];

    n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff + 1.f/12.f*(local_j[1]+local_j[2]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff - 1.f/12.f*(local_j[1]+local_j[2]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff + 1.f/12.f*(local_j[1]-local_j[2]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;
    n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff - 1.f/12.f*(local_j[1]-local_j[2]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;

  }
  }
}


/**calculate mass of the whole fluid kernel
 * @param *sum				Pointer to result storage value (Output)
 * @param n_a				Pointer to local node residing in array a (Input)
*/
__global__ void calc_mass(LB_nodes_gpu n_a, float *sum) {
  float mode[4];

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
    for(int ii=0;ii<LB_COMPONENTS;++ii) { 
      calc_mode(mode, n_a, index,ii);
      float Rho = mode[0] + para.rho[ii]*para.agrid*para.agrid*para.agrid;
      atomicadd(&(sum[0]), Rho);
    }
  }
}

/** (re-)initialization of the node force / set up of external force in lb units
 * @param node_f		Pointer to local node force (Input)
*/
__global__ void reinit_node_force(LB_node_force_gpu node_f){

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
   #pragma unroll
   for(int ii=0;ii<LB_COMPONENTS;++ii){
#ifdef EXTERNAL_FORCES
    if(para.external_force){
      node_f.force[(0+ii*3)*para.number_of_nodes + index] = para.ext_force[0]*para.agrid*para.agrid*para.agrid*para.agrid*para.tau*para.tau;
      node_f.force[(1+ii*3)*para.number_of_nodes + index] = para.ext_force[1]*para.agrid*para.agrid*para.agrid*para.agrid*para.tau*para.tau;
      node_f.force[(2+ii*3)*para.number_of_nodes + index] = para.ext_force[2]*para.agrid*para.agrid*para.agrid*para.agrid*para.tau*para.tau;
    }
    else{
      node_f.force[(0+ii*3)*para.number_of_nodes + index] = 0.0f;
      node_f.force[(1+ii*3)*para.number_of_nodes + index] = 0.0f;
      node_f.force[(2+ii*3)*para.number_of_nodes + index] = 0.0f;
    }
#else
    node_f.force[(0+ii*3)*para.number_of_nodes + index] = 0.0f;
    node_f.force[(1+ii*3)*para.number_of_nodes + index] = 0.0f;
    node_f.force[(2+ii*3)*para.number_of_nodes + index] = 0.0f;
#endif
   }
  }
}


/**set extern force on single nodes kernel
 * @param n_extern_nodeforces		number of nodes (Input)
 * @param *extern_nodeforces		Pointer to extern node force array (Input)
 * @param node_f			node force struct (Output)
*/
__global__ void init_extern_nodeforces(int n_extern_nodeforces, LB_extern_nodeforce_gpu *extern_nodeforces, LB_node_force_gpu node_f){

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
  float factor=powf(para.agrid,4)*para.tau*para.tau;
  if(index<n_extern_nodeforces){
   #pragma unroll
   for(int ii=0;ii<LB_COMPONENTS;++ii){
    node_f.force[(0+ii*3)*para.number_of_nodes + extern_nodeforces[index].index] = extern_nodeforces[index].force[0] * factor;
    node_f.force[(1+ii*3)*para.number_of_nodes + extern_nodeforces[index].index] = extern_nodeforces[index].force[1] * factor;
    node_f.force[(2+ii*3)*para.number_of_nodes + extern_nodeforces[index].index] = extern_nodeforces[index].force[2] * factor;
   }
  }
}

#ifdef SHANCHEN

/** 
 * @param single_nodeindex	Single node index        (Input)
 * @param *mode			Pointer to the local register values mode (Output)
 * @param n_a			Pointer to local node residing in array a(Input)
*/
__device__ __inline__ float calc_massmode(LB_nodes_gpu n_a, int single_nodeindex, int component_index){
	
  /** mass mode */
  float mode;
  mode = n_a.vd[(0 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(1 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(2 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] 
          + n_a.vd[(3 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(4 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(5 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex]
          + n_a.vd[(6 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(7 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(8 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex]
          + n_a.vd[(9 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(10 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(11 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(12 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex]
          + n_a.vd[(13 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(14 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(15 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(16 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex]
          + n_a.vd[(17 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(18 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex];
 mode += para.rho[component_index]*para.agrid*para.agrid*para.agrid;

 return mode;
}


__device__ __inline__ void calc_shanchen_contribution(LB_nodes_gpu n_a,int component_index, int x, int y, int z, float *p){ 
      float tmp_p[3]={0.f,0.f,0.f};
      float pseudo;
      int index;
      index  = (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z;
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[0]+=pseudo/18.f;

      index  = (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z;
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[0]-=pseudo/18.f;

      index  = x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z;
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[1]+=pseudo/18.f;

      index  = x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z;
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[1]-=pseudo/18.f;

      index  = x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z);
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[2]+=pseudo/18.f;	

      index  = x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z);
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[2]-=pseudo/18.f;

      index  = (x+1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z;
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[0]+=pseudo/36.f;
      tmp_p[1]+=pseudo/36.f;

      index  = (para.dim_x+x-1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z;
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[0]-=pseudo/36.f;
      tmp_p[1]-=pseudo/36.f;

      index  = (x+1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z;
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[0]+=pseudo/36.f;
      tmp_p[1]-=pseudo/36.f;

      index  = (para.dim_x+x-1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z;
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[0]-=pseudo/36.f;
      tmp_p[1]+=pseudo/36.f;

      index  = (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z);
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[0]+=pseudo/36.f;
      tmp_p[2]+=pseudo/36.f;

      index  = (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z);
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[0]-=pseudo/36.f;
      tmp_p[2]-=pseudo/36.f;

      index  = (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z);
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[0]+=pseudo/36.f;
      tmp_p[2]-=pseudo/36.f;

      index  = (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z);
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[0]-=pseudo/36.f;
      tmp_p[2]+=pseudo/36.f;

      index  = x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z);
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[1]+=pseudo/36.f;
      tmp_p[2]+=pseudo/36.f;

      index  = x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z);
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[1]-=pseudo/36.f;
      tmp_p[2]-=pseudo/36.f;

      index  = x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z);
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[1]+=pseudo/36.f;
      tmp_p[2]-=pseudo/36.f;

      index  = x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z);
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[1]-=pseudo/36.f;
      tmp_p[2]+=pseudo/36.f;
  
      p[0]=tmp_p[0];
      p[1]=tmp_p[1];
      p[2]=tmp_p[2];
}

/** function to calc shanchen forces 
 * @param *mode			Pointer to the local register values mode (Output)
 * @param n_a			Pointer to local node residing in array a(Input)
 * @param node_f		Pointer to local node force (Input)
*/
__global__ void lb_shanchen_GPU(LB_nodes_gpu n_a,LB_node_force_gpu node_f){
#ifndef D3Q19
#error Lattices other than D3Q19 not supported
#endif
#if ( LB_COMPONENTS == 1  ) 
  #warning shanchen forces not implemented 
#else  
  
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int xyz[3];
  float pseudo;

  if(index<para.number_of_nodes){
     /*Let's first identify the neighboring nodes */
     index_to_xyz(index, xyz);
     int x = xyz[0];
     int y = xyz[1];
     int z = xyz[2];
     
     #pragma unroll
     for(int ii=0;ii<LB_COMPONENTS;ii++){ 
       float p[3]={0.f,0.f,0.f};
       pseudo =  calc_massmode(n_a,index,ii);
       #pragma unroll
       for(int jj=0;jj<LB_COMPONENTS;jj++){ 
             float tmpp[3]={0.f,0.f,0.f};
             calc_shanchen_contribution(n_a, jj, x,y,z, tmpp);
// FIXME  coupling HAS to be rescaled with agrid....
             p[0] += - para.coupling[(LB_COMPONENTS)*ii+jj]  * pseudo  * tmpp[0];
             p[1] += - para.coupling[(LB_COMPONENTS)*ii+jj]  * pseudo  * tmpp[1];
             p[2] += - para.coupling[(LB_COMPONENTS)*ii+jj]  * pseudo  * tmpp[2];
       }
       node_f.force[(0+ii*3)*para.number_of_nodes + index]+=p[0];
       node_f.force[(1+ii*3)*para.number_of_nodes + index]+=p[1];
       node_f.force[(2+ii*3)*para.number_of_nodes + index]+=p[2];
     }
  }
#endif 
  return; 
}

#endif //SHANCHEN

/** kernel to set the local density
 *
 * @param n_a		   the current nodes array (double buffering!)
 * @param single_nodeindex the node to set the velocity for
 * @param rho              the density to set
 */
__global__ void set_rho(LB_nodes_gpu n_a,  LB_rho_v_gpu *d_v, int single_nodeindex,float *rho) {

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
  /*Note: this sets the velocities to zero */
  if(index == 0){
     float local_rho;
     #pragma unroll
     for(int ii=0;ii<LB_COMPONENTS;++ii) { 
       /** default values for fields in lattice units */
       local_rho = (rho[ii]-para.rho[ii])*para.agrid*para.agrid*para.agrid;
       d_v[single_nodeindex].rho[ii]=rho[ii];
       n_a.vd[(0  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/3.f * local_rho ;
       n_a.vd[(1  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/18.f * local_rho ;
       n_a.vd[(2  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/18.f * local_rho ;
       n_a.vd[(3  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/18.f * local_rho ;
       n_a.vd[(4  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/18.f * local_rho ;
       n_a.vd[(5  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/18.f * local_rho ;
       n_a.vd[(6  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/18.f * local_rho ;
       n_a.vd[(7  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/36.f * local_rho ;
       n_a.vd[(8  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/36.f * local_rho ;
       n_a.vd[(9  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/36.f * local_rho ;
       n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/36.f * local_rho ;
       n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/36.f * local_rho ;
       n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/36.f * local_rho ;
       n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/36.f * local_rho ;
       n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/36.f * local_rho ;
       n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/36.f * local_rho ;
       n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/36.f * local_rho ;
       n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/36.f * local_rho ;
       n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/36.f * local_rho ;
     }
  }
}

/**set the boundary flag for all boundary nodes
 * @param boundary_node_list    The indices of the boundary nodes
 * @param boundary_index_list   The flag representing the corresponding boundary
 * @param number_of_boundnodes	The number of boundary nodes
 * @param n_a			Pointer to local node residing in array a (Input)
 * @param n_b			Pointer to local node residing in array b (Input)
*/
__global__ void init_boundaries(int *boundary_node_list, int *boundary_index_list, int number_of_boundnodes, LB_nodes_gpu n_a, LB_nodes_gpu n_b){

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<number_of_boundnodes){
    n_a.boundary[boundary_node_list[index]] = boundary_index_list[index];
    n_b.boundary[boundary_node_list[index]] = boundary_index_list[index];
  }	
}

/**reset the boundary flag of every node
 * @param n_a		Pointer to local node residing in array a (Input)
 * @param n_b		Pointer to local node residing in array b (Input)	
*/
__global__ void reset_boundaries(LB_nodes_gpu n_a, LB_nodes_gpu n_b){

  size_t index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
    n_a.boundary[index] = n_b.boundary[index] = 0;
  }
}

/** integrationstep of the lb-fluid-solver
 * @param n_a		Pointer to local node residing in array a (Input)
 * @param n_b		Pointer to local node residing in array b (Input)
 * @param *d_v		Pointer to local device values (Input)
 * @param node_f	Pointer to local node force (Input)
*/
__global__ void integrate(LB_nodes_gpu n_a, LB_nodes_gpu n_b, LB_rho_v_gpu *d_v, LB_node_force_gpu node_f){
  /**every node is connected to a thread via the index*/
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
  /**the 19 moments (modes) are only temporary register values */
  float mode[19*LB_COMPONENTS];
  LB_randomnr_gpu rng;

  if( index < para.number_of_nodes ){
    /** storing the seed into a register value*/
    rng.seed = n_a.seed[index];
    /**calc_m_from_n*/
    calc_m_from_n(n_a, index, mode);
    /**lb_relax_modes*/
    relax_modes(mode, index, node_f,d_v);
    /**lb_thermalize_modes */
    if (para.fluct){thermalize_modes(mode, index, &rng);}
#if  defined(EXTERNAL_FORCES)  ||   defined (SHANCHEN)  
    /**if external force is used apply node force */
    apply_forces(index, mode, node_f,d_v);
#else
    /**if partcles are used apply node forces*/
    if (para.number_of_particles) apply_forces(index, mode, node_f,d_v); 
#endif
    /**lb_calc_n_from_modes_push*/
    normalize_modes(mode);
    /**calc of velocity densities and streaming with pbc*/
    calc_n_from_modes_push(n_b, mode, index);
    /** rewriting the seed back to the global memory*/
    n_b.seed[index] = rng.seed;
  }  
}

/** part interaction kernel
 * @param n_a				Pointer to local node residing in array a (Input)
 * @param *particle_data		Pointer to the particle position and velocity (Input)
 * @param *particle_force		Pointer to the particle force (Input)
 * @param *part				Pointer to the rn array of the particles (Input)
 * @param node_f			Pointer to local node force (Input)
*/
__global__ void calc_fluid_particle_ia(LB_nodes_gpu n_a, CUDA_particle_data *particle_data, CUDA_particle_force *particle_force, LB_node_force_gpu node_f, CUDA_particle_seed *part, LB_rho_v_gpu *d_v){
	
  unsigned int part_index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int node_index[8];
  float delta[8];
  float delta_j[3*LB_COMPONENTS]; 
  float partgrad1[8*LB_COMPONENTS]; 
  float partgrad2[8*LB_COMPONENTS]; 
  float partgrad3[8*LB_COMPONENTS]; 
  LB_randomnr_gpu rng_part;
  if(part_index<para.number_of_particles){

    rng_part.seed = part[part_index].seed;
    /**force acting on the particle. delta_j will be used later to compute the force that acts back onto the fluid. */
    calc_viscous_force(n_a, delta, partgrad1, partgrad2, partgrad3, particle_data, particle_force, part_index, &rng_part, delta_j, node_index,d_v);
    calc_node_force(delta, delta_j, partgrad1, partgrad2, partgrad3, node_index, node_f); 
    /**force which acts back to the fluid node */
    part[part_index].seed = rng_part.seed;		
  }
}

#ifdef LB_BOUNDARIES_GPU
/**Bounce back boundary read kernel
 * @param n_a					Pointer to local node residing in array a (Input)
 * @param n_b					Pointer to local node residing in array b (Input)
 * @param LB_boundary_velocity 			The constant velocity at the boundary, set by the user (Input)
 * @param LB_boundary_force 			The force on the boundary nodes (Output)
*/
__global__ void bb_read(LB_nodes_gpu n_a, LB_nodes_gpu n_b, float* LB_boundary_velocity, float* LB_boundary_force){

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
    bounce_back_read(n_b, n_a, index, LB_boundary_velocity, LB_boundary_force);
  }
}

/**Bounce back boundary write kernel
 * @param n_a					Pointer to local node residing in array a (Input)
 * @param n_b					Pointer to local node residing in array b (Input)
*/
__global__ void bb_write(LB_nodes_gpu n_a, LB_nodes_gpu n_b){

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
    bounce_back_write(n_b, n_a, index);
  }
}

#endif

/** get physical values of the nodes (density, velocity, ...)
 * @param n_a		Pointer to local node residing in array a (Input)
 * @param *p_v		Pointer to local print values (Output)
 * @param *d_v		Pointer to local device values (Input)
*/
__global__ void get_mesoscopic_values_in_MD_units(LB_nodes_gpu n_a, LB_rho_v_pi_gpu *p_v,LB_rho_v_gpu *d_v) {
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index < para.number_of_nodes) {
    float mode[19*LB_COMPONENTS];
    calc_m_from_n(n_a, index, mode);
    calc_values_in_MD_units(n_a, mode, p_v, d_v, index, index);
  }
}

/** get boundary flags
 *  @param n_a	              Pointer to local node residing in array a (Input)
 *  @param device_bound_array Pointer to local device values (Input)
 */
__global__ void lb_get_boundaries(LB_nodes_gpu n_a, unsigned int *device_bound_array){

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
   device_bound_array[index] = n_a.boundary[index];
  }
}



/**print single node values kernel
 * @param single_nodeindex		index of the node (Input)
 * @param *d_p_v			Pointer to result storage array (Input)
 * @param n_a				Pointer to local node residing in array a (Input)
*/
__global__ void lb_print_node(int single_nodeindex, LB_rho_v_pi_gpu *d_p_v, LB_nodes_gpu n_a, LB_rho_v_gpu * d_v){
	
  float mode[19*LB_COMPONENTS];
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index == 0) {
     calc_m_from_n(n_a, single_nodeindex, mode);
     
     /* the following actually copies rho and v from d_v, and calculates pi */
     calc_values_in_MD_units(n_a, mode, d_p_v, d_v, single_nodeindex, 0);
  }
}
__global__ void momentum(LB_nodes_gpu n_a, LB_rho_v_gpu * d_v, LB_node_force_gpu node_f, float *sum) {

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
  if(index<para.number_of_nodes){
    float j[3]={0.f,0.f,0.f};
    float mode[4];
    for(int ii=0 ; ii < LB_COMPONENTS ; ii++ ) { 
        calc_mode(mode, n_a, index,ii);
        j[0] += mode[1]+node_f.force[(0+ii*3)*para.number_of_nodes + index];
        j[1] += mode[2]+node_f.force[(1+ii*3)*para.number_of_nodes + index];
        j[2] += mode[3]+node_f.force[(2+ii*3)*para.number_of_nodes + index];
    }
#ifdef LB_BOUNDARIES_GPU
    if(n_a.boundary[index]){
	j[0]=j[1]=j[2]=0.0f;
    }
#endif
    atomicadd(&(sum[0]), j[0]); 
    atomicadd(&(sum[1]), j[1]); 
    atomicadd(&(sum[2]), j[2]); 
  }

}

/**print single node boundary flag
 * @param single_nodeindex		index of the node (Input)
 * @param *device_flag			Pointer to result storage array (Input)
 * @param n_a				Pointer to local node residing in array a (Input)
*/
__global__ void lb_get_boundary_flag(int single_nodeindex, unsigned int *device_flag, LB_nodes_gpu n_a){
	
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index == 0){
    device_flag[0] = n_a.boundary[single_nodeindex];
  }	
}

/**********************************************************************/
/* Host functions to setup and call kernels*/
/**********************************************************************/

void lb_get_para_pointer(LB_parameters_gpu** pointeradress) {
  if(hipGetSymbolAddress((void**) pointeradress, para) != hipSuccess) {
    printf("Trouble getting address of LB parameters.\n"); //TODO give proper error message
    exit(1);
  }
}

void lb_get_lbpar_pointer(LB_parameters_gpu** pointeradress) {
  *pointeradress = &lbpar_gpu;
}

/**initialization for the lb gpu fluid called from host
 * @param *lbpar_gpu	Pointer to parameters to setup the lb field
*/
void lb_init_GPU(LB_parameters_gpu *lbpar_gpu){
#define free_and_realloc(var,size)\
  { if( (var) != NULL ) hipFree((var)); cuda_safe_mem(hipMalloc((void**)&var, size)); } 


  size_of_rho_v     = lbpar_gpu->number_of_nodes * sizeof(LB_rho_v_gpu);
  size_of_rho_v_pi  = lbpar_gpu->number_of_nodes * sizeof(LB_rho_v_pi_gpu);


  /** Allocate structs in device memory*/
  if(extended_values_flag==0) { 
              free_and_realloc(device_rho_v, size_of_rho_v);
  } else { 
              /* see the notes to the stucture device_rho_v_pi above...*/
              free_and_realloc(device_rho_v_pi, size_of_rho_v_pi);
  }


  /* TODO: this is a almost a copy copy of  device_rho_v thik about eliminating it, and maybe pi can be added to device_rho_v in this case*/
  free_and_realloc(print_rho_v_pi  , size_of_rho_v_pi);
  free_and_realloc(nodes_a.vd      , lbpar_gpu->number_of_nodes * 19 * LB_COMPONENTS * sizeof(float));
  free_and_realloc(nodes_b.vd      , lbpar_gpu->number_of_nodes * 19 * LB_COMPONENTS * sizeof(float));   
  free_and_realloc(node_f.force    , lbpar_gpu->number_of_nodes * 3  * LB_COMPONENTS * sizeof(float));

  free_and_realloc(nodes_a.seed    , lbpar_gpu->number_of_nodes * sizeof( unsigned int));
  free_and_realloc(nodes_a.boundary, lbpar_gpu->number_of_nodes * sizeof( unsigned int));
  free_and_realloc(nodes_b.seed    , lbpar_gpu->number_of_nodes * sizeof( unsigned int));
  free_and_realloc(nodes_b.boundary, lbpar_gpu->number_of_nodes * sizeof( unsigned int));



  /**write parameters in const memory*/
  cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu, sizeof(LB_parameters_gpu)));

  /**check flag if lb gpu init works*/
  free_and_realloc(gpu_check, sizeof(int));
  if(h_gpu_check!=NULL) free(h_gpu_check) ;  
  h_gpu_check = (int*)malloc(sizeof(int));

  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu->number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  hipStreamCreate(&stream[0]);

  /** values for the particle kernel */
  int threads_per_block_particles = 64;
  int blocks_per_grid_particles_y = 4;
  int blocks_per_grid_particles_x = (lbpar_gpu->number_of_particles + threads_per_block_particles * blocks_per_grid_particles_y - 1)/(threads_per_block_particles * blocks_per_grid_particles_y);
  dim3 dim_grid_particles = make_uint3(blocks_per_grid_particles_x, blocks_per_grid_particles_y, 1);
        
  KERNELCALL(reset_boundaries, dim_grid, threads_per_block, (nodes_a, nodes_b));

  #ifdef SHANCHEN
  // TODO FIXME: 
  /* We must add shan-chen forces, which are zero only if the densities are uniform*/
  #endif

  /** calc of veloctiydensities from given parameters and initialize the Node_Force array with zero */
  KERNELCALL(calc_n_equilibrium, dim_grid, threads_per_block, (nodes_a, device_rho_v ,node_f, gpu_check));	
  

  
  KERNELCALL(reinit_node_force, dim_grid, threads_per_block, (node_f));


  intflag = 1;
  current_nodes = &nodes_a;
  h_gpu_check[0] = 0;
  cuda_safe_mem(hipMemcpy(h_gpu_check, gpu_check, sizeof(int), hipMemcpyDeviceToHost));
//fprintf(stderr, "initialization of lb gpu code %i\n", lbpar_gpu->number_of_nodes);
  hipDeviceSynchronize();
  if(!h_gpu_check[0]){
    fprintf(stderr, "initialization of lb gpu code failed! \n");
    errexit();	
  }	
}
/** reinitialization for the lb gpu fluid called from host
 * @param *lbpar_gpu	Pointer to parameters to setup the lb field
*/
void lb_reinit_GPU(LB_parameters_gpu *lbpar_gpu){

  /**write parameters in const memory*/
  cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu, sizeof(LB_parameters_gpu)));
  
  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu->number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  /** calc of veloctiydensities from given parameters and initialize the Node_Force array with zero */
  KERNELCALL(calc_n_equilibrium, dim_grid, threads_per_block, (nodes_a, device_rho_v, node_f, gpu_check));
}

/**setup and call particle reallocation from the host
 * @param *lbpar_gpu	Pointer to parameters to setup the lb field
*/
void lb_realloc_particle_GPU_leftovers(LB_parameters_gpu *lbpar_gpu){

  //copy parameters, especially number of parts to gpu mem
  cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu, sizeof(LB_parameters_gpu)));
}

#ifdef LB_BOUNDARIES_GPU
/** setup and call boundaries from the host
 * @param host_n_lb_boundaries number of LB boundaries
 * @param number_of_boundnodes	number of boundnodes
 * @param host_boundary_node_list    The indices of the boundary nodes
 * @param host_boundary_index_list   The flag representing the corresponding boundary
 * @param host_LB_Boundary_velocity 			The constant velocity at the boundary, set by the user (Input)
*/
void lb_init_boundaries_GPU(int host_n_lb_boundaries, int number_of_boundnodes, int *host_boundary_node_list, int* host_boundary_index_list, float* host_LB_Boundary_velocity){
  int temp = host_n_lb_boundaries;

  size_of_boundindex = number_of_boundnodes*sizeof(int);
  cuda_safe_mem(hipMalloc((void**)&boundary_node_list, size_of_boundindex));
  cuda_safe_mem(hipMalloc((void**)&boundary_index_list, size_of_boundindex));
  cuda_safe_mem(hipMemcpy(boundary_index_list, host_boundary_index_list, size_of_boundindex, hipMemcpyHostToDevice));
  cuda_safe_mem(hipMemcpy(boundary_node_list, host_boundary_node_list, size_of_boundindex, hipMemcpyHostToDevice));

  cuda_safe_mem(hipMalloc((void**)&LB_boundary_force   , 3*host_n_lb_boundaries*sizeof(float)));
  cuda_safe_mem(hipMalloc((void**)&LB_boundary_velocity, 3*host_n_lb_boundaries*sizeof(float)));
  cuda_safe_mem(hipMemcpy(LB_boundary_velocity, host_LB_Boundary_velocity, 3*n_lb_boundaries*sizeof(float), hipMemcpyHostToDevice));
  cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(n_lb_boundaries_gpu), &temp, sizeof(int)));

  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(reset_boundaries, dim_grid, threads_per_block, (nodes_a, nodes_b));

  if (n_lb_boundaries == 0) {
    hipDeviceSynchronize();
    return;
  }
  if(number_of_boundnodes == 0){
    fprintf(stderr, "WARNING: boundary cmd executed but no boundary node found!\n");
  } else{
    int threads_per_block_bound = 64;
    int blocks_per_grid_bound_y = 4;
    int blocks_per_grid_bound_x = (number_of_boundnodes + threads_per_block_bound * blocks_per_grid_bound_y - 1) /(threads_per_block_bound * blocks_per_grid_bound_y);
    dim3 dim_grid_bound = make_uint3(blocks_per_grid_bound_x, blocks_per_grid_bound_y, 1);

    KERNELCALL(init_boundaries, dim_grid_bound, threads_per_block_bound, (boundary_node_list, boundary_index_list, number_of_boundnodes, nodes_a, nodes_b));
  }

  hipDeviceSynchronize();
}
#endif
/**setup and call extern single node force initialization from the host
 * @param *lbpar_gpu				Pointer to host parameter struct
*/
void lb_reinit_extern_nodeforce_GPU(LB_parameters_gpu *lbpar_gpu){

  cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu, sizeof(LB_parameters_gpu))); 

  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu->number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(reinit_node_force, dim_grid, threads_per_block, (node_f));

}
/**setup and call extern single node force initialization from the host
 * @param n_extern_nodeforces			number of nodes on which the external force has to be applied
 * @param *host_extern_nodeforces		Pointer to the host extern node forces
 * @param *lbpar_gpu				Pointer to host parameter struct
*/
void lb_init_extern_nodeforces_GPU(int n_extern_nodeforces, LB_extern_nodeforce_gpu *host_extern_nodeforces, LB_parameters_gpu *lbpar_gpu){

  size_of_extern_nodeforces = n_extern_nodeforces*sizeof(LB_extern_nodeforce_gpu);
  cuda_safe_mem(hipMalloc((void**)&extern_nodeforces, size_of_extern_nodeforces));
  hipMemcpy(extern_nodeforces, host_extern_nodeforces, size_of_extern_nodeforces, hipMemcpyHostToDevice);

  if(lbpar_gpu->external_force == 0)cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu, sizeof(LB_parameters_gpu))); 

  int threads_per_block_exf = 64;
  int blocks_per_grid_exf_y = 4;
  int blocks_per_grid_exf_x = (n_extern_nodeforces + threads_per_block_exf * blocks_per_grid_exf_y - 1) /(threads_per_block_exf * blocks_per_grid_exf_y);
  dim3 dim_grid_exf = make_uint3(blocks_per_grid_exf_x, blocks_per_grid_exf_y, 1);
	
  KERNELCALL(init_extern_nodeforces, dim_grid_exf, threads_per_block_exf, (n_extern_nodeforces, extern_nodeforces, node_f));
  hipFree(extern_nodeforces);
}

/**setup and call particle kernel from the host
*/
void lb_calc_particle_lattice_ia_gpu(){
  if (lbpar_gpu.number_of_particles) {
    /** call of the particle kernel */
    /** values for the particle kernel */
    int threads_per_block_particles = 64;
    int blocks_per_grid_particles_y = 4;
    int blocks_per_grid_particles_x = (lbpar_gpu.number_of_particles + threads_per_block_particles * blocks_per_grid_particles_y - 1)/(threads_per_block_particles * blocks_per_grid_particles_y);
    dim3 dim_grid_particles = make_uint3(blocks_per_grid_particles_x, blocks_per_grid_particles_y, 1);

    KERNELCALL(calc_fluid_particle_ia, dim_grid_particles, threads_per_block_particles, (*current_nodes, gpu_get_particle_pointer(), gpu_get_particle_force_pointer(), node_f, gpu_get_particle_seed_pointer(),device_rho_v));
  }
}

/** setup and call kernel for getting macroscopic fluid values of all nodes
 * @param *host_values struct to save the gpu values
*/
void lb_get_values_GPU(LB_rho_v_pi_gpu *host_values){

  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(get_mesoscopic_values_in_MD_units, dim_grid, threads_per_block, (nodes_a, print_rho_v_pi, device_rho_v ));
  hipMemcpy(host_values, print_rho_v_pi, size_of_rho_v_pi, hipMemcpyDeviceToHost);

}

/** get all the boundary flags for all nodes
 *  @param host_bound_array here go the values of the boundary flag
 */
void lb_get_boundary_flags_GPU(unsigned int* host_bound_array){
   
  unsigned int* device_bound_array;
  cuda_safe_mem(hipMalloc((void**)&device_bound_array, lbpar_gpu.number_of_nodes*sizeof(unsigned int)));	
  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) / (threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(lb_get_boundaries, dim_grid, threads_per_block, (*current_nodes, device_bound_array));

  hipMemcpy(host_bound_array, device_bound_array, lbpar_gpu.number_of_nodes*sizeof(unsigned int), hipMemcpyDeviceToHost);

  hipFree(device_bound_array);

}

/** setup and call kernel for getting macroscopic fluid values of a single node*/
void lb_print_node_GPU(int single_nodeindex, LB_rho_v_pi_gpu *host_print_values){ 
      
  LB_rho_v_pi_gpu *device_print_values;
  cuda_safe_mem(hipMalloc((void**)&device_print_values, sizeof(LB_rho_v_pi_gpu)));	
  int threads_per_block_print = 1;
  int blocks_per_grid_print_y = 1;
  int blocks_per_grid_print_x = 1;
  dim3 dim_grid_print = make_uint3(blocks_per_grid_print_x, blocks_per_grid_print_y, 1);

  KERNELCALL(lb_print_node, dim_grid_print, threads_per_block_print, (single_nodeindex, device_print_values, *current_nodes, device_rho_v));

  hipMemcpy(host_print_values, device_print_values, sizeof(LB_rho_v_pi_gpu), hipMemcpyDeviceToHost);
  hipFree(device_print_values);

}

/** setup and call kernel to calculate the total momentum of the hole fluid
 * @param *mass value of the mass calcutated on the GPU
*/
void lb_calc_fluid_mass_GPU(double* mass){

  float* tot_mass;
  float cpu_mass =  0.f ;
  cuda_safe_mem(hipMalloc((void**)&tot_mass, sizeof(float)));
  hipMemcpy(tot_mass, &cpu_mass, sizeof(float), hipMemcpyHostToDevice);

  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(calc_mass, dim_grid, threads_per_block,(*current_nodes, tot_mass));

  hipMemcpy(&cpu_mass, tot_mass, sizeof(float), hipMemcpyDeviceToHost);
  
  hipFree(tot_mass);
  mass[0] = (double)(cpu_mass);
}

/** setup and call kernel to calculate the total momentum of the hole fluid
 *  @param host_mom value of the momentum calcutated on the GPU
 */
void lb_calc_fluid_momentum_GPU(double* host_mom){

  float* tot_momentum;
  float host_momentum[3] = { 0.f, 0.f, 0.f};
  cuda_safe_mem(hipMalloc((void**)&tot_momentum, 3*sizeof(float)));
  hipMemcpy(tot_momentum, host_momentum, 3*sizeof(float), hipMemcpyHostToDevice);

  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(momentum, dim_grid, threads_per_block,(*current_nodes, device_rho_v, node_f, tot_momentum));
  
  hipMemcpy(host_momentum, tot_momentum, 3*sizeof(float), hipMemcpyDeviceToHost);
  
  hipFree(tot_momentum);
  host_mom[0] = (double)(host_momentum[0]* lbpar_gpu.agrid/lbpar_gpu.tau);
  host_mom[1] = (double)(host_momentum[1]* lbpar_gpu.agrid/lbpar_gpu.tau);
  host_mom[2] = (double)(host_momentum[2]* lbpar_gpu.agrid/lbpar_gpu.tau);
}


/** setup and call kernel to calculate the temperature of the hole fluid
 *  @param host_temp value of the temperatur calcutated on the GPU
*/
void lb_calc_fluid_temperature_GPU(double* host_temp){

  float host_jsquared = 0.f;
  float* device_jsquared;
  cuda_safe_mem(hipMalloc((void**)&device_jsquared, sizeof(float)));
  hipMemcpy(device_jsquared, &host_jsquared, sizeof(float), hipMemcpyHostToDevice);

  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(temperature, dim_grid, threads_per_block,(*current_nodes, device_jsquared));

  hipMemcpy(&host_jsquared, device_jsquared, sizeof(float), hipMemcpyDeviceToHost);
  // TODO: check that temperature calculation is properly implemented for shanchen
  *host_temp=0;
  #pragma unroll
  for(int ii=0;ii<LB_COMPONENTS;++ii) { 
      *host_temp += (double)(host_jsquared*1./(3.f*lbpar_gpu.rho[ii]*lbpar_gpu.dim_x*lbpar_gpu.dim_y*lbpar_gpu.dim_z*lbpar_gpu.tau*lbpar_gpu.tau*lbpar_gpu.agrid));
  }
}


#ifdef SHANCHEN
void lb_calc_shanchen_GPU(){
  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(lb_shanchen_GPU, dim_grid, threads_per_block,(*current_nodes, node_f));

}

#endif // SHANCHEN






/** setup and call kernel for getting macroscopic fluid values of all nodes
 * @param *host_values struct to save the gpu values
*/
void lb_save_checkpoint_GPU(float *host_checkpoint_vd, unsigned int *host_checkpoint_seed, unsigned int *host_checkpoint_boundary, float *host_checkpoint_force){

  hipMemcpy(host_checkpoint_vd, current_nodes->vd, lbpar_gpu.number_of_nodes * 19 * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(host_checkpoint_seed, current_nodes->seed, lbpar_gpu.number_of_nodes * sizeof(unsigned int), hipMemcpyDeviceToHost);
  hipMemcpy(host_checkpoint_boundary, current_nodes->boundary, lbpar_gpu.number_of_nodes * sizeof(unsigned int), hipMemcpyDeviceToHost);
  hipMemcpy(host_checkpoint_force, node_f.force, lbpar_gpu.number_of_nodes * 3 * sizeof(float), hipMemcpyDeviceToHost);

}
/** setup and call kernel for setting macroscopic fluid values of all nodes
 * @param *host_values struct to set stored values
*/
void lb_load_checkpoint_GPU(float *host_checkpoint_vd, unsigned int *host_checkpoint_seed, unsigned int *host_checkpoint_boundary, float *host_checkpoint_force){

  hipMemcpy(current_nodes->vd, host_checkpoint_vd, lbpar_gpu.number_of_nodes * 19 * sizeof(float), hipMemcpyHostToDevice);
  intflag = 1;
  hipMemcpy(current_nodes->seed, host_checkpoint_seed, lbpar_gpu.number_of_nodes * sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy(current_nodes->boundary, host_checkpoint_boundary, lbpar_gpu.number_of_nodes * sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy(node_f.force, host_checkpoint_force, lbpar_gpu.number_of_nodes * 3 * sizeof(float), hipMemcpyHostToDevice);

}


/** setup and call kernel to get the boundary flag of a single node
 *  @param single_nodeindex number of the node to get the flag for
 *  @param host_flag her goes the value of the boundary flag
 */
void lb_get_boundary_flag_GPU(int single_nodeindex, unsigned int* host_flag){
   
  unsigned int* device_flag;
  cuda_safe_mem(hipMalloc((void**)&device_flag, sizeof(unsigned int)));	
  int threads_per_block_flag = 1;
  int blocks_per_grid_flag_y = 1;
  int blocks_per_grid_flag_x = 1;
  dim3 dim_grid_flag = make_uint3(blocks_per_grid_flag_x, blocks_per_grid_flag_y, 1);

  KERNELCALL(lb_get_boundary_flag, dim_grid_flag, threads_per_block_flag, (single_nodeindex, device_flag, *current_nodes));

  hipMemcpy(host_flag, device_flag, sizeof(unsigned int), hipMemcpyDeviceToHost);

  hipFree(device_flag);

}

/** set the density at a single node
 *  @param single_nodeindex the node to set the velocity for 
 *  @param host_velocity the velocity to set
 */
void lb_set_node_rho_GPU(int single_nodeindex, float* host_rho){
   
  float* device_rho;
  cuda_safe_mem(hipMalloc((void**)&device_rho, LB_COMPONENTS*sizeof(float)));	
  hipMemcpy(device_rho, host_rho, LB_COMPONENTS*sizeof(float), hipMemcpyHostToDevice);
  int threads_per_block_flag = 1;
  int blocks_per_grid_flag_y = 1;
  int blocks_per_grid_flag_x = 1;
  dim3 dim_grid_flag = make_uint3(blocks_per_grid_flag_x, blocks_per_grid_flag_y, 1);
  KERNELCALL(set_rho, dim_grid_flag, threads_per_block_flag, (*current_nodes, device_rho_v, single_nodeindex, device_rho)); 
  hipFree(device_rho);

}

/** set the net velocity at a single node
 *  @param single_nodeindex the node to set the velocity for 
 *  @param host_velocity the velocity to set
 */
void lb_set_node_velocity_GPU(int single_nodeindex, float* host_velocity){
   
  float* device_velocity;
  cuda_safe_mem(hipMalloc((void**)&device_velocity, 3*sizeof(float)));	
  hipMemcpy(device_velocity, host_velocity, 3*sizeof(float), hipMemcpyHostToDevice);
  int threads_per_block_flag = 1;
  int blocks_per_grid_flag_y = 1;
  int blocks_per_grid_flag_x = 1;
  dim3 dim_grid_flag = make_uint3(blocks_per_grid_flag_x, blocks_per_grid_flag_y, 1);

  KERNELCALL(set_u_equilibrium, dim_grid_flag, threads_per_block_flag, (*current_nodes, single_nodeindex, device_velocity)); 
  hipFree(device_velocity);

}

/** reinit of params 
 * @param *lbpar_gpu struct containing the paramters of the fluid
*/
void reinit_parameters_GPU(LB_parameters_gpu *lbpar_gpu){
  /**write parameters in const memory*/
  cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu, sizeof(LB_parameters_gpu)));
}

/**integration kernel for the lb gpu fluid update called from host */
void lb_integrate_GPU(){
  
  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

#ifdef LB_BOUNDARIES_GPU
  if (n_lb_boundaries > 0) 
    cuda_safe_mem(hipMemset	(	LB_boundary_force, 0, 3*n_lb_boundaries*sizeof(float)));
#endif


  /**call of fluid step*/
  /* NOTE: if pi is needed at every integration step, one should call an extended version 
           of the integrate kernel, or pass also device_rho_v_pi and make sure that either 
           it or device_rho_v are NULL depending on extended_values_flag */ 
  if (intflag == 1){
    KERNELCALL(integrate, dim_grid, threads_per_block, (nodes_a, nodes_b, device_rho_v, node_f));
    current_nodes = &nodes_b;
#ifdef LB_BOUNDARIES_GPU		

    if (n_lb_boundaries > 0) {
        KERNELCALL(bb_read, dim_grid, threads_per_block, (nodes_a, nodes_b, LB_boundary_velocity, LB_boundary_force));
      }
#endif
    intflag = 0;
  }
  else{
    KERNELCALL(integrate, dim_grid, threads_per_block, (nodes_b, nodes_a, device_rho_v, node_f));
    current_nodes = &nodes_a;
#ifdef LB_BOUNDARIES_GPU		

    if (n_lb_boundaries > 0) {
      KERNELCALL(bb_read, dim_grid, threads_per_block, (nodes_b, nodes_a, LB_boundary_velocity, LB_boundary_force));
    }
#endif
    intflag = 1;
  }             
}

void lb_gpu_get_boundary_forces(double* forces) {
#ifdef LB_BOUNDARIES_GPU
  float* temp = (float*) malloc(3*n_lb_boundaries*sizeof(float));
  cuda_safe_mem(hipMemcpy(temp, LB_boundary_force, 3*n_lb_boundaries*sizeof(float), hipMemcpyDeviceToHost));
  for (int i =0; i<3*n_lb_boundaries; i++) {
    forces[i]=(double)temp[i];
  }
  free(temp);
#endif
}

#endif /* LB_GPU */
