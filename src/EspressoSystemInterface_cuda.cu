#include "hip/hip_runtime.h"
#include "EspressoSystemInterface.hpp"
#include "cuda_interface.hpp"
#include "cuda_utils.hpp"

__global__ void split_kernel_rq(CUDA_particle_data *particles, float *r, float *q, int n) {
  int idx = blockDim.x*blockIdx.x + threadIdx.x;
  if(idx >= n)
    return;

  CUDA_particle_data p = particles[idx];

  r[idx + 0] = p.p[0];
  r[idx + 1] = p.p[1];
  r[idx + 2] = p.p[2];
  #ifdef ELECTROSTATICS
  q[idx] = p.q;
  #endif
}

__global__ void split_kernel_q(CUDA_particle_data *particles,float *q, int n) {
  int idx = blockDim.x*blockIdx.x + threadIdx.x;
  if(idx >= n)
    return;

  CUDA_particle_data p = particles[idx];

#ifdef ELECTROSTRATICS
  q[idx] = p.q;
#endif
}

__global__ void split_kernel_r(CUDA_particle_data *particles, float *r, int n) {
  int idx = blockDim.x*blockIdx.x + threadIdx.x;
  if(idx >= n)
    return;

  CUDA_particle_data p = particles[idx];

  idx *= 3;

  r[idx + 0] = p.p[0];
  r[idx + 1] = p.p[1];
  r[idx + 2] = p.p[2];
}

__global__ void split_kernel_v(CUDA_particle_data *particles, float *v, int n) {
  int idx = blockDim.x*blockIdx.x + threadIdx.x;
  if(idx >= n)
    return;

  CUDA_particle_data p = particles[idx];

  idx *= 3;

  v[idx + 0] = p.v[0];
  v[idx + 1] = p.v[1];
  v[idx + 2] = p.v[2];
}

void EspressoSystemInterface::split_particle_struct() {
  int n = gpu_get_global_particle_vars_pointer_host()->number_of_particles;
  if(n == 0) 
    return;
  
  if( (n != m_gpu_npart) ) {
    if(m_needsRGpu) {
      if(m_r_gpu_begin != 0)
	cuda_safe_mem(hipFree(m_r_gpu_begin));
      cuda_safe_mem(hipMalloc(&m_r_gpu_begin, 3*n*sizeof(float)));
      m_r_gpu_end = m_r_gpu_begin + 3*n;
    }
    if(m_needsVGpu) {
      if(m_v_gpu_begin != 0)
	cuda_safe_mem(hipFree(m_v_gpu_begin));
      cuda_safe_mem(hipMalloc(&m_v_gpu_begin, 3*n*sizeof(float)));
      m_v_gpu_end = m_v_gpu_begin + 3*n;
    }
    if(m_needsQGpu) {
      if(m_q_gpu_begin != 0)
	cuda_safe_mem(hipFree(m_q_gpu_begin));
      cuda_safe_mem(hipMalloc(&m_q_gpu_begin, n*sizeof(float)));
      m_q_gpu_end = m_q_gpu_begin + n;
    }
  }

  m_gpu_npart = n;
  
  dim3 grid(n/512+1,1,1);
  dim3 block(512,1,1);

  if(m_needsQGpu && m_needsRGpu)
    split_kernel_rq<<<grid,block>>>(gpu_get_particle_pointer(), m_r_gpu_begin,m_q_gpu_begin,n);
  if(m_needsQGpu && !m_needsRGpu)
    split_kernel_q<<<grid,block>>>(gpu_get_particle_pointer(), m_q_gpu_begin,n);
  if(!m_needsQGpu && m_needsRGpu)
    split_kernel_r<<<grid,block>>>(gpu_get_particle_pointer(), m_r_gpu_begin,n);
  if(m_needsVGpu)
    split_kernel_v<<<grid,block>>>(gpu_get_particle_pointer(), m_v_gpu_begin,n);
}
