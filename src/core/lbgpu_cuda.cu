#include "hip/hip_runtime.h"
/* 
   Copyright (C) 2010,2011,2012,2013,2014,2015,2016 The ESPResSo project

   This file is part of ESPResSo.
  
   ESPResSo is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.
   
   ESPResSo is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.
   
   You should have received a copy of the GNU General Public License
   along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

/** \file lbgpu_cuda.cu
 *
 * Cuda (.cu) file for the Lattice Boltzmann implementation on GPUs.
 * Header file for \ref lbgpu.hpp.
 */

#include "config.hpp"

#ifdef LB_GPU
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <vector>
#include <cassert>

#include "electrokinetics.hpp"
#include "electrokinetics_pdb_parse.hpp"
#include "lbgpu.hpp"
#include "cuda_interface.hpp"
#include "cuda_utils.hpp"
#include "observables/profiles.hpp"
#include "errorhandling.hpp"
#include "debug.hpp"

#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>


#if defined(OMPI_MPI_H) || defined(_MPI_H)
#error CU-file includes mpi.h! This should not happen!
#endif

#if (!defined(FLATNOISE) && !defined(GAUSSRANDOMCUT) && !defined(GAUSSRANDOM))
#define FLATNOISE
#endif

int extended_values_flag=0; /* TODO: this has to be set to one by
                               appropriate functions if there is 
                               the need to compute pi at every 
                               step (e.g. moving boundaries)*/

/**defining structures residing in global memory */

/** device_rho_v: struct for hydrodynamic fields: this is for internal use 
    (i.e. stores values in LB units) and should not used for 
    printing values  */
static LB_rho_v_gpu *device_rho_v= nullptr;

/** device_rho_v_pi: extended struct for hydrodynamic fields: this is the interface
    and stores values in MD units. It should not be used
    as an input for any LB calculations. TODO: This structure is not yet 
    used, and it is here to allow access to the stress tensor at any
    timestep, e.g. for future implementations of moving boundary codes */
static LB_rho_v_pi_gpu *device_rho_v_pi= nullptr;

/** print_rho_v_pi: struct for hydrodynamic fields: this is the interface
    and stores values in MD units. It should not used
    as an input for any LB calculations. TODO: in the future,
    one might want to have several structures for printing 
    separately rho, v, pi without having to compute/store 
    the complete set. */
static LB_rho_v_pi_gpu *print_rho_v_pi= nullptr;

/** structs for velocity densities */
static LB_nodes_gpu nodes_a = { nullptr, nullptr, nullptr};
static LB_nodes_gpu nodes_b = { nullptr, nullptr, nullptr};;
/** struct for node force */

LB_node_force_gpu node_f = {nullptr, nullptr} ;

static LB_extern_nodeforce_gpu *extern_nodeforces = nullptr;

#ifdef LB_BOUNDARIES_GPU
static float* lb_boundary_force = nullptr;

static float* lb_boundary_velocity = nullptr;

/** pointer for bound index array*/
static int *boundary_node_list;
static int *boundary_index_list;
static size_t size_of_boundindex;
#endif

EK_parameters* lb_ek_parameters_gpu;

/** pointers for additional cuda check flag*/
static int *gpu_check = nullptr;
static int *h_gpu_check = nullptr;

static unsigned int intflag = 1;
LB_nodes_gpu *current_nodes = nullptr;
/**defining size values for allocating global memory */
static size_t size_of_rho_v;
static size_t size_of_rho_v_pi;
static size_t size_of_extern_nodeforces;

/**parameters residing in constant memory */
static __device__ __constant__ LB_parameters_gpu para;
static const float c_sound_sq = 1.0f/3.0f;

/*-------------------------------------------------------*/
/*********************************************************/
/** \name device functions called by kernel functions */
/*********************************************************/
/*-------------------------------------------------------*/

/*-------------------------------------------------------*/

/** atomic add function for sveral cuda architectures 
*/
__device__ inline void atomicadd(float* address, float value){
#if !defined __CUDA_ARCH__ || __CUDA_ARCH__ >= 200 // for Fermi, atomicAdd supports floats
  atomicAdd(address, value);
#elif __CUDA_ARCH__ >= 110
#warning Using slower atomicAdd emulation
// float-atomic-add from 
// [url="http://forums.nvidia.com/index.php?showtopic=158039&view=findpost&p=991561"]
  float old = value;
  while ((old = atomicExch(address, atomicExch(address, 0.0f)+old))!=0.0f);
#else
#error I need at least compute capability 1.1
#endif
}

__device__ inline void atomicadd (double* address, double value) {
  unsigned long long oldval, newval, readback;
  oldval = __double_as_longlong(*address);
  newval = __double_as_longlong(__longlong_as_double(oldval) + value);
  while ((readback=atomicCAS((unsigned long long *)address, oldval, newval)) != oldval)
  {
    oldval = readback;
    newval = __double_as_longlong(__longlong_as_double(oldval) + value);
  }
}

/**randomgenerator which generates numbers [0,1]
 * @param *rn Pointer to randomnumber array of the local node or particle 
*/
__device__ void random_01(LB_randomnr_gpu *rn){

  const float mxi = 1.0f/(float)(1ul<<31);
  unsigned int curr = rn->seed;

  curr = 1103515245 * curr + 12345;
  rn->randomnr[0] = (float)(curr & ((1ul<<31)-1))*mxi;
  curr = 1103515245 * curr + 12345;
  rn->randomnr[1] = (float)(curr & ((1ul<<31)-1))*mxi;
  rn->seed = curr;

}

/**randomgenerator which generates numbers between -2 sigma and 2 sigma in the form of a Gaussian with standard deviation sigma=1.118591404 resulting in 
 * an actual standard deviation of 1.
 * @param *rn Pointer to randomnumber array of the local node or particle 
*/
__device__ void gaussian_random_cut(LB_randomnr_gpu *rn){

  float x1, x2;
  float r2, fac;
  /** On every second call two gaussian random numbers are calculated
   via the Box-Muller transformation.*/
  /** draw two uniform random numbers in the unit circle */
  do {
    random_01(rn);
    x1 = 2.0f*rn->randomnr[0] - 1.0f;
    x2 = 2.0f*rn->randomnr[1] - 1.0f;
    r2 = x1*x1 + x2*x2;
  } while (r2 >= 1.0f || r2 == 0.0f);

  /** perform Box-Muller transformation and cutoff the ends and replace with flat noise */
  /*
  fac = sqrtf(-2.0f*__logf(r2)/r2)*1.118591404f;
  rn->randomnr[0] = x2*fac;
  rn->randomnr[1] = x1*fac;
  random_01(rn);
  if ( fabs(rn->randomnr[0]) > 2.0f*1.118591404f) {
    rn->randomnr[0] = (2.0f*rn->randomnr[0]-1.0f)*2.0f*1.118591404f;
  }
  if ( fabs(rn->randomnr[1]) > 2.0f*1.118591404f ) {
    rn->randomnr[0] = (2.0f*rn->randomnr[1]-1.0f)*2.0f*1.118591404f;
  }
  */
  
  fac = sqrtf(-2.0f*__logf(r2)/r2)*1.042267973f;
  rn->randomnr[0] = x2*fac;
  rn->randomnr[1] = x1*fac;
  if ( fabs(rn->randomnr[0]) > 2.0f*1.042267973f) {
    if ( rn->randomnr[0] > 0 ) rn->randomnr[0] = 2.0f*1.042267973f;
    else rn->randomnr[0] = -2.0f*1.042267973f;
  }
  if ( fabs(rn->randomnr[1]) > 2.0f*1.042267973f ) {
    if ( rn->randomnr[1] > 0 ) rn->randomnr[1] = 2.0f*1.042267973f;
    else rn->randomnr[1] = -2.0f*1.042267973f;
  }
}

/** gaussian random nummber generator for thermalisation
 * @param *rn Pointer to randomnumber array of the local node node or particle 
*/
__device__ void gaussian_random(LB_randomnr_gpu *rn){

  float x1, x2;
  float r2, fac;
  /** On every second call two gaussian random numbers are calculated
   via the Box-Muller transformation.*/
  /** draw two uniform random numbers in the unit circle */
  do {
    random_01(rn);
    x1 = 2.0f*rn->randomnr[0]-1.0f;
    x2 = 2.0f*rn->randomnr[1]-1.0f;
    r2 = x1*x1 + x2*x2;
  } while (r2 >= 1.0f || r2 == 0.0f);

  /** perform Box-Muller transformation */
  fac = sqrtf(-2.0f*__logf(r2)/r2);
  rn->randomnr[0] = x2*fac;
  rn->randomnr[1] = x1*fac;
  
}
/* wrapper */
__device__ void random_wrapper(LB_randomnr_gpu *rn) { 

#if defined(FLATNOISE)
#define sqrt12 3.46410161514f
  random_01(rn);
  rn->randomnr[0]-=0.5f;
  rn->randomnr[0]*=sqrt12;
  rn->randomnr[1]-=0.5f;
  rn->randomnr[1]*=sqrt12;
#elif defined(GAUSSRANDOMCUT)
  gaussian_random_cut(rn);
#elif defined(GAUSSRANDOM)
  gaussian_random(rn);
#else
#error No noise type defined for the GPU LB
#endif  
  
}


/**tranformation from 1d array-index to xyz
 * @param index   node index / thread index (Input)
 * @param xyz     Pointer to calculated xyz array (Output)
 */
template < typename T >
__device__ void index_to_xyz(T index, T* xyz){
  xyz[0] = index%para.dim_x;
  index /= para.dim_x;
  xyz[1] = index%para.dim_y;
  index /= para.dim_y;
  xyz[2] = index;
}


/**tranformation from xyz to 1d array-index
 * @param xyz     Pointer xyz array (Input)
 * @param index   Calculated node index / thread index (Output)
 */
template < typename T >
__device__ T xyz_to_index(T* xyz){
  T x = (xyz[0] + para.dim_x) % para.dim_x;
  T y = (xyz[1] + para.dim_y) % para.dim_y;
  T z = (xyz[2] + para.dim_z) % para.dim_z;
  return x + para.dim_x*(y + para.dim_y*z);
}


/**calculation of the modes from the velocity densities (space-transform.)
 * @param n_a     Pointer to local node residing in array a (Input)
 * @param index   Node index / thread index (Input)
 * @param mode    Pointer to the local register values mode (Output)
*/
__device__ void calc_m_from_n(LB_nodes_gpu n_a, unsigned int index, float *mode){

  #pragma unroll
  for(int ii=0;ii<LB_COMPONENTS;++ii)
  {
    // The following convention is used:
    // The $\hat{c}_i$ form B. Duenweg's paper are given by:

    /* c_0  = { 0, 0, 0}
       c_1  = { 1, 0, 0}
       c_2  = {-1, 0, 0}
       c_3  = { 0, 1, 0}
       c_4  = { 0,-1, 0}
       c_5  = { 0, 0, 1}
       c_6  = { 0, 0,-1}
       c_7  = { 1, 1, 0}
       c_8  = {-1,-1, 0}
       c_9  = { 1,-1, 0}
       c_10 = {-1, 1, 0}
       c_11 = { 1, 0, 1}
       c_12 = {-1, 0,-1}
       c_13 = { 1, 0,-1}
       c_14 = {-1, 0, 1}
       c_15 = { 0, 1, 1}
       c_16 = { 0,-1,-1}
       c_17 = { 0, 1,-1}
       c_18 = { 0,-1, 1} */

    // The basis vectors (modes) are constructed as follows
    // $m_k = \sum_{i} e_{ki} n_{i}$, where the $e_{ki}$ form a 
    // linear transformation (matrix) that is given by

    /* $e{ 0,i} = 1$
       $e{ 1,i} = c_{i,x}$
       $e{ 2,i} = c_{i,y}$
       $e{ 3,i} = c_{i,z}$
       $e{ 4,i} = c_{i}^2 - 1$
       $e{ 5,i} = c_{i,x}^2 - c_{i,y}^2$
       $e{ 6,i} = c_{i}^2 - 3*c_{i,z}^2$
       $e{ 7,i} = c_{i,x}*c_{i,y}$
       $e{ 8,i} = c_{i,x}*c_{i,z}$
       $e{ 9,i} = c_{i,y}*c_{i,z}$
       $e{10,i} = (3*c_{i}^2 - 5)*c_{i,x}$
       $e{11,i} = (3*c_{i}^2 - 5)*c_{i,y}$
       $e{12,i} = (3*c_{i}^2 - 5)*c_{i,z}$
       $e{13,i} = (c_{i,y}^2 - c_{i,z}^2)*c_{i,x}$
       $e{14,i} = (c_{i,x}^2 - c_{i,z}^2)*c_{i,y}$
       $e{15,i} = (c_{i,x}^2 - c_{i,y}^2)*c_{i,z}$
       $e{16,i} = 3*c_{i}^2^2 - 6*c_{i}^2 + 1$
       $e{17,i} = (2*c_{i}^2 - 3)*(c_{i,x}^2 - c_{i,y}^2)$
       $e{18,i} = (2*c_{i}^2 - 3)*(c_{i}^2 - 3*c_{i,z}^2)$ */

    // Such that the transformation matrix is given by

    /* {{ 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1}, 
        { 0, 1,-1, 0, 0, 0, 0, 1,-1, 1,-1, 1,-1, 1,-1, 0, 0, 0, 0}, 
        { 0, 0, 0, 1,-1, 0, 0, 1,-1,-1, 1, 0, 0, 0, 0, 1,-1, 1,-1}, 
        { 0, 0, 0, 0, 0, 1,-1, 0, 0, 0, 0, 1,-1,-1, 1, 1,-1,-1, 1}, 
        {-1, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1}, 
        { 0, 1, 1,-1,-1, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1,-1,-1,-1,-1}, 
        { 0, 1, 1, 1, 1,-2,-2, 2, 2, 2, 2,-1,-1,-1,-1,-1,-1,-1,-1}, 
        { 0, 0, 0, 0, 0, 0, 0, 1, 1,-1,-1, 0, 0, 0, 0, 0, 0, 0, 0}, 
        { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1,-1,-1, 0, 0, 0, 0}, 
        { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1,-1,-1}, 
        { 0,-2, 2, 0, 0, 0, 0, 1,-1, 1,-1, 1,-1, 1,-1, 0, 0, 0, 0}, 
        { 0, 0, 0,-2, 2, 0, 0, 1,-1,-1, 1, 0, 0, 0, 0, 1,-1, 1,-1}, 
        { 0, 0, 0, 0, 0,-2, 2, 0, 0, 0, 0, 1,-1,-1, 1, 1,-1,-1, 1}, 
        { 0, 0, 0, 0, 0, 0, 0, 1,-1, 1,-1,-1, 1,-1, 1, 0, 0, 0, 0}, 
        { 0, 0, 0, 0, 0, 0, 0, 1,-1,-1, 1, 0, 0, 0, 0,-1, 1,-1, 1}, 
        { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1,-1,-1, 1,-1, 1, 1,-1}, 
        { 1,-2,-2,-2,-2,-2,-2, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1}, 
        { 0,-1,-1, 1, 1, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1,-1,-1,-1,-1}, 
        { 0,-1,-1,-1,-1, 2, 2, 2, 2, 2, 2,-1,-1,-1,-1,-1,-1,-1,-1}} */

    // With weights 

    /* q^{c_{i}} = { 1/3, 1/18, 1/18, 1/18,
                    1/18, 1/18, 1/18, 1/36,
                    1/36, 1/36, 1/36, 1/36, 
                    1/36, 1/36, 1/36, 1/36, 
                    1/36, 1/36, 1/36 } */

    // Which makes the transformation satisfy the following
    // orthogonality condition:
    // \sum_{i} q^{c_{i}} e_{ki} e_{li} = w_{k} \delta_{kl},
    // where the weights are:

    /* w_{i} = {  1, 1/3, 1/3, 1/3,
                2/3, 4/9, 4/3, 1/9,
                1/9, 1/9, 2/3, 2/3,
                2/3, 2/9, 2/9, 2/9, 
                  2, 4/9, 4/3 } */

    // mass mode

    mode[0 + ii * LBQ] =   n_a.vd[( 0 + ii*LBQ ) * para.number_of_nodes + index]
                         + n_a.vd[( 1 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[( 2 + ii*LBQ ) * para.number_of_nodes + index]
                         + n_a.vd[( 3 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[( 4 + ii*LBQ ) * para.number_of_nodes + index]
                         + n_a.vd[( 5 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[( 6 + ii*LBQ ) * para.number_of_nodes + index]
                         + n_a.vd[( 7 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[( 8 + ii*LBQ ) * para.number_of_nodes + index]
                         + n_a.vd[( 9 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index]
                         + n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index]
                         + n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index]
                         + n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index]
                         + n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index];

    // momentum modes

    mode[1 + ii * LBQ] =   (n_a.vd[( 1 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[( 2 + ii*LBQ ) * para.number_of_nodes + index])
                         + (n_a.vd[( 7 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[( 8 + ii*LBQ ) * para.number_of_nodes + index])
                         + (n_a.vd[( 9 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index])
                         + (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index])
                         + (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index]);

    mode[2 + ii * LBQ] =   (n_a.vd[( 3 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[( 4 + ii*LBQ ) * para.number_of_nodes + index])
                         + (n_a.vd[( 7 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[( 8 + ii*LBQ ) * para.number_of_nodes + index])
                         - (n_a.vd[( 9 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index])
                         + (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index])
                         + (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]);

    mode[3 + ii * LBQ] =   (n_a.vd[( 5 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[( 6 + ii*LBQ ) * para.number_of_nodes + index])
                         + (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index])
                         - (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index])
                         + (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index])
                         - (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]);

    // stress modes
    mode[4 + ii * LBQ] = - n_a.vd[( 0 + ii*LBQ ) * para.number_of_nodes + index]
                         + n_a.vd[( 7 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[( 8 + ii*LBQ ) * para.number_of_nodes + index]
                         + n_a.vd[( 9 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index]
                         + n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index]
                         + n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index]
                         + n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index]
                         + n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index];

    mode[5 + ii * LBQ] =   (n_a.vd[( 1 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[( 2 + ii*LBQ ) * para.number_of_nodes + index])
                         - (n_a.vd[( 3 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[( 4 + ii*LBQ ) * para.number_of_nodes + index])
                         + (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index])
                         + (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index])
                         - (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index])
                         - (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]);

    mode[6 + ii * LBQ] =   (n_a.vd[( 1 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[( 2 + ii*LBQ ) * para.number_of_nodes + index])
                         + (n_a.vd[( 3 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[( 4 + ii*LBQ ) * para.number_of_nodes + index])
                         - (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index])
                         - (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index])
                         - (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index])
                         - (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index])
                         - 2.0f*( 
                                    (n_a.vd[( 5 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[( 6 + ii*LBQ ) * para.number_of_nodes + index])
                                  - (n_a.vd[( 7 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[( 8 + ii*LBQ ) * para.number_of_nodes + index])
                                  - (n_a.vd[( 9 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index])
                                );

    mode[7 + ii * LBQ] =   (n_a.vd[( 7 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[( 8 + ii*LBQ ) * para.number_of_nodes + index])
                         - (n_a.vd[( 9 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index]);

    mode[8 + ii * LBQ] =   (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index])
                         - (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index]);

    mode[9 + ii * LBQ] =   (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index])
                         - (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]);

    // kinetic modes

    mode[10 + ii * LBQ] = - 2.0f*(n_a.vd[( 1 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[( 2 + ii*LBQ ) * para.number_of_nodes + index])
                               + (n_a.vd[( 7 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[( 8 + ii*LBQ ) * para.number_of_nodes + index])
                               + (n_a.vd[( 9 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index])
                               + (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index])
                               + (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index]);

    mode[11 + ii * LBQ] = - 2.0f*(n_a.vd[( 3 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[( 4 + ii*LBQ ) * para.number_of_nodes + index])
                               + (n_a.vd[( 7 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[( 8 + ii*LBQ ) * para.number_of_nodes + index])
                               - (n_a.vd[( 9 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index])
                               + (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index])
                               + (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]);

    mode[12 + ii * LBQ] = - 2.0f*(n_a.vd[( 5 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[( 6 + ii*LBQ ) * para.number_of_nodes + index])
                               + (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index])
                               - (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index])
                               + (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index])
                               - (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]);

    mode[13 + ii * LBQ] =   (n_a.vd[( 7 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[( 8 + ii*LBQ ) * para.number_of_nodes + index])
                          + (n_a.vd[( 9 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index])
                          - (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index])
                          - (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index]);

    mode[14 + ii * LBQ] =   (n_a.vd[( 7 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[( 8 + ii*LBQ ) * para.number_of_nodes + index])
                          - (n_a.vd[( 9 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index])
                          - (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index])
                          - (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]);

    mode[15 + ii * LBQ] =   (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index])
                          - (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index])
                          - (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index])
                          + (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]);

    mode[16 + ii * LBQ] =   n_a.vd[( 0 + ii*LBQ ) * para.number_of_nodes + index]
                          + n_a.vd[( 7 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[( 8 + ii*LBQ ) * para.number_of_nodes + index]
                          + n_a.vd[( 9 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index]
                          + n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index]
                          + n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index]
                          + n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index]
                          + n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]
                          - 2.0f*(
                                     (n_a.vd[( 1 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[( 2 + ii*LBQ ) * para.number_of_nodes + index])
                                   + (n_a.vd[( 3 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[( 4 + ii*LBQ ) * para.number_of_nodes + index])
                                   + (n_a.vd[( 5 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[( 6 + ii*LBQ ) * para.number_of_nodes + index])
                                 );

    mode[17 + ii * LBQ] = - (n_a.vd[( 1 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[( 2 + ii*LBQ ) * para.number_of_nodes + index])
                          + (n_a.vd[( 3 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[( 4 + ii*LBQ ) * para.number_of_nodes + index])
                          + (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index])
                          + (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index])
                          - (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index])
                          - (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]);

    mode[18 + ii * LBQ] = - (n_a.vd[( 1 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[( 2 + ii*LBQ ) * para.number_of_nodes + index])
                          - (n_a.vd[( 3 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[( 4 + ii*LBQ ) * para.number_of_nodes + index])
                          - (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index])
                          - (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index])
                          - (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index])
                          - (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index])
                          + 2.0f*(
                                     (n_a.vd[( 5 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[( 6 + ii*LBQ ) * para.number_of_nodes + index])
                                   + (n_a.vd[( 7 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[( 8 + ii*LBQ ) * para.number_of_nodes + index])
                                   + (n_a.vd[( 9 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index])
                                 );
  }
}

__device__ void reset_LB_forces(unsigned int index, LB_node_force_gpu node_f, bool buffer = true) {

  float force_factor=powf(para.agrid,2)*para.tau*para.tau;
  for(int ii=0;ii<LB_COMPONENTS;++ii)
  {

#if defined(IMMERSED_BOUNDARY) || defined(EK_DEBUG)
// Store backup of the node forces
    if (buffer)
    {
      node_f.force_buf[(0 + ii*3 ) * para.number_of_nodes + index] = node_f.force[(0 + ii*3 ) * para.number_of_nodes + index];
      node_f.force_buf[(1 + ii*3 ) * para.number_of_nodes + index] = node_f.force[(1 + ii*3 ) * para.number_of_nodes + index];
      node_f.force_buf[(2 + ii*3 ) * para.number_of_nodes + index] = node_f.force[(2 + ii*3 ) * para.number_of_nodes + index];
    }
#endif

#ifdef EXTERNAL_FORCES
      if(para.external_force)
      {
        node_f.force[(0 + ii*3 ) * para.number_of_nodes + index] = para.ext_force[0 + ii*3 ]*force_factor;
        node_f.force[(1 + ii*3 ) * para.number_of_nodes + index] = para.ext_force[1 + ii*3 ]*force_factor;
        node_f.force[(2 + ii*3 ) * para.number_of_nodes + index] = para.ext_force[2 + ii*3 ]*force_factor;
      }
      else
      {
        node_f.force[(0 + ii*3 ) * para.number_of_nodes + index] = 0.0f;
        node_f.force[(1 + ii*3 ) * para.number_of_nodes + index] = 0.0f;
        node_f.force[(2 + ii*3 ) * para.number_of_nodes + index] = 0.0f;
      }
#else
      /** reset force */
      node_f.force[(0 + ii*3 ) * para.number_of_nodes + index] = 0.0f;
      node_f.force[(1 + ii*3 ) * para.number_of_nodes + index] = 0.0f;
      node_f.force[(2 + ii*3 ) * para.number_of_nodes + index] = 0.0f;
#endif
  }
}

__global__ void reset_LB_forces_kernel(LB_node_force_gpu node_f, bool buffer = true) {
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if( index < para.number_of_nodes )
    reset_LB_forces(index, node_f, buffer);
}

void reset_LB_forces_GPU(bool buffer) {
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(reset_LB_forces_kernel, dim_grid, threads_per_block, (node_f, buffer));
}


__device__ void update_rho_v(float *mode, unsigned int index, LB_node_force_gpu node_f, LB_rho_v_gpu *d_v){

  float Rho_tot=0.0f;
  float u_tot[3]={0.0f,0.0f,0.0f};
  
  #pragma unroll
  for(int ii=0;ii<LB_COMPONENTS;++ii)
  { 
      /** re-construct the real density
      * remember that the populations are stored as differences to their
      * equilibrium value */

      d_v[index].rho[ii] = mode[0 + ii * LBQ] + para.rho[ii]*para.agrid*para.agrid*para.agrid;
      Rho_tot  += mode[0 + ii * LBQ] + para.rho[ii]*para.agrid*para.agrid*para.agrid;
      u_tot[0] += mode[1 + ii * LBQ];
      u_tot[1] += mode[2 + ii * LBQ];
      u_tot[2] += mode[3 + ii * LBQ];

      /** if forces are present, the momentum density is redefined to
      * inlcude one half-step of the force action.  See the
      * Chapman-Enskog expansion in [Ladd & Verberg]. */

      u_tot[0] += 0.5f*node_f.force[(0+ii*3)*para.number_of_nodes + index];
      u_tot[1] += 0.5f*node_f.force[(1+ii*3)*para.number_of_nodes + index];
      u_tot[2] += 0.5f*node_f.force[(2+ii*3)*para.number_of_nodes + index];
  }

  u_tot[0]/=Rho_tot;
  u_tot[1]/=Rho_tot;
  u_tot[2]/=Rho_tot;

  d_v[index].v[0]=u_tot[0]; 
  d_v[index].v[1]=u_tot[1]; 
  d_v[index].v[2]=u_tot[2]; 
}

/**lb_relax_modes, means collision update of the modes
 * @param index   node index / thread index (Input)
 * @param mode    Pointer to the local register values mode (Input/Output)
 * @param node_f  Pointer to local node force (Input)
 * @param *d_v    Pointer to local device values
*/
__device__ void relax_modes(float *mode, unsigned int index, LB_node_force_gpu node_f, LB_rho_v_gpu *d_v){
  float u_tot[3]={0.0f,0.0f,0.0f};

  update_rho_v(mode, index, node_f, d_v);

  u_tot[0]=d_v[index].v[0];  
  u_tot[1]=d_v[index].v[1];  
  u_tot[2]=d_v[index].v[2];  
 
  #pragma unroll
  for(int ii=0;ii<LB_COMPONENTS;++ii)
  { 
      float Rho; float j[3]; float modes_from_pi_eq[6];

      Rho = mode[0 + ii * LBQ] + para.rho[ii]*para.agrid*para.agrid*para.agrid ;
      j[0] = Rho * u_tot[0];
      j[1] = Rho * u_tot[1];
      j[2] = Rho * u_tot[2];

      /** equilibrium part of the stress modes (eq13 schiller)*/

      modes_from_pi_eq[0] = ((j[0]*j[0])+(j[1]*j[1])+(j[2]*j[2]))/Rho;
      modes_from_pi_eq[1] = ((j[0]*j[0])-(j[1]*j[1]))/Rho;
      modes_from_pi_eq[2] = (((j[0]*j[0])+(j[1]*j[1])+(j[2]*j[2])) - 3.0f*(j[2]*j[2]))/Rho;
      modes_from_pi_eq[3] = j[0]*j[1]/Rho;
      modes_from_pi_eq[4] = j[0]*j[2]/Rho;
      modes_from_pi_eq[5] = j[1]*j[2]/Rho;
 
      /** in Shan-Chen we have to relax the momentum modes as well using the mobility, but
          the total momentum is conserved */  

#ifdef SHANCHEN
      mode[1 + ii * LBQ] = j[0] + para.gamma_mobility[0]*(mode[1 + ii * LBQ] - j[0]);
      mode[2 + ii * LBQ] = j[1] + para.gamma_mobility[0]*(mode[2 + ii * LBQ] - j[1]);
      mode[3 + ii * LBQ] = j[2] + para.gamma_mobility[0]*(mode[3 + ii * LBQ] - j[2]);
#endif
 
      /** relax the stress modes (eq14 schiller)*/

      mode[4 + ii * LBQ] = modes_from_pi_eq[0] +  para.gamma_bulk[ii]*(mode[4 + ii * LBQ] - modes_from_pi_eq[0]);
      mode[5 + ii * LBQ] = modes_from_pi_eq[1] + para.gamma_shear[ii]*(mode[5 + ii * LBQ] - modes_from_pi_eq[1]);
      mode[6 + ii * LBQ] = modes_from_pi_eq[2] + para.gamma_shear[ii]*(mode[6 + ii * LBQ] - modes_from_pi_eq[2]);
      mode[7 + ii * LBQ] = modes_from_pi_eq[3] + para.gamma_shear[ii]*(mode[7 + ii * LBQ] - modes_from_pi_eq[3]);
      mode[8 + ii * LBQ] = modes_from_pi_eq[4] + para.gamma_shear[ii]*(mode[8 + ii * LBQ] - modes_from_pi_eq[4]);
      mode[9 + ii * LBQ] = modes_from_pi_eq[5] + para.gamma_shear[ii]*(mode[9 + ii * LBQ] - modes_from_pi_eq[5]);
    
      /** relax the ghost modes (project them out) */
      /** ghost modes have no equilibrium part due to orthogonality */

      mode[10 + ii * LBQ] =  para.gamma_odd[ii]*mode[10 + ii * LBQ];
      mode[11 + ii * LBQ] =  para.gamma_odd[ii]*mode[11 + ii * LBQ];
      mode[12 + ii * LBQ] =  para.gamma_odd[ii]*mode[12 + ii * LBQ];
      mode[13 + ii * LBQ] =  para.gamma_odd[ii]*mode[13 + ii * LBQ];
      mode[14 + ii * LBQ] =  para.gamma_odd[ii]*mode[14 + ii * LBQ];
      mode[15 + ii * LBQ] =  para.gamma_odd[ii]*mode[15 + ii * LBQ];
      mode[16 + ii * LBQ] = para.gamma_even[ii]*mode[16 + ii * LBQ];
      mode[17 + ii * LBQ] = para.gamma_even[ii]*mode[17 + ii * LBQ];
      mode[18 + ii * LBQ] = para.gamma_even[ii]*mode[18 + ii * LBQ];
  }
}


/**thermalization of the modes with gaussian random numbers
 * @param index   node index / thread index (Input)
 * @param mode    Pointer to the local register values mode (Input/Output)
 * @param *rn     Pointer to randomnumber array of the local node
*/
__device__ void thermalize_modes(float *mode, unsigned int index, LB_randomnr_gpu *rn){
  float Rho;
#ifdef SHANCHEN
  float Rho_tot=0.0,c;
  #pragma unroll
  for(int ii=0;ii<LB_COMPONENTS;++ii) { 
      Rho_tot  += mode[0 + ii * LBQ]+ para.rho[ii]*para.agrid*para.agrid*para.agrid;
  }
  c = (mode[0 + 0 * LBQ]+ para.rho[0]*para.agrid*para.agrid*para.agrid ) / Rho_tot;
  random_wrapper(rn);
  for(int ii=0;ii<LB_COMPONENTS;++ii) { 
      mode[1 + ii * LBQ] +=  sqrtf(c*(1-c)*Rho_tot*(para.mu[ii]*(2.0f/3.0f)*(1.0f-(para.gamma_mobility[0]*para.gamma_mobility[0])))) * (2*ii-1) * rn->randomnr[0];
      mode[2 + ii * LBQ] +=  sqrtf(c*(1-c)*Rho_tot*(para.mu[ii]*(2.0f/3.0f)*(1.0f-(para.gamma_mobility[0]*para.gamma_mobility[0])))) * (2*ii-1) * rn->randomnr[1];
  }                                      
  random_wrapper(rn);                    
  for(int ii=0;ii<LB_COMPONENTS;++ii)    
      mode[3 + ii * LBQ] +=  sqrtf(c*(1-c)*Rho_tot*(para.mu[ii]*(2.0f/3.0f)*(1.0f-( para.gamma_mobility[0]*para.gamma_mobility[0])))) * (2*ii-1) * rn->randomnr[0];
#endif
  
  
  for(int ii=0;ii<LB_COMPONENTS;++ii)
  {  

    /** mass mode */  
    Rho = mode[0 + ii * LBQ] + para.rho[ii]*para.agrid*para.agrid*para.agrid;

    /** momentum modes */

    /** stress modes */
    random_wrapper(rn);
    mode[4 + ii * LBQ] += sqrtf(Rho*(para.mu[ii]*(2.0f/3.0f)*(1.0f-( para.gamma_bulk[ii]* para.gamma_bulk[ii])))) * rn->randomnr[0];
    mode[5 + ii * LBQ] += sqrtf(Rho*(para.mu[ii]*(4.0f/9.0f)*(1.0f-(para.gamma_shear[ii]*para.gamma_shear[ii])))) * rn->randomnr[1];

    random_wrapper(rn);
    mode[6 + ii * LBQ] += sqrtf(Rho*(para.mu[ii]*(4.0f/3.0f)*(1.0f-(para.gamma_shear[ii]*para.gamma_shear[ii])))) * rn->randomnr[0];
    mode[7 + ii * LBQ] += sqrtf(Rho*(para.mu[ii]*(1.0f/9.0f)*(1.0f-(para.gamma_shear[ii]*para.gamma_shear[ii])))) * rn->randomnr[1];

    random_wrapper(rn);
    mode[8 + ii * LBQ] += sqrtf(Rho*(para.mu[ii]*(1.0f/9.0f)*(1.0f-(para.gamma_shear[ii]*para.gamma_shear[ii])))) * rn->randomnr[0];
    mode[9 + ii * LBQ] += sqrtf(Rho*(para.mu[ii]*(1.0f/9.0f)*(1.0f-(para.gamma_shear[ii]*para.gamma_shear[ii])))) * rn->randomnr[1];

    /** ghost modes */
    random_wrapper(rn);
    mode[10 + ii * LBQ] += sqrtf(Rho*(para.mu[ii]*(2.0f/3.0f)*(1.0f-(para.gamma_odd[ii]*para.gamma_odd[ii])))) * rn->randomnr[0];
    mode[11 + ii * LBQ] += sqrtf(Rho*(para.mu[ii]*(2.0f/3.0f)*(1.0f-(para.gamma_odd[ii]*para.gamma_odd[ii])))) * rn->randomnr[1];

    random_wrapper(rn);
    mode[12 + ii * LBQ] += sqrtf(Rho*(para.mu[ii]*(2.0f/3.0f)*(1.0f-(para.gamma_odd[ii]*para.gamma_odd[ii])))) * rn->randomnr[0];
    mode[13 + ii * LBQ] += sqrtf(Rho*(para.mu[ii]*(2.0f/9.0f)*(1.0f-(para.gamma_odd[ii]*para.gamma_odd[ii])))) * rn->randomnr[1];

    random_wrapper(rn);
    mode[14 + ii * LBQ] += sqrtf(Rho*(para.mu[ii]*(2.0f/9.0f)*(1.0f-(para.gamma_odd[ii]*para.gamma_odd[ii])))) * rn->randomnr[0];
    mode[15 + ii * LBQ] += sqrtf(Rho*(para.mu[ii]*(2.0f/9.0f)*(1.0f-(para.gamma_odd[ii]*para.gamma_odd[ii])))) * rn->randomnr[1];

    random_wrapper(rn);
    mode[16 + ii * LBQ] += sqrtf(Rho*(para.mu[ii]*(2.0f)*(1.0f-(para.gamma_even[ii]*para.gamma_even[ii]))))     * rn->randomnr[0];
    mode[17 + ii * LBQ] += sqrtf(Rho*(para.mu[ii]*(4.0f/9.0f)*(1.0f-(para.gamma_even[ii]*para.gamma_even[ii])))) * rn->randomnr[1];

    random_wrapper(rn);
    mode[18 + ii * LBQ] += sqrtf(Rho*(para.mu[ii]*(4.0f/3.0f)*(1.0f-(para.gamma_even[ii]*para.gamma_even[ii])))) * rn->randomnr[0];
  }
}


/*-------------------------------------------------------*/
/**normalization of the modes need befor backtransformation into velocity space
 * @param mode    Pointer to the local register values mode (Input/Output)
*/
__device__ void normalize_modes(float* mode){
  #pragma unroll
  for(int ii=0;ii<LB_COMPONENTS;++ii)
  { 
    /** normalization factors enter in the back transformation */
    mode[ 0 + ii * LBQ] *= 1.0f;
    mode[ 1 + ii * LBQ] *= 3.0f;
    mode[ 2 + ii * LBQ] *= 3.0f;
    mode[ 3 + ii * LBQ] *= 3.0f;
    mode[ 4 + ii * LBQ] *= 3.0f/2.0f;
    mode[ 5 + ii * LBQ] *= 9.0f/4.0f;
    mode[ 6 + ii * LBQ] *= 3.0f/4.0f;
    mode[ 7 + ii * LBQ] *= 9.0f;
    mode[ 8 + ii * LBQ] *= 9.0f;
    mode[ 9 + ii * LBQ] *= 9.0f;
    mode[10 + ii * LBQ] *= 3.0f/2.0f;
    mode[11 + ii * LBQ] *= 3.0f/2.0f;
    mode[12 + ii * LBQ] *= 3.0f/2.0f;
    mode[13 + ii * LBQ] *= 9.0f/2.0f;
    mode[14 + ii * LBQ] *= 9.0f/2.0f;
    mode[15 + ii * LBQ] *= 9.0f/2.0f;
    mode[16 + ii * LBQ] *= 1.0f/2.0f;
    mode[17 + ii * LBQ] *= 9.0f/4.0f;
    mode[18 + ii * LBQ] *= 3.0f/4.0f;
  }
}



/*-------------------------------------------------------*/
/**backtransformation from modespace to desityspace and streaming with the push method using pbc
 * @param index   node index / thread index (Input)
 * @param mode    Pointer to the local register values mode (Input)
 * @param *n_b    Pointer to local node residing in array b (Output)
*/
__device__ void calc_n_from_modes_push(LB_nodes_gpu n_b, float *mode, unsigned int index){

  unsigned int xyz[3];
  index_to_xyz(index, xyz);
  unsigned int x = xyz[0];
  unsigned int y = xyz[1];
  unsigned int z = xyz[2];

  #pragma unroll
  for(int ii=0;ii<LB_COMPONENTS;++ii)
  {
 
    n_b.vd[(0 + ii*LBQ ) * para.number_of_nodes + x 
                                                + para.dim_x*y
                                                + para.dim_x*para.dim_y*z] = 
      1.0f/3.0f * (mode[0 + ii * LBQ] - mode[4 + ii * LBQ] + mode[16 + ii * LBQ]);

    n_b.vd[(1 + ii*LBQ ) * para.number_of_nodes + (x+1)%para.dim_x
                                                + para.dim_x*y 
                                                + para.dim_x*para.dim_y*z] = 
      1.0f/18.0f * (
                       mode[ 0 + ii * LBQ] + mode[ 1 + ii * LBQ]
                     + mode[ 5 + ii * LBQ] + mode[ 6 + ii * LBQ]
                     - mode[17 + ii * LBQ] - mode[18 + ii * LBQ]
                     - 2.0f*(mode[10 + ii * LBQ] + mode[16 + ii * LBQ])
                   );

    n_b.vd[(2 + ii*LBQ ) * para.number_of_nodes + (para.dim_x+x-1)%para.dim_x
                                                + para.dim_x*y
                                                + para.dim_x*para.dim_y*z] =
      1.0f/18.0f * (
                       mode[ 0 + ii * LBQ] - mode[ 1 + ii * LBQ]
                     + mode[ 5 + ii * LBQ] + mode[ 6 + ii * LBQ]
                     - mode[17 + ii * LBQ] - mode[18 + ii * LBQ]
                     + 2.0f*(mode[10 + ii * LBQ] - mode[16 + ii * LBQ])
                   );

    n_b.vd[(3 + ii*LBQ ) * para.number_of_nodes + x
                                                + para.dim_x*((y+1)%para.dim_y)
                                                + para.dim_x*para.dim_y*z] =
      1.0f/18.0f * (
                       mode[ 0 + ii * LBQ] + mode[ 2 + ii * LBQ]
                     - mode[ 5 + ii * LBQ] + mode[ 6 + ii * LBQ]
                     + mode[17 + ii * LBQ] - mode[18 + ii * LBQ]
                     - 2.0f*(mode[11 + ii * LBQ] + mode[16 + ii * LBQ])
                   );

    n_b.vd[(4 + ii*LBQ ) * para.number_of_nodes + x
                                                + para.dim_x*((para.dim_y+y-1)%para.dim_y)
                                                + para.dim_x*para.dim_y*z] =
      1.0f/18.0f * (
                       mode[ 0 + ii * LBQ] - mode[ 2 + ii * LBQ]
                     - mode[ 5 + ii * LBQ] + mode[ 6 + ii * LBQ]
                     + mode[17 + ii * LBQ] - mode[18 + ii * LBQ]
                     + 2.0f*(mode[11 + ii * LBQ] - mode[16 + ii * LBQ])
                   );

    n_b.vd[(5 + ii*LBQ ) * para.number_of_nodes + x
                                                + para.dim_x*y
                                                + para.dim_x*para.dim_y*((z+1)%para.dim_z)] =
      1.0f/18.0f * (
                       mode[0 + ii * LBQ] + mode[3 + ii * LBQ]
                     - 2.0f*(   mode[ 6 + ii * LBQ] + mode[12 + ii * LBQ]
                              + mode[16 + ii * LBQ] - mode[18 + ii * LBQ])
                   );

    n_b.vd[(6 + ii*LBQ ) * para.number_of_nodes + x
                                                + para.dim_x*y
                                                + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] =
      1.0f/18.0f * (
                       mode[0 + ii * LBQ] - mode[3 + ii * LBQ]
                     - 2.0f*(   mode[6 + ii * LBQ] - mode[12 + ii * LBQ]
                              + mode[16 + ii * LBQ] - mode[18 + ii * LBQ])
                   );

    n_b.vd[(7 + ii*LBQ ) * para.number_of_nodes + (x+1)%para.dim_x
                                                + para.dim_x*((y+1)%para.dim_y)
                                                + para.dim_x*para.dim_y*z] =
      1.0f/36.0f * (
                       mode[ 0 + ii * LBQ] + mode[ 1 + ii * LBQ]
                     + mode[ 2 + ii * LBQ] + mode[ 4 + ii * LBQ]
                     + 2.0f*mode[ 6 + ii * LBQ] + mode[ 7 + ii * LBQ]
                     + mode[10 + ii * LBQ] + mode[11 + ii * LBQ]
                     + mode[13 + ii * LBQ] + mode[14 + ii * LBQ]
                     + mode[16 + ii * LBQ] + 2.0f*mode[18 + ii * LBQ]
                   );

    n_b.vd[(8 + ii*LBQ ) * para.number_of_nodes + (para.dim_x+x-1)%para.dim_x
                                                + para.dim_x*((para.dim_y+y-1)%para.dim_y)
                                                + para.dim_x*para.dim_y*z] =
      1.0f/36.0f * (
                       mode[ 0 + ii * LBQ] - mode[ 1 + ii * LBQ]
                     - mode[ 2 + ii * LBQ] + mode[ 4 + ii * LBQ]
                     + 2.0f*mode[ 6 + ii * LBQ] + mode[ 7 + ii * LBQ]
                     - mode[10 + ii * LBQ] - mode[11 + ii * LBQ]
                     - mode[13 + ii * LBQ] - mode[14 + ii * LBQ]
                     + mode[16 + ii * LBQ] + 2.0f*mode[18 + ii * LBQ]
                   );

    n_b.vd[(9 + ii*LBQ ) * para.number_of_nodes + (x+1)%para.dim_x
                                                + para.dim_x*((para.dim_y+y-1)%para.dim_y)
                                                + para.dim_x*para.dim_y*z] =
      1.0f/36.0f * (
                       mode[ 0 + ii * LBQ] + mode[ 1 + ii * LBQ]
                     - mode[ 2 + ii * LBQ] + mode[ 4 + ii * LBQ]
                     + 2.0f*mode[ 6 + ii * LBQ] - mode[ 7 + ii * LBQ]
                     + mode[10 + ii * LBQ] - mode[11 + ii * LBQ]
                     + mode[13 + ii * LBQ] - mode[14 + ii * LBQ]
                     + mode[16 + ii * LBQ] + 2.0f*mode[18 + ii * LBQ]
                   );

    n_b.vd[(10 + ii*LBQ ) * para.number_of_nodes + (para.dim_x+x-1)%para.dim_x
                                                 + para.dim_x*((y+1)%para.dim_y)
                                                 + para.dim_x*para.dim_y*z] = 
      1.0f/36.0f * (
                       mode[ 0 + ii * LBQ] - mode[ 1 + ii * LBQ]
                     + mode[ 2 + ii * LBQ] + mode[ 4 + ii * LBQ]
                     + 2.0f*mode[ 6 + ii * LBQ] - mode[ 7 + ii * LBQ]
                     - mode[10 + ii * LBQ] + mode[11 + ii * LBQ]
                     - mode[13 + ii * LBQ] + mode[14 + ii * LBQ]
                     + mode[16 + ii * LBQ] + 2.0f*mode[18 + ii * LBQ]
                   );

    n_b.vd[(11 + ii*LBQ ) * para.number_of_nodes + (x+1)%para.dim_x
                                                 + para.dim_x*y
                                                 + para.dim_x*para.dim_y*((z+1)%para.dim_z)] =
      1.0f/36.0f * (
                       mode[ 0 + ii * LBQ] + mode[ 1 + ii * LBQ]
                     + mode[ 3 + ii * LBQ] + mode[ 4 + ii * LBQ]
                     + mode[ 5 + ii * LBQ] - mode[ 6 + ii * LBQ]
                     + mode[ 8 + ii * LBQ] + mode[10 + ii * LBQ]
                     + mode[12 + ii * LBQ] - mode[13 + ii * LBQ]
                     + mode[15 + ii * LBQ] + mode[16 + ii * LBQ]
                     + mode[17 + ii * LBQ] - mode[18 + ii * LBQ]
                   );

    n_b.vd[(12 + ii*LBQ ) * para.number_of_nodes + (para.dim_x+x-1)%para.dim_x
                                                 + para.dim_x*y
                                                 + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] =
      1.0f/36.0f * (
                       mode[ 0 + ii * LBQ] - mode[ 1 + ii * LBQ]
                     - mode[ 3 + ii * LBQ] + mode[ 4 + ii * LBQ]
                     + mode[ 5 + ii * LBQ] - mode[ 6 + ii * LBQ]
                     + mode[ 8 + ii * LBQ] - mode[10 + ii * LBQ]
                     - mode[12 + ii * LBQ] + mode[13 + ii * LBQ]
                     - mode[15 + ii * LBQ] + mode[16 + ii * LBQ]
                     + mode[17 + ii * LBQ] - mode[18 + ii * LBQ]
                   );

    n_b.vd[(13 + ii*LBQ ) * para.number_of_nodes + (x+1)%para.dim_x
                                                 + para.dim_x*y
                                                 + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] =
      1.0f/36.0f * (
                       mode[ 0 + ii * LBQ] + mode[ 1 + ii * LBQ]
                     - mode[ 3 + ii * LBQ] + mode[ 4 + ii * LBQ]
                     + mode[ 5 + ii * LBQ] - mode[ 6 + ii * LBQ]
                     - mode[ 8 + ii * LBQ] + mode[10 + ii * LBQ]
                     - mode[12 + ii * LBQ] - mode[13 + ii * LBQ]
                     - mode[15 + ii * LBQ] + mode[16 + ii * LBQ]
                     + mode[17 + ii * LBQ] - mode[18 + ii * LBQ]
                   );

    n_b.vd[(14 + ii*LBQ ) * para.number_of_nodes + (para.dim_x+x-1)%para.dim_x
                                                 + para.dim_x*y
                                                 + para.dim_x*para.dim_y*((z+1)%para.dim_z)] =
      1.0f/36.0f * (
                       mode[ 0 + ii * LBQ] - mode[ 1 + ii * LBQ]
                     + mode[ 3 + ii * LBQ] + mode[ 4 + ii * LBQ]
                     + mode[ 5 + ii * LBQ] - mode[ 6 + ii * LBQ]
                     - mode[ 8 + ii * LBQ] - mode[10 + ii * LBQ]
                     + mode[12 + ii * LBQ] + mode[13 + ii * LBQ]
                     + mode[15 + ii * LBQ] + mode[16 + ii * LBQ]
                     + mode[17 + ii * LBQ] - mode[18 + ii * LBQ]
                   );

    n_b.vd[(15 + ii*LBQ ) * para.number_of_nodes + x
                                                 + para.dim_x*((y+1)%para.dim_y)
                                                 + para.dim_x*para.dim_y*((z+1)%para.dim_z)] =
      1.0f/36.0f * (
                       mode[ 0 + ii * LBQ] + mode[ 2 + ii * LBQ]
                     + mode[ 3 + ii * LBQ] + mode[ 4 + ii * LBQ]
                     - mode[ 5 + ii * LBQ] - mode[ 6 + ii * LBQ]
                     + mode[ 9 + ii * LBQ] + mode[11 + ii * LBQ]
                     + mode[12 + ii * LBQ] - mode[14 + ii * LBQ]
                     - mode[15 + ii * LBQ] + mode[16 + ii * LBQ]
                     - mode[17 + ii * LBQ] - mode[18 + ii * LBQ]
                   );

    n_b.vd[(16 + ii*LBQ ) * para.number_of_nodes + x
                                                 + para.dim_x*((para.dim_y+y-1)%para.dim_y)
                                                 + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] =
      1.0f/36.0f * (
                       mode[ 0 + ii * LBQ] - mode[ 2 + ii * LBQ]
                     - mode[ 3 + ii * LBQ] + mode[ 4 + ii * LBQ]
                     - mode[ 5 + ii * LBQ] - mode[ 6 + ii * LBQ]
                     + mode[ 9 + ii * LBQ] - mode[11 + ii * LBQ]
                     - mode[12 + ii * LBQ] + mode[14 + ii * LBQ]
                     + mode[15 + ii * LBQ] + mode[16 + ii * LBQ]
                     - mode[17 + ii * LBQ] - mode[18 + ii * LBQ]
                   );

    n_b.vd[(17 + ii*LBQ ) * para.number_of_nodes + x
                                                 + para.dim_x*((y+1)%para.dim_y)
                                                 + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] =
      1.0f/36.0f * (
                       mode[ 0 + ii * LBQ] + mode[ 2 + ii * LBQ]
                     - mode[ 3 + ii * LBQ] + mode[ 4 + ii * LBQ]
                     - mode[ 5 + ii * LBQ] - mode[ 6 + ii * LBQ]
                     - mode[ 9 + ii * LBQ] + mode[11 + ii * LBQ]
                     - mode[12 + ii * LBQ] - mode[14 + ii * LBQ]
                     + mode[15 + ii * LBQ] + mode[16 + ii * LBQ]
                     - mode[17 + ii * LBQ] - mode[18 + ii * LBQ]
                   );

    n_b.vd[(18 + ii*LBQ ) * para.number_of_nodes + x
                                                 + para.dim_x*((para.dim_y+y-1)%para.dim_y)
                                                 + para.dim_x*para.dim_y*((z+1)%para.dim_z)] =
      1.0f/36.0f * (
                       mode[ 0 + ii * LBQ] - mode[ 2 + ii * LBQ]
                     + mode[ 3 + ii * LBQ] + mode[ 4 + ii * LBQ]
                     - mode[ 5 + ii * LBQ] - mode[ 6 + ii * LBQ]
                     - mode[ 9 + ii * LBQ] - mode[11 + ii * LBQ]
                     + mode[12 + ii * LBQ] + mode[14 + ii * LBQ]
                     - mode[15 + ii * LBQ] + mode[16 + ii * LBQ]
                     - mode[17 + ii * LBQ] - mode[18 + ii * LBQ]
                   );
  }
}



/** Bounce back boundary conditions.
 * The populations that have propagated into a boundary node
 * are bounced back to the node they came from. This results
 * in no slip boundary conditions.
 *
 * [cf. Ladd and Verberg, J. Stat. Phys. 104(5/6):1191-1251, 2001]
 * @param index   node index / thread index (Input)
 * @param n_curr  Pointer to local node which receives the current node field (Input)
 * @param lb_boundary_velocity    The constant velocity at the boundary, set by the user (Input)
 * @param lb_boundary_force       The force on the boundary nodes (Output)
*/
__device__ void bounce_back_boundaries(LB_nodes_gpu n_curr, unsigned int index, \
    float* lb_boundary_velocity, float* lb_boundary_force){
    
  unsigned int xyz[3];
  int c[3];
  float v[3];
  float shift, weight, pop_to_bounce_back;
  float boundary_force[3] = {0.0f,0.0f,0.0f};
  size_t to_index, to_index_x, to_index_y, to_index_z;
  int population, inverse;
  int boundary_index;

  boundary_index= n_curr.boundary[index];
  if(boundary_index != 0)
  {
    
    v[0]=lb_boundary_velocity[3*(boundary_index-1)+0];
    v[1]=lb_boundary_velocity[3*(boundary_index-1)+1];
    v[2]=lb_boundary_velocity[3*(boundary_index-1)+2];

    index_to_xyz(index, xyz);

    unsigned int x = xyz[0];
    unsigned int y = xyz[1];
    unsigned int z = xyz[2];

    /* CPU analog of shift:
       lbpar.agrid*lbpar.agrid*lbpar.agrid*lbpar.rho*2*lbmodel.c[i][l]*lb_boundaries[lbfields[k].boundary-1].velocity[l] */
  
    /** store vd temporary in second lattice to avoid race conditions */

// TODO : PUT IN EQUILIBRIUM CONTRIBUTION TO THE BOUNCE-BACK DENSITY FOR THE BOUNDARY FORCE
// TODO : INITIALIZE BOUNDARY FORCE PROPERLY, HAS NONZERO ELEMENTS IN FIRST STEP
// TODO : SET INTERNAL BOUNDARY NODE VALUES TO ZERO

#ifndef SHANCHEN

#define BOUNCEBACK()  \
  shift = 2.0f*para.agrid*para.agrid*para.rho[0]*3.0f*weight*para.tau*(v[0]*c[0] + v[1]*c[1] + v[2]*c[2]); \
  pop_to_bounce_back =  n_curr.vd[population*para.number_of_nodes + index ]; \
  to_index_x = (x+c[0]+para.dim_x)%para.dim_x; \
  to_index_y = (y+c[1]+para.dim_y)%para.dim_y; \
  to_index_z = (z+c[2]+para.dim_z)%para.dim_z; \
  to_index = to_index_x + para.dim_x*to_index_y + para.dim_x*para.dim_y*to_index_z; \
  if ( n_curr.boundary[to_index] == 0) \
  { \
    boundary_force[0] += (2.0f*pop_to_bounce_back+shift)*c[0]/para.tau/para.tau/para.agrid; \
    boundary_force[1] += (2.0f*pop_to_bounce_back+shift)*c[1]/para.tau/para.tau/para.agrid; \
    boundary_force[2] += (2.0f*pop_to_bounce_back+shift)*c[2]/para.tau/para.tau/para.agrid; \
     n_curr.vd[inverse*para.number_of_nodes + to_index ] = pop_to_bounce_back + shift; \
  }
#else 

#define BOUNCEBACK()  \
  for(int component=0; component<LB_COMPONENTS;component++){\
     shift = 2.0f*para.agrid*para.agrid*para.rho[component]*3.0f*weight*para.tau*(v[0]*c[0] + v[1]*c[1] + v[2]*c[2]); \
     pop_to_bounce_back =  n_curr.vd[(population+component*LBQ)*para.number_of_nodes + index ]; \
     to_index_x = (x+c[0]+para.dim_x)%para.dim_x; \
     to_index_y = (y+c[1]+para.dim_y)%para.dim_y; \
     to_index_z = (z+c[2]+para.dim_z)%para.dim_z; \
     to_index = to_index_x + para.dim_x*to_index_y + para.dim_x*para.dim_y*to_index_z; \
     if ( n_curr.boundary[to_index] == 0) \
     { \
       boundary_force[0] += (2.0f*pop_to_bounce_back+shift)*c[0]/para.tau/para.tau/para.agrid; \
       boundary_force[1] += (2.0f*pop_to_bounce_back+shift)*c[1]/para.tau/para.tau/para.agrid; \
       boundary_force[2] += (2.0f*pop_to_bounce_back+shift)*c[2]/para.tau/para.tau/para.agrid; \
       n_curr.vd[(inverse+component*LBQ)*para.number_of_nodes + to_index ] = pop_to_bounce_back + shift; \
       n_curr.vd[(inverse+component*LBQ)*para.number_of_nodes + to_index ] = pop_to_bounce_back + shift; \
     } \
  }
#endif

    // the resting population does nothing, i.e., population 0.
    c[0]= 1;c[1]= 0;c[2]= 0; weight=1./18.; population= 2; inverse= 1; 
    BOUNCEBACK();
    
    c[0]=-1;c[1]= 0;c[2]= 0; weight=1./18.; population= 1; inverse= 2; 
    BOUNCEBACK();
    
    c[0]= 0;c[1]= 1;c[2]= 0; weight=1./18.; population= 4; inverse= 3; 
    BOUNCEBACK();

    c[0]= 0;c[1]=-1;c[2]= 0; weight=1./18.; population= 3; inverse= 4; 
    BOUNCEBACK();
    
    c[0]= 0;c[1]= 0;c[2]= 1; weight=1./18.; population= 6; inverse= 5; 
    BOUNCEBACK();

    c[0]= 0;c[1]= 0;c[2]=-1; weight=1./18.; population= 5; inverse= 6; 
    BOUNCEBACK(); 
    
    c[0]= 1;c[1]= 1;c[2]= 0; weight=1./36.; population= 8; inverse= 7; 
    BOUNCEBACK();
    
    c[0]=-1;c[1]=-1;c[2]= 0; weight=1./36.; population= 7; inverse= 8; 
    BOUNCEBACK();
    
    c[0]= 1;c[1]=-1;c[2]= 0; weight=1./36.; population=10; inverse= 9; 
    BOUNCEBACK();

    c[0]=-1;c[1]= 1;c[2]= 0; weight=1./36.; population= 9; inverse=10; 
    BOUNCEBACK();
    
    c[0]= 1;c[1]= 0;c[2]= 1; weight=1./36.; population=12; inverse=11; 
    BOUNCEBACK();
    
    c[0]=-1;c[1]= 0;c[2]=-1; weight=1./36.; population=11; inverse=12; 
    BOUNCEBACK();

    c[0]= 1;c[1]= 0;c[2]=-1; weight=1./36.; population=14; inverse=13; 
    BOUNCEBACK();
    
    c[0]=-1;c[1]= 0;c[2]= 1; weight=1./36.; population=13; inverse=14; 
    BOUNCEBACK();

    c[0]= 0;c[1]= 1;c[2]= 1; weight=1./36.; population=16; inverse=15; 
    BOUNCEBACK();
    
    c[0]= 0;c[1]=-1;c[2]=-1; weight=1./36.; population=15; inverse=16; 
    BOUNCEBACK();
    
    c[0]= 0;c[1]= 1;c[2]=-1; weight=1./36.; population=18; inverse=17; 
    BOUNCEBACK();
    
    c[0]= 0;c[1]=-1;c[2]= 1; weight=1./36.; population=17; inverse=18; 
    BOUNCEBACK();  
    
    atomicadd(&lb_boundary_force[3*( n_curr.boundary[index]-1)+0], boundary_force[0]);
    atomicadd(&lb_boundary_force[3*( n_curr.boundary[index]-1)+1], boundary_force[1]);
    atomicadd(&lb_boundary_force[3*( n_curr.boundary[index]-1)+2], boundary_force[2]);
  }
}

/** add of (external) forces within the modespace, needed for particle-interaction
 * @param index   node index / thread index (Input)
 * @param mode    Pointer to the local register values mode (Input/Output)
 * @param node_f  Pointer to local node force (Input)
 * @param *d_v    Pointer to local device values
*/
__device__ void apply_forces(unsigned int index, float *mode, LB_node_force_gpu node_f, LB_rho_v_gpu *d_v) {
  
  float u[3]={0.0f,0.0f,0.0f},
        C[6]={0.0f,0.0f,0.0f,0.0f,0.0f,0.0f};
  /* Note: the values d_v were calculated in relax_modes() */

  u[0]=d_v[index].v[0]; 
  u[1]=d_v[index].v[1]; 
  u[2]=d_v[index].v[2]; 

  #pragma unroll
  for(int ii=0;ii<LB_COMPONENTS;++ii)
  {  
       C[0] += (1.0f + para.gamma_bulk[ii])*u[0]*node_f.force[(0 + ii*3 ) * para.number_of_nodes + index] + 
                1.0f/3.0f*(para.gamma_bulk[ii]-para.gamma_shear[ii])*(
                                                                         u[0]*node_f.force[(0 + ii*3 ) * para.number_of_nodes + index]
                                                                       + u[1]*node_f.force[(1 + ii*3 ) * para.number_of_nodes + index]
                                                                       + u[2]*node_f.force[(2 + ii*3 ) * para.number_of_nodes + index]
                                                                     );

       C[2] += (1.0f + para.gamma_bulk[ii])*u[1]*node_f.force[(1 + ii*3 ) * para.number_of_nodes + index] + 
                1.0f/3.0f*(para.gamma_bulk[ii]-para.gamma_shear[ii])*(
                                                                         u[0]*node_f.force[(0 + ii*3 ) * para.number_of_nodes + index]
                                                                       + u[1]*node_f.force[(1 + ii*3 ) * para.number_of_nodes + index]
                                                                       + u[2]*node_f.force[(2 + ii*3 ) * para.number_of_nodes + index]
                                                                     );

       C[5] += (1.0f + para.gamma_bulk[ii])*u[2]*node_f.force[(2 + ii*3 ) * para.number_of_nodes + index] + 
                1.0f/3.0f*(para.gamma_bulk[ii]-para.gamma_shear[ii])*(
                                                                         u[0]*node_f.force[(0 + ii*3 ) * para.number_of_nodes + index] 
                                                                       + u[1]*node_f.force[(1 + ii*3 ) * para.number_of_nodes + index]
                                                                       + u[2]*node_f.force[(2 + ii*3 ) * para.number_of_nodes + index]
                                                                     );

       C[1] += 1.0f/2.0f*(1.0f+para.gamma_shear[ii])*(
                                                         u[0]*node_f.force[(1 + ii*3 ) * para.number_of_nodes + index]
                                                       + u[1]*node_f.force[(0 + ii*3 ) * para.number_of_nodes + index]
                                                     );

       C[3] += 1.0f/2.0f*(1.0f+para.gamma_shear[ii])*(
                                                         u[0]*node_f.force[(2 + ii*3 ) * para.number_of_nodes + index]
                                                       + u[2]*node_f.force[(0 + ii*3 ) * para.number_of_nodes + index]
                                                     );

       C[4] += 1.0f/2.0f*(1.0f+para.gamma_shear[ii])*(
                                                         u[1]*node_f.force[(2 + ii*3 ) * para.number_of_nodes + index]
                                                       + u[2]*node_f.force[(1 + ii*3 ) * para.number_of_nodes + index]
                                                     );
  }

  #pragma unroll
  for(int ii=0;ii<LB_COMPONENTS;++ii)
  {  
      /** update momentum modes */
#ifdef SHANCHEN
      float mobility_factor=1.0f/2.0f*(1.0f+para.gamma_mobility[0]);
#else
      float mobility_factor=1.0f;
#endif 
 /** update momentum modes */
      mode[1 + ii * LBQ] += mobility_factor * node_f.force[(0 + ii*3 ) * para.number_of_nodes + index];
      mode[2 + ii * LBQ] += mobility_factor * node_f.force[(1 + ii*3 ) * para.number_of_nodes + index];
      mode[3 + ii * LBQ] += mobility_factor * node_f.force[(2 + ii*3 ) * para.number_of_nodes + index];

      /** update stress modes */
      mode[4 + ii * LBQ] += C[0] + C[2] + C[5];
      mode[5 + ii * LBQ] += C[0] - C[2];
      mode[6 + ii * LBQ] += C[0] + C[2] - 2.0f*C[5];
      mode[7 + ii * LBQ] += C[1];
      mode[8 + ii * LBQ] += C[3];
      mode[9 + ii * LBQ] += C[4];
    
  }

//#if !defined(IMMERSED_BOUNDARY)
  // This must not be done here since we need the forces after LB update for the velocity interpolation
  // It is done by calling IBM_ResetLBForces_GPU from integrate_vv
  reset_LB_forces(index, node_f);
//#endif

#ifdef SHANCHEN
  for(int ii=0;ii<LB_COMPONENTS;++ii)
  {  
     node_f.force[(0 + ii*3 ) * para.number_of_nodes + index] +=node_f.scforce[(0+ii*3)*para.number_of_nodes + index];
     node_f.force[(1 + ii*3 ) * para.number_of_nodes + index] +=node_f.scforce[(1+ii*3)*para.number_of_nodes + index];
     node_f.force[(2 + ii*3 ) * para.number_of_nodes + index] +=node_f.scforce[(2+ii*3)*para.number_of_nodes + index];
  }
#endif
}

/**function used to calculate hydrodynamic fields in MD units.
 * @param n_a     Pointer to local node residing in array a for boundary flag(Input)
 * @param mode    Pointer to the local register values mode (Input)
 * @param d_p_v   Pointer to local print values (Output)
 * @param d_v     Pointer to local device values (Input)
 * @param node_f  Pointer to local node force (Input)
 * @param index   node index / thread index (Input)
 * @param print_index   node index / thread index (Output)
*/
__device__ void calc_values_in_MD_units(LB_nodes_gpu n_a, float *mode, LB_rho_v_pi_gpu *d_p_v, LB_rho_v_gpu *d_v, LB_node_force_gpu node_f, unsigned int index, unsigned int print_index) {
  
  float j[3]; 
  float modes_from_pi_eq[6]; 
  float pi[6]={0.0f,0.0f,0.0f,0.0f,0.0f,0.0f};

  if(n_a.boundary[index] == 0)
  {
    /* Ensure we are working with the current values of d_v */

    update_rho_v(mode, index, node_f, d_v);

    for(int ii= 0; ii < LB_COMPONENTS; ii++)
    {
      d_p_v[print_index].rho[ii] = d_v[index].rho[ii] / para.agrid / para.agrid / para.agrid;
    }
      
    d_p_v[print_index].v[0] = d_v[index].v[0] * para.agrid / para.tau;
    d_p_v[print_index].v[1] = d_v[index].v[1] * para.agrid / para.tau;
    d_p_v[print_index].v[2] = d_v[index].v[2] * para.agrid / para.tau;

    /* stress calculation */ 
    for(int ii = 0; ii < LB_COMPONENTS; ii++)
    {
      float Rho = d_v[index].rho[ii];
      
      /* note that d_v[index].v[] already includes the 1/2 f term, accounting for the pre- and post-collisional average */

      j[0] = Rho * d_v[index].v[0];
      j[1] = Rho * d_v[index].v[1];
      j[2] = Rho * d_v[index].v[2];

      // equilibrium part of the stress modes, which comes from 
      // the equality between modes and stress tensor components

      /* m4 = trace(pi) - rho
         m5 = pi_xx - pi_yy
         m6 = trace(pi) - 3 pi_zz
         m7 = pi_xy
         m8 = pi_xz
         m9 = pi_yz */

      // and pluggin in the Euler stress for the equilibrium:
      // pi_eq = rho_0*c_s^2*I3 + (j \otimes j)/rho
      // with I3 the 3D identity matrix and
      // rho = \trace(rho_0*c_s^2*I3), which yields

      /* m4_from_pi_eq = j.j
         m5_from_pi_eq = j_x*j_x - j_y*j_y
         m6_from_pi_eq = j.j - 3*j_z*j_z
         m7_from_pi_eq = j_x*j_y
         m8_from_pi_eq = j_x*j_z
         m9_from_pi_eq = j_y*j_z */

      // where the / Rho term has been dropped. We thus obtain: 

      modes_from_pi_eq[0] = ( j[0]*j[0] + j[1]*j[1] + j[2]*j[2] ) / Rho;
      modes_from_pi_eq[1] = ( j[0]*j[0] - j[1]*j[1] ) / Rho;
      modes_from_pi_eq[2] = ( j[0]*j[0] + j[1]*j[1] + j[2]*j[2] - 3.0f*j[2]*j[2] ) / Rho;
      modes_from_pi_eq[3] = j[0]*j[1] / Rho;
      modes_from_pi_eq[4] = j[0]*j[2] / Rho;
      modes_from_pi_eq[5] = j[1]*j[2] / Rho;
     
      /* Now we must predict the outcome of the next collision */
      /* We immediately average pre- and post-collision.  */
      /* TODO: need a reference for this.   */

      mode[4 + ii * LBQ ] = modes_from_pi_eq[0] + (0.5f + 0.5f*para.gamma_bulk[ii]) * (mode[4 + ii * LBQ] - modes_from_pi_eq[0]);
      mode[5 + ii * LBQ ] = modes_from_pi_eq[1] + (0.5f + 0.5f*para.gamma_shear[ii]) * (mode[5 + ii * LBQ] - modes_from_pi_eq[1]);
      mode[6 + ii * LBQ ] = modes_from_pi_eq[2] + (0.5f + 0.5f*para.gamma_shear[ii]) * (mode[6 + ii * LBQ] - modes_from_pi_eq[2]);
      mode[7 + ii * LBQ ] = modes_from_pi_eq[3] + (0.5f + 0.5f*para.gamma_shear[ii]) * (mode[7 + ii * LBQ] - modes_from_pi_eq[3]);
      mode[8 + ii * LBQ ] = modes_from_pi_eq[4] + (0.5f + 0.5f*para.gamma_shear[ii]) * (mode[8 + ii * LBQ] - modes_from_pi_eq[4]);
      mode[9 + ii * LBQ ] = modes_from_pi_eq[5] + (0.5f + 0.5f*para.gamma_shear[ii]) * (mode[9 + ii * LBQ] - modes_from_pi_eq[5]);

      // Transform the stress tensor components according to the modes that
      // correspond to those used by U. Schiller. In terms of populations this
      // expression then corresponds exactly to those in Eqs. 116 - 121 in the
      // Duenweg and Ladd paper, when these are written out in populations.
      // But to ensure this, the expression in Schiller's modes has to be different!

      pi[0] += (   2.0f*(mode[0 + ii * LBQ] + mode[4 + ii * LBQ])
                + mode[6 + ii * LBQ] + 3.0f*mode[5 + ii * LBQ] )/6.0f;  // xx
      pi[1] += mode[7 + ii * LBQ];                                      // xy
      pi[2] += (   2.0f*(mode[0 + ii * LBQ] + mode[4 + ii * LBQ])
                + mode[6 + ii * LBQ] - 3.0f*mode[5 + ii * LBQ] )/6.0f;  // yy
      pi[3] += mode[8 + ii * LBQ];                                      // xz
      pi[4] += mode[9 + ii * LBQ];                                      // yz
      pi[5] += (   mode[0 + ii * LBQ] + mode[4 + ii * LBQ]
                - mode[6 + ii * LBQ] )/3.0f;                            // zz

    }
     
    for(int i = 0; i < 6; i++)
    {
      d_p_v[print_index].pi[i] = pi[i] / para.tau
                                       / para.tau
                                       / para.agrid;
    }
  }
  else
  {
    for(int ii = 0; ii < LB_COMPONENTS; ii++)
      d_p_v[print_index].rho[ii] = 0.0f;
     
    for(int i = 0; i < 3; i++)
      d_p_v[print_index].v[i] = 0.0f;

    for(int i = 0; i < 6; i++)
      d_p_v[print_index].pi[i] = 0.0f;
  }
}

/**function used to calculate hydrodynamic fields in MD units.
 * @param mode_single   Pointer to the local register values mode (Input)
 * @param d_v_single    Pointer to local device values (Input)
 * @param rho_out       Pointer to density (Output)
 * @param j_out         Pointer to momentum (Output)
 * @param pi_out        Pointer to pressure tensor (Output)
*/
__device__ void calc_values_from_m_in_LB_units(float *mode_single, LB_rho_v_gpu *d_v_single, float* rho_out, float* j_out, float* pi_out) {

  float modes_from_pi_eq[6];
  float j[6];
  float Rho; 

  // stress calculation

  for(int ii = 0; ii < LB_COMPONENTS; ii++)
  {
    // Set the rho ouput value

    Rho = d_v_single->rho[ii];
    rho_out[ii] = Rho;
    
    // note that d_v_single->v[] already includes the 1/2 f term, 
    // accounting for the pre- and post-collisional average

    j[0] = Rho * d_v_single->v[0];
    j[1] = Rho * d_v_single->v[1];
    j[2] = Rho * d_v_single->v[2];

    j_out[3*ii + 0] = j[0];
    j_out[3*ii + 1] = j[1];
    j_out[3*ii + 2] = j[2];    

    // equilibrium part of the stress modes, which comes from 
    // the equality between modes and stress tensor components

    modes_from_pi_eq[0] = ( j[0]*j[0] + j[1]*j[1] + j[2]*j[2] ) / Rho;
    modes_from_pi_eq[1] = ( j[0]*j[0] - j[1]*j[1] ) / Rho;
    modes_from_pi_eq[2] = ( j[0]*j[0] + j[1]*j[1] + j[2]*j[2] - 3.0f*j[2]*j[2] ) / Rho;
    modes_from_pi_eq[3] = j[0]*j[1] / Rho;
    modes_from_pi_eq[4] = j[0]*j[2] / Rho;
    modes_from_pi_eq[5] = j[1]*j[2] / Rho;
   
    // Now we must predict the outcome of the next collision
    // We immediately average pre- and post-collision.

    mode_single[4 + ii * LBQ ] = modes_from_pi_eq[0] + (0.5f + 0.5f* para.gamma_bulk[ii]) * (mode_single[4 + ii * LBQ] - modes_from_pi_eq[0]);
    mode_single[5 + ii * LBQ ] = modes_from_pi_eq[1] + (0.5f + 0.5f*para.gamma_shear[ii]) * (mode_single[5 + ii * LBQ] - modes_from_pi_eq[1]);
    mode_single[6 + ii * LBQ ] = modes_from_pi_eq[2] + (0.5f + 0.5f*para.gamma_shear[ii]) * (mode_single[6 + ii * LBQ] - modes_from_pi_eq[2]);
    mode_single[7 + ii * LBQ ] = modes_from_pi_eq[3] + (0.5f + 0.5f*para.gamma_shear[ii]) * (mode_single[7 + ii * LBQ] - modes_from_pi_eq[3]);
    mode_single[8 + ii * LBQ ] = modes_from_pi_eq[4] + (0.5f + 0.5f*para.gamma_shear[ii]) * (mode_single[8 + ii * LBQ] - modes_from_pi_eq[4]);
    mode_single[9 + ii * LBQ ] = modes_from_pi_eq[5] + (0.5f + 0.5f*para.gamma_shear[ii]) * (mode_single[9 + ii * LBQ] - modes_from_pi_eq[5]);

    // Transform the stress tensor components according to the mode_singles.

    pi_out[6*ii + 0] = (   2.0f*(mode_single[0 + ii * LBQ] + mode_single[4 + ii * LBQ])
                         + mode_single[6 + ii * LBQ] + 3.0f*mode_single[5 + ii * LBQ] )/6.0f;   // xx
    pi_out[6*ii + 1] = mode_single[7 + ii * LBQ];                                               // xy
    pi_out[6*ii + 2] = (   2.0f*(mode_single[0 + ii * LBQ] + mode_single[4 + ii * LBQ])
                         + mode_single[6 + ii * LBQ] - 3.0f*mode_single[5 + ii * LBQ] )/6.0f;   // yy
    pi_out[6*ii + 3] = mode_single[8 + ii * LBQ];                                               // xz
    pi_out[6*ii + 4] = mode_single[9 + ii * LBQ];                                               // yz
    pi_out[6*ii + 5] = (   mode_single[0 + ii * LBQ] + mode_single[4 + ii * LBQ]
                         - mode_single[6 + ii * LBQ] )/3.0f;                                    // zz
  }
}

/**function used to calc physical values of every node
 * @param n_a     Pointer to local node residing in array a for boundary flag(Input)
 * @param mode    Pointer to the local register values mode (Input)
 * @param d_v     Pointer to local device values (Input/Output)
 * @param node_f  Pointer to local node force (Input)
 * @param index   node index / thread index (Input)
*/

/* FIXME this function is basically un-used, think about removing/replacing it */
__device__ void calc_values(LB_nodes_gpu n_a, float *mode, LB_rho_v_gpu *d_v, LB_node_force_gpu node_f, unsigned int index){ 

  float Rho_tot=0.0f;
  float u_tot[3]={0.0f,0.0f,0.0f};

  if(n_a.boundary[index] != 1){
      #pragma unroll
      for(int ii=0;ii<LB_COMPONENTS;++ii) { 
          /** re-construct the real density
          * remember that the populations are stored as differences to their
          * equilibrium value */
          d_v[index].rho[ii]= mode[0 + ii * 4]+ para.rho[ii]*para.agrid*para.agrid*para.agrid;
          Rho_tot  += mode[0 + ii * 4]+ para.rho[ii]*para.agrid*para.agrid*para.agrid;
          u_tot[0] += mode[1 + ii * 4];
          u_tot[1] += mode[2 + ii * 4];
          u_tot[2] += mode[3 + ii * 4];
    
          /** if forces are present, the momentum density is redefined to
          * inlcude one half-step of the force action.  See the
          * Chapman-Enskog expansion in [Ladd & Verberg]. */
    
          u_tot[0] += 0.5f*node_f.force[(0+ii*3)*para.number_of_nodes + index];
          u_tot[1] += 0.5f*node_f.force[(1+ii*3)*para.number_of_nodes + index];
          u_tot[2] += 0.5f*node_f.force[(2+ii*3)*para.number_of_nodes + index];
      }
      u_tot[0]/=Rho_tot;
      u_tot[1]/=Rho_tot;
      u_tot[2]/=Rho_tot;
    
      d_v[index].v[0]=u_tot[0]; 
      d_v[index].v[1]=u_tot[1]; 
      d_v[index].v[2]=u_tot[2]; 
  } else { 
    #pragma unroll
    for(int ii=0;ii<LB_COMPONENTS;++ii) { 
       d_v[index].rho[ii]   = 1.;
    }
    d_v[index].v[0] = 0.0f;
    d_v[index].v[1] = 0.0f; 
    d_v[index].v[2] = 0.0f; 
  }   
}


/** 
 * @param node_index  node index around (8) particle (Input)
 * @param *mode       Pointer to the local register values mode (Output)
 * @param n_a         Pointer to local node residing in array a(Input)
 * @param component_index   Shanchen component index        (Input)
*/
__device__ void calc_mode(float *mode, LB_nodes_gpu n_a, unsigned int node_index, int component_index){

  /** mass mode */
  mode[0] =   n_a.vd[( 0 + component_index*LBQ ) * para.number_of_nodes + node_index]
            + n_a.vd[( 1 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[( 2 + component_index*LBQ ) * para.number_of_nodes + node_index] 
            + n_a.vd[( 3 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[( 4 + component_index*LBQ ) * para.number_of_nodes + node_index]
            + n_a.vd[( 5 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[( 6 + component_index*LBQ ) * para.number_of_nodes + node_index]
            + n_a.vd[( 7 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[( 8 + component_index*LBQ ) * para.number_of_nodes + node_index]
            + n_a.vd[( 9 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(10 + component_index*LBQ ) * para.number_of_nodes + node_index]
            + n_a.vd[(11 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(12 + component_index*LBQ ) * para.number_of_nodes + node_index]
            + n_a.vd[(13 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(14 + component_index*LBQ ) * para.number_of_nodes + node_index]
            + n_a.vd[(15 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(16 + component_index*LBQ ) * para.number_of_nodes + node_index]
            + n_a.vd[(17 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(18 + component_index*LBQ ) * para.number_of_nodes + node_index];

  /** momentum modes */
  mode[1] =   (n_a.vd[( 1 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[( 2 + component_index*LBQ ) * para.number_of_nodes + node_index])
            + (n_a.vd[( 7 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[( 8 + component_index*LBQ ) * para.number_of_nodes + node_index])
            + (n_a.vd[( 9 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(10 + component_index*LBQ ) * para.number_of_nodes + node_index])
            + (n_a.vd[(11 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(12 + component_index*LBQ ) * para.number_of_nodes + node_index])
            + (n_a.vd[(13 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(14 + component_index*LBQ ) * para.number_of_nodes + node_index]);

  mode[2] =   (n_a.vd[( 3 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[( 4 + component_index*LBQ ) * para.number_of_nodes + node_index])
            + (n_a.vd[( 7 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[( 8 + component_index*LBQ ) * para.number_of_nodes + node_index])
            - (n_a.vd[( 9 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(10 + component_index*LBQ ) * para.number_of_nodes + node_index])
            + (n_a.vd[(15 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(16 + component_index*LBQ ) * para.number_of_nodes + node_index])
            + (n_a.vd[(17 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(18 + component_index*LBQ ) * para.number_of_nodes + node_index]);

  mode[3] =   (n_a.vd[( 5 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[( 6 + component_index*LBQ ) * para.number_of_nodes + node_index])
            + (n_a.vd[(11 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(12 + component_index*LBQ ) * para.number_of_nodes + node_index])
            - (n_a.vd[(13 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(14 + component_index*LBQ ) * para.number_of_nodes + node_index])
            + (n_a.vd[(15 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(16 + component_index*LBQ ) * para.number_of_nodes + node_index])
            - (n_a.vd[(17 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(18 + component_index*LBQ ) * para.number_of_nodes + node_index]);
}

/*********************************************************/
/** \name interpolation_three_point_coupling */
/*********************************************************/
/**(Eq. (12) Ahlrichs and Duenweg, JCP 111(17):8225 (1999))
 * @param n_a             Pointer to local node residing in array a (Input)
 * @param *delta          Pointer for the weighting of particle position (Output)
 * @param *particle_data  Pointer to the particle position and velocity (Input)
 * @param *particle_force Pointer to the particle force (Input)
 * @param part_index      particle id / thread id (Input)
 * @param node_index      node index around (8) particle (Output)
 * @param *d_v            Pointer to local device values
 * @param *interpolated_u Pointer to the interpolated velocity (Output)
*/
__device__ __inline__ void interpolation_three_point_coupling(LB_nodes_gpu n_a, float* particle_position, unsigned int *node_index, LB_rho_v_gpu *d_v, float *delta, float *interpolated_u) {

  int my_center[3];
  float temp_delta[27];
  float mode[19*LB_COMPONENTS];

  /** see Duenweg and Ladd http://arxiv.org/abs/0803.2826 eqn. 301 */
  /** the i index is left node, nearest node, right node */
  for(int i=0; i<3; ++i){
    /** note the -0.5f is to account for the shift of the LB grid relative to the MD */
    float scaledpos = particle_position[i]/para.agrid-0.5f;
    /** the +0.5 is to turn the floorf into a round function */
    my_center[i] = (int)(floorf(scaledpos+0.5f));
    scaledpos = scaledpos-1.0f*my_center[i];
    temp_delta[0+3*i] = ( 5.0f - 3.0f*abs(scaledpos+1.0f) - sqrtf( -2.0f + 6.0f*abs(scaledpos+1.0f) - 3.0f*powf(scaledpos+1.0f,2) ) )/6.0f;
    temp_delta[1+3*i] = ( 1.0f + sqrtf( 1.0f - 3.0f*powf(scaledpos,2) ) )/3.0f;
    temp_delta[2+3*i] = ( 5.0f - 3.0f*abs(scaledpos-1.0f) - sqrtf( -2.0f + 6.0f*abs(scaledpos-1.0f) - 3.0f*powf(scaledpos-1.0f,2) ) )/6.0f;
  }

  for (int i=-1; i<=1; i++) {
    for (int j=-1; j<=1; j++) {
      for (int k=-1; k<=1; k++) {
        delta[i+3*j+9*k+13] = temp_delta[i+1] * temp_delta[3+j+1] * temp_delta[6+k+1];
      }
    }
  }

  // modulo for negative numbers is strange at best, shift to make sure we are positive
  int x = my_center[0] + para.dim_x;
  int y = my_center[1] + para.dim_y;
  int z = my_center[2] + para.dim_z;
  /** Here we collect the nodes for the three point coupling scheme (27 nodes in 3d) with the analogous numbering scheme of the two point coupling scheme */
  for (int i=-1; i<=1; i++) {
    for (int j=-1; j<=1; j++) {
      for (int k=-1; k<=1; k++) {
        node_index[i+3*j+9*k+13] = (x+i+para.dim_x)%para.dim_x + para.dim_x*((y+j+para.dim_y)%para.dim_y) + para.dim_x*para.dim_y*((z+k+para.dim_z)%para.dim_z);
      }
    }
  }

  interpolated_u[0] = 0.0f;
  interpolated_u[1] = 0.0f;
  interpolated_u[2] = 0.0f;
#pragma unroll
  for(int i=0; i<27; ++i){
    float totmass=0.0f;
    calc_m_from_n(n_a,node_index[i],mode);
#pragma unroll
    for(int ii=0;ii<LB_COMPONENTS;ii++){
      totmass+=mode[0]+para.rho[ii]*para.agrid*para.agrid*para.agrid;
    } 
    /* The boolean expression (n_a.boundary[node_index[i]] == 0) causes boundary nodes
       to couple with velocity 0 to particles. This is necessary, since boundary nodes
       undergo the same LB dynamics as fluid nodes do. The flow within the boundaries
       does not interact with the physical fluid, since these populations are overwritten
       by the bounce back kernel. Particles close to walls can couple to this unphysical
       flow, though.
    */
    interpolated_u[0] += (mode[1]/totmass)*delta[i] * (n_a.boundary[node_index[i]] == 0);
    interpolated_u[1] += (mode[2]/totmass)*delta[i] * (n_a.boundary[node_index[i]] == 0);
    interpolated_u[2] += (mode[3]/totmass)*delta[i] * (n_a.boundary[node_index[i]] == 0);
  }
}


/*********************************************************/
/** \name calc_viscous_force_three_point_couple */
/*********************************************************/
/**(Eq. (12) Ahlrichs and Duenweg, JCP 111(17):8225 (1999))
 * @param n_a                Pointer to local node residing in array a (Input)
 * @param *delta             Pointer for the weighting of particle position (Output)
 * @param *delta_j           Pointer for the weighting of particle momentum (Output)
 * @param *particle_position Pointer to the particle position (Input)
 * @param *rn_part           Pointer to randomnumber array of the particle
 * @param node_index         node index around (8) particle (Output)
 * @param *d_v               Pointer to local device values
 * @param flag_cs            Determine if we are at the centre (0, typical) or at the source (1, swimmer only)
*/
__device__ void calc_viscous_force_three_point_couple(LB_nodes_gpu n_a, float *delta, CUDA_particle_data *particle_data, float *particle_force, unsigned int part_index, LB_randomnr_gpu *rn_part, float *delta_j, unsigned int *node_index, LB_rho_v_gpu *d_v, int flag_cs){

  float interpolated_u[3];
  float interpolated_rho[LB_COMPONENTS];
  float viscforce[3*LB_COMPONENTS];

  // Zero out workspace
#pragma unroll
  for(int ii=0; ii<LB_COMPONENTS; ++ii){ 
#pragma unroll
    for(int jj=0; jj<3; ++jj){ 
      viscforce[jj+ii*3]=0.0f;
      delta_j[jj+ii*3]  =0.0f;
    }
  }
  // Zero out only if we are at the centre of the particle <=> flag_cs = 0
  particle_force[3*part_index+0] = flag_cs * particle_force[3*part_index+0];
  particle_force[3*part_index+1] = flag_cs * particle_force[3*part_index+1];
  particle_force[3*part_index+2] = flag_cs * particle_force[3*part_index+2];

  float position[3];
  position[0] = particle_data[part_index].p[0];
  position[1] = particle_data[part_index].p[1];
  position[2] = particle_data[part_index].p[2];

  float velocity[3];
  velocity[0] = particle_data[part_index].v[0];
  velocity[1] = particle_data[part_index].v[1];
  velocity[2] = particle_data[part_index].v[2];

#ifdef ENGINE
  // First calculate interpolated velocity for dipole source,
  // such that we don't overwrite mode, d_v, etc. for the rest of the function
  float direction = float(particle_data[part_index].swim.push_pull) * particle_data[part_index].swim.dipole_length;
  // Extrapolate position by dipole length if we are at the centre of the particle
  position[0] += flag_cs * direction * particle_data[part_index].swim.quatu[0];
  position[1] += flag_cs * direction * particle_data[part_index].swim.quatu[1];
  position[2] += flag_cs * direction * particle_data[part_index].swim.quatu[2];
#endif

  // Do the velocity interpolation
  interpolation_three_point_coupling(n_a, position, node_index, d_v, delta, interpolated_u);

#ifdef ENGINE
  velocity[0] -= (particle_data[part_index].swim.v_swim*para.time_step)*particle_data[part_index].swim.quatu[0];
  velocity[1] -= (particle_data[part_index].swim.v_swim*para.time_step)*particle_data[part_index].swim.quatu[1];
  velocity[2] -= (particle_data[part_index].swim.v_swim*para.time_step)*particle_data[part_index].swim.quatu[2];

  // The first three components are v_center, the last three v_source
  // Do not use within LB, because these have already been converted back to MD units
  particle_data[part_index].swim.v_cs[0+3*flag_cs] = interpolated_u[0] * para.agrid / para.tau;
  particle_data[part_index].swim.v_cs[1+3*flag_cs] = interpolated_u[1] * para.agrid / para.tau;
  particle_data[part_index].swim.v_cs[2+3*flag_cs] = interpolated_u[2] * para.agrid / para.tau;
#endif

  /* for LB we do not reweight the friction force */
  for(int ii=0; ii<LB_COMPONENTS; ++ii){ 
    interpolated_rho[ii]=1.0;
  }

  /** calculate viscous force
   * take care to rescale velocities with time_step and transform to MD units
   * (Eq. (9) Ahlrichs and Duenweg, JCP 111(17):8225 (1999)) */
  float rhotot=0;

#pragma unroll
  for(int ii=0; ii<LB_COMPONENTS; ++ii){ 
    rhotot+=interpolated_rho[ii];
  }


  /* Viscous force */
  for(int ii=0; ii<LB_COMPONENTS; ++ii){ 
    viscforce[0+ii*3] -= interpolated_rho[ii]*para.friction[ii]*(velocity[0]/para.time_step - interpolated_u[0]*para.agrid/para.tau)/rhotot;
    viscforce[1+ii*3] -= interpolated_rho[ii]*para.friction[ii]*(velocity[1]/para.time_step - interpolated_u[1]*para.agrid/para.tau)/rhotot;
    viscforce[2+ii*3] -= interpolated_rho[ii]*para.friction[ii]*(velocity[2]/para.time_step - interpolated_u[2]*para.agrid/para.tau)/rhotot;

#ifdef LB_ELECTROHYDRODYNAMICS
    viscforce[0+ii*3] += interpolated_rho[ii]*para.friction[ii] * particle_data[part_index].mu_E[0]/rhotot;
    viscforce[1+ii*3] += interpolated_rho[ii]*para.friction[ii] * particle_data[part_index].mu_E[1]/rhotot;
    viscforce[2+ii*3] += interpolated_rho[ii]*para.friction[ii] * particle_data[part_index].mu_E[2]/rhotot;
#endif

    /** add stochastic force of zero mean (Ahlrichs, Duenweg equ. 15)*/
#ifdef FLATNOISE
    random_01(rn_part);
    viscforce[0+ii*3] += para.lb_coupl_pref[ii]*(rn_part->randomnr[0]-0.5f);
    viscforce[1+ii*3] += para.lb_coupl_pref[ii]*(rn_part->randomnr[1]-0.5f);
    random_01(rn_part);
    viscforce[2+ii*3] += para.lb_coupl_pref[ii]*(rn_part->randomnr[0]-0.5f);
#elif defined(GAUSSRANDOMCUT)
    gaussian_random_cut(rn_part);
    viscforce[0+ii*3] += para.lb_coupl_pref2[ii]*rn_part->randomnr[0];
    viscforce[1+ii*3] += para.lb_coupl_pref2[ii]*rn_part->randomnr[1];
    gaussian_random_cut(rn_part);
    viscforce[2+ii*3] += para.lb_coupl_pref2[ii]*rn_part->randomnr[0];
#elif defined(GAUSSRANDOM)
    gaussian_random(rn_part);
    viscforce[0+ii*3] += para.lb_coupl_pref2[ii]*rn_part->randomnr[0];
    viscforce[1+ii*3] += para.lb_coupl_pref2[ii]*rn_part->randomnr[1];
    gaussian_random(rn_part);
    viscforce[2+ii*3] += para.lb_coupl_pref2[ii]*rn_part->randomnr[0];
#else
#error No noise type defined for the GPU LB
#endif    
    /** delta_j for transform momentum transfer to lattice units which is done in calc_node_force
      (Eq. (12) Ahlrichs and Duenweg, JCP 111(17):8225 (1999)) */
    // only add to particle_force for particle centre <=> (1-flag_cs) = 1
    particle_force[3*part_index+0] += (1-flag_cs) * viscforce[0+ii*3];
    particle_force[3*part_index+1] += (1-flag_cs) * viscforce[1+ii*3];
    particle_force[3*part_index+2] += (1-flag_cs) * viscforce[2+ii*3];

    // only add to particle_force for particle centre <=> (1-flag_cs) = 1
    delta_j[0+3*ii] -= (1-flag_cs)*viscforce[0+ii*3]*para.time_step*para.tau/para.agrid;
    delta_j[1+3*ii] -= (1-flag_cs)*viscforce[1+ii*3]*para.time_step*para.tau/para.agrid;
    delta_j[2+3*ii] -= (1-flag_cs)*viscforce[2+ii*3]*para.time_step*para.tau/para.agrid;
#ifdef ENGINE
    // add swimming force to source position
    delta_j[0+3*ii] -= flag_cs*particle_data[part_index].swim.f_swim*particle_data[part_index].swim.quatu[0]*para.time_step*para.tau/para.agrid;
    delta_j[1+3*ii] -= flag_cs*particle_data[part_index].swim.f_swim*particle_data[part_index].swim.quatu[1]*para.time_step*para.tau/para.agrid;
    delta_j[2+3*ii] -= flag_cs*particle_data[part_index].swim.f_swim*particle_data[part_index].swim.quatu[2]*para.time_step*para.tau/para.agrid;
#endif
  }
}

/**calcutlation of the node force caused by the particles, with atomicadd due to avoiding race conditions 
  (Eq. (14) Ahlrichs and Duenweg, JCP 111(17):8225 (1999))
 * @param *delta    Pointer for the weighting of particle position (Input)
 * @param *delta_j    Pointer for the weighting of particle momentum (Input)
 * @param node_index    node index around (8) particle (Input)
 * @param node_f        Pointer to the node force (Output).
*/
__device__ void calc_node_force_three_point_couple(float *delta, float *delta_j, unsigned int *node_index, LB_node_force_gpu node_f){
/* TODO: should the drag depend on the density?? */

  for (int i=-1; i<=1; i++) {
    for (int j=-1; j<=1; j++) {
      for (int k=-1; k<=1; k++) {
        atomicadd(&(node_f.force[0*para.number_of_nodes + node_index[i+3*j+9*k+13]]), (delta[i+3*j+9*k+13]*delta_j[0]));
        atomicadd(&(node_f.force[1*para.number_of_nodes + node_index[i+3*j+9*k+13]]), (delta[i+3*j+9*k+13]*delta_j[1]));
        atomicadd(&(node_f.force[2*para.number_of_nodes + node_index[i+3*j+9*k+13]]), (delta[i+3*j+9*k+13]*delta_j[2]));
      }
    }
  }
}


/**calculate temperature of the fluid kernel
 * @param *cpu_jsquared   Pointer to result storage value (Output)
 * @param n_a             Pointer to local node residing in array a (Input)
*/
__global__ void temperature(LB_nodes_gpu n_a, float *cpu_jsquared, int *number_of_non_boundary_nodes ) {
  float mode[4];
  float jsquared = 0.0f;
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes)
  {
    if(!n_a.boundary[index])
    {
      for(int ii=0;ii<LB_COMPONENTS;++ii)
      {  
         calc_mode(mode, n_a, index,ii);
         jsquared = mode[1]*mode[1]+mode[2]*mode[2]+mode[3]*mode[3];
         atomicadd(cpu_jsquared, jsquared);
         atomicAdd(number_of_non_boundary_nodes, 1);
      }
    }
  }
}

/*********************************************************/
/** \name interpolation_two_point_coupling */
/*********************************************************/
/**(Eq. (12) Ahlrichs and Duenweg, JCP 111(17):8225 (1999))
 * @param n_a                   Pointer to local node residing in array a (Input)
 * @param *particle_position    Pointer to the particle position (Input)
 * @param node_index            node index around (8) particle (Output)
 * @param *mode                 Pointer to the 19 modes for current lattice point (Output)
 * @param *d_v                  Pointer to local device values
 * @param *delta                Pointer for the weighting of particle position (Output)
 * @param *interpolated_u       Pointer to the interpolated velocity (Output)
*/
__device__ __inline__ void interpolation_two_point_coupling(LB_nodes_gpu n_a, float *particle_position, unsigned int* node_index, float* mode, LB_rho_v_gpu *d_v, float* delta, float *interpolated_u) {
  int   left_node_index[3];
  float temp_delta[6];
  float temp_delta_half[6];

  // see ahlrichs + duenweg page 8227 equ (10) and (11)
#pragma unroll
  for(int i=0; i<3; ++i)
  {
    float scaledpos = particle_position[i]/para.agrid - 0.5f;
    left_node_index[i] = (int)(floorf(scaledpos));
    temp_delta[3+i] = scaledpos - left_node_index[i];
    temp_delta[i] = 1.0f - temp_delta[3+i];
    // further value used for interpolation of fluid velocity at part pos near boundaries
    temp_delta_half[3+i] = (scaledpos - left_node_index[i])*2.0f;
    temp_delta_half[i] = 2.0f - temp_delta_half[3+i];
  }

  delta[0] = temp_delta[0] * temp_delta[1] * temp_delta[2];
  delta[1] = temp_delta[3] * temp_delta[1] * temp_delta[2];
  delta[2] = temp_delta[0] * temp_delta[4] * temp_delta[2];
  delta[3] = temp_delta[3] * temp_delta[4] * temp_delta[2];
  delta[4] = temp_delta[0] * temp_delta[1] * temp_delta[5];
  delta[5] = temp_delta[3] * temp_delta[1] * temp_delta[5];
  delta[6] = temp_delta[0] * temp_delta[4] * temp_delta[5];
  delta[7] = temp_delta[3] * temp_delta[4] * temp_delta[5];

  // modulo for negative numbers is strange at best, shift to make sure we are positive
  int x = left_node_index[0] + para.dim_x;
  int y = left_node_index[1] + para.dim_y;
  int z = left_node_index[2] + para.dim_z;

  node_index[0] = x%para.dim_x     + para.dim_x*(y%para.dim_y)     + para.dim_x*para.dim_y*(z%para.dim_z);
  node_index[1] = (x+1)%para.dim_x + para.dim_x*(y%para.dim_y)     + para.dim_x*para.dim_y*(z%para.dim_z);
  node_index[2] = x%para.dim_x     + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*(z%para.dim_z);
  node_index[3] = (x+1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*(z%para.dim_z);
  node_index[4] = x%para.dim_x     + para.dim_x*(y%para.dim_y)     + para.dim_x*para.dim_y*((z+1)%para.dim_z);
  node_index[5] = (x+1)%para.dim_x + para.dim_x*(y%para.dim_y)     + para.dim_x*para.dim_y*((z+1)%para.dim_z);
  node_index[6] = x%para.dim_x     + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z);
  node_index[7] = (x+1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z);


  interpolated_u[0] = 0.0f;
  interpolated_u[1] = 0.0f;
  interpolated_u[2] = 0.0f;
#pragma unroll
  for(int i=0; i<8; ++i)
  {
    float totmass=0.0f;

    calc_m_from_n(n_a,node_index[i],mode);

#pragma unroll
    for(int ii=0;ii<LB_COMPONENTS;ii++)
    {
      totmass+=mode[0]+para.rho[ii]*para.agrid*para.agrid*para.agrid;
    } 

#ifdef SHANCHEN
    interpolated_u[0] += d_v[node_index[i]].v[0]/8.0f * (n_a.boundary[node_index[i]] == 0);  
    interpolated_u[1] += d_v[node_index[i]].v[1]/8.0f * (n_a.boundary[node_index[i]] == 0);
    interpolated_u[2] += d_v[node_index[i]].v[2]/8.0f * (n_a.boundary[node_index[i]] == 0);
#else
    /* The boolean expression (n_a.boundary[node_index[i]] == 0) causes boundary nodes
       to couple with velocity 0 to particles. This is necessary, since boundary nodes
       undergo the same LB dynamics as fluid nodes do. The flow within the boundaries
       does not interact with the physical fluid, since these populations are overwritten
       by the bounce back kernel. Particles close to walls can couple to this unphysical
       flow, though.
    */
    interpolated_u[0] += (mode[1]/totmass)*delta[i] * (n_a.boundary[node_index[i]] == 0);
    interpolated_u[1] += (mode[2]/totmass)*delta[i] * (n_a.boundary[node_index[i]] == 0);
    interpolated_u[2] += (mode[3]/totmass)*delta[i] * (n_a.boundary[node_index[i]] == 0);
#endif
  }
}

/*********************************************************/
/** \name calc_viscous_force */
/*********************************************************/
/**(Eq. (12) Ahlrichs and Duenweg, JCP 111(17):8225 (1999))
 * @param n_a                   Pointer to local node residing in array a (Input)
 * @param partgrad1             particle gradient for the Shan-Chen
 * @param partgrad2             particle gradient for the Shan-Chen
 * @param partgrad3             particle gradient for the Shan-Chen
 * @param *delta                Pointer for the weighting of particle position (Output)
 * @param *delta_j              Pointer for the weighting of particle momentum (Output)
 * @param *particle_data        Pointer to the particle position and velocity (Input)
 * @param *particle_force       Pointer to the particle force (Input)
 * @param *fluid_composition    Pointer to the fluid composition (Input)
 * @param part_index            particle id / thread id (Input)
 * @param *rn_part              Pointer to randomnumber array of the particle
 * @param node_index            node index around (8) particle (Output)
 * @param *d_v                  Pointer to local device values
 * @param flag_cs               Determine if we are at the centre (0, typical) or at the source (1, swimmer only)
*/
__device__ void calc_viscous_force(LB_nodes_gpu n_a, float *delta, float * partgrad1, float * partgrad2, float * partgrad3, CUDA_particle_data *particle_data, float *particle_force, CUDA_fluid_composition * fluid_composition, unsigned int part_index, LB_randomnr_gpu *rn_part, float *delta_j, unsigned int *node_index, LB_rho_v_gpu *d_v, int flag_cs){

  float interpolated_u[3];
  float interpolated_rho[LB_COMPONENTS];
  float viscforce[3*LB_COMPONENTS];
  float scforce[3*LB_COMPONENTS];
  float mode[19*LB_COMPONENTS];
#ifdef SHANCHEN
  float gradrho1, gradrho2, gradrho3;
  float Rho;
#endif
  // Zero out workspace
  #pragma unroll
  for(int ii=0; ii<LB_COMPONENTS; ++ii)
  { 
    #pragma unroll
    for(int jj=0; jj<3; ++jj)
    { 
      scforce[jj+ii*3]  =0.0f;
      viscforce[jj+ii*3]=0.0f;
      delta_j[jj+ii*3]  =0.0f;
    }
    
    #pragma unroll
    for(int jj=0; jj<8; ++jj)
    { 
      partgrad1[jj+ii*8]=0.0f;
      partgrad2[jj+ii*8]=0.0f;
      partgrad3[jj+ii*8]=0.0f;
    }
  }
  // Zero out only if we are at the centre of the particle <=> flag_cs = 0
  particle_force[3*part_index+0] = flag_cs * particle_force[3*part_index+0];
  particle_force[3*part_index+1] = flag_cs * particle_force[3*part_index+1];
  particle_force[3*part_index+2] = flag_cs * particle_force[3*part_index+2];

  float position[3];
  position[0] = particle_data[part_index].p[0];
  position[1] = particle_data[part_index].p[1];
  position[2] = particle_data[part_index].p[2];

  float velocity[3];
  velocity[0] = particle_data[part_index].v[0];
  velocity[1] = particle_data[part_index].v[1];
  velocity[2] = particle_data[part_index].v[2];

#ifdef ENGINE
  // First calculate interpolated velocity for dipole source,
  // such that we don't overwrite mode, d_v, etc. for the rest of the function
  float direction = float(particle_data[part_index].swim.push_pull) * particle_data[part_index].swim.dipole_length;
  // Extrapolate position by dipole length if we are at the centre of the particle
  position[0] += flag_cs * direction * particle_data[part_index].swim.quatu[0];
  position[1] += flag_cs * direction * particle_data[part_index].swim.quatu[1];
  position[2] += flag_cs * direction * particle_data[part_index].swim.quatu[2];
#endif

  // Do the velocity interpolation
  interpolation_two_point_coupling(n_a, position, node_index, mode, d_v, delta, interpolated_u);

#ifdef ENGINE
  velocity[0] -= (particle_data[part_index].swim.v_swim*para.time_step)*particle_data[part_index].swim.quatu[0];
  velocity[1] -= (particle_data[part_index].swim.v_swim*para.time_step)*particle_data[part_index].swim.quatu[1];
  velocity[2] -= (particle_data[part_index].swim.v_swim*para.time_step)*particle_data[part_index].swim.quatu[2];

  // The first three components are v_center, the last three v_source
  // Do not use within LB, because these have already been converted back to MD units
  particle_data[part_index].swim.v_cs[0+3*flag_cs] = interpolated_u[0] * para.agrid / para.tau;
  particle_data[part_index].swim.v_cs[1+3*flag_cs] = interpolated_u[1] * para.agrid / para.tau;
  particle_data[part_index].swim.v_cs[2+3*flag_cs] = interpolated_u[2] * para.agrid / para.tau;
#endif

#ifdef SHANCHEN

 #pragma unroll
  for(int ii=0; ii<LB_COMPONENTS; ++ii)
  { 
    float solvation2 = particle_data[part_index].solvation[2*ii + 1];
   
    interpolated_rho[ii]  = 0.0f;
    gradrho1 = gradrho2 = gradrho3 = 0.0f;
  
    // TODO: should one introduce a density-dependent friction ?
    calc_mode(mode, n_a, node_index[0],ii);
    Rho = mode[0] + para.rho[ii]*para.agrid*para.agrid*para.agrid;
    interpolated_rho[ii] += delta[0] * Rho; 
    partgrad1[ii*8 + 0] += Rho * solvation2;
    partgrad2[ii*8 + 0] += Rho * solvation2;
    partgrad3[ii*8 + 0] += Rho * solvation2;
    gradrho1 -=(delta[0] + delta[1]) * Rho; 
    gradrho2 -=(delta[0] + delta[2]) * Rho; 
    gradrho3 -=(delta[0] + delta[4]) * Rho; 

    calc_mode(mode, n_a, node_index[1],ii); 
    Rho = mode[0] +  para.rho[ii]*para.agrid*para.agrid*para.agrid; 
    interpolated_rho[ii] += delta[1] * Rho; 
    partgrad1[ii*8 + 1] -= Rho * solvation2;
    partgrad2[ii*8 + 1] += Rho * solvation2;
    partgrad3[ii*8 + 1] += Rho * solvation2;
    gradrho1 +=(delta[1] + delta[0]) * Rho; 
    gradrho2 -=(delta[1] + delta[3]) * Rho; 
    gradrho3 -=(delta[1] + delta[5]) * Rho; 

    calc_mode(mode, n_a, node_index[2],ii);
    Rho = mode[0] + para.rho[ii]*para.agrid*para.agrid*para.agrid;
    interpolated_rho[ii] += delta[2] * Rho; 
    partgrad1[ii*8 + 2] += Rho * solvation2;
    partgrad2[ii*8 + 2] -= Rho * solvation2;
    partgrad3[ii*8 + 2] += Rho * solvation2;
    gradrho1 -=(delta[2] + delta[3]) * Rho; 
    gradrho2 +=(delta[2] + delta[0]) * Rho; 
    gradrho3 -=(delta[2] + delta[6]) * Rho; 

    calc_mode(mode, n_a, node_index[3],ii);
    Rho = mode[0] + para.rho[ii]*para.agrid*para.agrid*para.agrid;
    interpolated_rho[ii] += delta[3] * Rho; 
    partgrad1[ii*8 + 3] -= Rho * solvation2;
    partgrad2[ii*8 + 3] -= Rho * solvation2;
    partgrad3[ii*8 + 3] += Rho * solvation2;
    gradrho1 +=(delta[3] + delta[2]) * Rho; 
    gradrho2 +=(delta[3] + delta[1]) * Rho; 
    gradrho3 -=(delta[3] + delta[7]) * Rho; 

    calc_mode(mode, n_a, node_index[4],ii);
    Rho = mode[0] + para.rho[ii]*para.agrid*para.agrid*para.agrid;
    interpolated_rho[ii] += delta[4] * Rho; 
    partgrad1[ii*8 + 4] += Rho * solvation2;
    partgrad2[ii*8 + 4] += Rho * solvation2;
    partgrad3[ii*8 + 4] -= Rho * solvation2;
    gradrho1 -=(delta[4] + delta[5]) * Rho; 
    gradrho2 -=(delta[4] + delta[6]) * Rho; 
    gradrho3 +=(delta[4] + delta[0]) * Rho; 

    calc_mode(mode, n_a, node_index[5],ii);
    Rho = mode[0] + para.rho[ii]*para.agrid*para.agrid*para.agrid;
    interpolated_rho[ii] += delta[5] * Rho; 
    partgrad1[ii*8 + 5] -= Rho * solvation2;
    partgrad2[ii*8 + 5] += Rho * solvation2;
    partgrad3[ii*8 + 5] -= Rho * solvation2;
    gradrho1 +=(delta[5] + delta[4]) * Rho; 
    gradrho2 -=(delta[5] + delta[7]) * Rho; 
    gradrho3 +=(delta[5] + delta[1]) * Rho; 

    calc_mode(mode, n_a, node_index[6],ii);
    Rho = mode[0] + para.rho[ii]*para.agrid*para.agrid*para.agrid;
    interpolated_rho[ii] += delta[6] * Rho; 
    partgrad1[ii*8 + 6] += Rho * solvation2;
    partgrad2[ii*8 + 6] -= Rho * solvation2;
    partgrad3[ii*8 + 6] -= Rho * solvation2;
    gradrho1 -=(delta[6] + delta[7]) * Rho; 
    gradrho2 +=(delta[6] + delta[4]) * Rho; 
    gradrho3 +=(delta[6] + delta[2]) * Rho; 

    calc_mode(mode, n_a, node_index[7],ii);
    Rho = mode[0] + para.rho[ii]*para.agrid*para.agrid*para.agrid;
    interpolated_rho[ii] += delta[7] * Rho; 
    partgrad1[ii*8 + 7] -= Rho * solvation2;
    partgrad2[ii*8 + 7] -= Rho * solvation2;
    partgrad3[ii*8 + 7] -= Rho * solvation2;
    gradrho1 +=(delta[7] + delta[6]) * Rho; 
    gradrho2 +=(delta[7] + delta[5]) * Rho; 
    gradrho3 +=(delta[7] + delta[3]) * Rho; 

    /* normalize the gradient to md units TODO: is that correct?*/
    gradrho1 *= para.agrid; 
    gradrho2 *= para.agrid; 
    gradrho3 *= para.agrid; 

    // scforce is 0 at the interpolated point where the swimming force gets put back on the fluid
    // only add sc_force up for particle centre <=> (1-flag_cs) = 1
    scforce[0+ii*3] += (1-flag_cs) * particle_data[part_index].solvation[2*ii] * gradrho1 ; 
    scforce[1+ii*3] += (1-flag_cs) * particle_data[part_index].solvation[2*ii] * gradrho2 ;
    scforce[2+ii*3] += (1-flag_cs) * particle_data[part_index].solvation[2*ii] * gradrho3 ;

    /* scforce is used also later...*/
    particle_force[3*part_index+0] += scforce[0+ii*3];
    particle_force[3*part_index+1] += scforce[1+ii*3];
    particle_force[3*part_index+2] += scforce[2+ii*3];
    // only set fluid_composition for particle centre <=> (1-flag_cs) = 1
    fluid_composition[part_index].weight[ii] = (1-flag_cs) * interpolated_rho[ii];
 }

#else // SHANCHEN is not defined

  /* for LB we do not reweight the friction force */
  for(int ii=0; ii<LB_COMPONENTS; ++ii){
    interpolated_rho[ii]=1.0;
  }

#endif // SHANCHEN

  /** calculate viscous force
   * take care to rescale velocities with time_step and transform to MD units
   * (Eq. (9) Ahlrichs and Duenweg, JCP 111(17):8225 (1999)) */
  float rhotot=0;

  #pragma unroll
  for(int ii=0; ii<LB_COMPONENTS; ++ii){
    rhotot+=interpolated_rho[ii];
  }

  /* Viscous force */
  for(int ii=0; ii<LB_COMPONENTS; ++ii)
  { 
    viscforce[0+ii*3] -= interpolated_rho[ii]*para.friction[ii]*(velocity[0]/para.time_step - interpolated_u[0]*para.agrid/para.tau)/rhotot;
    viscforce[1+ii*3] -= interpolated_rho[ii]*para.friction[ii]*(velocity[1]/para.time_step - interpolated_u[1]*para.agrid/para.tau)/rhotot;
    viscforce[2+ii*3] -= interpolated_rho[ii]*para.friction[ii]*(velocity[2]/para.time_step - interpolated_u[2]*para.agrid/para.tau)/rhotot;

#ifdef LB_ELECTROHYDRODYNAMICS
    viscforce[0+ii*3] += interpolated_rho[ii]*para.friction[ii] * particle_data[part_index].mu_E[0]/rhotot;
    viscforce[1+ii*3] += interpolated_rho[ii]*para.friction[ii] * particle_data[part_index].mu_E[1]/rhotot;
    viscforce[2+ii*3] += interpolated_rho[ii]*para.friction[ii] * particle_data[part_index].mu_E[2]/rhotot;
#endif

    /** add stochastic force of zero mean (Ahlrichs, Duenweg equ. 15)*/
#ifdef FLATNOISE
    random_01(rn_part);
    viscforce[0+ii*3] += para.lb_coupl_pref[ii]*(rn_part->randomnr[0]-0.5f);
    viscforce[1+ii*3] += para.lb_coupl_pref[ii]*(rn_part->randomnr[1]-0.5f);
    random_01(rn_part);
    viscforce[2+ii*3] += para.lb_coupl_pref[ii]*(rn_part->randomnr[0]-0.5f);
#elif defined(GAUSSRANDOMCUT)
    gaussian_random_cut(rn_part);
    viscforce[0+ii*3] += para.lb_coupl_pref2[ii]*rn_part->randomnr[0];
    viscforce[1+ii*3] += para.lb_coupl_pref2[ii]*rn_part->randomnr[1];
    gaussian_random_cut(rn_part);
    viscforce[2+ii*3] += para.lb_coupl_pref2[ii]*rn_part->randomnr[0];
#elif defined(GAUSSRANDOM)
    gaussian_random(rn_part);
    viscforce[0+ii*3] += para.lb_coupl_pref2[ii]*rn_part->randomnr[0];
    viscforce[1+ii*3] += para.lb_coupl_pref2[ii]*rn_part->randomnr[1];
    gaussian_random(rn_part);
    viscforce[2+ii*3] += para.lb_coupl_pref2[ii]*rn_part->randomnr[0];
#else
#error No noise type defined for the GPU LB
#endif 

    /** delta_j for transform momentum transfer to lattice units which is done in calc_node_force
      (Eq. (12) Ahlrichs and Duenweg, JCP 111(17):8225 (1999)) */

    // only add to particle_force for particle centre <=> (1-flag_cs) = 1
    particle_force[3*part_index+0] += (1-flag_cs) * viscforce[0+ii*3];
    particle_force[3*part_index+1] += (1-flag_cs) * viscforce[1+ii*3];
    particle_force[3*part_index+2] += (1-flag_cs) * viscforce[2+ii*3];

    /* the average force from the particle to surrounding nodes is transmitted back to preserve momentum */
    for(int node=0 ; node < 8 ; node++ ) { 
      particle_force[3*part_index+0] -= (1-flag_cs) * partgrad1[node+ii*8]/8.0f;
      particle_force[3*part_index+1] -= (1-flag_cs) * partgrad2[node+ii*8]/8.0f;
      particle_force[3*part_index+2] -= (1-flag_cs) * partgrad3[node+ii*8]/8.0f;
    }

    /* note that scforce is zero if SHANCHEN is not #defined */
    // only add to particle_force for particle centre <=> (1-flag_cs) = 1
    delta_j[0+3*ii] -= (scforce[0+ii*3]+(1-flag_cs)*viscforce[0+ii*3])*para.time_step*para.tau/para.agrid;
    delta_j[1+3*ii] -= (scforce[1+ii*3]+(1-flag_cs)*viscforce[1+ii*3])*para.time_step*para.tau/para.agrid;
    delta_j[2+3*ii] -= (scforce[2+ii*3]+(1-flag_cs)*viscforce[2+ii*3])*para.time_step*para.tau/para.agrid;

#ifdef ENGINE
    // add swimming force to source position
    delta_j[0+3*ii] -= flag_cs*particle_data[part_index].swim.f_swim*particle_data[part_index].swim.quatu[0]*para.time_step*para.tau/para.agrid;
    delta_j[1+3*ii] -= flag_cs*particle_data[part_index].swim.f_swim*particle_data[part_index].swim.quatu[1]*para.time_step*para.tau/para.agrid;
    delta_j[2+3*ii] -= flag_cs*particle_data[part_index].swim.f_swim*particle_data[part_index].swim.quatu[2]*para.time_step*para.tau/para.agrid;
#endif

  }

#ifdef SHANCHEN
  for(int node=0 ; node < 8 ; node++ ) { 
    for(int ii=0 ; ii < LB_COMPONENTS ; ii++ ) { 
      partgrad1[node+ii*8]*=(para.time_step*para.tau/para.agrid);
      partgrad2[node+ii*8]*=(para.time_step*para.tau/para.agrid);
      partgrad3[node+ii*8]*=(para.time_step*para.tau/para.agrid);
    }
  }
#endif

}



/**calculation of the node force caused by the particles, with atomicadd due to avoiding race conditions 
  (Eq. (14) Ahlrichs and Duenweg, JCP 111(17):8225 (1999))
 * @param *delta        Pointer for the weighting of particle position (Input)
 * @param partgrad1             particle gradient for the Shan-Chen
 * @param partgrad2             particle gradient for the Shan-Chen
 * @param partgrad3             particle gradient for the Shan-Chen
 * @param *delta_j      Pointer for the weighting of particle momentum (Input)
 * @param node_index    node index around (8) particle (Input)
 * @param node_f        Pointer to the node force (Output).
*/
__device__ void calc_node_force(float *delta, float *delta_j, float * partgrad1, float * partgrad2, float * partgrad3,  unsigned int *node_index, LB_node_force_gpu node_f){
/* TODO: should the drag depend on the density?? */
/* NOTE: partgrad is not zero only if SHANCHEN is defined. It is initialized in calc_node_force. Alternatively one could 
         specialize this function to the single component LB */ 
  for(int ii=0; ii < LB_COMPONENTS; ++ii)
  { 
    atomicadd(&(node_f.force[(0+ii*3)*para.number_of_nodes + node_index[0]]), (delta[0]*delta_j[0+ii*3] + partgrad1[ii*8+0]));
    atomicadd(&(node_f.force[(1+ii*3)*para.number_of_nodes + node_index[0]]), (delta[0]*delta_j[1+ii*3] + partgrad2[ii*8+0]));
    atomicadd(&(node_f.force[(2+ii*3)*para.number_of_nodes + node_index[0]]), (delta[0]*delta_j[2+ii*3] + partgrad3[ii*8+0]));
                                                                                                      
    atomicadd(&(node_f.force[(0+ii*3)*para.number_of_nodes + node_index[1]]), (delta[1]*delta_j[0+ii*3] + partgrad1[ii*8+1]));
    atomicadd(&(node_f.force[(1+ii*3)*para.number_of_nodes + node_index[1]]), (delta[1]*delta_j[1+ii*3] + partgrad2[ii*8+1]));
    atomicadd(&(node_f.force[(2+ii*3)*para.number_of_nodes + node_index[1]]), (delta[1]*delta_j[2+ii*3] + partgrad3[ii*8+1]));
                                                                                                      
    atomicadd(&(node_f.force[(0+ii*3)*para.number_of_nodes + node_index[2]]), (delta[2]*delta_j[0+ii*3] + partgrad1[ii*8+2]));
    atomicadd(&(node_f.force[(1+ii*3)*para.number_of_nodes + node_index[2]]), (delta[2]*delta_j[1+ii*3] + partgrad2[ii*8+2]));
    atomicadd(&(node_f.force[(2+ii*3)*para.number_of_nodes + node_index[2]]), (delta[2]*delta_j[2+ii*3] + partgrad3[ii*8+2]));
                                                                                                      
    atomicadd(&(node_f.force[(0+ii*3)*para.number_of_nodes + node_index[3]]), (delta[3]*delta_j[0+ii*3] + partgrad1[ii*8+3]));
    atomicadd(&(node_f.force[(1+ii*3)*para.number_of_nodes + node_index[3]]), (delta[3]*delta_j[1+ii*3] + partgrad2[ii*8+3]));
    atomicadd(&(node_f.force[(2+ii*3)*para.number_of_nodes + node_index[3]]), (delta[3]*delta_j[2+ii*3] + partgrad3[ii*8+3]));
                                                                                                      
    atomicadd(&(node_f.force[(0+ii*3)*para.number_of_nodes + node_index[4]]), (delta[4]*delta_j[0+ii*3] + partgrad1[ii*8+4]));
    atomicadd(&(node_f.force[(1+ii*3)*para.number_of_nodes + node_index[4]]), (delta[4]*delta_j[1+ii*3] + partgrad2[ii*8+4]));
    atomicadd(&(node_f.force[(2+ii*3)*para.number_of_nodes + node_index[4]]), (delta[4]*delta_j[2+ii*3] + partgrad3[ii*8+4]));
                                                                                                      
    atomicadd(&(node_f.force[(0+ii*3)*para.number_of_nodes + node_index[5]]), (delta[5]*delta_j[0+ii*3] + partgrad1[ii*8+5]));
    atomicadd(&(node_f.force[(1+ii*3)*para.number_of_nodes + node_index[5]]), (delta[5]*delta_j[1+ii*3] + partgrad2[ii*8+5]));
    atomicadd(&(node_f.force[(2+ii*3)*para.number_of_nodes + node_index[5]]), (delta[5]*delta_j[2+ii*3] + partgrad3[ii*8+5]));
                                                                                                      
    atomicadd(&(node_f.force[(0+ii*3)*para.number_of_nodes + node_index[6]]), (delta[6]*delta_j[0+ii*3] + partgrad1[ii*8+6]));
    atomicadd(&(node_f.force[(1+ii*3)*para.number_of_nodes + node_index[6]]), (delta[6]*delta_j[1+ii*3] + partgrad2[ii*8+6]));
    atomicadd(&(node_f.force[(2+ii*3)*para.number_of_nodes + node_index[6]]), (delta[6]*delta_j[2+ii*3] + partgrad3[ii*8+6]));
                                                                                                      
    atomicadd(&(node_f.force[(0+ii*3)*para.number_of_nodes + node_index[7]]), (delta[7]*delta_j[0+ii*3] + partgrad1[ii*8+7]));
    atomicadd(&(node_f.force[(1+ii*3)*para.number_of_nodes + node_index[7]]), (delta[7]*delta_j[1+ii*3] + partgrad2[ii*8+7]));
    atomicadd(&(node_f.force[(2+ii*3)*para.number_of_nodes + node_index[7]]), (delta[7]*delta_j[2+ii*3] + partgrad3[ii*8+7]));
  }
}

/*********************************************************/
/** \name System setup and Kernel functions */
/*********************************************************/

/**kernel to calculate local populations from hydrodynamic fields.
 * The mapping is given in terms of the equilibrium distribution.
 *
 * Eq. (2.15) Ladd, J. Fluid Mech. 271, 295-309 (1994)
 * Eq. (4) in Berk Usta, Ladd and Butler, JCP 122, 094902 (2005)
 *
 * @param n_a   Pointer to the lattice site (Input).
 * @param *gpu_check additional check if gpu kernel are executed(Input).
 * @param *d_v    Pointer to local device values
 * @param *node_f          Pointer to node forces
*/
__global__ void calc_n_from_rho_j_pi(LB_nodes_gpu n_a, LB_rho_v_gpu *d_v, LB_node_force_gpu node_f, int *gpu_check) {
   /* TODO: this can handle only a uniform density, something similar, but local, 
            has to be called every time the fields are set by the user ! */ 
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
  if(index<para.number_of_nodes)
  {
    float mode[19*LB_COMPONENTS];

    #pragma unroll
    for(int ii=0;ii<LB_COMPONENTS;++ii)
    { 
      /** default values for fields in lattice units */
      gpu_check[0] = 1;
     
      float Rho = para.rho[ii]*para.agrid*para.agrid*para.agrid;
      float v[3] = { 0.0f, 0.0f, 0.0f };
      float pi[6] = { Rho*c_sound_sq, 0.0f, Rho*c_sound_sq, 0.0f, 0.0f, Rho*c_sound_sq };
     
      float rhoc_sq = Rho*c_sound_sq;
      float avg_rho = para.rho[ii]*para.agrid*para.agrid*para.agrid;
      float local_rho, local_j[3], *local_pi, trace;
     
      local_rho  = Rho;
     
      local_j[0] = Rho * v[0];
      local_j[1] = Rho * v[1];
      local_j[2] = Rho * v[2];
     
      local_pi = pi;
     
      /** reduce the pressure tensor to the part needed here. 
          NOTE: this not true anymore for SHANCHEN 
          if the densities are not uniform. FIXME*/

      local_pi[0] -= rhoc_sq;
      local_pi[2] -= rhoc_sq;
      local_pi[5] -= rhoc_sq;
     
      trace = local_pi[0] + local_pi[2] + local_pi[5];
     
      float rho_times_coeff;
      float tmp1,tmp2;
     
      /** update the q=0 sublattice */
      n_a.vd[(0 + ii*LBQ ) * para.number_of_nodes + index] = 1.0f/3.0f * (local_rho-avg_rho) - 1.0f/2.0f*trace;
     
      /** update the q=1 sublattice */
      rho_times_coeff = 1.0f/18.0f * (local_rho-avg_rho);
     
      n_a.vd[(1 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff + 1.0f/6.0f*local_j[0] + 1.0f/4.0f*local_pi[0] - 1.0f/12.0f*trace;
      n_a.vd[(2 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff - 1.0f/6.0f*local_j[0] + 1.0f/4.0f*local_pi[0] - 1.0f/12.0f*trace;
      n_a.vd[(3 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff + 1.0f/6.0f*local_j[1] + 1.0f/4.0f*local_pi[2] - 1.0f/12.0f*trace;
      n_a.vd[(4 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff - 1.0f/6.0f*local_j[1] + 1.0f/4.0f*local_pi[2] - 1.0f/12.0f*trace;
      n_a.vd[(5 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff + 1.0f/6.0f*local_j[2] + 1.0f/4.0f*local_pi[5] - 1.0f/12.0f*trace;
      n_a.vd[(6 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff - 1.0f/6.0f*local_j[2] + 1.0f/4.0f*local_pi[5] - 1.0f/12.0f*trace;
     
      /** update the q=2 sublattice */
      rho_times_coeff = 1.0f/36.0f * (local_rho-avg_rho);
     
      tmp1 = local_pi[0] + local_pi[2];
      tmp2 = 2.0f*local_pi[1];
      n_a.vd[( 7 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff + 1.0f/12.0f*(local_j[0]+local_j[1]) + 1.0f/8.0f*(tmp1+tmp2) - 1.0f/24.0f*trace;
      n_a.vd[( 8 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff - 1.0f/12.0f*(local_j[0]+local_j[1]) + 1.0f/8.0f*(tmp1+tmp2) - 1.0f/24.0f*trace;
      n_a.vd[( 9 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff + 1.0f/12.0f*(local_j[0]-local_j[1]) + 1.0f/8.0f*(tmp1-tmp2) - 1.0f/24.0f*trace;
      n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff - 1.0f/12.0f*(local_j[0]-local_j[1]) + 1.0f/8.0f*(tmp1-tmp2) - 1.0f/24.0f*trace;
     
      tmp1 = local_pi[0] + local_pi[5];
      tmp2 = 2.0f*local_pi[3];
     
      n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff + 1.0f/12.0f*(local_j[0]+local_j[2]) + 1.0f/8.0f*(tmp1+tmp2) - 1.0f/24.0f*trace;
      n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff - 1.0f/12.0f*(local_j[0]+local_j[2]) + 1.0f/8.0f*(tmp1+tmp2) - 1.0f/24.0f*trace;
      n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff + 1.0f/12.0f*(local_j[0]-local_j[2]) + 1.0f/8.0f*(tmp1-tmp2) - 1.0f/24.0f*trace;
      n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff - 1.0f/12.0f*(local_j[0]-local_j[2]) + 1.0f/8.0f*(tmp1-tmp2) - 1.0f/24.0f*trace;
     
      tmp1 = local_pi[2] + local_pi[5];
      tmp2 = 2.0f*local_pi[4];
     
      n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff + 1.0f/12.0f*(local_j[1]+local_j[2]) + 1.0f/8.0f*(tmp1+tmp2) - 1.0f/24.0f*trace;
      n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff - 1.0f/12.0f*(local_j[1]+local_j[2]) + 1.0f/8.0f*(tmp1+tmp2) - 1.0f/24.0f*trace;
      n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff + 1.0f/12.0f*(local_j[1]-local_j[2]) + 1.0f/8.0f*(tmp1-tmp2) - 1.0f/24.0f*trace;
      n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff - 1.0f/12.0f*(local_j[1]-local_j[2]) + 1.0f/8.0f*(tmp1-tmp2) - 1.0f/24.0f*trace;
     
      /**set different seed for randomgen on every node */
      n_a.seed[index] = para.your_seed + index;
    }

    calc_m_from_n(n_a,index,mode);
    update_rho_v(mode,index,node_f,d_v);
  }
}

/** kernel to calculate local populations from hydrodynamic fields
 * from given flow field velocities. The mapping is given in terms of
 * the equilibrium distribution.
 *
 * Eq. (2.15) Ladd, J. Fluid Mech. 271, 295-309 (1994)
 * Eq. (4) in Berk Usta, Ladd and Butler, JCP 122, 094902 (2005)
 *
 * @param n_a              the current nodes array (double buffering!)
 * @param single_nodeindex the node to set the velocity for
 * @param velocity         the velocity to set
 * @param *d_v             Pointer to local device values (Input)
 * @param *node_f          Pointer to node forces (Input)
 */ 
__global__ void set_u_from_rho_v_pi( LB_nodes_gpu n_a, int single_nodeindex, float *velocity, LB_rho_v_gpu *d_v, LB_node_force_gpu node_f ) {

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index == 0)
  {
    float local_rho;
    float local_j[3];
    float local_pi[6];
    float trace, avg_rho;
    float rho_times_coeff;
    float tmp1, tmp2; 

    float mode_for_pi[19*LB_COMPONENTS];
    float rho_from_m[1*LB_COMPONENTS];
    float j_from_m[3*LB_COMPONENTS];
    float pi_from_m[6*LB_COMPONENTS];

    // Calculate the modes for this node

    calc_m_from_n(n_a, single_nodeindex, mode_for_pi);

    // Reset the d_v

    update_rho_v(mode_for_pi, single_nodeindex, node_f, d_v);

    // Calculate the density, velocity, and pressure tensor
    // in LB unit for this node

    calc_values_from_m_in_LB_units( mode_for_pi, &d_v[single_nodeindex], rho_from_m, j_from_m, pi_from_m);

    #pragma unroll
    for(int ii=0;ii<LB_COMPONENTS;++ii)
    { 
      // Take LB component density and calculate the equilibrium part

      local_rho = rho_from_m[ii];
      avg_rho = para.rho[ii]*para.agrid*para.agrid*para.agrid;

      // Take LB component velocity and make it a momentum

      local_j[0] = local_rho * velocity[0];
      local_j[1] = local_rho * velocity[1];
      local_j[2] = local_rho * velocity[2];

      // Take LB component pressure tensor and put in equilibrium

      local_pi[0] = pi_from_m[6*ii + 0];
      local_pi[1] = pi_from_m[6*ii + 1];
      local_pi[2] = pi_from_m[6*ii + 2];
      local_pi[3] = pi_from_m[6*ii + 3];
      local_pi[4] = pi_from_m[6*ii + 4];
      local_pi[5] = pi_from_m[6*ii + 5];

      trace = local_pi[0] + local_pi[2] + local_pi[5];

      // update the q=0 sublattice

      n_a.vd[(0 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.0f/3.0f * (local_rho - avg_rho) - 1.0f/2.0f*trace;

      // update the q=1 sublattice

      rho_times_coeff = 1.0f/18.0f * (local_rho - avg_rho);

      n_a.vd[(1 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] =   rho_times_coeff + 1.0f/6.0f*local_j[0]
                                                                        + 1.0f/4.0f*local_pi[0] - 1.0f/12.0f*trace;
      n_a.vd[(2 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] =   rho_times_coeff - 1.0f/6.0f*local_j[0]
                                                                        + 1.0f/4.0f*local_pi[0] - 1.0f/12.0f*trace;
      n_a.vd[(3 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] =   rho_times_coeff + 1.0f/6.0f*local_j[1]
                                                                        + 1.0f/4.0f*local_pi[2] - 1.0f/12.0f*trace;
      n_a.vd[(4 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] =   rho_times_coeff - 1.0f/6.0f*local_j[1]
                                                                        + 1.0f/4.0f*local_pi[2] - 1.0f/12.0f*trace;
      n_a.vd[(5 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] =   rho_times_coeff + 1.0f/6.0f*local_j[2]
                                                                        + 1.0f/4.0f*local_pi[5] - 1.0f/12.0f*trace;
      n_a.vd[(6 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] =   rho_times_coeff - 1.0f/6.0f*local_j[2]
                                                                        + 1.0f/4.0f*local_pi[5] - 1.0f/12.0f*trace;

      // update the q=2 sublattice

      rho_times_coeff = 1.0f/36.0f * (local_rho - avg_rho);

      tmp1 = local_pi[0] + local_pi[2];
      tmp2 = 2.0f*local_pi[1];

      n_a.vd[( 7 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] =   rho_times_coeff + 1.0f/12.0f*(local_j[0]+local_j[1])
                                                                         + 1.0f/8.0f*(tmp1+tmp2) - 1.0f/24.0f*trace;
      n_a.vd[( 8 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] =   rho_times_coeff - 1.0f/12.0f*(local_j[0]+local_j[1])
                                                                         + 1.0f/8.0f*(tmp1+tmp2) - 1.0f/24.0f*trace;
      n_a.vd[( 9 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] =   rho_times_coeff + 1.0f/12.0f*(local_j[0]-local_j[1])
                                                                         + 1.0f/8.0f*(tmp1-tmp2) - 1.0f/24.0f*trace;
      n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] =   rho_times_coeff - 1.0f/12.0f*(local_j[0]-local_j[1])
                                                                         + 1.0f/8.0f*(tmp1-tmp2) - 1.0f/24.0f*trace;

      tmp1 = local_pi[0] + local_pi[5];
      tmp2 = 2.0f*local_pi[3];

      n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] =   rho_times_coeff + 1.0f/12.0f*(local_j[0]+local_j[2])
                                                                         + 1.0f/8.0f*(tmp1+tmp2) - 1.0f/24.0f*trace;
      n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] =  rho_times_coeff - 1.0f/12.0f*(local_j[0]+local_j[2])
                                                                         + 1.0f/8.0f*(tmp1+tmp2) - 1.0f/24.0f*trace;
      n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] =  rho_times_coeff + 1.0f/12.0f*(local_j[0]-local_j[2])
                                                                         + 1.0f/8.0f*(tmp1-tmp2) - 1.0f/24.0f*trace;
      n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] =  rho_times_coeff - 1.0f/12.0f*(local_j[0]-local_j[2])
                                                                         + 1.0f/8.0f*(tmp1-tmp2) - 1.0f/24.0f*trace;

      tmp1 = local_pi[2] + local_pi[5];
      tmp2 = 2.0f*local_pi[4];

      n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] =   rho_times_coeff + 1.0f/12.0f*(local_j[1]+local_j[2])
                                                                         + 1.0f/8.0f*(tmp1+tmp2) - 1.0f/24.0f*trace;
      n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] =   rho_times_coeff - 1.0f/12.0f*(local_j[1]+local_j[2])
                                                                         + 1.0f/8.0f*(tmp1+tmp2) - 1.0f/24.0f*trace;
      n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] =   rho_times_coeff + 1.0f/12.0f*(local_j[1]-local_j[2])
                                                                         + 1.0f/8.0f*(tmp1-tmp2) - 1.0f/24.0f*trace;
      n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] =   rho_times_coeff - 1.0f/12.0f*(local_j[1]-local_j[2])
                                                                         + 1.0f/8.0f*(tmp1-tmp2) - 1.0f/24.0f*trace;
    }

    // Calculate the modes for this node

    calc_m_from_n(n_a, single_nodeindex, mode_for_pi);

    // Update the density and velocity field for this mode

    update_rho_v(mode_for_pi, single_nodeindex, node_f, d_v);
  }
}



/**calculate mass of the whole fluid kernel
 * @param *sum    Pointer to result storage value (Output)
 * @param n_a     Pointer to local node residing in array a (Input)
*/
__global__ void calc_mass(LB_nodes_gpu n_a, float *sum) {
  float mode[4];

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes)
  {
    for(int ii=0;ii<LB_COMPONENTS;++ii)
    { 
      calc_mode(mode, n_a, index,ii);
      float Rho = mode[0] + para.rho[ii]*para.agrid*para.agrid*para.agrid;
      atomicadd(&(sum[0]), Rho);
    }
  }
}

/** (re-)initialization of the node force / set up of external force in lb units
 * @param node_f  Pointer to local node force (Input)
*/
__global__ void reinit_node_force(LB_node_force_gpu node_f){

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes)
  {
    #pragma unroll
    for(int ii=0;ii<LB_COMPONENTS;++ii)
    {
#ifdef EXTERNAL_FORCES
      if(para.external_force)
      {
        node_f.force[(0+ii*3)*para.number_of_nodes + index] = para.ext_force[0+ii*3]*para.agrid*para.agrid*para.tau*para.tau;
        node_f.force[(1+ii*3)*para.number_of_nodes + index] = para.ext_force[1+ii*3]*para.agrid*para.agrid*para.tau*para.tau;
        node_f.force[(2+ii*3)*para.number_of_nodes + index] = para.ext_force[2+ii*3]*para.agrid*para.agrid*para.tau*para.tau;
      }
      else
      {
        node_f.force[(0+ii*3)*para.number_of_nodes + index] = 0.0f;
        node_f.force[(1+ii*3)*para.number_of_nodes + index] = 0.0f;
        node_f.force[(2+ii*3)*para.number_of_nodes + index] = 0.0f;
      }
#else
      node_f.force[(0+ii*3)*para.number_of_nodes + index] = 0.0f;
      node_f.force[(1+ii*3)*para.number_of_nodes + index] = 0.0f;
      node_f.force[(2+ii*3)*para.number_of_nodes + index] = 0.0f;
#endif
    }
  }
}


/**set extern force on single nodes kernel
 * @param n_extern_nodeforces   number of nodes (Input)
 * @param *extern_nodeforces    Pointer to extern node force array (Input)
 * @param node_f                node force struct (Output)
*/
__global__ void init_extern_nodeforces(int n_extern_nodeforces, LB_extern_nodeforce_gpu *extern_nodeforces, LB_node_force_gpu node_f){

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
  float factor=powf(para.agrid,2)*para.tau*para.tau;
  if(index<n_extern_nodeforces)
  {
    #pragma unroll
    for(int ii=0;ii<LB_COMPONENTS;++ii)
    {
      node_f.force[(0+ii*3)*para.number_of_nodes + extern_nodeforces[index].index] = extern_nodeforces[index].force[0] * factor;
      node_f.force[(1+ii*3)*para.number_of_nodes + extern_nodeforces[index].index] = extern_nodeforces[index].force[1] * factor;
      node_f.force[(2+ii*3)*para.number_of_nodes + extern_nodeforces[index].index] = extern_nodeforces[index].force[2] * factor;
    }
  }
}

#ifdef SHANCHEN

/** 
 * @param single_nodeindex  Single node index        (Input)
 * @param component_index   Shanchen component index        (Input)
 * @param n_a               Pointer to local node residing in array a(Input)
*/
__device__ __inline__ float calc_massmode(LB_nodes_gpu n_a, int single_nodeindex, int component_index){

  /** mass mode */
  float mode;
  mode =   n_a.vd[( 0 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex]
         + n_a.vd[( 1 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[( 2 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] 
         + n_a.vd[( 3 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[( 4 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex]
         + n_a.vd[( 5 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[( 6 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex]
         + n_a.vd[( 7 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[( 8 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex]
         + n_a.vd[( 9 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(10 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex]
         + n_a.vd[(11 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(12 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex]
         + n_a.vd[(13 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(14 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex]
         + n_a.vd[(15 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(16 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex]
         + n_a.vd[(17 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(18 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex];

 mode += para.rho[component_index]*para.agrid*para.agrid*para.agrid;

 return mode;
}

__device__ __inline__ void calc_shanchen_contribution(LB_nodes_gpu n_a,int component_index, int x, int y, int z, float *p){ 

  float tmp_p[3]={0.0f,0.0f,0.0f};
  float pseudo;
  int index;

  index  = (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z;
  pseudo =  calc_massmode(n_a,index,component_index);
  tmp_p[0]+=pseudo/18.0f;

  index  = (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z;
  pseudo =  calc_massmode(n_a,index,component_index);
  tmp_p[0]-=pseudo/18.0f;

  index  = x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z;
  pseudo =  calc_massmode(n_a,index,component_index);
  tmp_p[1]+=pseudo/18.0f;

  index  = x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z;
  pseudo =  calc_massmode(n_a,index,component_index);
  tmp_p[1]-=pseudo/18.0f;

  index  = x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z);
  pseudo =  calc_massmode(n_a,index,component_index);
  tmp_p[2]+=pseudo/18.0f;

  index  = x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z);
  pseudo =  calc_massmode(n_a,index,component_index);
  tmp_p[2]-=pseudo/18.0f;

  index  = (x+1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z;
  pseudo =  calc_massmode(n_a,index,component_index);
  tmp_p[0]+=pseudo/36.0f;
  tmp_p[1]+=pseudo/36.0f;

  index  = (para.dim_x+x-1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z;
  pseudo =  calc_massmode(n_a,index,component_index);
  tmp_p[0]-=pseudo/36.0f;
  tmp_p[1]-=pseudo/36.0f;

  index  = (x+1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z;
  pseudo =  calc_massmode(n_a,index,component_index);
  tmp_p[0]+=pseudo/36.0f;
  tmp_p[1]-=pseudo/36.0f;

  index  = (para.dim_x+x-1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z;
  pseudo =  calc_massmode(n_a,index,component_index);
  tmp_p[0]-=pseudo/36.0f;
  tmp_p[1]+=pseudo/36.0f;

  index  = (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z);
  pseudo =  calc_massmode(n_a,index,component_index);
  tmp_p[0]+=pseudo/36.0f;
  tmp_p[2]+=pseudo/36.0f;

  index  = (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z);
  pseudo =  calc_massmode(n_a,index,component_index);
  tmp_p[0]-=pseudo/36.0f;
  tmp_p[2]-=pseudo/36.0f;

  index  = (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z);
  pseudo =  calc_massmode(n_a,index,component_index);
  tmp_p[0]+=pseudo/36.0f;
  tmp_p[2]-=pseudo/36.0f;

  index  = (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z);
  pseudo =  calc_massmode(n_a,index,component_index);
  tmp_p[0]-=pseudo/36.0f;
  tmp_p[2]+=pseudo/36.0f;

  index  = x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z);
  pseudo =  calc_massmode(n_a,index,component_index);
  tmp_p[1]+=pseudo/36.0f;
  tmp_p[2]+=pseudo/36.0f;

  index  = x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z);
  pseudo =  calc_massmode(n_a,index,component_index);
  tmp_p[1]-=pseudo/36.0f;
  tmp_p[2]-=pseudo/36.0f;

  index  = x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z);
  pseudo =  calc_massmode(n_a,index,component_index);
  tmp_p[1]+=pseudo/36.0f;
  tmp_p[2]-=pseudo/36.0f;
  
  index  = x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z);
  pseudo =  calc_massmode(n_a,index,component_index);
  tmp_p[1]-=pseudo/36.0f;
  tmp_p[2]+=pseudo/36.0f;
 
  p[0]=tmp_p[0];
  p[1]=tmp_p[1];
  p[2]=tmp_p[2];
}

/** function to calc shanchen forces 
 * @param n_a     Pointer to local node residing in array a(Input)
 * @param node_f  Pointer to local node force (Input)
*/
__global__ void lb_shanchen_GPU(LB_nodes_gpu n_a,LB_node_force_gpu node_f){
#ifndef D3Q19
#error Lattices other than D3Q19 not supported
#endif
#if ( LB_COMPONENTS == 1  ) 
  #warning shanchen forces not implemented 
#else  

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int xyz[3];
  float pseudo;

  if(index<para.number_of_nodes)
  if( n_a.boundary[index] == 0 )
  {

    /* ShanChen forces are not reset at the end of the integration cycle, 
       in order to compute properly the hydrodynamic fields, so we have
       to reset them here. For the standard LB this is not needed */
     reset_LB_forces(index, node_f) ;
     /*Let's first identify the neighboring nodes */
     index_to_xyz(index, xyz);
     int x = xyz[0];
     int y = xyz[1];
     int z = xyz[2];
     
     #pragma unroll
     for(int ii=0;ii<LB_COMPONENTS;ii++)
     { 
       float p[3]={0.0f,0.0f,0.0f};
       pseudo =  calc_massmode(n_a,index,ii);

       #pragma unroll
       for(int jj=0;jj<LB_COMPONENTS;jj++)
       { 
             float tmpp[3]={0.0f,0.0f,0.0f};
             calc_shanchen_contribution(n_a, jj, x,y,z, tmpp);

             // FIXME  coupling HAS to be rescaled with agrid....
             p[0] += - para.coupling[(LB_COMPONENTS)*ii+jj]  * pseudo  * tmpp[0];
             p[1] += - para.coupling[(LB_COMPONENTS)*ii+jj]  * pseudo  * tmpp[1];
             p[2] += - para.coupling[(LB_COMPONENTS)*ii+jj]  * pseudo  * tmpp[2];
       }

       node_f.force[(0+ii*3)*para.number_of_nodes + index]+=p[0];
       node_f.force[(1+ii*3)*para.number_of_nodes + index]+=p[1];
       node_f.force[(2+ii*3)*para.number_of_nodes + index]+=p[2];
/* copy to be used when resetting forces */
       node_f.scforce[(0+ii*3)*para.number_of_nodes + index]=p[0];
       node_f.scforce[(1+ii*3)*para.number_of_nodes + index]=p[1];
       node_f.scforce[(2+ii*3)*para.number_of_nodes + index]=p[2];
    }
  }
#endif 
  return; 
}

#endif //SHANCHEN

/** kernel to set the local density
 *
 * @param n_a               the current nodes array (double buffering!)
 * @param single_nodeindex  the node to set the velocity for
 * @param rho               the density to set
 * @param d_v                Pointer to the local modes
*/
__global__ void set_rho(LB_nodes_gpu n_a,  LB_rho_v_gpu *d_v, int single_nodeindex,float *rho) {

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
  /*Note: this sets the velocities to zero */
  if(index == 0)
  {
    float local_rho;

    #pragma unroll
    for(int ii=0;ii<LB_COMPONENTS;++ii)
    { 
       /** default values for fields in lattice units */
       local_rho = (rho[ii]-para.rho[ii])*para.agrid*para.agrid*para.agrid;
       d_v[single_nodeindex].rho[ii]=rho[ii];

       n_a.vd[(0  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.0f/ 3.0f * local_rho ;
       n_a.vd[(1  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.0f/18.0f * local_rho ;
       n_a.vd[(2  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.0f/18.0f * local_rho ;
       n_a.vd[(3  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.0f/18.0f * local_rho ;
       n_a.vd[(4  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.0f/18.0f * local_rho ;
       n_a.vd[(5  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.0f/18.0f * local_rho ;
       n_a.vd[(6  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.0f/18.0f * local_rho ;
       n_a.vd[(7  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.0f/36.0f * local_rho ;
       n_a.vd[(8  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.0f/36.0f * local_rho ;
       n_a.vd[(9  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.0f/36.0f * local_rho ;
       n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.0f/36.0f * local_rho ;
       n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.0f/36.0f * local_rho ;
       n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.0f/36.0f * local_rho ;
       n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.0f/36.0f * local_rho ;
       n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.0f/36.0f * local_rho ;
       n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.0f/36.0f * local_rho ;
       n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.0f/36.0f * local_rho ;
       n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.0f/36.0f * local_rho ;
       n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.0f/36.0f * local_rho ;
    }
  }
}

/**set the boundary flag for all boundary nodes
 * @param boundary_node_list    The indices of the boundary nodes
 * @param boundary_index_list   The flag representing the corresponding boundary
 * @param number_of_boundnodes  The number of boundary nodes
 * @param n_a                   Pointer to local node residing in array a (Input)
 * @param n_b                   Pointer to local node residing in array b (Input)
*/
__global__ void init_boundaries(int *boundary_node_list, int *boundary_index_list, int number_of_boundnodes, LB_nodes_gpu n_a, LB_nodes_gpu n_b){

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<number_of_boundnodes)
  {
    n_a.boundary[boundary_node_list[index]] = boundary_index_list[index];
    n_b.boundary[boundary_node_list[index]] = boundary_index_list[index];
  }
}

/**reset the boundary flag of every node
 * @param n_a   Pointer to local node residing in array a (Input)
 * @param n_b   Pointer to local node residing in array b (Input)
*/
__global__ void reset_boundaries(LB_nodes_gpu n_a, LB_nodes_gpu n_b){

  size_t index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes)
    n_a.boundary[index] = n_b.boundary[index] = 0;
}

/** integrationstep of the lb-fluid-solver
 * @param n_a     Pointer to local node residing in array a (Input)
 * @param n_b     Pointer to local node residing in array b (Input)
 * @param *d_v    Pointer to local device values (Input)
 * @param node_f  Pointer to local node force (Input)
 * @param ek_parameters_gpu  Pointer to the parameters for the electrokinetics (Input)
*/


__global__ void integrate(LB_nodes_gpu n_a, LB_nodes_gpu n_b, LB_rho_v_gpu *d_v, LB_node_force_gpu node_f, EK_parameters* ek_parameters_gpu) {
  /**every node is connected to a thread via the index*/
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
  /**the 19 moments (modes) are only temporary register values */
  float mode[19*LB_COMPONENTS];
  LB_randomnr_gpu rng;

  if( index < para.number_of_nodes )
  {
    /** storing the seed into a register value*/
    rng.seed = n_a.seed[index];
    /**calc_m_from_n*/
    calc_m_from_n(n_a, index, mode);
    /**lb_relax_modes*/
    relax_modes(mode, index, node_f,d_v);
    /**lb_thermalize_modes */
    if (para.fluct)
    {
      thermalize_modes(mode, index, &rng);
    }
#if  defined(EXTERNAL_FORCES)  ||   defined (SHANCHEN)  
    /**if external force is used apply node force */
    apply_forces(index, mode, node_f,d_v);
#else
    /**if particles are used apply node forces*/
    if (para.number_of_particles) apply_forces(index, mode, node_f,d_v); 
#endif
    /**lb_calc_n_from_modes_push*/
    normalize_modes(mode);
    /**calc of velocity densities and streaming with pbc*/
    calc_n_from_modes_push(n_b, mode, index);
    /** rewriting the seed back to the global memory*/
    n_b.seed[index] = rng.seed;
  }  
}

/** part interaction kernel
 * @param n_a                Pointer to local node residing in array a (Input)
 * @param *particle_data     Pointer to the particle position and velocity (Input)
 * @param *particle_force    Pointer to the particle force (Input)
 * @param *part              Pointer to the rn array of the particles (Input)
 * @param node_f             Pointer to local node force (Input)
 * @param *fluid_composition Pointer to the local fluid composition for the Shanchen
 * @param *d_v               Pointer to local device values
*/
__global__ void calc_fluid_particle_ia(LB_nodes_gpu n_a, CUDA_particle_data *particle_data, float *particle_force, CUDA_fluid_composition * fluid_composition, LB_node_force_gpu node_f, CUDA_particle_seed *part, LB_rho_v_gpu *d_v){

  unsigned int part_index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int node_index[8];
  float delta[8];
  float delta_j[3*LB_COMPONENTS]; 
  float partgrad1[8*LB_COMPONENTS]; 
  float partgrad2[8*LB_COMPONENTS]; 
  float partgrad3[8*LB_COMPONENTS]; 
  LB_randomnr_gpu rng_part;
  if(part_index<para.number_of_particles)
  {
#if defined(IMMERSED_BOUNDARY) || defined(VIRTUAL_SITES_COM)
    if ( !particle_data[part_index].isVirtual )
#endif
    {
      rng_part.seed = part[part_index].seed;

      /**force acting on the particle. delta_j will be used later to compute the force that acts back onto the fluid. */
      calc_viscous_force(n_a, delta, partgrad1, partgrad2, partgrad3, particle_data, particle_force, fluid_composition,part_index, &rng_part, delta_j, node_index, d_v, 0);
      calc_node_force(delta, delta_j, partgrad1, partgrad2, partgrad3, node_index, node_f); 

#ifdef ENGINE
      if ( particle_data[part_index].swim.swimming ) {
        calc_viscous_force(n_a, delta, partgrad1, partgrad2, partgrad3, particle_data, particle_force, fluid_composition,part_index, &rng_part, delta_j, node_index, d_v, 1);
        calc_node_force(delta, delta_j, partgrad1, partgrad2, partgrad3, node_index, node_f);
      }
#endif

      /**force which acts back to the fluid node */
      part[part_index].seed = rng_part.seed;
    }
  }
}

/** part interaction kernel
 * @param n_a       Pointer to local node residing in array a (Input)
 * @param *particle_data    Pointer to the particle position and velocity (Input)
 * @param *particle_force   Pointer to the particle force (Input)
 * @param *part       Pointer to the rn array of the particles (Input)
 * @param node_f      Pointer to local node force (Input)
 * @param *d_v    Pointer to local device values
*/
__global__ void calc_fluid_particle_ia_three_point_couple(LB_nodes_gpu n_a, CUDA_particle_data *particle_data, float *particle_force, LB_node_force_gpu node_f, CUDA_particle_seed *part, LB_rho_v_gpu *d_v){

  unsigned int part_index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int node_index[27];
  float delta[27];
  float delta_j[3*LB_COMPONENTS]; 
  LB_randomnr_gpu rng_part;
  if(part_index<para.number_of_particles){

    rng_part.seed = part[part_index].seed;
    /**force acting on the particle. delta_j will be used later to compute the force that acts back onto the fluid. */
    calc_viscous_force_three_point_couple(n_a, delta, particle_data, particle_force, part_index, &rng_part, delta_j, node_index,d_v,0);
    calc_node_force_three_point_couple(delta, delta_j, node_index, node_f);

#ifdef ENGINE
    if ( particle_data[part_index].swim.swimming ) {
      calc_viscous_force_three_point_couple(n_a, delta, particle_data, particle_force, part_index, &rng_part, delta_j, node_index,d_v,1);
      calc_node_force_three_point_couple(delta, delta_j, node_index, node_f);
    }
#endif

    /**force which acts back to the fluid node */
    part[part_index].seed = rng_part.seed;    
  }
}


#ifdef LB_BOUNDARIES_GPU
/**Bounce back boundary kernel
 * @param n_a         Pointer to local node residing in array a (Input)
 * @param n_b         Pointer to local node residing in array b (Input)
 * @param lb_boundary_velocity    The constant velocity at the boundary, set by the user (Input)
 * @param lb_boundary_force       The force on the boundary nodes (Output)
*/
__global__ void apply_boundaries(LB_nodes_gpu n_curr, float* lb_boundary_velocity, float* lb_boundary_force){

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes)
    bounce_back_boundaries(n_curr, index, lb_boundary_velocity, lb_boundary_force);
}
#ifdef SHANCHEN
__global__ void lb_shanchen_set_boundaries(LB_nodes_gpu n_curr){
/* This implements neutral boundary conditions for the shanchen fluid (i.e., 90 deg contact angle) */

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int xyz[3];
  if(index<para.number_of_nodes){
   if(n_curr.boundary[index] != 0 ) { 
    index_to_xyz(index, xyz);
    unsigned int x = xyz[0];
    unsigned int y = xyz[1];
    unsigned int z = xyz[2];
    unsigned int to_index_x,to_index_y,to_index_z,to_index;
    int c[3],count=0;

    for(int ii=0;ii<LB_COMPONENTS;ii++)
       for(int comp=0;comp<19;comp++)
          n_curr.vd[(comp + ii*LBQ ) * para.number_of_nodes + index]  =  0.0 ; 
    for(c[0]=-1;c[0]<=1;c[0]++){
       for(c[1]=-1;c[1]<=1;c[1]++){
          for(c[2]=-1;c[2]<=1;c[2]++){
             to_index_x = (x+c[0]+para.dim_x)%para.dim_x; 
             to_index_y = (y+c[1]+para.dim_y)%para.dim_y; 
             to_index_z = (z+c[2]+para.dim_z)%para.dim_z; 
             to_index = to_index_x + para.dim_x*to_index_y + para.dim_x*para.dim_y*to_index_z;  
	     if(n_curr.boundary[to_index] == 0 ) { 
                  for(int ii=0;ii<LB_COMPONENTS;ii++){
                     for(int comp=0;comp<19;comp++){ /* We copy all velocities: at the end we will need 
                                                        only the density mode, but this introduces no overhead anyway */
	                 n_curr.vd[(comp + ii*LBQ ) * para.number_of_nodes + index] += 
	                    n_curr.vd[(comp + ii*LBQ ) * para.number_of_nodes + to_index] ;
                         count++;
                     }
                  }
             }	      	
          }
       }
    }
    if(count>0)
      for(int ii=0;ii<LB_COMPONENTS;ii++)
        for(int comp=0;comp<19;comp++)
           n_curr.vd[(comp + ii*LBQ ) * para.number_of_nodes + index]  /= count ; 
  }
 }
}
#endif /* SHANCHEN */


#endif

/** get physical values of the nodes (density, velocity, ...)
 * @param n_a     Pointer to local node residing in array a (Input)
 * @param *p_v    Pointer to local print values (Output)
 * @param *d_v    Pointer to local device values (Input)
 * @param node_f  The forces on the LB nodes
*/
__global__ void get_mesoscopic_values_in_MD_units(LB_nodes_gpu n_a, LB_rho_v_pi_gpu *p_v,LB_rho_v_gpu *d_v, LB_node_force_gpu node_f) {
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index < para.number_of_nodes)
  {
    float mode[19*LB_COMPONENTS];
    calc_m_from_n(n_a, index, mode);
    calc_values_in_MD_units(n_a, mode, p_v, d_v, node_f, index, index);
  }
}

/** get boundary flags
 *  @param n_a                Pointer to local node residing in array a (Input)
 *  @param device_bound_array Pointer to local device values (Input)
 */
__global__ void lb_get_boundaries(LB_nodes_gpu n_a, unsigned int *device_bound_array){

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes)
   device_bound_array[index] = n_a.boundary[index];
}



/**print single node values kernel
 * @param single_nodeindex  index of the node (Input)
 * @param *d_p_v            Pointer to result storage array (Input)
 * @param n_a               Pointer to local node residing in array a (Input)
 * @param *d_v    Pointer to local device values
 * @param node_f  Pointer to local node force
*/
__global__ void lb_print_node(int single_nodeindex, LB_rho_v_pi_gpu *d_p_v, LB_nodes_gpu n_a, LB_rho_v_gpu * d_v, LB_node_force_gpu node_f){

  float mode[19*LB_COMPONENTS];
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index == 0)
  {
    calc_m_from_n(n_a, single_nodeindex, mode);
     
    /* the following actually copies rho and v from d_v, and calculates pi */
    calc_values_in_MD_units(n_a, mode, d_p_v, d_v, node_f, single_nodeindex, 0);
  }
}
__global__ void momentum(LB_nodes_gpu n_a, LB_rho_v_gpu * d_v, LB_node_force_gpu node_f, float *sum) {

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes)
  {
    float j[3]={0.0f,0.0f,0.0f};
    float mode[4];

    for(int ii=0 ; ii < LB_COMPONENTS ; ii++ )
    { 
      calc_mode(mode, n_a, index,ii);

      j[0] += mode[1]+node_f.force[(0+ii*3)*para.number_of_nodes + index];
      j[1] += mode[2]+node_f.force[(1+ii*3)*para.number_of_nodes + index];
      j[2] += mode[3]+node_f.force[(2+ii*3)*para.number_of_nodes + index];
    }

#ifdef LB_BOUNDARIES_GPU
    if(n_a.boundary[index])
      j[0]=j[1]=j[2]=0.0f;
#endif

    atomicadd(&(sum[0]), j[0]); 
    atomicadd(&(sum[1]), j[1]); 
    atomicadd(&(sum[2]), j[2]); 
  }
}
__global__ void remove_momentum(LB_nodes_gpu n_a, LB_rho_v_gpu * d_v, LB_node_force_gpu node_f, float *sum) {

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
  if(index<para.number_of_nodes){
    for(int ii=0 ; ii < LB_COMPONENTS ; ii++ ) { 
        node_f.force[(0+ii*3)*para.number_of_nodes + index]-=sum[0]/para.number_of_nodes;
        node_f.force[(1+ii*3)*para.number_of_nodes + index]-=sum[1]/para.number_of_nodes;
        node_f.force[(2+ii*3)*para.number_of_nodes + index]-=sum[2]/para.number_of_nodes;
    }
  }
}

/**print single node boundary flag
 * @param single_nodeindex  index of the node (Input)
 * @param *device_flag      Pointer to result storage array (Input)
 * @param n_a               Pointer to local node residing in array a (Input)
*/
__global__ void lb_get_boundary_flag(int single_nodeindex, unsigned int *device_flag, LB_nodes_gpu n_a){

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index == 0)
    device_flag[0] = n_a.boundary[single_nodeindex];
}

/**********************************************************************/
/* Host functions to setup and call kernels*/
/**********************************************************************/

void lb_get_para_pointer(LB_parameters_gpu** pointeradress) {
  if(hipGetSymbolAddress((void**) pointeradress, para) != hipSuccess)
  {
    fprintf(stderr, "Trouble getting address of LB parameters.\n"); //TODO give proper error message
    errexit();
  }
}

void lb_get_lbpar_pointer(LB_parameters_gpu** pointeradress) {
  *pointeradress = &lbpar_gpu;
}


void lb_get_boundary_force_pointer(float** pointeradress) {
#ifdef LB_BOUNDARIES_GPU
  *pointeradress = lb_boundary_force;
#endif
}

void lb_get_device_values_pointer(LB_rho_v_gpu** pointeradress) {
  *pointeradress = device_rho_v;
}

/**initialization for the lb gpu fluid called from host
 * @param *lbpar_gpu  Pointer to parameters to setup the lb field
*/
void lb_init_GPU(LB_parameters_gpu *lbpar_gpu){
#define free_and_realloc(var,size)\
  { if( (var) != nullptr ) hipFree((var)); cuda_safe_mem(hipMalloc((void**)&var, size)); } 

  size_of_rho_v     = lbpar_gpu->number_of_nodes * sizeof(LB_rho_v_gpu);
  size_of_rho_v_pi  = lbpar_gpu->number_of_nodes * sizeof(LB_rho_v_pi_gpu);


  /** Allocate structs in device memory*/
  /* see the notes to the stucture device_rho_v_pi above...*/
  if(extended_values_flag==0) 
  {
    free_and_realloc(device_rho_v, size_of_rho_v);
  }
  else 
  {
    free_and_realloc(device_rho_v_pi, size_of_rho_v_pi);
  }

  /* TODO: this is a almost a copy copy of  device_rho_v think about eliminating it, and maybe pi can be added to device_rho_v in this case*/
  free_and_realloc(print_rho_v_pi  , size_of_rho_v_pi);
  free_and_realloc(nodes_a.vd      , lbpar_gpu->number_of_nodes * 19 * LB_COMPONENTS * sizeof(float));
  free_and_realloc(nodes_b.vd      , lbpar_gpu->number_of_nodes * 19 * LB_COMPONENTS * sizeof(float));   
  free_and_realloc(node_f.force    , lbpar_gpu->number_of_nodes *  3 * LB_COMPONENTS * sizeof(lbForceFloat));
#if defined(IMMERSED_BOUNDARY) || defined(EK_DEBUG)
  free_and_realloc(node_f.force_buf    , lbpar_gpu->number_of_nodes *  3 * LB_COMPONENTS * sizeof(lbForceFloat));
#endif
#ifdef SHANCHEN
  free_and_realloc(node_f.scforce  , lbpar_gpu->number_of_nodes *  3 * LB_COMPONENTS * sizeof(float));
#endif

  free_and_realloc(nodes_a.seed    , lbpar_gpu->number_of_nodes * sizeof( unsigned int));
  free_and_realloc(nodes_a.boundary, lbpar_gpu->number_of_nodes * sizeof( unsigned int));
  free_and_realloc(nodes_b.seed    , lbpar_gpu->number_of_nodes * sizeof( unsigned int));
  free_and_realloc(nodes_b.boundary, lbpar_gpu->number_of_nodes * sizeof( unsigned int));

  /**write parameters in const memory*/
  cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu, sizeof(LB_parameters_gpu)));

  /**check flag if lb gpu init works*/
  free_and_realloc(gpu_check, sizeof(int));

  if(h_gpu_check!=nullptr)
    free(h_gpu_check);  

  h_gpu_check = (int*)Utils::malloc(sizeof(int));

  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu->number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(reset_boundaries, dim_grid, threads_per_block, (nodes_a, nodes_b));

  #ifdef SHANCHEN
  // TODO FIXME: 
  /* We must add shan-chen forces, which are zero only if the densities are uniform*/
  #endif

  /** calc of velocitydensities from given parameters and initialize the Node_Force array with zero */
  KERNELCALL(reinit_node_force, dim_grid, threads_per_block, (node_f));
  KERNELCALL(calc_n_from_rho_j_pi, dim_grid, threads_per_block, (nodes_a, device_rho_v, node_f, gpu_check));
 
  intflag = 1;
  current_nodes = &nodes_a;
  h_gpu_check[0] = 0;
  cuda_safe_mem(hipMemcpy(h_gpu_check, gpu_check, sizeof(int), hipMemcpyDeviceToHost));
//fprintf(stderr, "initialization of lb gpu code %i\n", lbpar_gpu->number_of_nodes);
  hipDeviceSynchronize();

#if __CUDA_ARCH__ >= 200
  if(!h_gpu_check[0])
  {
    fprintf(stderr, "initialization of lb gpu code failed! \n");
    errexit();
  }
#endif
}

/** reinitialization for the lb gpu fluid called from host
 * @param *lbpar_gpu  Pointer to parameters to setup the lb field
*/
void lb_reinit_GPU(LB_parameters_gpu *lbpar_gpu){

  /**write parameters in const memory*/
  cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu, sizeof(LB_parameters_gpu)));
  
  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu->number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  /** calc of velocity densities from given parameters and initialize the Node_Force array with zero */
  KERNELCALL(calc_n_from_rho_j_pi, dim_grid, threads_per_block, (nodes_a, device_rho_v, node_f, gpu_check));
}

void lb_realloc_particles_GPU_leftovers(LB_parameters_gpu *lbpar_gpu){

  //copy parameters, especially number of parts to gpu mem
  cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu, sizeof(LB_parameters_gpu)));
}

#ifdef LB_BOUNDARIES_GPU
/** setup and call boundaries from the host
 * @param host_n_lb_boundaries  number of LB boundaries
 * @param number_of_boundnodes  number of boundnodes
 * @param host_boundary_node_list     The indices of the boundary nodes
 * @param host_boundary_index_list    The flag representing the corresponding boundary
 * @param host_lb_boundary_velocity   The constant velocity at the boundary, set by the user (Input)
*/
void lb_init_boundaries_GPU(int host_n_lb_boundaries, int number_of_boundnodes, int *host_boundary_node_list, int* host_boundary_index_list, float* host_lb_boundary_velocity){
  if (this_node != 0) return;
  
  size_of_boundindex = number_of_boundnodes*sizeof(int);
  cuda_safe_mem(hipMalloc((void**)&boundary_node_list, size_of_boundindex));
  cuda_safe_mem(hipMalloc((void**)&boundary_index_list, size_of_boundindex));
  cuda_safe_mem(hipMemcpy(boundary_index_list, host_boundary_index_list, size_of_boundindex, hipMemcpyHostToDevice));
  cuda_safe_mem(hipMemcpy(boundary_node_list, host_boundary_node_list, size_of_boundindex, hipMemcpyHostToDevice));
  cuda_safe_mem(hipMalloc((void**)&lb_boundary_force   , 3*host_n_lb_boundaries*sizeof(float)));
  cuda_safe_mem(hipMalloc((void**)&lb_boundary_velocity, 3*host_n_lb_boundaries*sizeof(float)));
  cuda_safe_mem(hipMemcpy(lb_boundary_velocity, host_lb_boundary_velocity, 3*LBBoundaries::lbboundaries.size()*sizeof(float), hipMemcpyHostToDevice));

  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(reset_boundaries, dim_grid, threads_per_block, (nodes_a, nodes_b));

  if (LBBoundaries::lbboundaries.size() == 0 && !pdb_boundary_lattice)
  {
    hipDeviceSynchronize();
    return;
  }

  if(number_of_boundnodes == 0)
  {
    fprintf(stderr, "WARNING: boundary cmd executed but no boundary node found!\n");
  }
  else
  {
    int threads_per_block_bound = 64;
    int blocks_per_grid_bound_y = 4;
    int blocks_per_grid_bound_x = (number_of_boundnodes + threads_per_block_bound * blocks_per_grid_bound_y - 1) /(threads_per_block_bound * blocks_per_grid_bound_y);
    dim3 dim_grid_bound = make_uint3(blocks_per_grid_bound_x, blocks_per_grid_bound_y, 1);

    KERNELCALL(init_boundaries, dim_grid_bound, threads_per_block_bound, (boundary_node_list, boundary_index_list, number_of_boundnodes, nodes_a, nodes_b));
  }

  hipDeviceSynchronize();
}
#endif
/**setup and call extern single node force initialization from the host
 * @param *lbpar_gpu    Pointer to host parameter struct
*/
void lb_reinit_extern_nodeforce_GPU(LB_parameters_gpu *lbpar_gpu){

  cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu, sizeof(LB_parameters_gpu))); 

  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu->number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(reinit_node_force, dim_grid, threads_per_block, (node_f));

}
/**setup and call extern single node force initialization from the host
 * @param n_extern_nodeforces       number of nodes on which the external force has to be applied
 * @param *host_extern_nodeforces   Pointer to the host extern node forces
 * @param *lbpar_gpu                Pointer to host parameter struct
*/
void lb_init_extern_nodeforces_GPU(int n_extern_nodeforces, LB_extern_nodeforce_gpu *host_extern_nodeforces, LB_parameters_gpu *lbpar_gpu){

  size_of_extern_nodeforces = n_extern_nodeforces*sizeof(LB_extern_nodeforce_gpu);
  cuda_safe_mem(hipMalloc((void**)&extern_nodeforces, size_of_extern_nodeforces));
  cuda_safe_mem(hipMemcpy(extern_nodeforces, host_extern_nodeforces, size_of_extern_nodeforces, hipMemcpyHostToDevice));

  if(lbpar_gpu->external_force == 0)
    cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu, sizeof(LB_parameters_gpu))); 

  int threads_per_block_exf = 64;
  int blocks_per_grid_exf_y = 4;
  int blocks_per_grid_exf_x = (n_extern_nodeforces + threads_per_block_exf * blocks_per_grid_exf_y - 1) / 
                              (threads_per_block_exf * blocks_per_grid_exf_y);
  dim3 dim_grid_exf = make_uint3(blocks_per_grid_exf_x, blocks_per_grid_exf_y, 1);

  KERNELCALL(init_extern_nodeforces, dim_grid_exf, threads_per_block_exf, (n_extern_nodeforces, extern_nodeforces, node_f));
  hipFree(extern_nodeforces);
}

/**setup and call particle kernel from the host
*/
void lb_calc_particle_lattice_ia_gpu(){
  if (lbpar_gpu.number_of_particles) 
  {
    /** call of the particle kernel */
    /** values for the particle kernel */
    int threads_per_block_particles = 64;
    int blocks_per_grid_particles_y = 4;
    int blocks_per_grid_particles_x = (lbpar_gpu.number_of_particles + threads_per_block_particles * blocks_per_grid_particles_y - 1) / 
                                      (threads_per_block_particles * blocks_per_grid_particles_y);
    dim3 dim_grid_particles = make_uint3(blocks_per_grid_particles_x, blocks_per_grid_particles_y, 1);

    if ( lbpar_gpu.lb_couple_switch & LB_COUPLE_TWO_POINT )
    {
      KERNELCALL( calc_fluid_particle_ia, dim_grid_particles, threads_per_block_particles, 
                  ( *current_nodes, gpu_get_particle_pointer(), 
                    gpu_get_particle_force_pointer(), gpu_get_fluid_composition_pointer(),
                    node_f, gpu_get_particle_seed_pointer(), device_rho_v )
                );
    }
    else { /** only other option is the three point coupling scheme */
#ifdef SHANCHEN
#if __CUDA_ARCH__ >= 200
      fprintf (stderr, "The three point particle coupling is not currently compatible with the Shan-Chen implementation of the LB\n");
      errexit(); 
#endif
#endif
      KERNELCALL( calc_fluid_particle_ia_three_point_couple, dim_grid_particles, threads_per_block_particles,
                   ( *current_nodes, gpu_get_particle_pointer(),
                     gpu_get_particle_force_pointer(), node_f,
                     gpu_get_particle_seed_pointer(), device_rho_v )
                );
    }
  }
}

/** setup and call kernel for getting macroscopic fluid values of all nodes
 * @param *host_values struct to save the gpu values
*/
void lb_get_values_GPU(LB_rho_v_pi_gpu *host_values){

  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) / 
                          (threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL( get_mesoscopic_values_in_MD_units, dim_grid, threads_per_block,
              ( *current_nodes, print_rho_v_pi, device_rho_v, node_f ) );
  cuda_safe_mem( hipMemcpy( host_values, print_rho_v_pi, size_of_rho_v_pi, hipMemcpyDeviceToHost ) );

}

/** get all the boundary flags for all nodes
 *  @param host_bound_array here go the values of the boundary flag
 */
void lb_get_boundary_flags_GPU(unsigned int* host_bound_array){
   
  unsigned int* device_bound_array;
  cuda_safe_mem(hipMalloc((void**)&device_bound_array, lbpar_gpu.number_of_nodes*sizeof(unsigned int)));
  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) / (threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(lb_get_boundaries, dim_grid, threads_per_block, (*current_nodes, device_bound_array));

  cuda_safe_mem(hipMemcpy(host_bound_array, device_bound_array, lbpar_gpu.number_of_nodes*sizeof(unsigned int), hipMemcpyDeviceToHost));

  hipFree(device_bound_array);

}

/** setup and call kernel for getting macroscopic fluid values of a single node*/
void lb_print_node_GPU(int single_nodeindex, LB_rho_v_pi_gpu *host_print_values){ 
      
  LB_rho_v_pi_gpu *device_print_values;
  cuda_safe_mem(hipMalloc((void**)&device_print_values, sizeof(LB_rho_v_pi_gpu)));
  int threads_per_block_print = 1;
  int blocks_per_grid_print_y = 1;
  int blocks_per_grid_print_x = 1;
  dim3 dim_grid_print = make_uint3(blocks_per_grid_print_x, blocks_per_grid_print_y, 1);

  KERNELCALL(lb_print_node, dim_grid_print, threads_per_block_print, (single_nodeindex, device_print_values, *current_nodes, device_rho_v, node_f));

  cuda_safe_mem(hipMemcpy(host_print_values, device_print_values, sizeof(LB_rho_v_pi_gpu), hipMemcpyDeviceToHost));
  hipFree(device_print_values);

}

/** setup and call kernel to calculate the total momentum of the hole fluid
 * @param *mass value of the mass calcutated on the GPU
*/
void lb_calc_fluid_mass_GPU(double* mass){

  float* tot_mass;
  float cpu_mass =  0.0f ;
  cuda_safe_mem(hipMalloc((void**)&tot_mass, sizeof(float)));
  cuda_safe_mem(hipMemcpy(tot_mass, &cpu_mass, sizeof(float), hipMemcpyHostToDevice));

  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(calc_mass, dim_grid, threads_per_block,(*current_nodes, tot_mass));

  cuda_safe_mem(hipMemcpy(&cpu_mass, tot_mass, sizeof(float), hipMemcpyDeviceToHost));
  
  hipFree(tot_mass);
  mass[0] = (double)(cpu_mass);
}

/** setup and call kernel to calculate the total momentum of the whole fluid
 *  @param host_mom value of the momentum calcutated on the GPU
 */
void lb_calc_fluid_momentum_GPU(double* host_mom){

  float* tot_momentum;
  float host_momentum[3] = { 0.0f, 0.0f, 0.0f};
  cuda_safe_mem(hipMalloc((void**)&tot_momentum, 3*sizeof(float)));
  cuda_safe_mem(hipMemcpy(tot_momentum, host_momentum, 3*sizeof(float), hipMemcpyHostToDevice));

  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(momentum, dim_grid, threads_per_block,(*current_nodes, device_rho_v, node_f, tot_momentum));
  
  cuda_safe_mem(hipMemcpy(host_momentum, tot_momentum, 3*sizeof(float), hipMemcpyDeviceToHost));
  
  hipFree(tot_momentum);
  host_mom[0] = (double)(host_momentum[0]* lbpar_gpu.agrid/lbpar_gpu.tau);
  host_mom[1] = (double)(host_momentum[1]* lbpar_gpu.agrid/lbpar_gpu.tau);
  host_mom[2] = (double)(host_momentum[2]* lbpar_gpu.agrid/lbpar_gpu.tau);
}

/** setup and call kernel to remove the net momentum of the whole fluid
 */
void lb_remove_fluid_momentum_GPU(void){
  float* tot_momentum;
  float host_momentum[3] = { 0.0f, 0.0f, 0.0f};
  cuda_safe_mem(hipMalloc((void**)&tot_momentum, 3*sizeof(float)));
  cuda_safe_mem(hipMemcpy(tot_momentum, host_momentum, 3*sizeof(float), hipMemcpyHostToDevice));

  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(momentum, dim_grid, threads_per_block,(*current_nodes, device_rho_v, node_f, tot_momentum));
  
  cuda_safe_mem(hipMemcpy(host_momentum, tot_momentum, 3*sizeof(float), hipMemcpyDeviceToHost));

  KERNELCALL(remove_momentum, dim_grid, threads_per_block,(*current_nodes, device_rho_v, node_f, tot_momentum));
  
  hipFree(tot_momentum);
}


/** setup and call kernel to calculate the temperature of the hole fluid
 *  @param host_temp value of the temperatur calcutated on the GPU
*/
void lb_calc_fluid_temperature_GPU(double* host_temp){

  int host_number_of_non_boundary_nodes = 0;
  int *device_number_of_non_boundary_nodes;
  cuda_safe_mem(hipMalloc((void**)&device_number_of_non_boundary_nodes, sizeof(int)));
  cuda_safe_mem(hipMemcpy(device_number_of_non_boundary_nodes, &host_number_of_non_boundary_nodes, sizeof(int), hipMemcpyHostToDevice));

  float host_jsquared = 0.0f;
  float* device_jsquared;
  cuda_safe_mem(hipMalloc((void**)&device_jsquared, sizeof(float)));
  cuda_safe_mem(hipMemcpy(device_jsquared, &host_jsquared, sizeof(float), hipMemcpyHostToDevice));

  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(temperature, dim_grid, threads_per_block,(*current_nodes, device_jsquared, device_number_of_non_boundary_nodes));

  cuda_safe_mem(hipMemcpy(&host_number_of_non_boundary_nodes, device_number_of_non_boundary_nodes, sizeof(int), hipMemcpyDeviceToHost));
  cuda_safe_mem(hipMemcpy(&host_jsquared, device_jsquared, sizeof(float), hipMemcpyDeviceToHost));

  // TODO: check that temperature calculation is properly implemented for shanchen
  *host_temp=0;

  #pragma unroll
  for(int ii=0;ii<LB_COMPONENTS;++ii)
  { 
      *host_temp += (double)(host_jsquared*1./(3.0f*lbpar_gpu.rho[ii]*host_number_of_non_boundary_nodes*lbpar_gpu.tau*lbpar_gpu.tau*lbpar_gpu.agrid));
  }
}


#ifdef SHANCHEN
void lb_calc_shanchen_GPU(){
  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

#ifdef LB_BOUNDARIES_GPU
  if (LBBoundaries::lbboundaries.size() != 0)
  {
    KERNELCALL(lb_shanchen_set_boundaries, dim_grid, threads_per_block,(*current_nodes));
    hipDeviceSynchronize();
  }
#endif
  KERNELCALL(lb_shanchen_GPU, dim_grid, threads_per_block,(*current_nodes, node_f));
}

#endif // SHANCHEN

/** setup and call kernel for getting macroscopic fluid values of all nodes
 * @param *host_checkpoint_vd struct to save the gpu populations
 * @param *host_checkpoint_seed struct to save the nodes' seeds for the lb on the gpu
 * @param *host_checkpoint_boundary struct to save the boundary nodes
 * @param *host_checkpoint_force struct to save the forces on the nodes
 */
void lb_save_checkpoint_GPU(float *host_checkpoint_vd, unsigned int *host_checkpoint_seed, unsigned int *host_checkpoint_boundary, lbForceFloat *host_checkpoint_force){

  cuda_safe_mem(hipMemcpy(host_checkpoint_vd, current_nodes->vd, lbpar_gpu.number_of_nodes * 19 * sizeof(float), hipMemcpyDeviceToHost));
  cuda_safe_mem(hipMemcpy(host_checkpoint_seed, current_nodes->seed, lbpar_gpu.number_of_nodes * sizeof(unsigned int), hipMemcpyDeviceToHost));
  cuda_safe_mem(hipMemcpy(host_checkpoint_boundary, current_nodes->boundary, lbpar_gpu.number_of_nodes * sizeof(unsigned int), hipMemcpyDeviceToHost));
  cuda_safe_mem(hipMemcpy(host_checkpoint_force, node_f.force, lbpar_gpu.number_of_nodes * 3 * sizeof(lbForceFloat), hipMemcpyDeviceToHost));

}

/** setup and call kernel for setting macroscopic fluid values of all nodes
 * @param *host_checkpoint_vd struct to save the gpu populations
 * @param *host_checkpoint_seed struct to save the nodes' seeds for the lb on the gpu
 * @param *host_checkpoint_boundary struct to save the boundary nodes
 * @param *host_checkpoint_force struct to save the forces on the nodes
*/
void lb_load_checkpoint_GPU(float *host_checkpoint_vd, unsigned int *host_checkpoint_seed, unsigned int *host_checkpoint_boundary, lbForceFloat *host_checkpoint_force){

  current_nodes = &nodes_a;
  intflag = 1;

  cuda_safe_mem(hipMemcpy(current_nodes->vd, host_checkpoint_vd, lbpar_gpu.number_of_nodes * 19 * sizeof(float), hipMemcpyHostToDevice));

  cuda_safe_mem(hipMemcpy(current_nodes->seed, host_checkpoint_seed, lbpar_gpu.number_of_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
  cuda_safe_mem(hipMemcpy(current_nodes->boundary, host_checkpoint_boundary, lbpar_gpu.number_of_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
  cuda_safe_mem(hipMemcpy(node_f.force, host_checkpoint_force, lbpar_gpu.number_of_nodes * 3 * sizeof(lbForceFloat), hipMemcpyHostToDevice));
}

/** setup and call kernel to get the boundary flag of a single node
 *  @param single_nodeindex number of the node to get the flag for
 *  @param host_flag her goes the value of the boundary flag
 */
void lb_get_boundary_flag_GPU(int single_nodeindex, unsigned int* host_flag){
   
  unsigned int* device_flag;
  cuda_safe_mem(hipMalloc((void**)&device_flag, sizeof(unsigned int)));
  int threads_per_block_flag = 1;
  int blocks_per_grid_flag_y = 1;
  int blocks_per_grid_flag_x = 1;
  dim3 dim_grid_flag = make_uint3(blocks_per_grid_flag_x, blocks_per_grid_flag_y, 1);

  KERNELCALL(lb_get_boundary_flag, dim_grid_flag, threads_per_block_flag, (single_nodeindex, device_flag, *current_nodes));

  cuda_safe_mem(hipMemcpy(host_flag, device_flag, sizeof(unsigned int), hipMemcpyDeviceToHost));

  hipFree(device_flag);
}

/** set the density at a single node
 *  @param single_nodeindex the node to set the velocity for 
 *  @param *host_rho the density to set
 */
void lb_set_node_rho_GPU(int single_nodeindex, float* host_rho){
   
  float* device_rho;
  cuda_safe_mem(hipMalloc((void**)&device_rho, LB_COMPONENTS*sizeof(float)));
  cuda_safe_mem(hipMemcpy(device_rho, host_rho, LB_COMPONENTS*sizeof(float), hipMemcpyHostToDevice));
  int threads_per_block_flag = 1;
  int blocks_per_grid_flag_y = 1;
  int blocks_per_grid_flag_x = 1;
  dim3 dim_grid_flag = make_uint3(blocks_per_grid_flag_x, blocks_per_grid_flag_y, 1);
  KERNELCALL(set_rho, dim_grid_flag, threads_per_block_flag, (*current_nodes, device_rho_v, single_nodeindex, device_rho)); 
  hipFree(device_rho);
}

/** set the net velocity at a single node
 *  @param single_nodeindex the node to set the velocity for 
 *  @param host_velocity the velocity to set
 */
void lb_set_node_velocity_GPU(int single_nodeindex, float* host_velocity){
   
  float* device_velocity;
  cuda_safe_mem(hipMalloc((void**)&device_velocity, 3*sizeof(float)));
  cuda_safe_mem(hipMemcpy(device_velocity, host_velocity, 3*sizeof(float), hipMemcpyHostToDevice));
  int threads_per_block_flag = 1;
  int blocks_per_grid_flag_y = 1;
  int blocks_per_grid_flag_x = 1;
  dim3 dim_grid_flag = make_uint3(blocks_per_grid_flag_x, blocks_per_grid_flag_y, 1);

  KERNELCALL(set_u_from_rho_v_pi, dim_grid_flag, threads_per_block_flag, (*current_nodes, single_nodeindex, device_velocity, device_rho_v, node_f));

  hipFree(device_velocity);
}

/** reinit of params 
 * @param *lbpar_gpu struct containing the paramters of the fluid
*/
void reinit_parameters_GPU(LB_parameters_gpu *lbpar_gpu){
  /**write parameters in const memory*/
  cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu, sizeof(LB_parameters_gpu)));
}

/**integration kernel for the lb gpu fluid update called from host */
void lb_integrate_GPU() {

  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

#ifdef LB_BOUNDARIES_GPU
  if (LBBoundaries::lbboundaries.size() > 0)
  {
    cuda_safe_mem(hipMemset( lb_boundary_force, 0, 3*LBBoundaries::lbboundaries.size()*sizeof(float)));
  }
#endif

  /**call of fluid step*/
  /* NOTE: if pi is needed at every integration step, one should call an extended version 
           of the integrate kernel, or pass also device_rho_v_pi and make sure that either 
           it or device_rho_v are nullptr depending on extended_values_flag */ 
  if (intflag == 1)
  {
    KERNELCALL(integrate, dim_grid, threads_per_block, (nodes_a, nodes_b, device_rho_v, node_f, lb_ek_parameters_gpu));
    current_nodes = &nodes_b;
    intflag = 0;
  }
  else
  {
    KERNELCALL(integrate, dim_grid, threads_per_block, (nodes_b, nodes_a, device_rho_v, node_f, lb_ek_parameters_gpu));
    current_nodes = &nodes_a;
    intflag = 1;
  }

#ifdef LB_BOUNDARIES_GPU
  if (LBBoundaries::lbboundaries.size() > 0)
    {
      KERNELCALL(apply_boundaries, dim_grid, threads_per_block, (*current_nodes, lb_boundary_velocity, lb_boundary_force));
    }
#endif
}

void lb_gpu_get_boundary_forces(double* forces) {
#ifdef LB_BOUNDARIES_GPU
  float* temp = (float*) Utils::malloc(3*LBBoundaries::lbboundaries.size()*sizeof(float));
  cuda_safe_mem(hipMemcpy(temp, lb_boundary_force, 3*LBBoundaries::lbboundaries.size()*sizeof(float), hipMemcpyDeviceToHost));

  for (int i =0; i<3*LBBoundaries::lbboundaries.size(); i++)
  {
    forces[i]=(double)temp[i];
  }
  free(temp);
#endif
}

struct lb_lbfluid_mass_of_particle
{
  __device__ float operator()(CUDA_particle_data particle) const
  {
#ifdef MASS
    return particle.mass;
#else
    return 1.;
#endif
  };
};

void lb_lbfluid_remove_total_momentum()
{
  // calculate momentum of fluid and particles
  float total_momentum[3] = { 0.0f, 0.0f, 0.0f };
  lb_lbfluid_calc_linear_momentum(total_momentum, /*include_particles*/ 1, /*include_lbfluid*/ 1);

  thrust::device_ptr<CUDA_particle_data> ptr(gpu_get_particle_pointer());
  float particles_mass = thrust::transform_reduce(
    ptr,
    ptr + lbpar_gpu.number_of_particles,
    lb_lbfluid_mass_of_particle(),
    0.0f,
    thrust::plus<float>());

  // lb_calc_fluid_mass_GPU has to be called with double but we don't
  // want narrowing warnings, that's why we narrow it down by hand.
  double lb_calc_fluid_mass_res;
  lb_calc_fluid_mass_GPU( &lb_calc_fluid_mass_res );
  float fluid_mass = lb_calc_fluid_mass_res;

  /* Momentum fraction of the particles */
  auto const part_frac = particles_mass / (fluid_mass + particles_mass);
  /* Mometum per particle */
  float momentum_particles[3] = {
    -total_momentum[0]*part_frac,
    -total_momentum[1]*part_frac,
    -total_momentum[2]*part_frac
  };

  auto const fluid_frac = fluid_mass / (fluid_mass + particles_mass);
  float momentum_fluid[3] = {
    -total_momentum[0]*fluid_frac,
    -total_momentum[1]*fluid_frac,
    -total_momentum[2]*fluid_frac
  };

  lb_lbfluid_particles_add_momentum( momentum_particles );
  lb_lbfluid_fluid_add_momentum( momentum_fluid );
}

__global__ void lb_lbfluid_fluid_add_momentum_kernel(
  float momentum[3],
  LB_nodes_gpu n_a,
  LB_node_force_gpu node_f,
  LB_rho_v_gpu *d_v)
{
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int number_of_nodes = para.number_of_nodes;
#ifdef LB_BOUNDARIES_GPU
  number_of_nodes -= para.number_of_boundnodes;
#endif
  if( index < para.number_of_nodes )
  {
    if( n_a.boundary[index] == 0 )
    {
      float force_factor=powf(para.agrid,2)*para.tau*para.tau;
      for(int i = 0 ; i < LB_COMPONENTS ; ++i )
      {
        // add force density onto each node (momentum / time_step / Volume)
        node_f.force[(0+i*3)*para.number_of_nodes + index] += momentum[0] / para.tau / (number_of_nodes * powf(para.agrid,3)) * force_factor;
        node_f.force[(1+i*3)*para.number_of_nodes + index] += momentum[1] / para.tau / (number_of_nodes * powf(para.agrid,3)) * force_factor;
        node_f.force[(2+i*3)*para.number_of_nodes + index] += momentum[2] / para.tau / (number_of_nodes * powf(para.agrid,3)) * force_factor;
      }
    }
  }
}


void lb_lbfluid_fluid_add_momentum( float momentum_host[3] )
{
  float* momentum_device;
  cuda_safe_mem(hipMalloc((void**)&momentum_device,3*sizeof(float)));
  cuda_safe_mem(hipMemcpy(momentum_device, momentum_host, 3*sizeof(float), hipMemcpyHostToDevice));

  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1)/(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL( lb_lbfluid_fluid_add_momentum_kernel, dim_grid, threads_per_block, (momentum_device, *current_nodes, node_f, device_rho_v));
}


/**set the populations of a specific node on the GPU
 * @param n_a            Pointer to local node residing in array a (Input)
 * @param population     Pointer to new population (Input)
 * @param x              x-coordinate of node (Input)
 * @param y              y-coordinate of node (Input)
 * @param z              z-coordinate of node (Input)
 * @param c              LB component (for SHANCHEN) (Input)
*/
__global__ void lb_lbfluid_set_population_kernel(LB_nodes_gpu n_a, float population[LBQ], int x, int y, int z, int c)
{
  int xyz[3] = { x, y, z };
  int index = xyz_to_index( xyz );

  for (int i = 0; i < LBQ; ++i)
  {
    n_a.vd[( i + c*LBQ ) * para.number_of_nodes + index] = population[i];
  }
}


/**interface to set the populations of a specific node for the GPU
 * @param xyz            coordinates of node (Input)
 * @param population     Pointer to population (Input)
 * @param c              LB component (for SHANCHEN) (Input)
*/
void lb_lbfluid_set_population( int xyz[3], float population_host[LBQ], int c )
{
  float* population_device;
  cuda_safe_mem(hipMalloc((void**)&population_device,LBQ*sizeof(float)));
  cuda_safe_mem(hipMemcpy(population_device, population_host, LBQ*sizeof(float), hipMemcpyHostToDevice));

  dim3 dim_grid = make_uint3(1, 1, 1);
  KERNELCALL( lb_lbfluid_set_population_kernel, dim_grid, 1,
              (*current_nodes, population_device, xyz[0], xyz[1], xyz[2], c));

  cuda_safe_mem(hipFree(population_device));
}


/**get the populations of a specific node on the GPU
 * @param n_a            Pointer to local node residing in array a (Input)
 * @param population     Pointer to population (Output)
 * @param x              x-coordinate of node (Input)
 * @param y              y-coordinate of node (Input)
 * @param z              z-coordinate of node (Input)
 * @param c              LB component (for SHANCHEN) (Input)
*/
__global__ void lb_lbfluid_get_population_kernel(LB_nodes_gpu n_a, float population[LBQ], int x, int y, int z, int c)
{
  int xyz[3] = { x, y, z };
  int index = xyz_to_index( xyz );

  for (int i = 0; i < LBQ; ++i)
  {
    population[i] = n_a.vd[( i + c*LBQ ) * para.number_of_nodes + index];
  }
}


/**interface to get the populations of a specific node for the GPU
 * @param xyz            coordinates of node (Input)
 * @param population     Pointer to population (Output)
 * @param c              LB component (for SHANCHEN) (Input)
*/
void lb_lbfluid_get_population( int xyz[3], float population_host[LBQ], int c )
{
  float* population_device;
  cuda_safe_mem(hipMalloc((void**)&population_device,LBQ*sizeof(float)));

  dim3 dim_grid = make_uint3(1, 1, 1);
  KERNELCALL( lb_lbfluid_get_population_kernel, dim_grid, 1,
              (*current_nodes, population_device, xyz[0], xyz[1], xyz[2], c));

  cuda_safe_mem(hipMemcpy(population_host, population_device, LBQ*sizeof(float), hipMemcpyDeviceToHost));

  cuda_safe_mem(hipFree(population_device));
}

struct two_point_interpolation { 
    LB_nodes_gpu current_nodes_gpu;
    LB_rho_v_gpu *d_v_gpu;
    two_point_interpolation(LB_nodes_gpu _current_nodes_gpu, LB_rho_v_gpu *_d_v_gpu) : current_nodes_gpu(_current_nodes_gpu), d_v_gpu(_d_v_gpu) {};
	__device__ float3 operator()(const float3 &position) const {
        unsigned int node_index[8];
        float delta[8];
        float u[3];
        float mode[19*LB_COMPONENTS];
        float _position[3] = {position.x, position.y, position.z};
        interpolation_two_point_coupling(current_nodes_gpu, _position, node_index, mode, d_v_gpu, delta, u);
        return make_float3(u[0], u[1], u[2]);
	} 
};

void lb_lbfluid_get_interpolated_velocity_at_positions(double *positions, double *velocities, int length) {
    thrust::host_vector<float3> positions_host(length);
    for (int p=0; p < 3 * length; p+=3) {
        // Cast double coming from python to float.
        positions_host[p/3].x = static_cast<float>(positions[p]);
        positions_host[p/3].y = static_cast<float>(positions[p+1]);
        positions_host[p/3].z = static_cast<float>(positions[p+2]);
    }
    thrust::device_vector<float3> positions_device = positions_host;
    thrust::device_vector<float3> velocities_device(length);
    thrust::transform(positions_device.begin(), positions_device.end(), velocities_device.begin(), two_point_interpolation(*current_nodes, device_rho_v));
    thrust::host_vector<float3> velocities_host = velocities_device;
    int index = 0;
    for (auto v : velocities_host) {
        velocities[index] = static_cast<double>(v.x) * lbpar_gpu.agrid/lbpar_gpu.tau;
        velocities[index+1] = static_cast<double>(v.y) * lbpar_gpu.agrid/lbpar_gpu.tau;
        velocities[index+2] = static_cast<double>(v.z) * lbpar_gpu.agrid/lbpar_gpu.tau;
        index += 3;
    }
}

#endif /* LB_GPU */
