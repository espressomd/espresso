/*
 * Copyright (C) 2010-2022 The ESPResSo project
 *
 * This file is part of ESPResSo.
 *
 * ESPResSo is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * ESPResSo is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>

#include "init.hpp"
#include "utils.cuh"

#include <utils/constants.hpp>

#include <cstring>
#include <string>

#if defined(OMPI_MPI_H) || defined(_MPI_H)
#error CU-file includes mpi.h! This should not happen!
#endif

#ifdef CUDA

/** \name minimally required compute capability. */
/**@{*/
static const int computeCapabilityMinMajor = 3;
static const int computeCapabilityMinMinor = 0;
/**@}*/

void cuda_init() { CUDA_CHECK(hipStreamCreate(&stream[0])) }

int cuda_get_n_gpus() {
  int deviceCount;
  CUDA_CHECK(hipGetDeviceCount(&deviceCount))
  return deviceCount;
}

int cuda_check_gpu_compute_capability(int dev) {
  hipDeviceProp_t deviceProp;
  CUDA_CHECK(hipGetDeviceProperties(&deviceProp, dev))
  if (deviceProp.major < computeCapabilityMinMajor ||
      (deviceProp.major == computeCapabilityMinMajor &&
       deviceProp.minor < computeCapabilityMinMinor)) {
    return ES_ERROR;
  }
  return ES_OK;
}

/**
 * @brief Safely copy the device name and pad the string with null characters.
 */
static void cuda_copy_gpu_name(char *const name, hipDeviceProp_t const &prop) {
  char buffer[256] = {'\0'};
  std::strncpy(buffer, prop.name, 256);
  name[255] = '\0';
  std::strncpy(name, buffer, 256);
}

void cuda_get_gpu_name(int dev, char *const name) {
  hipDeviceProp_t deviceProp;
  CUDA_CHECK(hipGetDeviceProperties(&deviceProp, dev))
  cuda_copy_gpu_name(name, deviceProp);
}

EspressoGpuDevice cuda_get_device_props(const int dev) {
  hipDeviceProp_t deviceProp;
  CUDA_CHECK(hipGetDeviceProperties(&deviceProp, dev))
  EspressoGpuDevice device{dev,
                           "",
                           "",
                           -1,
                           deviceProp.major,
                           deviceProp.minor,
                           deviceProp.totalGlobalMem,
                           deviceProp.multiProcessorCount};
  cuda_copy_gpu_name(device.name, deviceProp);
  return device;
}

void cuda_set_device(int dev) {
  CUDA_CHECK(hipSetDevice(dev))
  CUDA_CHECK(hipStreamDestroy(stream[0]))
  CUDA_CHECK(hipStreamCreate(&stream[0]))
}

int cuda_get_device() {
  int dev;
  CUDA_CHECK(hipGetDevice(&dev))
  return dev;
}

int cuda_test_device_access() {
  int *d = nullptr;
  int h = 42;
  hipError_t err;

  err = hipMalloc((void **)&d, sizeof(int));
  if (err != hipSuccess) {
    throw cuda_runtime_error_cuda(err);
  }
  err = hipMemcpy(d, &h, sizeof(int), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    hipFree(d);
    throw cuda_runtime_error_cuda(err);
  }
  h = 0;
  err = hipMemcpy(&h, d, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(d);
  if (err != hipSuccess) {
    throw cuda_runtime_error_cuda(err);
  }
  if (h != 42) {
    return ES_ERROR;
  }
  return ES_OK;
}

void cuda_check_device() {
  if (cuda_get_n_gpus() == 0) {
    throw cuda_runtime_error("No GPU was found.");
  }
  auto const devID = cuda_get_device();
  auto const compute_capability = cuda_check_gpu_compute_capability(devID);
  auto const communication_test = cuda_test_device_access();
  if (compute_capability != ES_OK or communication_test != ES_OK) {
    throw cuda_runtime_error("CUDA device " + std::to_string(devID) +
                             " is not capable of running ESPResSo.");
  }
}

#endif /* defined(CUDA) */
