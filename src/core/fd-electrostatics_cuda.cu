#include "hip/hip_runtime.h"
// TODO: throw exceptions upon errors initialization

#include <hip/hip_runtime.h>
#include <cuda_utils.hpp>
#include <hipfft/hipfft.h>
#include <fd-electrostatics.hpp>
#include <string>
//#include <cuda_interface.hpp>
#include <cstdio>

#if defined(OMPI_MPI_H) || defined(_MPI_H)
#error CU-file includes mpi.h! This should not happen!
#endif

__global__ void createGreensfcn();
__global__ void multiplyGreensfcn(hipfftComplex *charge_potential);

__device__ __constant__ FdElectrostatics::Parameters fde_parameters_gpu;

__device__ unsigned int fde_getThreadIndex() {

  return blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x +
         threadIdx.x;
}

__device__ hipfftReal fde_getNode(int x, int y, int z)
{
  hipfftReal* field = reinterpret_cast<hipfftReal*>(fde_parameters_gpu.charge_potential);
  return field[fde_parameters_gpu.dim_y*fde_parameters_gpu.dim_x_padded*z + fde_parameters_gpu.dim_x_padded*y + x];
}

__device__ void fde_setNode(int x, int y, int z, hipfftReal value)
{
  hipfftReal* field = reinterpret_cast<hipfftReal*>(fde_parameters_gpu.charge_potential);
  field[fde_parameters_gpu.dim_y*fde_parameters_gpu.dim_x_padded*z + fde_parameters_gpu.dim_x_padded*y + x] = value;
}

__device__ hipfftReal fde_getNode(int i)
{
  int x  = i % fde_parameters_gpu.dim_x_padded;
  i /= fde_parameters_gpu.dim_x_padded;
  int y  = i % fde_parameters_gpu.dim_y;
  int z  = i / fde_parameters_gpu.dim_y;
  return fde_getNode(x, y, z);
}

__device__ void fde_setNode(int i, hipfftReal value)
{
  int x  = i % fde_parameters_gpu.dim_x_padded;
  i /= fde_parameters_gpu.dim_x_padded;
  int y  = i % fde_parameters_gpu.dim_y;
  int z  = i / fde_parameters_gpu.dim_y;
  fde_setNode(x, y, z, value);
}

FdElectrostatics::~FdElectrostatics() {
  hipfftDestroy(plan_ifft);
  hipfftDestroy(plan_fft);

  void *symbol;
  hipGetSymbolAddress(&symbol, HIP_SYMBOL("fde_parameters_gpu"));
  cuda_safe_mem(hipFree(symbol));

  cuda_safe_mem(hipFree(parameters.greensfcn));
  cuda_safe_mem(hipFree(parameters.charge_potential));
}

FdElectrostatics::FdElectrostatics(InputParameters inputParameters,
                                   hipStream_t stream)
    : parameters(inputParameters), hip_stream(stream) {
  cuda_safe_mem(hipMalloc((void **)&parameters.charge_potential,
                           sizeof(hipfftComplex) * parameters.dim_z *
                               parameters.dim_y * (parameters.dim_x / 2 + 1)));

  cuda_safe_mem(hipMalloc((void **)&parameters.greensfcn,
                           sizeof(hipfftReal) * parameters.dim_z *
                               parameters.dim_y * (parameters.dim_x / 2 + 1)));

  if (hipGetLastError() != hipSuccess) {
    throw "Failed to allocate\n";
  }

  cuda_safe_mem(
      hipMemcpyToSymbol(HIP_SYMBOL(fde_parameters_gpu), &parameters, sizeof(Parameters)));

  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x =
      (parameters.dim_z * parameters.dim_y * (parameters.dim_x / 2 + 1) +
       threads_per_block * blocks_per_grid_y - 1) /
      (threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);
  KERNELCALL_stream(createGreensfcn, dim_grid, threads_per_block, stream, ());

  /* create 3D FFT plans */

  if (hipfftPlan3d(&plan_fft, parameters.dim_z, parameters.dim_y,
                  parameters.dim_x, HIPFFT_R2C) != HIPFFT_SUCCESS) {
    throw std::string("Unable to create fft plan");
  }

  if (hipfftSetStream(plan_fft, hip_stream) != HIPFFT_SUCCESS) {
    throw std::string("Unable to assign FFT to cuda stream");
  }

  if (hipfftPlan3d(&plan_ifft, parameters.dim_z, parameters.dim_y,
                  parameters.dim_x, HIPFFT_C2R) != HIPFFT_SUCCESS) {
    throw std::string("Unable to create ifft plan");
  }

  if (hipfftSetStream(plan_ifft, hip_stream) != HIPFFT_SUCCESS) {
    throw std::string("Unable to assign FFT to cuda stream");
  }

  initialized = true;
}

__global__ void createGreensfcn() {
  unsigned int index = fde_getThreadIndex();
  unsigned int tmp;
  unsigned int coord[3];

  coord[0] = index % (fde_parameters_gpu.dim_x / 2 + 1);
  tmp = index / (fde_parameters_gpu.dim_x / 2 + 1);
  coord[1] = tmp % fde_parameters_gpu.dim_y;
  coord[2] = tmp / fde_parameters_gpu.dim_y;

  if (index < fde_parameters_gpu.dim_z * fde_parameters_gpu.dim_y *
                  (fde_parameters_gpu.dim_x / 2 + 1)) {

    if (index == 0) {
      // setting 0th fourier mode to 0 enforces charge neutrality
      fde_parameters_gpu.greensfcn[index] = 0.0f;
    } else {
      fde_parameters_gpu.greensfcn[index] =
          -4.0f * PI_FLOAT * fde_parameters_gpu.prefactor *
          fde_parameters_gpu.agrid *
          fde_parameters_gpu.agrid * 0.5f /
          (cos(2.0f * PI_FLOAT * coord[0] /
               (hipfftReal)fde_parameters_gpu.dim_x) +
           cos(2.0f * PI_FLOAT * coord[1] /
               (hipfftReal)fde_parameters_gpu.dim_y) +
           cos(2.0f * PI_FLOAT * coord[2] /
               (hipfftReal)fde_parameters_gpu.dim_z) -
           3.0f) /
          (fde_parameters_gpu.dim_x * fde_parameters_gpu.dim_y *
           fde_parameters_gpu.dim_z);
    }

    // fde_parameters_gpu.greensfcn[index] = 0.0f; //TODO delete
  }
}

__global__ void multiplyGreensfcn(hipfftComplex *charge_potential) {

  unsigned int index = fde_getThreadIndex();

  if (index < fde_parameters_gpu.dim_z * fde_parameters_gpu.dim_y *
                  (fde_parameters_gpu.dim_x / 2 + 1)) {
    charge_potential[index].x *= fde_parameters_gpu.greensfcn[index];
    charge_potential[index].y *= fde_parameters_gpu.greensfcn[index];
  }
}

void FdElectrostatics::calculatePotential() {
  calculatePotential(parameters.charge_potential);
}

void FdElectrostatics::calculatePotential(hipfftComplex *charge_potential) {

  if (hipfftExecR2C(plan_fft, (hipfftReal *)charge_potential, charge_potential) !=
      HIPFFT_SUCCESS) {

    fprintf(stderr, "ERROR: Unable to execute FFT plan\n");
  }

  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x =
      (parameters.dim_z * parameters.dim_y * (parameters.dim_x / 2 + 1) +
       threads_per_block * blocks_per_grid_y - 1) /
      (threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(multiplyGreensfcn, dim_grid, threads_per_block,
             (charge_potential));

  if (hipfftExecC2R(plan_ifft, charge_potential,
                   (hipfftReal *)charge_potential) != HIPFFT_SUCCESS) {

    fprintf(stderr, "ERROR: Unable to execute iFFT plan\n");
  }
}

FdElectrostatics::Grid FdElectrostatics::getGrid() {
  Grid g = {(float *)parameters.charge_potential, parameters.dim_x,
            parameters.dim_y, parameters.dim_z, parameters.agrid};
  return g;
}
