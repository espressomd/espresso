#include "hip/hip_runtime.h"
/*
   Copyright (C) 2010-2018 The ESPResSo project

   This file is part of ESPResSo.

   ESPResSo is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.

   ESPResSo is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.

   You should have received a copy of the GNU General Public License
   along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "cuda_wrapper.hpp"

#include "config.hpp"
#include "debug.hpp"

#include "cuda_init.hpp"
#include "cuda_interface.hpp"
#include "cuda_utils.hpp"
#include "errorhandling.hpp"
#include "nonbonded_interactions/nonbonded_interaction_data.hpp"

#include <random>

#if defined(OMPI_MPI_H) || defined(_MPI_H)
#error CU-file includes mpi.h! This should not happen!
#endif

static CUDA_global_part_vars global_part_vars_host = {0, 0, 0};
__device__ __constant__ CUDA_global_part_vars global_part_vars_device[1];

/** struct for particle force */
static float *particle_forces_device = nullptr;
static float *particle_torques_device = nullptr;

/** struct for particle position and velocity */
static CUDA_particle_data *particle_data_device = nullptr;
/** struct for storing particle rn seed */
static CUDA_particle_seed *particle_seeds_device = nullptr;
/** struct for energies */
static CUDA_energy *energy_device = nullptr;

CUDA_particle_data *particle_data_host = nullptr;
std::vector<float> particle_forces_host;
CUDA_energy energy_host;

std::vector<float> particle_torques_host;
#ifdef ENGINE
std::vector<CUDA_v_cs> host_v_cs;
#endif

/**cuda streams for parallel computing on cpu and gpu */
hipStream_t stream[1];

hipError_t _err;
hipError_t CU_err;

void _cuda_safe_mem(hipError_t CU_err, const char *file, unsigned int line) {
  if (hipSuccess != CU_err) {
    fprintf(stderr, "Cuda Memory error at %s:%u.\n", file, line);
    printf("CUDA error: %s\n", hipGetErrorString(CU_err));
    if (CU_err == hipErrorInvalidValue)
      fprintf(stderr, "You may have tried to allocate zero memory at %s:%u.\n",
              file, line);
    errexit();
  } else {
    CU_err = hipGetLastError();
    if (CU_err != hipSuccess) {
      fprintf(stderr,
              "Error found during memory operation. Possibly however "
              "from an failed operation before. %s:%u.\n",
              file, line);
      printf("CUDA error: %s\n", hipGetErrorString(CU_err));
      if (CU_err == hipErrorInvalidValue)
        fprintf(stderr,
                "You may have tried to allocate zero memory before %s:%u.\n",
                file, line);
      errexit();
    }
  }
}

void _cuda_check_errors(const dim3 &block, const dim3 &grid,
                        const char *function, const char *file,
                        unsigned int line) {
/** If debugging is enabled, wait for Kernels to terminate before checking for
 * errors. This removes parallelism between host and device and should only be
 * enabled while debugging. */
#ifdef CUDA_DEBUG
  hipDeviceSynchronize();
#endif
  CU_err = hipGetLastError();
  if (CU_err != hipSuccess) {
    fprintf(stderr,
            "%d: error \"%s\" calling %s with dim %d %d %d, grid %d %d "
            "%d in %s:%u\n",
            this_node, hipGetErrorString(CU_err), function, block.x, block.y,
            block.z, grid.x, grid.y, grid.z, file, line);
    errexit();
  }
}

__device__ unsigned int getThreadIndex() {

  return blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x +
         threadIdx.x;
}

/** Kernel for the initialisation of the particle force array
 * @param[out] particle_forces_device    Local particle force
 * @param[out] particle_torques_device   Local particle torque
 * @param[out] particle_seeds_device     Particle random seed
 */
__global__ void init_particle_force(float *particle_forces_device,
                                    float *particle_torques_device,
                                    CUDA_particle_seed *particle_seeds_device) {

  unsigned int part_index = getThreadIndex();

  if (part_index < global_part_vars_device->number_of_particles) {
    particle_forces_device[3 * part_index + 0] = 0.0f;
    particle_forces_device[3 * part_index + 1] = 0.0f;
    particle_forces_device[3 * part_index + 2] = 0.0f;

#ifdef ROTATION
    particle_torques_device[3 * part_index] = 0.0f;
    particle_torques_device[3 * part_index + 1] = 0.0f;
    particle_torques_device[3 * part_index + 2] = 0.0f;
#endif

    particle_seeds_device[part_index].seed =
        global_part_vars_device->seed + part_index;
  }
}

/** Kernel for the initialisation of the particle force array
 * @param[out] particle_forces_device    Local particle force
 * @param[out] particle_torques_device   Local particle torque
 */
__global__ void reset_particle_force(float *particle_forces_device,
                                     float *particle_torques_device) {

  unsigned int part_index = getThreadIndex();

  if (part_index < global_part_vars_device->number_of_particles) {
    particle_forces_device[3 * part_index + 0] = 0.0f;
    particle_forces_device[3 * part_index + 1] = 0.0f;
    particle_forces_device[3 * part_index + 2] = 0.0f;
#ifdef ROTATION
    particle_torques_device[3 * part_index + 0] = 0.0f;
    particle_torques_device[3 * part_index + 1] = 0.0f;
    particle_torques_device[3 * part_index + 2] = 0.0f;
#endif
  }
}

/** change number of particles to be communicated to the GPU
 *  Note that in addition to calling this function the parameters must be
 * broadcast with either:
 * 1) cuda_bcast_global_part_params(); (when just being executed on the master
 * node) or
 * 2) MPI_Bcast(gpu_get_global_particle_vars_pointer_host(),
 * sizeof(CUDA_global_part_vars), MPI_BYTE, 0, comm_cart); (when executed on all
 * nodes)
 */
void gpu_change_number_of_part_to_comm() {
  // we only run the function if there are new particles which have been created
  // since the last call of this function

  if (global_part_vars_host.number_of_particles != n_part &&
      global_part_vars_host.communication_enabled == 1 && this_node == 0) {

    global_part_vars_host.seed = (unsigned int)std::random_device{}();
    global_part_vars_host.number_of_particles = n_part;

    cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(global_part_vars_device),
                                     &global_part_vars_host,
                                     sizeof(CUDA_global_part_vars)));

    // if the arrays exists free them to prevent memory leaks
    particle_forces_host.clear();
    if (particle_data_host) {
      cuda_safe_mem(hipHostFree(particle_data_host));
      particle_data_host = nullptr;
    }
    if (particle_forces_device) {
      hipFree(particle_forces_device);
      particle_forces_device = nullptr;
    }
    if (particle_data_device) {
      hipFree(particle_data_device);
      particle_data_device = nullptr;
    }
    if (particle_seeds_device) {
      cuda_safe_mem(hipFree(particle_seeds_device));
      particle_seeds_device = nullptr;
    }
#ifdef ENGINE
    host_v_cs.clear();
#endif
#ifdef ROTATION
    particle_torques_host.clear();
#endif

#ifdef ROTATION
    if (particle_torques_device) {
      cuda_safe_mem(hipFree(particle_torques_device));
      particle_torques_device = nullptr;
    }
#endif

    if (global_part_vars_host.number_of_particles) {

      /**pinned memory mode - use special function to get OS-pinned memory*/
      cuda_safe_mem(hipHostAlloc((void **)&particle_data_host,
                                  global_part_vars_host.number_of_particles *
                                      sizeof(CUDA_particle_data),
                                  hipHostMallocWriteCombined));
      particle_forces_host.resize(3 *
                                  global_part_vars_host.number_of_particles);
#ifdef ENGINE
      host_v_cs.resize(global_part_vars_host.number_of_particles);
#endif
#if (defined DIPOLES || defined ROTATION)
      particle_torques_host.resize(3 *
                                   global_part_vars_host.number_of_particles);
#endif

      cuda_safe_mem(hipMalloc((void **)&particle_forces_device,
                               3 * global_part_vars_host.number_of_particles *
                                   sizeof(float)));
#ifdef ROTATION
      cuda_safe_mem(hipMalloc((void **)&particle_torques_device,
                               3 * global_part_vars_host.number_of_particles *
                                   sizeof(float)));
#endif

      cuda_safe_mem(hipMalloc((void **)&particle_data_device,
                               global_part_vars_host.number_of_particles *
                                   sizeof(CUDA_particle_data)));
      cuda_safe_mem(hipMalloc((void **)&particle_seeds_device,
                               global_part_vars_host.number_of_particles *
                                   sizeof(CUDA_particle_seed)));

      /** values for the particle kernel */
      int threads_per_block_particles = 64;
      int blocks_per_grid_particles_y = 4;
      int blocks_per_grid_particles_x =
          (global_part_vars_host.number_of_particles +
           threads_per_block_particles * blocks_per_grid_particles_y - 1) /
          (threads_per_block_particles * blocks_per_grid_particles_y);
      dim3 dim_grid_particles = make_uint3(blocks_per_grid_particles_x,
                                           blocks_per_grid_particles_y, 1);

      KERNELCALL(init_particle_force, dim_grid_particles,
                 threads_per_block_particles, particle_forces_device,
                 particle_torques_device, particle_seeds_device);
    }
  }
}

/** setup and call particle reallocation from the host
 *  Note that in addition to calling this function the parameters must be
 * broadcast with either:
 * 1) cuda_bcast_global_part_params(); (when just being executed on the master
 * node) or
 * 2) MPI_Bcast(gpu_get_global_particle_vars_pointer_host(),
 * sizeof(CUDA_global_part_vars), MPI_BYTE, 0, comm_cart); (when executed on all
 * nodes)
 */
void gpu_init_particle_comm() {
  if (this_node == 0 && global_part_vars_host.communication_enabled == 0) {
    if (cuda_get_n_gpus() == -1) {
      runtimeErrorMsg()
          << "Unable to initialize CUDA as no sufficient GPU is available.";
      errexit();
    }
    if (cuda_get_n_gpus() > 1) {
      runtimeWarningMsg() << "More than one GPU detected, please note ESPResSo "
                             "uses device 0 by default regardless of usage or "
                             "capability. The GPU to be used can be modified "
                             "by setting System.cuda_init_handle.device.";
      if (cuda_check_gpu(0) != ES_OK) {
        runtimeWarningMsg()
            << "CUDA device 0 is not capable of running ESPResSo but is used "
               "by default. Espresso has detected a CUDA capable card but it "
               "is not the one used by ESPResSo by default. Please set the "
               "GPU to use by setting System.cuda_init_handle.device. A list "
               "of avalable GPUs is available through "
               "System.cuda_init_handle.device_list.";
      }
    }
  }
  global_part_vars_host.communication_enabled = 1;
  gpu_change_number_of_part_to_comm();
}

CUDA_particle_data *gpu_get_particle_pointer() { return particle_data_device; }
CUDA_global_part_vars *gpu_get_global_particle_vars_pointer_host() {
  return &global_part_vars_host;
}
CUDA_global_part_vars *gpu_get_global_particle_vars_pointer() {
  return global_part_vars_device;
}
float *gpu_get_particle_force_pointer() { return particle_forces_device; }
CUDA_energy *gpu_get_energy_pointer() { return energy_device; }
float *gpu_get_particle_torque_pointer() { return particle_torques_device; }

CUDA_particle_seed *gpu_get_particle_seed_pointer() {
  return particle_seeds_device;
}

void copy_part_data_to_gpu(ParticleRange particles) {
  COMM_TRACE(printf("global_part_vars_host.communication_enabled = %d && "
                    "global_part_vars_host.number_of_particles = %d\n",
                    global_part_vars_host.communication_enabled,
                    global_part_vars_host.number_of_particles));
  if (global_part_vars_host.communication_enabled == 1 &&
      global_part_vars_host.number_of_particles) {
    cuda_mpi_get_particles(particles, particle_data_host);

    /** get espresso md particle values*/
    if (this_node == 0)
      hipMemcpyAsync(particle_data_device, particle_data_host,
                      global_part_vars_host.number_of_particles *
                          sizeof(CUDA_particle_data),
                      hipMemcpyHostToDevice, stream[0]);
  }
}

/** setup and call kernel to copy particle forces to host
 */
void copy_forces_from_GPU(ParticleRange particles) {

  if (global_part_vars_host.communication_enabled == 1 &&
      global_part_vars_host.number_of_particles) {

    /** Copy result from device memory to host memory*/
    if (this_node == 0) {
      cuda_safe_mem(hipMemcpy(
          &(particle_forces_host[0]), particle_forces_device,
          3 * global_part_vars_host.number_of_particles * sizeof(float),
          hipMemcpyDeviceToHost));
#ifdef ROTATION
      cuda_safe_mem(hipMemcpy(
          &(particle_torques_host[0]), particle_torques_device,
          global_part_vars_host.number_of_particles * 3 * sizeof(float),
          hipMemcpyDeviceToHost));
#endif

      /** values for the particle kernel */
      int threads_per_block_particles = 64;
      int blocks_per_grid_particles_y = 4;
      int blocks_per_grid_particles_x =
          (global_part_vars_host.number_of_particles +
           threads_per_block_particles * blocks_per_grid_particles_y - 1) /
          (threads_per_block_particles * blocks_per_grid_particles_y);
      dim3 dim_grid_particles = make_uint3(blocks_per_grid_particles_x,
                                           blocks_per_grid_particles_y, 1);

      /** reset part forces with zero*/

      KERNELCALL(reset_particle_force, dim_grid_particles,
                 threads_per_block_particles, particle_forces_device,
                 particle_torques_device);
      hipDeviceSynchronize();
    }

    cuda_mpi_send_forces(particles, particle_forces_host,
                         particle_torques_host);
  }
}

#if defined(ENGINE) && defined(LB_GPU)
// setup and call kernel to copy v_cs to host
void copy_v_cs_from_GPU(ParticleRange particles) {
  if (global_part_vars_host.communication_enabled == 1 &&
      global_part_vars_host.number_of_particles) {
    // Copy result from device memory to host memory
    if (this_node == 0) {
      cuda_safe_mem(hipMemcpy2D(
          host_v_cs.data(), sizeof(CUDA_v_cs), particle_data_device,
          sizeof(CUDA_particle_data), sizeof(CUDA_v_cs),
          global_part_vars_host.number_of_particles, hipMemcpyDeviceToHost));
    }
    cuda_mpi_send_v_cs(particles, host_v_cs);
  }
}
#endif

void clear_energy_on_GPU() {
  if (!global_part_vars_host.communication_enabled)
    // || !global_part_vars_host.number_of_particles )
    return;
  if (energy_device == nullptr)
    cuda_safe_mem(hipMalloc((void **)&energy_device, sizeof(CUDA_energy)));
  cuda_safe_mem(hipMemset(energy_device, 0, sizeof(CUDA_energy)));
}

void copy_energy_from_GPU() {
  if (!global_part_vars_host.communication_enabled ||
      !global_part_vars_host.number_of_particles)
    return;
  cuda_safe_mem(hipMemcpy(&energy_host, energy_device, sizeof(CUDA_energy),
                           hipMemcpyDeviceToHost));
  copy_CUDA_energy_to_energy(energy_host);
}

/** Generic copy functions from an to device **/

void cuda_copy_to_device(void *host_data, void *device_data, size_t n) {
  cuda_safe_mem(hipMemcpy(host_data, device_data, n, hipMemcpyHostToDevice));
}

void cuda_copy_to_host(void *host_device, void *device_host, size_t n) {
  cuda_safe_mem(
      hipMemcpy(host_device, device_host, n, hipMemcpyDeviceToHost));
}
