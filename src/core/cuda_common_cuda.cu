/*
 * Copyright (C) 2010-2019 The ESPResSo project
 *
 * This file is part of ESPResSo.
 *
 * ESPResSo is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * ESPResSo is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */
#include "cuda_wrapper.hpp"

#include "config.hpp"

#include "ParticleRange.hpp"
#include "cuda_init.hpp"
#include "cuda_interface.hpp"
#include "cuda_utils.hpp"
#include "errorhandling.hpp"

#include "CudaDeviceAllocator.hpp"
#include "CudaHostAllocator.hpp"

#include <thrust/device_vector.h>
#include <utils/constants.hpp>

extern int this_node;

template <class T>
using device_vector = thrust::device_vector<T, CudaDeviceAllocator<T>>;

static CUDA_global_part_vars global_part_vars_host = {};

template <class T, class A>
T *raw_data_pointer(thrust::device_vector<T, A> &vec) {
  return thrust::raw_pointer_cast(vec.data());
}

template <class SpanLike> size_t byte_size(SpanLike const &v) {
  return v.size() * sizeof(typename SpanLike::value_type);
}

/** struct for particle force */
static device_vector<float> particle_forces_device;
static device_vector<float> particle_torques_device;

/** struct for particle position and velocity */
static device_vector<CUDA_particle_data> particle_data_device;
/** struct for energies */
static CUDA_energy *energy_device = nullptr;

pinned_vector<CUDA_particle_data> particle_data_host;
pinned_vector<float> particle_forces_host;
CUDA_energy energy_host;

pinned_vector<float> particle_torques_host;

/**cuda streams for parallel computing on cpu and gpu */
hipStream_t stream[1];

hipError_t _err;
hipError_t CU_err;

void _cuda_check_errors(const dim3 &block, const dim3 &grid,
                        const char *function, const char *file,
                        unsigned int line) {
  CU_err = hipGetLastError();
  if (CU_err != hipSuccess) {
    fprintf(stderr,
            "%d: error \"%s\" calling %s with dim %d %d %d, grid %d %d "
            "%d in %s:%u\n",
            this_node, hipGetErrorString(CU_err), function, block.x, block.y,
            block.z, grid.x, grid.y, grid.z, file, line);
    errexit();
  }
}

/**
 * @brief Resize a @ref device_vector.
 *
 * Due to a bug in thrust (https://github.com/thrust/thrust/issues/939),
 * resizing or appending to default constructed containers causes undefined
 * behavior by dereferencing a null-pointer for certain types. This
 * function is used instead of the resize member function to side-step
 * the problem. This is done by replacing the existing vector by a new
 * one constructed with the desired size if resizing from capacity zero.
 * Behaves as-if vec.resize(n) was called.
 *
 * @tparam T Type contained in the vector.
 * @param vec Vector To resize.
 * @param n Desired new size of the element.
 */
template <class T> void resize_or_replace(device_vector<T> &vec, size_t n) {
  if (vec.capacity() == 0) {
    vec = device_vector<T>(n);
  } else {
    vec.resize(n);
  }
}

void resize_buffers(size_t number_of_particles) {
  particle_data_host.resize(number_of_particles);
  resize_or_replace(particle_data_device, number_of_particles);

  particle_forces_host.resize(3 * number_of_particles);
  resize_or_replace(particle_forces_device, 3 * number_of_particles);

#ifdef ROTATION
  particle_torques_host.resize(3 * number_of_particles);
  resize_or_replace(particle_torques_device, 3 * number_of_particles);
#endif
}

/** setup and call particle reallocation from the host
 *  Note that in addition to calling this function the parameters must be
 * broadcast with either:
 * 1) cuda_bcast_global_part_params(); (when just being executed on the master
 * node) or
 * 2) MPI_Bcast(gpu_get_global_particle_vars_pointer_host(),
 * sizeof(CUDA_global_part_vars), MPI_BYTE, 0, comm_cart); (when executed on all
 * nodes)
 */
void gpu_init_particle_comm() {
  if (this_node == 0 && global_part_vars_host.communication_enabled == 0) {
    if (cuda_get_n_gpus() == -1) {
      runtimeErrorMsg()
          << "Unable to initialize CUDA as no sufficient GPU is available.";
      errexit();
    }
    if (cuda_get_n_gpus() > 1) {
      runtimeWarningMsg() << "More than one GPU detected, please note ESPResSo "
                             "uses device 0 by default regardless of usage or "
                             "capability. The GPU to be used can be modified "
                             "by setting System.cuda_init_handle.device.";
      if (cuda_check_gpu(0) != ES_OK) {
        runtimeWarningMsg()
            << "CUDA device 0 is not capable of running ESPResSo but is used "
               "by default. ESPResSo has detected a CUDA capable card but it "
               "is not the one used by ESPResSo by default. Please set the "
               "GPU to use by setting System.cuda_init_handle.device. A list "
               "of available GPUs is available through "
               "System.cuda_init_handle.device_list.";
      }
    }
  }
  global_part_vars_host.communication_enabled = 1;
}

Utils::Span<CUDA_particle_data> gpu_get_particle_pointer() {
  return {raw_data_pointer(particle_data_device), particle_data_device.size()};
}
CUDA_global_part_vars *gpu_get_global_particle_vars_pointer_host() {
  return &global_part_vars_host;
}
float *gpu_get_particle_force_pointer() {
  return raw_data_pointer(particle_forces_device);
}
CUDA_energy *gpu_get_energy_pointer() { return energy_device; }
float *gpu_get_particle_torque_pointer() {
  return raw_data_pointer(particle_torques_device);
}

void copy_part_data_to_gpu(ParticleRange particles) {
  if (global_part_vars_host.communication_enabled == 1) {
    cuda_mpi_get_particles(particles, particle_data_host);

    resize_buffers(particle_data_host.size());

    /* get espressomd particle values */
    if (this_node == 0) {
      hipMemsetAsync(raw_data_pointer(particle_forces_device), 0x0,
                      byte_size(particle_forces_device), stream[0]);
#ifdef ROTATION
      hipMemsetAsync(raw_data_pointer(particle_torques_device), 0x0,
                      byte_size(particle_torques_device), stream[0]);
#endif
      hipMemcpyAsync(raw_data_pointer(particle_data_device),
                      particle_data_host.data(), byte_size(particle_data_host),
                      hipMemcpyHostToDevice, stream[0]);
    }
  }
}

/** setup and call kernel to copy particle forces to host
 */
void copy_forces_from_GPU(ParticleRange &particles) {
  if (global_part_vars_host.communication_enabled == 1) {
    /* Copy result from device memory to host memory*/
    if (this_node == 0 && (not particle_forces_device.empty())) {
      thrust::copy(particle_forces_device.begin(), particle_forces_device.end(),
                   particle_forces_host.begin());
#ifdef ROTATION
      thrust::copy(particle_torques_device.begin(),
                   particle_torques_device.end(),
                   particle_torques_host.begin());
#endif
    }

    cuda_mpi_send_forces(
        particles, {particle_forces_host.data(), particle_forces_host.size()},
        {particle_torques_host.data(), particle_torques_host.size()});
  }
}

void clear_energy_on_GPU() {
  if (!global_part_vars_host.communication_enabled)
    return;
  if (energy_device == nullptr)
    cuda_safe_mem(hipMalloc((void **)&energy_device, sizeof(CUDA_energy)));
  cuda_safe_mem(hipMemset(energy_device, 0, sizeof(CUDA_energy)));
}

CUDA_energy copy_energy_from_GPU() {
  if (!global_part_vars_host.communication_enabled)
    return {};
  cuda_safe_mem(hipMemcpy(&energy_host, energy_device, sizeof(CUDA_energy),
                           hipMemcpyDeviceToHost));
  return energy_host;
}

void _cuda_safe_mem(hipError_t CU_err, const char *file, unsigned int line) {
  if (hipSuccess != CU_err) {
    fprintf(stderr, "Cuda Memory error at %s:%u.\n", file, line);
    printf("CUDA error: %s\n", hipGetErrorString(CU_err));
    if (CU_err == hipErrorInvalidValue)
      fprintf(stderr, "You may have tried to allocate zero memory at %s:%u.\n",
              file, line);
    errexit();
  } else {
    CU_err = hipGetLastError();
    if (CU_err != hipSuccess) {
      fprintf(stderr,
              "Error found during memory operation. Possibly however "
              "from a failed operation before. %s:%u.\n",
              file, line);
      printf("CUDA error: %s\n", hipGetErrorString(CU_err));
      if (CU_err == hipErrorInvalidValue)
        fprintf(stderr,
                "You may have tried to allocate zero memory before %s:%u.\n",
                file, line);
      errexit();
    }
  }
}
