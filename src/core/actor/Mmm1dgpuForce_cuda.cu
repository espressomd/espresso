#include "hip/hip_runtime.h"
/*
  Copyright (C) 2014-2018 The ESPResSo project

  This file is part of ESPResSo.

  ESPResSo is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  ESPResSo is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "cuda_wrapper.hpp"

#include "actor/Mmm1dgpuForce.hpp"
#include "cuda_utils.hpp"

#if defined(OMPI_MPI_H) || defined(_MPI_H)
#error CU-file includes mpi.h! This should not happen!
#endif

#ifdef MMM1D_GPU

// the code is mostly multi-GPU capable, but Espresso is not yet
const int deviceCount = 1;
float multigpu_factors[] = {1.0};
#define hipSetDevice(d)

#include "EspressoSystemInterface.hpp"
#include "electrostatics_magnetostatics/coulomb.hpp"
#include "electrostatics_magnetostatics/mmm1d.hpp"
#include "mmm-common_cuda.hpp"

#if defined(OMPI_MPI_H) || defined(_MPI_H)
#error CU-file includes mpi.h! This should not happen!
#endif

const mmm1dgpu_real C_GAMMAf = C_GAMMA;
const mmm1dgpu_real C_2PIf = C_2PI;

__constant__ mmm1dgpu_real far_switch_radius_2[1] = {0.05 * 0.05};
__constant__ mmm1dgpu_real boxz[1];
__constant__ mmm1dgpu_real uz[1];
__constant__ mmm1dgpu_real coulomb_prefactor[1] = {1.0};
__constant__ int bessel_cutoff[1] = {5};
__constant__ mmm1dgpu_real maxPWerror[1] = {1e-5};

Mmm1dgpuForce::Mmm1dgpuForce(SystemInterface &s,
                             mmm1dgpu_real _coulomb_prefactor,
                             mmm1dgpu_real _maxPWerror,
                             mmm1dgpu_real _far_switch_radius,
                             int _bessel_cutoff)
    : numThreads(64), host_boxz(0), host_npart(0), need_tune(true), pairs(-1),
      dev_forcePairs(nullptr), dev_energyBlocks(nullptr),
      coulomb_prefactor(_coulomb_prefactor), maxPWerror(_maxPWerror),
      far_switch_radius(_far_switch_radius), bessel_cutoff(_bessel_cutoff) {
  // interface sanity checks
  if (!s.requestFGpu())
    std::cerr << "Mmm1dgpuForce needs access to forces on GPU!" << std::endl;

  if (!s.requestRGpu())
    std::cerr << "Mmm1dgpuForce needs access to positions on GPU!" << std::endl;

  if (!s.requestQGpu())
    std::cerr << "Mmm1dgpuForce needs access to charges on GPU!" << std::endl;

  // system sanity checks
  check_periodicity();

  modpsi_init();
}

void Mmm1dgpuForce::setup(SystemInterface &s) {
  if (s.box()[2] <= 0) {
    throw std::runtime_error(
        "Error: Please set box length before initializing MMM1D!");
  }
  if (need_tune == true && s.npart_gpu() > 0) {
    set_params(s.box()[2], coulomb.prefactor, maxPWerror, far_switch_radius,
               bessel_cutoff);
    tune(s, maxPWerror, far_switch_radius, bessel_cutoff);
  }
  if (s.box()[2] != host_boxz) {
    set_params(s.box()[2], 0, -1, -1, -1);
  }
  if (s.npart_gpu() == host_npart) // unchanged
  {
    return;
  }

  // For all but the largest systems, it is faster to store force pairs and then
  // sum them up. Atomics are just so slow: so unless we're limited by memory,
  // do the latter.
  pairs = 2;
  for (int d = 0; d < deviceCount; d++) {
    hipSetDevice(d);

    size_t freeMem, totalMem;
    hipMemGetInfo(&freeMem, &totalMem);
    if (freeMem / 2 <
        3 * s.npart_gpu() * s.npart_gpu() *
            sizeof(
                mmm1dgpu_real)) // don't use more than half the device's memory
    {
      std::cerr << "Switching to atomicAdd due to memory constraints."
                << std::endl;
      pairs = 0;
      break;
    }
  }
  if (dev_forcePairs)
    hipFree(dev_forcePairs);
  if (pairs) // we need memory to store force pairs
  {
    cuda_safe_mem(
        hipMalloc((void **)&dev_forcePairs,
                   3 * s.npart_gpu() * s.npart_gpu() * sizeof(mmm1dgpu_real)));
  }
  if (dev_energyBlocks)
    hipFree(dev_energyBlocks);
  cuda_safe_mem(hipMalloc((void **)&dev_energyBlocks,
                           numBlocks(s) * sizeof(mmm1dgpu_real)));
  host_npart = s.npart_gpu();
}

unsigned int Mmm1dgpuForce::numBlocks(SystemInterface &s) {
  int b = s.npart_gpu() * s.npart_gpu() / numThreads + 1;
  if (b > 65535)
    b = 65535;
  return b;
}

Mmm1dgpuForce::~Mmm1dgpuForce() {
  modpsi_destroy();
  hipFree(dev_forcePairs);
}

__forceinline__ __device__ mmm1dgpu_real sqpow(mmm1dgpu_real x) {
  return x * x;
}
__forceinline__ __device__ mmm1dgpu_real cbpow(mmm1dgpu_real x) {
  return x * x * x;
}

__device__ void sumReduction(mmm1dgpu_real *input, mmm1dgpu_real *sum) {
  int tid = threadIdx.x;
  for (int i = blockDim.x / 2; i > 0; i /= 2) {
    __syncthreads();
    if (tid < i)
      input[tid] += input[i + tid];
  }
  __syncthreads();
  if (tid == 0)
    sum[0] = input[0];
}

__global__ void sumKernel(mmm1dgpu_real *data, int N) {
  HIP_DYNAMIC_SHARED(mmm1dgpu_real, partialsums)
  if (blockIdx.x != 0)
    return;
  int tid = threadIdx.x;
  mmm1dgpu_real result = 0;

  for (int i = 0; i < N; i += blockDim.x) {
    if (i + tid >= N)
      partialsums[tid] = 0;
    else
      partialsums[tid] = data[i + tid];

    sumReduction(partialsums, &result);
    if (tid == 0) {
      if (i == 0)
        data[0] = 0;
      data[0] += result;
    }
  }
}

__global__ void besselTuneKernel(int *result, mmm1dgpu_real far_switch_radius,
                                 int maxCut) {
  mmm1dgpu_real arg = C_2PIf * *uz * far_switch_radius;
  mmm1dgpu_real pref = 4 * *uz * max(1.0f, C_2PIf * *uz);
  mmm1dgpu_real err;
  int P = 1;
  do {
    err = pref * dev_K1(arg * P) * exp(arg) / arg * (P - 1 + 1 / arg);
    P++;
  } while (err > *maxPWerror && P <= maxCut);
  P--;

  result[0] = P;
}

void Mmm1dgpuForce::tune(SystemInterface &s, mmm1dgpu_real _maxPWerror,
                         mmm1dgpu_real _far_switch_radius, int _bessel_cutoff) {
  mmm1dgpu_real far_switch_radius = _far_switch_radius;
  int bessel_cutoff = _bessel_cutoff;
  mmm1dgpu_real maxrad = host_boxz;

  if (_far_switch_radius < 0 && _bessel_cutoff < 0)
  // autodetermine switching radius and Bessel cutoff
  {
    mmm1dgpu_real bestrad = 0, besttime = INFINITY;

    for (far_switch_radius = 0.05 * maxrad; far_switch_radius < maxrad;
         far_switch_radius += 0.05 * maxrad) {
      set_params(0, 0, _maxPWerror, far_switch_radius, bessel_cutoff);
      tune(s, _maxPWerror, far_switch_radius, -2); // tune Bessel cutoff
      int runtime = force_benchmark(s);
      if (runtime < besttime) {
        besttime = runtime;
        bestrad = far_switch_radius;
      }
    }
    far_switch_radius = bestrad;

    set_params(0, 0, _maxPWerror, far_switch_radius, bessel_cutoff);
    tune(s, _maxPWerror, far_switch_radius, -2); // tune Bessel cutoff
  }

  else if (_bessel_cutoff < 0)
  // autodetermine Bessel cutoff
  {
    int *dev_cutoff;
    int maxCut = 30;
    cuda_safe_mem(hipMalloc((void **)&dev_cutoff, sizeof(int)));
    hipLaunchKernelGGL(besselTuneKernel, dim3(1), dim3(1), 0, 0, dev_cutoff,
                       far_switch_radius, maxCut);
    cuda_safe_mem(hipMemcpy(&bessel_cutoff, dev_cutoff, sizeof(int),
                             hipMemcpyDeviceToHost));
    hipFree(dev_cutoff);
    if (_bessel_cutoff != -2 &&
        bessel_cutoff >=
            maxCut) // we already have our switching radius and only need to
                    // determine the cutoff, i.e. this is the final tuning round
    {
      throw std::runtime_error(
          "No reasonable Bessel cutoff could be determined.");
    }

    set_params(0, 0, _maxPWerror, far_switch_radius, bessel_cutoff);
  }
}

void Mmm1dgpuForce::set_params(mmm1dgpu_real _boxz,
                               mmm1dgpu_real _coulomb_prefactor,
                               mmm1dgpu_real _maxPWerror,
                               mmm1dgpu_real _far_switch_radius,
                               int _bessel_cutoff, bool manual) {
  if (_boxz > 0 && _far_switch_radius > _boxz) {
    throw std::runtime_error(
        "switching radius must not be larger than box length");
  }
  mmm1dgpu_real _far_switch_radius_2 = _far_switch_radius * _far_switch_radius;
  mmm1dgpu_real _uz = 1.0 / _boxz;
  for (int d = 0; d < deviceCount; d++) {
    // double colons are needed to access the constant memory variables because
    // they are file globals and we have identically named class variables
    hipSetDevice(d);
    if (manual) // tuning needs to be performed again
    {
      far_switch_radius = _far_switch_radius;
      bessel_cutoff = _bessel_cutoff;
    }
    if (_far_switch_radius >= 0) {
      mmm1d_params.far_switch_radius_2 =
          _far_switch_radius * _far_switch_radius;
      cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(::far_switch_radius_2),
                                       &_far_switch_radius_2,
                                       sizeof(mmm1dgpu_real)));
      far_switch_radius = _far_switch_radius;
    }
    if (_boxz > 0) {
      host_boxz = _boxz;
      cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(::boxz), &_boxz,
                                       sizeof(mmm1dgpu_real)));
      cuda_safe_mem(
          hipMemcpyToSymbol(HIP_SYMBOL(::uz), &_uz, sizeof(mmm1dgpu_real)));
    }
    if (_coulomb_prefactor != 0) {
      cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(::coulomb_prefactor),
                                       &_coulomb_prefactor,
                                       sizeof(mmm1dgpu_real)));
      coulomb_prefactor = _coulomb_prefactor;
    }
    if (_bessel_cutoff > 0) {
      mmm1d_params.bessel_cutoff = _bessel_cutoff;
      cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(::bessel_cutoff),
                                       &_bessel_cutoff, sizeof(int)));
      bessel_cutoff = _bessel_cutoff;
    }
    if (_maxPWerror > 0) {
      mmm1d_params.maxPWerror = _maxPWerror;
      cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(::maxPWerror), &_maxPWerror,
                                       sizeof(mmm1dgpu_real)));
      maxPWerror = _maxPWerror;
    }
  }
  need_tune = true;

  // The changed parameters in mmm1d_params do not need to be broadcast: they
  // are only accessed by the TCL print function (on node 0) when you call inter
  // coulomb. The CUDA code only runs on node 0, so other nodes do not need the
  // parameters. We couldn't broadcast from here anyway because set_params()
  // might be called from inside computeForces() which is not a time at which
  // the MPI loop on the slave nodes is waiting for broadcasts.
}

__global__ void forcesKernel(const mmm1dgpu_real *__restrict__ r,
                             const mmm1dgpu_real *__restrict__ q,
                             mmm1dgpu_real *__restrict__ force, int N,
                             int pairs, int tStart, int tStop) {
  if (tStop < 0)
    tStop = N * N;

  for (int tid = threadIdx.x + blockIdx.x * blockDim.x + tStart; tid < tStop;
       tid += blockDim.x * gridDim.x) {
    int p1 = tid % N, p2 = tid / N;
    mmm1dgpu_real x = r[3 * p2] - r[3 * p1], y = r[3 * p2 + 1] - r[3 * p1 + 1],
                  z = r[3 * p2 + 2] - r[3 * p1 + 2];
    mmm1dgpu_real rxy2 = sqpow(x) + sqpow(y);
    mmm1dgpu_real rxy = sqrt(rxy2);
    mmm1dgpu_real sum_r = 0, sum_z = 0;

    // if (*boxz <= 0.0) return; // in case we are not initialized yet

    while (fabs(z) > *boxz / 2) // make sure we take the shortest distance
      z -= (z > 0 ? 1 : -1) * *boxz;

    if (p1 == p2) // particle exerts no force on itself
    {
      rxy = 1; // so the division at the end doesn't fail with NaN (sum_r is 0
               // anyway)
    } else if (rxy2 <= *far_switch_radius_2) // near formula
    {
      mmm1dgpu_real uzz = *uz * z;
      mmm1dgpu_real uzr = *uz * rxy;
      sum_z = dev_mod_psi_odd(0, uzz);
      mmm1dgpu_real uzrpow = uzr;
      for (int n = 1; n < *device_n_modPsi; n++) {
        mmm1dgpu_real sum_r_old = sum_r;
        mmm1dgpu_real mpe = dev_mod_psi_even(n, uzz);
        mmm1dgpu_real mpo = dev_mod_psi_odd(n, uzz);

        sum_r += 2 * n * mpe * uzrpow;
        uzrpow *= uzr;
        sum_z += mpo * uzrpow;
        uzrpow *= uzr;

        if (fabs(sum_r_old - sum_r) < *maxPWerror)
          break;
      }

      sum_r *= sqpow(*uz);
      sum_z *= sqpow(*uz);

      sum_r += rxy * cbpow(rsqrt(rxy2 + sqpow(z)));
      sum_r += rxy * cbpow(rsqrt(rxy2 + sqpow(z + *boxz)));
      sum_r += rxy * cbpow(rsqrt(rxy2 + sqpow(z - *boxz)));

      sum_z += z * cbpow(rsqrt(rxy2 + sqpow(z)));
      sum_z += (z + *boxz) * cbpow(rsqrt(rxy2 + sqpow(z + *boxz)));
      sum_z += (z - *boxz) * cbpow(rsqrt(rxy2 + sqpow(z - *boxz)));

      if (rxy == 0) // particles at the same radial position only exert a force
                    // in z direction
      {
        rxy = 1; // so the division at the end doesn't fail with NaN (sum_r is 0
                 // anyway)
      }
    } else // far formula
    {
      for (int p = 1; p < *bessel_cutoff; p++) {
        mmm1dgpu_real arg = C_2PIf * *uz * p;
        sum_r += p * dev_K1(arg * rxy) * cos(arg * z);
        sum_z += p * dev_K0(arg * rxy) * sin(arg * z);
      }
      sum_r *= sqpow(*uz) * 4 * C_2PIf;
      sum_z *= sqpow(*uz) * 4 * C_2PIf;
      sum_r += 2 * *uz / rxy;
    }

    mmm1dgpu_real pref = *coulomb_prefactor * q[p1] * q[p2];
    if (pairs) {
      force[3 * (p1 + p2 * N - tStart)] = pref * sum_r / rxy * x;
      force[3 * (p1 + p2 * N - tStart) + 1] = pref * sum_r / rxy * y;
      force[3 * (p1 + p2 * N - tStart) + 2] = pref * sum_z;
    } else {
      atomicAdd(&force[3 * p2], pref * sum_r / rxy * x);
      atomicAdd(&force[3 * p2 + 1], pref * sum_r / rxy * y);
      atomicAdd(&force[3 * p2 + 2], pref * sum_z);
    }
  }
}

__global__ void energiesKernel(const mmm1dgpu_real *__restrict__ r,
                               const mmm1dgpu_real *__restrict__ q,
                               mmm1dgpu_real *__restrict__ energy, int N,
                               int pairs, int tStart, int tStop) {
  if (tStop < 0)
    tStop = N * N;

  HIP_DYNAMIC_SHARED(mmm1dgpu_real, partialsums)
  if (!pairs) {
    partialsums[threadIdx.x] = 0;
    __syncthreads();
  }
  for (int tid = threadIdx.x + blockIdx.x * blockDim.x + tStart; tid < tStop;
       tid += blockDim.x * gridDim.x) {
    int p1 = tid % N, p2 = tid / N;
    mmm1dgpu_real z = r[3 * p2 + 2] - r[3 * p1 + 2];
    mmm1dgpu_real rxy2 =
        sqpow(r[3 * p2] - r[3 * p1]) + sqpow(r[3 * p2 + 1] - r[3 * p1 + 1]);
    mmm1dgpu_real rxy = sqrt(rxy2);
    mmm1dgpu_real sum_e = 0;

    // if (*boxz <= 0.0) return; // in case we are not initialized yet

    while (fabs(z) > *boxz / 2) // make sure we take the shortest distance
      z -= (z > 0 ? 1 : -1) * *boxz;

    if (p1 == p2) // particle exerts no force on itself
    {
    } else if (rxy2 <= *far_switch_radius_2) // near formula
    {
      mmm1dgpu_real uzz = *uz * z;
      mmm1dgpu_real uzr2 = sqpow(*uz * rxy);
      mmm1dgpu_real uzrpow = uzr2;
      sum_e = dev_mod_psi_even(0, uzz);
      for (int n = 1; n < *device_n_modPsi; n++) {
        mmm1dgpu_real sum_e_old = sum_e;
        mmm1dgpu_real mpe = dev_mod_psi_even(n, uzz);
        sum_e += mpe * uzrpow;
        uzrpow *= uzr2;

        if (fabs(sum_e_old - sum_e) < *maxPWerror)
          break;
      }

      sum_e *= -1 * *uz;
      sum_e -= 2 * *uz * C_GAMMAf;
      sum_e += rsqrt(rxy2 + sqpow(z));
      sum_e += rsqrt(rxy2 + sqpow(z + *boxz));
      sum_e += rsqrt(rxy2 + sqpow(z - *boxz));
    } else // far formula
    {
      sum_e = -(log(rxy * *uz / 2) + C_GAMMAf) / 2;
      for (int p = 1; p < *bessel_cutoff; p++) {
        mmm1dgpu_real arg = C_2PIf * *uz * p;
        sum_e += dev_K0(arg * rxy) * cos(arg * z);
      }
      sum_e *= *uz * 4;
    }

    if (pairs) {
      energy[p1 + p2 * N - tStart] = *coulomb_prefactor * q[p1] * q[p2] * sum_e;
    } else {
      partialsums[threadIdx.x] += *coulomb_prefactor * q[p1] * q[p2] * sum_e;
    }
  }
  if (!pairs) {
    sumReduction(partialsums, &energy[blockIdx.x]);
  }
}

__global__ void vectorReductionKernel(mmm1dgpu_real *src, mmm1dgpu_real *dst,
                                      int N, int tStart, int tStop) {
  if (tStop < 0)
    tStop = N * N;

  for (int tid = threadIdx.x + blockIdx.x * blockDim.x; tid < N;
       tid += blockDim.x * gridDim.x) {
    int offset = ((tid + (tStart % N)) % N);

    for (int i = 0; tid + i * N < (tStop - tStart); i++) {
#pragma unroll 3
      for (int d = 0; d < 3; d++) {
        dst[3 * offset + d] -= src[3 * (tid + i * N) + d];
      }
    }
  }
}

void Mmm1dgpuForce::computeForces(SystemInterface &s) {
  if (coulomb.method !=
      COULOMB_MMM1D_GPU) // MMM1DGPU was disabled. nobody cares about our
                         // calculations anymore
  {
    std::cerr << "MMM1D: coulomb.method has been changed, skipping calculation"
              << std::endl;
    return;
  }
  setup(s);

  if (pairs < 0) {
    throw std::runtime_error("MMM1D was not initialized correctly");
  }

  if (pairs) // if we calculate force pairs, we need to reduce them to forces
  {
    int blocksRed = s.npart_gpu() / numThreads + 1;
    KERNELCALL(forcesKernel, numBlocks(s), numThreads, s.rGpuBegin(),
               s.qGpuBegin(), dev_forcePairs, s.npart_gpu(), pairs, 0, -1)
    KERNELCALL(vectorReductionKernel, blocksRed, numThreads, dev_forcePairs,
               s.fGpuBegin(), s.npart_gpu(), 0, -1)
  } else {
    KERNELCALL(forcesKernel, numBlocks(s), numThreads, s.rGpuBegin(),
               s.qGpuBegin(), s.fGpuBegin(), s.npart_gpu(), pairs, 0, -1)
  }
}

__global__ void scaleAndAddKernel(mmm1dgpu_real *dst, mmm1dgpu_real *src, int N,
                                  mmm1dgpu_real factor) {
  for (int tid = threadIdx.x + blockIdx.x * blockDim.x; tid < N;
       tid += blockDim.x * gridDim.x) {
    dst[tid] += src[tid] * factor;
  }
}

void Mmm1dgpuForce::computeEnergy(SystemInterface &s) {
  if (coulomb.method !=
      COULOMB_MMM1D_GPU) // MMM1DGPU was disabled. nobody cares about our
                         // calculations anymore
  {
    std::cerr << "MMM1D: coulomb.method has been changed, skipping calculation"
              << std::endl;
    return;
  }
  setup(s);

  if (pairs < 0) {
    throw std::runtime_error("MMM1D was not initialized correctly");
  }
  int shared = numThreads * sizeof(mmm1dgpu_real);

  KERNELCALL_shared(energiesKernel, numBlocks(s), numThreads, shared,
                    s.rGpuBegin(), s.qGpuBegin(), dev_energyBlocks,
                    s.npart_gpu(), 0, 0, -1);
  KERNELCALL_shared(sumKernel, 1, numThreads, shared, dev_energyBlocks,
                    numBlocks(s));
  KERNELCALL(scaleAndAddKernel, 1, 1, &(((CUDA_energy *)s.eGpu())->coulomb),
             &dev_energyBlocks[0], 1,
             0.5); // we have counted every interaction twice, so halve the
                   // total energy
}

float Mmm1dgpuForce::force_benchmark(SystemInterface &s) {
  hipEvent_t eventStart, eventStop;
  float elapsedTime;
  mmm1dgpu_real *dev_f_benchmark;

  cuda_safe_mem(hipMalloc((void **)&dev_f_benchmark,
                           3 * s.npart_gpu() * sizeof(mmm1dgpu_real)));
  cuda_safe_mem(hipEventCreate(&eventStart));
  cuda_safe_mem(hipEventCreate(&eventStop));
  cuda_safe_mem(hipEventRecord(eventStart, stream[0]));
  KERNELCALL(forcesKernel, numBlocks(s), numThreads, s.rGpuBegin(),
             s.qGpuBegin(), dev_f_benchmark, s.npart_gpu(), 0, 0, -1)
  cuda_safe_mem(hipEventRecord(eventStop, stream[0]));
  cuda_safe_mem(hipEventSynchronize(eventStop));
  cuda_safe_mem(hipEventElapsedTime(&elapsedTime, eventStart, eventStop));
  cuda_safe_mem(hipEventDestroy(eventStart));
  cuda_safe_mem(hipEventDestroy(eventStop));
  cuda_safe_mem(hipFree(dev_f_benchmark));

  return elapsedTime;
}

#endif /* MMM1D_GPU */
