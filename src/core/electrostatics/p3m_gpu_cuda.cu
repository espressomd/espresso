#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2010-2022 The ESPResSo project
 *
 * This file is part of ESPResSo.
 *
 * ESPResSo is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * ESPResSo is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

/**
 * @file
 *
 * P3M electrostatics on GPU.
 *
 * The corresponding header file is @ref p3m_gpu_cuda.cuh.
 */

#include "config/config.hpp"

#ifdef ELECTROSTATICS

#define P3M_GPU_FLOAT
// #define P3M_GPU_REAL_DOUBLE

#ifdef P3M_GPU_FLOAT
#define REAL_TYPE float
#define FFT_TYPE_COMPLEX hipfftComplex
#define FFT_FORW_FFT hipfftExecR2C
#define FFT_BACK_FFT hipfftExecC2R
#define FFT_PLAN_FORW_FLAG HIPFFT_R2C
#define FFT_PLAN_BACK_FLAG HIPFFT_C2R
#endif

#ifdef P3M_GPU_REAL_DOUBLE
#define REAL_TYPE double
#define FFT_TYPE_COMPLEX hipfftDoubleComplex
#define FFT_FORW_FFT hipfftExecD2Z
#define FFT_BACK_FFT hipfftExecZ2D
#define FFT_PLAN_FORW_FLAG HIPFFT_D2Z
#define FFT_PLAN_BACK_FLAG HIPFFT_Z2D
#endif

#include "electrostatics/p3m_gpu_cuda.cuh"

#include "cuda/utils.cuh"
#include "system/System.hpp"

#include <utils/math/bspline.hpp>
#include <utils/math/int_pow.hpp>
#include <utils/math/sinc.hpp>
#include <utils/math/sqr.hpp>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <cstdio>
#include <cstdlib>
#include <stdexcept>

#if defined(OMPI_MPI_H) || defined(_MPI_H)
#error CU-file includes mpi.h! This should not happen!
#endif

using Utils::int_pow;
using Utils::sqr;

struct P3MGpuData {
  /** Charge mesh */
  FFT_TYPE_COMPLEX *charge_mesh;
  /** Force meshes */
  FFT_TYPE_COMPLEX *force_mesh_x;
  FFT_TYPE_COMPLEX *force_mesh_y;
  FFT_TYPE_COMPLEX *force_mesh_z;
  /** Influence Function */
  REAL_TYPE *G_hat;
  /** Charge assignment order */
  int cao;
  /** Total number of mesh points (including padding) */
  int mesh_size;
  /** Ewald parameter */
  REAL_TYPE alpha;
  /** Number of particles */
  unsigned int n_part;
  /** Box size */
  REAL_TYPE box[3];
  /** Mesh dimensions */
  int mesh[3];
  /** Padded size */
  int mesh_z_padded;
  /** Inverse mesh spacing */
  REAL_TYPE hi[3];
  /** Position shift */
  REAL_TYPE pos_shift;
};

struct P3MGpuFftPlan {
  /** Forward FFT plan */
  hipfftHandle forw_plan;
  /** Backward FFT plan */
  hipfftHandle back_plan;
};

struct P3MGpuParams {
  P3MGpuData p3m_gpu_data;
  P3MGpuFftPlan p3m_fft;
  bool is_initialized;

  ~P3MGpuParams() { free_device_memory(); }

  void free_device_memory() {
    auto const free_device_pointer = [](auto *&ptr) {
      if (ptr != nullptr) {
        cuda_safe_mem(hipFree(reinterpret_cast<void *>(ptr)));
        ptr = nullptr;
      }
    };
    free_device_pointer(p3m_gpu_data.charge_mesh);
    free_device_pointer(p3m_gpu_data.force_mesh_x);
    free_device_pointer(p3m_gpu_data.force_mesh_y);
    free_device_pointer(p3m_gpu_data.force_mesh_z);
    free_device_pointer(p3m_gpu_data.G_hat);
    hipfftDestroy(p3m_fft.forw_plan);
    hipfftDestroy(p3m_fft.back_plan);
    is_initialized = false;
  }
};

template <int cao>
__device__ void static Aliasing_sums_ik(const P3MGpuData p, int NX, int NY,
                                        int NZ, REAL_TYPE *Zaehler,
                                        REAL_TYPE *Nenner) {
  REAL_TYPE S1, S2, S3;
  REAL_TYPE zwi;
  int MX, MY, MZ;
  REAL_TYPE NMX, NMY, NMZ;
  REAL_TYPE NM2;
  REAL_TYPE TE;
  REAL_TYPE Leni[3];
  REAL_TYPE Meshi[3];
  for (int i = 0; i < 3; ++i) {
    Leni[i] = 1.0f / p.box[i];
    Meshi[i] = 1.0f / static_cast<REAL_TYPE>(p.mesh[i]);
  }

  Zaehler[0] = Zaehler[1] = Zaehler[2] = *Nenner = 0.0;

  for (MX = -P3M_BRILLOUIN; MX <= P3M_BRILLOUIN; MX++) {
    NMX = static_cast<REAL_TYPE>(((NX > p.mesh[0] / 2) ? NX - p.mesh[0] : NX) +
                                 p.mesh[0] * MX);
    S1 = int_pow<2 * cao>(Utils::sinc(Meshi[0] * NMX));
    for (MY = -P3M_BRILLOUIN; MY <= P3M_BRILLOUIN; MY++) {
      NMY = static_cast<REAL_TYPE>(
          ((NY > p.mesh[1] / 2) ? NY - p.mesh[1] : NY) + p.mesh[1] * MY);
      S2 = S1 * int_pow<2 * cao>(Utils::sinc(Meshi[1] * NMY));
      for (MZ = -P3M_BRILLOUIN; MZ <= P3M_BRILLOUIN; MZ++) {
        NMZ = static_cast<REAL_TYPE>(
            ((NZ > p.mesh[2] / 2) ? NZ - p.mesh[2] : NZ) + p.mesh[2] * MZ);
        S3 = S2 * int_pow<2 * cao>(Utils::sinc(Meshi[2] * NMZ));

        NM2 = sqr(NMX * Leni[0]) + sqr(NMY * Leni[1]) + sqr(NMZ * Leni[2]);
        *Nenner += S3;

        TE = exp(-sqr(Utils::pi<REAL_TYPE>() / (p.alpha)) * NM2);
        zwi = S3 * TE / NM2;
        Zaehler[0] += NMX * zwi * Leni[0];
        Zaehler[1] += NMY * zwi * Leni[1];
        Zaehler[2] += NMZ * zwi * Leni[2];
      }
    }
  }
}

/* Calculate influence function */
template <int cao>
__global__ void calculate_influence_function_device(const P3MGpuData p) {

  const auto NX = static_cast<int>(blockDim.x * blockIdx.x + threadIdx.x);
  const auto NY = static_cast<int>(blockDim.y * blockIdx.y + threadIdx.y);
  const auto NZ = static_cast<int>(blockDim.z * blockIdx.z + threadIdx.z);
  REAL_TYPE Dnx, Dny, Dnz;
  REAL_TYPE Zaehler[3] = {0.0, 0.0, 0.0}, Nenner = 0.0;
  REAL_TYPE zwi;
  auto index = 0;
  REAL_TYPE Leni[3];
  for (int i = 0; i < 3; ++i) {
    Leni[i] = 1.0f / p.box[i];
  }

  if ((NX >= p.mesh[0]) || (NY >= p.mesh[1]) || (NZ >= (p.mesh[2] / 2 + 1)))
    return;

  index = NX * p.mesh[1] * (p.mesh[2] / 2 + 1) + NY * (p.mesh[2] / 2 + 1) + NZ;

  if (((NX == 0) && (NY == 0) && (NZ == 0)) ||
      ((NX % (p.mesh[0] / 2) == 0) && (NY % (p.mesh[1] / 2) == 0) &&
       (NZ % (p.mesh[2] / 2) == 0))) {
    p.G_hat[index] = 0;
  } else {
    Aliasing_sums_ik<cao>(p, NX, NY, NZ, Zaehler, &Nenner);

    Dnx = static_cast<REAL_TYPE>((NX > p.mesh[0] / 2) ? NX - p.mesh[0] : NX);
    Dny = static_cast<REAL_TYPE>((NY > p.mesh[1] / 2) ? NY - p.mesh[1] : NY);
    Dnz = static_cast<REAL_TYPE>((NZ > p.mesh[2] / 2) ? NZ - p.mesh[2] : NZ);

    zwi = Dnx * Zaehler[0] * Leni[0] + Dny * Zaehler[1] * Leni[1] +
          Dnz * Zaehler[2] * Leni[2];
    zwi /= ((sqr(Dnx * Leni[0]) + sqr(Dny * Leni[1]) + sqr(Dnz * Leni[2])) *
            sqr(Nenner));
    p.G_hat[index] = 2 * zwi / Utils::pi<REAL_TYPE>();
  }
}

#ifdef P3M_GPU_REAL_DOUBLE
__device__ double atomicAdd(double *address, double val) {
  unsigned long long int *address_as_ull = (unsigned long long int *)address;
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val + __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}
#endif

namespace {
__device__ inline auto linear_index_r(P3MGpuData const &p, int i, int j,
                                      int k) {
  return static_cast<unsigned int>(p.mesh[1] * p.mesh_z_padded * i +
                                   p.mesh_z_padded * j + k);
}

__device__ inline auto linear_index_k(P3MGpuData const &p, int i, int j,
                                      int k) {
  return static_cast<unsigned int>(p.mesh[1] * (p.mesh[2] / 2 + 1) * i +
                                   (p.mesh[2] / 2 + 1) * j + k);
}
} // namespace

__global__ void apply_diff_op(const P3MGpuData p) {
  auto const linear_index = linear_index_k(p, static_cast<int>(blockIdx.x),
                                           static_cast<int>(blockIdx.y),
                                           static_cast<int>(threadIdx.x));

  auto const bidx = static_cast<int>(blockIdx.x);
  auto const bidy = static_cast<int>(blockIdx.y);
  auto const nx = (bidx > p.mesh[0] / 2) ? bidx - p.mesh[0] : bidx;
  auto const ny = (bidy > p.mesh[1] / 2) ? bidy - p.mesh[1] : bidy;
  auto const nz = static_cast<int>(threadIdx.x);

  const FFT_TYPE_COMPLEX meshw = p.charge_mesh[linear_index];
  FFT_TYPE_COMPLEX buf;
  buf.x = -2.0f * Utils::pi<float>() * meshw.y;
  buf.y = 2.0f * Utils::pi<float>() * meshw.x;

  p.force_mesh_x[linear_index].x =
      static_cast<decltype(FFT_TYPE_COMPLEX::x)>(nx) * buf.x / p.box[0];
  p.force_mesh_x[linear_index].y =
      static_cast<decltype(FFT_TYPE_COMPLEX::x)>(nx) * buf.y / p.box[0];

  p.force_mesh_y[linear_index].x =
      static_cast<decltype(FFT_TYPE_COMPLEX::x)>(ny) * buf.x / p.box[1];
  p.force_mesh_y[linear_index].y =
      static_cast<decltype(FFT_TYPE_COMPLEX::x)>(ny) * buf.y / p.box[1];

  p.force_mesh_z[linear_index].x =
      static_cast<decltype(FFT_TYPE_COMPLEX::x)>(nz) * buf.x / p.box[2];
  p.force_mesh_z[linear_index].y =
      static_cast<decltype(FFT_TYPE_COMPLEX::x)>(nz) * buf.y / p.box[2];
}

__device__ inline int wrap_index(const int ind, const int mesh) {
  if (ind < 0)
    return ind + mesh;
  if (ind >= mesh)
    return ind - mesh;
  return ind;
}

__global__ void apply_influence_function(const P3MGpuData p) {
  auto const linear_index = linear_index_k(p, static_cast<int>(blockIdx.x),
                                           static_cast<int>(blockIdx.y),
                                           static_cast<int>(threadIdx.x));

  p.charge_mesh[linear_index].x *= p.G_hat[linear_index];
  p.charge_mesh[linear_index].y *= p.G_hat[linear_index];
}

template <int cao, bool shared>
__global__ void assign_charge_kernel(P3MGpuData const params,
                                     float const *const __restrict__ part_pos,
                                     float const *const __restrict__ part_q,
                                     unsigned int const parts_per_block) {
  auto const part_in_block = threadIdx.x / static_cast<unsigned int>(cao);
  auto const cao_id_x =
      threadIdx.x - part_in_block * static_cast<unsigned int>(cao);
  /* id of the particle */
  auto const id =
      parts_per_block * (blockIdx.x * gridDim.y + blockIdx.y) + part_in_block;
  if (id >= params.n_part)
    return;
  /* position relative to the closest gird point */
  REAL_TYPE m_pos[3];
  /* index of the nearest mesh point */
  int nmp_x, nmp_y, nmp_z;

  auto *charge_mesh = (REAL_TYPE *)params.charge_mesh;

  m_pos[0] = part_pos[3 * id + 0] * params.hi[0] - params.pos_shift;
  m_pos[1] = part_pos[3 * id + 1] * params.hi[1] - params.pos_shift;
  m_pos[2] = part_pos[3 * id + 2] * params.hi[2] - params.pos_shift;

  nmp_x = static_cast<int>(floorf(m_pos[0] + 0.5f));
  nmp_y = static_cast<int>(floorf(m_pos[1] + 0.5f));
  nmp_z = static_cast<int>(floorf(m_pos[2] + 0.5f));

  m_pos[0] -= static_cast<REAL_TYPE>(nmp_x);
  m_pos[1] -= static_cast<REAL_TYPE>(nmp_y);
  m_pos[2] -= static_cast<REAL_TYPE>(nmp_z);

  nmp_x = wrap_index(nmp_x + static_cast<int>(cao_id_x), params.mesh[0]);
  nmp_y = wrap_index(nmp_y + static_cast<int>(threadIdx.y), params.mesh[1]);
  nmp_z = wrap_index(nmp_z + static_cast<int>(threadIdx.z), params.mesh[2]);

  auto const index = linear_index_r(params, nmp_x, nmp_y, nmp_z);

  extern __shared__ float weights[];

  if (shared) {
    auto const offset = static_cast<unsigned int>(cao) * part_in_block;
    if ((threadIdx.y < 3u) && (threadIdx.z == 0u)) {
      weights[3u * offset + 3u * cao_id_x + threadIdx.y] =
          Utils::bspline<cao>(static_cast<int>(cao_id_x), m_pos[threadIdx.y]);
    }

    __syncthreads();

    auto const c = weights[3u * offset + 3u * cao_id_x] *
                   weights[3u * offset + 3u * threadIdx.y + 1u] *
                   weights[3u * offset + 3u * threadIdx.z + 2u] * part_q[id];
    atomicAdd(&(charge_mesh[index]), c);

  } else {
    auto const c =
        Utils::bspline<cao>(static_cast<int>(cao_id_x), m_pos[0]) * part_q[id] *
        Utils::bspline<cao>(static_cast<int>(threadIdx.y), m_pos[1]) *
        Utils::bspline<cao>(static_cast<int>(threadIdx.z), m_pos[2]);
    atomicAdd(&(charge_mesh[index]), c);
  }
}

void assign_charges(P3MGpuData const &params,
                    float const *const __restrict__ part_pos,
                    float const *const __restrict__ part_q) {
  auto const cao = static_cast<unsigned int>(params.cao);
  auto const cao3 = int_pow<3>(cao);
  unsigned int parts_per_block = 1u, n_blocks = 1u;

  while ((parts_per_block + 1u) * cao3 <= 1024u) {
    parts_per_block++;
  }
  if ((params.n_part % parts_per_block) == 0u)
    n_blocks = std::max<unsigned>(1u, params.n_part / parts_per_block);
  else
    n_blocks = params.n_part / parts_per_block + 1u;

  dim3 block(parts_per_block * cao, cao, cao);
  dim3 grid(n_blocks, 1u, 1u);
  while (grid.x > 65536u) {
    grid.y++;
    if ((n_blocks % grid.y) == 0u)
      grid.x = std::max<unsigned>(1u, n_blocks / grid.y);
    else
      grid.x = n_blocks / grid.y + 1u;
  }

  auto const data_length =
      3 * static_cast<std::size_t>(parts_per_block * cao) * sizeof(REAL_TYPE);
  switch (cao) {
  case 1:
    (assign_charge_kernel<1, false>)<<<grid, block, 0, nullptr>>>(
        params, part_pos, part_q, parts_per_block);
    break;
  case 2:
    (assign_charge_kernel<2, false>)<<<grid, block, 0, nullptr>>>(
        params, part_pos, part_q, parts_per_block);
    break;
  case 3:
    (assign_charge_kernel<3, true>)<<<grid, block, data_length, nullptr>>>(
        params, part_pos, part_q, parts_per_block);
    break;
  case 4:
    (assign_charge_kernel<4, true>)<<<grid, block, data_length, nullptr>>>(
        params, part_pos, part_q, parts_per_block);
    break;
  case 5:
    (assign_charge_kernel<5, true>)<<<grid, block, data_length, nullptr>>>(
        params, part_pos, part_q, parts_per_block);
    break;
  case 6:
    (assign_charge_kernel<6, true>)<<<grid, block, data_length, nullptr>>>(
        params, part_pos, part_q, parts_per_block);
    break;
  case 7:
    (assign_charge_kernel<7, true>)<<<grid, block, data_length, nullptr>>>(
        params, part_pos, part_q, parts_per_block);
    break;
  default:
    break;
  }
  cuda_check_errors_exit(block, grid, "assign_charge", __FILE__, __LINE__);
}

template <int cao, bool shared>
__global__ void assign_forces_kernel(P3MGpuData const params,
                                     float const *const __restrict__ part_pos,
                                     float const *const __restrict__ part_q,
                                     float *const __restrict__ part_f,
                                     REAL_TYPE prefactor,
                                     unsigned int const parts_per_block) {
  auto const part_in_block = threadIdx.x / static_cast<unsigned int>(cao);
  auto const cao_id_x =
      threadIdx.x - part_in_block * static_cast<unsigned int>(cao);
  /* id of the particle */
  auto const id =
      parts_per_block * (blockIdx.x * gridDim.y + blockIdx.y) + part_in_block;
  if (id >= params.n_part)
    return;
  /* position relative to the closest grid point */
  REAL_TYPE m_pos[3];
  /* index of the nearest mesh point */
  int nmp_x, nmp_y, nmp_z;

  m_pos[0] = part_pos[3 * id + 0] * params.hi[0] - params.pos_shift;
  m_pos[1] = part_pos[3 * id + 1] * params.hi[1] - params.pos_shift;
  m_pos[2] = part_pos[3 * id + 2] * params.hi[2] - params.pos_shift;

  nmp_x = static_cast<int>(floorf(m_pos[0] + REAL_TYPE{0.5}));
  nmp_y = static_cast<int>(floorf(m_pos[1] + REAL_TYPE{0.5}));
  nmp_z = static_cast<int>(floorf(m_pos[2] + REAL_TYPE{0.5}));

  m_pos[0] -= static_cast<REAL_TYPE>(nmp_x);
  m_pos[1] -= static_cast<REAL_TYPE>(nmp_y);
  m_pos[2] -= static_cast<REAL_TYPE>(nmp_z);

  nmp_x = wrap_index(nmp_x + static_cast<int>(cao_id_x), params.mesh[0]);
  nmp_y = wrap_index(nmp_y + static_cast<int>(threadIdx.y), params.mesh[1]);
  nmp_z = wrap_index(nmp_z + static_cast<int>(threadIdx.z), params.mesh[2]);

  auto const index = linear_index_r(params, nmp_x, nmp_y, nmp_z);

  extern __shared__ float weights[];

  REAL_TYPE c;
  if (shared) {
    auto const offset = static_cast<unsigned int>(cao) * part_in_block;
    if ((threadIdx.y < 3u) && (threadIdx.z == 0u)) {
      weights[3u * offset + 3u * cao_id_x + threadIdx.y] =
          Utils::bspline<cao>(static_cast<int>(cao_id_x), m_pos[threadIdx.y]);
    }

    __syncthreads();

    c = -prefactor * weights[3u * offset + 3u * cao_id_x] *
        weights[3u * offset + 3u * threadIdx.y + 1u] *
        weights[3u * offset + 3u * threadIdx.z + 2u] * part_q[id];
  } else {
    c = -prefactor * part_q[id] *
        Utils::bspline<cao>(static_cast<int>(cao_id_x), m_pos[0]) *
        Utils::bspline<cao>(static_cast<int>(threadIdx.y), m_pos[1]) *
        Utils::bspline<cao>(static_cast<int>(threadIdx.z), m_pos[2]);
  }

  const REAL_TYPE *force_mesh_x = (REAL_TYPE *)params.force_mesh_x;
  const REAL_TYPE *force_mesh_y = (REAL_TYPE *)params.force_mesh_y;
  const REAL_TYPE *force_mesh_z = (REAL_TYPE *)params.force_mesh_z;

  atomicAdd(&(part_f[3u * id + 0u]), c * force_mesh_x[index]);
  atomicAdd(&(part_f[3u * id + 1u]), c * force_mesh_y[index]);
  atomicAdd(&(part_f[3u * id + 2u]), c * force_mesh_z[index]);
}

void assign_forces(P3MGpuData const &params,
                   float const *const __restrict__ part_pos,
                   float const *const __restrict__ part_q,
                   float *const __restrict__ part_f,
                   REAL_TYPE const prefactor) {
  auto const cao = params.cao;
  auto const cao3 = int_pow<3>(cao);
  unsigned int parts_per_block = 1u, n_blocks = 1u;

  while ((parts_per_block + 1u) * static_cast<unsigned int>(cao3) <= 1024u) {
    parts_per_block++;
  }

  if ((params.n_part % parts_per_block) == 0u)
    n_blocks = std::max<unsigned>(1u, params.n_part / parts_per_block);
  else
    n_blocks = params.n_part / parts_per_block + 1u;

  dim3 block(parts_per_block * static_cast<unsigned int>(cao),
             static_cast<unsigned int>(cao), static_cast<unsigned int>(cao));
  dim3 grid(n_blocks, 1u, 1u);
  while (grid.x > 65536u) {
    grid.y++;
    if (n_blocks % grid.y == 0u)
      grid.x = std::max<unsigned>(1u, n_blocks / grid.y);
    else
      grid.x = n_blocks / grid.y + 1u;
  }

  /* Switch for assignment templates, the shared version only is faster for cao
   * > 2 */
  auto const data_length =
      3u *
      static_cast<std::size_t>(parts_per_block *
                               static_cast<unsigned int>(cao)) *
      sizeof(float);
  switch (cao) {
  case 1:
    (assign_forces_kernel<1, false>)<<<grid, block, 0, nullptr>>>(
        params, part_pos, part_q, part_f, prefactor, parts_per_block);
    break;
  case 2:
    (assign_forces_kernel<2, false>)<<<grid, block, 0, nullptr>>>(
        params, part_pos, part_q, part_f, prefactor, parts_per_block);
    break;
  case 3:
    (assign_forces_kernel<3, true>)<<<grid, block, data_length, nullptr>>>(
        params, part_pos, part_q, part_f, prefactor, parts_per_block);
    break;
  case 4:
    (assign_forces_kernel<4, true>)<<<grid, block, data_length, nullptr>>>(
        params, part_pos, part_q, part_f, prefactor, parts_per_block);
    break;
  case 5:
    (assign_forces_kernel<5, true>)<<<grid, block, data_length, nullptr>>>(
        params, part_pos, part_q, part_f, prefactor, parts_per_block);
    break;
  case 6:
    (assign_forces_kernel<6, true>)<<<grid, block, data_length, nullptr>>>(
        params, part_pos, part_q, part_f, prefactor, parts_per_block);
    break;
  case 7:
    (assign_forces_kernel<7, true>)<<<grid, block, data_length, nullptr>>>(
        params, part_pos, part_q, part_f, prefactor, parts_per_block);
    break;
  default:
    break;
  }
  cuda_check_errors_exit(block, grid, "assign_forces", __FILE__, __LINE__);
}

/* Init the internal data structures of the P3M GPU.
 * Mainly allocation on the device and influence function calculation.
 * Be advised: this needs mesh^3*5*sizeof(REAL_TYPE) of device memory.
 * We use real to complex FFTs, so the size of the reciprocal mesh
 * is (cuFFT convention) Nx x Ny x [ Nz /2 + 1 ].
 */
void p3m_gpu_init(std::shared_ptr<P3MGpuParams> &data, int cao,
                  const int mesh[3], double alpha, Utils::Vector3d const &box_l,
                  unsigned n_part) {
  if (mesh[0] == -1 && mesh[1] == -1 && mesh[2] == -1)
    throw std::runtime_error("P3M: invalid mesh size");

  if (not data) {
    data = std::make_shared<P3MGpuParams>();
  }

  auto &p3m_gpu_data = data->p3m_gpu_data;
  bool do_reinit = false, mesh_changed = false;
  p3m_gpu_data.n_part = n_part;

  if (not data->is_initialized or p3m_gpu_data.alpha != alpha) {
    p3m_gpu_data.alpha = static_cast<REAL_TYPE>(alpha);
    do_reinit = true;
  }

  if (not data->is_initialized or p3m_gpu_data.cao != cao) {
    p3m_gpu_data.cao = cao;
    // NOLINTNEXTLINE(bugprone-integer-division)
    p3m_gpu_data.pos_shift = static_cast<REAL_TYPE>((p3m_gpu_data.cao - 1) / 2);
    do_reinit = true;
  }

  if (not data->is_initialized or (p3m_gpu_data.mesh[0] != mesh[0]) or
      (p3m_gpu_data.mesh[1] != mesh[1]) or (p3m_gpu_data.mesh[2] != mesh[2])) {
    std::copy(mesh, mesh + 3, p3m_gpu_data.mesh);
    mesh_changed = true;
    do_reinit = true;
  }

  if (not data->is_initialized or (p3m_gpu_data.box[0] != box_l[0]) or
      (p3m_gpu_data.box[1] != box_l[1]) or (p3m_gpu_data.box[2] != box_l[2])) {
    std::copy(box_l.begin(), box_l.end(), p3m_gpu_data.box);
    do_reinit = true;
  }

  p3m_gpu_data.mesh_z_padded = (mesh[2] / 2 + 1) * 2;
  p3m_gpu_data.mesh_size = mesh[0] * mesh[1] * p3m_gpu_data.mesh_z_padded;

  for (int i = 0; i < 3; i++) {
    p3m_gpu_data.hi[i] =
        static_cast<REAL_TYPE>(p3m_gpu_data.mesh[i]) / p3m_gpu_data.box[i];
  }

  if (data->is_initialized and mesh_changed) {
    data->free_device_memory();
    data->is_initialized = false;
  }

  if (not data->is_initialized and p3m_gpu_data.mesh_size > 0) {
    /* Size of the complex mesh Nx * Ny * ( Nz / 2 + 1 ) */
    auto const cmesh_size =
        static_cast<std::size_t>(p3m_gpu_data.mesh[0]) *
        static_cast<std::size_t>(p3m_gpu_data.mesh[1]) *
        static_cast<std::size_t>(p3m_gpu_data.mesh[2] / 2 + 1);
    auto const mesh_len = cmesh_size * sizeof(FFT_TYPE_COMPLEX);
    cuda_safe_mem(hipMalloc((void **)&(p3m_gpu_data.charge_mesh), mesh_len));
    cuda_safe_mem(hipMalloc((void **)&(p3m_gpu_data.force_mesh_x), mesh_len));
    cuda_safe_mem(hipMalloc((void **)&(p3m_gpu_data.force_mesh_y), mesh_len));
    cuda_safe_mem(hipMalloc((void **)&(p3m_gpu_data.force_mesh_z), mesh_len));
    cuda_safe_mem(hipMalloc((void **)&(p3m_gpu_data.G_hat),
                             cmesh_size * sizeof(REAL_TYPE)));

    if (hipfftPlan3d(&(data->p3m_fft.forw_plan), mesh[0], mesh[1], mesh[2],
                    FFT_PLAN_FORW_FLAG) != HIPFFT_SUCCESS or
        hipfftPlan3d(&(data->p3m_fft.back_plan), mesh[0], mesh[1], mesh[2],
                    FFT_PLAN_BACK_FLAG) != HIPFFT_SUCCESS) {
      throw std::runtime_error("Unable to create fft plan");
    }
  }

  if ((do_reinit or not data->is_initialized) and p3m_gpu_data.mesh_size > 0) {
    dim3 grid(1, 1, 1);
    dim3 block(1, 1, 1);
    block.x = static_cast<unsigned>(512 / mesh[0] + 1);
    block.y = static_cast<unsigned>(mesh[1]);
    block.z = 1;
    grid.x = static_cast<unsigned>(mesh[0]) / block.x + 1;
    grid.z = static_cast<unsigned>(mesh[2]) / 2 + 1;

    switch (p3m_gpu_data.cao) {
    case 1:
      KERNELCALL(calculate_influence_function_device<1>, grid, block,
                 p3m_gpu_data);
      break;
    case 2:
      KERNELCALL(calculate_influence_function_device<2>, grid, block,
                 p3m_gpu_data);
      break;
    case 3:
      KERNELCALL(calculate_influence_function_device<3>, grid, block,
                 p3m_gpu_data);
      break;
    case 4:
      KERNELCALL(calculate_influence_function_device<4>, grid, block,
                 p3m_gpu_data);
      break;
    case 5:
      KERNELCALL(calculate_influence_function_device<5>, grid, block,
                 p3m_gpu_data);
      break;
    case 6:
      KERNELCALL(calculate_influence_function_device<6>, grid, block,
                 p3m_gpu_data);
      break;
    case 7:
      KERNELCALL(calculate_influence_function_device<7>, grid, block,
                 p3m_gpu_data);
      break;
    }
  }
  if (p3m_gpu_data.mesh_size > 0)
    data->is_initialized = true;
}

/**
 *  \brief The long-range part of the P3M algorithm.
 */
void p3m_gpu_add_farfield_force(P3MGpuParams &data, GpuParticleData &gpu,
                                double prefactor, unsigned n_part) {
  auto &p3m_gpu_data = data.p3m_gpu_data;
  p3m_gpu_data.n_part = n_part;

  if (p3m_gpu_data.n_part == 0u)
    return;

  auto const positions_device = gpu.get_particle_positions_device();
  auto const charges_device = gpu.get_particle_charges_device();
  auto const forces_device = gpu.get_particle_forces_device();

  dim3 gridConv(static_cast<unsigned>(p3m_gpu_data.mesh[0]),
                static_cast<unsigned>(p3m_gpu_data.mesh[1]), 1u);
  dim3 threadsConv(static_cast<unsigned>(p3m_gpu_data.mesh[2] / 2 + 1), 1u, 1u);

  auto const volume =
      Utils::product(Utils::Vector3<REAL_TYPE>(p3m_gpu_data.box));
  auto const pref = static_cast<REAL_TYPE>(prefactor) / (volume * REAL_TYPE{2});

  cuda_safe_mem(hipMemset(p3m_gpu_data.charge_mesh, 0,
                           static_cast<std::size_t>(p3m_gpu_data.mesh_size) *
                               sizeof(REAL_TYPE)));

  /* Interpolate the charges to the mesh */
  assign_charges(p3m_gpu_data, positions_device, charges_device);

  /* Do forward FFT of the charge mesh */
  if (FFT_FORW_FFT(data.p3m_fft.forw_plan,
                   (REAL_TYPE *)p3m_gpu_data.charge_mesh,
                   p3m_gpu_data.charge_mesh) != HIPFFT_SUCCESS) {
    fprintf(stderr, "CUFFT error: Forward FFT failed\n");
    return;
  }

  /* Do convolution */
  KERNELCALL(apply_influence_function, gridConv, threadsConv, p3m_gpu_data);

  /* Take derivative */
  KERNELCALL(apply_diff_op, gridConv, threadsConv, p3m_gpu_data);

  /* Transform the components of the electric field back */
  FFT_BACK_FFT(data.p3m_fft.back_plan, p3m_gpu_data.force_mesh_x,
               (REAL_TYPE *)p3m_gpu_data.force_mesh_x);
  FFT_BACK_FFT(data.p3m_fft.back_plan, p3m_gpu_data.force_mesh_y,
               (REAL_TYPE *)p3m_gpu_data.force_mesh_y);
  FFT_BACK_FFT(data.p3m_fft.back_plan, p3m_gpu_data.force_mesh_z,
               (REAL_TYPE *)p3m_gpu_data.force_mesh_z);

  /* Assign the forces from the mesh back to the particles */
  assign_forces(p3m_gpu_data, positions_device, charges_device, forces_device,
                pref);
}

#endif // ELECTROSTATICS
