#include "hip/hip_runtime.h"
/*
   Copyright (C) 2010-2018 The ESPResSo project

   This file is part of ESPResSo.

   ESPResSo is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.

   ESPResSo is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.

   You should have received a copy of the GNU General Public License
   along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/
/** \file
 *  %Lattice Boltzmann on GPUs.
 *
 *  The corresponding header file is lbgpu.cuh.
 */

#include "cuda_wrapper.hpp"
#include "curand_wrapper.hpp"

#include "config.hpp"

#ifdef LB_GPU
#include <cassert>
#include <stdio.h>
#include <stdlib.h>
#include <vector>

#include "cuda_interface.hpp"
#include "cuda_utils.hpp"
#include "debug.hpp"
#include "errorhandling.hpp"
#include "grid_based_algorithms/electrokinetics.hpp"
#include "grid_based_algorithms/electrokinetics_pdb_parse.hpp"
#include "grid_based_algorithms/lbgpu.cuh"
#include "grid_based_algorithms/lbgpu.hpp"
#include "utils/Array.hpp"
#include "utils/Counter.hpp"

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/transform_reduce.h>

#include <cassert>
#include <cstdint>
#include <stdio.h>
#include <stdlib.h>
#include <vector>

/** defining structures residing in global memory */

/** device_rho_v: struct for hydrodynamic fields: this is for internal use
 *  (i.e. stores values in LB units) and should not used for
 *  printing values
 */
static LB_rho_v_gpu *device_rho_v = nullptr;

/** print_rho_v_pi: struct for hydrodynamic fields: this is the interface
 *  and stores values in MD units. It should not used
 *  as an input for any LB calculations. TODO: in the future,
 *  one might want to have several structures for printing
 *  separately rho, v, pi without having to compute/store
 *  the complete set.
 */
static LB_rho_v_pi_gpu *print_rho_v_pi = nullptr;

/** @name structs for velocity densities */
/*@{*/
static LB_nodes_gpu nodes_a = {nullptr, nullptr};
static LB_nodes_gpu nodes_b = {nullptr, nullptr};
/*@}*/

/** struct for node force density */
LB_node_force_density_gpu node_f = {
    // force_density
    nullptr,
    // scforce_density
    nullptr,
#if defined(VIRTUAL_SITES_INERTIALESS_TRACERS) || defined(EK_DEBUG)
    // force_density_buf
    nullptr
#endif
};

static LB_extern_nodeforcedensity_gpu *extern_node_force_densities = nullptr;

#ifdef LB_BOUNDARIES_GPU
/** @brief Force on the boundary nodes */
static float *lb_boundary_force = nullptr;

/** @brief Velocity at the boundary */
static float *lb_boundary_velocity = nullptr;

/** @name pointers for bound index array */
/*@{*/
static int *boundary_node_list;
static int *boundary_index_list;
static size_t size_of_boundindex;
/*@}*/
#endif

EK_parameters *lb_ek_parameters_gpu;

/** @name pointers for additional cuda check flag */
/*@{*/
static int *gpu_check = nullptr;
static int *h_gpu_check = nullptr;
/*@}*/

/** @brief Direction of data transfer between @ref nodes_a and @ref nodes_b
 *  during integration in @ref lb_integrate_GPU
 */
static bool intflag = true;
LB_nodes_gpu *current_nodes = nullptr;
/** @name defining size values for allocating global memory */
/*@{*/
static size_t size_of_rho_v;
static size_t size_of_rho_v_pi;
static size_t size_of_extern_node_force_densities;
/*@}*/

/** Parameters residing in constant memory */
__device__ __constant__ LB_parameters_gpu para[1];
static const float c_sound_sq = 1.0f / 3.0f;

/*-------------------------------------------------------*/
/*********************************************************/
/** \name device functions called by kernel functions */
/*********************************************************/
/*-------------------------------------------------------*/

/*-------------------------------------------------------*/

static constexpr float sqrt12 = 3.4641016151377544f;
static Utils::Counter<uint64_t> rng_counter_coupling_gpu;
Utils::Counter<uint64_t> rng_counter_fluid_gpu;

/** Transformation from 1d array-index to xyz
 *  @param[in]  index   Node index / thread index
 */
template <typename T> __device__ uint3 index_to_xyz(T index) {
  auto const x = index % para->dim_x;
  index /= para->dim_x;
  auto const y = index % para->dim_y;
  index /= para->dim_y;
  auto const z = index;
  return {x, y, z};
}

/** Transformation from xyz to 1d array-index
 *  @param[in] x,y,z     The xyz array
 */
template <typename T> __device__ T xyz_to_index(T x, T y, T z) {
  return x + para->dim_x * (y + para->dim_y * z);
}

__device__ __inline__ float calc_mode_x_from_n(LB_nodes_gpu n_a,
                                               unsigned int index, int x) {
  auto const flat_index = [&index](int population) {
    return population * para->number_of_nodes + index;
  };
  switch (x) {
  case 0:
    return n_a.vd[flat_index(0)] + n_a.vd[flat_index(1)] +
           n_a.vd[flat_index(2)] + n_a.vd[flat_index(3)] +
           n_a.vd[flat_index(4)] + n_a.vd[flat_index(5)] +
           n_a.vd[flat_index(6)] + n_a.vd[flat_index(7)] +
           n_a.vd[flat_index(8)] + n_a.vd[flat_index(9)] +
           n_a.vd[flat_index(10)] + n_a.vd[flat_index(11)] +
           n_a.vd[flat_index(12)] + n_a.vd[flat_index(13)] +
           n_a.vd[flat_index(14)] + n_a.vd[flat_index(15)] +
           n_a.vd[flat_index(16)] + n_a.vd[flat_index(17)] +
           n_a.vd[flat_index(18)];
  case 1:
    return (n_a.vd[flat_index(1)] - n_a.vd[flat_index(2)]) +
           (n_a.vd[flat_index(7)] - n_a.vd[flat_index(8)]) +
           (n_a.vd[flat_index(9)] - n_a.vd[flat_index(10)]) +
           (n_a.vd[flat_index(11)] - n_a.vd[flat_index(12)]) +
           (n_a.vd[flat_index(13)] - n_a.vd[flat_index(14)]);
  case 2:
    return (n_a.vd[flat_index(3)] - n_a.vd[flat_index(4)]) +
           (n_a.vd[flat_index(7)] - n_a.vd[flat_index(8)]) -
           (n_a.vd[flat_index(9)] - n_a.vd[flat_index(10)]) +
           (n_a.vd[flat_index(15)] - n_a.vd[flat_index(16)]) +
           (n_a.vd[flat_index(17)] - n_a.vd[flat_index(18)]);
  case 3:
    return (n_a.vd[flat_index(5)] - n_a.vd[flat_index(6)]) +
           (n_a.vd[flat_index(11)] - n_a.vd[flat_index(12)]) -
           (n_a.vd[flat_index(13)] - n_a.vd[flat_index(14)]) +
           (n_a.vd[flat_index(15)] - n_a.vd[flat_index(16)]) -
           (n_a.vd[flat_index(17)] - n_a.vd[flat_index(18)]);
  case 4:
    return -n_a.vd[flat_index(0)] + n_a.vd[flat_index(7)] +
           n_a.vd[flat_index(8)] + n_a.vd[flat_index(9)] +
           n_a.vd[flat_index(10)] + n_a.vd[flat_index(11)] +
           n_a.vd[flat_index(12)] + n_a.vd[flat_index(13)] +
           n_a.vd[flat_index(14)] + n_a.vd[flat_index(15)] +
           n_a.vd[flat_index(16)] + n_a.vd[flat_index(17)] +
           n_a.vd[flat_index(18)];
  case 5:
    return (n_a.vd[flat_index(1)] + n_a.vd[flat_index(2)]) -
           (n_a.vd[flat_index(3)] + n_a.vd[flat_index(4)]) +
           (n_a.vd[flat_index(11)] + n_a.vd[flat_index(12)]) +
           (n_a.vd[flat_index(13)] + n_a.vd[flat_index(14)]) -
           (n_a.vd[flat_index(15)] + n_a.vd[flat_index(16)]) -
           (n_a.vd[flat_index(17)] + n_a.vd[flat_index(18)]);
  case 6:
    return (n_a.vd[flat_index(1)] + n_a.vd[flat_index(2)]) +
           (n_a.vd[flat_index(3)] + n_a.vd[flat_index(4)]) -
           (n_a.vd[flat_index(11)] + n_a.vd[flat_index(12)]) -
           (n_a.vd[flat_index(13)] + n_a.vd[flat_index(14)]) -
           (n_a.vd[flat_index(15)] + n_a.vd[flat_index(16)]) -
           (n_a.vd[flat_index(17)] + n_a.vd[flat_index(18)]) -
           2.0f * ((n_a.vd[flat_index(5)] + n_a.vd[flat_index(6)]) -
                   (n_a.vd[flat_index(7)] + n_a.vd[flat_index(8)]) -
                   (n_a.vd[flat_index(9)] + n_a.vd[flat_index(10)]));
  case 7:
    return (n_a.vd[flat_index(7)] + n_a.vd[flat_index(8)]) -
           (n_a.vd[flat_index(9)] + n_a.vd[flat_index(10)]);
  case 8:
    return (n_a.vd[flat_index(11)] + n_a.vd[flat_index(12)]) -
           (n_a.vd[flat_index(13)] + n_a.vd[flat_index(14)]);
  case 9:
    return (n_a.vd[flat_index(15)] + n_a.vd[flat_index(16)]) -
           (n_a.vd[flat_index(17)] + n_a.vd[flat_index(18)]);
  case 10:
    return -2.0f * (n_a.vd[flat_index(1)] - n_a.vd[flat_index(2)]) +
           (n_a.vd[flat_index(7)] - n_a.vd[flat_index(8)]) +
           (n_a.vd[flat_index(9)] - n_a.vd[flat_index(10)]) +
           (n_a.vd[flat_index(11)] - n_a.vd[flat_index(12)]) +
           (n_a.vd[flat_index(13)] - n_a.vd[flat_index(14)]);
  case 11:
    return -2.0f * (n_a.vd[flat_index(3)] - n_a.vd[flat_index(4)]) +
           (n_a.vd[flat_index(7)] - n_a.vd[flat_index(8)]) -
           (n_a.vd[flat_index(9)] - n_a.vd[flat_index(10)]) +
           (n_a.vd[flat_index(15)] - n_a.vd[flat_index(16)]) +
           (n_a.vd[flat_index(17)] - n_a.vd[flat_index(18)]);
  case 12:
    return -2.0f * (n_a.vd[flat_index(5)] - n_a.vd[flat_index(6)]) +
           (n_a.vd[flat_index(11)] - n_a.vd[flat_index(12)]) -
           (n_a.vd[flat_index(13)] - n_a.vd[flat_index(14)]) +
           (n_a.vd[flat_index(15)] - n_a.vd[flat_index(16)]) -
           (n_a.vd[flat_index(17)] - n_a.vd[flat_index(18)]);
  case 13:
    return (n_a.vd[flat_index(7)] - n_a.vd[flat_index(8)]) +
           (n_a.vd[flat_index(9)] - n_a.vd[flat_index(10)]) -
           (n_a.vd[flat_index(11)] - n_a.vd[flat_index(12)]) -
           (n_a.vd[flat_index(13)] - n_a.vd[flat_index(14)]);
  case 14:
    return (n_a.vd[flat_index(7)] - n_a.vd[flat_index(8)]) -
           (n_a.vd[flat_index(9)] - n_a.vd[flat_index(10)]) -
           (n_a.vd[flat_index(15)] - n_a.vd[flat_index(16)]) -
           (n_a.vd[flat_index(17)] - n_a.vd[flat_index(18)]);
  case 15:
    return (n_a.vd[flat_index(11)] - n_a.vd[flat_index(12)]) -
           (n_a.vd[flat_index(13)] - n_a.vd[flat_index(14)]) -
           (n_a.vd[flat_index(15)] - n_a.vd[flat_index(16)]) +
           (n_a.vd[flat_index(17)] - n_a.vd[flat_index(18)]);
  case 16:
    return n_a.vd[flat_index(0)] + n_a.vd[flat_index(7)] +
           n_a.vd[flat_index(8)] + n_a.vd[flat_index(9)] +
           n_a.vd[flat_index(10)] + n_a.vd[flat_index(11)] +
           n_a.vd[flat_index(12)] + n_a.vd[flat_index(13)] +
           n_a.vd[flat_index(14)] + n_a.vd[flat_index(15)] +
           n_a.vd[flat_index(16)] + n_a.vd[flat_index(17)] +
           n_a.vd[flat_index(18)] -
           2.0f * ((n_a.vd[flat_index(1)] + n_a.vd[flat_index(2)]) +
                   (n_a.vd[flat_index(3)] + n_a.vd[flat_index(4)]) +
                   (n_a.vd[flat_index(5)] + n_a.vd[flat_index(6)]));
  case 17:
    return -(n_a.vd[flat_index(1)] + n_a.vd[flat_index(2)]) +
           (n_a.vd[flat_index(3)] + n_a.vd[flat_index(4)]) +
           (n_a.vd[flat_index(11)] + n_a.vd[flat_index(12)]) +
           (n_a.vd[flat_index(13)] + n_a.vd[flat_index(14)]) -
           (n_a.vd[flat_index(15)] + n_a.vd[flat_index(16)]) -
           (n_a.vd[flat_index(17)] + n_a.vd[flat_index(18)]);
  case 18:
    return -(n_a.vd[flat_index(1)] + n_a.vd[flat_index(2)]) -
           (n_a.vd[flat_index(3)] + n_a.vd[flat_index(4)]) -
           (n_a.vd[flat_index(11)] + n_a.vd[flat_index(12)]) -
           (n_a.vd[flat_index(13)] + n_a.vd[flat_index(14)]) -
           (n_a.vd[flat_index(15)] + n_a.vd[flat_index(16)]) -
           (n_a.vd[flat_index(17)] + n_a.vd[flat_index(18)]) +
           2.0f * ((n_a.vd[flat_index(5)] + n_a.vd[flat_index(6)]) +
                   (n_a.vd[flat_index(7)] + n_a.vd[flat_index(8)]) +
                   (n_a.vd[flat_index(9)] + n_a.vd[flat_index(10)]));
  }
  return 0.0;
}

/** Calculate modes from the velocity densities (space-transform)
 *  @param[in]  n_a     Local node residing in array a
 *  @param[in]  index   Node index / thread index
 *  @param[out] mode    Local register values mode
 */
__device__ void calc_m_from_n(LB_nodes_gpu n_a, unsigned int index,
                              Utils::Array<float, 19> &mode) {
  // The following convention is used:
  // The $\hat{c}_i$ form B. Duenweg's paper are given by:

  /* c_0  = { 0, 0, 0}
     c_1  = { 1, 0, 0}
     c_2  = {-1, 0, 0}
     c_3  = { 0, 1, 0}
     c_4  = { 0,-1, 0}
     c_5  = { 0, 0, 1}
     c_6  = { 0, 0,-1}
     c_7  = { 1, 1, 0}
     c_8  = {-1,-1, 0}
     c_9  = { 1,-1, 0}
     c_10 = {-1, 1, 0}
     c_11 = { 1, 0, 1}
     c_12 = {-1, 0,-1}
     c_13 = { 1, 0,-1}
     c_14 = {-1, 0, 1}
     c_15 = { 0, 1, 1}
     c_16 = { 0,-1,-1}
     c_17 = { 0, 1,-1}
     c_18 = { 0,-1, 1} */

  // The basis vectors (modes) are constructed as follows
  // $m_k = \sum_{i} e_{ki} n_{i}$, where the $e_{ki}$ form a
  // linear transformation (matrix) that is given by

  /* $e{ 0,i} = 1$
     $e{ 1,i} = c_{i,x}$
     $e{ 2,i} = c_{i,y}$
     $e{ 3,i} = c_{i,z}$
     $e{ 4,i} = c_{i}^2 - 1$
     $e{ 5,i} = c_{i,x}^2 - c_{i,y}^2$
     $e{ 6,i} = c_{i}^2 - 3*c_{i,z}^2$
     $e{ 7,i} = c_{i,x}*c_{i,y}$
     $e{ 8,i} = c_{i,x}*c_{i,z}$
     $e{ 9,i} = c_{i,y}*c_{i,z}$
     $e{10,i} = (3*c_{i}^2 - 5)*c_{i,x}$
     $e{11,i} = (3*c_{i}^2 - 5)*c_{i,y}$
     $e{12,i} = (3*c_{i}^2 - 5)*c_{i,z}$
     $e{13,i} = (c_{i,y}^2 - c_{i,z}^2)*c_{i,x}$
     $e{14,i} = (c_{i,x}^2 - c_{i,z}^2)*c_{i,y}$
     $e{15,i} = (c_{i,x}^2 - c_{i,y}^2)*c_{i,z}$
     $e{16,i} = 3*c_{i}^2^2 - 6*c_{i}^2 + 1$
     $e{17,i} = (2*c_{i}^2 - 3)*(c_{i,x}^2 - c_{i,y}^2)$
     $e{18,i} = (2*c_{i}^2 - 3)*(c_{i}^2 - 3*c_{i,z}^2)$ */

  // Such that the transformation matrix is given by

  /* {{ 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1},
      { 0, 1,-1, 0, 0, 0, 0, 1,-1, 1,-1, 1,-1, 1,-1, 0, 0, 0, 0},
      { 0, 0, 0, 1,-1, 0, 0, 1,-1,-1, 1, 0, 0, 0, 0, 1,-1, 1,-1},
      { 0, 0, 0, 0, 0, 1,-1, 0, 0, 0, 0, 1,-1,-1, 1, 1,-1,-1, 1},
      {-1, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1},
      { 0, 1, 1,-1,-1, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1,-1,-1,-1,-1},
      { 0, 1, 1, 1, 1,-2,-2, 2, 2, 2, 2,-1,-1,-1,-1,-1,-1,-1,-1},
      { 0, 0, 0, 0, 0, 0, 0, 1, 1,-1,-1, 0, 0, 0, 0, 0, 0, 0, 0},
      { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1,-1,-1, 0, 0, 0, 0},
      { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1,-1,-1},
      { 0,-2, 2, 0, 0, 0, 0, 1,-1, 1,-1, 1,-1, 1,-1, 0, 0, 0, 0},
      { 0, 0, 0,-2, 2, 0, 0, 1,-1,-1, 1, 0, 0, 0, 0, 1,-1, 1,-1},
      { 0, 0, 0, 0, 0,-2, 2, 0, 0, 0, 0, 1,-1,-1, 1, 1,-1,-1, 1},
      { 0, 0, 0, 0, 0, 0, 0, 1,-1, 1,-1,-1, 1,-1, 1, 0, 0, 0, 0},
      { 0, 0, 0, 0, 0, 0, 0, 1,-1,-1, 1, 0, 0, 0, 0,-1, 1,-1, 1},
      { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1,-1,-1, 1,-1, 1, 1,-1},
      { 1,-2,-2,-2,-2,-2,-2, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1},
      { 0,-1,-1, 1, 1, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1,-1,-1,-1,-1},
      { 0,-1,-1,-1,-1, 2, 2, 2, 2, 2, 2,-1,-1,-1,-1,-1,-1,-1,-1}} */

  // With weights

  /* q^{c_{i}} = { 1/3, 1/18, 1/18, 1/18,
                  1/18, 1/18, 1/18, 1/36,
                  1/36, 1/36, 1/36, 1/36,
                  1/36, 1/36, 1/36, 1/36,
                  1/36, 1/36, 1/36 } */

  // Which makes the transformation satisfy the following
  // orthogonality condition:
  // \sum_{i} q^{c_{i}} e_{ki} e_{li} = w_{k} \delta_{kl},
  // where the weights are:

  /* w_{i} = {  1, 1/3, 1/3, 1/3,
              2/3, 4/9, 4/3, 1/9,
              1/9, 1/9, 2/3, 2/3,
              2/3, 2/9, 2/9, 2/9,
                2, 4/9, 4/3 } */
  for (int i = 0; i < 19; ++i) {
    mode[i] = calc_mode_x_from_n(n_a, index, i);
  }
}

__device__ void reset_LB_force_densities(unsigned int index,
                                         LB_node_force_density_gpu node_f,
                                         bool buffer = true) {
#if defined(VIRTUAL_SITES_INERTIALESS_TRACERS) || defined(EK_DEBUG)
  // Store backup of the node forces
  if (buffer) {
    node_f.force_density_buf[0 * para->number_of_nodes + index] =
        node_f.force_density[0 * para->number_of_nodes + index];
    node_f.force_density_buf[1 * para->number_of_nodes + index] =
        node_f.force_density[1 * para->number_of_nodes + index];
    node_f.force_density_buf[2 * para->number_of_nodes + index] =
        node_f.force_density[2 * para->number_of_nodes + index];
  }
#endif

#ifdef EXTERNAL_FORCES
  if (para->external_force_density) {
    node_f.force_density[0 * para->number_of_nodes + index] =
        para->ext_force_density[0];
    node_f.force_density[1 * para->number_of_nodes + index] =
        para->ext_force_density[1];
    node_f.force_density[2 * para->number_of_nodes + index] =
        para->ext_force_density[2];
  } else {
    node_f.force_density[0 * para->number_of_nodes + index] = 0.0f;
    node_f.force_density[1 * para->number_of_nodes + index] = 0.0f;
    node_f.force_density[2 * para->number_of_nodes + index] = 0.0f;
  }
#else
  /* reset force */
  node_f.force_density[0 * para->number_of_nodes + index] = 0.0f;
  node_f.force_density[1 * para->number_of_nodes + index] = 0.0f;
  node_f.force_density[2 * para->number_of_nodes + index] = 0.0f;
#endif
}

__global__ void
reset_LB_force_densities_kernel(LB_node_force_density_gpu node_f,
                                bool buffer = true) {
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x +
                       blockDim.x * blockIdx.x + threadIdx.x;

  if (index < para->number_of_nodes)
    reset_LB_force_densities(index, node_f, buffer);
}

void reset_LB_force_densities_GPU(bool buffer) {
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x =
      (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /
      (threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(reset_LB_force_densities_kernel, dim_grid, threads_per_block,
             node_f, buffer);
}

/**
 *  @param[in]  mode    Local register values mode
 *  @param[in]  index   Node index / thread index
 *  @param[in]  node_f  Local node force
 *  @param[out] d_v     Local device values
 */
__device__ void update_rho_v(Utils::Array<float, 19> const &mode,
                             unsigned int index,
                             LB_node_force_density_gpu node_f,
                             LB_rho_v_gpu *d_v) {
  float Rho_tot = 0.0f;
  float u_tot[3] = {0.0f, 0.0f, 0.0f};

  /* re-construct the real density
   * remember that the populations are stored as differences to their
   * equilibrium value */

  d_v[index].rho = mode[0] + para->rho;
  Rho_tot += mode[0] + para->rho;
  u_tot[0] += mode[1];
  u_tot[1] += mode[2];
  u_tot[2] += mode[3];

  /** if forces are present, the momentum density is redefined to
   * include one half-step of the force action.  See the
   * Chapman-Enskog expansion in [Ladd & Verberg]. */

  u_tot[0] += 0.5f * node_f.force_density[0 * para->number_of_nodes + index];
  u_tot[1] += 0.5f * node_f.force_density[1 * para->number_of_nodes + index];
  u_tot[2] += 0.5f * node_f.force_density[2 * para->number_of_nodes + index];

  u_tot[0] /= Rho_tot;
  u_tot[1] /= Rho_tot;
  u_tot[2] /= Rho_tot;

  d_v[index].v[0] = u_tot[0];
  d_v[index].v[1] = u_tot[1];
  d_v[index].v[2] = u_tot[2];
}

/** lb_relax_modes, means collision update of the modes
 *  @param[in] index     Node index / thread index
 *  @param[in,out] mode  Local register values mode
 *  @param[in] node_f    Local node force
 *  @param[in,out] d_v   Local device values
 */
__device__ void relax_modes(Utils::Array<float, 19> &mode, unsigned int index,
                            LB_node_force_density_gpu node_f,
                            LB_rho_v_gpu *d_v) {
  float u_tot[3] = {0.0f, 0.0f, 0.0f};

  update_rho_v(mode, index, node_f, d_v);

  u_tot[0] = d_v[index].v[0];
  u_tot[1] = d_v[index].v[1];
  u_tot[2] = d_v[index].v[2];

  float Rho;
  float j[3];
  Utils::Array<float, 6> modes_from_pi_eq;

  Rho = mode[0] + para->rho;
  j[0] = Rho * u_tot[0];
  j[1] = Rho * u_tot[1];
  j[2] = Rho * u_tot[2];

  /** equilibrium part of the stress modes (eq13 schiller) */

  modes_from_pi_eq[0] = ((j[0] * j[0]) + (j[1] * j[1]) + (j[2] * j[2])) / Rho;
  modes_from_pi_eq[1] = ((j[0] * j[0]) - (j[1] * j[1])) / Rho;
  modes_from_pi_eq[2] =
      (((j[0] * j[0]) + (j[1] * j[1]) + (j[2] * j[2])) - 3.0f * (j[2] * j[2])) /
      Rho;
  modes_from_pi_eq[3] = j[0] * j[1] / Rho;
  modes_from_pi_eq[4] = j[0] * j[2] / Rho;
  modes_from_pi_eq[5] = j[1] * j[2] / Rho;

  /** relax the stress modes (eq14 schiller) */

  mode[4] =
      modes_from_pi_eq[0] + para->gamma_bulk * (mode[4] - modes_from_pi_eq[0]);
  mode[5] =
      modes_from_pi_eq[1] + para->gamma_shear * (mode[5] - modes_from_pi_eq[1]);
  mode[6] =
      modes_from_pi_eq[2] + para->gamma_shear * (mode[6] - modes_from_pi_eq[2]);
  mode[7] =
      modes_from_pi_eq[3] + para->gamma_shear * (mode[7] - modes_from_pi_eq[3]);
  mode[8] =
      modes_from_pi_eq[4] + para->gamma_shear * (mode[8] - modes_from_pi_eq[4]);
  mode[9] =
      modes_from_pi_eq[5] + para->gamma_shear * (mode[9] - modes_from_pi_eq[5]);

  /** relax the ghost modes (project them out) */
  /** ghost modes have no equilibrium part due to orthogonality */

  mode[10] = para->gamma_odd * mode[10];
  mode[11] = para->gamma_odd * mode[11];
  mode[12] = para->gamma_odd * mode[12];
  mode[13] = para->gamma_odd * mode[13];
  mode[14] = para->gamma_odd * mode[14];
  mode[15] = para->gamma_odd * mode[15];
  mode[16] = para->gamma_even * mode[16];
  mode[17] = para->gamma_even * mode[17];
  mode[18] = para->gamma_even * mode[18];
}

/** Thermalization of the modes with Gaussian random numbers
 *  @param[in] index     Node index / thread index
 *  @param[in,out] mode  Local register values mode
 *  @param[in] philox_counter
 */
__device__ void thermalize_modes(Utils::Array<float, 19> &mode,
                                 unsigned int index, uint64_t philox_counter) {
  float Rho;
  float4 random_floats;
  /** mass mode */
  Rho = mode[0] + para->rho;

  /* momentum modes */

  /* stress modes */
  random_floats = random_wrapper_philox(index, 4, philox_counter);
  mode[4] += sqrtf(Rho * (para->mu * (2.0f / 3.0f) *
                          (1.0f - (para->gamma_bulk * para->gamma_bulk)))) *
             (random_floats.w - 0.5f) * sqrt12;
  mode[5] += sqrtf(Rho * (para->mu * (4.0f / 9.0f) *
                          (1.0f - (para->gamma_shear * para->gamma_shear)))) *
             (random_floats.x - 0.5f) * sqrt12;

  mode[6] += sqrtf(Rho * (para->mu * (4.0f / 3.0f) *
                          (1.0f - (para->gamma_shear * para->gamma_shear)))) *
             (random_floats.y - 0.5f) * sqrt12;
  mode[7] += sqrtf(Rho * (para->mu * (1.0f / 9.0f) *
                          (1.0f - (para->gamma_shear * para->gamma_shear)))) *
             (random_floats.z - 0.5f) * sqrt12;

  random_floats = random_wrapper_philox(index, 8, philox_counter);
  mode[8] += sqrtf(Rho * (para->mu * (1.0f / 9.0f) *
                          (1.0f - (para->gamma_shear * para->gamma_shear)))) *
             (random_floats.w - 0.5f) * sqrt12;
  mode[9] += sqrtf(Rho * (para->mu * (1.0f / 9.0f) *
                          (1.0f - (para->gamma_shear * para->gamma_shear)))) *
             (random_floats.x - 0.5f) * sqrt12;

  /* ghost modes */
  mode[10] += sqrtf(Rho * (para->mu * (2.0f / 3.0f) *
                           (1.0f - (para->gamma_odd * para->gamma_odd)))) *
              (random_floats.y - 0.5f) * sqrt12;
  mode[11] += sqrtf(Rho * (para->mu * (2.0f / 3.0f) *
                           (1.0f - (para->gamma_odd * para->gamma_odd)))) *
              (random_floats.z - 0.5f) * sqrt12;

  random_floats = random_wrapper_philox(index, 12, philox_counter);
  mode[12] += sqrtf(Rho * (para->mu * (2.0f / 3.0f) *
                           (1.0f - (para->gamma_odd * para->gamma_odd)))) *
              (random_floats.w - 0.5f) * sqrt12;
  mode[13] += sqrtf(Rho * (para->mu * (2.0f / 9.0f) *
                           (1.0f - (para->gamma_odd * para->gamma_odd)))) *
              (random_floats.x - 0.5f) * sqrt12;

  mode[14] += sqrtf(Rho * (para->mu * (2.0f / 9.0f) *
                           (1.0f - (para->gamma_odd * para->gamma_odd)))) *
              (random_floats.y - 0.5f) * sqrt12;
  mode[15] += sqrtf(Rho * (para->mu * (2.0f / 9.0f) *
                           (1.0f - (para->gamma_odd * para->gamma_odd)))) *
              (random_floats.z - 0.5f) * sqrt12;

  random_floats = random_wrapper_philox(index, 16, philox_counter);
  mode[16] += sqrtf(Rho * (para->mu * (2.0f) *
                           (1.0f - (para->gamma_even * para->gamma_even)))) *
              (random_floats.w - 0.5f) * sqrt12;
  mode[17] += sqrtf(Rho * (para->mu * (4.0f / 9.0f) *
                           (1.0f - (para->gamma_even * para->gamma_even)))) *
              (random_floats.x - 0.5f) * sqrt12;

  mode[18] += sqrtf(Rho * (para->mu * (4.0f / 3.0f) *
                           (1.0f - (para->gamma_even * para->gamma_even)))) *
              (random_floats.y - 0.5f) * sqrt12;
}

/*-------------------------------------------------------*/
/** Normalization of the modes need before back-transformation into velocity
 *  space
 *  @param[in,out] mode  Local register values mode
 */
__device__ void normalize_modes(Utils::Array<float, 19> &mode) {
  /* normalization factors enter in the back transformation */
  mode[0] *= 1.0f;
  mode[1] *= 3.0f;
  mode[2] *= 3.0f;
  mode[3] *= 3.0f;
  mode[4] *= 3.0f / 2.0f;
  mode[5] *= 9.0f / 4.0f;
  mode[6] *= 3.0f / 4.0f;
  mode[7] *= 9.0f;
  mode[8] *= 9.0f;
  mode[9] *= 9.0f;
  mode[10] *= 3.0f / 2.0f;
  mode[11] *= 3.0f / 2.0f;
  mode[12] *= 3.0f / 2.0f;
  mode[13] *= 9.0f / 2.0f;
  mode[14] *= 9.0f / 2.0f;
  mode[15] *= 9.0f / 2.0f;
  mode[16] *= 1.0f / 2.0f;
  mode[17] *= 9.0f / 4.0f;
  mode[18] *= 3.0f / 4.0f;
}

/*-------------------------------------------------------*/
/** Back-transformation from modespace to densityspace and streaming with
 *  the push method using pbc
 *  @param[in]  index  Node index / thread index
 *  @param[in]  mode   Local register values mode
 *  @param[out] n_b    Local node residing in array b
 */
__device__ void calc_n_from_modes_push(LB_nodes_gpu n_b,
                                       Utils::Array<float, 19> const &mode,
                                       unsigned int index) {
  auto const xyz = index_to_xyz(index);
  unsigned int x = xyz.x;
  unsigned int y = xyz.y;
  unsigned int z = xyz.z;

  n_b.vd[0 * para->number_of_nodes + x + para->dim_x * y +
         para->dim_x * para->dim_y * z] =
      1.0f / 3.0f * (mode[0] - mode[4] + mode[16]);

  n_b.vd[1 * para->number_of_nodes + (x + 1) % para->dim_x + para->dim_x * y +
         para->dim_x * para->dim_y * z] =
      1.0f / 18.0f *
      (mode[0] + mode[1] + mode[5] + mode[6] - mode[17] - mode[18] -
       2.0f * (mode[10] + mode[16]));

  n_b.vd[2 * para->number_of_nodes + (para->dim_x + x - 1) % para->dim_x +
         para->dim_x * y + para->dim_x * para->dim_y * z] =
      1.0f / 18.0f *
      (mode[0] - mode[1] + mode[5] + mode[6] - mode[17] - mode[18] +
       2.0f * (mode[10] - mode[16]));

  n_b.vd[3 * para->number_of_nodes + x + para->dim_x * ((y + 1) % para->dim_y) +
         para->dim_x * para->dim_y * z] =
      1.0f / 18.0f *
      (mode[0] + mode[2] - mode[5] + mode[6] + mode[17] - mode[18] -
       2.0f * (mode[11] + mode[16]));

  n_b.vd[4 * para->number_of_nodes + x +
         para->dim_x * ((para->dim_y + y - 1) % para->dim_y) +
         para->dim_x * para->dim_y * z] =
      1.0f / 18.0f *
      (mode[0] - mode[2] - mode[5] + mode[6] + mode[17] - mode[18] +
       2.0f * (mode[11] - mode[16]));

  n_b.vd[5 * para->number_of_nodes + x + para->dim_x * y +
         para->dim_x * para->dim_y * ((z + 1) % para->dim_z)] =
      1.0f / 18.0f *
      (mode[0] + mode[3] - 2.0f * (mode[6] + mode[12] + mode[16] - mode[18]));

  n_b.vd[6 * para->number_of_nodes + x + para->dim_x * y +
         para->dim_x * para->dim_y * ((para->dim_z + z - 1) % para->dim_z)] =
      1.0f / 18.0f *
      (mode[0] - mode[3] - 2.0f * (mode[6] - mode[12] + mode[16] - mode[18]));

  n_b.vd[7 * para->number_of_nodes + (x + 1) % para->dim_x +
         para->dim_x * ((y + 1) % para->dim_y) +
         para->dim_x * para->dim_y * z] =
      1.0f / 36.0f *
      (mode[0] + mode[1] + mode[2] + mode[4] + 2.0f * mode[6] + mode[7] +
       mode[10] + mode[11] + mode[13] + mode[14] + mode[16] + 2.0f * mode[18]);

  n_b.vd[8 * para->number_of_nodes + (para->dim_x + x - 1) % para->dim_x +
         para->dim_x * ((para->dim_y + y - 1) % para->dim_y) +
         para->dim_x * para->dim_y * z] =
      1.0f / 36.0f *
      (mode[0] - mode[1] - mode[2] + mode[4] + 2.0f * mode[6] + mode[7] -
       mode[10] - mode[11] - mode[13] - mode[14] + mode[16] + 2.0f * mode[18]);

  n_b.vd[9 * para->number_of_nodes + (x + 1) % para->dim_x +
         para->dim_x * ((para->dim_y + y - 1) % para->dim_y) +
         para->dim_x * para->dim_y * z] =
      1.0f / 36.0f *
      (mode[0] + mode[1] - mode[2] + mode[4] + 2.0f * mode[6] - mode[7] +
       mode[10] - mode[11] + mode[13] - mode[14] + mode[16] + 2.0f * mode[18]);

  n_b.vd[10 * para->number_of_nodes + (para->dim_x + x - 1) % para->dim_x +
         para->dim_x * ((y + 1) % para->dim_y) +
         para->dim_x * para->dim_y * z] =
      1.0f / 36.0f *
      (mode[0] - mode[1] + mode[2] + mode[4] + 2.0f * mode[6] - mode[7] -
       mode[10] + mode[11] - mode[13] + mode[14] + mode[16] + 2.0f * mode[18]);

  n_b.vd[11 * para->number_of_nodes + (x + 1) % para->dim_x + para->dim_x * y +
         para->dim_x * para->dim_y * ((z + 1) % para->dim_z)] =
      1.0f / 36.0f *
      (mode[0] + mode[1] + mode[3] + mode[4] + mode[5] - mode[6] + mode[8] +
       mode[10] + mode[12] - mode[13] + mode[15] + mode[16] + mode[17] -
       mode[18]);

  n_b.vd[12 * para->number_of_nodes + (para->dim_x + x - 1) % para->dim_x +
         para->dim_x * y +
         para->dim_x * para->dim_y * ((para->dim_z + z - 1) % para->dim_z)] =
      1.0f / 36.0f *
      (mode[0] - mode[1] - mode[3] + mode[4] + mode[5] - mode[6] + mode[8] -
       mode[10] - mode[12] + mode[13] - mode[15] + mode[16] + mode[17] -
       mode[18]);

  n_b.vd[13 * para->number_of_nodes + (x + 1) % para->dim_x + para->dim_x * y +
         para->dim_x * para->dim_y * ((para->dim_z + z - 1) % para->dim_z)] =
      1.0f / 36.0f *
      (mode[0] + mode[1] - mode[3] + mode[4] + mode[5] - mode[6] - mode[8] +
       mode[10] - mode[12] - mode[13] - mode[15] + mode[16] + mode[17] -
       mode[18]);

  n_b.vd[14 * para->number_of_nodes + (para->dim_x + x - 1) % para->dim_x +
         para->dim_x * y +
         para->dim_x * para->dim_y * ((z + 1) % para->dim_z)] =
      1.0f / 36.0f *
      (mode[0] - mode[1] + mode[3] + mode[4] + mode[5] - mode[6] - mode[8] -
       mode[10] + mode[12] + mode[13] + mode[15] + mode[16] + mode[17] -
       mode[18]);

  n_b.vd[15 * para->number_of_nodes + x +
         para->dim_x * ((y + 1) % para->dim_y) +
         para->dim_x * para->dim_y * ((z + 1) % para->dim_z)] =
      1.0f / 36.0f *
      (mode[0] + mode[2] + mode[3] + mode[4] - mode[5] - mode[6] + mode[9] +
       mode[11] + mode[12] - mode[14] - mode[15] + mode[16] - mode[17] -
       mode[18]);

  n_b.vd[16 * para->number_of_nodes + x +
         para->dim_x * ((para->dim_y + y - 1) % para->dim_y) +
         para->dim_x * para->dim_y * ((para->dim_z + z - 1) % para->dim_z)] =
      1.0f / 36.0f *
      (mode[0] - mode[2] - mode[3] + mode[4] - mode[5] - mode[6] + mode[9] -
       mode[11] - mode[12] + mode[14] + mode[15] + mode[16] - mode[17] -
       mode[18]);

  n_b.vd[17 * para->number_of_nodes + x +
         para->dim_x * ((y + 1) % para->dim_y) +
         para->dim_x * para->dim_y * ((para->dim_z + z - 1) % para->dim_z)] =
      1.0f / 36.0f *
      (mode[0] + mode[2] - mode[3] + mode[4] - mode[5] - mode[6] - mode[9] +
       mode[11] - mode[12] - mode[14] + mode[15] + mode[16] - mode[17] -
       mode[18]);

  n_b.vd[18 * para->number_of_nodes + x +
         para->dim_x * ((para->dim_y + y - 1) % para->dim_y) +
         para->dim_x * para->dim_y * ((z + 1) % para->dim_z)] =
      1.0f / 36.0f *
      (mode[0] - mode[2] + mode[3] + mode[4] - mode[5] - mode[6] - mode[9] -
       mode[11] + mode[12] + mode[14] - mode[15] + mode[16] - mode[17] -
       mode[18]);
}

/** Bounce back boundary conditions.
 *
 *  The populations that have propagated into a boundary node
 *  are bounced back to the node they came from. This results
 *  in no slip boundary conditions.
 *
 *  [cf. Ladd and Verberg, J. Stat. Phys. 104(5/6):1191-1251, 2001]
 *  @param[in]  index   Node index / thread index
 *  @param[in]  n_curr  Local node receiving the current node field
 *  @param[in]  lb_boundary_velocity  Constant velocity at the boundary,
 *                                    set by the user
 *  @param[out] lb_boundary_force     Force on the boundary nodes
 */
__device__ void bounce_back_boundaries(LB_nodes_gpu n_curr, unsigned int index,
                                       float *lb_boundary_velocity,
                                       float *lb_boundary_force) {
  int c[3];
  float v[3];
  float shift, weight, pop_to_bounce_back;
  float boundary_force[3] = {0.0f, 0.0f, 0.0f};
  size_t to_index, to_index_x, to_index_y, to_index_z;
  int population, inverse;
  int boundary_index;

  boundary_index = n_curr.boundary[index];
  if (boundary_index != 0) {
    v[0] = lb_boundary_velocity[3 * (boundary_index - 1) + 0];
    v[1] = lb_boundary_velocity[3 * (boundary_index - 1) + 1];
    v[2] = lb_boundary_velocity[3 * (boundary_index - 1) + 2];

    auto const xyz = index_to_xyz(index);

    unsigned int x = xyz.x;
    unsigned int y = xyz.y;
    unsigned int z = xyz.z;

    /** store vd temporary in second lattice to avoid race conditions */

    // TODO : PUT IN EQUILIBRIUM CONTRIBUTION TO THE BOUNCE-BACK DENSITY FOR THE
    // BOUNDARY FORCE
    // TODO : INITIALIZE BOUNDARY FORCE PROPERLY, HAS NONZERO ELEMENTS IN FIRST
    // STEP
    // TODO : SET INTERNAL BOUNDARY NODE VALUES TO ZERO

#define BOUNCEBACK()                                                           \
  shift = 2.0f / para->agrid * para->rho * 3.0f * weight * para->tau *         \
          (v[0] * c[0] + v[1] * c[1] + v[2] * c[2]);                           \
  pop_to_bounce_back = n_curr.vd[population * para->number_of_nodes + index];  \
  to_index_x = (x + c[0] + para->dim_x) % para->dim_x;                         \
  to_index_y = (y + c[1] + para->dim_y) % para->dim_y;                         \
  to_index_z = (z + c[2] + para->dim_z) % para->dim_z;                         \
  to_index = to_index_x + para->dim_x * to_index_y +                           \
             para->dim_x * para->dim_y * to_index_z;                           \
  if (n_curr.boundary[to_index] == 0) {                                        \
    boundary_force[0] += (2.0f * pop_to_bounce_back + shift) * c[0] /          \
                         para->tau / para->tau / para->agrid;                  \
    boundary_force[1] += (2.0f * pop_to_bounce_back + shift) * c[1] /          \
                         para->tau / para->tau / para->agrid;                  \
    boundary_force[2] += (2.0f * pop_to_bounce_back + shift) * c[2] /          \
                         para->tau / para->tau / para->agrid;                  \
    n_curr.vd[inverse * para->number_of_nodes + to_index] =                    \
        pop_to_bounce_back + shift;                                            \
  }

    // the resting population does nothing, i.e., population 0.
    c[0] = 1;
    c[1] = 0;
    c[2] = 0;
    weight = 1. / 18.;
    population = 2;
    inverse = 1;
    BOUNCEBACK();

    c[0] = -1;
    c[1] = 0;
    c[2] = 0;
    weight = 1. / 18.;
    population = 1;
    inverse = 2;
    BOUNCEBACK();

    c[0] = 0;
    c[1] = 1;
    c[2] = 0;
    weight = 1. / 18.;
    population = 4;
    inverse = 3;
    BOUNCEBACK();

    c[0] = 0;
    c[1] = -1;
    c[2] = 0;
    weight = 1. / 18.;
    population = 3;
    inverse = 4;
    BOUNCEBACK();

    c[0] = 0;
    c[1] = 0;
    c[2] = 1;
    weight = 1. / 18.;
    population = 6;
    inverse = 5;
    BOUNCEBACK();

    c[0] = 0;
    c[1] = 0;
    c[2] = -1;
    weight = 1. / 18.;
    population = 5;
    inverse = 6;
    BOUNCEBACK();

    c[0] = 1;
    c[1] = 1;
    c[2] = 0;
    weight = 1. / 36.;
    population = 8;
    inverse = 7;
    BOUNCEBACK();

    c[0] = -1;
    c[1] = -1;
    c[2] = 0;
    weight = 1. / 36.;
    population = 7;
    inverse = 8;
    BOUNCEBACK();

    c[0] = 1;
    c[1] = -1;
    c[2] = 0;
    weight = 1. / 36.;
    population = 10;
    inverse = 9;
    BOUNCEBACK();

    c[0] = -1;
    c[1] = 1;
    c[2] = 0;
    weight = 1. / 36.;
    population = 9;
    inverse = 10;
    BOUNCEBACK();

    c[0] = 1;
    c[1] = 0;
    c[2] = 1;
    weight = 1. / 36.;
    population = 12;
    inverse = 11;
    BOUNCEBACK();

    c[0] = -1;
    c[1] = 0;
    c[2] = -1;
    weight = 1. / 36.;
    population = 11;
    inverse = 12;
    BOUNCEBACK();

    c[0] = 1;
    c[1] = 0;
    c[2] = -1;
    weight = 1. / 36.;
    population = 14;
    inverse = 13;
    BOUNCEBACK();

    c[0] = -1;
    c[1] = 0;
    c[2] = 1;
    weight = 1. / 36.;
    population = 13;
    inverse = 14;
    BOUNCEBACK();

    c[0] = 0;
    c[1] = 1;
    c[2] = 1;
    weight = 1. / 36.;
    population = 16;
    inverse = 15;
    BOUNCEBACK();

    c[0] = 0;
    c[1] = -1;
    c[2] = -1;
    weight = 1. / 36.;
    population = 15;
    inverse = 16;
    BOUNCEBACK();

    c[0] = 0;
    c[1] = 1;
    c[2] = -1;
    weight = 1. / 36.;
    population = 18;
    inverse = 17;
    BOUNCEBACK();

    c[0] = 0;
    c[1] = -1;
    c[2] = 1;
    weight = 1. / 36.;
    population = 17;
    inverse = 18;
    BOUNCEBACK();

    atomicAdd(&lb_boundary_force[3 * (n_curr.boundary[index] - 1) + 0],
              boundary_force[0]);
    atomicAdd(&lb_boundary_force[3 * (n_curr.boundary[index] - 1) + 1],
              boundary_force[1]);
    atomicAdd(&lb_boundary_force[3 * (n_curr.boundary[index] - 1) + 2],
              boundary_force[2]);
  }
}

/** Add external forces within the modespace, needed for particle-interaction
 *  @param[in]     index   Node index / thread index
 *  @param[in,out] mode    Local register values mode
 *  @param[in,out] node_f  Local node force
 *  @param[in]     d_v     Local device values
 */
__device__ void apply_forces(unsigned int index, Utils::Array<float, 19> &mode,
                             LB_node_force_density_gpu node_f,
                             LB_rho_v_gpu *d_v) {
  float u[3] = {0.0f, 0.0f, 0.0f}, C[6] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
  /* Note: the values d_v were calculated in relax_modes() */

  u[0] = d_v[index].v[0];
  u[1] = d_v[index].v[1];
  u[2] = d_v[index].v[2];

  C[0] += (1.0f + para->gamma_bulk) * u[0] *
              node_f.force_density[0 * para->number_of_nodes + index] +
          1.0f / 3.0f * (para->gamma_bulk - para->gamma_shear) *
              (u[0] * node_f.force_density[0 * para->number_of_nodes + index] +
               u[1] * node_f.force_density[1 * para->number_of_nodes + index] +
               u[2] * node_f.force_density[2 * para->number_of_nodes + index]);

  C[2] += (1.0f + para->gamma_bulk) * u[1] *
              node_f.force_density[1 * para->number_of_nodes + index] +
          1.0f / 3.0f * (para->gamma_bulk - para->gamma_shear) *
              (u[0] * node_f.force_density[0 * para->number_of_nodes + index] +
               u[1] * node_f.force_density[1 * para->number_of_nodes + index] +
               u[2] * node_f.force_density[2 * para->number_of_nodes + index]);

  C[5] += (1.0f + para->gamma_bulk) * u[2] *
              node_f.force_density[2 * para->number_of_nodes + index] +
          1.0f / 3.0f * (para->gamma_bulk - para->gamma_shear) *
              (u[0] * node_f.force_density[0 * para->number_of_nodes + index] +
               u[1] * node_f.force_density[1 * para->number_of_nodes + index] +
               u[2] * node_f.force_density[2 * para->number_of_nodes + index]);

  C[1] += 1.0f / 2.0f * (1.0f + para->gamma_shear) *
          (u[0] * node_f.force_density[1 * para->number_of_nodes + index] +
           u[1] * node_f.force_density[0 * para->number_of_nodes + index]);

  C[3] += 1.0f / 2.0f * (1.0f + para->gamma_shear) *
          (u[0] * node_f.force_density[2 * para->number_of_nodes + index] +
           u[2] * node_f.force_density[0 * para->number_of_nodes + index]);

  C[4] += 1.0f / 2.0f * (1.0f + para->gamma_shear) *
          (u[1] * node_f.force_density[2 * para->number_of_nodes + index] +
           u[2] * node_f.force_density[1 * para->number_of_nodes + index]);

  /* update momentum modes */
  mode[1] += node_f.force_density[0 * para->number_of_nodes + index];
  mode[2] += node_f.force_density[1 * para->number_of_nodes + index];
  mode[3] += node_f.force_density[2 * para->number_of_nodes + index];

  /* update stress modes */
  mode[4] += C[0] + C[2] + C[5];
  mode[5] += C[0] - C[2];
  mode[6] += C[0] + C[2] - 2.0f * C[5];
  mode[7] += C[1];
  mode[8] += C[3];
  mode[9] += C[4];

  reset_LB_force_densities(index, node_f);
}

/** Calculate hydrodynamic fields in LB units
 *  @param[in]  n_a     Local node residing in array a for boundary flag
 *  @param[out] mode    Local register values mode
 *  @param[out] d_p_v   Local print values
 *  @param[out] d_v     Local device values
 *  @param[in]  node_f  Local node force
 *  @param[in]  index   Node index / thread index
 *  @param[in]  print_index  Node index / thread index
 *  TODO: code duplication with \ref calc_values_from_m_in_LB_units
 */
__device__ void
calc_values_in_LB_units(LB_nodes_gpu n_a, Utils::Array<float, 19> &mode,
                        LB_rho_v_pi_gpu *d_p_v, LB_rho_v_gpu *d_v,
                        LB_node_force_density_gpu node_f, unsigned int index,
                        unsigned int print_index) {
  Utils::Array<float, 3> j{};
  Utils::Array<float, 6> modes_from_pi_eq{};
  Utils::Array<float, 6> pi{};

  if (n_a.boundary[index] == 0) {
    /* Ensure we are working with the current values of d_v */

    update_rho_v(mode, index, node_f, d_v);

    d_p_v[print_index].rho = d_v[index].rho;

    d_p_v[print_index].v[0] = d_v[index].v[0];
    d_p_v[print_index].v[1] = d_v[index].v[1];
    d_p_v[print_index].v[2] = d_v[index].v[2];
    /* stress calculation */
    float Rho = d_v[index].rho;

    /* note that d_v[index].v[] already includes the 1/2 f term, accounting
     * for the pre- and post-collisional average
     */

    j[0] = Rho * d_v[index].v[0];
    j[1] = Rho * d_v[index].v[1];
    j[2] = Rho * d_v[index].v[2];

    // equilibrium part of the stress modes, which comes from
    // the equality between modes and stress tensor components

    /* m4 = trace(pi) - rho
       m5 = pi_xx - pi_yy
       m6 = trace(pi) - 3 pi_zz
       m7 = pi_xy
       m8 = pi_xz
       m9 = pi_yz */

    // and plugging in the Euler stress for the equilibrium:
    // pi_eq = rho_0*c_s^2*I3 + (j \otimes j)/rho
    // with I3 the 3D identity matrix and
    // rho = \trace(rho_0*c_s^2*I3), which yields

    /* m4_from_pi_eq = j.j
       m5_from_pi_eq = j_x*j_x - j_y*j_y
       m6_from_pi_eq = j.j - 3*j_z*j_z
       m7_from_pi_eq = j_x*j_y
       m8_from_pi_eq = j_x*j_z
       m9_from_pi_eq = j_y*j_z */

    // where the / Rho term has been dropped. We thus obtain:

    modes_from_pi_eq[0] = (j[0] * j[0] + j[1] * j[1] + j[2] * j[2]) / Rho;
    modes_from_pi_eq[1] = (j[0] * j[0] - j[1] * j[1]) / Rho;
    modes_from_pi_eq[2] =
        (j[0] * j[0] + j[1] * j[1] + j[2] * j[2] - 3.0f * j[2] * j[2]) / Rho;
    modes_from_pi_eq[3] = j[0] * j[1] / Rho;
    modes_from_pi_eq[4] = j[0] * j[2] / Rho;
    modes_from_pi_eq[5] = j[1] * j[2] / Rho;

    /* Now we must predict the outcome of the next collision */
    /* We immediately average pre- and post-collision.  */
    /* TODO: need a reference for this.   */

    mode[4] = modes_from_pi_eq[0] + (0.5f + 0.5f * para->gamma_bulk) *
                                        (mode[4] - modes_from_pi_eq[0]);
    mode[5] = modes_from_pi_eq[1] + (0.5f + 0.5f * para->gamma_shear) *
                                        (mode[5] - modes_from_pi_eq[1]);
    mode[6] = modes_from_pi_eq[2] + (0.5f + 0.5f * para->gamma_shear) *
                                        (mode[6] - modes_from_pi_eq[2]);
    mode[7] = modes_from_pi_eq[3] + (0.5f + 0.5f * para->gamma_shear) *
                                        (mode[7] - modes_from_pi_eq[3]);
    mode[8] = modes_from_pi_eq[4] + (0.5f + 0.5f * para->gamma_shear) *
                                        (mode[8] - modes_from_pi_eq[4]);
    mode[9] = modes_from_pi_eq[5] + (0.5f + 0.5f * para->gamma_shear) *
                                        (mode[9] - modes_from_pi_eq[5]);

    // Transform the stress tensor components according to the modes that
    // correspond to those used by U. Schiller. In terms of populations this
    // expression then corresponds exactly to those in Eqs. 116 - 121 in the
    // Duenweg and Ladd paper, when these are written out in populations.
    // But to ensure this, the expression in Schiller's modes has to be
    // different!

    pi[0] +=
        (2.0f * (mode[0] + mode[4]) + mode[6] + 3.0f * mode[5]) / 6.0f; // xx
    pi[1] += mode[7];                                                   // xy
    pi[2] +=
        (2.0f * (mode[0] + mode[4]) + mode[6] - 3.0f * mode[5]) / 6.0f; // yy
    pi[3] += mode[8];                                                   // xz
    pi[4] += mode[9];                                                   // yz
    pi[5] += (mode[0] + mode[4] - mode[6]) / 3.0f;                      // zz

    for (int i = 0; i < 6; i++) {
      d_p_v[print_index].pi[i] = pi[i];
    }
  } else {
    d_p_v[print_index].rho = 0.0f;

    for (int i = 0; i < 3; i++)
      d_p_v[print_index].v[i] = 0.0f;

    for (int i = 0; i < 6; i++)
      d_p_v[print_index].pi[i] = 0.0f;
  }
}

/** Calculate hydrodynamic fields in MD units
 *  @param[out] mode_single   Local register values mode
 *  @param[in]  d_v_single    Local device values
 *  @param[out] rho_out       Density
 *  @param[out] j_out         Momentum
 *  @param[out] pi_out        Pressure tensor
 */
__device__ void
calc_values_from_m_in_LB_units(Utils::Array<float, 19> &mode_single,
                               LB_rho_v_gpu *d_v_single, float *rho_out,
                               float *j_out, float *pi_out) {
  Utils::Array<float, 6> modes_from_pi_eq{};
  Utils::Array<float, 6> j{};
  float Rho;

  // stress calculation

  // Set the rho output value

  Rho = d_v_single->rho;
  *rho_out = d_v_single->rho;

  // note that d_v_single->v[] already includes the 1/2 f term,
  // accounting for the pre- and post-collisional average

  j[0] = Rho * d_v_single->v[0];
  j[1] = Rho * d_v_single->v[1];
  j[2] = Rho * d_v_single->v[2];

  j_out[3] = j[0];
  j_out[3] = j[1];
  j_out[3] = j[2];

  // equilibrium part of the stress modes, which comes from
  // the equality between modes and stress tensor components

  modes_from_pi_eq[0] = (j[0] * j[0] + j[1] * j[1] + j[2] * j[2]) / Rho;
  modes_from_pi_eq[1] = (j[0] * j[0] - j[1] * j[1]) / Rho;
  modes_from_pi_eq[2] =
      (j[0] * j[0] + j[1] * j[1] + j[2] * j[2] - 3.0f * j[2] * j[2]) / Rho;
  modes_from_pi_eq[3] = j[0] * j[1] / Rho;
  modes_from_pi_eq[4] = j[0] * j[2] / Rho;
  modes_from_pi_eq[5] = j[1] * j[2] / Rho;

  // Now we must predict the outcome of the next collision
  // We immediately average pre- and post-collision.

  mode_single[4] =
      modes_from_pi_eq[0] +
      (0.5f + 0.5f * para->gamma_bulk) * (mode_single[4] - modes_from_pi_eq[0]);
  mode_single[5] =
      modes_from_pi_eq[1] + (0.5f + 0.5f * para->gamma_shear) *
                                (mode_single[5] - modes_from_pi_eq[1]);
  mode_single[6] =
      modes_from_pi_eq[2] + (0.5f + 0.5f * para->gamma_shear) *
                                (mode_single[6] - modes_from_pi_eq[2]);
  mode_single[7] =
      modes_from_pi_eq[3] + (0.5f + 0.5f * para->gamma_shear) *
                                (mode_single[7] - modes_from_pi_eq[3]);
  mode_single[8] =
      modes_from_pi_eq[4] + (0.5f + 0.5f * para->gamma_shear) *
                                (mode_single[8] - modes_from_pi_eq[4]);
  mode_single[9] =
      modes_from_pi_eq[5] + (0.5f + 0.5f * para->gamma_shear) *
                                (mode_single[9] - modes_from_pi_eq[5]);

  // Transform the stress tensor components according to the mode_singles.

  pi_out[0] = (2.0f * (mode_single[0] + mode_single[4]) + mode_single[6] +
               3.0f * mode_single[5]) /
              6.0f;           // xx
  pi_out[1] = mode_single[7]; // xy
  pi_out[2] = (2.0f * (mode_single[0] + mode_single[4]) + mode_single[6] -
               3.0f * mode_single[5]) /
              6.0f;                                                      // yy
  pi_out[3] = mode_single[8];                                            // xz
  pi_out[4] = mode_single[9];                                            // yz
  pi_out[5] = (mode_single[0] + mode_single[4] - mode_single[6]) / 3.0f; // zz
}

/**
 *  @param[in]  node_index        Node index around (8) particle
 *  @param[out] mode              Local register values mode
 *  @param[in]  n_a               Local node residing in array a
 */
__device__ void calc_mode(Utils::Array<float, 4> &mode, LB_nodes_gpu n_a,
                          unsigned int node_index) {
  /* mass mode */
  mode[0] = n_a.vd[0 * para->number_of_nodes + node_index] +
            n_a.vd[1 * para->number_of_nodes + node_index] +
            n_a.vd[2 * para->number_of_nodes + node_index] +
            n_a.vd[3 * para->number_of_nodes + node_index] +
            n_a.vd[4 * para->number_of_nodes + node_index] +
            n_a.vd[5 * para->number_of_nodes + node_index] +
            n_a.vd[6 * para->number_of_nodes + node_index] +
            n_a.vd[7 * para->number_of_nodes + node_index] +
            n_a.vd[8 * para->number_of_nodes + node_index] +
            n_a.vd[9 * para->number_of_nodes + node_index] +
            n_a.vd[10 * para->number_of_nodes + node_index] +
            n_a.vd[11 * para->number_of_nodes + node_index] +
            n_a.vd[12 * para->number_of_nodes + node_index] +
            n_a.vd[13 * para->number_of_nodes + node_index] +
            n_a.vd[14 * para->number_of_nodes + node_index] +
            n_a.vd[15 * para->number_of_nodes + node_index] +
            n_a.vd[16 * para->number_of_nodes + node_index] +
            n_a.vd[17 * para->number_of_nodes + node_index] +
            n_a.vd[18 * para->number_of_nodes + node_index];

  /* momentum modes */
  mode[1] = (n_a.vd[1 * para->number_of_nodes + node_index] -
             n_a.vd[2 * para->number_of_nodes + node_index]) +
            (n_a.vd[7 * para->number_of_nodes + node_index] -
             n_a.vd[8 * para->number_of_nodes + node_index]) +
            (n_a.vd[9 * para->number_of_nodes + node_index] -
             n_a.vd[10 * para->number_of_nodes + node_index]) +
            (n_a.vd[11 * para->number_of_nodes + node_index] -
             n_a.vd[12 * para->number_of_nodes + node_index]) +
            (n_a.vd[13 * para->number_of_nodes + node_index] -
             n_a.vd[14 * para->number_of_nodes + node_index]);

  mode[2] = (n_a.vd[3 * para->number_of_nodes + node_index] -
             n_a.vd[4 * para->number_of_nodes + node_index]) +
            (n_a.vd[7 * para->number_of_nodes + node_index] -
             n_a.vd[8 * para->number_of_nodes + node_index]) -
            (n_a.vd[9 * para->number_of_nodes + node_index] -
             n_a.vd[10 * para->number_of_nodes + node_index]) +
            (n_a.vd[15 * para->number_of_nodes + node_index] -
             n_a.vd[16 * para->number_of_nodes + node_index]) +
            (n_a.vd[17 * para->number_of_nodes + node_index] -
             n_a.vd[18 * para->number_of_nodes + node_index]);

  mode[3] = (n_a.vd[5 * para->number_of_nodes + node_index] -
             n_a.vd[6 * para->number_of_nodes + node_index]) +
            (n_a.vd[11 * para->number_of_nodes + node_index] -
             n_a.vd[12 * para->number_of_nodes + node_index]) -
            (n_a.vd[13 * para->number_of_nodes + node_index] -
             n_a.vd[14 * para->number_of_nodes + node_index]) +
            (n_a.vd[15 * para->number_of_nodes + node_index] -
             n_a.vd[16 * para->number_of_nodes + node_index]) -
            (n_a.vd[17 * para->number_of_nodes + node_index] -
             n_a.vd[18 * para->number_of_nodes + node_index]);
}

/** Calculate temperature of the fluid kernel
 *  @param[out] cpu_jsquared  Result
 *  @param[in]  n_a           Local node residing in array a
 *  @param[out] number_of_non_boundary_nodes  Local node residing in array a
 */
__global__ void temperature(LB_nodes_gpu n_a, float *cpu_jsquared,
                            int *number_of_non_boundary_nodes) {
  Utils::Array<float, 4> mode;
  float jsquared = 0.0f;
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x +
                       blockDim.x * blockIdx.x + threadIdx.x;

  if (index < para->number_of_nodes) {
    if (!n_a.boundary[index]) {
      calc_mode(mode, n_a, index);
      jsquared = mode[1] * mode[1] + mode[2] * mode[2] + mode[3] * mode[3];
      atomicAdd(cpu_jsquared, jsquared);
      atomicAdd(number_of_non_boundary_nodes, 1);
    }
  }
}

/**
 * @param u Distance to grid point in units of agrid
 * @retval Value for the interpolation function.
 * see Duenweg and Ladd http://arxiv.org/abs/0803.2826
 */
__device__ __inline__ float
three_point_polynomial_smallerequal_than_half(float u) {
  return 1.f / 3.f * (1.f + sqrtf(1.f - 3.f * u * u));
}

__device__ __inline__ float three_point_polynomial_larger_than_half(float u) {
  return 1.f / 6.f *
         (5.f + -3 * fabsf(u) - sqrtf(-2.f + 6.f * fabsf(u) - 3.f * u * u));
}

__device__ __inline__ float3
velocity_interpolation(LB_nodes_gpu n_a, float *particle_position,
                       float *lb_boundary_velocity,
                       Utils::Array<unsigned int, 27> &node_indices,
                       Utils::Array<float, 27> &delta) {
  Utils::Array<int, 3> center_node_index{};
  Utils::Array<float3, 3> temp_delta{};

#pragma unroll
  for (int i = 0; i < 3; ++i) {
    // position of particle in units of agrid.
    auto const scaled_pos = particle_position[i] / para->agrid - 0.5f;
    center_node_index[i] = static_cast<int>(rint(scaled_pos));
    // distance to center node in agrid
    auto const dist = scaled_pos - static_cast<float>(center_node_index[i]);
    // distance to left node in agrid
    auto const dist_m1 =
        scaled_pos - static_cast<float>(center_node_index[i] - 1.f);
    // distance to right node in agrid
    auto const dist_p1 =
        scaled_pos - static_cast<float>(center_node_index[i] + 1.f);
    if (i == 0) {
      temp_delta[0].x = three_point_polynomial_larger_than_half(dist_m1);
      temp_delta[1].x = three_point_polynomial_smallerequal_than_half(dist);
      temp_delta[2].x = three_point_polynomial_larger_than_half(dist_p1);
    } else if (i == 1) {
      temp_delta[0].y = three_point_polynomial_larger_than_half(dist_m1);
      temp_delta[1].y = three_point_polynomial_smallerequal_than_half(dist);
      temp_delta[2].y = three_point_polynomial_larger_than_half(dist_p1);
    } else if (i == 2) {
      temp_delta[0].z = three_point_polynomial_larger_than_half(dist_m1);
      temp_delta[1].z = three_point_polynomial_smallerequal_than_half(dist);
      temp_delta[2].z = three_point_polynomial_larger_than_half(dist_p1);
    }
  }

  auto fold_if_necessary = [](int ind, int dim) {
    if (ind >= dim) {
      return ind - dim;
    } else if (ind < 0) {
      return ind + dim;
    }
    return ind;
  };

  int cnt = 0;
  float3 interpolated_u{0.0f, 0.0f, 0.0f};
#pragma unroll
  for (int i = 0; i < 3; ++i) {
    for (int j = 0; j < 3; ++j) {
      for (int k = 0; k < 3; ++k) {
        auto const x =
            fold_if_necessary(center_node_index[0] - 1 + i, para->dim_x);
        auto const y =
            fold_if_necessary(center_node_index[1] - 1 + j, para->dim_y);
        auto const z =
            fold_if_necessary(center_node_index[2] - 1 + k, para->dim_z);
        delta[cnt] = temp_delta[i].x * temp_delta[j].y * temp_delta[k].z;
        node_indices[cnt] = xyz_to_index(x, y, z);
        auto const boundary_index = n_a.boundary[node_indices[cnt]];
        if (not boundary_index) {
          float totmass = 0.0f;
          auto const mass_mode = calc_mode_x_from_n(n_a, node_indices[cnt], 0);

          totmass += mass_mode + para->rho;

          auto const j_x = calc_mode_x_from_n(n_a, node_indices[cnt], 1);
          auto const j_y = calc_mode_x_from_n(n_a, node_indices[cnt], 2);
          auto const j_z = calc_mode_x_from_n(n_a, node_indices[cnt], 3);
          interpolated_u.x += (j_x / totmass) * delta[cnt];
          interpolated_u.y += (j_y / totmass) * delta[cnt];
          interpolated_u.z += (j_z / totmass) * delta[cnt];
        } else {
          interpolated_u.x +=
              lb_boundary_velocity[3 * (boundary_index - 1) + 0] * para->tau /
              para->agrid * delta[cnt];
          interpolated_u.y +=
              lb_boundary_velocity[3 * (boundary_index - 1) + 1] * para->tau /
              para->agrid * delta[cnt];
          interpolated_u.z +=
              lb_boundary_velocity[3 * (boundary_index - 1) + 2] * para->tau /
              para->agrid * delta[cnt];
        }
        ++cnt;
      }
    }
  }
  return interpolated_u;
}

/**
 *  (Eq. (12) Ahlrichs and Duenweg, JCP 111(17):8225 (1999))
 *  @param[in]  n_a                Local node residing in array a
 *  @param[in]  particle_position  Particle position
 *  @param[out] node_index         Node index around (8) particle
 *  @param[out] delta              Weighting of particle position
 *  @param[in]  lb_boundary_velocity Velocity at the boundary
 *  @retval Interpolated velocity
 */
__device__ __inline__ float3 velocity_interpolation(
    LB_nodes_gpu n_a, float *particle_position, float *lb_boundary_velocity,
    Utils::Array<unsigned int, 8> &node_index, Utils::Array<float, 8> &delta) {
  Utils::Array<int, 3> left_node_index;
  Utils::Array<float, 6> temp_delta;
  // see ahlrichs + duenweg page 8227 equ (10) and (11)
#pragma unroll
  for (int i = 0; i < 3; ++i) {
    auto const scaledpos = particle_position[i] / para->agrid - 0.5f;
    left_node_index[i] = static_cast<int>(floorf(scaledpos));
    temp_delta[3 + i] = scaledpos - left_node_index[i];
    temp_delta[i] = 1.0f - temp_delta[3 + i];
  }

  delta[0] = temp_delta[0] * temp_delta[1] * temp_delta[2];
  delta[1] = temp_delta[3] * temp_delta[1] * temp_delta[2];
  delta[2] = temp_delta[0] * temp_delta[4] * temp_delta[2];
  delta[3] = temp_delta[3] * temp_delta[4] * temp_delta[2];
  delta[4] = temp_delta[0] * temp_delta[1] * temp_delta[5];
  delta[5] = temp_delta[3] * temp_delta[1] * temp_delta[5];
  delta[6] = temp_delta[0] * temp_delta[4] * temp_delta[5];
  delta[7] = temp_delta[3] * temp_delta[4] * temp_delta[5];

  // modulo for negative numbers is strange at best, shift to make sure we are
  // positive
  int x = (left_node_index[0] + para->dim_x) % para->dim_x;
  int y = (left_node_index[1] + para->dim_y) % para->dim_y;
  int z = (left_node_index[2] + para->dim_z) % para->dim_z;
  auto xp1 = x + 1;
  auto yp1 = y + 1;
  auto zp1 = z + 1;
  auto fold_if_necessary = [](int ind, int dim) {
    return ind >= dim ? ind % dim : ind;
  };
  xp1 = fold_if_necessary(xp1, para->dim_x);
  yp1 = fold_if_necessary(yp1, para->dim_y);
  zp1 = fold_if_necessary(zp1, para->dim_z);
  node_index[0] = xyz_to_index(x, y, z);
  node_index[1] = xyz_to_index(xp1, y, z);
  node_index[2] = xyz_to_index(x, yp1, z);
  node_index[3] = xyz_to_index(xp1, yp1, z);
  node_index[4] = xyz_to_index(x, y, zp1);
  node_index[5] = xyz_to_index(xp1, y, zp1);
  node_index[6] = xyz_to_index(x, yp1, zp1);
  node_index[7] = xyz_to_index(xp1, yp1, zp1);

  float3 interpolated_u{0.0f, 0.0f, 0.0f};
#pragma unroll
  for (int i = 0; i < 8; ++i) {
    float totmass = 0.0f;
    Utils::Array<float, 19> mode;

    calc_m_from_n(n_a, node_index[i], mode);
    auto const mass_mode = calc_mode_x_from_n(n_a, node_index[i], 0);

    totmass += mass_mode + para->rho;

    /* The boolean expression (n_a.boundary[node_index[i]] == 0) causes boundary
       nodes to couple with velocity 0 to particles. This is necessary, since
       boundary nodes undergo the same LB dynamics as fluid nodes do. The flow
       within the boundaries does not interact with the physical fluid, since
       these populations are overwritten by the bounce back kernel. Particles
       close to walls can couple to this unphysical flow, though.
    */
    auto const j_x = calc_mode_x_from_n(n_a, node_index[i], 1);
    auto const j_y = calc_mode_x_from_n(n_a, node_index[i], 2);
    auto const j_z = calc_mode_x_from_n(n_a, node_index[i], 3);
    interpolated_u.x +=
        (j_x / totmass) * delta[i] * (n_a.boundary[node_index[i]] == 0);
    interpolated_u.y +=
        (j_y / totmass) * delta[i] * (n_a.boundary[node_index[i]] == 0);
    interpolated_u.z +=
        (j_z / totmass) * delta[i] * (n_a.boundary[node_index[i]] == 0);
  }
  return interpolated_u;
}

/**
 *  (Eq. (12) Ahlrichs and Duenweg, JCP 111(17):8225 (1999))
 *  @param[in]  n_a                Local node residing in array a
 *  @param[out] delta              Weighting of particle position
 *  @param[out] delta_j            Weighting of particle momentum
 *  @param[in,out] particle_data   Particle position and velocity
 *  @param[in,out] particle_force  Particle force
 *  @param[in]  part_index         Particle id / thread id
 *  @param[out] node_index         Node index around (8) particle
 *  @param[in]  d_v                Local device values
 *  @param[in]  flag_cs            Determine if we are at the centre (0,
 *                                 typical) or at the source (1, swimmer only)
 *  @param[in]  philox_counter
 *  @param[in]  friction           Friction constant for the particle coupling
 *  @param[in]  lb_boundary_velocity Velocity at the boundary
 *  @tparam no_of_neighbours       The number of neighbours to consider for
 * interpolation
 */
template <std::size_t no_of_neighbours>
__device__ void
calc_viscous_force(LB_nodes_gpu n_a,
                   Utils::Array<float, no_of_neighbours> &delta,
                   CUDA_particle_data *particle_data, float *particle_force,
                   unsigned int part_index, float *delta_j,
                   Utils::Array<unsigned int, no_of_neighbours> &node_index,
                   LB_rho_v_gpu *d_v, int flag_cs, uint64_t philox_counter,
                   float friction, float *lb_boundary_velocity) {
// Zero out workspace
#pragma unroll
  for (int jj = 0; jj < 3; ++jj) {
    delta_j[jj] = 0.0f;
  }

  // Zero out only if we are at the centre of the particle <=> flag_cs = 0
  particle_force[3 * part_index + 0] =
      flag_cs * particle_force[3 * part_index + 0];
  particle_force[3 * part_index + 1] =
      flag_cs * particle_force[3 * part_index + 1];
  particle_force[3 * part_index + 2] =
      flag_cs * particle_force[3 * part_index + 2];

  float position[3];
  position[0] = particle_data[part_index].p[0];
  position[1] = particle_data[part_index].p[1];
  position[2] = particle_data[part_index].p[2];

  float velocity[3];
  velocity[0] = particle_data[part_index].v[0];
  velocity[1] = particle_data[part_index].v[1];
  velocity[2] = particle_data[part_index].v[2];

#ifdef ENGINE
  // First calculate interpolated velocity for dipole source,
  // such that we don't overwrite mode, d_v, etc. for the rest of the function
  float direction = float(particle_data[part_index].swim.push_pull) *
                    particle_data[part_index].swim.dipole_length;
  // Extrapolate position by dipole length if we are at the centre of the
  // particle
  position[0] +=
      flag_cs * direction * particle_data[part_index].swim.director[0];
  position[1] +=
      flag_cs * direction * particle_data[part_index].swim.director[1];
  position[2] +=
      flag_cs * direction * particle_data[part_index].swim.director[2];
#endif

  float3 const interpolated_u = velocity_interpolation(
      n_a, position, lb_boundary_velocity, node_index, delta);

#ifdef ENGINE
  velocity[0] -= particle_data[part_index].swim.v_swim *
                 particle_data[part_index].swim.director[0];
  velocity[1] -= particle_data[part_index].swim.v_swim *
                 particle_data[part_index].swim.director[1];
  velocity[2] -= particle_data[part_index].swim.v_swim *
                 particle_data[part_index].swim.director[2];

  // The first three components are v_center, the last three v_source
  // Do not use within LB, because these have already been converted back to MD
  // units
  particle_data[part_index].swim.v_cs[0 + 3 * flag_cs] =
      interpolated_u.x * para->agrid / para->tau;
  particle_data[part_index].swim.v_cs[1 + 3 * flag_cs] =
      interpolated_u.y * para->agrid / para->tau;
  particle_data[part_index].swim.v_cs[2 + 3 * flag_cs] =
      interpolated_u.z * para->agrid / para->tau;
#endif

  /** calculate viscous force
   * take care to rescale velocities with time_step and transform to MD units
   * (Eq. (9) Ahlrichs and Duenweg, JCP 111(17):8225 (1999)) */

  /* Viscous force */
  float3 viscforce_density{0.0f, 0.0f, 0.0f};
  viscforce_density.x -=
      friction * (velocity[0] - interpolated_u.x * para->agrid / para->tau);
  viscforce_density.y -=
      friction * (velocity[1] - interpolated_u.y * para->agrid / para->tau);
  viscforce_density.z -=
      friction * (velocity[2] - interpolated_u.z * para->agrid / para->tau);

#ifdef LB_ELECTROHYDRODYNAMICS
  viscforce_density.x += friction * particle_data[part_index].mu_E[0];
  viscforce_density.y += friction * particle_data[part_index].mu_E[1];
  viscforce_density.z += friction * particle_data[part_index].mu_E[2];
#endif

  /** add stochastic force of zero mean (Ahlrichs, Duenweg equ. 15)*/
  float4 random_floats = random_wrapper_philox(
      particle_data[part_index].identity, LBQ * 32, philox_counter);
  /* lb_coupl_pref is stored in MD units (force)
   * Eq. (16) Ahlrichs and Duenweg, JCP 111(17):8225 (1999).
   * The factor 12 comes from the fact that we use random numbers
   * from -0.5 to 0.5 (equally distributed) which have variance 1/12.
   * time_step comes from the discretization.
   */
  float lb_coupl_pref =
      sqrtf(12.f * 2.f * friction * para->kT / para->time_step);
  viscforce_density.x += lb_coupl_pref * (random_floats.w - 0.5f);
  viscforce_density.y += lb_coupl_pref * (random_floats.x - 0.5f);
  viscforce_density.z += lb_coupl_pref * (random_floats.y - 0.5f);

  /** delta_j for transform momentum transfer to lattice units which is done
    in calc_node_force (Eq. (12) Ahlrichs and Duenweg, JCP 111(17):8225
    (1999)) */

  // only add to particle_force for particle centre <=> (1-flag_cs) = 1
  particle_force[3 * part_index + 0] += (1 - flag_cs) * viscforce_density.x;
  particle_force[3 * part_index + 1] += (1 - flag_cs) * viscforce_density.y;
  particle_force[3 * part_index + 2] += (1 - flag_cs) * viscforce_density.z;

  // only add to particle_force for particle centre <=> (1-flag_cs) = 1
  delta_j[0] -= ((1 - flag_cs) * viscforce_density.x) * para->time_step *
                para->tau / para->agrid;
  delta_j[1] -= ((1 - flag_cs) * viscforce_density.y) * para->time_step *
                para->tau / para->agrid;
  delta_j[2] -= ((1 - flag_cs) * viscforce_density.z) * para->time_step *
                para->tau / para->agrid;

#ifdef ENGINE
  // add swimming force to source position
  delta_j[0] -= flag_cs * particle_data[part_index].swim.f_swim *
                particle_data[part_index].swim.director[0] * para->time_step *
                para->tau / para->agrid;
  delta_j[1] -= flag_cs * particle_data[part_index].swim.f_swim *
                particle_data[part_index].swim.director[1] * para->time_step *
                para->tau / para->agrid;
  delta_j[2] -= flag_cs * particle_data[part_index].swim.f_swim *
                particle_data[part_index].swim.director[2] * para->time_step *
                para->tau / para->agrid;
#endif
}

/** Calculate the node force caused by the particles, with atomicAdd due to
 *  avoiding race conditions
 *  (Eq. (14) Ahlrichs and Duenweg, JCP 111(17):8225 (1999))
 *  @param[in]  delta              Weighting of particle position
 *  @param[in]  delta_j            Weighting of particle momentum
 *  @param[in]  node_index         Node index around (8) particle
 *  @param[out] node_f             Node force
 *  @tparam no_of_neighbours       The number of neighbours to consider for
 * interpolation
 */
template <std::size_t no_of_neighbours>
__device__ void
calc_node_force(Utils::Array<float, no_of_neighbours> const &delta,
                float *delta_j,
                Utils::Array<unsigned int, no_of_neighbours> const &node_index,
                LB_node_force_density_gpu node_f) {
  for (int node = 0; node < no_of_neighbours; ++node) {
    for (int i = 0; i < 3; ++i) {
      atomicAdd(
          &(node_f.force_density[i * para->number_of_nodes + node_index[node]]),
          delta[node] * delta_j[i]);
    }
  }
}

/*********************************************************/
/** \name System setup and Kernel functions */
/*********************************************************/

/** Kernel to calculate local populations from hydrodynamic fields.
 *  The mapping is given in terms of the equilibrium distribution.
 *
 *  Eq. (2.15) Ladd, J. Fluid Mech. 271, 295-309 (1994)
 *  Eq. (4) in Berk Usta, Ladd and Butler, JCP 122, 094902 (2005)
 *
 *  @param[out] n_a        %Lattice site
 *  @param[out] gpu_check  Additional check if GPU kernel are executed
 *  @param[out] d_v        Local device values
 *  @param[in]  node_f     Node forces
 */
__global__ void calc_n_from_rho_j_pi(LB_nodes_gpu n_a, LB_rho_v_gpu *d_v,
                                     LB_node_force_density_gpu node_f,
                                     int *gpu_check) {
  /* TODO: this can handle only a uniform density, something similar, but local,
           has to be called every time the fields are set by the user ! */
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x +
                       blockDim.x * blockIdx.x + threadIdx.x;
  if (index < para->number_of_nodes) {
    Utils::Array<float, 19> mode;

    /* default values for fields in lattice units */
    gpu_check[0] = 1;

    float Rho = para->rho;
    Utils::Array<float, 3> v{};
    Utils::Array<float, 6> pi = {
        Rho * c_sound_sq, 0.0f, Rho * c_sound_sq, 0.0f, 0.0f, Rho * c_sound_sq};
    Utils::Array<float, 6> local_pi{};
    float rhoc_sq = Rho * c_sound_sq;
    float avg_rho = para->rho;
    float local_rho, trace;
    Utils::Array<float, 3> local_j{};

    local_rho = Rho;

    local_j[0] = Rho * v[0];
    local_j[1] = Rho * v[1];
    local_j[2] = Rho * v[2];

    local_pi = pi;

    // reduce the pressure tensor to the part needed here.

    local_pi[0] -= rhoc_sq;
    local_pi[2] -= rhoc_sq;
    local_pi[5] -= rhoc_sq;

    trace = local_pi[0] + local_pi[2] + local_pi[5];

    float rho_times_coeff;
    float tmp1, tmp2;

    /* update the q=0 sublattice */
    n_a.vd[(0) * para->number_of_nodes + index] =
        1.0f / 3.0f * (local_rho - avg_rho) - 1.0f / 2.0f * trace;

    /* update the q=1 sublattice */
    rho_times_coeff = 1.0f / 18.0f * (local_rho - avg_rho);

    n_a.vd[(1) * para->number_of_nodes + index] =
        rho_times_coeff + 1.0f / 6.0f * local_j[0] + 1.0f / 4.0f * local_pi[0] -
        1.0f / 12.0f * trace;
    n_a.vd[(2) * para->number_of_nodes + index] =
        rho_times_coeff - 1.0f / 6.0f * local_j[0] + 1.0f / 4.0f * local_pi[0] -
        1.0f / 12.0f * trace;
    n_a.vd[(3) * para->number_of_nodes + index] =
        rho_times_coeff + 1.0f / 6.0f * local_j[1] + 1.0f / 4.0f * local_pi[2] -
        1.0f / 12.0f * trace;
    n_a.vd[(4) * para->number_of_nodes + index] =
        rho_times_coeff - 1.0f / 6.0f * local_j[1] + 1.0f / 4.0f * local_pi[2] -
        1.0f / 12.0f * trace;
    n_a.vd[(5) * para->number_of_nodes + index] =
        rho_times_coeff + 1.0f / 6.0f * local_j[2] + 1.0f / 4.0f * local_pi[5] -
        1.0f / 12.0f * trace;
    n_a.vd[(6) * para->number_of_nodes + index] =
        rho_times_coeff - 1.0f / 6.0f * local_j[2] + 1.0f / 4.0f * local_pi[5] -
        1.0f / 12.0f * trace;

    /* update the q=2 sublattice */
    rho_times_coeff = 1.0f / 36.0f * (local_rho - avg_rho);

    tmp1 = local_pi[0] + local_pi[2];
    tmp2 = 2.0f * local_pi[1];
    n_a.vd[(7) * para->number_of_nodes + index] =
        rho_times_coeff + 1.0f / 12.0f * (local_j[0] + local_j[1]) +
        1.0f / 8.0f * (tmp1 + tmp2) - 1.0f / 24.0f * trace;
    n_a.vd[(8) * para->number_of_nodes + index] =
        rho_times_coeff - 1.0f / 12.0f * (local_j[0] + local_j[1]) +
        1.0f / 8.0f * (tmp1 + tmp2) - 1.0f / 24.0f * trace;
    n_a.vd[(9) * para->number_of_nodes + index] =
        rho_times_coeff + 1.0f / 12.0f * (local_j[0] - local_j[1]) +
        1.0f / 8.0f * (tmp1 - tmp2) - 1.0f / 24.0f * trace;
    n_a.vd[(10) * para->number_of_nodes + index] =
        rho_times_coeff - 1.0f / 12.0f * (local_j[0] - local_j[1]) +
        1.0f / 8.0f * (tmp1 - tmp2) - 1.0f / 24.0f * trace;

    tmp1 = local_pi[0] + local_pi[5];
    tmp2 = 2.0f * local_pi[3];

    n_a.vd[(11) * para->number_of_nodes + index] =
        rho_times_coeff + 1.0f / 12.0f * (local_j[0] + local_j[2]) +
        1.0f / 8.0f * (tmp1 + tmp2) - 1.0f / 24.0f * trace;
    n_a.vd[(12) * para->number_of_nodes + index] =
        rho_times_coeff - 1.0f / 12.0f * (local_j[0] + local_j[2]) +
        1.0f / 8.0f * (tmp1 + tmp2) - 1.0f / 24.0f * trace;
    n_a.vd[(13) * para->number_of_nodes + index] =
        rho_times_coeff + 1.0f / 12.0f * (local_j[0] - local_j[2]) +
        1.0f / 8.0f * (tmp1 - tmp2) - 1.0f / 24.0f * trace;
    n_a.vd[(14) * para->number_of_nodes + index] =
        rho_times_coeff - 1.0f / 12.0f * (local_j[0] - local_j[2]) +
        1.0f / 8.0f * (tmp1 - tmp2) - 1.0f / 24.0f * trace;

    tmp1 = local_pi[2] + local_pi[5];
    tmp2 = 2.0f * local_pi[4];

    n_a.vd[(15) * para->number_of_nodes + index] =
        rho_times_coeff + 1.0f / 12.0f * (local_j[1] + local_j[2]) +
        1.0f / 8.0f * (tmp1 + tmp2) - 1.0f / 24.0f * trace;
    n_a.vd[(16) * para->number_of_nodes + index] =
        rho_times_coeff - 1.0f / 12.0f * (local_j[1] + local_j[2]) +
        1.0f / 8.0f * (tmp1 + tmp2) - 1.0f / 24.0f * trace;
    n_a.vd[(17) * para->number_of_nodes + index] =
        rho_times_coeff + 1.0f / 12.0f * (local_j[1] - local_j[2]) +
        1.0f / 8.0f * (tmp1 - tmp2) - 1.0f / 24.0f * trace;
    n_a.vd[(18) * para->number_of_nodes + index] =
        rho_times_coeff - 1.0f / 12.0f * (local_j[1] - local_j[2]) +
        1.0f / 8.0f * (tmp1 - tmp2) - 1.0f / 24.0f * trace;

    calc_m_from_n(n_a, index, mode);
    update_rho_v(mode, index, node_f, d_v);
  }
}

/** Kernel to calculate local populations from hydrodynamic fields
 *  from given flow field velocities. The mapping is given in terms of
 *  the equilibrium distribution.
 *
 *  Eq. (2.15) Ladd, J. Fluid Mech. 271, 295-309 (1994)
 *  Eq. (4) in Berk Usta, Ladd and Butler, JCP 122, 094902 (2005)
 *
 *  @param[out] n_a               Current nodes array (double buffering!)
 *  @param[in]  single_nodeindex  Single node index
 *  @param[in]  velocity          Velocity
 *  @param[out] d_v               Local device values
 *  @param[in]  node_f            Node forces
 */
__global__ void set_u_from_rho_v_pi(LB_nodes_gpu n_a, int single_nodeindex,
                                    float *velocity, LB_rho_v_gpu *d_v,
                                    LB_node_force_density_gpu node_f) {
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x +
                       blockDim.x * blockIdx.x + threadIdx.x;

  if (index == 0) {
    float local_rho;
    float local_j[3];
    float local_pi[6];
    float trace, avg_rho;
    float rho_times_coeff;
    float tmp1, tmp2;

    Utils::Array<float, 19> mode_for_pi;
    float rho_from_m;
    float j_from_m[3];
    float pi_from_m[6];

    // Calculate the modes for this node

    calc_m_from_n(n_a, single_nodeindex, mode_for_pi);

    // Reset the d_v

    update_rho_v(mode_for_pi, single_nodeindex, node_f, d_v);

    // Calculate the density, velocity, and pressure tensor
    // in LB unit for this node

    calc_values_from_m_in_LB_units(mode_for_pi, &d_v[single_nodeindex],
                                   &rho_from_m, j_from_m, pi_from_m);

    // Take LB component density and calculate the equilibrium part
    local_rho = rho_from_m;
    avg_rho = para->rho;

    // Take LB component velocity and make it a momentum

    local_j[0] = local_rho * velocity[0];
    local_j[1] = local_rho * velocity[1];
    local_j[2] = local_rho * velocity[2];
    // Take LB component pressure tensor and put in equilibrium

    local_pi[0] = pi_from_m[0];
    local_pi[1] = pi_from_m[1];
    local_pi[2] = pi_from_m[2];
    local_pi[3] = pi_from_m[3];
    local_pi[4] = pi_from_m[4];
    local_pi[5] = pi_from_m[5];

    trace = local_pi[0] + local_pi[2] + local_pi[5];

    // update the q=0 sublattice

    n_a.vd[(0) * para->number_of_nodes + single_nodeindex] =
        1.0f / 3.0f * (local_rho - avg_rho) - 1.0f / 2.0f * trace;

    // update the q=1 sublattice

    rho_times_coeff = 1.0f / 18.0f * (local_rho - avg_rho);

    n_a.vd[(1) * para->number_of_nodes + single_nodeindex] =
        rho_times_coeff + 1.0f / 6.0f * local_j[0] + 1.0f / 4.0f * local_pi[0] -
        1.0f / 12.0f * trace;
    n_a.vd[(2) * para->number_of_nodes + single_nodeindex] =
        rho_times_coeff - 1.0f / 6.0f * local_j[0] + 1.0f / 4.0f * local_pi[0] -
        1.0f / 12.0f * trace;
    n_a.vd[(3) * para->number_of_nodes + single_nodeindex] =
        rho_times_coeff + 1.0f / 6.0f * local_j[1] + 1.0f / 4.0f * local_pi[2] -
        1.0f / 12.0f * trace;
    n_a.vd[(4) * para->number_of_nodes + single_nodeindex] =
        rho_times_coeff - 1.0f / 6.0f * local_j[1] + 1.0f / 4.0f * local_pi[2] -
        1.0f / 12.0f * trace;
    n_a.vd[(5) * para->number_of_nodes + single_nodeindex] =
        rho_times_coeff + 1.0f / 6.0f * local_j[2] + 1.0f / 4.0f * local_pi[5] -
        1.0f / 12.0f * trace;
    n_a.vd[(6) * para->number_of_nodes + single_nodeindex] =
        rho_times_coeff - 1.0f / 6.0f * local_j[2] + 1.0f / 4.0f * local_pi[5] -
        1.0f / 12.0f * trace;

    // update the q=2 sublattice

    rho_times_coeff = 1.0f / 36.0f * (local_rho - avg_rho);

    tmp1 = local_pi[0] + local_pi[2];
    tmp2 = 2.0f * local_pi[1];

    n_a.vd[(7) * para->number_of_nodes + single_nodeindex] =
        rho_times_coeff + 1.0f / 12.0f * (local_j[0] + local_j[1]) +
        1.0f / 8.0f * (tmp1 + tmp2) - 1.0f / 24.0f * trace;
    n_a.vd[(8) * para->number_of_nodes + single_nodeindex] =
        rho_times_coeff - 1.0f / 12.0f * (local_j[0] + local_j[1]) +
        1.0f / 8.0f * (tmp1 + tmp2) - 1.0f / 24.0f * trace;
    n_a.vd[(9) * para->number_of_nodes + single_nodeindex] =
        rho_times_coeff + 1.0f / 12.0f * (local_j[0] - local_j[1]) +
        1.0f / 8.0f * (tmp1 - tmp2) - 1.0f / 24.0f * trace;
    n_a.vd[(10) * para->number_of_nodes + single_nodeindex] =
        rho_times_coeff - 1.0f / 12.0f * (local_j[0] - local_j[1]) +
        1.0f / 8.0f * (tmp1 - tmp2) - 1.0f / 24.0f * trace;

    tmp1 = local_pi[0] + local_pi[5];
    tmp2 = 2.0f * local_pi[3];

    n_a.vd[(11) * para->number_of_nodes + single_nodeindex] =
        rho_times_coeff + 1.0f / 12.0f * (local_j[0] + local_j[2]) +
        1.0f / 8.0f * (tmp1 + tmp2) - 1.0f / 24.0f * trace;
    n_a.vd[(12) * para->number_of_nodes + single_nodeindex] =
        rho_times_coeff - 1.0f / 12.0f * (local_j[0] + local_j[2]) +
        1.0f / 8.0f * (tmp1 + tmp2) - 1.0f / 24.0f * trace;
    n_a.vd[(13) * para->number_of_nodes + single_nodeindex] =
        rho_times_coeff + 1.0f / 12.0f * (local_j[0] - local_j[2]) +
        1.0f / 8.0f * (tmp1 - tmp2) - 1.0f / 24.0f * trace;
    n_a.vd[(14) * para->number_of_nodes + single_nodeindex] =
        rho_times_coeff - 1.0f / 12.0f * (local_j[0] - local_j[2]) +
        1.0f / 8.0f * (tmp1 - tmp2) - 1.0f / 24.0f * trace;

    tmp1 = local_pi[2] + local_pi[5];
    tmp2 = 2.0f * local_pi[4];

    n_a.vd[(15) * para->number_of_nodes + single_nodeindex] =
        rho_times_coeff + 1.0f / 12.0f * (local_j[1] + local_j[2]) +
        1.0f / 8.0f * (tmp1 + tmp2) - 1.0f / 24.0f * trace;
    n_a.vd[(16) * para->number_of_nodes + single_nodeindex] =
        rho_times_coeff - 1.0f / 12.0f * (local_j[1] + local_j[2]) +
        1.0f / 8.0f * (tmp1 + tmp2) - 1.0f / 24.0f * trace;
    n_a.vd[(17) * para->number_of_nodes + single_nodeindex] =
        rho_times_coeff + 1.0f / 12.0f * (local_j[1] - local_j[2]) +
        1.0f / 8.0f * (tmp1 - tmp2) - 1.0f / 24.0f * trace;
    n_a.vd[(18) * para->number_of_nodes + single_nodeindex] =
        rho_times_coeff - 1.0f / 12.0f * (local_j[1] - local_j[2]) +
        1.0f / 8.0f * (tmp1 - tmp2) - 1.0f / 24.0f * trace;

    // Calculate the modes for this node

    calc_m_from_n(n_a, single_nodeindex, mode_for_pi);

    // Update the density and velocity field for this mode

    update_rho_v(mode_for_pi, single_nodeindex, node_f, d_v);
  }
}

/** Calculate the mass of the whole fluid kernel
 *  @param[out] sum  Resulting mass
 *  @param[in]  n_a  Local node residing in array a
 */
__global__ void calc_mass(LB_nodes_gpu n_a, float *sum) {

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x +
                       blockDim.x * blockIdx.x + threadIdx.x;

  if (index < para->number_of_nodes) {
    Utils::Array<float, 4> mode;
    calc_mode(mode, n_a, index);
    float Rho = mode[0] + para->rho;
    atomicAdd(&(sum[0]), Rho);
  }
}

/** (Re-)initialize the node force density / set the external force
 *  density in lb units
 *  @param[out] node_f  Local node force density
 */
__global__ void reinit_node_force(LB_node_force_density_gpu node_f) {
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x +
                       blockDim.x * blockIdx.x + threadIdx.x;

  if (index < para->number_of_nodes) {
    node_f.force_density[0 * para->number_of_nodes + index] = 0.0f;
    node_f.force_density[1 * para->number_of_nodes + index] = 0.0f;
    node_f.force_density[2 * para->number_of_nodes + index] = 0.0f;
  }
}

/** Set external force on single nodes kernel
 *  @param[in]  n_extern_node_force_densities  Number of nodes
 *  @param[in]  extern_node_force_densities    External node force array
 *  @param[out] node_f                         Node force struct
 */
__global__ void init_extern_node_force_densities(
    int n_extern_node_force_densities,
    LB_extern_nodeforcedensity_gpu *extern_node_force_densities,
    LB_node_force_density_gpu node_f) {
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x +
                       blockDim.x * blockIdx.x + threadIdx.x;
  float factor = powf(para->agrid, 2) * para->tau * para->tau;
  if (index < n_extern_node_force_densities) {
    node_f.force_density[0 * para->number_of_nodes +
                         extern_node_force_densities[index].index] =
        extern_node_force_densities[index].force_density[0] * factor;
    node_f.force_density[1 * para->number_of_nodes +
                         extern_node_force_densities[index].index] =
        extern_node_force_densities[index].force_density[1] * factor;
    node_f.force_density[2 * para->number_of_nodes +
                         extern_node_force_densities[index].index] =
        extern_node_force_densities[index].force_density[2] * factor;
  }
}

/** Kernel to set the local density
 *
 *  @param[out] n_a              Current nodes array (double buffering!)
 *  @param[in] single_nodeindex  Node to set the velocity for
 *  @param[in] rho               Density to set
 *  @param[in] d_v               Local modes
 */
__global__ void set_rho(LB_nodes_gpu n_a, LB_rho_v_gpu *d_v,
                        int single_nodeindex, float rho) {
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x +
                       blockDim.x * blockIdx.x + threadIdx.x;
  /*Note: this sets the velocities to zero */
  if (index == 0) {
    float local_rho;

    /** default values for fields in lattice units */
    local_rho = (rho - para->rho);
    d_v[single_nodeindex].rho = rho;

    n_a.vd[0 * para->number_of_nodes + single_nodeindex] =
        1.0f / 3.0f * local_rho;
    n_a.vd[1 * para->number_of_nodes + single_nodeindex] =
        1.0f / 18.0f * local_rho;
    n_a.vd[2 * para->number_of_nodes + single_nodeindex] =
        1.0f / 18.0f * local_rho;
    n_a.vd[3 * para->number_of_nodes + single_nodeindex] =
        1.0f / 18.0f * local_rho;
    n_a.vd[4 * para->number_of_nodes + single_nodeindex] =
        1.0f / 18.0f * local_rho;
    n_a.vd[5 * para->number_of_nodes + single_nodeindex] =
        1.0f / 18.0f * local_rho;
    n_a.vd[6 * para->number_of_nodes + single_nodeindex] =
        1.0f / 18.0f * local_rho;
    n_a.vd[7 * para->number_of_nodes + single_nodeindex] =
        1.0f / 36.0f * local_rho;
    n_a.vd[8 * para->number_of_nodes + single_nodeindex] =
        1.0f / 36.0f * local_rho;
    n_a.vd[9 * para->number_of_nodes + single_nodeindex] =
        1.0f / 36.0f * local_rho;
    n_a.vd[10 * para->number_of_nodes + single_nodeindex] =
        1.0f / 36.0f * local_rho;
    n_a.vd[11 * para->number_of_nodes + single_nodeindex] =
        1.0f / 36.0f * local_rho;
    n_a.vd[12 * para->number_of_nodes + single_nodeindex] =
        1.0f / 36.0f * local_rho;
    n_a.vd[13 * para->number_of_nodes + single_nodeindex] =
        1.0f / 36.0f * local_rho;
    n_a.vd[14 * para->number_of_nodes + single_nodeindex] =
        1.0f / 36.0f * local_rho;
    n_a.vd[15 * para->number_of_nodes + single_nodeindex] =
        1.0f / 36.0f * local_rho;
    n_a.vd[16 * para->number_of_nodes + single_nodeindex] =
        1.0f / 36.0f * local_rho;
    n_a.vd[17 * para->number_of_nodes + single_nodeindex] =
        1.0f / 36.0f * local_rho;
    n_a.vd[18 * para->number_of_nodes + single_nodeindex] =
        1.0f / 36.0f * local_rho;
  }
}

/** Set the boundary flag for all boundary nodes
 *  @param[in]  boundary_node_list    Indices of the boundary nodes
 *  @param[in]  boundary_index_list   Flag for the corresponding boundary
 *  @param[in]  number_of_boundnodes  Number of boundary nodes
 *  @param[out] n_a                   Local node residing in array a
 *  @param[out] n_b                   Local node residing in array b
 */
__global__ void init_boundaries(int *boundary_node_list,
                                int *boundary_index_list,
                                int number_of_boundnodes, LB_nodes_gpu n_a,
                                LB_nodes_gpu n_b) {
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x +
                       blockDim.x * blockIdx.x + threadIdx.x;

  if (index < number_of_boundnodes) {
    n_a.boundary[boundary_node_list[index]] = boundary_index_list[index];
    n_b.boundary[boundary_node_list[index]] = boundary_index_list[index];
  }
}

/** Reset the boundary flag of every node
 *  @param[out] n_a   Local node residing in array a
 *  @param[out] n_b   Local node residing in array b
 */
__global__ void reset_boundaries(LB_nodes_gpu n_a, LB_nodes_gpu n_b) {
  size_t index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x +
                 threadIdx.x;
  if (index < para->number_of_nodes)
    n_a.boundary[index] = n_b.boundary[index] = 0;
}

/** Integration step of the LB-fluid-solver
 *  @param[in]     n_a     Local node residing in array a
 *  @param[out]    n_b     Local node residing in array b
 *  @param[in,out] d_v     Local device values
 *  @param[in,out] node_f  Local node force density
 *  @param[in]     ek_parameters_gpu  Parameters for the electrokinetics
 *  @param[in]     philox_counter
 */
__global__ void integrate(LB_nodes_gpu n_a, LB_nodes_gpu n_b, LB_rho_v_gpu *d_v,
                          LB_node_force_density_gpu node_f,
                          EK_parameters *ek_parameters_gpu,
                          unsigned int philox_counter) {
  /*every node is connected to a thread via the index*/
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x +
                       blockDim.x * blockIdx.x + threadIdx.x;
  /*the 19 moments (modes) are only temporary register values */
  Utils::Array<float, 19> mode;

  if (index < para->number_of_nodes) {
    calc_m_from_n(n_a, index, mode);
    relax_modes(mode, index, node_f, d_v);
    if (para->kT > 0.0) {
      thermalize_modes(mode, index, philox_counter);
    }
    apply_forces(index, mode, node_f, d_v);
    normalize_modes(mode);
    calc_n_from_modes_push(n_b, mode, index);
  }
}

/** Particle interaction kernel
 *  @param[in]  n_a                 Local node residing in array a
 *  @param[in,out]  particle_data   Particle position and velocity
 *  @param[in,out]  particle_force  Particle force
 *  @param[out] node_f              Local node force
 *  @param[in]  d_v                 Local device values
 *  @param[in]  couple_virtual
 *  @param[in]  philox_counter
 *  @param[in]  friction            Friction constant for the particle coupling
 *  @param[in]  lb_boundary_velocity Velocity at the boundary
 *  @tparam     no_of_neighbours    The number of neighbours to consider for
 * interpolation
 */
template <std::size_t no_of_neighbours>
__global__ void calc_fluid_particle_ia(
    LB_nodes_gpu n_a, CUDA_particle_data *particle_data, float *particle_force,
    LB_node_force_density_gpu node_f, LB_rho_v_gpu *d_v, bool couple_virtual,
    uint64_t philox_counter, float friction, float *lb_boundary_velocity) {

  unsigned int part_index = blockIdx.y * gridDim.x * blockDim.x +
                            blockDim.x * blockIdx.x + threadIdx.x;
  Utils::Array<unsigned int, no_of_neighbours> node_index;
  Utils::Array<float, no_of_neighbours> delta;
  float delta_j[3];
  if (part_index < para->number_of_particles) {
#if defined(VIRTUAL_SITES)
    if (!particle_data[part_index].is_virtual || couple_virtual)
#endif
    {
      /* force acting on the particle. delta_j will be used later to compute the
       * force that acts back onto the fluid. */
      calc_viscous_force<no_of_neighbours>(
          n_a, delta, particle_data, particle_force, part_index, delta_j,
          node_index, d_v, 0, philox_counter, friction, lb_boundary_velocity);
      calc_node_force<no_of_neighbours>(delta, delta_j, node_index, node_f);

#ifdef ENGINE
      if (particle_data[part_index].swim.swimming) {
        calc_viscous_force<no_of_neighbours>(
            n_a, delta, particle_data, particle_force, part_index, delta_j,
            node_index, d_v, 1, philox_counter, friction, lb_boundary_velocity);
        calc_node_force<no_of_neighbours>(delta, delta_j, node_index, node_f);
      }
#endif
    }
  }
}

#ifdef LB_BOUNDARIES_GPU
/** Bounce back boundary kernel
 *  @param[in]  n_curr  Pointer to local node receiving the current node field
 *  @param[in]  lb_boundary_velocity  Constant velocity at the boundary,
 *                                    set by the user
 *  @param[out] lb_boundary_force     Force on the boundary nodes
 */
__global__ void apply_boundaries(LB_nodes_gpu n_curr,
                                 float *lb_boundary_velocity,
                                 float *lb_boundary_force) {
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x +
                       blockDim.x * blockIdx.x + threadIdx.x;

  if (index < para->number_of_nodes)
    bounce_back_boundaries(n_curr, index, lb_boundary_velocity,
                           lb_boundary_force);
}

#endif

/** Get physical values of the nodes (density, velocity, ...)
 *  @param[in]  n_a     Local node residing in array a
 *  @param[out] p_v     Local print values
 *  @param[out] d_v     Local device values
 *  @param[in]  node_f  Local node force
 */
__global__ void
get_mesoscopic_values_in_LB_units(LB_nodes_gpu n_a, LB_rho_v_pi_gpu *p_v,
                                  LB_rho_v_gpu *d_v,
                                  LB_node_force_density_gpu node_f) {
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x +
                       blockDim.x * blockIdx.x + threadIdx.x;

  if (index < para->number_of_nodes) {
    Utils::Array<float, 19> mode;
    calc_m_from_n(n_a, index, mode);
    calc_values_in_LB_units(n_a, mode, p_v, d_v, node_f, index, index);
  }
}

/** Get boundary flags
 *  @param[in]  n_a                 Local node residing in array a
 *  @param[out] device_bound_array  Local device values
 */
__global__ void lb_get_boundaries(LB_nodes_gpu n_a,
                                  unsigned int *device_bound_array) {
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x +
                       blockDim.x * blockIdx.x + threadIdx.x;

  if (index < para->number_of_nodes)
    device_bound_array[index] = n_a.boundary[index];
}

/** Print single node values kernel
 *  @param[in]  single_nodeindex  Node index
 *  @param[out] d_p_v   Result
 *  @param[in]  n_a     Local node residing in array a
 *  @param[out] d_v     Local device values
 *  @param[in]  node_f  Local node force
 */
__global__ void lb_print_node(int single_nodeindex, LB_rho_v_pi_gpu *d_p_v,
                              LB_nodes_gpu n_a, LB_rho_v_gpu *d_v,
                              LB_node_force_density_gpu node_f) {
  Utils::Array<float, 19> mode;
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x +
                       blockDim.x * blockIdx.x + threadIdx.x;

  if (index == 0) {
    calc_m_from_n(n_a, single_nodeindex, mode);

    /* the following actually copies rho and v from d_v, and calculates pi */
    calc_values_in_LB_units(n_a, mode, d_p_v, d_v, node_f, single_nodeindex, 0);
  }
}

__global__ void momentum(LB_nodes_gpu n_a, LB_rho_v_gpu *d_v,
                         LB_node_force_density_gpu node_f, float *sum) {
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x +
                       blockDim.x * blockIdx.x + threadIdx.x;

  if (index < para->number_of_nodes) {
    float j[3] = {0.0f, 0.0f, 0.0f};
    Utils::Array<float, 4> mode{};

    calc_mode(mode, n_a, index);

    j[0] += mode[1] + node_f.force_density[0 * para->number_of_nodes + index];
    j[1] += mode[2] + node_f.force_density[1 * para->number_of_nodes + index];
    j[2] += mode[3] + node_f.force_density[2 * para->number_of_nodes + index];

#ifdef LB_BOUNDARIES_GPU
    if (n_a.boundary[index])
      j[0] = j[1] = j[2] = 0.0f;
#endif

    atomicAdd(&(sum[0]), j[0]);
    atomicAdd(&(sum[1]), j[1]);
    atomicAdd(&(sum[2]), j[2]);
  }
}
__global__ void remove_momentum(LB_nodes_gpu n_a, LB_rho_v_gpu *d_v,
                                LB_node_force_density_gpu node_f, float *sum) {
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x +
                       blockDim.x * blockIdx.x + threadIdx.x;
  if (index < para->number_of_nodes) {
    node_f.force_density[0 * para->number_of_nodes + index] -=
        sum[0] / para->number_of_nodes;
    node_f.force_density[1 * para->number_of_nodes + index] -=
        sum[1] / para->number_of_nodes;
    node_f.force_density[2 * para->number_of_nodes + index] -=
        sum[2] / para->number_of_nodes;
  }
}

/** Print single node boundary flag
 *  @param[in]  single_nodeindex  Node index
 *  @param[out] device_flag       Result
 *  @param[in]  n_a               Local node residing in array a
 */
__global__ void lb_get_boundary_flag(int single_nodeindex,
                                     unsigned int *device_flag,
                                     LB_nodes_gpu n_a) {
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x +
                       blockDim.x * blockIdx.x + threadIdx.x;

  if (index == 0)
    device_flag[0] = n_a.boundary[single_nodeindex];
}

/**********************************************************************/
/* Host functions to setup and call kernels*/
/**********************************************************************/

void lb_get_para_pointer(LB_parameters_gpu **pointeradress) {
  if (hipGetSymbolAddress((void **)pointeradress, HIP_SYMBOL(para)) !=
      hipSuccess) {
    fprintf(stderr,
            "Trouble getting address of LB parameters.\n"); // TODO give proper
                                                            // error message
    errexit();
  }
}

void lb_get_lbpar_pointer(LB_parameters_gpu **pointeradress) {
  *pointeradress = &lbpar_gpu;
}

void lb_get_boundary_force_pointer(float **pointeradress) {
#ifdef LB_BOUNDARIES_GPU
  *pointeradress = lb_boundary_force;
#endif
}

void lb_get_device_values_pointer(LB_rho_v_gpu **pointeradress) {
  *pointeradress = device_rho_v;
}

/** Initialization for the lb gpu fluid called from host
 *  @param lbpar_gpu   Pointer to parameters to setup the lb field
 */
void lb_init_GPU(LB_parameters_gpu *lbpar_gpu) {
#define free_realloc_and_clear(var, size)                                      \
  {                                                                            \
    if ((var) != nullptr)                                                      \
      cuda_safe_mem(hipFree((var)));                                          \
    cuda_safe_mem(hipMalloc((void **)&var, size));                            \
    hipMemset(var, 0, size);                                                  \
  }

  size_of_rho_v = lbpar_gpu->number_of_nodes * sizeof(LB_rho_v_gpu);
  size_of_rho_v_pi = lbpar_gpu->number_of_nodes * sizeof(LB_rho_v_pi_gpu);

  /** Allocate structs in device memory*/
  free_realloc_and_clear(device_rho_v, size_of_rho_v);

  /* TODO: this is a almost a copy of device_rho_v; think about eliminating
   * it, and maybe pi can be added to device_rho_v in this case */
  free_realloc_and_clear(print_rho_v_pi, size_of_rho_v_pi);
  free_realloc_and_clear(nodes_a.vd,
                         lbpar_gpu->number_of_nodes * 19 * sizeof(float));
  free_realloc_and_clear(nodes_b.vd,
                         lbpar_gpu->number_of_nodes * 19 * sizeof(float));
  free_realloc_and_clear(node_f.force_density,
                         lbpar_gpu->number_of_nodes * 3 * sizeof(lbForceFloat));
#if defined(VIRTUAL_SITES_INERTIALESS_TRACERS) || defined(EK_DEBUG)
  free_realloc_and_clear(node_f.force_density_buf,
                         lbpar_gpu->number_of_nodes * 3 * sizeof(lbForceFloat));
#endif
  free_realloc_and_clear(nodes_a.boundary,
                         lbpar_gpu->number_of_nodes * sizeof(unsigned int));
  free_realloc_and_clear(nodes_b.boundary,
                         lbpar_gpu->number_of_nodes * sizeof(unsigned int));

  /*write parameters in const memory*/
  cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu,
                                   sizeof(LB_parameters_gpu)));

  /*check flag if lb gpu init works*/
  free_realloc_and_clear(gpu_check, sizeof(int));

  if (h_gpu_check != nullptr)
    free(h_gpu_check);

  h_gpu_check = (int *)Utils::malloc(sizeof(int));

  /* values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x =
      (lbpar_gpu->number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /
      (threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(reset_boundaries, dim_grid, threads_per_block, nodes_a, nodes_b);

  /* calc of velocitydensities from given parameters and initialize the
   * Node_Force array with zero */
  KERNELCALL(reinit_node_force, dim_grid, threads_per_block, (node_f));
  KERNELCALL(calc_n_from_rho_j_pi, dim_grid, threads_per_block, nodes_a,
             device_rho_v, node_f, gpu_check);

  intflag = true;
  current_nodes = &nodes_a;
  h_gpu_check[0] = 0;
  cuda_safe_mem(
      hipMemcpy(h_gpu_check, gpu_check, sizeof(int), hipMemcpyDeviceToHost));
  hipDeviceSynchronize();

  if (!h_gpu_check[0]) {
    fprintf(stderr, "initialization of lb gpu code failed! \n");
    errexit();
  }
}

/** Reinitialization for the lb gpu fluid called from host
 *  @param lbpar_gpu   Pointer to parameters to setup the lb field
 */
void lb_reinit_GPU(LB_parameters_gpu *lbpar_gpu) {
  /* write parameters in const memory */
  cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu,
                                   sizeof(LB_parameters_gpu)));

  /* values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x =
      (lbpar_gpu->number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /
      (threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  /* calc of velocity densities from given parameters and initialize the
   * Node_Force array with zero */
  KERNELCALL(calc_n_from_rho_j_pi, dim_grid, threads_per_block, nodes_a,
             device_rho_v, node_f, gpu_check);
}

void lb_realloc_particles_GPU_leftovers(LB_parameters_gpu *lbpar_gpu) {
  // copy parameters, especially number of parts to gpu mem
  cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu,
                                   sizeof(LB_parameters_gpu)));
}

#ifdef LB_BOUNDARIES_GPU
/** Setup and call boundaries from the host
 *  @param host_n_lb_boundaries        Number of LB boundaries
 *  @param number_of_boundnodes        Number of boundnodes
 *  @param host_boundary_node_list     The indices of the boundary nodes
 *  @param host_boundary_index_list    The flag representing the corresponding
 *                                     boundary
 *  @param host_lb_boundary_velocity   The constant velocity at the boundary,
 *                                     set by the user
 */
void lb_init_boundaries_GPU(int host_n_lb_boundaries, int number_of_boundnodes,
                            int *host_boundary_node_list,
                            int *host_boundary_index_list,
                            float *host_lb_boundary_velocity) {
  if (this_node != 0)
    return;

  size_of_boundindex = number_of_boundnodes * sizeof(int);
  cuda_safe_mem(hipMalloc((void **)&boundary_node_list, size_of_boundindex));
  cuda_safe_mem(hipMalloc((void **)&boundary_index_list, size_of_boundindex));
  cuda_safe_mem(hipMemcpy(boundary_index_list, host_boundary_index_list,
                           size_of_boundindex, hipMemcpyHostToDevice));
  cuda_safe_mem(hipMemcpy(boundary_node_list, host_boundary_node_list,
                           size_of_boundindex, hipMemcpyHostToDevice));
  cuda_safe_mem(hipMalloc((void **)&lb_boundary_force,
                           3 * host_n_lb_boundaries * sizeof(float)));
  cuda_safe_mem(hipMalloc((void **)&lb_boundary_velocity,
                           3 * host_n_lb_boundaries * sizeof(float)));
  cuda_safe_mem(
      hipMemcpy(lb_boundary_velocity, host_lb_boundary_velocity,
                 3 * LBBoundaries::lbboundaries.size() * sizeof(float),
                 hipMemcpyHostToDevice));

  /* values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x =
      (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /
      (threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(reset_boundaries, dim_grid, threads_per_block, nodes_a, nodes_b);

  if (LBBoundaries::lbboundaries.size() == 0 && !pdb_boundary_lattice) {
    hipDeviceSynchronize();
    return;
  }

  if (number_of_boundnodes == 0) {
    fprintf(stderr,
            "WARNING: boundary cmd executed but no boundary node found!\n");
  } else {
    int threads_per_block_bound = 64;
    int blocks_per_grid_bound_y = 4;
    int blocks_per_grid_bound_x =
        (number_of_boundnodes +
         threads_per_block_bound * blocks_per_grid_bound_y - 1) /
        (threads_per_block_bound * blocks_per_grid_bound_y);
    dim3 dim_grid_bound =
        make_uint3(blocks_per_grid_bound_x, blocks_per_grid_bound_y, 1);

    KERNELCALL(init_boundaries, dim_grid_bound, threads_per_block_bound,
               boundary_node_list, boundary_index_list, number_of_boundnodes,
               nodes_a, nodes_b);
  }

  hipDeviceSynchronize();
}
#endif
/** Setup and call extern single node force initialization from the host
 *  @param lbpar_gpu    Host parameter struct
 */
void lb_reinit_extern_nodeforce_GPU(LB_parameters_gpu *lbpar_gpu) {
  cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu,
                                   sizeof(LB_parameters_gpu)));

  /* values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x =
      (lbpar_gpu->number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /
      (threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(reinit_node_force, dim_grid, threads_per_block, node_f);
}
/** Setup and call extern single node force initialization from the host
 *  @param n_extern_node_force_densities     Number of nodes on which the
 *                                           external force has to be applied
 *  @param host_extern_node_force_densities  Host extern node forces
 *  @param lbpar_gpu                         Host parameter struct
 */
void lb_init_extern_nodeforcedensities_GPU(
    int n_extern_node_force_densities,
    LB_extern_nodeforcedensity_gpu *host_extern_node_force_densities,
    LB_parameters_gpu *lbpar_gpu) {

  size_of_extern_node_force_densities =
      n_extern_node_force_densities * sizeof(LB_extern_nodeforcedensity_gpu);
  cuda_safe_mem(hipMalloc((void **)&extern_node_force_densities,
                           size_of_extern_node_force_densities));
  cuda_safe_mem(
      hipMemcpy(extern_node_force_densities, host_extern_node_force_densities,
                 size_of_extern_node_force_densities, hipMemcpyHostToDevice));

  cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu,
                                   sizeof(LB_parameters_gpu)));

  int threads_per_block_exf = 64;
  int blocks_per_grid_exf_y = 4;
  int blocks_per_grid_exf_x =
      (n_extern_node_force_densities +
       threads_per_block_exf * blocks_per_grid_exf_y - 1) /
      (threads_per_block_exf * blocks_per_grid_exf_y);
  dim3 dim_grid_exf =
      make_uint3(blocks_per_grid_exf_x, blocks_per_grid_exf_y, 1);

  KERNELCALL(init_extern_node_force_densities, dim_grid_exf,
             threads_per_block_exf, n_extern_node_force_densities,
             extern_node_force_densities, node_f);
  hipFree(extern_node_force_densities);
}

/** Setup and call particle kernel from the host
 *  @tparam no_of_neighbours       The number of neighbours to consider for
 * interpolation
 */
template <std::size_t no_of_neighbours>
void lb_calc_particle_lattice_ia_gpu(bool couple_virtual, double friction) {
  if (lbpar_gpu.number_of_particles) {
    /* call of the particle kernel */
    /* values for the particle kernel */
    int threads_per_block_particles = 64;
    int blocks_per_grid_particles_y = 4;
    int blocks_per_grid_particles_x =
        (lbpar_gpu.number_of_particles +
         threads_per_block_particles * blocks_per_grid_particles_y - 1) /
        (threads_per_block_particles * blocks_per_grid_particles_y);
    dim3 dim_grid_particles =
        make_uint3(blocks_per_grid_particles_x, blocks_per_grid_particles_y, 1);

    KERNELCALL(
        calc_fluid_particle_ia<no_of_neighbours>, dim_grid_particles,
        threads_per_block_particles, *current_nodes, gpu_get_particle_pointer(),
        gpu_get_particle_force_pointer(), node_f, device_rho_v, couple_virtual,
        rng_counter_coupling_gpu.value(), friction, lb_boundary_velocity);
    rng_counter_coupling_gpu.increment();
  }
}
template void lb_calc_particle_lattice_ia_gpu<8>(bool couple_virtual,
                                                 double friction);
template void lb_calc_particle_lattice_ia_gpu<27>(bool couple_virtual,
                                                  double friction);

/** Setup and call kernel for getting macroscopic fluid values of all nodes
 *  @param host_values   struct to save the gpu values
 */
void lb_get_values_GPU(LB_rho_v_pi_gpu *host_values) {
  /* values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x =
      (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /
      (threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(get_mesoscopic_values_in_LB_units, dim_grid, threads_per_block,
             *current_nodes, print_rho_v_pi, device_rho_v, node_f);
  cuda_safe_mem(hipMemcpy(host_values, print_rho_v_pi, size_of_rho_v_pi,
                           hipMemcpyDeviceToHost));
}

/** Get all the boundary flags for all nodes
 *  @param host_bound_array   here go the values of the boundary flag
 */
void lb_get_boundary_flags_GPU(unsigned int *host_bound_array) {
  unsigned int *device_bound_array;
  cuda_safe_mem(hipMalloc((void **)&device_bound_array,
                           lbpar_gpu.number_of_nodes * sizeof(unsigned int)));
  /* values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x =
      (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /
      (threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(lb_get_boundaries, dim_grid, threads_per_block, *current_nodes,
             device_bound_array);

  cuda_safe_mem(hipMemcpy(host_bound_array, device_bound_array,
                           lbpar_gpu.number_of_nodes * sizeof(unsigned int),
                           hipMemcpyDeviceToHost));

  hipFree(device_bound_array);
}

/** Setup and call kernel for getting macroscopic fluid values of a single
 *  node
 */
void lb_print_node_GPU(int single_nodeindex,
                       LB_rho_v_pi_gpu *host_print_values) {
  LB_rho_v_pi_gpu *device_print_values;
  cuda_safe_mem(
      hipMalloc((void **)&device_print_values, sizeof(LB_rho_v_pi_gpu)));
  int threads_per_block_print = 1;
  int blocks_per_grid_print_y = 1;
  int blocks_per_grid_print_x = 1;
  dim3 dim_grid_print =
      make_uint3(blocks_per_grid_print_x, blocks_per_grid_print_y, 1);

  KERNELCALL(lb_print_node, dim_grid_print, threads_per_block_print,
             single_nodeindex, device_print_values, *current_nodes,
             device_rho_v, node_f);

  cuda_safe_mem(hipMemcpy(host_print_values, device_print_values,
                           sizeof(LB_rho_v_pi_gpu), hipMemcpyDeviceToHost));
  hipFree(device_print_values);
}

/** Setup and call kernel to calculate the total momentum of the hole fluid
 *  @param mass   value of the mass calculated on the GPU
 */
void lb_calc_fluid_mass_GPU(double *mass) {
  float *tot_mass;
  float cpu_mass = 0.0f;
  cuda_safe_mem(hipMalloc((void **)&tot_mass, sizeof(float)));
  cuda_safe_mem(
      hipMemcpy(tot_mass, &cpu_mass, sizeof(float), hipMemcpyHostToDevice));

  /* values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x =
      (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /
      (threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(calc_mass, dim_grid, threads_per_block, *current_nodes, tot_mass);

  cuda_safe_mem(
      hipMemcpy(&cpu_mass, tot_mass, sizeof(float), hipMemcpyDeviceToHost));

  hipFree(tot_mass);
  mass[0] = (double)(cpu_mass);
}

/** Setup and call kernel to calculate the total momentum of the whole fluid
 *  @param host_mom   value of the momentum calculated on the GPU
 */
void lb_calc_fluid_momentum_GPU(double *host_mom) {
  float *tot_momentum;
  float host_momentum[3] = {0.0f, 0.0f, 0.0f};
  cuda_safe_mem(hipMalloc((void **)&tot_momentum, 3 * sizeof(float)));
  cuda_safe_mem(hipMemcpy(tot_momentum, host_momentum, 3 * sizeof(float),
                           hipMemcpyHostToDevice));

  /* values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x =
      (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /
      (threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(momentum, dim_grid, threads_per_block, *current_nodes,
             device_rho_v, node_f, tot_momentum);

  cuda_safe_mem(hipMemcpy(host_momentum, tot_momentum, 3 * sizeof(float),
                           hipMemcpyDeviceToHost));

  hipFree(tot_momentum);
  host_mom[0] = (double)(host_momentum[0] * lbpar_gpu.agrid / lbpar_gpu.tau);
  host_mom[1] = (double)(host_momentum[1] * lbpar_gpu.agrid / lbpar_gpu.tau);
  host_mom[2] = (double)(host_momentum[2] * lbpar_gpu.agrid / lbpar_gpu.tau);
}

/** Setup and call kernel to remove the net momentum of the whole fluid
 */
void lb_remove_fluid_momentum_GPU(void) {
  float *tot_momentum;
  float host_momentum[3] = {0.0f, 0.0f, 0.0f};
  cuda_safe_mem(hipMalloc((void **)&tot_momentum, 3 * sizeof(float)));
  cuda_safe_mem(hipMemcpy(tot_momentum, host_momentum, 3 * sizeof(float),
                           hipMemcpyHostToDevice));

  /* values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x =
      (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /
      (threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(momentum, dim_grid, threads_per_block, *current_nodes,
             device_rho_v, node_f, tot_momentum);

  cuda_safe_mem(hipMemcpy(host_momentum, tot_momentum, 3 * sizeof(float),
                           hipMemcpyDeviceToHost));

  KERNELCALL(remove_momentum, dim_grid, threads_per_block, *current_nodes,
             device_rho_v, node_f, tot_momentum);

  hipFree(tot_momentum);
}

/** Setup and call kernel to calculate the temperature of the hole fluid
 *  @param host_temp   value of the temperature calculated on the GPU
 */
void lb_calc_fluid_temperature_GPU(double *host_temp) {
  int host_number_of_non_boundary_nodes = 0;
  int *device_number_of_non_boundary_nodes;
  cuda_safe_mem(
      hipMalloc((void **)&device_number_of_non_boundary_nodes, sizeof(int)));
  cuda_safe_mem(hipMemcpy(device_number_of_non_boundary_nodes,
                           &host_number_of_non_boundary_nodes, sizeof(int),
                           hipMemcpyHostToDevice));

  float host_jsquared = 0.0f;
  float *device_jsquared;
  cuda_safe_mem(hipMalloc((void **)&device_jsquared, sizeof(float)));
  cuda_safe_mem(hipMemcpy(device_jsquared, &host_jsquared, sizeof(float),
                           hipMemcpyHostToDevice));

  /* values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x =
      (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /
      (threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(temperature, dim_grid, threads_per_block, *current_nodes,
             device_jsquared, device_number_of_non_boundary_nodes);

  cuda_safe_mem(hipMemcpy(&host_number_of_non_boundary_nodes,
                           device_number_of_non_boundary_nodes, sizeof(int),
                           hipMemcpyDeviceToHost));
  cuda_safe_mem(hipMemcpy(&host_jsquared, device_jsquared, sizeof(float),
                           hipMemcpyDeviceToHost));

  *host_temp = 0;

  *host_temp +=
      (double)(host_jsquared * 1. /
               (3.0f * lbpar_gpu.rho / lbpar_gpu.agrid / lbpar_gpu.agrid /
                lbpar_gpu.agrid * host_number_of_non_boundary_nodes *
                lbpar_gpu.tau * lbpar_gpu.tau * lbpar_gpu.agrid));
}

/** Setup and call kernel for getting macroscopic fluid values of all nodes
 *  @param[out] host_checkpoint_vd   LB populations
 */
void lb_save_checkpoint_GPU(float *const host_checkpoint_vd) {
  cuda_safe_mem(hipMemcpy(host_checkpoint_vd, current_nodes->vd,
                           lbpar_gpu.number_of_nodes * 19 * sizeof(float),
                           hipMemcpyDeviceToHost));
}

/** Setup and call kernel for getting macroscopic fluid values of all nodes
 *  @param[in] host_checkpoint_vd    LB populations
 */
void lb_load_checkpoint_GPU(float const *const host_checkpoint_vd) {
  current_nodes = &nodes_a;
  intflag = true;

  cuda_safe_mem(hipMemcpy(current_nodes->vd, host_checkpoint_vd,
                           lbpar_gpu.number_of_nodes * 19 * sizeof(float),
                           hipMemcpyHostToDevice));
}

/** Setup and call kernel to get the boundary flag of a single node
 *  @param single_nodeindex   number of the node to get the flag for
 *  @param host_flag          here goes the value of the boundary flag
 */
void lb_get_boundary_flag_GPU(int single_nodeindex, unsigned int *host_flag) {
  unsigned int *device_flag;
  cuda_safe_mem(hipMalloc((void **)&device_flag, sizeof(unsigned int)));
  int threads_per_block_flag = 1;
  int blocks_per_grid_flag_y = 1;
  int blocks_per_grid_flag_x = 1;
  dim3 dim_grid_flag =
      make_uint3(blocks_per_grid_flag_x, blocks_per_grid_flag_y, 1);

  KERNELCALL(lb_get_boundary_flag, dim_grid_flag, threads_per_block_flag,
             single_nodeindex, device_flag, *current_nodes);

  cuda_safe_mem(hipMemcpy(host_flag, device_flag, sizeof(unsigned int),
                           hipMemcpyDeviceToHost));

  hipFree(device_flag);
}

/** Set the density at a single node
 *  @param single_nodeindex   the node to set the velocity for
 *  @param host_rho           the density to set
 */
void lb_set_node_rho_GPU(int single_nodeindex, float host_rho) {
  int threads_per_block_flag = 1;
  int blocks_per_grid_flag_y = 1;
  int blocks_per_grid_flag_x = 1;
  dim3 dim_grid_flag =
      make_uint3(blocks_per_grid_flag_x, blocks_per_grid_flag_y, 1);
  KERNELCALL(set_rho, dim_grid_flag, threads_per_block_flag, *current_nodes,
             device_rho_v, single_nodeindex, host_rho);
}

/** Set the net velocity at a single node
 *  @param single_nodeindex   the node to set the velocity for
 *  @param host_velocity      the velocity to set
 */
void lb_set_node_velocity_GPU(int single_nodeindex, float *host_velocity) {
  float *device_velocity;
  cuda_safe_mem(hipMalloc((void **)&device_velocity, 3 * sizeof(float)));
  cuda_safe_mem(hipMemcpy(device_velocity, host_velocity, 3 * sizeof(float),
                           hipMemcpyHostToDevice));
  int threads_per_block_flag = 1;
  int blocks_per_grid_flag_y = 1;
  int blocks_per_grid_flag_x = 1;
  dim3 dim_grid_flag =
      make_uint3(blocks_per_grid_flag_x, blocks_per_grid_flag_y, 1);

  KERNELCALL(set_u_from_rho_v_pi, dim_grid_flag, threads_per_block_flag,
             *current_nodes, single_nodeindex, device_velocity, device_rho_v,
             node_f);

  hipFree(device_velocity);
}

/** Reinitialize parameters
 *  @param lbpar_gpu   struct containing the parameters of the fluid
 */
void reinit_parameters_GPU(LB_parameters_gpu *lbpar_gpu) {
  /*write parameters in const memory*/
  cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu,
                                   sizeof(LB_parameters_gpu)));
}

/** Integration kernel for the lb gpu fluid update called from host */
void lb_integrate_GPU() {
  /* values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x =
      (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /
      (threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);
#ifdef LB_BOUNDARIES_GPU
  if (LBBoundaries::lbboundaries.size() > 0) {
    cuda_safe_mem(
        hipMemset(lb_boundary_force, 0,
                   3 * LBBoundaries::lbboundaries.size() * sizeof(float)));
  }
#endif

  /* call of fluid step */
  if (intflag) {
    KERNELCALL(integrate, dim_grid, threads_per_block, nodes_a, nodes_b,
               device_rho_v, node_f, lb_ek_parameters_gpu,
               rng_counter_fluid_gpu.value());
    current_nodes = &nodes_b;
    intflag = false;
  } else {
    KERNELCALL(integrate, dim_grid, threads_per_block, nodes_b, nodes_a,
               device_rho_v, node_f, lb_ek_parameters_gpu,
               rng_counter_fluid_gpu.value());
    current_nodes = &nodes_a;
    intflag = true;
  }

#ifdef LB_BOUNDARIES_GPU
  if (LBBoundaries::lbboundaries.size() > 0) {
    KERNELCALL(apply_boundaries, dim_grid, threads_per_block, *current_nodes,
               lb_boundary_velocity, lb_boundary_force);
  }
#endif
}

void lb_gpu_get_boundary_forces(double *forces) {
#ifdef LB_BOUNDARIES_GPU
  float *temp = (float *)Utils::malloc(3 * LBBoundaries::lbboundaries.size() *
                                       sizeof(float));
  cuda_safe_mem(
      hipMemcpy(temp, lb_boundary_force,
                 3 * LBBoundaries::lbboundaries.size() * sizeof(float),
                 hipMemcpyDeviceToHost));

  for (int i = 0; i < 3 * LBBoundaries::lbboundaries.size(); i++) {
    forces[i] = (double)temp[i];
  }
  free(temp);
#endif
}

struct lb_lbfluid_mass_of_particle {
  __host__ __device__ float operator()(CUDA_particle_data particle) const {
#ifdef MASS
    return particle.mass;
#else
    return 1.;
#endif
  };
};

void lb_lbfluid_remove_total_momentum() {
  // calculate momentum of fluid and particles
  float total_momentum[3] = {0.0f, 0.0f, 0.0f};
  lb_lbfluid_calc_linear_momentum(total_momentum, /*include_particles*/ 1,
                                  /*include_lbfluid*/ 1);

  thrust::device_ptr<CUDA_particle_data> ptr(gpu_get_particle_pointer());
  float particles_mass = thrust::transform_reduce(
      ptr, ptr + lbpar_gpu.number_of_particles, lb_lbfluid_mass_of_particle(),
      0.0f, thrust::plus<float>());

  // lb_calc_fluid_mass_GPU has to be called with double but we don't
  // want narrowing warnings, that's why we narrow it down by hand.
  double lb_calc_fluid_mass_res;
  lb_calc_fluid_mass_GPU(&lb_calc_fluid_mass_res);
  float fluid_mass = lb_calc_fluid_mass_res;

  /* Momentum fraction of the particles */
  auto const part_frac = particles_mass / (fluid_mass + particles_mass);
  /* Momentum per particle */
  float momentum_particles[3] = {-total_momentum[0] * part_frac,
                                 -total_momentum[1] * part_frac,
                                 -total_momentum[2] * part_frac};

  auto const fluid_frac = fluid_mass / (fluid_mass + particles_mass);
  float momentum_fluid[3] = {-total_momentum[0] * fluid_frac,
                             -total_momentum[1] * fluid_frac,
                             -total_momentum[2] * fluid_frac};

  lb_lbfluid_particles_add_momentum(momentum_particles);
  lb_lbfluid_fluid_add_momentum(momentum_fluid);
}

__global__ void
lb_lbfluid_fluid_add_momentum_kernel(float momentum[3], LB_nodes_gpu n_a,
                                     LB_node_force_density_gpu node_f,
                                     LB_rho_v_gpu *d_v) {
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x +
                       blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int number_of_nodes = para->number_of_nodes;
#ifdef LB_BOUNDARIES_GPU
  number_of_nodes -= para->number_of_boundnodes;
#endif
  if (index < para->number_of_nodes) {
    if (n_a.boundary[index] == 0) {
      float force_factor = powf(para->agrid, 2) * para->tau * para->tau;
      // add force density onto each node (momentum / time_step / Volume)
      node_f.force_density[0 * para->number_of_nodes + index] +=
          momentum[0] / para->tau / (number_of_nodes * powf(para->agrid, 3)) *
          force_factor;
      node_f.force_density[1 * para->number_of_nodes + index] +=
          momentum[1] / para->tau / (number_of_nodes * powf(para->agrid, 3)) *
          force_factor;
      node_f.force_density[2 * para->number_of_nodes + index] +=
          momentum[2] / para->tau / (number_of_nodes * powf(para->agrid, 3)) *
          force_factor;
    }
  }
}

void lb_lbfluid_fluid_add_momentum(float momentum_host[3]) {
  float *momentum_device;
  cuda_safe_mem(hipMalloc((void **)&momentum_device, 3 * sizeof(float)));
  cuda_safe_mem(hipMemcpy(momentum_device, momentum_host, 3 * sizeof(float),
                           hipMemcpyHostToDevice));

  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x =
      (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /
      (threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(lb_lbfluid_fluid_add_momentum_kernel, dim_grid, threads_per_block,
             momentum_device, *current_nodes, node_f, device_rho_v);
}

/** Set the populations of a specific node on the GPU
 *  @param[out] n_a         Local node residing in array a
 *  @param[in]  population  New population
 *  @param[in]  x           x-coordinate of node
 *  @param[in]  y           y-coordinate of node
 *  @param[in]  z           z-coordinate of node
 */
__global__ void lb_lbfluid_set_population_kernel(LB_nodes_gpu n_a,
                                                 float population[LBQ], int x,
                                                 int y, int z) {
  auto const index = xyz_to_index(x, y, z);

  for (int i = 0; i < LBQ; ++i) {
    n_a.vd[i * para->number_of_nodes + index] = population[i];
  }
}

/** Interface to set the populations of a specific node for the GPU
 *  @param[in] xyz              Node coordinates
 *  @param[in] population_host  Population
 */
void lb_lbfluid_set_population(const Vector3i &xyz,
                               float population_host[LBQ]) {
  float *population_device;
  cuda_safe_mem(hipMalloc((void **)&population_device, LBQ * sizeof(float)));
  cuda_safe_mem(hipMemcpy(population_device, population_host,
                           LBQ * sizeof(float), hipMemcpyHostToDevice));

  dim3 dim_grid = make_uint3(1, 1, 1);
  KERNELCALL(lb_lbfluid_set_population_kernel, dim_grid, 1, *current_nodes,
             population_device, xyz[0], xyz[1], xyz[2]);

  cuda_safe_mem(hipFree(population_device));
}

/** Get the populations of a specific node on the GPU
 *  @param[in]  n_a         Local node residing in array a
 *  @param[out] population  Population
 *  @param[in]  x           x-coordinate of node
 *  @param[in]  y           y-coordinate of node
 *  @param[in]  z           z-coordinate of node
 */
__global__ void lb_lbfluid_get_population_kernel(LB_nodes_gpu n_a,
                                                 float population[LBQ], int x,
                                                 int y, int z) {
  auto const index = xyz_to_index(x, y, z);

  for (int i = 0; i < LBQ; ++i) {
    population[i] = n_a.vd[i * para->number_of_nodes + index];
  }
}

/** Interface to get the populations of a specific node for the GPU
 *  @param[in]  xyz              Node coordinates
 *  @param[out] population_host  Population
 */
void lb_lbfluid_get_population(const Vector3i &xyz,
                               float population_host[LBQ]) {
  float *population_device;
  cuda_safe_mem(hipMalloc((void **)&population_device, LBQ * sizeof(float)));

  dim3 dim_grid = make_uint3(1, 1, 1);
  KERNELCALL(lb_lbfluid_get_population_kernel, dim_grid, 1, *current_nodes,
             population_device, xyz[0], xyz[1], xyz[2]);

  cuda_safe_mem(hipMemcpy(population_host, population_device,
                           LBQ * sizeof(float), hipMemcpyDeviceToHost));

  cuda_safe_mem(hipFree(population_device));
}

/**
 * @brief Velocity interpolation functor
 * @tparam no_of_neighbours The number of neighbours to consider for
 * interpolation
 */
template <std::size_t no_of_neighbours> struct interpolation {
  LB_nodes_gpu current_nodes_gpu;
  LB_rho_v_gpu *d_v_gpu;
  float *lb_boundary_velocity;
  interpolation(LB_nodes_gpu _current_nodes_gpu, LB_rho_v_gpu *_d_v_gpu,
                float *lb_boundary_velocity)
      : current_nodes_gpu(_current_nodes_gpu), d_v_gpu(_d_v_gpu),
        lb_boundary_velocity(lb_boundary_velocity){};
  __device__ float3 operator()(const float3 &position) const {
    float _position[3] = {position.x, position.y, position.z};
    Utils::Array<unsigned int, no_of_neighbours> node_indices;
    Utils::Array<float, no_of_neighbours> delta;
    return velocity_interpolation(current_nodes_gpu, _position,
                                  lb_boundary_velocity, node_indices, delta);
  }
};

template <std::size_t no_of_neighbours>
void lb_get_interpolated_velocity_gpu(double const *positions,
                                      double *velocities, int length) {
  thrust::host_vector<float3> positions_host(length);
  for (int p = 0; p < 3 * length; p += 3) {
    // Cast double coming from python to float.
    positions_host[p / 3].x = static_cast<float>(positions[p]);
    positions_host[p / 3].y = static_cast<float>(positions[p + 1]);
    positions_host[p / 3].z = static_cast<float>(positions[p + 2]);
  }
  thrust::device_vector<float3> positions_device = positions_host;
  thrust::device_vector<float3> velocities_device(length);
  thrust::transform(positions_device.begin(), positions_device.end(),
                    velocities_device.begin(),
                    interpolation<no_of_neighbours>(
                        *current_nodes, device_rho_v, lb_boundary_velocity));
  thrust::host_vector<float3> velocities_host = velocities_device;
  int index = 0;
  for (auto v : velocities_host) {
    velocities[index] = static_cast<double>(v.x);
    velocities[index + 1] = static_cast<double>(v.y);
    velocities[index + 2] = static_cast<double>(v.z);
    index += 3;
  }
}
template void lb_get_interpolated_velocity_gpu<8>(double const *positions,
                                                  double *velocities,
                                                  int length);
template void lb_get_interpolated_velocity_gpu<27>(double const *positions,
                                                   double *velocities,
                                                   int length);

void linear_velocity_interpolation(double const *positions, double *velocities,
                                   int length) {
  return lb_get_interpolated_velocity_gpu<8>(positions, velocities, length);
}

void quadratic_velocity_interpolation(double const *positions,
                                      double *velocities, int length) {
  return lb_get_interpolated_velocity_gpu<27>(positions, velocities, length);
}

void lb_coupling_set_rng_state_gpu(uint64_t counter) {
  rng_counter_coupling_gpu = Utils::Counter<uint64_t>(counter);
}

void lb_fluid_set_rng_state_gpu(uint64_t counter) {
  rng_counter_fluid_gpu = Utils::Counter<uint64_t>(counter);
#ifdef ELECTROKINETICS
  ek_set_rng_state(counter);
#endif // ELECTROKINETICS
}

uint64_t lb_coupling_get_rng_state_gpu() {
  return rng_counter_coupling_gpu.value();
}
uint64_t lb_fluid_get_rng_state_gpu() { return rng_counter_fluid_gpu.value(); }

#endif /* LB_GPU */
