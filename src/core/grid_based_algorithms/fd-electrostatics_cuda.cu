#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2010-2019 The ESPResSo project
 *
 * This file is part of ESPResSo.
 *
 * ESPResSo is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * ESPResSo is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */
#include "cuda_wrapper.hpp"

// TODO: throw exceptions upon errors initialization

#include "cuda_utils.hpp"
#include "cufft_wrapper.hpp"
#include "grid_based_algorithms/fd-electrostatics.cuh"
#include <stdexcept>
#include <string>
//#include <cuda_interface.hpp>
#include <cstdio>

#if defined(OMPI_MPI_H) || defined(_MPI_H)
#error CU-file includes mpi.h! This should not happen!
#endif

__global__ void createGreensfcn();
__global__ void multiplyGreensfcn(hipfftComplex *charge_potential);

__device__ __constant__ FdElectrostatics::Parameters fde_parameters_gpu[1];

__device__ unsigned int fde_getThreadIndex() {

  return blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x +
         threadIdx.x;
}

__device__ hipfftReal fde_getNode(int x, int y, int z) {
  hipfftReal *field =
      reinterpret_cast<hipfftReal *>(fde_parameters_gpu->charge_potential);
  return field[fde_parameters_gpu->dim_y * fde_parameters_gpu->dim_x_padded *
                   z +
               fde_parameters_gpu->dim_x_padded * y + x];
}

__device__ void fde_setNode(int x, int y, int z, hipfftReal value) {
  hipfftReal *field =
      reinterpret_cast<hipfftReal *>(fde_parameters_gpu->charge_potential);
  field[fde_parameters_gpu->dim_y * fde_parameters_gpu->dim_x_padded * z +
        fde_parameters_gpu->dim_x_padded * y + x] = value;
}

__device__ hipfftReal fde_getNode(int i) {
  int x = i % fde_parameters_gpu->dim_x_padded;
  i /= fde_parameters_gpu->dim_x_padded;
  int y = i % fde_parameters_gpu->dim_y;
  int z = i / fde_parameters_gpu->dim_y;
  return fde_getNode(x, y, z);
}

__device__ void fde_setNode(int i, hipfftReal value) {
  int x = i % fde_parameters_gpu->dim_x_padded;
  i /= fde_parameters_gpu->dim_x_padded;
  int y = i % fde_parameters_gpu->dim_y;
  int z = i / fde_parameters_gpu->dim_y;
  fde_setNode(x, y, z, value);
}

FdElectrostatics::~FdElectrostatics() {
  hipfftDestroy(plan_ifft);
  hipfftDestroy(plan_fft);

  cuda_safe_mem(hipFree(parameters.greensfcn));
  cuda_safe_mem(hipFree(parameters.charge_potential));
}

FdElectrostatics::FdElectrostatics(InputParameters inputParameters,
                                   hipStream_t stream)
    : parameters(inputParameters), hip_stream(stream) {
  cuda_safe_mem(hipMalloc((void **)&parameters.charge_potential,
                           sizeof(hipfftComplex) * parameters.dim_z *
                               parameters.dim_y * (parameters.dim_x / 2 + 1)));

  cuda_safe_mem(hipMalloc((void **)&parameters.greensfcn,
                           sizeof(hipfftReal) * parameters.dim_z *
                               parameters.dim_y * (parameters.dim_x / 2 + 1)));

  if (hipGetLastError() != hipSuccess) {
    throw std::runtime_error("Failed to allocate");
  }

  cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(fde_parameters_gpu), &parameters,
                                   sizeof(Parameters)));

  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x =
      (parameters.dim_z * parameters.dim_y * (parameters.dim_x / 2 + 1) +
       threads_per_block * blocks_per_grid_y - 1) /
      (threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);
  KERNELCALL_stream(createGreensfcn, dim_grid, threads_per_block, stream);

  /* create 3D FFT plans */

  if (hipfftPlan3d(&plan_fft, parameters.dim_z, parameters.dim_y,
                  parameters.dim_x, HIPFFT_R2C) != HIPFFT_SUCCESS) {
    throw std::runtime_error("Unable to create fft plan");
  }

  if (hipfftSetStream(plan_fft, hip_stream) != HIPFFT_SUCCESS) {
    throw std::runtime_error("Unable to assign FFT to cuda stream");
  }

  if (hipfftPlan3d(&plan_ifft, parameters.dim_z, parameters.dim_y,
                  parameters.dim_x, HIPFFT_C2R) != HIPFFT_SUCCESS) {
    throw std::runtime_error("Unable to create ifft plan");
  }

  if (hipfftSetStream(plan_ifft, hip_stream) != HIPFFT_SUCCESS) {
    throw std::runtime_error("Unable to assign FFT to cuda stream");
  }

  initialized = true;
}

__global__ void createGreensfcn() {
  unsigned int index = fde_getThreadIndex();
  unsigned int tmp;
  unsigned int coord[3];

  coord[0] = index % (fde_parameters_gpu->dim_x / 2 + 1);
  tmp = index / (fde_parameters_gpu->dim_x / 2 + 1);
  coord[1] = tmp % fde_parameters_gpu->dim_y;
  coord[2] = tmp / fde_parameters_gpu->dim_y;

  if (index < fde_parameters_gpu->dim_z * fde_parameters_gpu->dim_y *
                  (fde_parameters_gpu->dim_x / 2 + 1)) {

    if (index == 0) {
      // setting 0th Fourier mode to 0 enforces charge neutrality
      fde_parameters_gpu->greensfcn[index] = 0.0f;
    } else {
      fde_parameters_gpu->greensfcn[index] =
          -4.0f * PI_FLOAT * fde_parameters_gpu->prefactor *
          fde_parameters_gpu->agrid * fde_parameters_gpu->agrid * 0.5f /
          (cos(2.0f * PI_FLOAT * coord[0] /
               (hipfftReal)fde_parameters_gpu->dim_x) +
           cos(2.0f * PI_FLOAT * coord[1] /
               (hipfftReal)fde_parameters_gpu->dim_y) +
           cos(2.0f * PI_FLOAT * coord[2] /
               (hipfftReal)fde_parameters_gpu->dim_z) -
           3.0f) /
          (fde_parameters_gpu->dim_x * fde_parameters_gpu->dim_y *
           fde_parameters_gpu->dim_z);
    }

    // fde_parameters_gpu->greensfcn[index] = 0.0f; //TODO delete
  }
}

__global__ void multiplyGreensfcn(hipfftComplex *charge_potential) {

  unsigned int index = fde_getThreadIndex();

  if (index < fde_parameters_gpu->dim_z * fde_parameters_gpu->dim_y *
                  (fde_parameters_gpu->dim_x / 2 + 1)) {
    charge_potential[index].x *= fde_parameters_gpu->greensfcn[index];
    charge_potential[index].y *= fde_parameters_gpu->greensfcn[index];
  }
}

void FdElectrostatics::calculatePotential() {
  calculatePotential(parameters.charge_potential);
}

void FdElectrostatics::calculatePotential(hipfftComplex *charge_potential) {

  if (hipfftExecR2C(plan_fft, (hipfftReal *)charge_potential, charge_potential) !=
      HIPFFT_SUCCESS) {

    fprintf(stderr, "ERROR: Unable to execute FFT plan\n");
  }

  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x =
      (parameters.dim_z * parameters.dim_y * (parameters.dim_x / 2 + 1) +
       threads_per_block * blocks_per_grid_y - 1) /
      (threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(multiplyGreensfcn, dim_grid, threads_per_block, charge_potential);

  if (hipfftExecC2R(plan_ifft, charge_potential,
                   (hipfftReal *)charge_potential) != HIPFFT_SUCCESS) {

    fprintf(stderr, "ERROR: Unable to execute iFFT plan\n");
  }
}

FdElectrostatics::Grid FdElectrostatics::getGrid() {
  Grid g = {(float *)parameters.charge_potential, parameters.dim_x,
            parameters.dim_y, parameters.dim_z, parameters.agrid};
  return g;
}
