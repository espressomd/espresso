#include "hip/hip_runtime.h"
#include "cuda_wrapper.hpp"

// *******
// This is an internal file of the IMMERSED BOUNDARY implementation
// It should not be included by any main Espresso routines
// Functions to be exported for Espresso are in ibm_main.hpp

#include "config.hpp"

#if defined(VIRTUAL_SITES_INERTIALESS_TRACERS) && defined(CUDA)

#include "cuda_interface.hpp"
#include "cuda_utils.hpp"
#include "grid_based_algorithms/lb_boundaries.hpp"
#include "grid_based_algorithms/lbgpu.cuh"
#include "grid_based_algorithms/lbgpu.hpp"
#include "particle_data.hpp"
#include "virtual_sites/lb_inertialess_tracers.hpp"
#include "virtual_sites/lb_inertialess_tracers_cuda_interface.hpp"

// To avoid include of communication.hpp in cuda file
extern int this_node;

// ****** Kernel functions for internal use ********
__global__ void ResetLBForces_Kernel(LB_node_force_density_gpu node_f,
                                     const LB_parameters_gpu *const paraP);
__global__ void ParticleVelocitiesFromLB_Kernel(
    LB_nodes_gpu n_curr,
    const IBM_CUDA_ParticleDataInput *const particles_input,
    IBM_CUDA_ParticleDataOutput *const particles_output,
    LB_node_force_density_gpu node_f, const float *const lb_boundary_velocity,
    const LB_parameters_gpu *const para);
__global__ void
ForcesIntoFluid_Kernel(const IBM_CUDA_ParticleDataInput *const particle_input,
                       LB_node_force_density_gpu node_f,
                       const LB_parameters_gpu *const paraP);

// ***** Other functions for internal use *****
void InitCUDA_IBM(const int numParticles);

// ***** Our own global variables ********
IBM_CUDA_ParticleDataInput *IBM_ParticleDataInput_device = nullptr;
IBM_CUDA_ParticleDataOutput *IBM_ParticleDataOutput_device = nullptr;
int IBM_numParticlesCache = -1; // To detect a change in particle number which
                                // requires reallocation of memory

// ****** These variables are defined in lbgpu_cuda.cu, but we also want them
// here ****
extern LB_node_force_density_gpu node_f;
extern LB_nodes_gpu *current_nodes;

// ** These variables are static in lbgpu_cuda.cu, so we need to duplicate them
// here They are initialized in ForcesIntoFluid The pointers are on the host,
// but point into device memory
LB_parameters_gpu *para_gpu = nullptr;
float *lb_boundary_velocity_IBM = nullptr;

/****************
   IBM_ResetLBForces_GPU
Calls a kernel to reset the forces on the LB nodes to the external force
*****************/

void IBM_ResetLBForces_GPU() {
  if (this_node == 0) {
    // Setup for kernel call
    int threads_per_block = 64;
    int blocks_per_grid_y = 4;
    int blocks_per_grid_x = (lbpar_gpu.number_of_nodes +
                             threads_per_block * blocks_per_grid_y - 1) /
                            (threads_per_block * blocks_per_grid_y);
    dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

    KERNELCALL(ResetLBForces_Kernel, dim_grid, threads_per_block, node_f,
               para_gpu);
  }
}

/******************
   IBM_ForcesIntoFluid_GPU
Called from integrate_vv to put the forces into the fluid
This must be the first CUDA-IBM function to be called because it also does some
initialization
*******************/

void IBM_ForcesIntoFluid_GPU(ParticleRange particles) {
  // This function does
  // (1) Gather forces from all particles via MPI
  // (2) Copy forces to the GPU
  // (3) interpolate on the LBM grid and spread forces

  const int numParticles =
      gpu_get_global_particle_vars_pointer_host()->number_of_particles;

  // Storage only needed on master and allocated only once at the first time
  // step if ( IBM_ParticleDataInput_host == nullptr && this_node == 0 )
  if (IBM_ParticleDataInput_host == NULL ||
      numParticles != IBM_numParticlesCache)
    InitCUDA_IBM(numParticles);

  // We gather particle positions and forces from all nodes
  IBM_cuda_mpi_get_particles(particles);

  // ***** GPU stuff only on master *****
  if (this_node == 0 && numParticles > 0) {

    // Copy data to device
    cuda_safe_mem(hipMemcpy(IBM_ParticleDataInput_device,
                             IBM_ParticleDataInput_host,
                             numParticles * sizeof(IBM_CUDA_ParticleDataInput),
                             hipMemcpyHostToDevice));

    // Kernel call for spreading the forces on the LB grid
    int threads_per_block_particles = 64;
    int blocks_per_grid_particles_y = 4;
    int blocks_per_grid_particles_x =
        (lbpar_gpu.number_of_particles +
         threads_per_block_particles * blocks_per_grid_particles_y - 1) /
        (threads_per_block_particles * blocks_per_grid_particles_y);
    dim3 dim_grid_particles =
        make_uint3(blocks_per_grid_particles_x, blocks_per_grid_particles_y, 1);

    KERNELCALL(ForcesIntoFluid_Kernel, dim_grid_particles,
               threads_per_block_particles, IBM_ParticleDataInput_device,
               node_f, para_gpu);
  }
}

/***************
   InitCUDA_IBM
***************/

void InitCUDA_IBM(const int numParticles) {

  if (this_node == 0) // GPU only on master
  {

    // Check if we have to delete
    if (IBM_ParticleDataInput_host != NULL) {
      delete[] IBM_ParticleDataInput_host;
      delete[] IBM_ParticleDataOutput_host;
      cuda_safe_mem(hipFree(IBM_ParticleDataInput_device));
      cuda_safe_mem(hipFree(IBM_ParticleDataOutput_device));
      cuda_safe_mem(hipFree(lb_boundary_velocity_IBM));
    }

    // Back and forth communication of positions and velocities
    IBM_ParticleDataInput_host = new IBM_CUDA_ParticleDataInput[numParticles];
    cuda_safe_mem(
        hipMalloc((void **)&IBM_ParticleDataInput_device,
                   numParticles * sizeof(IBM_CUDA_ParticleDataInput)));
    cuda_safe_mem(
        hipMalloc((void **)&IBM_ParticleDataOutput_device,
                   numParticles * sizeof(IBM_CUDA_ParticleDataOutput)));
    IBM_ParticleDataOutput_host = new IBM_CUDA_ParticleDataOutput[numParticles];

    // Use LB parameters
    lb_get_para_pointer(&para_gpu);

    // Copy boundary velocities to the GPU
    // First put them into correct format
#ifdef LB_BOUNDARIES_GPU
    float *host_lb_boundary_velocity =
        new float[3 * (LBBoundaries::lbboundaries.size() + 1)];

    for (int n = 0; n < LBBoundaries::lbboundaries.size(); n++) {
      host_lb_boundary_velocity[3 * n + 0] =
          LBBoundaries::lbboundaries[n]->velocity()[0];
      host_lb_boundary_velocity[3 * n + 1] =
          LBBoundaries::lbboundaries[n]->velocity()[1];
      host_lb_boundary_velocity[3 * n + 2] =
          LBBoundaries::lbboundaries[n]->velocity()[2];
    }

    host_lb_boundary_velocity[3 * LBBoundaries::lbboundaries.size() + 0] = 0.0f;
    host_lb_boundary_velocity[3 * LBBoundaries::lbboundaries.size() + 1] = 0.0f;
    host_lb_boundary_velocity[3 * LBBoundaries::lbboundaries.size() + 2] = 0.0f;

    cuda_safe_mem(
        hipMalloc((void **)&lb_boundary_velocity_IBM,
                   3 * LBBoundaries::lbboundaries.size() * sizeof(float)));
    cuda_safe_mem(
        hipMemcpy(lb_boundary_velocity_IBM, host_lb_boundary_velocity,
                   3 * LBBoundaries::lbboundaries.size() * sizeof(float),
                   hipMemcpyHostToDevice));

    delete[] host_lb_boundary_velocity;
#endif

    IBM_numParticlesCache = numParticles;
  }
}

/**************
   Calc_m_from_n_IBM
This is our own version of the calc_m_from_n function in lbgpu_cuda.cu
It does exactly the same, but calculates only the first four modes
***************/

__device__ void Calc_m_from_n_IBM(const LB_nodes_gpu n_a,
                                  const unsigned int index, float *mode,
                                  const LB_parameters_gpu *const paraP) {
  const LB_parameters_gpu &para = *paraP;
  // mass mode
  mode[0] = n_a.vd[0 * para.number_of_nodes + index] +
            n_a.vd[1 * para.number_of_nodes + index] +
            n_a.vd[2 * para.number_of_nodes + index] +
            n_a.vd[3 * para.number_of_nodes + index] +
            n_a.vd[4 * para.number_of_nodes + index] +
            n_a.vd[5 * para.number_of_nodes + index] +
            n_a.vd[6 * para.number_of_nodes + index] +
            n_a.vd[7 * para.number_of_nodes + index] +
            n_a.vd[8 * para.number_of_nodes + index] +
            n_a.vd[9 * para.number_of_nodes + index] +
            n_a.vd[10 * para.number_of_nodes + index] +
            n_a.vd[11 * para.number_of_nodes + index] +
            n_a.vd[12 * para.number_of_nodes + index] +
            n_a.vd[13 * para.number_of_nodes + index] +
            n_a.vd[14 * para.number_of_nodes + index] +
            n_a.vd[15 * para.number_of_nodes + index] +
            n_a.vd[16 * para.number_of_nodes + index] +
            n_a.vd[17 * para.number_of_nodes + index] +
            n_a.vd[18 * para.number_of_nodes + index];

  // momentum modes

  mode[1] = (n_a.vd[1 * para.number_of_nodes + index] -
             n_a.vd[2 * para.number_of_nodes + index]) +
            (n_a.vd[7 * para.number_of_nodes + index] -
             n_a.vd[8 * para.number_of_nodes + index]) +
            (n_a.vd[9 * para.number_of_nodes + index] -
             n_a.vd[10 * para.number_of_nodes + index]) +
            (n_a.vd[11 * para.number_of_nodes + index] -
             n_a.vd[12 * para.number_of_nodes + index]) +
            (n_a.vd[13 * para.number_of_nodes + index] -
             n_a.vd[14 * para.number_of_nodes + index]);

  mode[2] = (n_a.vd[3 * para.number_of_nodes + index] -
             n_a.vd[4 * para.number_of_nodes + index]) +
            (n_a.vd[7 * para.number_of_nodes + index] -
             n_a.vd[8 * para.number_of_nodes + index]) -
            (n_a.vd[9 * para.number_of_nodes + index] -
             n_a.vd[10 * para.number_of_nodes + index]) +
            (n_a.vd[15 * para.number_of_nodes + index] -
             n_a.vd[16 * para.number_of_nodes + index]) +
            (n_a.vd[17 * para.number_of_nodes + index] -
             n_a.vd[18 * para.number_of_nodes + index]);

  mode[3] = (n_a.vd[5 * para.number_of_nodes + index] -
             n_a.vd[6 * para.number_of_nodes + index]) +
            (n_a.vd[11 * para.number_of_nodes + index] -
             n_a.vd[12 * para.number_of_nodes + index]) -
            (n_a.vd[13 * para.number_of_nodes + index] -
             n_a.vd[14 * para.number_of_nodes + index]) +
            (n_a.vd[15 * para.number_of_nodes + index] -
             n_a.vd[16 * para.number_of_nodes + index]) -
            (n_a.vd[17 * para.number_of_nodes + index] -
             n_a.vd[18 * para.number_of_nodes + index]);
}

/**************
   ParticleVelocitiesFromLB_GPU
Calls a kernel function to interpolate the velocity at each IBM particle's
position Store velocity in the particle data structure
**************/

void ParticleVelocitiesFromLB_GPU(ParticleRange particles) {
  // This function performs three steps:
  // (1) interpolate velocities on GPU
  // (2) transfer velocities back to CPU
  // (3) spread velocities to local cells via MPI

  const int numParticles =
      gpu_get_global_particle_vars_pointer_host()->number_of_particles;

  // **** GPU stuff only on master ****
  if (this_node == 0 && numParticles > 0) {
    // Kernel call
    int threads_per_block_particles = 64;
    int blocks_per_grid_particles_y = 4;
    int blocks_per_grid_particles_x =
        (lbpar_gpu.number_of_particles +
         threads_per_block_particles * blocks_per_grid_particles_y - 1) /
        (threads_per_block_particles * blocks_per_grid_particles_y);
    dim3 dim_grid_particles =
        make_uint3(blocks_per_grid_particles_x, blocks_per_grid_particles_y, 1);
    KERNELCALL(ParticleVelocitiesFromLB_Kernel, dim_grid_particles,
               threads_per_block_particles, *current_nodes,
               IBM_ParticleDataInput_device, IBM_ParticleDataOutput_device,
               node_f, lb_boundary_velocity_IBM, para_gpu);

    // Copy velocities from device to host
    cuda_safe_mem(hipMemcpy(IBM_ParticleDataOutput_host,
                             IBM_ParticleDataOutput_device,
                             numParticles * sizeof(IBM_CUDA_ParticleDataOutput),
                             hipMemcpyDeviceToHost));
  }

  // ***** Back to all nodes ****
  // Spread using MPI
  IBM_cuda_mpi_send_velocities(particles);
}

/***************
   ForcesIntoFluid_Kernel
****************/

__global__ void
ForcesIntoFluid_Kernel(const IBM_CUDA_ParticleDataInput *const particle_input,
                       LB_node_force_density_gpu node_f,
                       const LB_parameters_gpu *const paraP) {
  const unsigned int particleIndex = blockIdx.y * gridDim.x * blockDim.x +
                                     blockDim.x * blockIdx.x + threadIdx.x;
  const LB_parameters_gpu &para = *paraP;

  if (particleIndex < para.number_of_particles &&
      particle_input[particleIndex].is_virtual) {

    //    const float factor = powf( para.agrid,2)*para.tau*para.tau; --> Old
    //    version. Worked, but not when agrid != 1
    // MD to LB units: mass is not affected, length are scaled by agrid, times
    // by para.tau
    const float factor = 1 / para.agrid * para.tau * para.tau;
    const float particleForce[3] = {particle_input[particleIndex].f[0] * factor,
                                    particle_input[particleIndex].f[1] * factor,
                                    particle_input[particleIndex].f[2] *
                                        factor};
    const float pos[3] = {particle_input[particleIndex].pos[0],
                          particle_input[particleIndex].pos[1],
                          particle_input[particleIndex].pos[2]};

    // First part is the same as for interpolation --> merge into a single
    // function
    float temp_delta[6];
    float delta[8];
    int my_left[3];
    int node_index[8];
    for (int i = 0; i < 3; ++i) {
      const float scaledpos = pos[i] / para.agrid - 0.5f;
      my_left[i] = (int)(floorf(scaledpos));
      temp_delta[3 + i] = scaledpos - my_left[i];
      temp_delta[i] = 1.f - temp_delta[3 + i];
    }

    delta[0] = temp_delta[0] * temp_delta[1] * temp_delta[2];
    delta[1] = temp_delta[3] * temp_delta[1] * temp_delta[2];
    delta[2] = temp_delta[0] * temp_delta[4] * temp_delta[2];
    delta[3] = temp_delta[3] * temp_delta[4] * temp_delta[2];
    delta[4] = temp_delta[0] * temp_delta[1] * temp_delta[5];
    delta[5] = temp_delta[3] * temp_delta[1] * temp_delta[5];
    delta[6] = temp_delta[0] * temp_delta[4] * temp_delta[5];
    delta[7] = temp_delta[3] * temp_delta[4] * temp_delta[5];

    // modulo for negative numbers is strange at best, shift to make sure we are
    // positive
    const int x = my_left[0] + para.dim_x;
    const int y = my_left[1] + para.dim_y;
    const int z = my_left[2] + para.dim_z;

    node_index[0] = x % para.dim_x + para.dim_x * (y % para.dim_y) +
                    para.dim_x * para.dim_y * (z % para.dim_z);
    node_index[1] = (x + 1) % para.dim_x + para.dim_x * (y % para.dim_y) +
                    para.dim_x * para.dim_y * (z % para.dim_z);
    node_index[2] = x % para.dim_x + para.dim_x * ((y + 1) % para.dim_y) +
                    para.dim_x * para.dim_y * (z % para.dim_z);
    node_index[3] = (x + 1) % para.dim_x + para.dim_x * ((y + 1) % para.dim_y) +
                    para.dim_x * para.dim_y * (z % para.dim_z);
    node_index[4] = x % para.dim_x + para.dim_x * (y % para.dim_y) +
                    para.dim_x * para.dim_y * ((z + 1) % para.dim_z);
    node_index[5] = (x + 1) % para.dim_x + para.dim_x * (y % para.dim_y) +
                    para.dim_x * para.dim_y * ((z + 1) % para.dim_z);
    node_index[6] = x % para.dim_x + para.dim_x * ((y + 1) % para.dim_y) +
                    para.dim_x * para.dim_y * ((z + 1) % para.dim_z);
    node_index[7] = (x + 1) % para.dim_x + para.dim_x * ((y + 1) % para.dim_y) +
                    para.dim_x * para.dim_y * ((z + 1) % para.dim_z);

    for (int i = 0; i < 8; ++i) {

      // Atomic add is essential because this runs in parallel!
      atomicAdd(
          &(node_f.force_density[0 * para.number_of_nodes + node_index[i]]),
          (particleForce[0] * delta[i]));
      atomicAdd(
          &(node_f.force_density[1 * para.number_of_nodes + node_index[i]]),
          (particleForce[1] * delta[i]));
      atomicAdd(
          &(node_f.force_density[2 * para.number_of_nodes + node_index[i]]),
          (particleForce[2] * delta[i]));
    }
  }
}

/**************
   ParticleVelocitiesFromLB_Kernel
**************/

__global__ void ParticleVelocitiesFromLB_Kernel(
    LB_nodes_gpu n_curr,
    const IBM_CUDA_ParticleDataInput *const particles_input,
    IBM_CUDA_ParticleDataOutput *const particles_output,
    LB_node_force_density_gpu node_f, const float *const lb_boundary_velocity,
    const LB_parameters_gpu *const paraP) {

  const unsigned int particleIndex = blockIdx.y * gridDim.x * blockDim.x +
                                     blockDim.x * blockIdx.x + threadIdx.x;

  const LB_parameters_gpu &para = *paraP;

  if (particleIndex < para.number_of_particles &&
      particles_input[particleIndex].is_virtual) {

    // Get position
    float pos[3] = {particles_input[particleIndex].pos[0],
                    particles_input[particleIndex].pos[1],
                    particles_input[particleIndex].pos[2]};
    float v[3] = {0};

    // ***** This part is copied from get_interpolated_velocity
    // ***** + we add the force + we consider boundaries

    float temp_delta[6];
    float delta[8];
    int my_left[3];
    int node_index[8];
    float mode[4];
#pragma unroll
    for (int i = 0; i < 3; ++i) {
      const float scaledpos = pos[i] / para.agrid - 0.5f;
      my_left[i] = (int)(floorf(scaledpos));
      temp_delta[3 + i] = scaledpos - my_left[i];
      temp_delta[i] = 1.f - temp_delta[3 + i];
    }

    delta[0] = temp_delta[0] * temp_delta[1] * temp_delta[2];
    delta[1] = temp_delta[3] * temp_delta[1] * temp_delta[2];
    delta[2] = temp_delta[0] * temp_delta[4] * temp_delta[2];
    delta[3] = temp_delta[3] * temp_delta[4] * temp_delta[2];
    delta[4] = temp_delta[0] * temp_delta[1] * temp_delta[5];
    delta[5] = temp_delta[3] * temp_delta[1] * temp_delta[5];
    delta[6] = temp_delta[0] * temp_delta[4] * temp_delta[5];
    delta[7] = temp_delta[3] * temp_delta[4] * temp_delta[5];

    // modulo for negative numbers is strange at best, shift to make sure we are
    // positive
    int x = my_left[0] + para.dim_x;
    int y = my_left[1] + para.dim_y;
    int z = my_left[2] + para.dim_z;

    node_index[0] = x % para.dim_x + para.dim_x * (y % para.dim_y) +
                    para.dim_x * para.dim_y * (z % para.dim_z);
    node_index[1] = (x + 1) % para.dim_x + para.dim_x * (y % para.dim_y) +
                    para.dim_x * para.dim_y * (z % para.dim_z);
    node_index[2] = x % para.dim_x + para.dim_x * ((y + 1) % para.dim_y) +
                    para.dim_x * para.dim_y * (z % para.dim_z);
    node_index[3] = (x + 1) % para.dim_x + para.dim_x * ((y + 1) % para.dim_y) +
                    para.dim_x * para.dim_y * (z % para.dim_z);
    node_index[4] = x % para.dim_x + para.dim_x * (y % para.dim_y) +
                    para.dim_x * para.dim_y * ((z + 1) % para.dim_z);
    node_index[5] = (x + 1) % para.dim_x + para.dim_x * (y % para.dim_y) +
                    para.dim_x * para.dim_y * ((z + 1) % para.dim_z);
    node_index[6] = x % para.dim_x + para.dim_x * ((y + 1) % para.dim_y) +
                    para.dim_x * para.dim_y * ((z + 1) % para.dim_z);
    node_index[7] = (x + 1) % para.dim_x + para.dim_x * ((y + 1) % para.dim_y) +
                    para.dim_x * para.dim_y * ((z + 1) % para.dim_z);

    for (int i = 0; i < 8; ++i) {
      double local_rho;
      double local_j[3];
#ifdef LB_BOUNDARIES_GPU
      if (n_curr.boundary[node_index[i]]) {
        // Boundary node
        const int boundary_index = n_curr.boundary[node_index[i]];

        // lb_boundary_velocity is given in MD units --> convert to LB and
        // reconvert back at the end of this function
        local_rho = para.rho;
        local_j[0] =
            para.rho * lb_boundary_velocity[3 * (boundary_index - 1) + 0];
        local_j[1] =
            para.rho * lb_boundary_velocity[3 * (boundary_index - 1) + 1];
        local_j[2] =
            para.rho * lb_boundary_velocity[3 * (boundary_index - 1) + 2];

      } else
#endif
      {
        Calc_m_from_n_IBM(n_curr, node_index[i], mode, paraP);
        local_rho = para.rho + mode[0];

        // Add the +f/2 contribution!!
        local_j[0] =
            mode[1] +
            node_f.force_density_buf[0 * para.number_of_nodes + node_index[i]] /
                2.f;
        local_j[1] =
            mode[2] +
            node_f.force_density_buf[1 * para.number_of_nodes + node_index[i]] /
                2.f;
        local_j[2] =
            mode[3] +
            node_f.force_density_buf[2 * para.number_of_nodes + node_index[i]] /
                2.f;
      }

      // Interpolate velocity
      v[0] += delta[i] * local_j[0] / (local_rho);
      v[1] += delta[i] * local_j[1] / (local_rho);
      v[2] += delta[i] * local_j[2] / (local_rho);
    }

    // Rescale and store output
    particles_output[particleIndex].v[0] = v[0] * para.agrid / para.tau;
    particles_output[particleIndex].v[1] = v[1] * para.agrid / para.tau;
    particles_output[particleIndex].v[2] = v[2] * para.agrid / para.tau;
  }
}

/****************
   ResetLBForces_Kernel
*****************/

__global__ void ResetLBForces_Kernel(LB_node_force_density_gpu node_f,
                                     const LB_parameters_gpu *const paraP) {

  const size_t index = blockIdx.y * gridDim.x * blockDim.x +
                       blockDim.x * blockIdx.x + threadIdx.x;
  const LB_parameters_gpu &para = *paraP;

  if (index < para.number_of_nodes) {
    const float force_factor = powf(para.agrid, 2) * para.tau * para.tau;
    if (para.external_force_density) {
      node_f.force_density[0 * para.number_of_nodes + index] =
          para.ext_force_density[0] * force_factor;
      node_f.force_density[1 * para.number_of_nodes + index] =
          para.ext_force_density[1] * force_factor;
      node_f.force_density[2 * para.number_of_nodes + index] =
          para.ext_force_density[2] * force_factor;
    } else {
      node_f.force_density[0 * para.number_of_nodes + index] = 0.0f;
      node_f.force_density[1 * para.number_of_nodes + index] = 0.0f;
      node_f.force_density[2 * para.number_of_nodes + index] = 0.0f;
    }
  }
}

#endif
