#include "hip/hip_runtime.h"
/* 
   Copyright (C) 2010,2011,2012 The ESPResSo project

   This file is part of ESPResSo.
  
   ESPResSo is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.
   
   ESPResSo is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.
   
   You should have received a copy of the GNU General Public License
   along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "config.hpp"
#ifdef CUDA /* Terminates at end of file */



#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>



#include <stdio.h>
#include "lb-boundaries.hpp"
#include "electrokinetics.hpp"
#include "cuda_interface.hpp"
#include "cuda_utils.hpp"
#include "lbgpu.hpp"
#include "constraint.hpp"



#ifdef ELECTROKINETICS /* Terminates at end of file */

  /* TODO: get rid of this code duplication with lb-boundaries.h by solving the
           cuda-mpi incompatibility */

#define LATTICE_OFF      0
#define LATTICE_LB_CPU   1
#define LATTICE_LB_GPU   2
extern int lattice_switch;
extern int ek_initialized;
extern EK_parameters* lb_ek_parameters_gpu;

// Used to limit register use for the pressure calculation
#define EK_LINK_U00_pressure 0
#define EK_LINK_0U0_pressure 1
#define EK_LINK_00U_pressure 2
#define EK_LINK_D00_pressure 3
#define EK_LINK_0D0_pressure 4
#define EK_LINK_00D_pressure 5
     
#ifdef EK_BOUNDARIES
  extern int n_lb_boundaries;
  extern LB_Boundary *lb_boundaries;

  void lb_init_boundaries();
#endif
  /* end of code duplication */

  extern hipStream_t stream[1];
  extern hipError_t _err;

  #define PI_FLOAT 3.14159265358979323846f

  EK_parameters ek_parameters = { -1.0, -1.0, -1.0,
                                     0,    0,    0,
                                     0,
                                  -1.0, -1.0,  0.0,
                                   0.0,  0.0, -1.0,
                                  -1.0,
                                     0,    0,
                                  -1.0, -1.0, -1.0, 
                                  {0.0,  0.0, 0.0},
                                  { -1,   -1,  -1},
                                  -1.0, -1.0, -1.0,
                                  -1.0, -1.0, -1.0,
                                  -1.0, -1.0, -1.0,
                                  -1.0
                                };
                                
  static __device__ __constant__ EK_parameters ek_parameters_gpu;
  static __device__ float ek_accelerated_frame_boundary_force [3] = { 0.0f, 0.0f, 0.0f };
  static float* ek_accelerated_frame_boundary_velocity;
  EK_parameters *ek_parameters_gpu_pointer;
  LB_parameters_gpu *ek_lbparameters_gpu;
  CUDA_particle_data *particle_data_gpu;
  float *ek_lb_boundary_force;
  char *ek_node_is_catalyst;
  unsigned int old_number_of_species = 0;
  unsigned int old_number_of_boundaries = 0;

  hipfftHandle plan_fft;
  hipfftHandle plan_ifft;
  
  bool initialized = false;
  
  extern LB_parameters_gpu lbpar_gpu;
  extern LB_node_force_gpu node_f;
  extern LB_nodes_gpu *current_nodes;
  extern EK_parameters *lb_ek_parameters;
  
  LB_rho_v_gpu *ek_lb_device_values;



__device__ inline void atomicadd( float* address,
                                  float value
                                ) {

#if !defined __CUDA_ARCH__ || __CUDA_ARCH__ >= 200 // for Fermi, atomicAdd supports floats
  atomicAdd(address, value);
#elif __CUDA_ARCH__ >= 110

  #warning Using slower atomicAdd emulation
  
  //float-atomic-add from 
  //[url="http://forums.nvidia.com/index.php?showtopic=158039&view=findpost&p=991561"]
  
  float old = value;
  while( ( old = atomicExch( address, atomicExch( address, 0.0f ) + old ) ) != 0.0f );
  
#else
  #error CUDA compute capability 1.1 or higher required
#endif
}


__device__ unsigned int ek_getThreadIndex() {

  return blockIdx.y * gridDim.x * blockDim.x +
         blockDim.x * blockIdx.x +
         threadIdx.x;
}


__device__ void rhoindex_linear2cartesian( unsigned int index,
                                           unsigned int * coord
                                         ) {

  coord[0]  = index % ek_parameters_gpu.dim_x;
  index    /= ek_parameters_gpu.dim_x;
  coord[1]  = index % ek_parameters_gpu.dim_y;
  coord[2]  = index / ek_parameters_gpu.dim_y;
}


__device__ unsigned int rhoindex_cartesian2linear( unsigned int x,
                                                   unsigned int y,
                                                   unsigned int z
                                                 ) {

  return z * ek_parameters_gpu.dim_y * ek_parameters_gpu.dim_x +
         y * ek_parameters_gpu.dim_x +
         x;
}


__device__ void jindex_linear2cartesian( unsigned int index,
                                         unsigned int * coord,
                                         unsigned int * c
                                       ) {

  coord[0]  = index % ek_parameters_gpu.dim_x;
  index    /= ek_parameters_gpu.dim_x;
  coord[1]  = index % ek_parameters_gpu.dim_y;
  index    /= ek_parameters_gpu.dim_y;
  coord[2]  = index % ek_parameters_gpu.dim_z;
  *c        = index / ek_parameters_gpu.dim_z;
}


__device__ unsigned int jindex_cartesian2linear( unsigned int x,
                                                 unsigned int y,
                                                 unsigned int z,
                                                 unsigned int c
                                               ) {
                                                 
  return c * ek_parameters_gpu.number_of_nodes + 
         z * ek_parameters_gpu.dim_y * ek_parameters_gpu.dim_x +
         y * ek_parameters_gpu.dim_x +
         x;
}


//TODO fluxindex fastest running might improve caching
__device__ unsigned int jindex_getByRhoLinear( unsigned int rho_index,
                                               unsigned int c
                                             ) {
                                               
  return c * ek_parameters_gpu.number_of_nodes +
         rho_index;
}


__device__ void ek_displacement( float * dx,
                                 LB_nodes_gpu n,
                                 unsigned int node_index,
                                 LB_parameters_gpu * ek_lbparameters_gpu
                               ) {
                                 
  float rho = ek_lbparameters_gpu->rho[0] *
              ek_lbparameters_gpu->agrid *
              ek_lbparameters_gpu->agrid *
              ek_lbparameters_gpu->agrid;

  float mode [19];

  for ( int i = 0; i < 19; i++ )
  {
    mode[i] = n.vd[  i * ek_lbparameters_gpu->number_of_nodes + node_index ];
  }
  
  rho += mode[  0 ] +
         mode[  1 ] +
         mode[  2 ] +
         mode[  3 ] +
         mode[  4 ] +
         mode[  5 ] +
         mode[  6 ] +
         mode[  7 ] +
         mode[  8 ] +
         mode[  9 ] +
         mode[ 10 ] +
         mode[ 11 ] +
         mode[ 12 ] +
         mode[ 13 ] +
         mode[ 14 ] +
         mode[ 15 ] +
         mode[ 16 ] +
         mode[ 17 ] +
         mode[ 18 ];

  dx[0] = ( mode[  1 ] - mode[  2 ] ) +
          ( mode[  7 ] - mode[  8 ] ) +
          ( mode[  9 ] - mode[ 10 ] ) +
          ( mode[ 11 ] - mode[ 12 ] ) +
          ( mode[ 13 ] - mode[ 14 ] );
                 
  dx[1] = ( mode[  3 ] - mode[  4 ] ) +
          ( mode[  7 ] - mode[  8 ] ) -
          ( mode[  9 ] - mode[ 10 ] ) +
          ( mode[ 15 ] - mode[ 16 ] ) +
          ( mode[ 17 ] - mode[ 18 ] );
          
  dx[2] = ( mode[  5 ] - mode[  6 ] ) +
          ( mode[ 11 ] - mode[ 12 ] ) -
          ( mode[ 13 ] - mode[ 14 ] ) +
          ( mode[ 15 ] - mode[ 16 ] ) -
          ( mode[ 17 ] - mode[ 18 ] );

  // Velocity requires half the force in the previous time step

  dx[0] += 0.5f * ek_parameters_gpu.lb_force_previous[ node_index ];
  dx[1] += 0.5f * ek_parameters_gpu.lb_force_previous[ ek_parameters_gpu.number_of_nodes + node_index ];
  dx[2] += 0.5f * ek_parameters_gpu.lb_force_previous[ 2 * ek_parameters_gpu.number_of_nodes + node_index ];

  dx[0] *= 1.0f / rho;
  dx[1] *= 1.0f / rho;
  dx[2] *= 1.0f / rho;
}

#ifdef EK_REACTION
__global__ void ek_pressure(
                             LB_nodes_gpu n_a,
                             LB_parameters_gpu *ek_lbparameters_gpu,
                             LB_rho_v_gpu *d_v
                           )
{
  unsigned int index = ek_getThreadIndex ();

  if( index < ek_parameters_gpu.number_of_nodes )
  {  
    ek_parameters_gpu.pressure[ index ] = 0.0f;
 
    // Add the ideal-gas contribution f from the EK
    // species, which is given by n_i * k. In MD units
    // the proper expression is n_i * T / ag^2, where 
    // there is a 1/ag^3 factor coming from converting the
    // internal EK particle number back to a density,
    // and an ag factor that is required to get the 
    // proper pressure difference

    for ( int i = 0; i < ek_parameters_gpu.number_of_species; i++ )
    {
      ek_parameters_gpu.pressure[ index ] += ek_parameters_gpu.rho[ i ][ index ] *
                                             ek_parameters_gpu.T /
                                             powf(ek_parameters_gpu.agrid, 2);
    }

    // Set pressure to zero inside boundary

    ek_parameters_gpu.pressure[ index ] *= (n_a.boundary[index] == 0);
  }
}

__global__ void ek_add_ideal_pressure_to_lb_force(
                                                   LB_nodes_gpu lb_node,
                                                   LB_node_force_gpu node_f,
                                                   LB_parameters_gpu *ek_lbparameters_gpu
                                                 ) 
{
  unsigned int coord[3];
  unsigned int neighborindex[6];
  unsigned int index = ek_getThreadIndex ();

  if(index < ek_parameters_gpu.number_of_nodes)
  {
    float pressure_gradient;
    
    rhoindex_linear2cartesian( index, coord );

    // Calculate the indices of the neighbours to which
    // the force is to be applied
       
    neighborindex[EK_LINK_U00_pressure] =
      rhoindex_cartesian2linear(
        (coord[0] + 1) % ek_parameters_gpu.dim_x,
         coord[1],
         coord[2]
      );
      
    neighborindex[EK_LINK_0U0_pressure] =
      rhoindex_cartesian2linear(
         coord[0],
        (coord[1] + 1) % ek_parameters_gpu.dim_y,
         coord[2]
      );
      
    neighborindex[EK_LINK_00U_pressure] =
      rhoindex_cartesian2linear(
         coord[0],
         coord[1],
        (coord[2] + 1) % ek_parameters_gpu.dim_z
      );

    neighborindex[EK_LINK_D00_pressure] =
      rhoindex_cartesian2linear(
        (coord[0] - 1 + ek_parameters_gpu.dim_x) % ek_parameters_gpu.dim_x,
         coord[1],
         coord[2]
      );
      
    neighborindex[EK_LINK_0D0_pressure] =
      rhoindex_cartesian2linear(
         coord[0],
        (coord[1] - 1 + ek_parameters_gpu.dim_y) % ek_parameters_gpu.dim_y,
         coord[2]
      );
      
    neighborindex[EK_LINK_00D_pressure] =
      rhoindex_cartesian2linear(
         coord[0],
         coord[1],
        (coord[2] - 1 + ek_parameters_gpu.dim_z) % ek_parameters_gpu.dim_z
      );

    // Force in x direction (multiplicative factor
    // comes from converting MD force into LB force)

    pressure_gradient = (   ek_parameters_gpu.pressure[ neighborindex[EK_LINK_D00_pressure] ]
                          - ek_parameters_gpu.pressure[ neighborindex[EK_LINK_U00_pressure] ] )/
                        ( 2.0f * ek_parameters_gpu.agrid );

    pressure_gradient *= powf(ek_parameters_gpu.agrid, 3) *
                         ek_parameters_gpu.time_step *
                         ek_parameters_gpu.time_step;

    pressure_gradient *= ( (   lb_node.boundary[ neighborindex[EK_LINK_U00_pressure] ]
                             + lb_node.boundary[ index ]
                             + lb_node.boundary[ neighborindex[EK_LINK_D00_pressure] ] ) == 0 );

    atomicadd( &node_f.force[index], pressure_gradient );
    
    // Force in y direction

    pressure_gradient = (   ek_parameters_gpu.pressure[ neighborindex[EK_LINK_0D0_pressure] ]
                          - ek_parameters_gpu.pressure[ neighborindex[EK_LINK_0U0_pressure] ] )/
                        ( 2.0f * ek_parameters_gpu.agrid );

    pressure_gradient *= powf(ek_parameters_gpu.agrid, 3) *
                         ek_parameters_gpu.time_step *
                         ek_parameters_gpu.time_step;

    pressure_gradient *= ( (   lb_node.boundary[ neighborindex[EK_LINK_0U0_pressure] ]
                             + lb_node.boundary[ index ]
                             + lb_node.boundary[ neighborindex[EK_LINK_0D0_pressure] ] ) == 0 );

    atomicadd( &node_f.force[ek_parameters_gpu.number_of_nodes + index], pressure_gradient );
              
    // Force in z direction

    pressure_gradient = (   ek_parameters_gpu.pressure[ neighborindex[EK_LINK_00D_pressure] ]
                          - ek_parameters_gpu.pressure[ neighborindex[EK_LINK_00U_pressure] ] )/
                        ( 2.0f * ek_parameters_gpu.agrid );

    pressure_gradient *= powf(ek_parameters_gpu.agrid, 3) *
                         ek_parameters_gpu.time_step *
                         ek_parameters_gpu.time_step;

    pressure_gradient *= ( (   lb_node.boundary[ neighborindex[EK_LINK_00U_pressure] ]
                             + lb_node.boundary[ index ]
                             + lb_node.boundary[ neighborindex[EK_LINK_00D_pressure] ] ) == 0 );

    atomicadd( &node_f.force[2*ek_parameters_gpu.number_of_nodes + index], pressure_gradient );
  }
}
#endif

__global__ void ek_accelerated_frame_transformation( LB_node_force_gpu node_f )
{
                                       
  unsigned int index = ek_getThreadIndex ();

  if(index < ek_parameters_gpu.number_of_nodes)
  {
    //face in x

    atomicadd( &node_f.force[index],
               ek_accelerated_frame_boundary_force[0] );
    
    //face in y

    atomicadd( &node_f.force[ek_parameters_gpu.number_of_nodes + index],
               ek_accelerated_frame_boundary_force[1] );
              
    //face in z

    atomicadd( &node_f.force[2*ek_parameters_gpu.number_of_nodes + index],
               ek_accelerated_frame_boundary_force[2] );
  }
}

__global__ void ek_calculate_quantities( unsigned int species_index,
                                         LB_nodes_gpu lb_node,
                                         LB_node_force_gpu node_f,
                                         LB_parameters_gpu *ek_lbparameters_gpu,
                                         LB_rho_v_gpu *d_v
                                       ) {
                                       
  unsigned int index = ek_getThreadIndex ();

  if(index < ek_parameters_gpu.number_of_nodes)
  {
  
    unsigned int coord[3];
    unsigned int neighborindex[9];
    float dx[3];
    int di[3];
    int node;
    float flux, force;
    float boltzmannfactor_local, boltzmannfactor_neighbor;
    
    rhoindex_linear2cartesian( index, coord );
    
    /* Calculate the diffusive fluxes between this node and its neighbors. Only 
       the 9 fluxes along the directions of the LB velocities c_i with i odd are
       stored with a node to avoid redundencies. */
       
    neighborindex[EK_LINK_U00] =
      rhoindex_cartesian2linear(
        (coord[0] + 1) % ek_parameters_gpu.dim_x,
         coord[1],
         coord[2]
      );
      
    neighborindex[EK_LINK_0U0] =
      rhoindex_cartesian2linear(
         coord[0],
        (coord[1] + 1) % ek_parameters_gpu.dim_y,
         coord[2]
      );
      
    neighborindex[EK_LINK_00U] =
      rhoindex_cartesian2linear(
         coord[0],
         coord[1],
        (coord[2] + 1) % ek_parameters_gpu.dim_z
      );
      
    neighborindex[EK_LINK_UU0] =
      rhoindex_cartesian2linear(
        (coord[0] + 1) % ek_parameters_gpu.dim_x,
        (coord[1] + 1) % ek_parameters_gpu.dim_y,
         coord[2]
      );
      
    neighborindex[EK_LINK_UD0] =
      rhoindex_cartesian2linear(
        (coord[0] + 1                          ) % ek_parameters_gpu.dim_x,
        (coord[1] - 1 + ek_parameters_gpu.dim_y) % ek_parameters_gpu.dim_y,
         coord[2]
      );
      
    neighborindex[EK_LINK_U0U] =
      rhoindex_cartesian2linear(
        (coord[0] + 1) % ek_parameters_gpu.dim_x,
         coord[1],
        (coord[2] + 1) % ek_parameters_gpu.dim_z
      );
      
    neighborindex[EK_LINK_U0D] =
      rhoindex_cartesian2linear(
        (coord[0] + 1                          ) % ek_parameters_gpu.dim_x,
         coord[1],
        (coord[2] - 1 + ek_parameters_gpu.dim_z) % ek_parameters_gpu.dim_z
      );
      
    neighborindex[EK_LINK_0UU] =
      rhoindex_cartesian2linear(
         coord[0],
        (coord[1] + 1) % ek_parameters_gpu.dim_y,
        (coord[2] + 1) % ek_parameters_gpu.dim_z
      );
      
    neighborindex[EK_LINK_0UD] =
      rhoindex_cartesian2linear(
         coord[0],
        (coord[1] + 1                          ) % ek_parameters_gpu.dim_y,
        (coord[2] - 1 + ek_parameters_gpu.dim_z) % ek_parameters_gpu.dim_z
      );
    
    
    /* diffusive contribution to flux and LB force*/
    
    /* TODO: take out all of the boltzmann factor based calculations and replace
             them with direct gradient evaluations. */
             
    boltzmannfactor_local = 
      exp( 1.0f / ek_parameters_gpu.T *
           ek_parameters_gpu.valency[species_index] *
           ((hipfftReal*) ek_parameters_gpu.charge_potential)[index]
         );
 
    //face in x
    boltzmannfactor_neighbor =
      exp( 1.0f / ek_parameters_gpu.T *
           ( ek_parameters_gpu.valency[species_index] *
             ((hipfftReal*) ek_parameters_gpu.charge_potential)[neighborindex[EK_LINK_U00]] -
             ek_parameters_gpu.ext_force[0][species_index] * ek_parameters_gpu.agrid
           )
         );
         
    flux = ( ek_parameters_gpu.d[species_index] / ek_parameters_gpu.agrid ) *
           ( 1.0f / boltzmannfactor_local +
             1.0f / boltzmannfactor_neighbor
           ) / 2.0f *
           ( ek_parameters_gpu.rho[species_index][index] *
             boltzmannfactor_local -
             ek_parameters_gpu.rho[species_index][neighborindex[EK_LINK_U00]] *
             boltzmannfactor_neighbor
           ) / ek_parameters_gpu.agrid;
           
    atomicadd( &ek_parameters_gpu.j[jindex_getByRhoLinear(index, EK_LINK_U00)],
               flux * ek_parameters_gpu.time_step );

    force  = -1.0f * ek_parameters_gpu.valency[species_index] *
             ( ((hipfftReal*)ek_parameters_gpu.charge_potential)[neighborindex[EK_LINK_U00]] -
               ((hipfftReal*)ek_parameters_gpu.charge_potential)[index]
             ) / ek_parameters_gpu.agrid;

    force *= powf(ek_parameters_gpu.agrid, 1) *
             ek_parameters_gpu.time_step *
             ek_parameters_gpu.time_step;
             
    atomicadd( &node_f.force[index],
               ek_parameters_gpu.rho[species_index][index] *
               (
                 force / 2.0f +
                 ek_parameters_gpu.ext_force[0][species_index] *
                 (
                   powf(ek_parameters_gpu.agrid, 1) *
                   ek_parameters_gpu.time_step *
                   ek_parameters_gpu.time_step
                 )
               )
             );

    atomicadd( &node_f.force[neighborindex[EK_LINK_U00]],
                ek_parameters_gpu.rho[species_index][neighborindex[EK_LINK_U00]] *
                force / 2.0f );
    
    //face in y
    boltzmannfactor_neighbor =
      exp( 1.0f / ek_parameters_gpu.T *
           ( ek_parameters_gpu.valency[species_index] *
             ((hipfftReal*) ek_parameters_gpu.charge_potential)[neighborindex[EK_LINK_0U0]] -
             ek_parameters_gpu.ext_force[1][species_index] * ek_parameters_gpu.agrid
           )
         );
         
    flux = ( ek_parameters_gpu.d[species_index] / ek_parameters_gpu.agrid ) *
           ( 1.0f / boltzmannfactor_local +
             1.0f / boltzmannfactor_neighbor
           ) / 2.0f *
           ( ek_parameters_gpu.rho[species_index][index] *
             boltzmannfactor_local -
             ek_parameters_gpu.rho[species_index][neighborindex[EK_LINK_0U0]] *
             boltzmannfactor_neighbor
           ) / ek_parameters_gpu.agrid;
           
    atomicadd( &ek_parameters_gpu.j[jindex_getByRhoLinear(index, EK_LINK_0U0)],
               flux * ek_parameters_gpu.time_step );
              
    force  = -1.0f * ek_parameters_gpu.valency[species_index] *
             ( ((hipfftReal*) ek_parameters_gpu.charge_potential)[neighborindex[EK_LINK_0U0]] -
               ((hipfftReal*) ek_parameters_gpu.charge_potential)[index]
             ) / ek_parameters_gpu.agrid;

    force *= powf(ek_parameters_gpu.agrid, 1) *
             ek_parameters_gpu.time_step *
             ek_parameters_gpu.time_step;

    atomicadd( &node_f.force[ek_parameters_gpu.number_of_nodes + index],
               ek_parameters_gpu.rho[species_index][index] *
               (
                 force / 2.0f +
                 ek_parameters_gpu.ext_force[1][species_index] *
                 (
                   powf(ek_parameters_gpu.agrid, 1) *
                   ek_parameters_gpu.time_step *
                   ek_parameters_gpu.time_step
                 )
               )
             );

    atomicadd( &node_f.force[ek_parameters_gpu.number_of_nodes + neighborindex[EK_LINK_0U0]],
                ek_parameters_gpu.rho[species_index][neighborindex[EK_LINK_0U0]] *
                force / 2.0f );
              
    //face in z
    boltzmannfactor_neighbor =
      exp( 1.0f / ek_parameters_gpu.T *
           ( ek_parameters_gpu.valency[species_index] *
             ( (hipfftReal*) ek_parameters_gpu.charge_potential)[neighborindex[EK_LINK_00U]] -
               ek_parameters_gpu.ext_force[2][species_index] *
               ek_parameters_gpu.agrid
             )
           );
           
    flux = ( ek_parameters_gpu.d[species_index] / ek_parameters_gpu.agrid ) *
           ( 1.0f / boltzmannfactor_local +
             1.0f / boltzmannfactor_neighbor
           ) / 2.0f *
           ( ek_parameters_gpu.rho[species_index][index] *
             boltzmannfactor_local -
             ek_parameters_gpu.rho[species_index][neighborindex[EK_LINK_00U]] *
             boltzmannfactor_neighbor
           ) / ek_parameters_gpu.agrid;

    atomicadd( &ek_parameters_gpu.j[jindex_getByRhoLinear(index, EK_LINK_00U)],
               flux * ek_parameters_gpu.time_step );

    force  = -1.0f * ek_parameters_gpu.valency[species_index] *
             ( ((hipfftReal*) ek_parameters_gpu.charge_potential)[neighborindex[EK_LINK_00U]] -
               ((hipfftReal*) ek_parameters_gpu.charge_potential)[index]
             ) / ek_parameters_gpu.agrid;

    force *= powf(ek_parameters_gpu.agrid, 1) *
             ek_parameters_gpu.time_step *
             ek_parameters_gpu.time_step;

    atomicadd( &node_f.force[2*ek_parameters_gpu.number_of_nodes + index],
               ek_parameters_gpu.rho[species_index][index] *
               (
                 force / 2.0f +
                 ek_parameters_gpu.ext_force[2][species_index] *
                 (
                   powf(ek_parameters_gpu.agrid, 1) *
                   ek_parameters_gpu.time_step *
                   ek_parameters_gpu.time_step
                 )
               )
             );

    atomicadd( &node_f.force[2*ek_parameters_gpu.number_of_nodes + neighborindex[EK_LINK_00U]],
                ek_parameters_gpu.rho[species_index][neighborindex[EK_LINK_00U]] *
                force / 2.0f );
    
    //edge in z
    boltzmannfactor_neighbor =
      exp( 1.0f / ek_parameters_gpu.T *
           ( ek_parameters_gpu.valency[species_index] *
             ((hipfftReal*) ek_parameters_gpu.charge_potential)[neighborindex[EK_LINK_UU0]] -
             ( ek_parameters_gpu.ext_force[0][species_index] +
               ek_parameters_gpu.ext_force[1][species_index]
             ) * ek_parameters_gpu.agrid
           )
         );
             
    flux = ( ek_parameters_gpu.d[species_index] / ek_parameters_gpu.agrid ) *
           ( 1.0f / boltzmannfactor_local +
             1.0f/boltzmannfactor_neighbor
           ) / 2.0f *
           ( ek_parameters_gpu.rho[species_index][index] *
             boltzmannfactor_local -
             ek_parameters_gpu.rho[species_index][neighborindex[EK_LINK_UU0]] *
             boltzmannfactor_neighbor
           ) /
           ( sqrt(2.0f) * ek_parameters_gpu.agrid );
           
    atomicadd( &ek_parameters_gpu.j[jindex_getByRhoLinear(index, EK_LINK_UU0)],
               flux * ek_parameters_gpu.time_step
             );

    boltzmannfactor_neighbor =
      exp( 1.0f / ek_parameters_gpu.T *
           ( ek_parameters_gpu.valency[species_index] *
             ((hipfftReal*) ek_parameters_gpu.charge_potential)[neighborindex[EK_LINK_UD0]] -
             ( ek_parameters_gpu.ext_force[0][species_index] -
               ek_parameters_gpu.ext_force[1][species_index]
             ) * ek_parameters_gpu.agrid
           )
         );
    
    flux = ( ek_parameters_gpu.d[species_index] / ek_parameters_gpu.agrid ) *
           ( 1.0f / boltzmannfactor_local +
             1.0f / boltzmannfactor_neighbor
           ) / 2.0f *
           ( ek_parameters_gpu.rho[species_index][index] *
             boltzmannfactor_local -
             ek_parameters_gpu.rho[species_index][neighborindex[EK_LINK_UD0]] *
             boltzmannfactor_neighbor
           ) /
           ( sqrt(2.0f) * ek_parameters_gpu.agrid );
    
    atomicadd( &ek_parameters_gpu.j[jindex_getByRhoLinear(index, EK_LINK_UD0)],
               flux * ek_parameters_gpu.time_step );
        
    boltzmannfactor_neighbor =
      exp( 1.0f / ek_parameters_gpu.T *
           ( ek_parameters_gpu.valency[species_index] *
             ((hipfftReal*) ek_parameters_gpu.charge_potential)[neighborindex[EK_LINK_U0U]] -
             ( ek_parameters_gpu.ext_force[0][species_index] +
               ek_parameters_gpu.ext_force[2][species_index]
             ) * ek_parameters_gpu.agrid
           )
         );
    
    flux = ( ek_parameters_gpu.d[species_index] / ek_parameters_gpu.agrid ) *
           ( 1.0f / boltzmannfactor_local +
             1.0f / boltzmannfactor_neighbor
           ) / 2.0f *
           ( ek_parameters_gpu.rho[species_index][index] *
             boltzmannfactor_local -
             ek_parameters_gpu.rho[species_index][neighborindex[EK_LINK_U0U]]
             * boltzmannfactor_neighbor
           ) /
           ( sqrt(2.0f) * ek_parameters_gpu.agrid );
    
    atomicadd( &ek_parameters_gpu.j[jindex_getByRhoLinear(index, EK_LINK_U0U)],
               flux * ek_parameters_gpu.time_step );
        
    boltzmannfactor_neighbor =
      exp( 1.0f / ek_parameters_gpu.T *
           ( ek_parameters_gpu.valency[species_index] *
             ((hipfftReal*) ek_parameters_gpu.charge_potential)[neighborindex[EK_LINK_U0D]] -
             ( ek_parameters_gpu.ext_force[0][species_index] -
               ek_parameters_gpu.ext_force[2][species_index]
             ) * ek_parameters_gpu.agrid
           )
         );
    
    flux = ( ek_parameters_gpu.d[species_index] / ek_parameters_gpu.agrid ) *
           ( 1.0f / boltzmannfactor_local +
             1.0f / boltzmannfactor_neighbor
           ) / 2.0f *
           ( ek_parameters_gpu.rho[species_index][index] *
             boltzmannfactor_local -
             ek_parameters_gpu.rho[species_index][neighborindex[EK_LINK_U0D]] *
             boltzmannfactor_neighbor
           ) /
           ( sqrt(2.0f) * ek_parameters_gpu.agrid );

    atomicadd( &ek_parameters_gpu.j[jindex_getByRhoLinear(index, EK_LINK_U0D)],
               flux * ek_parameters_gpu.time_step );
    
    boltzmannfactor_neighbor =
      exp( 1.0f / ek_parameters_gpu.T *
           ( ek_parameters_gpu.valency[species_index] *
             ((hipfftReal*) ek_parameters_gpu.charge_potential)[neighborindex[EK_LINK_0UU]] -
             ( ek_parameters_gpu.ext_force[1][species_index] +
               ek_parameters_gpu.ext_force[2][species_index]
             ) * ek_parameters_gpu.agrid
           )
         );
    
    flux = ( ek_parameters_gpu.d[species_index] / ek_parameters_gpu.agrid ) *
           ( 1.0f / boltzmannfactor_local +
             1.0f / boltzmannfactor_neighbor
           ) / 2.0f *
           ( ek_parameters_gpu.rho[species_index][index] * boltzmannfactor_local -
             ek_parameters_gpu.rho[species_index][neighborindex[EK_LINK_0UU]] *
             boltzmannfactor_neighbor
           ) /
           ( sqrt(2.0f) * ek_parameters_gpu.agrid );

    atomicadd( &ek_parameters_gpu.j[jindex_getByRhoLinear(index, EK_LINK_0UU)],
               flux * ek_parameters_gpu.time_step );
    
    boltzmannfactor_neighbor =
      exp( 1.0f / ek_parameters_gpu.T *
           ( ek_parameters_gpu.valency[species_index] *
             ((hipfftReal*) ek_parameters_gpu.charge_potential)[neighborindex[EK_LINK_0UD]] -
             ( ek_parameters_gpu.ext_force[1][species_index] -
               ek_parameters_gpu.ext_force[2][species_index]
             ) * ek_parameters_gpu.agrid
           )
         );
    
    flux = ( ek_parameters_gpu.d[species_index] / ek_parameters_gpu.agrid ) *
           ( 1.0f / boltzmannfactor_local +
             1.0f / boltzmannfactor_neighbor
           ) / 2.0f *
           ( ek_parameters_gpu.rho[species_index][index] *
             boltzmannfactor_local -
             ek_parameters_gpu.rho[species_index][neighborindex[EK_LINK_0UD]] *
             boltzmannfactor_neighbor
           ) /
           ( sqrt(2.0f) * ek_parameters_gpu.agrid );

    atomicadd( &ek_parameters_gpu.j[jindex_getByRhoLinear(index, EK_LINK_0UD)],
               flux * ek_parameters_gpu.time_step );    
    
    /* advective contribution to flux */

    ek_displacement( dx, lb_node, index, ek_lbparameters_gpu );

    di[0] = 1 - signbit(dx[0]);
    di[1] = 1 - signbit(dx[1]);
    di[2] = 1 - signbit(dx[2]);

    dx[0] = fabs(dx[0]);
    dx[1] = fabs(dx[1]);
    dx[2] = fabs(dx[2]);

    //face in x
    node =
      rhoindex_cartesian2linear(
        (coord[0] + di[0] - 1 + ek_parameters_gpu.dim_x) % ek_parameters_gpu.dim_x,
        coord[1],
        coord[2]
      );
    
    atomicadd( &ek_parameters_gpu.j[jindex_getByRhoLinear( node, EK_LINK_U00 )],
               (2 * di[0] - 1) * ek_parameters_gpu.rho[species_index][index] *
               dx[0] * (1.0f - dx[1]) * (1.0f - dx[2])
             );
    
    //face in y
    node =
      rhoindex_cartesian2linear(
        coord[0],
        (coord[1] + di[1] - 1 + ek_parameters_gpu.dim_y) % ek_parameters_gpu.dim_y,
        coord[2]
      );
      
    atomicadd( &ek_parameters_gpu.j[jindex_getByRhoLinear( node, EK_LINK_0U0 )],
              (2 * di[1] - 1) * ek_parameters_gpu.rho[species_index][index] *
              (1.0f - dx[0]) * dx[1] * (1.0f - dx[2]) );
    
    //face in z
    node =
      rhoindex_cartesian2linear(
        coord[0],
        coord[1],
        (coord[2] + di[2] - 1 + ek_parameters_gpu.dim_z) % ek_parameters_gpu.dim_z
      );
      
    atomicadd( &ek_parameters_gpu.j[jindex_getByRhoLinear( node, EK_LINK_00U )],
               (2 * di[2] - 1) * ek_parameters_gpu.rho[species_index][index] *
               (1.0f - dx[0]) * (1.0f - dx[1]) * dx[2] );
    
    //edge in x
    node =
      rhoindex_cartesian2linear(
        coord[0],
        (coord[1] + di[1] - 1                   + ek_parameters_gpu.dim_y) % ek_parameters_gpu.dim_y,
        (coord[2] + (1 - di[1]) * (2*di[2] - 1) + ek_parameters_gpu.dim_z) % ek_parameters_gpu.dim_z
      );
        
    atomicadd( &ek_parameters_gpu.j[jindex_getByRhoLinear( node, EK_LINK_0UU + (di[1] + di[2] == 1) )],
               (2 * di[1] - 1) * ek_parameters_gpu.rho[species_index][index] *
               (1.0f - dx[0]) * dx[1] * dx[2]
             );
    
    //edge in y
    node =
      rhoindex_cartesian2linear(
        (coord[0] + di[0] - 1                   + ek_parameters_gpu.dim_x) % ek_parameters_gpu.dim_x,
        coord[1],
        (coord[2] + (1 - di[0]) * (2*di[2] - 1) + ek_parameters_gpu.dim_z) % ek_parameters_gpu.dim_z
      );
      
    atomicadd( &ek_parameters_gpu.j[jindex_getByRhoLinear( node, EK_LINK_U0U + (di[0] + di[2] == 1) )],
               (2 * di[0] - 1) * ek_parameters_gpu.rho[species_index][index] *
               dx[0] * (1.0f - dx[1]) * dx[2] );
    
    //edge in z
    node =
      rhoindex_cartesian2linear(
        (coord[0] + di[0] - 1                   + ek_parameters_gpu.dim_x) % ek_parameters_gpu.dim_x,
        (coord[1] + (1 - di[0]) * (2*di[1] - 1) + ek_parameters_gpu.dim_y) % ek_parameters_gpu.dim_y,
        coord[2]
      );
      
    atomicadd( &ek_parameters_gpu.j[jindex_getByRhoLinear( node, EK_LINK_UU0 + (di[0] + di[1] == 1) )],
               (2 * di[0] - 1) * ek_parameters_gpu.rho[species_index][index] *
               dx[0] * dx[1] * (1.0f - dx[2]) );
    
    //corner
    node =
      rhoindex_cartesian2linear(
        (coord[0] + di[0] - 1                   + ek_parameters_gpu.dim_x) % ek_parameters_gpu.dim_x,
        (coord[1] + (1 - di[0]) * (2*di[1] - 1) + ek_parameters_gpu.dim_y) % ek_parameters_gpu.dim_y,
        (coord[2] + (1 - di[0]) * (2*di[2] - 1) + ek_parameters_gpu.dim_z) % ek_parameters_gpu.dim_z
      );
      
    atomicadd( &ek_parameters_gpu.j[
                jindex_getByRhoLinear( node, (1 - di[0]) *
                                             (EK_LINK_UUU + 2*di[1] + di[2]) +
                                             di[0] * (EK_LINK_UDD - 2*di[1] - di[2])
                                     ) ],
               (2 * di[0] - 1) * ek_parameters_gpu.rho[species_index][index] *
               dx[0] * dx[1] * dx[2] );
  }
}


__global__ void ek_propagate_densities( unsigned int species_index
                                      ) {
                                      
  unsigned int index = ek_getThreadIndex();
  
  if( index < ek_parameters_gpu.number_of_nodes ) 
  {
  
    unsigned int neighborindex[13];
    unsigned int coord[3];
    
    rhoindex_linear2cartesian(index, coord);
    
    /* Indices of the neighbors storing the other half
       of the fluxes associated with this link */
    neighborindex[EK_LINK_D00-13] =
      rhoindex_cartesian2linear(
        (coord[0] - 1 + ek_parameters_gpu.dim_x) % ek_parameters_gpu.dim_x,
        coord[1],
        coord[2]
      );
      
    neighborindex[EK_LINK_0D0-13] =
      rhoindex_cartesian2linear(
        coord[0],
        (coord[1] - 1 + ek_parameters_gpu.dim_y) % ek_parameters_gpu.dim_y,
        coord[2]
      );
      
    neighborindex[EK_LINK_00D-13] =
      rhoindex_cartesian2linear(
        coord[0],
        coord[1],
        (coord[2] - 1 + ek_parameters_gpu.dim_z) % ek_parameters_gpu.dim_z
      );
      
    neighborindex[EK_LINK_DD0-13] =
      rhoindex_cartesian2linear(
        (coord[0] - 1 + ek_parameters_gpu.dim_x) % ek_parameters_gpu.dim_x,
        (coord[1] - 1 + ek_parameters_gpu.dim_y) % ek_parameters_gpu.dim_y,
        coord[2]
      );
      
    neighborindex[EK_LINK_DU0-13] =
      rhoindex_cartesian2linear(
        (coord[0] - 1 + ek_parameters_gpu.dim_x) % ek_parameters_gpu.dim_x,
        (coord[1] + 1                          ) % ek_parameters_gpu.dim_y,
        coord[2]
      );
      
    neighborindex[EK_LINK_D0D-13] =
      rhoindex_cartesian2linear(
        (coord[0] - 1 + ek_parameters_gpu.dim_x) % ek_parameters_gpu.dim_x,
        coord[1],
        (coord[2] - 1 + ek_parameters_gpu.dim_z) % ek_parameters_gpu.dim_z
      );
      
    neighborindex[EK_LINK_D0U-13] =
      rhoindex_cartesian2linear(
        (coord[0] - 1 + ek_parameters_gpu.dim_x) % ek_parameters_gpu.dim_x,
        coord[1],
        (coord[2] + 1                          ) % ek_parameters_gpu.dim_z
      );
      
    neighborindex[EK_LINK_0DD-13] =
      rhoindex_cartesian2linear(
        coord[0],
        (coord[1] - 1 + ek_parameters_gpu.dim_y) % ek_parameters_gpu.dim_y,
        (coord[2] - 1 + ek_parameters_gpu.dim_z) % ek_parameters_gpu.dim_z
      );
      
    neighborindex[EK_LINK_0DU-13] =
      rhoindex_cartesian2linear(
       coord[0],
       (coord[1] - 1 + ek_parameters_gpu.dim_y) % ek_parameters_gpu.dim_y,
       (coord[2] + 1                          ) % ek_parameters_gpu.dim_z
     );
      
    
    neighborindex[EK_LINK_DDD-13] =
      rhoindex_cartesian2linear(
       (coord[0] - 1 + ek_parameters_gpu.dim_x) % ek_parameters_gpu.dim_x,
       (coord[1] - 1 + ek_parameters_gpu.dim_y) % ek_parameters_gpu.dim_y,
       (coord[2] - 1 + ek_parameters_gpu.dim_z) % ek_parameters_gpu.dim_z
     );
      
    neighborindex[EK_LINK_DDU-13] =
      rhoindex_cartesian2linear(
       (coord[0] - 1 + ek_parameters_gpu.dim_x) % ek_parameters_gpu.dim_x,
       (coord[1] - 1 + ek_parameters_gpu.dim_y) % ek_parameters_gpu.dim_y,
       (coord[2] + 1                          ) % ek_parameters_gpu.dim_z
     );
      
    neighborindex[EK_LINK_DUD-13] = 
      rhoindex_cartesian2linear(
       (coord[0] - 1 + ek_parameters_gpu.dim_x) % ek_parameters_gpu.dim_x,
       (coord[1] + 1                          ) % ek_parameters_gpu.dim_y,
       (coord[2] - 1 + ek_parameters_gpu.dim_z) % ek_parameters_gpu.dim_z
     );
      
    neighborindex[EK_LINK_DUU-13] =
      rhoindex_cartesian2linear(
       (coord[0] - 1 + ek_parameters_gpu.dim_x) % ek_parameters_gpu.dim_x,
       (coord[1] + 1                          ) % ek_parameters_gpu.dim_y,
       (coord[2] + 1                          ) % ek_parameters_gpu.dim_z
     );
      
    
    /* Calculate change of densities due to diffusive fluxes */
    ek_parameters_gpu.rho[ species_index ][index] -=
      ek_parameters_gpu.j[ jindex_getByRhoLinear( index, EK_LINK_U00 ) ];
    ek_parameters_gpu.rho[ species_index ][index] +=
      ek_parameters_gpu.j[ jindex_getByRhoLinear( neighborindex[EK_LINK_D00-13], EK_LINK_U00 ) ];
    
    ek_parameters_gpu.rho[ species_index ][index] -=
      ek_parameters_gpu.j[ jindex_getByRhoLinear( index, EK_LINK_0U0 ) ];
    ek_parameters_gpu.rho[ species_index ][index] +=
      ek_parameters_gpu.j[ jindex_getByRhoLinear( neighborindex[EK_LINK_0D0-13], EK_LINK_0U0 ) ];

    ek_parameters_gpu.rho[ species_index ][index] -=
      ek_parameters_gpu.j[ jindex_getByRhoLinear( index, EK_LINK_00U ) ];
    ek_parameters_gpu.rho[species_index ][index]  +=
      ek_parameters_gpu.j[ jindex_getByRhoLinear( neighborindex[EK_LINK_00D-13], EK_LINK_00U ) ];
    
    ek_parameters_gpu.rho[ species_index ][index] -=
      ek_parameters_gpu.j[ jindex_getByRhoLinear( index, EK_LINK_UU0 ) ];
    ek_parameters_gpu.rho[ species_index ][index] +=
      ek_parameters_gpu.j[ jindex_getByRhoLinear( neighborindex[EK_LINK_DD0-13], EK_LINK_UU0 ) ];
    
    ek_parameters_gpu.rho[ species_index ][index] -=
      ek_parameters_gpu.j[ jindex_getByRhoLinear( index, EK_LINK_UD0 ) ];
    ek_parameters_gpu.rho[ species_index ][index] +=
      ek_parameters_gpu.j[ jindex_getByRhoLinear( neighborindex[EK_LINK_DU0-13], EK_LINK_UD0 ) ];
    
    ek_parameters_gpu.rho[ species_index ][index] -=
      ek_parameters_gpu.j[ jindex_getByRhoLinear( index, EK_LINK_U0U ) ];
    ek_parameters_gpu.rho[ species_index ][index] +=
      ek_parameters_gpu.j[ jindex_getByRhoLinear( neighborindex[EK_LINK_D0D-13], EK_LINK_U0U ) ];
    
    ek_parameters_gpu.rho[ species_index ][index] -=
      ek_parameters_gpu.j[ jindex_getByRhoLinear( index, EK_LINK_U0D ) ];
    ek_parameters_gpu.rho[ species_index ][index] +=
      ek_parameters_gpu.j[ jindex_getByRhoLinear( neighborindex[EK_LINK_D0U-13], EK_LINK_U0D ) ];
    
    ek_parameters_gpu.rho[ species_index ][index] -=
      ek_parameters_gpu.j[ jindex_getByRhoLinear( index, EK_LINK_0UU ) ];
    ek_parameters_gpu.rho[ species_index ][index] +=
      ek_parameters_gpu.j[ jindex_getByRhoLinear( neighborindex[EK_LINK_0DD-13], EK_LINK_0UU ) ];
    
    ek_parameters_gpu.rho[ species_index ][index] -=
      ek_parameters_gpu.j[ jindex_getByRhoLinear( index, EK_LINK_0UD ) ];
    ek_parameters_gpu.rho[ species_index ][index] +=
      ek_parameters_gpu.j[ jindex_getByRhoLinear( neighborindex[EK_LINK_0DU-13], EK_LINK_0UD ) ];
    
    ek_parameters_gpu.rho[ species_index ][index] -=
      ek_parameters_gpu.j[ jindex_getByRhoLinear( index, EK_LINK_UUU ) ];
    ek_parameters_gpu.rho[ species_index ][index] +=
      ek_parameters_gpu.j[ jindex_getByRhoLinear( neighborindex[EK_LINK_DDD-13], EK_LINK_UUU ) ];
      
    ek_parameters_gpu.rho[ species_index ][index] -=
      ek_parameters_gpu.j[ jindex_getByRhoLinear( index, EK_LINK_UUD ) ];
    ek_parameters_gpu.rho[ species_index ][index] +=
      ek_parameters_gpu.j[ jindex_getByRhoLinear( neighborindex[EK_LINK_DDU-13], EK_LINK_UUD ) ];
      
    ek_parameters_gpu.rho[ species_index ][index] -=
      ek_parameters_gpu.j[ jindex_getByRhoLinear( index, EK_LINK_UDU ) ];
    ek_parameters_gpu.rho[ species_index ][index] +=
      ek_parameters_gpu.j[ jindex_getByRhoLinear( neighborindex[EK_LINK_DUD-13], EK_LINK_UDU ) ];
      
    ek_parameters_gpu.rho[ species_index ][index] -=
      ek_parameters_gpu.j[ jindex_getByRhoLinear( index, EK_LINK_UDD ) ];
    ek_parameters_gpu.rho[ species_index ][index] +=
      ek_parameters_gpu.j[ jindex_getByRhoLinear( neighborindex[EK_LINK_DUU-13], EK_LINK_UDD ) ];
  }
}


__global__ void ek_apply_boundaries( unsigned int species_index,
                                     LB_nodes_gpu lbnode,
                                     LB_node_force_gpu node_f
                                   ) {

  unsigned int index = ek_getThreadIndex();
  unsigned int neighborindex[22];
  unsigned int coord[3];

  if( index < ek_parameters_gpu.number_of_nodes ) 
  {
    if( lbnode.boundary[index] ) 
    {
    
      rhoindex_linear2cartesian(index, coord);
      
      /* Indices of the neighbors */
      neighborindex[EK_LINK_D00-13] =
        rhoindex_cartesian2linear(
          (coord[0] - 1 + ek_parameters_gpu.dim_x) % ek_parameters_gpu.dim_x,
          coord[1],
          coord[2]
        );
          
      neighborindex[EK_LINK_0D0-13] =
        rhoindex_cartesian2linear(
          coord[0], 
          (coord[1] - 1 + ek_parameters_gpu.dim_y) % ek_parameters_gpu.dim_y,
          coord[2]
        );
        
      neighborindex[EK_LINK_00D-13] =
        rhoindex_cartesian2linear(
          coord[0],
          coord[1],
          (coord[2] - 1 + ek_parameters_gpu.dim_z) % ek_parameters_gpu.dim_z
        );
        
      neighborindex[EK_LINK_DD0-13] =
        rhoindex_cartesian2linear(
          (coord[0] - 1 + ek_parameters_gpu.dim_x) % ek_parameters_gpu.dim_x,
          (coord[1] - 1 + ek_parameters_gpu.dim_y) % ek_parameters_gpu.dim_y,
          coord[2]
        );
        
      neighborindex[EK_LINK_DU0-13] =
        rhoindex_cartesian2linear(
          (coord[0] - 1 + ek_parameters_gpu.dim_x) % ek_parameters_gpu.dim_x,
          (coord[1] + 1                          ) % ek_parameters_gpu.dim_y,
          coord[2]
        );
        
      neighborindex[EK_LINK_D0D-13] =
        rhoindex_cartesian2linear(
          (coord[0] - 1 + ek_parameters_gpu.dim_x) % ek_parameters_gpu.dim_x,
          coord[1],
          (coord[2] - 1 + ek_parameters_gpu.dim_z) % ek_parameters_gpu.dim_z
        );
        
      neighborindex[EK_LINK_D0U-13] =
        rhoindex_cartesian2linear(
          (coord[0] - 1 + ek_parameters_gpu.dim_x) % ek_parameters_gpu.dim_x,
          coord[1],
          (coord[2] + 1                          ) % ek_parameters_gpu.dim_z
        );
        
      neighborindex[EK_LINK_0DD-13] =
        rhoindex_cartesian2linear(
          coord[0],
          (coord[1] - 1 + ek_parameters_gpu.dim_y) % ek_parameters_gpu.dim_y,
          (coord[2] - 1 + ek_parameters_gpu.dim_z) % ek_parameters_gpu.dim_z
        );
        
      neighborindex[EK_LINK_0DU-13] =
        rhoindex_cartesian2linear(
          coord[0],
          (coord[1] - 1 + ek_parameters_gpu.dim_y) % ek_parameters_gpu.dim_y,
          (coord[2] + 1                          ) % ek_parameters_gpu.dim_z
        );
        
      neighborindex[EK_LINK_DDD-13] =
        rhoindex_cartesian2linear(
          (coord[0] - 1 + ek_parameters_gpu.dim_x) % ek_parameters_gpu.dim_x,
          (coord[1] - 1 + ek_parameters_gpu.dim_y) % ek_parameters_gpu.dim_y,
          (coord[2] - 1 + ek_parameters_gpu.dim_z) % ek_parameters_gpu.dim_z
        );
      
      neighborindex[EK_LINK_DDU-13] =
        rhoindex_cartesian2linear(
          (coord[0] - 1 + ek_parameters_gpu.dim_x) % ek_parameters_gpu.dim_x,
          (coord[1] - 1 + ek_parameters_gpu.dim_y) % ek_parameters_gpu.dim_y,
          (coord[2] + 1                          ) % ek_parameters_gpu.dim_z
        );
      
      neighborindex[EK_LINK_DUD-13] =
        rhoindex_cartesian2linear(
          (coord[0] - 1 + ek_parameters_gpu.dim_x) % ek_parameters_gpu.dim_x,
          (coord[1] + 1                          ) % ek_parameters_gpu.dim_y,
          (coord[2] - 1 + ek_parameters_gpu.dim_z) % ek_parameters_gpu.dim_z
        );
      
      neighborindex[EK_LINK_DUU-13] =
        rhoindex_cartesian2linear(
          (coord[0] - 1 + ek_parameters_gpu.dim_x) % ek_parameters_gpu.dim_x,
          (coord[1] + 1                          ) % ek_parameters_gpu.dim_y,
          (coord[2] + 1                          ) % ek_parameters_gpu.dim_z
        );
      
      /* Clear fluxes on links connecting a boundary node */
      for( int i = 0; i < 13; i++ )
        ek_parameters_gpu.j[jindex_getByRhoLinear(index, i)] = 0.0f;
        
      ek_parameters_gpu.j[ jindex_getByRhoLinear( neighborindex[ EK_LINK_D00-13 ], EK_LINK_U00 ) ] = 0.0f;
      ek_parameters_gpu.j[ jindex_getByRhoLinear( neighborindex[ EK_LINK_0D0-13 ], EK_LINK_0U0 ) ] = 0.0f;
      ek_parameters_gpu.j[ jindex_getByRhoLinear( neighborindex[ EK_LINK_00D-13 ], EK_LINK_00U ) ] = 0.0f;
      ek_parameters_gpu.j[ jindex_getByRhoLinear( neighborindex[ EK_LINK_DD0-13 ], EK_LINK_UU0 ) ] = 0.0f;
      ek_parameters_gpu.j[ jindex_getByRhoLinear( neighborindex[ EK_LINK_DU0-13 ], EK_LINK_UD0 ) ] = 0.0f;
      ek_parameters_gpu.j[ jindex_getByRhoLinear( neighborindex[ EK_LINK_D0D-13 ], EK_LINK_U0U ) ] = 0.0f;
      ek_parameters_gpu.j[ jindex_getByRhoLinear( neighborindex[ EK_LINK_D0U-13 ], EK_LINK_U0D ) ] = 0.0f;
      ek_parameters_gpu.j[ jindex_getByRhoLinear( neighborindex[ EK_LINK_0DD-13 ], EK_LINK_0UU ) ] = 0.0f;
      ek_parameters_gpu.j[ jindex_getByRhoLinear( neighborindex[ EK_LINK_0DU-13 ], EK_LINK_0UD ) ] = 0.0f;
      ek_parameters_gpu.j[ jindex_getByRhoLinear( neighborindex[ EK_LINK_DDD-13 ], EK_LINK_UUU ) ] = 0.0f;
      ek_parameters_gpu.j[ jindex_getByRhoLinear( neighborindex[ EK_LINK_DDU-13 ], EK_LINK_UUD ) ] = 0.0f;
      ek_parameters_gpu.j[ jindex_getByRhoLinear( neighborindex[ EK_LINK_DUD-13 ], EK_LINK_UDU ) ] = 0.0f;
      ek_parameters_gpu.j[ jindex_getByRhoLinear( neighborindex[ EK_LINK_DUU-13 ], EK_LINK_UDD ) ] = 0.0f;
    }
  }
}


//TODO maybe make this obsolete by a multiplication in the advective fluxes, just as it's done for the diffusive ones
__global__ void ek_clear_fluxes() {

  unsigned int index = ek_getThreadIndex();

  if( index < ek_parameters_gpu.number_of_nodes ) 
  {
    for( int i = 0; i < 13; i++ ) 
    {
      ek_parameters_gpu.j[ jindex_getByRhoLinear( index, i ) ] = 0.0f;
    }
  }
}


__global__ void ek_init_species_density_homogeneous() {

  unsigned int index = ek_getThreadIndex();

  if(index < ek_parameters_gpu.number_of_nodes) 
  {  
    for(int i = 0; i < ek_parameters_gpu.number_of_species; i++) 
    {
      ek_parameters_gpu.rho[ i ][ index ] = ek_parameters_gpu.density[ i ] *
                                            ek_parameters_gpu.agrid *
                                            ek_parameters_gpu.agrid *
                                            ek_parameters_gpu.agrid;
    }
  }
}


__global__ void ek_multiply_greensfcn() {

  unsigned int index = ek_getThreadIndex();
  
  if( index < ek_parameters_gpu.dim_z *
              ek_parameters_gpu.dim_y *
              (ek_parameters_gpu.dim_x / 2 + 1) ) 
  {
    ek_parameters_gpu.charge_potential[ index ].x *= ek_parameters_gpu.greensfcn[ index ];
    ek_parameters_gpu.charge_potential[ index ].y *= ek_parameters_gpu.greensfcn[ index ];
  }
}


__global__ void ek_gather_species_charge_density() {

  unsigned int index = ek_getThreadIndex();

  if( index < ek_parameters_gpu.number_of_nodes ) 
  {
    ((hipfftReal*) ek_parameters_gpu.charge_potential)[ index ] = 0.0f;
    
    for( int i = 0; i < ek_parameters_gpu.number_of_species; i++ ) 
    {
    
      ((hipfftReal*) ek_parameters_gpu.charge_potential)[ index ] +=
        ek_parameters_gpu.valency[ i ] * ek_parameters_gpu.rho[ i ][ index ] /
        powf( ek_parameters_gpu.agrid, 3 );
    }
  }
}


__global__ void ek_gather_particle_charge_density( CUDA_particle_data * particle_data,
                                                   LB_parameters_gpu * ek_lbparameters_gpu
                                                 ) {

  unsigned int index = ek_getThreadIndex();
  unsigned int lowernode[3];
  float cellpos[3];
  float gridpos;

  if( index < ek_lbparameters_gpu->number_of_particles ) 
  {  
    gridpos      = particle_data[ index ].p[0] / ek_parameters_gpu.agrid - 0.5f;
    lowernode[0] = (int) floorf( gridpos );
    cellpos[0]   = gridpos - lowernode[0];
  
    gridpos      = particle_data[ index ].p[1] / ek_parameters_gpu.agrid - 0.5f;
    lowernode[1] = (int) floorf( gridpos );
    cellpos[1]   = gridpos - lowernode[1];
  
    gridpos      = particle_data[ index ].p[2] / ek_parameters_gpu.agrid - 0.5f;
    lowernode[2] = (int) floorf( gridpos );
    cellpos[2]   = gridpos - lowernode[2];

    lowernode[0] = (lowernode[0] + ek_lbparameters_gpu->dim_x) % ek_lbparameters_gpu->dim_x;
    lowernode[1] = (lowernode[1] + ek_lbparameters_gpu->dim_y) % ek_lbparameters_gpu->dim_y;
    lowernode[2] = (lowernode[2] + ek_lbparameters_gpu->dim_z) % ek_lbparameters_gpu->dim_z;

    atomicadd( &((hipfftReal*) ek_parameters_gpu.charge_potential)[
                 rhoindex_cartesian2linear( lowernode[0],
                                            lowernode[1],
                                            lowernode[2]  )
               ],
               particle_data[ index ].q *
               ( 1 - cellpos[0] ) * ( 1 - cellpos[1] ) * ( 1 - cellpos[2] )
    );
    
    atomicadd( &((hipfftReal*) ek_parameters_gpu.charge_potential)[
                 rhoindex_cartesian2linear( ( lowernode[0] + 1 ) % ek_parameters_gpu.dim_x,
                                            lowernode[1],
                                            lowernode[2]                                    )
               ],
               particle_data[ index ].q *
               cellpos[0] * ( 1 - cellpos[1] ) * ( 1 - cellpos[2] )
    );
    
    atomicadd( &((hipfftReal*) ek_parameters_gpu.charge_potential)[
                 rhoindex_cartesian2linear( lowernode[0],
                                            ( lowernode[1] + 1 ) % ek_parameters_gpu.dim_y,
                                            lowernode[2]                                    )
               ],
               particle_data[ index ].q *
               ( 1 - cellpos[0] ) * cellpos[1] * ( 1 - cellpos[2] )
    );
    
    atomicadd( &((hipfftReal*) ek_parameters_gpu.charge_potential)[
                 rhoindex_cartesian2linear( lowernode[0],
                                            lowernode[1],
                                            ( lowernode[2] + 1 ) % ek_parameters_gpu.dim_z  )
               ],
               particle_data[ index ].q *
               ( 1 - cellpos[0] ) * ( 1 - cellpos[1] ) * cellpos[2]
    );
    
    atomicadd( &((hipfftReal*) ek_parameters_gpu.charge_potential)[
                 rhoindex_cartesian2linear( ( lowernode[0] + 1 ) % ek_parameters_gpu.dim_x,
                                            ( lowernode[1] + 1 ) % ek_parameters_gpu.dim_y,
                                            lowernode[2]                                    )
               ],
               particle_data[ index ].q *
               cellpos[0] * cellpos[1] * ( 1 - cellpos[2] )
    );
    
    atomicadd( &((hipfftReal*) ek_parameters_gpu.charge_potential)[
                 rhoindex_cartesian2linear( ( lowernode[0] + 1 ) % ek_parameters_gpu.dim_x,
                                            lowernode[1],
                                            ( lowernode[2] + 1 ) % ek_parameters_gpu.dim_z  )
               ],
               particle_data[ index ].q *
               cellpos[0] * ( 1 - cellpos[1] ) * cellpos[2]
    );
    
    atomicadd( &((hipfftReal*) ek_parameters_gpu.charge_potential)[
                 rhoindex_cartesian2linear( lowernode[0],
                                            ( lowernode[1] + 1 ) % ek_parameters_gpu.dim_y,
                                            ( lowernode[2] + 1 ) % ek_parameters_gpu.dim_z  )
               ],
               particle_data[ index ].q *
               ( 1 - cellpos[0] ) * cellpos[1] * cellpos[2]
    );
    
    atomicadd( &((hipfftReal*) ek_parameters_gpu.charge_potential)[
                 rhoindex_cartesian2linear( ( lowernode[0] + 1 ) % ek_parameters_gpu.dim_x,
                                            ( lowernode[1] + 1 ) % ek_parameters_gpu.dim_y,
                                            ( lowernode[2] + 1 ) % ek_parameters_gpu.dim_z  )
               ],
               particle_data[ index ].q *
               cellpos[0] * cellpos[1] * cellpos[2]
    );
    
    //((hipfftReal*) ek_parameters_gpu.charge_potential)[ index ] = 0.0f;
    //printf("particle %d (%d):\n  charge %f\n  pos %f %f %f\n  lowernode %d %d %d\n  cellpos %f %f %f\n\n", index, ek_lbparameters_gpu->number_of_particles, particle_data[index].q, particle_data[index].p[0], particle_data[index].p[1], particle_data[index].p[2], lowernode[0], lowernode[1], lowernode[2], cellpos[0], cellpos[1], cellpos[2]); //TODO delete
  }
}


__global__ void ek_create_greensfcn() {

  unsigned int index = ek_getThreadIndex();
  unsigned int tmp;
  unsigned int coord[3];
  
  coord[0] = index % ( ek_parameters_gpu.dim_x / 2 + 1 );
  tmp      = index / ( ek_parameters_gpu.dim_x / 2 + 1 );
  coord[1] = tmp % ek_parameters_gpu.dim_y;
  coord[2] = tmp / ek_parameters_gpu.dim_y;
  
  if( index < ek_parameters_gpu.dim_z *
              ek_parameters_gpu.dim_y *
              ( ek_parameters_gpu.dim_x / 2 + 1 ) ) 
  {
              
    if( index == 0 ) 
    {
    
      //setting 0th fourier mode to 0 enforces charge neutrality
      ek_parameters_gpu.greensfcn[index] = 0.0f;
    }
    else 
    {
    
      ek_parameters_gpu.greensfcn[ index ] =
        -4.0f * PI_FLOAT * ek_parameters_gpu.bjerrumlength *
        ek_parameters_gpu.T * ek_parameters_gpu.agrid * ek_parameters_gpu.agrid *
        0.5f /
        ( cos( 2.0f * PI_FLOAT * coord[0] / (hipfftReal) ek_parameters_gpu.dim_x ) +
          cos( 2.0f * PI_FLOAT * coord[1] / (hipfftReal) ek_parameters_gpu.dim_y ) +
          cos( 2.0f * PI_FLOAT * coord[2] / (hipfftReal) ek_parameters_gpu.dim_z ) -
          3.0f
        ) /
        ( ek_parameters_gpu.dim_x *
          ek_parameters_gpu.dim_y *
          ek_parameters_gpu.dim_z
        );
    }
  }
}


__global__ void ek_clear_boundary_densities( LB_nodes_gpu lbnode ) {

  unsigned int index = ek_getThreadIndex();

  if( index < ek_parameters_gpu.number_of_nodes ) 
  {  
    if( lbnode.boundary[ index ] ) 
    {
    
      for( int i = 0; i < ek_parameters_gpu.number_of_species; i++ ) 
      {     
        ek_parameters_gpu.rho[ i ][ index ] = 0.0f;
      }
    }
  }
}


//TODO delete ?? (it has the previous step setting now)
__global__ void ek_clear_node_force( LB_node_force_gpu node_f ) {

  unsigned int index = ek_getThreadIndex();

  if( index < ek_parameters_gpu.number_of_nodes )
  {
    ek_parameters_gpu.lb_force_previous[ index ] = 
                           node_f.force[ index ];
    ek_parameters_gpu.lb_force_previous[ ek_parameters_gpu.number_of_nodes + index ] =
                           node_f.force[ ek_parameters_gpu.number_of_nodes + index ];
    ek_parameters_gpu.lb_force_previous[ 2 * ek_parameters_gpu.number_of_nodes + index ] = 
                           node_f.force[ 2 * ek_parameters_gpu.number_of_nodes + index ];

    node_f.force[ index ]                                         = 0.0f;
    node_f.force[ ek_parameters_gpu.number_of_nodes + index ]     = 0.0f;
    node_f.force[ 2 * ek_parameters_gpu.number_of_nodes + index ] = 0.0f;
  }
}


#ifdef EK_REACTION
__global__ void ek_reaction( ) {

  unsigned int index = ek_getThreadIndex();
  unsigned int coord[3];

  float* rho_reactant = &ek_parameters_gpu.rho[ek_parameters_gpu.reaction_species[0]][index];
  float* rho_product0 = &ek_parameters_gpu.rho[ek_parameters_gpu.reaction_species[1]][index];
  float* rho_product1 = &ek_parameters_gpu.rho[ek_parameters_gpu.reaction_species[2]][index];

  float dt = ek_parameters_gpu.time_step;
  float ct_rate = ek_parameters_gpu.reaction_ct_rate;
  float fraction_0 = ek_parameters_gpu.reaction_fraction_0;
  float fraction_1 = ek_parameters_gpu.reaction_fraction_1;

  float rho_change = *rho_reactant * ( 1.0f - expf(-dt*ct_rate) );

  rhoindex_linear2cartesian(index, coord);

  if ( index < ek_parameters_gpu.number_of_nodes )
  {
    if ( ek_parameters_gpu.node_is_catalyst[index] == 1 )
    {
      *rho_reactant -= rho_change;
      *rho_product0 += rho_change * fraction_0;
      *rho_product1 += rho_change * fraction_1;
    }
    else if ( ek_parameters_gpu.node_is_catalyst[index] == 2 )
    {
      *rho_reactant = ek_parameters_gpu.rho_reactant_reservoir * powf(ek_parameters_gpu.agrid,3);
      *rho_product0 = ek_parameters_gpu.rho_product0_reservoir * powf(ek_parameters_gpu.agrid,3);
      *rho_product1 = ek_parameters_gpu.rho_product1_reservoir * powf(ek_parameters_gpu.agrid,3); 
    } 
  }
}
#endif


__global__ void ek_calculate_boundary_forces( int n_lb_boundaries, 
                                              float* ek_lb_boundary_force, 
                                              float* ek_accelerated_frame_boundary_velocity, 
                                              LB_parameters_gpu *ek_lbparameters_gpu )
{
  // Set force to zero

  ek_accelerated_frame_boundary_force[0] = 0.0f;
  ek_accelerated_frame_boundary_force[1] = 0.0f;
  ek_accelerated_frame_boundary_force[2] = 0.0f;

  if ( ek_parameters_gpu.accelerated_frame_enabled == 1 )
  {
    // The number of nodes belonging to the fluid

    float number_of_fluid_nodes = ek_parameters_gpu.accelerated_frame_fluid_mass / 
                                  ek_lbparameters_gpu->rho[0] / 
                                  powf(ek_lbparameters_gpu->agrid,3);

    // Calculate the correct external force, on the Tcl level a volume density
    // is specified, which only acts on the fluid nodes!

    double ext_acc_force[3];
    ext_acc_force[0] = ek_parameters_gpu.ext_acceleration_force[0] * number_of_fluid_nodes *  
                       powf(ek_lbparameters_gpu->agrid,3);
    ext_acc_force[1] = ek_parameters_gpu.ext_acceleration_force[1] * number_of_fluid_nodes *  
                       powf(ek_lbparameters_gpu->agrid,3);
    ext_acc_force[2] = ek_parameters_gpu.ext_acceleration_force[2] * number_of_fluid_nodes *  
                       powf(ek_lbparameters_gpu->agrid,3);

    // Add/Start with the external force applied to the particle, which is
    // technically a boundary composite, this has a positive sign because it is
    // applied to the particle

    ek_accelerated_frame_boundary_force[0] += ext_acc_force[0];
    ek_accelerated_frame_boundary_force[1] += ext_acc_force[1];
    ek_accelerated_frame_boundary_force[2] += ext_acc_force[2];

// TODO : REMOVE
//printf("ext_force %f ", ek_accelerated_frame_boundary_force[2]);

    for ( int i = 0; i < n_lb_boundaries; i++)
    {
// TODO : REMOVE
//printf("bndry_force %f ", -ek_lb_boundary_force[3*i + 2]);

      // Sum over all the boundaries that make up the composite and add the total
      // friction force to the external force vector wise. The boundary force
      // coming from the LB has the incorrect sign, it points in the direction
      // of the fluid, not in the direction of the boundary, hence the minus.

      ek_accelerated_frame_boundary_force[0] += -ek_lb_boundary_force[3*i + 0];
      ek_accelerated_frame_boundary_force[1] += -ek_lb_boundary_force[3*i + 1];
      ek_accelerated_frame_boundary_force[2] += -ek_lb_boundary_force[3*i + 2];
    }

// TODO : REMOVE
//printf("ext_force+bndry_force %f ", ek_accelerated_frame_boundary_force[2]);

    // INTERMEZZO: Calculate the velocity of the frame, and thus obtain the 
    // velocity of the boundary with respect to the center of mass of the system.
    // In the finite box, the fluid's center of mass moves in the opposite
    // direction.

    ek_accelerated_frame_boundary_velocity[0] += ( ek_accelerated_frame_boundary_force[0] / 
                                                   ek_parameters_gpu.accelerated_frame_boundary_mass *
                                                   ek_parameters_gpu.time_step );
    ek_accelerated_frame_boundary_velocity[1] += ( ek_accelerated_frame_boundary_force[1] / 
                                                   ek_parameters_gpu.accelerated_frame_boundary_mass *
                                                   ek_parameters_gpu.time_step );
    ek_accelerated_frame_boundary_velocity[2] += ( ek_accelerated_frame_boundary_force[2] / 
                                                   ek_parameters_gpu.accelerated_frame_boundary_mass *
                                                   ek_parameters_gpu.time_step );

    // Now calculate the acceleration on the particle by dividing the total force
    // on the particle by the boundary mass. This acceleration is applied on the
    // fluid in the transformation (with a minus sign) and must be multiplied 
    // by the fluid mass to get the right force on the fluid
  
    ek_accelerated_frame_boundary_force[0] *= - (   ek_parameters_gpu.accelerated_frame_fluid_mass
                                                  / ek_parameters_gpu.accelerated_frame_boundary_mass );
    ek_accelerated_frame_boundary_force[1] *= - (   ek_parameters_gpu.accelerated_frame_fluid_mass
                                                  / ek_parameters_gpu.accelerated_frame_boundary_mass );
    ek_accelerated_frame_boundary_force[2] *= - (   ek_parameters_gpu.accelerated_frame_fluid_mass
                                                  / ek_parameters_gpu.accelerated_frame_boundary_mass );

// TODO : REMOVE
//printf("(ef+bf)*(mf/mp) %f ", ek_accelerated_frame_boundary_force[2]);

    // In a finite system there is also always the negative of the external force 
    // on the particle acting on the fluid. This force ensures that the there is
    // no total force acting on the system, hence momentum is conserved. That is,
    // for a moving particle there would otherwise be no stationary state.

    ek_accelerated_frame_boundary_force[0] -= ext_acc_force[0];
    ek_accelerated_frame_boundary_force[1] -= ext_acc_force[1];
    ek_accelerated_frame_boundary_force[2] -= ext_acc_force[2];

// TODO : REMOVE
//printf("(ef+bf)*(mf/mp) + ef %f ", ek_accelerated_frame_boundary_force[2]);

    // Do the unit conversion from LB units (boundary force) to units that 
    // can be added back into the LB fluid via the LB external force (MD units),
    // apparently this requires scaling by the fluid node number

    ek_accelerated_frame_boundary_force[0] *= ( ek_parameters_gpu.agrid * powf(ek_parameters_gpu.time_step, 2) / 
                                                number_of_fluid_nodes );
    ek_accelerated_frame_boundary_force[1] *= ( ek_parameters_gpu.agrid * powf(ek_parameters_gpu.time_step, 2) / 
                                                number_of_fluid_nodes );
    ek_accelerated_frame_boundary_force[2] *= ( ek_parameters_gpu.agrid * powf(ek_parameters_gpu.time_step, 2) / 
                                                number_of_fluid_nodes );
  }
}



void ek_integrate_electrostatics() {

  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x =
    ( ek_parameters.number_of_nodes + threads_per_block * blocks_per_grid_y - 1 ) /
    ( threads_per_block * blocks_per_grid_y );
  dim3 dim_grid = make_uint3( blocks_per_grid_x, blocks_per_grid_y, 1 );
  
  KERNELCALL( ek_gather_species_charge_density, dim_grid, threads_per_block, () );
  
  if ( lbpar_gpu.number_of_particles != 0 ) //TODO make it an if number_of_charged_particles != 0
  { 
  
    blocks_per_grid_x =
      ( lbpar_gpu.number_of_particles + threads_per_block * blocks_per_grid_y - 1 ) /
      ( threads_per_block * blocks_per_grid_y );
    dim_grid = make_uint3( blocks_per_grid_x, blocks_per_grid_y, 1 );
    
    particle_data_gpu = gpu_get_particle_pointer();
  
    KERNELCALL( ek_gather_particle_charge_density,
                dim_grid, threads_per_block,
                ( particle_data_gpu, ek_lbparameters_gpu ) );
  }
  
  if( hipfftExecR2C( plan_fft,
                    (hipfftReal*) ek_parameters.charge_potential,
                    ek_parameters.charge_potential               ) != HIPFFT_SUCCESS ) 
  {
                    
    fprintf(stderr, "ERROR: Unable to execute FFT plan\n");
  }
  
  blocks_per_grid_x =
    ( ek_parameters.dim_z * ek_parameters.dim_y * ( ek_parameters.dim_x / 2 + 1 ) +
      threads_per_block * blocks_per_grid_y - 1) / 
    ( threads_per_block * blocks_per_grid_y );
  dim_grid = make_uint3( blocks_per_grid_x, blocks_per_grid_y, 1 );
  
  KERNELCALL( ek_multiply_greensfcn, dim_grid, threads_per_block, () );
    
  if( hipfftExecC2R( plan_ifft,
                    ek_parameters.charge_potential,
                    (hipfftReal*) ek_parameters.charge_potential ) != HIPFFT_SUCCESS )
  {
                    
    fprintf(stderr, "ERROR: Unable to execute iFFT plan\n");
  }
}


void ek_integrate() {

  /** values for the kernel call */
  
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x =
    ( ek_parameters.number_of_nodes + threads_per_block * blocks_per_grid_y - 1 )
    / (threads_per_block * blocks_per_grid_y );
  dim3 dim_grid = make_uint3( blocks_per_grid_x, blocks_per_grid_y, 1 );



  /* Clears the force on the nodes and must be called before fluxes are calculated,
     since in the reaction set up the previous-step LB force is added to the flux
     (in ek_calculate_quantities / ek_displacement), which is copied in this routine;
     it also needs to be called before ek_accelerated_frame_transformation */

  KERNELCALL( ek_clear_node_force, dim_grid, threads_per_block, ( node_f ) );



#ifdef EK_REACTION
  if ( ek_parameters.reaction_species[0] != -1 &&
       ek_parameters.reaction_species[1] != -1 &&
       ek_parameters.reaction_species[2] != -1 )
  {
    /* Performs the catalytic reaction and sets the reservoir densities at 
       the boundary of the simulation box */

    KERNELCALL( ek_reaction, dim_grid, threads_per_block, ());

    /* Determines the excess pressure that follows from the creation of 
       species by the reaction */

    KERNELCALL( ek_pressure, dim_grid, threads_per_block, ( *current_nodes, 
                                                            ek_lbparameters_gpu, 
                                                            ek_lb_device_values ) );
  }
#endif


#ifdef EK_BOUNDARIES
  if ( ek_parameters.accelerated_frame_enabled == 1 && n_lb_boundaries > 0 )
  {
    /* Adds the force required to perform the accelerated frame tranformation,
       must be done before the integration of the LB is called; force is 
       technically from the previous step and is calculated in the kernel
       ek_calculate_boundary_forces */

    KERNELCALL( ek_accelerated_frame_transformation, dim_grid, threads_per_block, ( node_f ) );
  }
#endif


  /* Integrate diffusion-advection */
  
  for( int i = 0; i < ek_parameters.number_of_species; i++ )
  {
  
    KERNELCALL( ek_clear_fluxes, dim_grid, threads_per_block, () );
    KERNELCALL( ek_calculate_quantities, dim_grid, threads_per_block,
                ( i, *current_nodes, node_f, ek_lbparameters_gpu, ek_lb_device_values ) );
              
#ifdef EK_BOUNDARIES
    KERNELCALL( ek_apply_boundaries, dim_grid, threads_per_block,
                ( i, *current_nodes, node_f ) );
#endif

    KERNELCALL( ek_propagate_densities, dim_grid, threads_per_block, ( i ) );
  }



#ifdef EK_REACTION
  if ( ek_parameters.reaction_species[0] != -1 &&
       ek_parameters.reaction_species[1] != -1 &&
       ek_parameters.reaction_species[2] != -1 )
  {
    /* Add pressure force to LB must be done outside of loop,
       otherwise the force gets added several times */

    KERNELCALL( ek_add_ideal_pressure_to_lb_force, dim_grid, threads_per_block,
                  ( *current_nodes, node_f, ek_lbparameters_gpu ) );
  }
#endif



  /* Integrate electrostatics */
  
  ek_integrate_electrostatics();
  
  /* Integrate Navier-Stokes */
  
  lb_integrate_GPU();


  
#ifdef EK_BOUNDARIES
  if ( ek_parameters.accelerated_frame_enabled == 1 && n_lb_boundaries > 0 )
  {
    /* Calculate the total force on the boundaries for the accelerated frame transformation,
       can only be done after the LB integration is called */

    ek_calculate_boundary_forces<<<1,1>>>( n_lb_boundaries, ek_lb_boundary_force, ek_accelerated_frame_boundary_velocity, ek_lbparameters_gpu );
  }
#endif


  
  //TODO delete - needed for printfs
  hipDeviceSynchronize();

// TODO : REMOVE
/*
LB_rho_v_pi_gpu *host_values = (LB_rho_v_pi_gpu*) malloc( lbpar_gpu.number_of_nodes *
                                                        sizeof( LB_rho_v_pi_gpu ) );
lb_get_values_GPU( host_values ); 
printf( "ve %e %e %e\n", host_values[ 0 ].v[0], host_values[ 0 ].v[1], host_values[ 0 ].v[2] );
free(host_values);
*/
}


#ifdef EK_BOUNDARIES
void ek_init_species_density_wallcharge( float* wallcharge_species_density,
                                         int wallcharge_species             ) {
  
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x =
    ( ek_parameters.number_of_nodes + threads_per_block * blocks_per_grid_y - 1 )
    / ( threads_per_block * blocks_per_grid_y );
  dim3 dim_grid = make_uint3( blocks_per_grid_x, blocks_per_grid_y, 1 );
  
  KERNELCALL( ek_init_species_density_homogeneous, dim_grid, threads_per_block, () );
  KERNELCALL( ek_clear_boundary_densities, dim_grid, threads_per_block, ( *current_nodes ) );
  
  if( wallcharge_species != -1 ) 
  {  
    cuda_safe_mem( hipMemcpy( ek_parameters.rho[wallcharge_species], 
                               wallcharge_species_density,
                               ek_parameters.number_of_nodes * sizeof( float ),
                               hipMemcpyHostToDevice )
                 );
  }
}
#endif


void ek_init_species( int species ) {

  if( !initialized ) 
  {  
    ek_init();
  }
  
  if( ek_parameters.species_index[ species ] == -1 ) 
  {  
    ek_parameters.species_index[ species ] = ek_parameters.number_of_species;
    ek_parameters.number_of_species++;
    
    cuda_safe_mem( hipMalloc( (void**) &ek_parameters.rho[ ek_parameters.species_index[ species ] ],
                               ek_parameters.number_of_nodes * sizeof( float )                        ) );
    
    ek_parameters.density[      ek_parameters.species_index[ species ] ] = 0.0;
    ek_parameters.D[            ek_parameters.species_index[ species ] ] = 0.0;
    ek_parameters.valency[      ek_parameters.species_index[ species ] ] = 0.0;
    ek_parameters.ext_force[0][ ek_parameters.species_index[ species ] ] = 0.0;
    ek_parameters.ext_force[1][ ek_parameters.species_index[ species ] ] = 0.0;
    ek_parameters.ext_force[2][ ek_parameters.species_index[ species ] ] = 0.0;
    ek_parameters.d[            ek_parameters.species_index[ species ] ] =
      ek_parameters.D[          ek_parameters.species_index[ species ] ] / ( 1.0 + 2.0 * sqrt( 2.0 ) );
  }
}


int ek_init() {

  if( ek_parameters.agrid < 0.0 ||
      ek_parameters.viscosity < 0.0 ||
      ek_parameters.T < 0.0 ||
      ek_parameters.bjerrumlength < 0.0 ) 
  {
      
    fprintf( stderr, "ERROR: invalid agrid, viscosity, T or bjerrum_length\n" );
    
    return 1;
  }
    
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x;
  dim3 dim_grid;
  
  if(!initialized) 
  {
    if( hipGetSymbolAddress( (void**) &ek_parameters_gpu_pointer, ek_parameters_gpu ) != hipSuccess) 
    {
      fprintf( stderr, "ERROR: Fetching constant memory pointer\n" );

      return 1;
    }
    
    for( int i = 0; i < MAX_NUMBER_OF_SPECIES; i++ ) 
    {    
      ek_parameters.species_index[i] = -1;
    }

    if ( lattice_switch != LATTICE_OFF ) 
    {
      fprintf( stderr, "ERROR: Electrokinetics automatically intializes the LB on the GPU and can therefore not be used in conjunction with LB.\n");
      fprintf( stderr, "ERROR: Please run either electrokinetics or LB.\n");
      
      return 1;
    }

    lattice_switch = LATTICE_LB_GPU;
    ek_initialized = 1;         

    lbpar_gpu.agrid = ek_parameters.agrid;
    lbpar_gpu.viscosity[0] = ek_parameters.viscosity;
    lbpar_gpu.bulk_viscosity[0] = ek_parameters.bulk_viscosity;
    lbpar_gpu.friction[0] = ek_parameters.friction;

    lbpar_gpu.rho[0] = ( ek_parameters.lb_density < 0.0 ? 1.0 : ek_parameters.lb_density );
    lb_reinit_parameters_gpu();

    lbpar_gpu.external_force = 0;
    lbpar_gpu.ext_force[0] = 0.0;
    lbpar_gpu.ext_force[1] = 0.0;
    lbpar_gpu.ext_force[2] = 0.0;
    
    lb_init_gpu();

    ek_parameters.dim_x = lbpar_gpu.dim_x;
    ek_parameters.dim_y = lbpar_gpu.dim_y;
    ek_parameters.dim_z = lbpar_gpu.dim_z;
    ek_parameters.time_step = lbpar_gpu.time_step;
    ek_parameters.number_of_nodes = ek_parameters.dim_x * ek_parameters.dim_y * ek_parameters.dim_z;

    cuda_safe_mem( hipMalloc( (void**) &ek_parameters.j,
                             ek_parameters.number_of_nodes * 13 * sizeof( float ) ) );
    cuda_safe_mem( hipMemcpyToSymbol(HIP_SYMBOL( ek_parameters_gpu), &ek_parameters, sizeof( EK_parameters ) ) );
    
    lb_get_para_pointer( &ek_lbparameters_gpu );
    lb_set_ek_pointer( ek_parameters_gpu_pointer );

    cuda_safe_mem( hipMalloc( (void**) &ek_parameters.lb_force_previous,
                             ek_parameters.number_of_nodes * 3 * sizeof( float ) ) );

#ifdef EK_REACTION
    cuda_safe_mem( hipMalloc( (void**) &ek_parameters.pressure,
                             ek_parameters.number_of_nodes * sizeof( float ) ) );
    ek_node_is_catalyst = (char*) calloc( ek_parameters.number_of_nodes , sizeof( char ) );
#endif

    lb_get_device_values_pointer( &ek_lb_device_values );
    
    if( hipGetLastError() != hipSuccess ) 
    {
      fprintf(stderr, "ERROR: Failed to allocate\n");
      return 1;
    }
    
    cuda_safe_mem( hipMalloc( (void**) &ek_parameters.greensfcn,
                             sizeof( hipfftReal ) * 
                ek_parameters.dim_z * ek_parameters.dim_y * ( ek_parameters.dim_x / 2 + 1 ) ) );
    
    if( hipGetLastError() != hipSuccess ) 
    {
      fprintf(stderr, "ERROR: Failed to allocate\n");
      return 1;
    }

    hipHostMalloc((void**) &ek_parameters.node_is_catalyst,
                             sizeof( char ) * 
                ek_parameters.dim_z*ek_parameters.dim_y*ek_parameters.dim_x );
    
    if(hipGetLastError() != hipSuccess) 
    {
      fprintf(stderr, "ERROR: Failed to allocate\n");
      return 1;
    }
    
    cuda_safe_mem( hipMemcpyToSymbol(HIP_SYMBOL( ek_parameters_gpu), &ek_parameters, sizeof( EK_parameters ) ) );
    
    blocks_per_grid_x =
      ( ek_parameters.dim_z * ek_parameters.dim_y * (ek_parameters.dim_x / 2 + 1) +
        threads_per_block * blocks_per_grid_y - 1
      ) / ( threads_per_block * blocks_per_grid_y );
    dim_grid = make_uint3( blocks_per_grid_x, blocks_per_grid_y, 1 );
    KERNELCALL( ek_create_greensfcn, dim_grid, threads_per_block, () );

    /* create 3D FFT plans */
    
    if( hipfftPlan3d( &plan_fft,
                     ek_parameters.dim_z,
                     ek_parameters.dim_y,
                     ek_parameters.dim_x,
                     HIPFFT_R2C            ) != HIPFFT_SUCCESS ) 
    {
      fprintf(stderr, "ERROR: Unable to create fft plan\n");
      return 1;
    }
    
    if( cufftSetCompatibilityMode( plan_fft, CUFFT_COMPATIBILITY_NATIVE ) != HIPFFT_SUCCESS ) 
    {    
      fprintf(stderr, "ERROR: Unable to set fft compatibility mode to native\n");
      return 1;
    }
    
    if( hipfftSetStream( plan_fft, stream[0]) != HIPFFT_SUCCESS ) 
    {
        fprintf(stderr, "ERROR: Unable to assign FFT to cuda stream\n");
        return 1;
    }

    if( hipfftPlan3d( &plan_ifft,
                     ek_parameters.dim_z,
                     ek_parameters.dim_y,
                     ek_parameters.dim_x,
                     HIPFFT_C2R            ) != HIPFFT_SUCCESS ) 
    {   
      fprintf(stderr, "ERROR: Unable to create ifft plan\n");
      return 1;
    }
    
    if( cufftSetCompatibilityMode( plan_ifft, CUFFT_COMPATIBILITY_NATIVE ) != HIPFFT_SUCCESS) 
    {   
      fprintf(stderr, "ERROR: Unable to set ifft compatibility mode to native\n");
      return 1;
    }
    
    if( hipfftSetStream( plan_ifft, stream[0] ) != HIPFFT_SUCCESS )
    {    
      fprintf(stderr, "ERROR: Unable to assign FFT to cuda stream\n");
      return 1;
    }
    
    blocks_per_grid_x =
      ( ek_parameters.dim_z * ek_parameters.dim_y * (ek_parameters.dim_x ) +
        threads_per_block * blocks_per_grid_y - 1
      ) / ( threads_per_block * blocks_per_grid_y );
    dim_grid = make_uint3( blocks_per_grid_x, blocks_per_grid_y, 1 );
    KERNELCALL( ek_clear_node_force, dim_grid, threads_per_block, ( node_f ) );
    
    cuda_safe_mem( hipMalloc( (void**) &ek_parameters.charge_potential,
                             sizeof( hipfftComplex ) *
                             ek_parameters.dim_z * ek_parameters.dim_y * ( ek_parameters.dim_x / 2 + 1 ) ) );

    cuda_safe_mem( hipMalloc( (void**) &ek_accelerated_frame_boundary_velocity,
                               3 * sizeof( float ) ) );

    cuda_safe_mem( hipMemset( ek_accelerated_frame_boundary_velocity,
                               0,
                               3 * sizeof( float ) ) );

    initialized = true;

    cuda_safe_mem( hipMemcpyToSymbol(HIP_SYMBOL( ek_parameters_gpu), &ek_parameters, sizeof( EK_parameters ) ) );
  }
  else
  {
    if ( lbpar_gpu.agrid != ek_parameters.agrid ||
         lbpar_gpu.viscosity[0] != ek_parameters.viscosity ||
         lbpar_gpu.bulk_viscosity[0] != ek_parameters.bulk_viscosity ||
         lbpar_gpu.friction[0] != ek_parameters.friction ||
         ( ( lbpar_gpu.rho[0] != 1.0 ) && ( lbpar_gpu.rho[0] != ek_parameters.lb_density ) )
       )
    {
      fprintf( stderr, "ERROR: The LB parameters on the GPU cannot be reinitialized.\n");
      
      return 1;
    }
    else
    {
      cuda_safe_mem( hipMemcpyToSymbol(HIP_SYMBOL( ek_parameters_gpu), &ek_parameters, sizeof( EK_parameters ) ) );

#ifdef EK_BOUNDARIES
      if ( old_number_of_boundaries != n_lb_boundaries )
      {
        lb_init_boundaries();
        lb_get_boundary_force_pointer( &ek_lb_boundary_force );
        old_number_of_boundaries = n_lb_boundaries;
      }

      // Determine the total boundary mass and the fluid mass

      unsigned int number_of_boundary_nodes = ek_calculate_boundary_mass( );
      unsigned int number_of_fluid_nodes = ek_parameters.number_of_nodes - number_of_boundary_nodes;
      
      ek_parameters.accelerated_frame_boundary_mass =   static_cast<float>( number_of_boundary_nodes )
                                                      * ek_parameters.accelerated_frame_boundary_mass_density
                                                      * powf(lbpar_gpu.agrid,3);

      ek_parameters.accelerated_frame_fluid_mass =   static_cast<float>( number_of_fluid_nodes )
                                                   * lbpar_gpu.rho[0] * powf(lbpar_gpu.agrid,3);
        
      cuda_safe_mem( hipMemcpyToSymbol(HIP_SYMBOL( ek_parameters_gpu), &ek_parameters, sizeof( EK_parameters ) ) );
#else
      blocks_per_grid_x =
        ( ek_parameters.number_of_nodes + threads_per_block * blocks_per_grid_y - 1 )
        / (threads_per_block * blocks_per_grid_y );
      dim_grid = make_uint3( blocks_per_grid_x, blocks_per_grid_y, 1 );
      

      if ( old_number_of_species != ek_parameters.number_of_species )
      {
        KERNELCALL( ek_init_species_density_homogeneous, dim_grid, threads_per_block, () );
        old_number_of_species = ek_parameters.number_of_species;
      }
#endif

#ifdef EK_REACTION
      // added to ensure that the pressure is set to the proper value in the first time step
      blocks_per_grid_x = (ek_parameters.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) / (threads_per_block * blocks_per_grid_y);
      dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);
      KERNELCALL( ek_pressure, dim_grid, threads_per_block, ( *current_nodes, ek_lbparameters_gpu, ek_lb_device_values ) );
#endif

      ek_integrate_electrostatics();
    }
  }

  //ek_print_parameters(); //TODO delete
      
  return 0; 
}


void lb_set_ek_pointer(EK_parameters* pointeradress) {
  lb_ek_parameters_gpu = pointeradress;
}


unsigned int ek_calculate_boundary_mass( )
{
  unsigned int* bound_array = (unsigned int*) malloc( lbpar_gpu.number_of_nodes*sizeof(unsigned int) );

  lb_get_boundary_flags_GPU(bound_array);

  unsigned int boundary_node_number = 0;

  for( int j=0; j<ek_parameters.number_of_nodes; j++)
    if( bound_array[j] != 0 ) boundary_node_number++;

  free(bound_array);

  return boundary_node_number;
}


void rhoindex_linear2cartesian_host( unsigned int index,
                                     unsigned int * coord
                                   ) {

  coord[0]  = index % ek_parameters.dim_x;
  index    /= ek_parameters.dim_x;
  coord[1]  = index % ek_parameters.dim_y;
  coord[2]  = index / ek_parameters.dim_y;
}


unsigned int jindex_cartesian2linear_host( unsigned int x,
                                           unsigned int y,
                                           unsigned int z,
                                           unsigned int c
                                         ) {
                                    
  x = ( x + ek_parameters.dim_x ) % ek_parameters.dim_x; //this does not happen in the GPU version of this function
  y = ( y + ek_parameters.dim_y ) % ek_parameters.dim_y;
  z = ( z + ek_parameters.dim_z ) % ek_parameters.dim_z;
  
  return c * ek_parameters.number_of_nodes + 
         z * ek_parameters.dim_y * ek_parameters.dim_x +
         y * ek_parameters.dim_x +
         x;
}


unsigned int jindex_getByRhoLinear_host( unsigned int rho_index,
                                         unsigned int c
                                       ) {
                                               
  return c * ek_parameters.number_of_nodes +
         rho_index;
}


unsigned int rhoindex_cartesian2linear_host( unsigned int x,
                                             unsigned int y,
                                             unsigned int z
                                           ) {

  return z * ek_parameters.dim_y * ek_parameters.dim_x +
         y * ek_parameters.dim_x +
         x;
}


int ek_lb_print_vtk_velocity( char* filename ) {

  FILE* fp = fopen( filename, "w" );

  if( fp == NULL ) 
  {  
    return 1;
  }
  
  LB_rho_v_pi_gpu *host_values = (LB_rho_v_pi_gpu*) malloc( lbpar_gpu.number_of_nodes *
                                                        sizeof( LB_rho_v_pi_gpu ) );
  lb_get_values_GPU( host_values );
  
  fprintf( fp, "\
# vtk DataFile Version 2.0\n\
velocity\n\
ASCII\n\
\n\
DATASET STRUCTURED_POINTS\n\
DIMENSIONS %u %u %u\n\
ORIGIN %f %f %f\n\
SPACING %f %f %f\n\
\nPOINT_DATA %u\n\
SCALARS velocity float 3\n\
LOOKUP_TABLE default\n",
           lbpar_gpu.dim_x, lbpar_gpu.dim_y, lbpar_gpu.dim_z,
           lbpar_gpu.agrid*0.5f, lbpar_gpu.agrid*0.5f, lbpar_gpu.agrid*0.5f,
           lbpar_gpu.agrid, lbpar_gpu.agrid, lbpar_gpu.agrid,
           lbpar_gpu.number_of_nodes                                      );

  for( int i = 0; i < lbpar_gpu.number_of_nodes; i++ ) 
  {  
    fprintf( fp, "%e %e %e ", host_values[ i ].v[0],
                              host_values[ i ].v[1],
                              host_values[ i ].v[2]  );
  }
  
  free(host_values);
  fclose(fp);
  
  return 0;
}


int ek_node_print_velocity( int x, int y, int z, double* velocity ) { //TODO only calculate single node velocity
  
  LB_rho_v_pi_gpu *host_values = (LB_rho_v_pi_gpu*) malloc( lbpar_gpu.number_of_nodes *
                                                        sizeof( LB_rho_v_pi_gpu ) );
  lb_get_values_GPU( host_values );
  
  int i = z * ek_parameters.dim_y * ek_parameters.dim_x + y * ek_parameters.dim_x + x;
  
  velocity[0] = host_values[i].v[0];
  velocity[1] = host_values[i].v[1];
  velocity[2] = host_values[i].v[2];
  
  free(host_values);
  
  return 0;
}


int ek_node_print_mass_flux( int x, int y, int z, double* mass_flux ) { //TODO only calculate single node mass_flux

  float current_mass_fraction;
  float total_mass =   ek_parameters.rho_reactant_reservoir*ek_parameters.mass_reactant
                     + ek_parameters.rho_product0_reservoir*ek_parameters.mass_product0
                     + ek_parameters.rho_product1_reservoir*ek_parameters.mass_product1;

  unsigned int coord[3];
  float flux_local_cartesian[3]; // temporary variable for converting fluxes into cartesian coordinates for output
  float mass_flux_local_cartesian[3] = {0.0,0.0,0.0}; // total mass flux

  float* fluxes = (float*) malloc( ek_parameters.number_of_nodes * 13 * sizeof( float ) );
  
  for ( int species_number = 0; species_number < 3; species_number++ )
  {
    int species = ek_parameters.reaction_species[species_number];

    if ( species_number == 0 )
      current_mass_fraction = ek_parameters.lb_density*ek_parameters.mass_reactant/total_mass;
    else if ( species_number == 1 )
      current_mass_fraction = ek_parameters.lb_density*ek_parameters.mass_product0/total_mass;
    else if ( species_number == 2 )
      current_mass_fraction = ek_parameters.lb_density*ek_parameters.mass_product1/total_mass;
    else 
      return 1;

    if( ek_parameters.species_index[ species ] != -1 ) {
    
      int threads_per_block = 64;
      int blocks_per_grid_y = 4;
      int blocks_per_grid_x =
        ( ek_parameters.number_of_nodes + threads_per_block * blocks_per_grid_y - 1 )
        / (threads_per_block * blocks_per_grid_y );
      dim3 dim_grid = make_uint3( blocks_per_grid_x, blocks_per_grid_y, 1 );
      
      KERNELCALL( ek_clear_fluxes, dim_grid, threads_per_block, () );
      KERNELCALL( ek_calculate_quantities, dim_grid, threads_per_block,
                  ( ek_parameters.species_index[ species ], *current_nodes, node_f, ek_lbparameters_gpu, ek_lb_device_values ) );
                
#ifdef EK_BOUNDARIES
      KERNELCALL( ek_apply_boundaries, dim_grid, threads_per_block,
                  ( ek_parameters.species_index[ species ], *current_nodes, node_f ) );
#endif
    
      cuda_safe_mem( hipMemcpy( fluxes, 
                                 ek_parameters.j,
                                 ek_parameters.number_of_nodes * 13*sizeof( float ),
                                 hipMemcpyDeviceToHost )
                   );
    }
    else
      return 1;

    int i = z * ek_parameters.dim_y * ek_parameters.dim_x + y * ek_parameters.dim_x + x;

    rhoindex_linear2cartesian_host(i, coord);
     
    flux_local_cartesian[0]  = 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_U00) ];

    flux_local_cartesian[0] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UU0) ];
    flux_local_cartesian[0] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UD0) ];
    flux_local_cartesian[0] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_U0U) ];
    flux_local_cartesian[0] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_U0D) ];

    flux_local_cartesian[0] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UUU) ];
    flux_local_cartesian[0] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UUD) ];
    flux_local_cartesian[0] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UDU) ];
    flux_local_cartesian[0] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UDD) ];

    flux_local_cartesian[0] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1], coord[2], EK_LINK_D00-13) ];

    flux_local_cartesian[0] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]-1, coord[2], EK_LINK_DD0-13) ];
    flux_local_cartesian[0] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]+1, coord[2], EK_LINK_DU0-13) ];
    flux_local_cartesian[0] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1], coord[2]-1, EK_LINK_D0D-13) ];
    flux_local_cartesian[0] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1], coord[2]+1, EK_LINK_D0U-13) ];

    flux_local_cartesian[0] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]-1, coord[2]-1, EK_LINK_DDD-13) ];
    flux_local_cartesian[0] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]-1, coord[2]+1, EK_LINK_DDU-13) ];
    flux_local_cartesian[0] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]+1, coord[2]-1, EK_LINK_DUD-13) ];
    flux_local_cartesian[0] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]+1, coord[2]+1, EK_LINK_DUU-13) ];


    flux_local_cartesian[1]  = 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_0U0) ];

    flux_local_cartesian[1] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UU0) ];
    flux_local_cartesian[1] -= 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UD0) ];
    flux_local_cartesian[1] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_0UU) ];
    flux_local_cartesian[1] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_0UD) ];

    flux_local_cartesian[1] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UUU) ];
    flux_local_cartesian[1] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UUD) ];
    flux_local_cartesian[1] -= 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UDU) ];
    flux_local_cartesian[1] -= 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UDD) ];

    flux_local_cartesian[1] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0], coord[1]-1, coord[2], EK_LINK_0D0-13) ];

    flux_local_cartesian[1] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]-1, coord[2], EK_LINK_DD0-13) ];
    flux_local_cartesian[1] -= 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]+1, coord[2], EK_LINK_DU0-13) ];
    flux_local_cartesian[1] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0], coord[1]-1, coord[2]-1, EK_LINK_0DD-13) ];
    flux_local_cartesian[1] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0], coord[1]-1, coord[2]+1, EK_LINK_0DU-13) ];

    flux_local_cartesian[1] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]-1, coord[2]-1, EK_LINK_DDD-13) ];
    flux_local_cartesian[1] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]-1, coord[2]+1, EK_LINK_DDU-13) ];
    flux_local_cartesian[1] -= 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]+1, coord[2]-1, EK_LINK_DUD-13) ];
    flux_local_cartesian[1] -= 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]+1, coord[2]+1, EK_LINK_DUU-13) ];


    flux_local_cartesian[2]  = 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_00U) ];

    flux_local_cartesian[2] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_U0U) ];
    flux_local_cartesian[2] -= 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_U0D) ];
    flux_local_cartesian[2] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_0UU) ];
    flux_local_cartesian[2] -= 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_0UD) ];

    flux_local_cartesian[2] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UUU) ];
    flux_local_cartesian[2] -= 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UUD) ];
    flux_local_cartesian[2] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UDU) ];
    flux_local_cartesian[2] -= 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UDD) ];

    flux_local_cartesian[2] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0], coord[1], coord[2]-1, EK_LINK_00D-13) ];

    flux_local_cartesian[2] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1], coord[2]-1, EK_LINK_D0D-13) ];
    flux_local_cartesian[2] -= 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1], coord[2]+1, EK_LINK_D0U-13) ];
    flux_local_cartesian[2] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0], coord[1]-1, coord[2]-1, EK_LINK_0DD-13) ];
    flux_local_cartesian[2] -= 0.5*fluxes[ jindex_cartesian2linear_host(coord[0], coord[1]-1, coord[2]+1, EK_LINK_0DU-13) ];

    flux_local_cartesian[2] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]-1, coord[2]-1, EK_LINK_DDD-13) ];
    flux_local_cartesian[2] -= 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]-1, coord[2]+1, EK_LINK_DDU-13) ];
    flux_local_cartesian[2] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]+1, coord[2]-1, EK_LINK_DUD-13) ];
    flux_local_cartesian[2] -= 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]+1, coord[2]+1, EK_LINK_DUU-13) ];

    mass_flux_local_cartesian[0] += current_mass_fraction * flux_local_cartesian[0] 
                                    / ( ek_parameters.time_step * ek_parameters.agrid * ek_parameters.agrid );
    mass_flux_local_cartesian[1] += current_mass_fraction * flux_local_cartesian[1] 
                                    / ( ek_parameters.time_step * ek_parameters.agrid * ek_parameters.agrid );
    mass_flux_local_cartesian[2] += current_mass_fraction * flux_local_cartesian[2] 
                                    / ( ek_parameters.time_step * ek_parameters.agrid * ek_parameters.agrid );
  }

  mass_flux[0] = mass_flux_local_cartesian[0];
  mass_flux[1] = mass_flux_local_cartesian[1];
  mass_flux[2] = mass_flux_local_cartesian[2];

  free( fluxes );

  return 0;
}


int ek_lb_print_vtk_density( char* filename ) {

  FILE* fp = fopen( filename, "w" );

  if( fp == NULL ) 
  {
    return 1;
  }
  
  LB_rho_v_pi_gpu *host_values = (LB_rho_v_pi_gpu*) malloc( lbpar_gpu.number_of_nodes *
                                                        sizeof( LB_rho_v_pi_gpu ) );
  lb_get_values_GPU( host_values );
  
  fprintf( fp, "\
# vtk DataFile Version 2.0\n\
density_lb\n\
ASCII\n\
\n\
DATASET STRUCTURED_POINTS\n\
DIMENSIONS %u %u %u\n\
ORIGIN %f %f %f\n\
SPACING %f %f %f\n\
\n\
POINT_DATA %u\n\
SCALARS density_lb float 1\n\
LOOKUP_TABLE default\n",
           lbpar_gpu.dim_x, lbpar_gpu.dim_y, lbpar_gpu.dim_z,
           lbpar_gpu.agrid*0.5f, lbpar_gpu.agrid*0.5f, lbpar_gpu.agrid*0.5f,
           lbpar_gpu.agrid, lbpar_gpu.agrid, lbpar_gpu.agrid,
           lbpar_gpu.number_of_nodes                                      );

  for( int i = 0; i < lbpar_gpu.number_of_nodes; i++ ) 
  {  
    fprintf( fp, "%e ", host_values[ i ].rho[ 0 ] );
  }
  
  free( host_values );
  fclose( fp );
  
  return 0;
}


int ek_print_vtk_density( int species, char* filename ) {

  FILE* fp = fopen( filename, "w" );

  if( fp == NULL ){
    return 1;
  }

  float* densities = (float*) malloc( ek_parameters.number_of_nodes *
                                      sizeof( float )                 );
  
  if( ek_parameters.species_index[ species ] != -1 ) 
  {  
    cuda_safe_mem( hipMemcpy( densities, 
                               ek_parameters.rho[ ek_parameters.species_index[ species ] ],
                               ek_parameters.number_of_nodes * sizeof( float ),
                               hipMemcpyDeviceToHost )
                 );
  }
  else
    return 1;
  
  fprintf( fp, "\
# vtk DataFile Version 2.0\n\
density_%d\n\
ASCII\n\
\n\
DATASET STRUCTURED_POINTS\n\
DIMENSIONS %u %u %u\n\
ORIGIN %f %f %f\n\
SPACING %f %f %f\n\
\n\
POINT_DATA %u\n\
SCALARS density_%d float 1\n\
LOOKUP_TABLE default\n",
           species,
           ek_parameters.dim_x, ek_parameters.dim_y, ek_parameters.dim_z,
           ek_parameters.agrid*0.5f, ek_parameters.agrid*0.5f, ek_parameters.agrid*0.5f,
           ek_parameters.agrid, ek_parameters.agrid, ek_parameters.agrid,
           ek_parameters.number_of_nodes,
           species                                                                    );

  for( int i = 0; i < ek_parameters.number_of_nodes; i++ ) 
  {  
    fprintf( fp, "%e\n", densities[ i ] / (ek_parameters.agrid*ek_parameters.agrid*ek_parameters.agrid) );
  }
  
  free( densities );
  fclose( fp );
  
  return 0;
}


int ek_node_print_density( int species, int x, int y, int z, double* density ) {

  float* densities = (float*) malloc( ek_parameters.number_of_nodes *
                                      sizeof( float )                 );
  
  if( ek_parameters.species_index[ species ] != -1 ) 
  {  
    cuda_safe_mem( hipMemcpy( densities, 
                               ek_parameters.rho[ ek_parameters.species_index[ species ] ],
                               ek_parameters.number_of_nodes * sizeof( float ),
                               hipMemcpyDeviceToHost )
                 );
  }
  else
    return 1;
  
  *density = densities[z * ek_parameters.dim_y * ek_parameters.dim_x + y * ek_parameters.dim_x + x] / (ek_parameters.agrid*ek_parameters.agrid*ek_parameters.agrid);
  
  free( densities );
  
  return 0;
}


int ek_print_vtk_flux( int species, char* filename ) {

  FILE* fp = fopen( filename, "w" );
  float flux_local_cartesian[3]; //temporary variable for converting fluxes into cartesian coordinates for output

  unsigned int coord[3];

  if( fp == NULL ){
    return 1;
  }

  float* fluxes = (float*) malloc( ek_parameters.number_of_nodes * 13 * sizeof( float ) );
  
  if( ek_parameters.species_index[ species ] != -1 ) 
  {  
    int threads_per_block = 64;
    int blocks_per_grid_y = 4;
    int blocks_per_grid_x =
      ( ek_parameters.number_of_nodes + threads_per_block * blocks_per_grid_y - 1 )
      / (threads_per_block * blocks_per_grid_y );
    dim3 dim_grid = make_uint3( blocks_per_grid_x, blocks_per_grid_y, 1 );
    
    KERNELCALL( ek_clear_fluxes, dim_grid, threads_per_block, () );
    KERNELCALL( ek_calculate_quantities, dim_grid, threads_per_block,
                ( ek_parameters.species_index[ species ], *current_nodes, node_f, ek_lbparameters_gpu, ek_lb_device_values )    );
              
#ifdef EK_BOUNDARIES
    KERNELCALL( ek_apply_boundaries, dim_grid, threads_per_block,
                ( ek_parameters.species_index[ species ], *current_nodes, node_f )                     );
#endif
  
    cuda_safe_mem( hipMemcpy( fluxes, 
                               ek_parameters.j,
                               ek_parameters.number_of_nodes * 13*sizeof( float ),
                               hipMemcpyDeviceToHost )
                 );
  }
  else
    return 1;
  
  fprintf( fp, "\
# vtk DataFile Version 2.0\n\
flux_%d\n\
ASCII\n\
\n\
DATASET STRUCTURED_POINTS\n\
DIMENSIONS %u %u %u\n\
ORIGIN %f %f %f\n\
SPACING %f %f %f\n\
\n\
POINT_DATA %u\n\
SCALARS flux_%d float 3\n\
LOOKUP_TABLE default\n",
           species,
           ek_parameters.dim_x, ek_parameters.dim_y, ek_parameters.dim_z,
           ek_parameters.agrid*0.5f, ek_parameters.agrid*0.5f, ek_parameters.agrid*0.5f,
           ek_parameters.agrid, ek_parameters.agrid, ek_parameters.agrid,
           ek_parameters.number_of_nodes,
           species                                                                    );

  for( int i = 0; i < ek_parameters.number_of_nodes; i++ ) 
  {    
    rhoindex_linear2cartesian_host(i, coord);
     
    flux_local_cartesian[0]  = 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_U00) ];

    flux_local_cartesian[0] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UU0) ];
    flux_local_cartesian[0] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UD0) ];
    flux_local_cartesian[0] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_U0U) ];
    flux_local_cartesian[0] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_U0D) ];

    flux_local_cartesian[0] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UUU) ];
    flux_local_cartesian[0] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UUD) ];
    flux_local_cartesian[0] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UDU) ];
    flux_local_cartesian[0] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UDD) ];

    flux_local_cartesian[0] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1], coord[2], EK_LINK_D00-13) ];

    flux_local_cartesian[0] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]-1, coord[2], EK_LINK_DD0-13) ];
    flux_local_cartesian[0] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]+1, coord[2], EK_LINK_DU0-13) ];
    flux_local_cartesian[0] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1], coord[2]-1, EK_LINK_D0D-13) ];
    flux_local_cartesian[0] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1], coord[2]+1, EK_LINK_D0U-13) ];

    flux_local_cartesian[0] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]-1, coord[2]-1, EK_LINK_DDD-13) ];
    flux_local_cartesian[0] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]-1, coord[2]+1, EK_LINK_DDU-13) ];
    flux_local_cartesian[0] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]+1, coord[2]-1, EK_LINK_DUD-13) ];
    flux_local_cartesian[0] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]+1, coord[2]+1, EK_LINK_DUU-13) ];


    flux_local_cartesian[1]  = 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_0U0) ];

    flux_local_cartesian[1] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UU0) ];
    flux_local_cartesian[1] -= 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UD0) ];
    flux_local_cartesian[1] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_0UU) ];
    flux_local_cartesian[1] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_0UD) ];

    flux_local_cartesian[1] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UUU) ];
    flux_local_cartesian[1] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UUD) ];
    flux_local_cartesian[1] -= 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UDU) ];
    flux_local_cartesian[1] -= 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UDD) ];

    flux_local_cartesian[1] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0], coord[1]-1, coord[2], EK_LINK_0D0-13) ];

    flux_local_cartesian[1] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]-1, coord[2], EK_LINK_DD0-13) ];
    flux_local_cartesian[1] -= 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]+1, coord[2], EK_LINK_DU0-13) ];
    flux_local_cartesian[1] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0], coord[1]-1, coord[2]-1, EK_LINK_0DD-13) ];
    flux_local_cartesian[1] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0], coord[1]-1, coord[2]+1, EK_LINK_0DU-13) ];

    flux_local_cartesian[1] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]-1, coord[2]-1, EK_LINK_DDD-13) ];
    flux_local_cartesian[1] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]-1, coord[2]+1, EK_LINK_DDU-13) ];
    flux_local_cartesian[1] -= 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]+1, coord[2]-1, EK_LINK_DUD-13) ];
    flux_local_cartesian[1] -= 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]+1, coord[2]+1, EK_LINK_DUU-13) ];


    flux_local_cartesian[2]  = 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_00U) ];

    flux_local_cartesian[2] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_U0U) ];
    flux_local_cartesian[2] -= 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_U0D) ];
    flux_local_cartesian[2] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_0UU) ];
    flux_local_cartesian[2] -= 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_0UD) ];

    flux_local_cartesian[2] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UUU) ];
    flux_local_cartesian[2] -= 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UUD) ];
    flux_local_cartesian[2] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UDU) ];
    flux_local_cartesian[2] -= 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UDD) ];

    flux_local_cartesian[2] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0], coord[1], coord[2]-1, EK_LINK_00D-13) ];

    flux_local_cartesian[2] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1], coord[2]-1, EK_LINK_D0D-13) ];
    flux_local_cartesian[2] -= 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1], coord[2]+1, EK_LINK_D0U-13) ];
    flux_local_cartesian[2] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0], coord[1]-1, coord[2]-1, EK_LINK_0DD-13) ];
    flux_local_cartesian[2] -= 0.5*fluxes[ jindex_cartesian2linear_host(coord[0], coord[1]-1, coord[2]+1, EK_LINK_0DU-13) ];

    flux_local_cartesian[2] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]-1, coord[2]-1, EK_LINK_DDD-13) ];
    flux_local_cartesian[2] -= 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]-1, coord[2]+1, EK_LINK_DDU-13) ];
    flux_local_cartesian[2] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]+1, coord[2]-1, EK_LINK_DUD-13) ];
    flux_local_cartesian[2] -= 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]+1, coord[2]+1, EK_LINK_DUU-13) ];


    fprintf( fp, "%e %e %e\n",
             flux_local_cartesian[0] / ( ek_parameters.time_step * ek_parameters.agrid * ek_parameters.agrid ),
             flux_local_cartesian[1] / ( ek_parameters.time_step * ek_parameters.agrid * ek_parameters.agrid ),
             flux_local_cartesian[2] / ( ek_parameters.time_step * ek_parameters.agrid * ek_parameters.agrid ) );
  }
  
  free( fluxes );
  fclose( fp );
  
  return 0;
}


int ek_print_vtk_potential( char* filename ) {

  FILE* fp = fopen( filename, "w" );

  if( fp == NULL ) 
  {  
    return 1;
  }

  float* potential = (float*) malloc( ek_parameters.number_of_nodes * sizeof( hipfftReal ) );
  
  cuda_safe_mem( hipMemcpy( potential, 
                             ek_parameters.charge_potential,
                             ek_parameters.number_of_nodes * sizeof( hipfftReal ),
                             hipMemcpyDeviceToHost )                          
               );
  
  fprintf(fp, "\
# vtk DataFile Version 2.0\n\
potential\n\
ASCII\n\
\n\
DATASET STRUCTURED_POINTS\n\
DIMENSIONS %u %u %u\n\
ORIGIN %f %f %f\n\
SPACING %f %f %f\n\
\n\
POINT_DATA %u\n\
SCALARS potential float 1\n\
LOOKUP_TABLE default\n",
          ek_parameters.dim_x, ek_parameters.dim_y, ek_parameters.dim_z,
          ek_parameters.agrid*0.5f, ek_parameters.agrid*0.5f, ek_parameters.agrid*0.5f,
          ek_parameters.agrid, ek_parameters.agrid, ek_parameters.agrid,
          ek_parameters.number_of_nodes                                              );

  for( int i = 0; i < ek_parameters.number_of_nodes; i++ ) 
  {  
    fprintf( fp, "%e\n", potential[ i ] );
  }
  
  free( potential );
  fclose( fp );
  
  return 0;
}


int ek_print_vtk_lbforce( char* filename ) {

  FILE* fp = fopen( filename, "w" );

  if( fp == NULL ) 
  {
    return 1;
  }

  float* lbforce = (float*) malloc( ek_parameters.number_of_nodes * 3 *sizeof( float ) );
  
  cuda_safe_mem( hipMemcpy( lbforce, 
                             node_f.force,
                             ek_parameters.number_of_nodes * 3 * sizeof( float ),
                             hipMemcpyDeviceToHost )
               );
  
  fprintf( fp, "\
# vtk DataFile Version 2.0\n\
lbforce\n\
ASCII\n\
\n\
DATASET STRUCTURED_POINTS\n\
DIMENSIONS %u %u %u\n\
ORIGIN %f %f %f\n\
SPACING %f %f %f\n\
\n\
POINT_DATA %u\n\
SCALARS lbforce float 3\n\
LOOKUP_TABLE default\n",
           ek_parameters.dim_x, ek_parameters.dim_y, ek_parameters.dim_z,
           ek_parameters.agrid*0.5f, ek_parameters.agrid*0.5f, ek_parameters.agrid*0.5f,
           ek_parameters.agrid, ek_parameters.agrid, ek_parameters.agrid,
           ek_parameters.number_of_nodes                                              );

  for( int i = 0; i < ek_parameters.number_of_nodes; i++ ) 
  {
    fprintf( fp, "%e %e %e\n", lbforce[ i ] / 
                                 ( 2.0 * lbpar_gpu.rho[0] * powf( ek_parameters.time_step , 2.0 ) * powf( ek_parameters.agrid, 4.0 ) ),
                               lbforce[ i + ek_parameters.number_of_nodes ] /
                                 ( 2.0 * lbpar_gpu.rho[0] * powf( ek_parameters.time_step , 2.0 ) * powf( ek_parameters.agrid, 4.0 ) ),
                               lbforce[ i + 2 * ek_parameters.number_of_nodes ] /
                                 ( 2.0 * lbpar_gpu.rho[0] * powf( ek_parameters.time_step , 2.0 ) * powf( ek_parameters.agrid, 4.0 ) ) );
  }
  
  free( lbforce );
  fclose( fp );
  
  return 0;
}


#ifdef EK_REACTION
int ek_print_vtk_pressure( char* filename ) {

  FILE* fp = fopen( filename, "w" );

  if( fp == NULL ) 
  {
    return 1;
  }

  float* pressure = (float*) malloc( ek_parameters.number_of_nodes * sizeof( float ) );
  
  cuda_safe_mem( hipMemcpy( pressure, 
                             ek_parameters.pressure,
                             ek_parameters.number_of_nodes * sizeof( float ),
                             hipMemcpyDeviceToHost )
               );
  
  fprintf(fp, "\
# vtk DataFile Version 2.0\n\
pressure\n\
ASCII\n\
\n\
DATASET STRUCTURED_POINTS\n\
DIMENSIONS %u %u %u\n\
ORIGIN %f %f %f\n\
SPACING %f %f %f\n\
\n\
POINT_DATA %u\n\
SCALARS pressure float 1\n\
LOOKUP_TABLE default\n",
          ek_parameters.dim_x, ek_parameters.dim_y, ek_parameters.dim_z,
          ek_parameters.agrid*0.5f, ek_parameters.agrid*0.5f, ek_parameters.agrid*0.5f,
          ek_parameters.agrid, ek_parameters.agrid, ek_parameters.agrid,
          ek_parameters.number_of_nodes                                              );

  for( int i = 0; i < ek_parameters.number_of_nodes; i++ ) 
  { 
    fprintf( fp, "%e\n", pressure[ i ] / ek_parameters.agrid );
  }
  
  free( pressure );
  fclose( fp );
  
  return 0;
}


int ek_print_vtk_reaction_tags( char* filename ) {

  FILE* fp = fopen( filename, "w" );

  if( fp == NULL ) 
  {
    return 1;
  }

  fprintf(fp, "\
# vtk DataFile Version 2.0\n\
rection_tags\n\
ASCII\n\
\n\
DATASET STRUCTURED_POINTS\n\
DIMENSIONS %u %u %u\n\
ORIGIN %f %f %f\n\
SPACING %f %f %f\n\
\n\
POINT_DATA %u\n\
SCALARS reaction_tags int 1\n\
LOOKUP_TABLE default\n",
          ek_parameters.dim_x, ek_parameters.dim_y, ek_parameters.dim_z,
          ek_parameters.agrid*0.5f, ek_parameters.agrid*0.5f, ek_parameters.agrid*0.5f,
          ek_parameters.agrid, ek_parameters.agrid, ek_parameters.agrid,
          ek_parameters.number_of_nodes                                              );

  for( int i = 0; i < ek_parameters.number_of_nodes; i++ ) 
  {  
    fprintf( fp, "%d\n", ek_node_is_catalyst[ i ] );
  }
  
  fclose( fp );
  
  return 0;
}


int ek_print_vtk_mass_flux( char* filename ) {

  FILE* fp = fopen( filename, "w" );

  if( fp == NULL ){
    return 1;
  }

  fprintf( fp, "\
# vtk DataFile Version 2.0\n\
mass_flux\n\
ASCII\n\
\n\
DATASET STRUCTURED_POINTS\n\
DIMENSIONS %u %u %u\n\
ORIGIN %f %f %f\n\
SPACING %f %f %f\n\
\n\
POINT_DATA %u\n\
SCALARS mass_flux float 3\n\
LOOKUP_TABLE default\n",
           ek_parameters.dim_x, ek_parameters.dim_y, ek_parameters.dim_z,
           ek_parameters.agrid*0.5f, ek_parameters.agrid*0.5f, ek_parameters.agrid*0.5f,
           ek_parameters.agrid, ek_parameters.agrid, ek_parameters.agrid,
           ek_parameters.number_of_nodes
         );

  float current_mass_fraction;
  float total_mass =   ek_parameters.rho_reactant_reservoir*ek_parameters.mass_reactant
                     + ek_parameters.rho_product0_reservoir*ek_parameters.mass_product0
                     + ek_parameters.rho_product1_reservoir*ek_parameters.mass_product1;

  unsigned int coord[3];
  float flux_local_cartesian[3]; // temporary variable for converting fluxes into cartesian coordinates for output

  float* fluxes = (float*) malloc( ek_parameters.number_of_nodes * 13 * sizeof( float ) );
  float* mass_flux_local_cartesian = (float*) calloc( ek_parameters.number_of_nodes * 3 , sizeof( float ) );
  
  for ( int species_number = 0; species_number < 3; species_number++ )
  {
    int species = ek_parameters.reaction_species[species_number];

    if ( species_number == 0 )
      current_mass_fraction = ek_parameters.lb_density*ek_parameters.mass_reactant/total_mass;
    else if ( species_number == 1 )
      current_mass_fraction = ek_parameters.lb_density*ek_parameters.mass_product0/total_mass;
    else if ( species_number == 2 )
      current_mass_fraction = ek_parameters.lb_density*ek_parameters.mass_product1/total_mass;
    else 
      return 1;

    if( ek_parameters.species_index[ species ] != -1 ) 
    {    
      int threads_per_block = 64;
      int blocks_per_grid_y = 4;
      int blocks_per_grid_x =
        ( ek_parameters.number_of_nodes + threads_per_block * blocks_per_grid_y - 1 )
        / (threads_per_block * blocks_per_grid_y );
      dim3 dim_grid = make_uint3( blocks_per_grid_x, blocks_per_grid_y, 1 );
      
      KERNELCALL( ek_clear_fluxes, dim_grid, threads_per_block, () );
      KERNELCALL( ek_calculate_quantities, dim_grid, threads_per_block,
                  ( ek_parameters.species_index[ species ], *current_nodes, node_f, ek_lbparameters_gpu, ek_lb_device_values ) );
                
#ifdef EK_BOUNDARIES
      KERNELCALL( ek_apply_boundaries, dim_grid, threads_per_block,
                  ( ek_parameters.species_index[ species ], *current_nodes, node_f ) );
#endif
    
      cuda_safe_mem( hipMemcpy( fluxes, 
                                 ek_parameters.j,
                                 ek_parameters.number_of_nodes * 13*sizeof( float ),
                                 hipMemcpyDeviceToHost )
                   );
    }
    else
      return 1;

    for( int i = 0; i < ek_parameters.number_of_nodes; i++ ) 
    {      
      rhoindex_linear2cartesian_host(i, coord);
       
      flux_local_cartesian[0]  = 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_U00) ];

      flux_local_cartesian[0] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UU0) ];
      flux_local_cartesian[0] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UD0) ];
      flux_local_cartesian[0] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_U0U) ];
      flux_local_cartesian[0] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_U0D) ];

      flux_local_cartesian[0] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UUU) ];
      flux_local_cartesian[0] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UUD) ];
      flux_local_cartesian[0] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UDU) ];
      flux_local_cartesian[0] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UDD) ];

      flux_local_cartesian[0] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1], coord[2], EK_LINK_D00-13) ];

      flux_local_cartesian[0] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]-1, coord[2], EK_LINK_DD0-13) ];
      flux_local_cartesian[0] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]+1, coord[2], EK_LINK_DU0-13) ];
      flux_local_cartesian[0] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1], coord[2]-1, EK_LINK_D0D-13) ];
      flux_local_cartesian[0] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1], coord[2]+1, EK_LINK_D0U-13) ];

      flux_local_cartesian[0] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]-1, coord[2]-1, EK_LINK_DDD-13) ];
      flux_local_cartesian[0] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]-1, coord[2]+1, EK_LINK_DDU-13) ];
      flux_local_cartesian[0] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]+1, coord[2]-1, EK_LINK_DUD-13) ];
      flux_local_cartesian[0] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]+1, coord[2]+1, EK_LINK_DUU-13) ];


      flux_local_cartesian[1]  = 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_0U0) ];

      flux_local_cartesian[1] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UU0) ];
      flux_local_cartesian[1] -= 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UD0) ];
      flux_local_cartesian[1] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_0UU) ];
      flux_local_cartesian[1] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_0UD) ];

      flux_local_cartesian[1] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UUU) ];
      flux_local_cartesian[1] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UUD) ];
      flux_local_cartesian[1] -= 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UDU) ];
      flux_local_cartesian[1] -= 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UDD) ];

      flux_local_cartesian[1] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0], coord[1]-1, coord[2], EK_LINK_0D0-13) ];

      flux_local_cartesian[1] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]-1, coord[2], EK_LINK_DD0-13) ];
      flux_local_cartesian[1] -= 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]+1, coord[2], EK_LINK_DU0-13) ];
      flux_local_cartesian[1] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0], coord[1]-1, coord[2]-1, EK_LINK_0DD-13) ];
      flux_local_cartesian[1] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0], coord[1]-1, coord[2]+1, EK_LINK_0DU-13) ];

      flux_local_cartesian[1] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]-1, coord[2]-1, EK_LINK_DDD-13) ];
      flux_local_cartesian[1] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]-1, coord[2]+1, EK_LINK_DDU-13) ];
      flux_local_cartesian[1] -= 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]+1, coord[2]-1, EK_LINK_DUD-13) ];
      flux_local_cartesian[1] -= 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]+1, coord[2]+1, EK_LINK_DUU-13) ];


      flux_local_cartesian[2]  = 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_00U) ];

      flux_local_cartesian[2] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_U0U) ];
      flux_local_cartesian[2] -= 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_U0D) ];
      flux_local_cartesian[2] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_0UU) ];
      flux_local_cartesian[2] -= 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_0UD) ];

      flux_local_cartesian[2] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UUU) ];
      flux_local_cartesian[2] -= 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UUD) ];
      flux_local_cartesian[2] += 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UDU) ];
      flux_local_cartesian[2] -= 0.5*fluxes[ jindex_getByRhoLinear_host(i, EK_LINK_UDD) ];

      flux_local_cartesian[2] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0], coord[1], coord[2]-1, EK_LINK_00D-13) ];

      flux_local_cartesian[2] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1], coord[2]-1, EK_LINK_D0D-13) ];
      flux_local_cartesian[2] -= 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1], coord[2]+1, EK_LINK_D0U-13) ];
      flux_local_cartesian[2] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0], coord[1]-1, coord[2]-1, EK_LINK_0DD-13) ];
      flux_local_cartesian[2] -= 0.5*fluxes[ jindex_cartesian2linear_host(coord[0], coord[1]-1, coord[2]+1, EK_LINK_0DU-13) ];

      flux_local_cartesian[2] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]-1, coord[2]-1, EK_LINK_DDD-13) ];
      flux_local_cartesian[2] -= 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]-1, coord[2]+1, EK_LINK_DDU-13) ];
      flux_local_cartesian[2] += 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]+1, coord[2]-1, EK_LINK_DUD-13) ];
      flux_local_cartesian[2] -= 0.5*fluxes[ jindex_cartesian2linear_host(coord[0]-1, coord[1]+1, coord[2]+1, EK_LINK_DUU-13) ];

      mass_flux_local_cartesian[3*i + 0] += current_mass_fraction * flux_local_cartesian[0]
                                            / ( ek_parameters.time_step * ek_parameters.agrid * ek_parameters.agrid );
      mass_flux_local_cartesian[3*i + 1] += current_mass_fraction * flux_local_cartesian[1]
                                            / ( ek_parameters.time_step * ek_parameters.agrid * ek_parameters.agrid );
      mass_flux_local_cartesian[3*i + 2] += current_mass_fraction * flux_local_cartesian[2]
                                            / ( ek_parameters.time_step * ek_parameters.agrid * ek_parameters.agrid );

    }
  }
  
  for( int i = 0; i < ek_parameters.number_of_nodes; i++ ) 
  {
    fprintf( fp, "%e %e %e\n",
             mass_flux_local_cartesian[3*i + 0],
             mass_flux_local_cartesian[3*i + 1],
             mass_flux_local_cartesian[3*i + 2] );
  }

  free( fluxes );
  fclose( fp );
  
  return 0;
}
#endif


void ek_print_parameters() {

  printf( "ek_parameters {\n" );
  
  printf( "  float agrid = %f;\n",                      ek_parameters.agrid );
  printf( "  float time_step = %f;\n",                  ek_parameters.time_step );
  printf( "  float lb_density = %f;\n",                 ek_parameters.lb_density );
  printf( "  unsigned int dim_x = %d;\n",               ek_parameters.dim_x );
  printf( "  unsigned int dim_y = %d;\n",               ek_parameters.dim_y );
  printf( "  unsigned int dim_z = %d;\n",               ek_parameters.dim_z );
  printf( "  unsigned int number_of_nodes = %d;\n",     ek_parameters.number_of_nodes );
  printf( "  float viscosity = %f;\n",                  ek_parameters.viscosity );
  printf( "  float bulk_viscosity = %f;\n",             ek_parameters.bulk_viscosity );
  printf( "  float gamma_odd = %f;\n",                  ek_parameters.gamma_odd );
  printf( "  float gamma_even = %f;\n",                 ek_parameters.gamma_even );
  printf( "  float friction = %f;\n",                   ek_parameters.friction );
  printf( "  float T = %f;\n",                          ek_parameters.T );
  printf( "  float bjerrumlength = %f;\n",              ek_parameters.bjerrumlength );
  printf( "  unsigned int number_of_species = %d;\n",   ek_parameters.number_of_species);
  printf( "  unsigned int accelerated_frame_enabled = %d;\n",             ek_parameters.accelerated_frame_enabled);
  printf( "  float accelerated_frame_boundary_mass_density = %f;\n",      ek_parameters.accelerated_frame_boundary_mass_density);
  printf( "  float accelerated_frame_boundary_mass = %f;\n",              ek_parameters.accelerated_frame_boundary_mass);
  printf( "  float accelerated_frame_fluid_mass = %f;\n",                 ek_parameters.accelerated_frame_fluid_mass);
  printf( "  float ext_acceleration_force[] = {%f, %f, %f};\n",           ek_parameters.ext_acceleration_force[0], 
                                                                          ek_parameters.ext_acceleration_force[1], 
                                                                          ek_parameters.ext_acceleration_force[2] );
  printf( "  int reaction_species[] = {%d, %d, %d};\n", ek_parameters.reaction_species[0], 
                                                        ek_parameters.reaction_species[1], 
                                                        ek_parameters.reaction_species[2] );
  printf( "  float rho_reactant_reservoir = %f;\n",     ek_parameters.rho_reactant_reservoir);
  printf( "  float rho_product0_reservoir = %f;\n",     ek_parameters.rho_product0_reservoir);
  printf( "  float rho_product1_reservoir = %f;\n",     ek_parameters.rho_product1_reservoir);
  printf( "  float reaction_ct_rate = %f;\n",           ek_parameters.reaction_ct_rate); 
  printf( "  float reaction_fraction_0 = %f;\n",        ek_parameters.reaction_fraction_0);
  printf( "  float reaction_fraction_1 = %f;\n",        ek_parameters.reaction_fraction_0);
  printf( "  float reset_mode_0 = %f;\n",               ek_parameters.reset_mode_0);
  printf( "  float* j = %p;\n",                         ek_parameters.j );
  
  printf( "  float* rho[] = {%p, %p, %p, %p, %p, %p, %p, %p, %p, %p};\n",
          ek_parameters.rho[0], ek_parameters.rho[1], ek_parameters.rho[2],
          ek_parameters.rho[3], ek_parameters.rho[4], ek_parameters.rho[5],
          ek_parameters.rho[6], ek_parameters.rho[7], ek_parameters.rho[8],
          ek_parameters.rho[9]                                              );
  
  printf( "  int species_index[] = {%d, %d, %d, %d, %d, %d, %d, %d, %d, %d};\n",
          ek_parameters.species_index[0], ek_parameters.species_index[1],
          ek_parameters.species_index[2], ek_parameters.species_index[3],
          ek_parameters.species_index[4], ek_parameters.species_index[5],
          ek_parameters.species_index[6], ek_parameters.species_index[7],
          ek_parameters.species_index[8], ek_parameters.species_index[9]         );
  
  printf( "  float density = {%f, %f, %f, %f, %f, %f, %f, %f, %f, %f};\n",
          ek_parameters.density[0], ek_parameters.density[1],
          ek_parameters.density[2], ek_parameters.density[3],
          ek_parameters.density[4], ek_parameters.density[5],
          ek_parameters.density[6], ek_parameters.density[7],
          ek_parameters.density[8], ek_parameters.density[9]                );
  
  printf( "  float D[] = {%f, %f, %f, %f, %f, %f, %f, %f, %f, %f};\n",
          ek_parameters.D[0], ek_parameters.D[1], ek_parameters.D[2],
          ek_parameters.D[3], ek_parameters.D[4], ek_parameters.D[5],
          ek_parameters.D[6], ek_parameters.D[7], ek_parameters.D[8],
          ek_parameters.D[9]                                           );
  
  printf( "  float d[] = {%f, %f, %f, %f, %f, %f, %f, %f, %f, %f};\n",
          ek_parameters.d[0], ek_parameters.d[1], ek_parameters.d[2],
          ek_parameters.d[3], ek_parameters.d[4], ek_parameters.d[5],
          ek_parameters.d[6], ek_parameters.d[7], ek_parameters.d[8],
          ek_parameters.d[9]                                                   );
  
  printf( "  float valency[] = {%f, %f, %f, %f, %f, %f, %f, %f, %f, %f};\n",
          ek_parameters.valency[0], ek_parameters.valency[1], ek_parameters.valency[2],
          ek_parameters.valency[3], ek_parameters.valency[4], ek_parameters.valency[5],
          ek_parameters.valency[6], ek_parameters.valency[7], ek_parameters.valency[8],
          ek_parameters.valency[9]                                                      );
  
  printf( "  float ext_force[0][] = {%f, %f, %f, %f, %f, %f, %f, %f, %f, %f};\n",
          ek_parameters.ext_force[0][0], ek_parameters.ext_force[0][1], ek_parameters.ext_force[0][2],
          ek_parameters.ext_force[0][3], ek_parameters.ext_force[0][4], ek_parameters.ext_force[0][5],
          ek_parameters.ext_force[0][6], ek_parameters.ext_force[0][7], ek_parameters.ext_force[0][8],
          ek_parameters.ext_force[0][9]                                                                );
  
  printf( "  float ext_force[1][] = {%f, %f, %f, %f, %f, %f, %f, %f, %f, %f};\n",
          ek_parameters.ext_force[1][0], ek_parameters.ext_force[1][1], ek_parameters.ext_force[1][2],
          ek_parameters.ext_force[1][3], ek_parameters.ext_force[1][4], ek_parameters.ext_force[1][5],
          ek_parameters.ext_force[1][6], ek_parameters.ext_force[1][7], ek_parameters.ext_force[1][8],
          ek_parameters.ext_force[1][9]                                                                );
  
  printf( "  float ext_force[2][] = {%f, %f, %f, %f, %f, %f, %f, %f, %f, %f};\n",
          ek_parameters.ext_force[2][0], ek_parameters.ext_force[2][1], ek_parameters.ext_force[2][2],
          ek_parameters.ext_force[2][3], ek_parameters.ext_force[2][4], ek_parameters.ext_force[2][5],
          ek_parameters.ext_force[2][6], ek_parameters.ext_force[2][7], ek_parameters.ext_force[2][8],
          ek_parameters.ext_force[2][9]                                                                );
  
  printf( "}\n" );
}


void ek_print_lbpar() {

  printf("lbpar_gpu {\n");
  
  printf("    float rho = %f;\n",                        lbpar_gpu.rho[0] );
  printf("    float mu = %f;\n",                         lbpar_gpu.mu[0] );
  printf("    float viscosity = %f;\n",                  lbpar_gpu.viscosity[0] );
  printf("    float gamma_shear = %f;\n",                lbpar_gpu.gamma_shear[0] );
  printf("    float gamma_bulk = %f;\n",                 lbpar_gpu.gamma_bulk[0] );
  printf("    float gamma_odd = %f;\n",                  lbpar_gpu.gamma_odd[0] );
  printf("    float gamma_even = %f;\n",                 lbpar_gpu.gamma_even[0] );
  printf("    float agrid = %f;\n",                      lbpar_gpu.agrid );
  printf("    float tau = %f;\n",                        lbpar_gpu.tau );
  printf("    float friction = %f;\n",                   lbpar_gpu.friction[0] );
  printf("    float time_step = %f;\n",                  lbpar_gpu.time_step );
  printf("    float lb_coupl_pref = %f;\n",              lbpar_gpu.lb_coupl_pref[0] );
  printf("    float lb_coupl_pref2 = %f;\n",             lbpar_gpu.lb_coupl_pref2[0] );
  printf("    float bulk_viscosity = %f;\n",             lbpar_gpu.bulk_viscosity[0] );
  printf("    unsigned int dim_x = %d;\n",               lbpar_gpu.dim_x );
  printf("    unsigned int dim_y = %d;\n",               lbpar_gpu.dim_y );
  printf("    unsigned int dim_z = %d;\n",               lbpar_gpu.dim_z );
  printf("    unsigned int number_of_nodes = %d;\n",     lbpar_gpu.number_of_nodes );
  printf("    unsigned int number_of_particles = %d;\n", lbpar_gpu.number_of_particles );
  printf("    int fluct = %d;\n",                        lbpar_gpu.fluct );
  printf("    int calc_val = %d;\n",                     lbpar_gpu.calc_val );
  printf("    int external_force = %d;\n",               lbpar_gpu.external_force );
  printf("    float ext_force[3] = {%f, %f, %f};\n",     lbpar_gpu.ext_force[0],
                                                         lbpar_gpu.ext_force[1],
                                                         lbpar_gpu.ext_force[2] );
  printf("    unsigned int your_seed = %d;\n",           lbpar_gpu.your_seed );
  printf("    unsigned int reinit = %d;\n",              lbpar_gpu.reinit );
  
  printf("}\n");
}


int ek_set_agrid( double agrid ) {  

  ek_parameters.agrid = agrid;    
  return 0;
}


int ek_set_lb_density( double lb_density ) {  

  ek_parameters.lb_density = lb_density;    
  return 0;
}


int ek_set_bjerrumlength( double bjerrumlength ) {

  ek_parameters.bjerrumlength = bjerrumlength;
  return 0;
}


int ek_set_viscosity( double viscosity ) {

  ek_parameters.viscosity = viscosity;  
  return 0;
}


int ek_set_friction( double friction ) {

  ek_parameters.friction = friction;  
  return 0;
}


int ek_set_bulk_viscosity( double bulk_viscosity ) {

  ek_parameters.bulk_viscosity = bulk_viscosity;  
  return 0;
}


int ek_set_gamma_odd( double gamma_odd ) {

  ek_parameters.gamma_odd = gamma_odd;  
  return 0;
}


int ek_set_gamma_even( double gamma_even ) {

  ek_parameters.gamma_even = gamma_even;  
  return 0;
}


int ek_set_density( int species, double density ) {

  ek_init_species( species );

  ek_parameters.density[ ek_parameters.species_index[ species ] ] = density;
   
  return 0;
}


int ek_set_D( int species, double D ) {

  ek_init_species( species );
  
  ek_parameters.D[ ek_parameters.species_index[ species ] ] = D;
  ek_parameters.d[ ek_parameters.species_index[ species ] ] = D / ( 1.0 + 2.0 * sqrt(2.0)) ;
  
  return 0;
}

int ek_set_T(double T) {

  ek_parameters.T = T;
  
  return 0;
}


int ek_set_valency( int species, double valency ) {

  ek_init_species( species );
  
  ek_parameters.valency[ ek_parameters.species_index[ species ] ] = valency;
  
  return 0;
}


int ek_set_ext_force( int species,
                      double ext_force_x,
                      double ext_force_y,
                      double ext_force_z
                    ) {
                    
  ek_init_species( species );
  
  ek_parameters.ext_force[0][ ek_parameters.species_index[ species ] ] = ext_force_x;
  ek_parameters.ext_force[1][ ek_parameters.species_index[ species ] ] = ext_force_y;
  ek_parameters.ext_force[2][ ek_parameters.species_index[ species ] ] = ext_force_z;
  
  return 0;
}

int ek_set_accelerated_frame( int enabled, double boundary_mass_density, double* ext_acceleration_force ) {

#ifdef EK_BOUNDARIES
  ek_parameters.accelerated_frame_enabled = enabled;
  ek_parameters.accelerated_frame_boundary_mass_density = boundary_mass_density;

  ek_parameters.ext_acceleration_force[0] = ext_acceleration_force[0];
  ek_parameters.ext_acceleration_force[1] = ext_acceleration_force[1];
  ek_parameters.ext_acceleration_force[2] = ext_acceleration_force[2];

  return 0;
#else 
  printf("ERROR: Need boundaries (EK_BOUNDARIES) for the accelerated frame.\n");
  return 1;
#endif

}

int ek_accelerated_frame_print_boundary_velocity( double* accelerated_boundary_velocity ) {

#ifdef EK_BOUNDARIES
  float* temp_boundary_velocity = (float*) malloc( 3 * sizeof( float ) );

  cuda_safe_mem( hipMemcpy( temp_boundary_velocity, 
                             ek_accelerated_frame_boundary_velocity,
                             3 * sizeof( float ),
                             hipMemcpyDeviceToHost )
               );

  accelerated_boundary_velocity[0] = static_cast<double>(temp_boundary_velocity[0]);
  accelerated_boundary_velocity[1] = static_cast<double>(temp_boundary_velocity[1]);
  accelerated_boundary_velocity[2] = static_cast<double>(temp_boundary_velocity[2]);

  return 0;
#else 
  printf("ERROR: Need boundaries (EK_BOUNDARIES) for the accelerated frame.\n");
  return 1;
#endif

}


#ifdef EK_REACTION
int ek_set_reaction( int reactant, int product0, int product1, 
                     float rho_reactant_reservoir, float rho_product0_reservoir, float rho_product1_reservoir, 
                     float reaction_ct_rate, float reaction_fraction_0, float reaction_fraction_1,
                     float mass_reactant, float mass_product0, float mass_product1 ) 
{
  if ( ek_parameters.species_index[reactant] == -1 ||
       ek_parameters.species_index[product0] == -1 ||
       ek_parameters.species_index[product1] == -1 ) 
    return 1;

  ek_parameters.reaction_species[0] = reactant;
  ek_parameters.reaction_species[1] = product0;
  ek_parameters.reaction_species[2] = product1;

  ek_parameters.rho_reactant_reservoir = rho_reactant_reservoir;
  ek_parameters.rho_product0_reservoir = rho_product0_reservoir;
  ek_parameters.rho_product1_reservoir = rho_product1_reservoir;

  ek_parameters.reaction_ct_rate = reaction_ct_rate;

  ek_parameters.mass_reactant = mass_reactant;
  ek_parameters.mass_product0 = mass_product0;
  ek_parameters.mass_product1 = mass_product1;

  ek_parameters.reaction_fraction_0 = reaction_fraction_0;
  ek_parameters.reaction_fraction_1 = reaction_fraction_1;  

  return 0;
}

int ek_reset_mode_zero( double reset_mode_0 ) {

  if ( ek_parameters.reaction_species[0] == -1 ||
       ek_parameters.reaction_species[1] == -1 ||
       ek_parameters.reaction_species[2] == -1 ) 
    return 1;
                    
  ek_parameters.reset_mode_0 = reset_mode_0;
  
  return 0;
}

int ek_tag_reaction_nodes( LB_Boundary *boundary, char reaction_type )
{

#ifdef EK_BOUNDARIES
  char *errtxt;
  double pos[3], dist, dist_vec[3];

  for(int z=0; z<int(ek_parameters.dim_z); z++) {
  for(int y=0; y<int(ek_parameters.dim_y); y++) {
  for(int x=0; x<int(ek_parameters.dim_x); x++) {	 

    pos[0] = (x + 0.5)*lbpar_gpu.agrid;
    pos[1] = (y + 0.5)*lbpar_gpu.agrid;
    pos[2] = (z + 0.5)*lbpar_gpu.agrid;

    switch (boundary->type)
    {
      case LB_BOUNDARY_WAL:
        calculate_wall_dist((Particle*) NULL, pos, (Particle*) NULL, &boundary->c.wal, &dist, dist_vec);
        break;
                
      case LB_BOUNDARY_SPH:
        calculate_sphere_dist((Particle*) NULL, pos, (Particle*) NULL, &boundary->c.sph, &dist, dist_vec);
        break;
                
      case LB_BOUNDARY_CYL:
        calculate_cylinder_dist((Particle*) NULL, pos, (Particle*) NULL, &boundary->c.cyl, &dist, dist_vec);
        break;
                
      case LB_BOUNDARY_RHOMBOID:
        calculate_rhomboid_dist((Particle*) NULL, pos, (Particle*) NULL, &boundary->c.rhomboid, &dist, dist_vec);
        break;
                
      case LB_BOUNDARY_POR:
        calculate_pore_dist((Particle*) NULL, pos, (Particle*) NULL, &boundary->c.pore, &dist, dist_vec);
        break;
                
      case LB_BOUNDARY_STOMATOCYTE:
        calculate_stomatocyte_dist((Particle*) NULL, pos, (Particle*) NULL, &boundary->c.stomatocyte, &dist, dist_vec);
        break;

      case LB_BOUNDARY_BOX:
        dist = -1.0;
        break;
                
      case LB_BOUNDARY_HOLLOW_CONE:
        calculate_hollow_cone_dist((Particle*) NULL, pos, (Particle*) NULL, &boundary->c.hollow_cone, &dist, dist_vec);
        break;
                
      default:
        errtxt = runtime_error(128);
        ERROR_SPRINTF(errtxt, "{109 lbboundary type %d not implemented in ek_tag_reaction_nodes()\n", boundary->type);
    }

    if( dist <= 0.0 )
    {
      ek_node_is_catalyst[
                           z * ek_parameters.dim_y * ek_parameters.dim_x +
                           y * ek_parameters.dim_x +
                           x
                         ] = reaction_type;
    }

  }}}

  cuda_safe_mem( hipMemcpy( ek_parameters.node_is_catalyst, 
                             ek_node_is_catalyst, 
                             ek_parameters.number_of_nodes * sizeof( char ), 
                             hipMemcpyHostToDevice ) 
               );

  return 0;
#else 
  printf("ERROR: Need boundaries (EK_BOUNDARIES) for the catalytic reaction tagging.\n");
  return 1;
#endif

}
#endif


#endif /* ELECTROKINETICS */

#endif /* CUDA */
